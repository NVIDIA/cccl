/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/inner_product.h>

#include "nvbench_helper.cuh"

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  auto generator               = generate(elements);
  thrust::device_vector<T> lhs = generator;
  thrust::device_vector<T> rhs = generator;

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements * 2);
  state.add_global_memory_writes<T>(1);

  caching_allocator_t alloc;
  thrust::inner_product(policy(alloc), lhs.begin(), lhs.end(), rhs.begin(), T{0});

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::inner_product(policy(alloc, launch), lhs.begin(), lhs.end(), rhs.begin(), T{0});
  });
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(all_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));
