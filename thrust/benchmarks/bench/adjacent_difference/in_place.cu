/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/adjacent_difference.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include "nvbench_helper.cuh"

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<T> vec(elements, 0);

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);

  caching_allocator_t alloc;
  thrust::adjacent_difference(policy(alloc), vec.begin(), vec.end(), vec.begin());

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::adjacent_difference(policy(alloc, launch), vec.begin(), vec.end(), vec.begin());
  });
}

using types = nvbench::type_list<int8_t, int16_t, int32_t, int64_t, float, double>;

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));
