/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include "nvbench_helper.cuh"

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements      = static_cast<std::size_t>(state.get_int64("Elements"));
  const auto needles_ratio = static_cast<std::size_t>(state.get_int64("NeedlesRatio"));
  const auto needles       = needles_ratio * static_cast<std::size_t>(static_cast<double>(elements) / 100.0);

  thrust::device_vector<T> data = generate(elements + needles);
  thrust::device_vector<T> result(needles);
  thrust::sort(data.begin(), data.begin() + elements);

  state.add_element_count(needles);

  caching_allocator_t alloc;
  thrust::lower_bound(
    policy(alloc), data.begin(), data.begin() + elements, data.begin() + elements, data.end(), result.begin());

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::lower_bound(
      policy(alloc, launch), data.begin(), data.begin() + elements, data.begin() + elements, data.end(), result.begin());
  });
}

using types = nvbench::type_list<int8_t, int16_t, int32_t, int64_t>;

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_int64_axis("NeedlesRatio", {1, 25, 50});
