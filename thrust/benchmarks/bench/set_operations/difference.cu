/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "base.cuh"

struct op_t
{
  template <class PolicyT, class InputIterator1, class InputIterator2, class OutputIterator>
  __host__ OutputIterator operator()(
    const PolicyT& policy,
    InputIterator1 first1,
    InputIterator1 last1,
    InputIterator2 first2,
    InputIterator2 last2,
    OutputIterator result) const
  {
    return thrust::set_difference(policy, first1, last1, first2, last2, result);
  }
};

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T> tl)
{
  basic(state, tl, op_t{});
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"})
  .add_int64_axis("SizeRatio", {25, 50, 75});
