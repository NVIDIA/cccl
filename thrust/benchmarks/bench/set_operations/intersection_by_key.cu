/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "by_key.cuh"

struct op_t
{
  static constexpr bool read_all_values = false;

  template <class PolicyT,
            class InputIterator1,
            class InputIterator2,
            class InputIterator3,
            class InputIterator4,
            class OutputIterator1,
            class OutputIterator2>
  __host__ thrust::pair<OutputIterator1, OutputIterator2> operator()(
    const PolicyT& policy,
    InputIterator1 keys_first1,
    InputIterator1 keys_last1,
    InputIterator2 keys_first2,
    InputIterator2 keys_last2,
    InputIterator3 values_first1,
    InputIterator4 /* values_first2 */,
    OutputIterator1 keys_result,
    OutputIterator2 values_result) const
  {
    return thrust::set_intersection_by_key(
      policy, keys_first1, keys_last1, keys_first2, keys_last2, values_first1, keys_result, values_result);
  }
};

template <class KeyT, class ValueT>
static void basic(nvbench::state& state, nvbench::type_list<KeyT, ValueT> tl)
{
  basic(state, tl, op_t{});
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(key_types, value_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "ValueT{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"})
  .add_int64_axis("SizeRatio", {25, 50, 75});
