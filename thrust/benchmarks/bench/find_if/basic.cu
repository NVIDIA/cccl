/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/count.h>
#include <thrust/fill.h>

#include "nvbench_helper.cuh"

template <typename T>
struct equals
{
  T val;

  __device__ __host__ bool operator()(T i)
  {
    return i == val;
  }
};

template <typename T>
void find_if(nvbench::state& state, nvbench::type_list<T>)
{
  T val = 1;
  // set up input
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements"));
  const auto common_prefix  = state.get_float64("MismatchAt");
  const auto mismatch_point = elements * common_prefix;

  thrust::device_vector<T> dinput(elements, 0);
  thrust::fill(dinput.begin() + mismatch_point, dinput.end(), val);
  ///

  caching_allocator_t alloc;
  thrust::find_if(policy(alloc), dinput.begin(), dinput.end(), equals<T>{val});

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::find_if(policy(alloc, launch), dinput.begin(), dinput.end(), equals<T>{val});
  });
}

NVBENCH_BENCH_TYPES(find_if, NVBENCH_TYPE_AXES(fundamental_types))
  .set_name("thrust::find_if")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_float64_axis("MismatchAt", std::vector{1.0, 0.5, 0.0});
