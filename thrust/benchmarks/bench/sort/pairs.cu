/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "nvbench_helper.cuh"

template <class KeyT, class ValueT>
static void basic(nvbench::state& state, nvbench::type_list<KeyT, ValueT>)
{
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  thrust::device_vector<KeyT> in_keys = generate(elements, entropy);
  thrust::device_vector<KeyT> keys(elements);

  thrust::device_vector<ValueT> in_vals = generate(elements);
  thrust::device_vector<ValueT> vals(elements);

  state.add_element_count(elements);
  state.add_global_memory_reads<KeyT>(elements);
  state.add_global_memory_reads<ValueT>(elements);
  state.add_global_memory_writes<KeyT>(elements);
  state.add_global_memory_writes<ValueT>(elements);

  caching_allocator_t alloc;
  thrust::sort_by_key(policy(alloc), keys.begin(), keys.end(), vals.begin());

  state.exec(nvbench::exec_tag::timer | nvbench::exec_tag::sync, [&](nvbench::launch& launch, auto& timer) {
    keys = in_keys;
    vals = in_vals;
    timer.start();
    thrust::sort_by_key(policy(alloc, launch), keys.begin(), keys.end(), vals.begin());
    timer.stop();
  });
}

using key_types   = integral_types;
using value_types = integral_types;

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(key_types, value_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "ValueT{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"});
