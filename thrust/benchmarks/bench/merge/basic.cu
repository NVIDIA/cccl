/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>
#include <thrust/sort.h>

#include "nvbench_helper.cuh"

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements        = static_cast<std::size_t>(state.get_int64("Elements"));
  const auto size_ratio      = static_cast<std::size_t>(state.get_int64("InputSizeRatio"));
  const auto entropy         = str_to_entropy(state.get_string("Entropy"));
  const auto elements_in_lhs = static_cast<std::size_t>(static_cast<double>(size_ratio * elements) / 100.0);

  thrust::device_vector<T> out(elements);
  thrust::device_vector<T> in = generate(elements, entropy);
  thrust::sort(in.begin(), in.begin() + elements_in_lhs);
  thrust::sort(in.begin() + elements_in_lhs, in.end());

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);

  caching_allocator_t alloc;
  thrust::merge(
    policy(alloc), in.cbegin(), in.cbegin() + elements_in_lhs, in.cbegin() + elements_in_lhs, in.cend(), out.begin());

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::merge(
      policy(alloc, launch),
      in.cbegin(),
      in.cbegin() + elements_in_lhs,
      in.cbegin() + elements_in_lhs,
      in.cend(),
      out.begin());
  });
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(fundamental_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"})
  .add_int64_axis("InputSizeRatio", {25, 50, 75});
