/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <nvbench_helper.cuh>

template <typename T>
static void basic(nvbench::state& state, nvbench::type_list<T>)
{
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<T> input(elements, 1);
  thrust::device_vector<T> output(elements);

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);

  caching_allocator_t alloc;
  thrust::copy(policy(alloc), input.cbegin(), input.cend(), output.begin());

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    thrust::copy(policy(alloc, launch), input.cbegin(), input.cend(), output.begin());
  });
}

using types = nvbench::type_list<nvbench::uint8_t, nvbench::uint16_t, nvbench::uint32_t, nvbench::uint64_t>;

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types))
  .set_name("base")
  .set_type_axes_names({"T{ct}"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));
