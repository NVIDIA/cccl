#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <cmath>
#include <limits>
#include <iostream>

// This example computes several statistical properties of a data
// series in a single reduction.  The algorithm is described in detail here:
// http://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Parallel_algorithm
//
// Thanks to Joseph Rhoads for contributing this example


// structure used to accumulate the moments and other 
// statistical properties encountered so far.
template <typename T>
struct summary_stats_data
{
    T n;
    T min;
    T max;
    T mean;
    T M2;
    T M3;
    T M4;
    
    // initialize to the identity element
    void initialize()
    {
      n = mean = M2 = M3 = M4 = 0;
      min = std::numeric_limits<T>::max();
      max = std::numeric_limits<T>::min();
    }

    T variance()   { return M2 / (n - 1); }
    T variance_n() { return M2 / n; }
    T skewness()   { return std::sqrt(n) * M3 / std::pow(M2, (T) 1.5); }
    T kurtosis()   { return n * M4 / (M2 * M2); }
};

// stats_unary_op is a functor that takes in a value x and
// returns a variace_data whose mean value is initialized to x.
template <typename T>
struct summary_stats_unary_op
{
    __host__ __device__
    summary_stats_data<T> operator()(const T& x) const
    {
         summary_stats_data<T> result;
         result.n    = 1;
         result.min  = x;
         result.max  = x;
         result.mean = x;
         result.M2   = 0;
         result.M3   = 0;
         result.M4   = 0;

         return result;
    }
};

// summary_stats_binary_op is a functor that accepts two summary_stats_data 
// structs and returns a new summary_stats_data which are an
// approximation to the summary_stats for 
// all values that have been agregated so far
template <typename T>
struct summary_stats_binary_op 
    : public thrust::binary_function<const summary_stats_data<T>&, 
                                     const summary_stats_data<T>&,
                                           summary_stats_data<T> >
{
    __host__ __device__
    summary_stats_data<T> operator()(const summary_stats_data<T>& x, const summary_stats_data <T>& y) const
    {
        summary_stats_data<T> result;
        
        // precompute some common subexpressions
        T n  = x.n + y.n;
        T n2 = n  * n;
        T n3 = n2 * n;

        T delta  = y.mean - x.mean;
        T delta2 = delta  * delta;
        T delta3 = delta2 * delta;
        T delta4 = delta3 * delta;
        
        //Basic number of samples (n), min, and max
        result.n   = n;
        result.min = thrust::min(x.min, y.min);
        result.max = thrust::max(x.max, y.max);

        result.mean = x.mean + delta * y.n / n;

        result.M2  = x.M2 + y.M2;
        result.M2 += delta2 * x.n * y.n / n;

        result.M3  = x.M3 + y.M3;
        result.M3 += delta3 * x.n * y.n * (x.n - y.n) / n2; 
        result.M3 += (T) 3.0 * delta * (x.n * y.M2 - y.n * x.M2) / n;
    
        result.M4  = x.M4 + y.M4;
        result.M4 += delta4 * x.n * y.n * (x.n * x.n - x.n * y.n + y.n * y.n) / n3;
        result.M4 += (T) 6.0 * delta2 * (x.n * x.n * y.M2 + y.n * y.n * x.M2) / n2;
        result.M4 += (T) 4.0 * delta * (x.n * y.M3 - y.n * x.M3) / n;
        
        return result;
    }
};

template <typename Iterator>
void print_range(const std::string& name, Iterator first, Iterator last)
{
    typedef typename std::iterator_traits<Iterator>::value_type T;

    std::cout << name << ": ";
    thrust::copy(first, last, std::ostream_iterator<T>(std::cout, " "));  
    std::cout << "\n";
}


int main(void)
{
    typedef float T;

    // initialize host array
    T h_x[] = {4, 7, 13, 16};

    // transfer to device
    thrust::device_vector<T> d_x(h_x, h_x + sizeof(h_x) / sizeof(T));

    // setup arguments
    summary_stats_unary_op<T>  unary_op;
    summary_stats_binary_op<T> binary_op;
    summary_stats_data<T>      init;

    init.initialize();

    // compute summary statistics
    summary_stats_data<T> result = thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op);

    std::cout <<"******Summary Statistics Example*****"<<std::endl;
    print_range("The data", d_x.begin(), d_x.end());

    std::cout <<"Count              : "<< result.n << std::endl;
    std::cout <<"Minimum            : "<< result.min <<std::endl;
    std::cout <<"Maximum            : "<< result.max <<std::endl;
    std::cout <<"Mean               : "<< result.mean << std::endl;
    std::cout <<"Variance           : "<< result.variance() << std::endl;
    std::cout <<"Standard Deviation : "<< std::sqrt(result.variance_n()) << std::endl;
    std::cout <<"Skewness           : "<< result.skewness() << std::endl;
    std::cout <<"Kurtosis           : "<< result.kurtosis() << std::endl;

    return 0;
}

