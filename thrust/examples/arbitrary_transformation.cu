#include <thrust/detail/config.h>

#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/zip_function.h>

#include <iostream>

// This example shows how to implement an arbitrary transformation of
// the form output[i] = F(first[i], second[i], third[i], ... ).
// In this example, we use a function with 3 inputs and 1 output.
//
// Iterators for all four vectors (3 inputs + 1 output) are "zipped"
// into a single sequence of tuples with the zip_iterator.
//
// The arbitrary_functor receives a tuple that contains four elements,
// which are references to values in each of the four sequences. When we
// access the tuple 't' with the get() function,
//      get<0>(t) returns a reference to A[i],
//      get<1>(t) returns a reference to B[i],
//      get<2>(t) returns a reference to C[i],
//      get<3>(t) returns a reference to D[i].
//
// In this example, we can implement the transformation,
//      D[i] = A[i] + B[i] * C[i];
// by invoking arbitrary_functor() on each of the tuples using for_each.
//
// If we are using a functor that is not designed for zip iterators by taking a
// tuple instead of individual arguments we can adapt this function using the
// zip_function adaptor (C++11 only).
//
// Note that we could extend this example to implement functions with an
// arbitrary number of input arguments by zipping more sequence together.
// With the same approach we can have multiple *output* sequences, if we
// wanted to implement something like
//      D[i] = A[i] + B[i] * C[i];
//      E[i] = A[i] + B[i] + C[i];
//
// The possibilities are endless! :)

struct arbitrary_functor1
{
  template <typename Tuple>
  __host__ __device__ void operator()(Tuple t)
  {
    // D[i] = A[i] + B[i] * C[i];
    thrust::get<3>(t) = thrust::get<0>(t) + thrust::get<1>(t) * thrust::get<2>(t);
  }
};

struct arbitrary_functor2
{
  __host__ __device__ void operator()(const float& a, const float& b, const float& c, float& d)
  {
    // D[i] = A[i] + B[i] * C[i];
    d = a + b * c;
  }
};

int main()
{
  // allocate and initialize storage using initializer lists
  thrust::device_vector<float> A{3, 4, 0, 8, 2};
  thrust::device_vector<float> B{6, 7, 2, 1, 8};
  thrust::device_vector<float> C{2, 5, 7, 4, 3};
  thrust::device_vector<float> D1(5);

  // apply the transformation
  thrust::for_each(thrust::make_zip_iterator(A.begin(), B.begin(), C.begin(), D1.begin()),
                   thrust::make_zip_iterator(A.end(), B.end(), C.end(), D1.end()),
                   arbitrary_functor1());

  // print the output
  std::cout << "Tuple functor" << std::endl;
  for (size_t i = 0; i < A.size(); i++)
  {
    std::cout << A[i] << " + " << B[i] << " * " << C[i] << " = " << D1[i] << std::endl;
  }

  // apply the transformation using zip_function
  thrust::device_vector<float> D2(5);
  thrust::for_each(thrust::make_zip_iterator(A.begin(), B.begin(), C.begin(), D2.begin()),
                   thrust::make_zip_iterator(A.end(), B.end(), C.end(), D2.end()),
                   thrust::make_zip_function(arbitrary_functor2()));

  // print the output
  std::cout << "N-ary functor" << std::endl;
  for (size_t i = 0; i < A.size(); i++)
  {
    std::cout << A[i] << " + " << B[i] << " * " << C[i] << " = " << D2[i] << std::endl;
  }
}
