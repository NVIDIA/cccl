#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>

int main(void)
{
    size_t N = 10;

    // create a device_ptr 
    thrust::device_ptr<int> dev_ptr = thrust::device_malloc<int>(N);
     
    // extract raw pointer from device_ptr
    int * raw_ptr = thrust::raw_pointer_cast(dev_ptr);

    // use raw_ptr in CUDA API functions
    hipMemset(raw_ptr, 0, N * sizeof(int));

    // free memory
    thrust::device_free(dev_ptr);
    
    // we can use the same approach for device_vector
    thrust::device_vector<int> d_vec(N);

    // note: d_vec.data() returns a device_ptr
    raw_ptr = thrust::raw_pointer_cast(d_vec.data());

    return 0;
}
