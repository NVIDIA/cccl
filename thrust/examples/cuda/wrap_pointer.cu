#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <hip/hip_runtime.h>

int main(void)
{
    size_t N = 10;

    // obtain raw pointer to device memory
    int * raw_ptr;
    hipMalloc((void **) &raw_ptr, N * sizeof(int));

    // wrap raw pointer with a device_ptr 
    thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(raw_ptr);

    // use device_ptr in Thrust algorithms
    thrust::fill(dev_ptr, dev_ptr + N, (int) 0);

    // access device memory transparently through device_ptr
    dev_ptr[0] = 1;

    // free memory
    hipFree(raw_ptr);

    return 0;
}
