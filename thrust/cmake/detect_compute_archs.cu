/*
 *  Copyright 2019-2020 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <set>
#include <string>

int main(int argc, char** argv) {
  std::set<std::string> archs;
  int devices;
  if ((hipGetDeviceCount(&devices) == hipSuccess) && (devices > 0)) {
    for (int dev = 0; dev < devices; ++dev) {
      char buff[32];
      hipDeviceProp_t prop;
      if(hipGetDeviceProperties(&prop, dev) != hipSuccess) continue;
      sprintf(buff, "%d%d", prop.major, prop.minor);
      archs.insert(buff);
    }
  }
  if (archs.empty()) {
    printf("NONE");
  } else {
    bool first = true;
    for(const auto& arch : archs) {
      printf(first ? "%s" : ";%s", arch.c_str());
      first = false;
    }
  }
  printf("\n");
}
