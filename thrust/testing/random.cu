#include "hip/hip_runtime.h"
#include <thrust/generate.h>
#include <thrust/random.h>

#include <sstream>

#include <unittest/unittest.h>

template <typename Engine>
struct ValidateEngine
{
  _CCCL_HOST_DEVICE ValidateEngine(const typename Engine::result_type value_10000)
      : m_value_10000(value_10000)
  {}

  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    Engine e;
    e.discard(9999);

    // get the 10Kth result
    return e() == m_value_10000;
  }

  const typename Engine::result_type m_value_10000;
}; // end ValidateEngine

template <typename Engine, bool trivial_min = (Engine::min == 0)>
struct ValidateEngineMin
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    Engine e;

    bool result = true;

    for (int i = 0; i < 10000; ++i)
    {
      result &= (e() >= Engine::min);
    }

    return result;
  }
}; // end ValidateEngineMin

template <typename Engine>
struct ValidateEngineMin<Engine, true>
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    return true;
  }
};

template <typename Engine>
struct ValidateEngineMax
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    Engine e;

    bool result = true;

    for (int i = 0; i < 10000; ++i)
    {
      result &= (e() <= Engine::max);
    }

    return result;
  }
}; // end ValidateEngineMax

template <typename Engine>
struct ValidateEngineEqual
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    bool result = true;

    // test from default constructor
    Engine e0, e1;
    result &= (e0 == e1);

    // advance engines
    e0.discard(10000);
    e1.discard(10000);
    result &= (e0 == e1);

    // test from identical seeds
    Engine e2(13), e3(13);
    result &= (e2 == e3);

    // test different seeds aren't equal
    Engine e4(7), e5(13);
    result &= !(e4 == e5);

    // test reseeding engine to the same seed causes equality
    e4.seed(13);
    result &= (e4 == e5);

    return result;
  }
};

template <typename Engine>
struct ValidateEngineUnequal
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    bool result = true;

    // test from default constructor
    Engine e0, e1;
    result &= !(e0 != e1);

    // advance engines
    e0.discard(1000);
    e1.discard(1000);
    result &= !(e0 != e1);

    // test from identical seeds
    Engine e2(13), e3(13);
    result &= !(e2 != e3);

    // test different seeds aren't equal
    Engine e4(7), e5(13);
    result &= (e4 != e5);

    // test reseeding engine to the same seed causes equality
    e4.seed(13);
    result &= !(e4 != e5);

    // test different discards causes inequality
    Engine e6(13), e7(13);
    e6.discard(500);
    e7.discard(1000);
    result &= (e6 != e7);

    return result;
  }
};

template <typename Distribution, typename Engine>
struct ValidateDistributionMin
{
  using random_engine = Engine;

  _CCCL_HOST_DEVICE ValidateDistributionMin(const Distribution& dd)
      : d(dd)
  {}

  _CCCL_HOST_DEVICE bool operator()(void)
  {
    Engine e;

    bool result = true;

    for (int i = 0; i < 10000; ++i)
    {
      result &= (d(e) >= d.min());
    }

    return result;
  }

  Distribution d;
};

template <typename Distribution, typename Engine>
struct ValidateDistributionMax
{
  using random_engine = Engine;

  _CCCL_HOST_DEVICE ValidateDistributionMax(const Distribution& dd)
      : d(dd)
  {}

  _CCCL_HOST_DEVICE bool operator()(void)
  {
    Engine e;

    bool result = true;

    for (int i = 0; i < 10000; ++i)
    {
      result &= (d(e) <= d.max());
    }

    return result;
  }

  Distribution d;
};

template <typename Distribution>
struct ValidateDistributionEqual
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    return d0 == d1;
  }

  Distribution d0, d1;
};

template <typename Distribution>
struct ValidateDistributionUnqual
{
  _CCCL_HOST_DEVICE bool operator()(void) const
  {
    return d0 != d1;
  }

  Distribution d0, d1;
};

template <typename Engine, std::uint64_t value_10000>
void TestEngineValidation()
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngine<Engine>(value_10000));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngine<Engine>(value_10000));

  ASSERT_EQUAL(true, d[0]);
}

template <typename Engine>
void TestEngineMax()
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngineMax<Engine>());

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngineMax<Engine>());

  ASSERT_EQUAL(true, d[0]);
}

template <typename Engine>
void TestEngineMin()
{
  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), ValidateEngineMin<Engine>());

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), ValidateEngineMin<Engine>());

  ASSERT_EQUAL(true, d[0]);
}

template <typename Engine>
void TestEngineSaveRestore()
{
  // create a default engine
  Engine e0;

  // run it for a while
  e0.discard(10000);

  // save it
  std::stringstream ss;
  ss << e0;

  // run it a while longer
  e0.discard(10000);

  // restore old state
  Engine e1;
  ss >> e1;

  // run e1 a while longer
  e1.discard(10000);

  // both should return the same result

  ASSERT_EQUAL(e0(), e1());
}

template <typename Engine>
void TestEngineEqual()
{
  ValidateEngineEqual<Engine> f;

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), f);

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), f);

  ASSERT_EQUAL(true, d[0]);
}

template <typename Engine>
void TestEngineUnequal()
{
  ValidateEngineUnequal<Engine> f;

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), f);

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), f);

  ASSERT_EQUAL(true, d[0]);
}

void TestRanlux24BaseValidation()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineValidation<Engine, 7937952u>();
}
DECLARE_UNITTEST(TestRanlux24BaseValidation);

void TestRanlux24BaseMin()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseMin);

void TestRanlux24BaseMax()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseMax);

void TestRanlux24BaseSaveRestore()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseSaveRestore);

void TestRanlux24BaseEqual()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseEqual);

void TestRanlux24BaseUnequal()
{
  using Engine = thrust::random::ranlux24_base;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux24BaseUnequal);

void TestRanlux48BaseValidation()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineValidation<Engine, 192113843633948ull>();
}
DECLARE_UNITTEST(TestRanlux48BaseValidation);

void TestRanlux48BaseMin()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseMin);

void TestRanlux48BaseMax()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseMax);

void TestRanlux48BaseSaveRestore()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseSaveRestore);

void TestRanlux48BaseEqual()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux48BaseEqual);

#if defined(__INTEL_COMPILER) && 1800 >= __INTEL_COMPILER
void TestRanlux48BaseUnequal()
{
  // ICPC has a known failure with this test.
  // See nvbug 200414000.
  KNOWN_FAILURE;
}
#else
void TestRanlux48BaseUnequal()
{
  using Engine = thrust::random::ranlux48_base;

  TestEngineUnequal<Engine>();
}
#endif
DECLARE_UNITTEST(TestRanlux48BaseUnequal);

void TestMinstdRandValidation()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineValidation<Engine, 399268537u>();
}
DECLARE_UNITTEST(TestMinstdRandValidation);

void TestMinstdRandMin()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandMin);

void TestMinstdRandMax()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandMax);

void TestMinstdRandSaveRestore()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandSaveRestore);

void TestMinstdRandEqual()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandEqual);

void TestMinstdRandUnequal()
{
  using Engine = thrust::random::minstd_rand;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestMinstdRandUnequal);

void TestMinstdRand0Validation()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineValidation<Engine, 1043618065u>();
}
DECLARE_UNITTEST(TestMinstdRand0Validation);

void TestMinstdRand0Min()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Min);

void TestMinstdRand0Max()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Max);

void TestMinstdRand0SaveRestore()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0SaveRestore);

void TestMinstdRand0Equal()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Equal);

void TestMinstdRand0Unequal()
{
  using Engine = thrust::random::minstd_rand0;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestMinstdRand0Unequal);

void TestTaus88Validation()
{
  using Engine = thrust::random::taus88;

  TestEngineValidation<Engine, 3535848941ull>();
}
DECLARE_UNITTEST(TestTaus88Validation);

void TestTaus88Min()
{
  using Engine = thrust::random::taus88;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestTaus88Min);

void TestTaus88Max()
{
  using Engine = thrust::random::taus88;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestTaus88Max);

void TestTaus88SaveRestore()
{
  using Engine = thrust::random::taus88;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestTaus88SaveRestore);

void TestTaus88Equal()
{
  using Engine = thrust::random::taus88;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestTaus88Equal);

void TestTaus88Unequal()
{
  using Engine = thrust::random::taus88;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestTaus88Unequal);

void TestRanlux24Validation()
{
  using Engine = thrust::random::ranlux24;

  TestEngineValidation<Engine, 9901578>();
}
DECLARE_UNITTEST(TestRanlux24Validation);

void TestRanlux24Min()
{
  using Engine = thrust::random::ranlux24;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Min);

void TestRanlux24Max()
{
  using Engine = thrust::random::ranlux24;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Max);

void TestRanlux24SaveRestore()
{
  using Engine = thrust::random::ranlux24;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux24SaveRestore);

void TestRanlux24Equal()
{
  using Engine = thrust::random::ranlux24;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Equal);

void TestRanlux24Unequal()
{
  using Engine = thrust::random::ranlux24;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux24Unequal);

void TestRanlux48Validation()
{
  using Engine = thrust::random::ranlux48;

  TestEngineValidation<Engine, 88229545517833ull>();
}
DECLARE_UNITTEST(TestRanlux48Validation);

void TestRanlux48Min()
{
  using Engine = thrust::random::ranlux48;

  TestEngineMin<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Min);

void TestRanlux48Max()
{
  using Engine = thrust::random::ranlux48;

  TestEngineMax<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Max);

void TestRanlux48SaveRestore()
{
  using Engine = thrust::random::ranlux48;

  TestEngineSaveRestore<Engine>();
}
DECLARE_UNITTEST(TestRanlux48SaveRestore);

void TestRanlux48Equal()
{
  using Engine = thrust::random::ranlux48;

  TestEngineEqual<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Equal);

void TestRanlux48Unequal()
{
  using Engine = thrust::random::ranlux48;

  TestEngineUnequal<Engine>();
}
DECLARE_UNITTEST(TestRanlux48Unequal);

THRUST_DISABLE_MSVC_WARNING_BEGIN(4305) // truncation warning
template <typename Distribution, typename Validator>
void ValidateDistributionCharacteristic()
{
  using Engine = typename Validator::random_engine;

  // test default-constructed Distribution

  // test host
  thrust::host_vector<bool> h(1);
  thrust::generate(h.begin(), h.end(), Validator(Distribution()));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::device_vector<bool> d(1);
  thrust::generate(d.begin(), d.end(), Validator(Distribution()));

  ASSERT_EQUAL(true, d[0]);

  // test distribution & engine with comparable ranges
  // only do this if they have the same result_type
  if (::cuda::std::is_same<typename Distribution::result_type, typename Engine::result_type>::value)
  {
    // test Distribution with same range as engine

    // test host
    thrust::generate(h.begin(), h.end(), Validator(Distribution(Engine::min, Engine::max)));

    ASSERT_EQUAL(true, h[0]);

    // test device
    thrust::generate(d.begin(), d.end(), Validator(Distribution(Engine::min, Engine::max)));

    ASSERT_EQUAL(true, d[0]);

    // test Distribution with smaller range than engine

    // test host
    typename Distribution::result_type engine_range = Engine::max - Engine::min;
    thrust::generate(h.begin(), h.end(), Validator(Distribution(engine_range / 3, (2 * engine_range) / 3)));

    ASSERT_EQUAL(true, h[0]);

    // test device
    thrust::generate(d.begin(), d.end(), Validator(Distribution(engine_range / 3, (2 * engine_range) / 3)));

    ASSERT_EQUAL(true, d[0]);
  }

  // test Distribution with a very small range

  // test host
  thrust::generate(h.begin(), h.end(), Validator(Distribution(1, 6)));

  ASSERT_EQUAL(true, h[0]);

  // test device
  thrust::generate(d.begin(), d.end(), Validator(Distribution(1, 6)));

  ASSERT_EQUAL(true, d[0]);
}
THRUST_DISABLE_MSVC_WARNING_END(4305)

template <typename Distribution>
void TestDistributionSaveRestore()
{
  // create a default distribution
  Distribution d0(7, 13);

  // save it
  std::stringstream ss;
  ss << d0;

  // restore old state
  Distribution d1;
  ss >> d1;

  ASSERT_EQUAL(d0, d1);
}

void TestUniformIntDistributionMin()
{
  using int_dist  = thrust::random::uniform_int_distribution<int>;
  using uint_dist = thrust::random::uniform_int_distribution<unsigned int>;

  ValidateDistributionCharacteristic<int_dist, ValidateDistributionMin<int_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<uint_dist, ValidateDistributionMin<uint_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestUniformIntDistributionMin);

void TestUniformIntDistributionMax()
{
  using int_dist  = thrust::random::uniform_int_distribution<int>;
  using uint_dist = thrust::random::uniform_int_distribution<unsigned int>;

  ValidateDistributionCharacteristic<int_dist, ValidateDistributionMax<int_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<uint_dist, ValidateDistributionMax<uint_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestUniformIntDistributionMax);

void TestUniformIntDistributionSaveRestore()
{
  using int_dist  = thrust::random::uniform_int_distribution<int>;
  using uint_dist = thrust::random::uniform_int_distribution<unsigned int>;

  TestDistributionSaveRestore<int_dist>();
  TestDistributionSaveRestore<uint_dist>();
}
DECLARE_UNITTEST(TestUniformIntDistributionSaveRestore);

void TestUniformRealDistributionMin()
{
  using float_dist  = thrust::random::uniform_real_distribution<float>;
  using double_dist = thrust::random::uniform_real_distribution<double>;

  ValidateDistributionCharacteristic<float_dist, ValidateDistributionMin<float_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMin<double_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestUniformRealDistributionMin);

void TestUniformRealDistributionMax()
{
  using float_dist  = thrust::random::uniform_real_distribution<float>;
  using double_dist = thrust::random::uniform_real_distribution<double>;

  ValidateDistributionCharacteristic<float_dist, ValidateDistributionMax<float_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMax<double_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestUniformRealDistributionMax);

void TestUniformRealDistributionSaveRestore()
{
  using float_dist  = thrust::random::uniform_real_distribution<float>;
  using double_dist = thrust::random::uniform_real_distribution<double>;

  TestDistributionSaveRestore<float_dist>();
  TestDistributionSaveRestore<double_dist>();
}
DECLARE_UNITTEST(TestUniformRealDistributionSaveRestore);

void TestNormalDistributionMin()
{
  using float_dist  = thrust::random::normal_distribution<float>;
  using double_dist = thrust::random::normal_distribution<double>;

  ValidateDistributionCharacteristic<float_dist, ValidateDistributionMin<float_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMin<double_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestNormalDistributionMin);

void TestNormalDistributionMax()
{
  using float_dist  = thrust::random::normal_distribution<float>;
  using double_dist = thrust::random::normal_distribution<double>;

  ValidateDistributionCharacteristic<float_dist, ValidateDistributionMax<float_dist, thrust::minstd_rand>>();
  ValidateDistributionCharacteristic<double_dist, ValidateDistributionMax<double_dist, thrust::minstd_rand>>();
}
DECLARE_UNITTEST(TestNormalDistributionMax);

void TestNormalDistributionSaveRestore()
{
  using float_dist  = thrust::random::normal_distribution<float>;
  using double_dist = thrust::random::normal_distribution<double>;

  TestDistributionSaveRestore<float_dist>();
  TestDistributionSaveRestore<double_dist>();
}
DECLARE_UNITTEST(TestNormalDistributionSaveRestore);
