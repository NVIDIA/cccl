#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/iterator/retag.h>


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_difference(my_system &system,
                              InputIterator1,
                              InputIterator1,
                              InputIterator2,
                              InputIterator2,
                              OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestSetDifferenceDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::set_difference(sys,
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin(),
                         vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestSetDifferenceDispatchExplicit);


template<typename InputIterator1,
         typename InputIterator2,
         typename OutputIterator>
OutputIterator set_difference(my_tag,
                              InputIterator1,
                              InputIterator1,
                              InputIterator2,
                              InputIterator2,
                              OutputIterator result)
{
  *result = 13;
  return result;
}

void TestSetDifferenceDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::set_difference(thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()),
                         thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestSetDifferenceDispatchImplicit);


template<typename Vector>
void TestSetDifferenceSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  Iterator end = thrust::set_difference(a.begin(), a.end(),
                                        b.begin(), b.end(),
                                        result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetDifferenceSimple);


template<typename T>
void TestSetDifference(const size_t n)
{
  size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  thrust::host_vector<T> random = unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<T> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<T> h_b(random.begin() + n, random.end());
  
  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());
  
  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  for (size_t i = 0; i < num_sizes; i++)
  {
    size_t size = sizes[i];
    
    thrust::host_vector<T>   h_result(n + size);
    thrust::device_vector<T> d_result(n + size);

    typename thrust::host_vector<T>::iterator   h_end;
    typename thrust::device_vector<T>::iterator d_end;
    
    h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.begin() + size,
                                   h_result.begin());
    h_result.resize(h_end - h_result.begin());

    d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.begin() + size,
                                   d_result.begin());
    d_result.resize(d_end - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}
DECLARE_VARIABLE_UNITTEST(TestSetDifference);


template<typename T>
void TestSetDifferenceEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_a = temp; thrust::sort(h_a.begin(), h_a.end());
  thrust::host_vector<T> h_b = h_a;

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                 h_b.begin(), h_b.end(),
                                 h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                 d_b.begin(), d_b.end(),
                                 d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetDifferenceEquivalentRanges);


template<typename T>
void TestSetDifferenceMultiset(const size_t n)
{
  thrust::host_vector<T> vec = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = vec.begin();
      i != vec.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a(vec.begin(), vec.begin() + n);
  thrust::host_vector<T> h_b(vec.begin() + n, vec.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_difference(h_a.begin(), h_a.end(),
                                 h_b.begin(), h_b.end(),
                                 h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_difference(d_a.begin(), d_a.end(),
                                 d_b.begin(), d_b.end(),
                                 d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetDifferenceMultiset);

// FIXME: disabled on Windows, because it causes a failure on the internal CI system in one specific configuration.
// That failure will be tracked in a new NVBug, this is disabled to unblock submitting all the other changes.
#if THRUST_HOST_COMPILER != THRUST_HOST_COMPILER_MSVC
void TestSetDifferenceWithBigIndexesHelper(int magnitude)
{
    thrust::counting_iterator<long long> begin(0);
    thrust::counting_iterator<long long> end = begin + (1ll << magnitude);
    thrust::counting_iterator<long long> end_longer = end + 1;
    ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

    thrust::device_vector<long long> result;
    result.resize(1);
    thrust::set_difference(thrust::device, begin, end_longer, begin, end, result.begin());

    thrust::host_vector<long long> expected;
    expected.push_back(*end);

    ASSERT_EQUAL(result, expected);
}

void TestSetDifferenceWithBigIndexes()
{
    TestSetDifferenceWithBigIndexesHelper(30);
    TestSetDifferenceWithBigIndexesHelper(31);
    TestSetDifferenceWithBigIndexesHelper(32);
    TestSetDifferenceWithBigIndexesHelper(33);
}
DECLARE_UNITTEST(TestSetDifferenceWithBigIndexes);
#endif
