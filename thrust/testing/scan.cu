#include "hip/hip_runtime.h"
#include <thrust/detail/config.h>

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/retag.h>
#include <thrust/scan.h>

#include <cuda/std/array>

#include <unittest/unittest.h>

template <typename T>
struct max_functor
{
  _CCCL_HOST_DEVICE T operator()(T rhs, T lhs) const
  {
    return thrust::max(rhs, lhs);
  }
};

template <class Vector>
void TestScanSimple()
{
  using T = typename Vector::value_type;

  // icc miscompiles the intermediate sum updates for custom_numeric.
  // The issue doesn't happen with opts disabled, or on other compilers.
  // Printing the intermediate sum each iteration "fixes" the issue,
  // so likely a bad optimization.
#if _CCCL_COMPILER(ICC)
  if (std::is_same<T, custom_numeric>::value)
  {
    return;
  }
#endif

  typename Vector::iterator iter;

  Vector input(5);
  Vector result(5);
  Vector output(5);

  input = {1, 3, -2, 4, -5};
  Vector input_copy(input);

  // inclusive scan
  iter   = thrust::inclusive_scan(input.begin(), input.end(), output.begin());
  result = {1, 4, 2, 6, 1};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // exclusive scan
  iter   = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(0));
  result = {0, 1, 4, 2, 6};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // exclusive scan with init
  iter   = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(3));
  result = {3, 4, 7, 5, 9};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // inclusive scan with op
  iter   = thrust::inclusive_scan(input.begin(), input.end(), output.begin(), thrust::plus<T>());
  result = {1, 4, 2, 6, 1};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // inclusive scan with init and op
  iter   = thrust::inclusive_scan(input.begin(), input.end(), output.begin(), T(-1), thrust::multiplies<T>());
  result = {-1, -3, 6, 24, -120};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // exclusive scan with init and op
  iter   = thrust::exclusive_scan(input.begin(), input.end(), output.begin(), T(3), thrust::plus<T>());
  result = {3, 4, 7, 5, 9};
  ASSERT_EQUAL(std::size_t(iter - output.begin()), input.size());
  ASSERT_EQUAL(input, input_copy);
  ASSERT_EQUAL(output, result);

  // inplace inclusive scan
  input  = input_copy;
  iter   = thrust::inclusive_scan(input.begin(), input.end(), input.begin());
  result = {1, 4, 2, 6, 1};
  ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
  ASSERT_EQUAL(input, result);

  // inplace inclusive scan with init and op
  input  = input_copy;
  iter   = thrust::inclusive_scan(input.begin(), input.end(), input.begin(), T(3), thrust::plus<T>());
  result = {4, 7, 5, 9, 4};
  ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
  ASSERT_EQUAL(input, result);

  // inplace exclusive scan with init
  input  = input_copy;
  iter   = thrust::exclusive_scan(input.begin(), input.end(), input.begin(), T(3));
  result = {3, 4, 7, 5, 9};
  ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
  ASSERT_EQUAL(input, result);

  // inplace exclusive scan with implicit init=0
  input  = input_copy;
  iter   = thrust::exclusive_scan(input.begin(), input.end(), input.begin());
  result = {0, 1, 4, 2, 6};
  ASSERT_EQUAL(std::size_t(iter - input.begin()), input.size());
  ASSERT_EQUAL(input, result);
}
DECLARE_VECTOR_UNITTEST(TestScanSimple);

template <typename InputIterator, typename OutputIterator>
OutputIterator inclusive_scan(my_system& system, InputIterator, InputIterator, OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestInclusiveScanDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::inclusive_scan(sys, vec.begin(), vec.begin(), vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestInclusiveScanDispatchExplicit);

template <typename InputIterator, typename OutputIterator>
OutputIterator inclusive_scan(my_tag, InputIterator, InputIterator, OutputIterator result)
{
  *result = 13;
  return result;
}

void TestInclusiveScanDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::inclusive_scan(
    thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestInclusiveScanDispatchImplicit);

template <typename InputIterator, typename OutputIterator>
OutputIterator exclusive_scan(my_system& system, InputIterator, InputIterator, OutputIterator result)
{
  system.validate_dispatch();
  return result;
}

void TestExclusiveScanDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::exclusive_scan(sys, vec.begin(), vec.begin(), vec.begin());

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestExclusiveScanDispatchExplicit);

template <typename InputIterator, typename OutputIterator>
OutputIterator exclusive_scan(my_tag, InputIterator, InputIterator, OutputIterator result)
{
  *result = 13;
  return result;
}

void TestExclusiveScanDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::exclusive_scan(
    thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.begin()));

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestExclusiveScanDispatchImplicit);

void TestInclusiveScan32()
{
  using T  = int;
  size_t n = 32;

  thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(n);
  thrust::device_vector<T> d_output(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
  thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());

  ASSERT_EQUAL(d_output, h_output);
}
DECLARE_UNITTEST(TestInclusiveScan32);

void TestExclusiveScan32()
{
  using T  = int;
  size_t n = 32;
  T init   = 13;

  thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(n);
  thrust::device_vector<T> d_output(n);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), init);
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), init);

  ASSERT_EQUAL(d_output, h_output);
}
DECLARE_UNITTEST(TestExclusiveScan32);

template <class IntVector, class FloatVector>
void TestScanMixedTypes()
{
  // make sure we get types for default args and operators correct
  IntVector int_input{1, 2, 3, 4};
  FloatVector float_input{1.5, 2.5, 3.5, 4.5};
  IntVector int_output(4);
  FloatVector float_output(4);

  // float -> int should use plus<void> operator and float accumulator by default
  thrust::inclusive_scan(float_input.begin(), float_input.end(), int_output.begin());
  ASSERT_EQUAL(int_output[0], 1); // in: 1.5 accum: 1.5f out: 1
  ASSERT_EQUAL(int_output[1], 4); // in: 2.5 accum: 4.0f out: 4
  ASSERT_EQUAL(int_output[2], 7); // in: 3.5 accum: 7.5f out: 7
  ASSERT_EQUAL(int_output[3], 12); // in: 4.5 accum: 12.f out: 12

  // float -> float with plus<int> operator (float accumulator)
  thrust::inclusive_scan(float_input.begin(), float_input.end(), float_output.begin(), thrust::plus<int>());
  ASSERT_EQUAL(float_output[0], 1.5f); // in: 1.5 accum: 1.5f out: 1.5f
  ASSERT_EQUAL(float_output[1], 3.0f); // in: 2.5 accum: 3.0f out: 3.0f
  ASSERT_EQUAL(float_output[2], 6.0f); // in: 3.5 accum: 6.0f out: 6.0f
  ASSERT_EQUAL(float_output[3], 10.0f); // in: 4.5 accum: 10.f out: 10.f

  // float -> int should use plus<void> operator and float accumulator by default
  thrust::exclusive_scan(float_input.begin(), float_input.end(), int_output.begin());
  ASSERT_EQUAL(int_output[0], 0); // out: 0.0f  in: 1.5 accum: 1.5f
  ASSERT_EQUAL(int_output[1], 1); // out: 1.5f  in: 2.5 accum: 4.0f
  ASSERT_EQUAL(int_output[2], 4); // out: 4.0f  in: 3.5 accum: 7.5f
  ASSERT_EQUAL(int_output[3], 7); // out: 7.5f  in: 4.5 accum: 12.f

  // float -> int should use plus<> operator and float accumulator by default
  thrust::exclusive_scan(float_input.begin(), float_input.end(), int_output.begin(), (float) 5.5);
  ASSERT_EQUAL(int_output[0], 5); // out: 5.5f  in: 1.5 accum: 7.0f
  ASSERT_EQUAL(int_output[1], 7); // out: 7.0f  in: 2.5 accum: 9.5f
  ASSERT_EQUAL(int_output[2], 9); // out: 9.5f  in: 3.5 accum: 13.0f
  ASSERT_EQUAL(int_output[3], 13); // out: 13.f  in: 4.5 accum: 17.4f

  // int -> float should use using plus<> operator and int accumulator by default
  thrust::inclusive_scan(int_input.begin(), int_input.end(), float_output.begin());
  ASSERT_EQUAL(float_output[0], 1.f); // in: 1 accum: 1  out: 1
  ASSERT_EQUAL(float_output[1], 3.f); // in: 2 accum: 3  out: 3
  ASSERT_EQUAL(float_output[2], 6.f); // in: 3 accum: 6  out: 6
  ASSERT_EQUAL(float_output[3], 10.f); // in: 4 accum: 10 out: 10

  // int -> float + float init_value should use using plus<> operator and
  // float accumulator by default
  thrust::exclusive_scan(int_input.begin(), int_input.end(), float_output.begin(), (float) 5.5);
  ASSERT_EQUAL(float_output[0], 5.5f); // out: 5.5f  in: 1 accum: 6.5f
  ASSERT_EQUAL(float_output[1], 6.5f); // out: 6.0f  in: 2 accum: 8.5f
  ASSERT_EQUAL(float_output[2], 8.5f); // out: 8.0f  in: 3 accum: 11.5f
  ASSERT_EQUAL(float_output[3], 11.5f); // out: 11.f  in: 4 accum: 15.5f
}
void TestScanMixedTypesHost()
{
  TestScanMixedTypes<thrust::host_vector<int>, thrust::host_vector<float>>();
}
DECLARE_UNITTEST(TestScanMixedTypesHost);
void TestScanMixedTypesDevice()
{
  TestScanMixedTypes<thrust::device_vector<int>, thrust::device_vector<float>>();
}
DECLARE_UNITTEST(TestScanMixedTypesDevice);

template <typename T>
struct TestScanWithOperator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T> h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), max_functor<T>());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), max_functor<T>());
    ASSERT_EQUAL(d_output, h_output);

    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), T(13), max_functor<T>());
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), T(13), max_functor<T>());
    ASSERT_EQUAL(d_output, h_output);
  }
};
VariableUnitTest<TestScanWithOperator, SignedIntegralTypes> TestScanWithOperatorInstance;

template <typename T>
struct TestScanWithOperatorToDiscardIterator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::discard_iterator<> reference(n);

    thrust::discard_iterator<> h_result =
      thrust::inclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), max_functor<T>());

    thrust::discard_iterator<> d_result =
      thrust::inclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), max_functor<T>());

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);

    h_result =
      thrust::exclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), T(13), max_functor<T>());

    d_result =
      thrust::exclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), T(13), max_functor<T>());

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
  }
};
VariableUnitTest<TestScanWithOperatorToDiscardIterator,
                 unittest::type_list<unittest::int8_t, unittest::int16_t, unittest::int32_t>>
  TestScanWithOperatorToDiscardIteratorInstance;

template <typename T>
struct TestScan
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::host_vector<T> h_output(n);
    thrust::device_vector<T> d_output(n);

    thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);

    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);

    thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), (T) 11);
    thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), (T) 11);
    ASSERT_EQUAL(d_output, h_output);

    // in-place scans
    h_output = h_input;
    d_output = d_input;
    thrust::inclusive_scan(h_output.begin(), h_output.end(), h_output.begin());
    thrust::inclusive_scan(d_output.begin(), d_output.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);

    h_output = h_input;
    d_output = d_input;
    thrust::exclusive_scan(h_output.begin(), h_output.end(), h_output.begin());
    thrust::exclusive_scan(d_output.begin(), d_output.end(), d_output.begin());
    ASSERT_EQUAL(d_output, h_output);
  }
};
VariableUnitTest<TestScan, IntegralTypes> TestScanInstance;

template <typename T>
struct TestScanToDiscardIterator
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T> h_input   = unittest::random_integers<T>(n);
    thrust::device_vector<T> d_input = h_input;

    thrust::discard_iterator<> h_result =
      thrust::inclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator());

    thrust::discard_iterator<> d_result =
      thrust::inclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator());

    thrust::discard_iterator<> reference(n);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);

    h_result = thrust::exclusive_scan(h_input.begin(), h_input.end(), thrust::make_discard_iterator(), (T) 11);

    d_result = thrust::exclusive_scan(d_input.begin(), d_input.end(), thrust::make_discard_iterator(), (T) 11);

    ASSERT_EQUAL_QUIET(reference, h_result);
    ASSERT_EQUAL_QUIET(reference, d_result);
  }
};
VariableUnitTest<TestScanToDiscardIterator, unittest::type_list<unittest::int8_t, unittest::int16_t, unittest::int32_t>>
  TestScanToDiscardIteratorInstance;

void TestScanMixedTypes()
{
  const unsigned int n = 113;

  thrust::host_vector<unsigned int> h_input = unittest::random_integers<unsigned int>(n);
  for (size_t i = 0; i < n; i++)
  {
    h_input[i] %= 10;
  }
  thrust::device_vector<unsigned int> d_input = h_input;

  thrust::host_vector<float> h_float_output(n);
  thrust::device_vector<float> d_float_output(n);
  thrust::host_vector<int> h_int_output(n);
  thrust::device_vector<int> d_int_output(n);

  // mixed input/output types
  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin());
  thrust::inclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin());
  ASSERT_EQUAL(d_float_output, h_float_output);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin(), (float) 3.5);
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin(), (float) 3.5);
  ASSERT_EQUAL(d_float_output, h_float_output);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_float_output.begin(), (int) 3);
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_float_output.begin(), (int) 3);
  ASSERT_EQUAL(d_float_output, h_float_output);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_int_output.begin(), (int) 3);
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_int_output.begin(), (int) 3);
  ASSERT_EQUAL(d_int_output, h_int_output);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_int_output.begin(), (float) 3.5);
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_int_output.begin(), (float) 3.5);
  ASSERT_EQUAL(d_int_output, h_int_output);
}
DECLARE_UNITTEST(TestScanMixedTypes);

template <typename T, unsigned int N>
void _TestScanWithLargeTypes()
{
  size_t n = (1024 * 1024) / sizeof(FixedVector<T, N>);

  thrust::host_vector<FixedVector<T, N>> h_input(n);
  thrust::host_vector<FixedVector<T, N>> h_output(n);

  for (size_t i = 0; i < h_input.size(); i++)
  {
    h_input[i] = FixedVector<T, N>(static_cast<T>(i));
  }

  thrust::device_vector<FixedVector<T, N>> d_input = h_input;
  thrust::device_vector<FixedVector<T, N>> d_output(n);

  thrust::inclusive_scan(h_input.begin(), h_input.end(), h_output.begin());
  thrust::inclusive_scan(d_input.begin(), d_input.end(), d_output.begin());

  ASSERT_EQUAL_QUIET(h_output, d_output);

  thrust::exclusive_scan(h_input.begin(), h_input.end(), h_output.begin(), FixedVector<T, N>(0));
  thrust::exclusive_scan(d_input.begin(), d_input.end(), d_output.begin(), FixedVector<T, N>(0));

  ASSERT_EQUAL_QUIET(h_output, d_output);
}

void TestScanWithLargeTypes()
{
  _TestScanWithLargeTypes<int, 1>();

#if !defined(__QNX__)
  _TestScanWithLargeTypes<int, 8>();
  _TestScanWithLargeTypes<int, 64>();
#else
  KNOWN_FAILURE;
#endif
}
DECLARE_UNITTEST(TestScanWithLargeTypes);

template <typename T>
struct plus_mod3
{
  T* table;

  plus_mod3(T* table)
      : table(table)
  {}

  _CCCL_HOST_DEVICE T operator()(T a, T b)
  {
    return table[(int) (a + b)];
  }
};

template <typename Vector>
void TestInclusiveScanWithIndirection()
{
  // add numbers modulo 3 with external lookup table
  using T = typename Vector::value_type;

  Vector data{0, 1, 2, 1, 2, 0, 1};
  Vector table{0, 1, 2, 0, 1, 2};
  thrust::inclusive_scan(data.begin(), data.end(), data.begin(), plus_mod3<T>(thrust::raw_pointer_cast(&table[0])));

  ASSERT_EQUAL(data, (Vector{0, 1, 0, 1, 0, 0, 1}));
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestInclusiveScanWithIndirection);

template <typename T>
struct const_ref_plus_mod3
{
  T* table;

  const_ref_plus_mod3(T* table)
      : table(table)
  {}

  _CCCL_HOST_DEVICE const T& operator()(T a, T b)
  {
    return table[(int) (a + b)];
  }
};

template <typename Vector>
void TestInclusiveScanWithConstAccumulator()
{
  // add numbers modulo 3 with external lookup table
  using T = typename Vector::value_type;

  Vector data{0, 1, 2, 1, 2, 0, 1};
  Vector table{0, 1, 2, 0, 1, 2};
  thrust::inclusive_scan(
    data.begin(), data.end(), data.begin(), const_ref_plus_mod3<T>(thrust::raw_pointer_cast(&table[0])));

  ASSERT_EQUAL(data, (Vector{0, 1, 0, 1, 0, 0, 1}));
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestInclusiveScanWithConstAccumulator);

struct only_set_when_expected_it
{
  long long expected;
  bool* flag;

  _CCCL_HOST_DEVICE only_set_when_expected_it operator++() const
  {
    return *this;
  }
  _CCCL_HOST_DEVICE only_set_when_expected_it operator*() const
  {
    return *this;
  }
  template <typename Difference>
  _CCCL_HOST_DEVICE only_set_when_expected_it operator+(Difference) const
  {
    return *this;
  }
  template <typename Index>
  _CCCL_HOST_DEVICE only_set_when_expected_it operator[](Index) const
  {
    return *this;
  }

  _CCCL_DEVICE void operator=(long long value) const
  {
    if (value == expected)
    {
      *flag = true;
    }
  }
};

THRUST_NAMESPACE_BEGIN
template <>
struct iterator_traits<only_set_when_expected_it>
{
  using value_type = long long;
  using reference  = only_set_when_expected_it;
};
THRUST_NAMESPACE_END

void TestInclusiveScanWithBigIndexesHelper(int magnitude)
{
  thrust::constant_iterator<long long> begin(1);
  thrust::constant_iterator<long long> end = begin + (1ll << magnitude);
  ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

  thrust::device_ptr<bool> has_executed = thrust::device_malloc<bool>(1);
  *has_executed                         = false;

  only_set_when_expected_it out = {(1ll << magnitude), thrust::raw_pointer_cast(has_executed)};

  thrust::inclusive_scan(thrust::device, begin, end, out);

  bool has_executed_h = *has_executed;
  thrust::device_free(has_executed);

  ASSERT_EQUAL(has_executed_h, true);
}

void TestInclusiveScanWithBigIndexes()
{
  TestInclusiveScanWithBigIndexesHelper(30);
  TestInclusiveScanWithBigIndexesHelper(31);
#ifndef THRUST_FORCE_32_BIT_OFFSET_TYPE
  TestInclusiveScanWithBigIndexesHelper(32);
  TestInclusiveScanWithBigIndexesHelper(33);
#endif
}

DECLARE_UNITTEST(TestInclusiveScanWithBigIndexes);

void TestExclusiveScanWithBigIndexesHelper(int magnitude)
{
  thrust::constant_iterator<long long> begin(1);
  thrust::constant_iterator<long long> end = begin + (1ll << magnitude);
  ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

  thrust::device_ptr<bool> has_executed = thrust::device_malloc<bool>(1);
  *has_executed                         = false;

  only_set_when_expected_it out = {(1ll << magnitude) - 1, thrust::raw_pointer_cast(has_executed)};

  thrust::exclusive_scan(thrust::device, begin, end, out, 0ll);

  bool has_executed_h = *has_executed;
  thrust::device_free(has_executed);

  ASSERT_EQUAL(has_executed_h, true);
}

void TestExclusiveScanWithBigIndexes()
{
  TestExclusiveScanWithBigIndexesHelper(30);
  TestExclusiveScanWithBigIndexesHelper(31);
#ifndef THRUST_FORCE_32_BIT_OFFSET_TYPE
  TestExclusiveScanWithBigIndexesHelper(32);
  TestExclusiveScanWithBigIndexesHelper(33);
#endif
}

DECLARE_UNITTEST(TestExclusiveScanWithBigIndexes);

struct Int
{
  int i{};
  _CCCL_HOST_DEVICE explicit Int(int num)
      : i(num)
  {}
  _CCCL_HOST_DEVICE Int()
      : i{}
  {}
  _CCCL_HOST_DEVICE Int operator+(Int const& o) const
  {
    return Int{this->i + o.i};
  }
};

void TestInclusiveScanWithUserDefinedType()
{
  thrust::device_vector<Int> vec(5, Int{1});

  thrust::inclusive_scan(thrust::device, vec.cbegin(), vec.cend(), vec.begin());

  ASSERT_EQUAL(static_cast<Int>(vec.back()).i, 5);
}
DECLARE_UNITTEST(TestInclusiveScanWithUserDefinedType);

// Represents a permutation as a tuple of integers, see also: https://en.wikipedia.org/wiki/Permutation
// We need a distinct type (instead of an alias) for operator<< to be found via ADL
struct permutation_t : ::cuda::std::array<int, 5>
{
  permutation_t() = default;

  constexpr _CCCL_HOST_DEVICE permutation_t(int a, int b, int c, int d, int e)
      : ::cuda::std::array<int, 5>{a, b, c, d, e}
  {}

  friend std::ostream& operator<<(std::ostream& os, const permutation_t& p)
  {
    os << '{';
    for (std::size_t i = 0; i < p.size(); i++)
    {
      if (i > 0)
      {
        os << ", ";
      }
      os << p[i];
    }
    return os << '}';
  }
};

// Composes two permutations. This operation is associative, but not commutative.
struct composition_op_t
{
  _CCCL_HOST_DEVICE permutation_t operator()(permutation_t lhs, permutation_t rhs) const
  {
    permutation_t result;
    for (std::size_t i = 0; i < lhs.size(); i++)
    {
      result[i] = rhs[lhs[i]];
    }
    return result;
  }
};

void TestInclusiveScanWithNonCommutativeOp()
{
  const thrust::device_vector<permutation_t> input = {
    {3, 2, 0, 1, 4},
    {2, 4, 0, 1, 3},
    {3, 2, 1, 4, 0},
    {4, 3, 1, 0, 2},
    {0, 3, 2, 4, 1},
    {3, 2, 1, 0, 4},
    {3, 4, 1, 2, 0},
    {4, 2, 1, 0, 3},
    {4, 0, 1, 3, 2},
    {0, 2, 3, 1, 4}};
  thrust::device_vector<permutation_t> output(10);
  constexpr auto identity = permutation_t{0, 1, 2, 3, 4};

  thrust::inclusive_scan(input.begin(), input.end(), output.begin(), composition_op_t{});
  ASSERT_EQUAL(
    output,
    (thrust::device_vector<permutation_t>{
      {3, 2, 0, 1, 4},
      {1, 0, 2, 4, 3},
      {2, 3, 1, 0, 4},
      {1, 0, 3, 4, 2},
      {3, 0, 4, 1, 2},
      {0, 3, 4, 2, 1},
      {3, 2, 0, 1, 4},
      {0, 1, 4, 2, 3},
      {4, 0, 2, 1, 3},
      {4, 0, 3, 2, 1}}));

  thrust::exclusive_scan(input.begin(), input.end(), output.begin(), identity, composition_op_t{});
  ASSERT_EQUAL(
    output,
    (thrust::device_vector<permutation_t>{
      {0, 1, 2, 3, 4},
      {3, 2, 0, 1, 4},
      {1, 0, 2, 4, 3},
      {2, 3, 1, 0, 4},
      {1, 0, 3, 4, 2},
      {3, 0, 4, 1, 2},
      {0, 3, 4, 2, 1},
      {3, 2, 0, 1, 4},
      {0, 1, 4, 2, 3},
      {4, 0, 2, 1, 3}}));
}
DECLARE_UNITTEST(TestInclusiveScanWithNonCommutativeOp);
