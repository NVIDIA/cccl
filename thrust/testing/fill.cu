#include "hip/hip_runtime.h"
#include <thrust/fill.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/retag.h>
#include <thrust/iterator/zip_iterator.h>

#include <algorithm>

#include <unittest/unittest.h>

_CCCL_DIAG_PUSH
_CCCL_DIAG_SUPPRESS_MSVC(4244 4267) // possible loss of data

template <class Vector>
void TestFillSimple()
{
  using T = typename Vector::value_type;

  Vector v{0, 1, 2, 3, 4};

  thrust::fill(v.begin() + 1, v.begin() + 4, (T) 7);

  Vector ref{0, 7, 7, 7, 4};
  ASSERT_EQUAL(v, ref);

  thrust::fill(v.begin() + 0, v.begin() + 3, (T) 8);

  ref = {8, 8, 8, 7, 4};
  ASSERT_EQUAL(v, ref);

  thrust::fill(v.begin() + 2, v.end(), (T) 9);

  ref = {8, 8, 9, 9, 9};
  ASSERT_EQUAL(v, ref);

  thrust::fill(v.begin(), v.end(), (T) 1);

  ref = Vector(5, 1);
  ASSERT_EQUAL(v, ref);
}
DECLARE_VECTOR_UNITTEST(TestFillSimple);

void TestFillDiscardIterator()
{
  // there's no result to check because fill returns void
  thrust::fill(
    thrust::discard_iterator<thrust::host_system_tag>(), thrust::discard_iterator<thrust::host_system_tag>(10), 13);

  thrust::fill(
    thrust::discard_iterator<thrust::device_system_tag>(), thrust::discard_iterator<thrust::device_system_tag>(10), 13);
}
DECLARE_UNITTEST(TestFillDiscardIterator);

template <class Vector>
void TestFillMixedTypes()
{
  Vector v(4);

  thrust::fill(v.begin(), v.end(), bool(true));

  Vector ref(4, 1);
  ASSERT_EQUAL(v, ref);

  thrust::fill(v.begin(), v.end(), char(20));

  ref = Vector(4, 20);
  ASSERT_EQUAL(v, ref);
}
DECLARE_VECTOR_UNITTEST(TestFillMixedTypes);

template <typename T>
void TestFill(size_t n)
{
  thrust::host_vector<T> h_data   = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;

  thrust::fill(h_data.begin() + std::min((size_t) 1, n), h_data.begin() + std::min((size_t) 3, n), (T) 0);
  thrust::fill(d_data.begin() + std::min((size_t) 1, n), d_data.begin() + std::min((size_t) 3, n), (T) 0);

  ASSERT_EQUAL(h_data, d_data);

  thrust::fill(h_data.begin() + std::min((size_t) 117, n), h_data.begin() + std::min((size_t) 367, n), (T) 1);
  thrust::fill(d_data.begin() + std::min((size_t) 117, n), d_data.begin() + std::min((size_t) 367, n), (T) 1);

  ASSERT_EQUAL(h_data, d_data);

  thrust::fill(h_data.begin() + std::min((size_t) 8, n), h_data.begin() + std::min((size_t) 259, n), (T) 2);
  thrust::fill(d_data.begin() + std::min((size_t) 8, n), d_data.begin() + std::min((size_t) 259, n), (T) 2);

  ASSERT_EQUAL(h_data, d_data);

  thrust::fill(h_data.begin() + std::min((size_t) 3, n), h_data.end(), (T) 3);
  thrust::fill(d_data.begin() + std::min((size_t) 3, n), d_data.end(), (T) 3);

  ASSERT_EQUAL(h_data, d_data);

  thrust::fill(h_data.begin(), h_data.end(), (T) 4);
  thrust::fill(d_data.begin(), d_data.end(), (T) 4);

  ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestFill);

template <class Vector>
void TestFillNSimple()
{
  using T = typename Vector::value_type;

  Vector v{0, 1, 2, 3, 4};

  typename Vector::iterator iter = thrust::fill_n(v.begin() + 1, 3, (T) 7);

  Vector ref{0, 7, 7, 7, 4};
  ASSERT_EQUAL(v, ref);

  ASSERT_EQUAL_QUIET(v.begin() + 4, iter);

  iter = thrust::fill_n(v.begin() + 0, 3, (T) 8);

  ref = {8, 8, 8, 7, 4};
  ASSERT_EQUAL(v, ref);

  ASSERT_EQUAL_QUIET(v.begin() + 3, iter);

  iter = thrust::fill_n(v.begin() + 2, 3, (T) 9);

  ref = {8, 8, 9, 9, 9};
  ASSERT_EQUAL(v, ref);

  ASSERT_EQUAL_QUIET(v.end(), iter);

  iter = thrust::fill_n(v.begin(), v.size(), (T) 1);

  ref = Vector(5, 1);
  ASSERT_EQUAL(v, ref);

  ASSERT_EQUAL_QUIET(v.end(), iter);
}
DECLARE_VECTOR_UNITTEST(TestFillNSimple);

void TestFillNDiscardIterator()
{
  thrust::discard_iterator<thrust::host_system_tag> h_result =
    thrust::fill_n(thrust::discard_iterator<thrust::host_system_tag>(), 10, 13);

  thrust::discard_iterator<thrust::device_system_tag> d_result =
    thrust::fill_n(thrust::discard_iterator<thrust::device_system_tag>(), 10, 13);

  thrust::discard_iterator<> reference(10);

  ASSERT_EQUAL_QUIET(reference, h_result);
  ASSERT_EQUAL_QUIET(reference, d_result);
}
DECLARE_UNITTEST(TestFillNDiscardIterator);

template <class Vector>
void TestFillNMixedTypes()
{
  Vector v(4);

  typename Vector::iterator iter = thrust::fill_n(v.begin(), v.size(), bool(true));

  Vector ref(4, 1);
  ASSERT_EQUAL(v, ref);
  ASSERT_EQUAL_QUIET(v.end(), iter);

  iter = thrust::fill_n(v.begin(), v.size(), char(20));

  ref = Vector(4, 20);
  ASSERT_EQUAL(v, ref);
  ASSERT_EQUAL_QUIET(v.end(), iter);
}
DECLARE_VECTOR_UNITTEST(TestFillNMixedTypes);

template <typename T>
void TestFillN(size_t n)
{
  thrust::host_vector<T> h_data   = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;

  size_t begin_offset = std::min<size_t>(1, n);
  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t) 3, n) - begin_offset, (T) 0);
  thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t) 3, n) - begin_offset, (T) 0);

  ASSERT_EQUAL(h_data, d_data);

  begin_offset = std::min<size_t>(117, n);
  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t) 367, n) - begin_offset, (T) 1);
  thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t) 367, n) - begin_offset, (T) 1);

  ASSERT_EQUAL(h_data, d_data);

  begin_offset = std::min<size_t>(8, n);
  thrust::fill_n(h_data.begin() + begin_offset, std::min((size_t) 259, n) - begin_offset, (T) 2);
  thrust::fill_n(d_data.begin() + begin_offset, std::min((size_t) 259, n) - begin_offset, (T) 2);

  ASSERT_EQUAL(h_data, d_data);

  begin_offset = std::min<size_t>(3, n);
  thrust::fill_n(h_data.begin() + begin_offset, h_data.size() - begin_offset, (T) 3);
  thrust::fill_n(d_data.begin() + begin_offset, d_data.size() - begin_offset, (T) 3);

  ASSERT_EQUAL(h_data, d_data);

  thrust::fill_n(h_data.begin(), h_data.size(), (T) 4);
  thrust::fill_n(d_data.begin(), d_data.size(), (T) 4);

  ASSERT_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestFillN);

template <typename Vector>
void TestFillZipIterator()
{
  using T = typename Vector::value_type;

  Vector v1(3, T(0));
  Vector v2(3, T(0));
  Vector v3(3, T(0));

  thrust::fill(thrust::make_zip_iterator(thrust::make_tuple(v1.begin(), v2.begin(), v3.begin())),
               thrust::make_zip_iterator(thrust::make_tuple(v1.end(), v2.end(), v3.end())),
               thrust::tuple<T, T, T>(4, 7, 13));

  Vector ref1{4, 4, 4};
  ASSERT_EQUAL(ref1, v1);

  Vector ref2{7, 7, 7};
  ASSERT_EQUAL(ref2, v2);

  Vector ref3{13, 13, 13};
  ASSERT_EQUAL(ref3, v3);
};
DECLARE_VECTOR_UNITTEST(TestFillZipIterator);

void TestFillTuple()
{
  using T     = int;
  using Tuple = thrust::tuple<T, T>;

  thrust::host_vector<Tuple> h(3, Tuple(0, 0));
  thrust::device_vector<Tuple> d(3, Tuple(0, 0));

  thrust::fill(h.begin(), h.end(), Tuple(4, 7));
  thrust::fill(d.begin(), d.end(), Tuple(4, 7));

  ASSERT_EQUAL_QUIET(h, d);
};
DECLARE_UNITTEST(TestFillTuple);

struct TypeWithTrivialAssigment
{
  int x, y, z;
};

void TestFillWithTrivialAssignment()
{
  using T = TypeWithTrivialAssigment;

  thrust::host_vector<T> h(1);
  thrust::device_vector<T> d(1);

  ASSERT_EQUAL(h[0].x, 0);
  ASSERT_EQUAL(h[0].y, 0);
  ASSERT_EQUAL(h[0].z, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).x, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).y, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).z, 0);

  T val;
  val.x = 10;
  val.y = 20;
  val.z = -1;

  thrust::fill(h.begin(), h.end(), val);
  thrust::fill(d.begin(), d.end(), val);

  ASSERT_EQUAL(h[0].x, 10);
  ASSERT_EQUAL(h[0].y, 20);
  ASSERT_EQUAL(h[0].z, -1);
  ASSERT_EQUAL(static_cast<T>(d[0]).x, 10);
  ASSERT_EQUAL(static_cast<T>(d[0]).y, 20);
  ASSERT_EQUAL(static_cast<T>(d[0]).z, -1);
};
DECLARE_UNITTEST(TestFillWithTrivialAssignment);

struct TypeWithNonTrivialAssigment
{
  int x, y, z;

  _CCCL_HOST_DEVICE TypeWithNonTrivialAssigment()
      : x(0)
      , y(0)
      , z(0)
  {}

  TypeWithNonTrivialAssigment(const TypeWithNonTrivialAssigment&) = default;

  _CCCL_HOST_DEVICE TypeWithNonTrivialAssigment& operator=(const TypeWithNonTrivialAssigment& t)
  {
    x = t.x;
    y = t.y;
    z = t.x + t.y;
    return *this;
  }

  _CCCL_HOST_DEVICE bool operator==(const TypeWithNonTrivialAssigment& t) const
  {
    return x == t.x && y == t.y && z == t.z;
  }
};

void TestFillWithNonTrivialAssignment()
{
  using T = TypeWithNonTrivialAssigment;

  thrust::host_vector<T> h(1);
  thrust::device_vector<T> d(1);

  ASSERT_EQUAL(h[0].x, 0);
  ASSERT_EQUAL(h[0].y, 0);
  ASSERT_EQUAL(h[0].z, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).x, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).y, 0);
  ASSERT_EQUAL(static_cast<T>(d[0]).z, 0);

  T val;
  val.x = 10;
  val.y = 20;
  val.z = -1;

  thrust::fill(h.begin(), h.end(), val);
  thrust::fill(d.begin(), d.end(), val);

  ASSERT_EQUAL(h[0].x, 10);
  ASSERT_EQUAL(h[0].y, 20);
  ASSERT_EQUAL(h[0].z, 30);
  ASSERT_EQUAL(static_cast<T>(d[0]).x, 10);
  ASSERT_EQUAL(static_cast<T>(d[0]).y, 20);
  ASSERT_EQUAL(static_cast<T>(d[0]).z, 30);
};
DECLARE_UNITTEST(TestFillWithNonTrivialAssignment);

template <typename ForwardIterator, typename T>
void fill(my_system& system, ForwardIterator /*first*/, ForwardIterator, const T&)
{
  system.validate_dispatch();
}

void TestFillDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::fill(sys, vec.begin(), vec.end(), 0);

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestFillDispatchExplicit);

template <typename ForwardIterator, typename T>
void fill(my_tag, ForwardIterator first, ForwardIterator, const T&)
{
  *first = 13;
}

void TestFillDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::fill(thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.end()), 0);

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestFillDispatchImplicit);

template <typename OutputIterator, typename Size, typename T>
OutputIterator fill_n(my_system& system, OutputIterator first, Size, const T&)
{
  system.validate_dispatch();
  return first;
}

void TestFillNDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::fill_n(sys, vec.begin(), vec.size(), 0);

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestFillNDispatchExplicit);

template <typename OutputIterator, typename Size, typename T>
OutputIterator fill_n(my_tag, OutputIterator first, Size, const T&)
{
  *first = 13;
  return first;
}

void TestFillNDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::fill_n(thrust::retag<my_tag>(vec.begin()), vec.size(), 0);

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestFillNDispatchImplicit);

_CCCL_DIAG_POP
