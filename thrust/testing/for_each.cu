#include "hip/hip_runtime.h"
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/retag.h>

#include <algorithm>

#include <unittest/unittest.h>

_CCCL_DIAG_PUSH
_CCCL_DIAG_SUPPRESS_MSVC(4244 4267) // possible loss of data

template <typename T>
class mark_present_for_each
{
public:
  T* ptr;
  _CCCL_HOST_DEVICE void operator()(T x)
  {
    ptr[(int) x] = 1;
  }
};

template <class Vector>
void TestForEachSimple()
{
  using T = typename Vector::value_type;

  Vector input{3, 2, 3, 4, 6};
  Vector output(7, (T) 0);

  mark_present_for_each<T> f;
  f.ptr = thrust::raw_pointer_cast(output.data());

  typename Vector::iterator result = thrust::for_each(input.begin(), input.end(), f);

  Vector ref{0, 0, 1, 1, 1, 0, 1};
  ASSERT_EQUAL(output, ref);
  ASSERT_EQUAL_QUIET(result, input.end());
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestForEachSimple);

template <typename InputIterator, typename Function>
InputIterator for_each(my_system& system, InputIterator first, InputIterator, Function)
{
  system.validate_dispatch();
  return first;
}

void TestForEachDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::for_each(sys, vec.begin(), vec.end(), 0);

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestForEachDispatchExplicit);

template <typename InputIterator, typename Function>
InputIterator for_each(my_tag, InputIterator first, InputIterator, Function)
{
  *first = 13;
  return first;
}

void TestForEachDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::for_each(thrust::retag<my_tag>(vec.begin()), thrust::retag<my_tag>(vec.end()), 0);

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestForEachDispatchImplicit);

template <class Vector>
void TestForEachNSimple()
{
  using T = typename Vector::value_type;

  Vector input{3, 2, 3, 4, 6};
  Vector output(7, (T) 0);

  mark_present_for_each<T> f;
  f.ptr = thrust::raw_pointer_cast(output.data());

  typename Vector::iterator result = thrust::for_each_n(input.begin(), input.size(), f);

  Vector ref{0, 0, 1, 1, 1, 0, 1};
  ASSERT_EQUAL(output, ref);
  ASSERT_EQUAL_QUIET(result, input.end());
}
DECLARE_INTEGRAL_VECTOR_UNITTEST(TestForEachNSimple);

template <typename InputIterator, typename Size, typename Function>
InputIterator for_each_n(my_system& system, InputIterator first, Size, Function)
{
  system.validate_dispatch();
  return first;
}

void TestForEachNDispatchExplicit()
{
  thrust::device_vector<int> vec(1);

  my_system sys(0);
  thrust::for_each_n(sys, vec.begin(), vec.size(), 0);

  ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestForEachNDispatchExplicit);

template <typename InputIterator, typename Size, typename Function>
InputIterator for_each_n(my_tag, InputIterator first, Size, Function)
{
  *first = 13;
  return first;
}

void TestForEachNDispatchImplicit()
{
  thrust::device_vector<int> vec(1);

  thrust::for_each_n(thrust::retag<my_tag>(vec.begin()), vec.size(), 0);

  ASSERT_EQUAL(13, vec.front());
}
DECLARE_UNITTEST(TestForEachNDispatchImplicit);

void TestForEachSimpleAnySystem()
{
  thrust::device_vector<int> output(7, 0);

  mark_present_for_each<int> f;
  f.ptr = thrust::raw_pointer_cast(output.data());

  thrust::counting_iterator<int> result =
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(5), f);

  thrust::device_vector<int> ref{1, 1, 1, 1, 1, 0, 0};
  ASSERT_EQUAL(output, ref);
  ASSERT_EQUAL_QUIET(result, thrust::make_counting_iterator(5));
}
DECLARE_UNITTEST(TestForEachSimpleAnySystem);

void TestForEachNSimpleAnySystem()
{
  thrust::device_vector<int> output(7, 0);

  mark_present_for_each<int> f;
  f.ptr = thrust::raw_pointer_cast(output.data());

  thrust::counting_iterator<int> result = thrust::for_each_n(thrust::make_counting_iterator(0), 5, f);

  thrust::device_vector<int> ref{1, 1, 1, 1, 1, 0, 0};
  ASSERT_EQUAL(output, ref);
  ASSERT_EQUAL_QUIET(result, thrust::make_counting_iterator(5));
}
DECLARE_UNITTEST(TestForEachNSimpleAnySystem);

template <typename T>
void TestForEach(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = ((size_t) h_input[i]) % output_size;
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  typename thrust::host_vector<T>::iterator h_result = thrust::for_each(h_input.begin(), h_input.end(), h_f);

  typename thrust::device_vector<T>::iterator d_result = thrust::for_each(d_input.begin(), d_input.end(), d_f);

  ASSERT_EQUAL(h_output, d_output);
  ASSERT_EQUAL_QUIET(h_result, h_input.end());
  ASSERT_EQUAL_QUIET(d_result, d_input.end());
}
DECLARE_VARIABLE_UNITTEST(TestForEach);

template <typename T>
void TestForEachN(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = ((size_t) h_input[i]) % output_size;
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  typename thrust::host_vector<T>::iterator h_result = thrust::for_each_n(h_input.begin(), h_input.size(), h_f);

  typename thrust::device_vector<T>::iterator d_result = thrust::for_each_n(d_input.begin(), d_input.size(), d_f);

  ASSERT_EQUAL(h_output, d_output);
  ASSERT_EQUAL_QUIET(h_result, h_input.end());
  ASSERT_EQUAL_QUIET(d_result, d_input.end());
}
DECLARE_VARIABLE_UNITTEST(TestForEachN);

template <typename T, unsigned int N>
struct SetFixedVectorToConstant
{
  FixedVector<T, N> exemplar;

  SetFixedVectorToConstant(T scalar)
      : exemplar(scalar)
  {}

  _CCCL_HOST_DEVICE void operator()(FixedVector<T, N>& t)
  {
    t = exemplar;
  }
};

template <typename T, unsigned int N>
void _TestForEachWithLargeTypes()
{
  size_t n = (64 * 1024) / sizeof(FixedVector<T, N>);

  thrust::host_vector<FixedVector<T, N>> h_data(n);

  for (size_t i = 0; i < h_data.size(); i++)
  {
    h_data[i] = FixedVector<T, N>(i);
  }

  thrust::device_vector<FixedVector<T, N>> d_data = h_data;

  SetFixedVectorToConstant<T, N> func(123);

  thrust::for_each(h_data.begin(), h_data.end(), func);
  thrust::for_each(d_data.begin(), d_data.end(), func);

  ASSERT_EQUAL_QUIET(h_data, d_data);
}

void TestForEachWithLargeTypes()
{
  _TestForEachWithLargeTypes<int, 1>();
  _TestForEachWithLargeTypes<int, 2>();
  _TestForEachWithLargeTypes<int, 4>();
  _TestForEachWithLargeTypes<int, 8>();
  _TestForEachWithLargeTypes<int, 16>();

  _TestForEachWithLargeTypes<int, 32>(); // fails on Linux 32 w/ gcc 4.1
  _TestForEachWithLargeTypes<int, 64>();
  _TestForEachWithLargeTypes<int, 128>();
  _TestForEachWithLargeTypes<int, 256>();
  _TestForEachWithLargeTypes<int, 512>();

  // XXX parallel_for doens't support large types
  //    _TestForEachWithLargeTypes<int, 1024>();  // fails on Vista 64 w/ VS2008
}
DECLARE_UNITTEST(TestForEachWithLargeTypes);

template <typename T, unsigned int N>
void _TestForEachNWithLargeTypes()
{
  size_t n = (64 * 1024) / sizeof(FixedVector<T, N>);

  thrust::host_vector<FixedVector<T, N>> h_data(n);

  for (size_t i = 0; i < h_data.size(); i++)
  {
    h_data[i] = FixedVector<T, N>(i);
  }

  thrust::device_vector<FixedVector<T, N>> d_data = h_data;

  SetFixedVectorToConstant<T, N> func(123);

  thrust::for_each_n(h_data.begin(), h_data.size(), func);
  thrust::for_each_n(d_data.begin(), d_data.size(), func);

  ASSERT_EQUAL_QUIET(h_data, d_data);
}

void TestForEachNWithLargeTypes()
{
  _TestForEachNWithLargeTypes<int, 1>();
  _TestForEachNWithLargeTypes<int, 2>();
  _TestForEachNWithLargeTypes<int, 4>();
  _TestForEachNWithLargeTypes<int, 8>();
  _TestForEachNWithLargeTypes<int, 16>();

  _TestForEachNWithLargeTypes<int, 32>(); // fails on Linux 32 w/ gcc 4.1
  _TestForEachNWithLargeTypes<int, 64>();
  _TestForEachNWithLargeTypes<int, 128>();
  _TestForEachNWithLargeTypes<int, 256>();
  _TestForEachNWithLargeTypes<int, 512>();

  // XXX parallel_for doens't support large types
  //    _TestForEachNWithLargeTypes<int, 1024>();  // fails on Vista 64 w/ VS2008
}
DECLARE_UNITTEST(TestForEachNWithLargeTypes);

_CCCL_DIAG_POP

struct only_set_when_expected
{
  unsigned long long expected;
  bool* flag;

  _CCCL_DEVICE void operator()(unsigned long long x)
  {
    if (x == expected)
    {
      *flag = true;
    }
  }
};

void TestForEachWithBigIndexesHelper(int magnitude)
{
  thrust::counting_iterator<unsigned long long> begin(0);
  thrust::counting_iterator<unsigned long long> end = begin + (1ull << magnitude);
  ASSERT_EQUAL(thrust::distance(begin, end), 1ll << magnitude);

  thrust::device_ptr<bool> has_executed = thrust::device_malloc<bool>(1);
  *has_executed                         = false;

  only_set_when_expected fn = {(1ull << magnitude) - 1, thrust::raw_pointer_cast(has_executed)};

  thrust::for_each(thrust::device, begin, end, fn);

  bool has_executed_h = *has_executed;
  thrust::device_free(has_executed);

  ASSERT_EQUAL(has_executed_h, true);
}

void TestForEachWithBigIndexes()
{
  TestForEachWithBigIndexesHelper(30);
  TestForEachWithBigIndexesHelper(31);
  TestForEachWithBigIndexesHelper(32);
  TestForEachWithBigIndexesHelper(33);
}
DECLARE_UNITTEST(TestForEachWithBigIndexes);
