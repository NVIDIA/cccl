#include "hip/hip_runtime.h"
#include <unittest/unittest.h>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>


THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN


template <typename Iterator1, typename Iterator2>
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
__global__
#endif 
void simple_copy_on_device(Iterator1 first1, Iterator1 last1, Iterator2 first2)
{
    while(first1 != last1)
        *(first2++) = *(first1++);
}

template <typename Iterator1, typename Iterator2>
void simple_copy(Iterator1 first1, Iterator1 last1, Iterator2 first2)
{
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    simple_copy_on_device<<<1,1>>>(first1, last1, first2);
#else
    simple_copy_on_device(first1, last1, first2);
#endif
}


void TestDeviceDereferenceDeviceVectorIterator(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100); 
    thrust::device_vector<int> output(input.size(), 0);

    simple_copy(input.begin(), input.end(), output.begin());

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceDeviceVectorIterator);

void TestDeviceDereferenceDevicePtr(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100); 
    thrust::device_vector<int> output(input.size(), 0);

    thrust::device_ptr<int> _first1 = &input[0];
    thrust::device_ptr<int> _last1  = _first1 + input.size();
    thrust::device_ptr<int> _first2 = &output[0];

    simple_copy(_first1, _last1, _first2);

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceDevicePtr);

void TestDeviceDereferenceTransformIterator(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100); 
    thrust::device_vector<int> output(input.size(), 0);

    simple_copy(thrust::make_transform_iterator(input.begin(), thrust::identity<int>()),
                thrust::make_transform_iterator(input.end (),  thrust::identity<int>()),
                output.begin());

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceTransformIterator);

void TestDeviceDereferenceCountingIterator(void)
{
    thrust::counting_iterator<int> first(1);
    thrust::counting_iterator<int> last(6);

    thrust::device_vector<int> output(5);

    simple_copy(first, last, output.begin());

    ASSERT_EQUAL(output[0], 1);
    ASSERT_EQUAL(output[1], 2);
    ASSERT_EQUAL(output[2], 3);
    ASSERT_EQUAL(output[3], 4);
    ASSERT_EQUAL(output[4], 5);
}
DECLARE_UNITTEST(TestDeviceDereferenceCountingIterator);

void TestDeviceDereferenceTransformedCountingIterator(void)
{
    thrust::counting_iterator<int> first(1);
    thrust::counting_iterator<int> last(6);

    thrust::device_vector<int> output(5);

    simple_copy(thrust::make_transform_iterator(first, thrust::negate<int>()),
                thrust::make_transform_iterator(last,  thrust::negate<int>()),
                output.begin());

    ASSERT_EQUAL(output[0], -1);
    ASSERT_EQUAL(output[1], -2);
    ASSERT_EQUAL(output[2], -3);
    ASSERT_EQUAL(output[3], -4);
    ASSERT_EQUAL(output[4], -5);
}
DECLARE_UNITTEST(TestDeviceDereferenceTransformedCountingIterator);

THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
