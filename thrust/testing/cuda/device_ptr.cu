#include <thrust/device_vector.h>

#include <unittest/unittest.h>

#if _CCCL_HAS_CTK()
void TestConversionToCudeviceptr()
{
  thrust::device_vector<int> vec(3);
  int* p = thrust::raw_pointer_cast(vec.data());

  hipDeviceptr_t cdevptr = static_cast<hipDeviceptr_t>(vec.data());
  ASSERT_EQUAL(cdevptr, reinterpret_cast<hipDeviceptr_t>(p));
}
DECLARE_UNITTEST(TestConversionToCudeviceptr);
#endif
