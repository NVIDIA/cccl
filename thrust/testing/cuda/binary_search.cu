#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>

#include <unittest/unittest.h>

void TestEqualRangeOnStream()
{ // Regression test for GH issue #921 (nvbug 2173437)
  using vector_t   = typename thrust::device_vector<int>;
  using iterator_t = typename vector_t::iterator;
  using result_t   = thrust::pair<iterator_t, iterator_t>;

  vector_t input(10);
  thrust::sequence(thrust::device, input.begin(), input.end(), 0);
  hipStream_t stream = 0;
  result_t result     = thrust::equal_range(thrust::cuda::par.on(stream), input.begin(), input.end(), 5);

  ASSERT_EQUAL(5, thrust::distance(input.begin(), result.first));
  ASSERT_EQUAL(6, thrust::distance(input.begin(), result.second));
}
DECLARE_UNITTEST(TestEqualRangeOnStream);
