#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator>
__global__ void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::sequence(exec, first, last);
}

template <typename ExecutionPolicy, typename Iterator, typename T>
__global__ void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init)
{
  thrust::sequence(exec, first, last, init);
}

template <typename ExecutionPolicy, typename Iterator, typename T>
__global__ void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, T step)
{
  thrust::sequence(exec, first, last, init, step);
}

template <typename ExecutionPolicy>
void TestSequenceDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> v(5);

  sequence_kernel<<<1, 1>>>(exec, v.begin(), v.end());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  thrust::device_vector<int> ref{0, 1, 2, 3, 4};
  ASSERT_EQUAL(v, ref);

  sequence_kernel<<<1, 1>>>(exec, v.begin(), v.end(), 10);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ref = {10, 11, 12, 13, 14};
  ASSERT_EQUAL(v, ref);

  sequence_kernel<<<1, 1>>>(exec, v.begin(), v.end(), 10, 2);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ref = {10, 12, 14, 16, 18};
  ASSERT_EQUAL(v, ref);
}

void TestSequenceDeviceSeq()
{
  TestSequenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSequenceDeviceSeq);

void TestSequenceDeviceDevice()
{
  TestSequenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSequenceDeviceDevice);
#endif

void TestSequenceCudaStreams()
{
  using Vector = thrust::device_vector<int>;

  Vector v(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end());
  hipStreamSynchronize(s);

  Vector ref{0, 1, 2, 3, 4};
  ASSERT_EQUAL(v, ref);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10);
  hipStreamSynchronize(s);

  ref = {10, 11, 12, 13, 14};
  ASSERT_EQUAL(v, ref);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10, 2);
  hipStreamSynchronize(s);

  ref = {10, 12, 14, 16, 18};
  ASSERT_EQUAL(v, ref);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSequenceCudaStreams);
