#include "hip/hip_runtime.h"
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__ void set_intersection_kernel(
  ExecutionPolicy exec,
  Iterator1 first1,
  Iterator1 last1,
  Iterator2 first2,
  Iterator2 last2,
  Iterator3 result1,
  Iterator4 result2)
{
  *result2 = thrust::set_intersection(exec, first1, last1, first2, last2, result1);
}

template <typename ExecutionPolicy>
void TestSetIntersectionDevice(ExecutionPolicy exec)
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = Vector::iterator;

  Vector a{0, 2, 4}, b{0, 3, 3, 4};

  Vector ref{0, 4};
  Vector result(2);
  thrust::device_vector<Iterator> end_vec(1);

  set_intersection_kernel<<<1, 1>>>(exec, a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec.front();

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}

void TestSetIntersectionDeviceSeq()
{
  TestSetIntersectionDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceSeq);

void TestSetIntersectionDeviceDevice()
{
  TestSetIntersectionDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceDevice);

void TestSetIntersectionDeviceNoSync()
{
  TestSetIntersectionDevice(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestSetIntersectionDeviceNoSync);
#endif

template <typename ExecutionPolicy>
void TestSetIntersectionCudaStreams(ExecutionPolicy policy)
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = Vector::iterator;

  Vector a{0, 2, 4}, b{0, 3, 3, 4};

  Vector ref{0, 4};
  Vector result(2);

  hipStream_t s;
  hipStreamCreate(&s);

  auto streampolicy = policy.on(s);

  Iterator end = thrust::set_intersection(streampolicy, a.begin(), a.end(), b.begin(), b.end(), result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}

void TestSetIntersectionCudaStreamsSync()
{
  TestSetIntersectionCudaStreams(thrust::cuda::par);
}
DECLARE_UNITTEST(TestSetIntersectionCudaStreamsSync);

void TestSetIntersectionCudaStreamsNoSync()
{
  TestSetIntersectionCudaStreams(thrust::cuda::par_nosync);
}
DECLARE_UNITTEST(TestSetIntersectionCudaStreamsNoSync);
