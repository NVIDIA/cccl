#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/set_operations.h>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__ void set_symmetric_difference_kernel(
  ExecutionPolicy exec,
  Iterator1 first1,
  Iterator1 last1,
  Iterator2 first2,
  Iterator2 last2,
  Iterator3 result1,
  Iterator4 result2)
{
  *result2 = thrust::set_symmetric_difference(exec, first1, last1, first2, last2, result1);
}

template <typename ExecutionPolicy>
void TestSetSymmetricDifferenceDevice(ExecutionPolicy exec)
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = typename Vector::iterator;

  Vector a{0, 2, 4, 6}, b{0, 3, 3, 4, 7};

  Vector ref{2, 3, 3, 6, 7};
  Vector result(5);
  thrust::device_vector<Iterator> end_vec(1);

  set_symmetric_difference_kernel<<<1, 1>>>(
    exec, a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec[0];

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}

void TestSetSymmetricDifferenceDeviceSeq()
{
  TestSetSymmetricDifferenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceDeviceSeq);

void TestSetSymmetricDifferenceDeviceDevice()
{
  TestSetSymmetricDifferenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceDeviceDevice);
#endif

void TestSetSymmetricDifferenceCudaStreams()
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = Vector::iterator;

  Vector a{0, 2, 4, 6}, b{0, 3, 3, 4, 7};

  Vector ref{2, 3, 3, 6, 7};
  Vector result(5);

  hipStream_t s;
  hipStreamCreate(&s);

  Iterator end =
    thrust::set_symmetric_difference(thrust::cuda::par.on(s), a.begin(), a.end(), b.begin(), b.end(), result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSetSymmetricDifferenceCudaStreams);
