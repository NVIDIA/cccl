#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>

#include <algorithm>

#include "thrust/device_vector.h"
#include <unittest/unittest.h>

static const size_t NUM_REGISTERS = 64;

template <size_t N>
_CCCL_HOST_DEVICE void f(int* x)
{
  int temp = *x;
  f<N - 1>(x + 1);
  *x = temp;
};
template <>
_CCCL_HOST_DEVICE void f<0>(int* /*x*/)
{}
template <size_t N>
struct CopyFunctorWithManyRegisters
{
  _CCCL_HOST_DEVICE void operator()(int* ptr)
  {
    f<N>(ptr);
  }
};

void TestForEachLargeRegisterFootprint()
{
  int current_device = -1;
  hipGetDevice(&current_device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, current_device);

  thrust::device_vector<int> data(NUM_REGISTERS, 12345);

  thrust::device_vector<int*> input(1, thrust::raw_pointer_cast(&data[0])); // length is irrelevant

  thrust::for_each(input.begin(), input.end(), CopyFunctorWithManyRegisters<NUM_REGISTERS>());
}
DECLARE_UNITTEST(TestForEachLargeRegisterFootprint);

void TestForEachNLargeRegisterFootprint()
{
  int current_device = -1;
  hipGetDevice(&current_device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, current_device);

  thrust::device_vector<int> data(NUM_REGISTERS, 12345);

  thrust::device_vector<int*> input(1, thrust::raw_pointer_cast(&data[0])); // length is irrelevant

  thrust::for_each_n(input.begin(), input.size(), CopyFunctorWithManyRegisters<NUM_REGISTERS>());
}
DECLARE_UNITTEST(TestForEachNLargeRegisterFootprint);

template <typename T>
struct mark_present_for_each
{
  T* ptr;
  _CCCL_HOST_DEVICE void operator()(T x)
  {
    ptr[(int) x] = 1;
  }
};

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator, typename Function>
__global__ void for_each_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::for_each(exec, first, last, f);
}

template <typename T>
void TestForEachDeviceSeq(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = ((size_t) h_input[i]) % output_size;
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  thrust::for_each(h_input.begin(), h_input.end(), h_f);

  for_each_kernel<<<1, 1>>>(thrust::seq, d_input.begin(), d_input.end(), d_f);
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachDeviceSeq);

template <typename T>
void TestForEachDeviceDevice(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = ((size_t) h_input[i]) % output_size;
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  thrust::for_each(h_input.begin(), h_input.end(), h_f);

  for_each_kernel<<<1, 1>>>(thrust::device, d_input.begin(), d_input.end(), d_f);
  {
    hipError_t const err = hipGetLastError();
    ASSERT_EQUAL(hipSuccess, err);
  }
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachDeviceDevice);

template <typename ExecutionPolicy, typename Iterator, typename Size, typename Function>
__global__ void for_each_n_kernel(ExecutionPolicy exec, Iterator first, Size n, Function f)
{
  thrust::for_each_n(exec, first, n, f);
}

template <typename T>
void TestForEachNDeviceSeq(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = static_cast<T>(((size_t) h_input[i]) % output_size);
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  thrust::for_each_n(h_input.begin(), h_input.size(), h_f);

  for_each_n_kernel<<<1, 1>>>(thrust::seq, d_input.begin(), d_input.size(), d_f);
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachNDeviceSeq);

template <typename T>
void TestForEachNDeviceDevice(const size_t n)
{
  const size_t output_size = std::min((size_t) 10, 2 * n);

  thrust::host_vector<T> h_input = unittest::random_integers<T>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_input[i] = static_cast<T>(((size_t) h_input[i]) % output_size);
  }

  thrust::device_vector<T> d_input = h_input;

  thrust::host_vector<T> h_output(output_size, (T) 0);
  thrust::device_vector<T> d_output(output_size, (T) 0);

  mark_present_for_each<T> h_f;
  mark_present_for_each<T> d_f;
  h_f.ptr = &h_output[0];
  d_f.ptr = (&d_output[0]).get();

  thrust::for_each_n(h_input.begin(), h_input.size(), h_f);

  for_each_n_kernel<<<1, 1>>>(thrust::device, d_input.begin(), d_input.size(), d_f);
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(h_output, d_output);
}
DECLARE_VARIABLE_UNITTEST(TestForEachNDeviceDevice);
#endif

void TestForEachCudaStreams()
{
  hipStream_t s;
  hipStreamCreate(&s);

  thrust::device_vector<int> input{3, 2, 3, 4, 6};
  thrust::device_vector<int> output(7, 0);

  mark_present_for_each<int> f;
  f.ptr = thrust::raw_pointer_cast(output.data());

  thrust::for_each(thrust::cuda::par.on(s), input.begin(), input.end(), f);

  hipStreamSynchronize(s);

  thrust::device_vector<int> ref{0, 0, 1, 1, 1, 0, 1};
  ASSERT_EQUAL(output, ref);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestForEachCudaStreams);
