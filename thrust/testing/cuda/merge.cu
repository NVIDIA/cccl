#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/merge.h>
#include <thrust/sort.h>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__ void merge_kernel(
  ExecutionPolicy exec,
  Iterator1 first1,
  Iterator1 last1,
  Iterator2 first2,
  Iterator2 last2,
  Iterator3 result1,
  Iterator4 result2)
{
  *result2 = thrust::merge(exec, first1, last1, first2, last2, result1);
}

template <typename ExecutionPolicy>
void TestMergeDevice(ExecutionPolicy exec)
{
  const size_t n         = 10000;
  const size_t sizes[]   = {0, 1, n / 2, n, n + 1, 2 * n};
  const size_t num_sizes = sizeof(sizes) / sizeof(size_t);

  const thrust::host_vector<int> random =
    unittest::random_integers<unittest::int8_t>(n + *thrust::max_element(sizes, sizes + num_sizes));

  thrust::host_vector<int> h_a(random.begin(), random.begin() + n);
  thrust::host_vector<int> h_b(random.begin() + n, random.end());

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());

  const thrust::device_vector<int> d_a = h_a;
  const thrust::device_vector<int> d_b = h_b;

  for (size_t i = 0; i < num_sizes; i++)
  {
    const size_t size = sizes[i];

    thrust::host_vector<int> h_result(n + size);
    thrust::device_vector<int> d_result(n + size);

    using iter_type = typename thrust::device_vector<int>::iterator;
    thrust::device_vector<iter_type> d_end(1);

    const auto h_end = thrust::merge(h_a.begin(), h_a.end(), h_b.begin(), h_b.begin() + size, h_result.begin());
    h_result.resize(h_end - h_result.begin());

    merge_kernel<<<1, 1>>>(
      exec, d_a.begin(), d_a.end(), d_b.begin(), d_b.begin() + size, d_result.begin(), d_end.begin());
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);

    d_result.resize(static_cast<iter_type>(d_end[0]) - d_result.begin());

    ASSERT_EQUAL(h_result, d_result);
  }
}

void TestMergeDeviceSeq()
{
  TestMergeDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMergeDeviceSeq);

void TestMergeDeviceDevice()
{
  TestMergeDevice(thrust::device);
}
DECLARE_UNITTEST(TestMergeDeviceDevice);
#endif

void TestMergeCudaStreams()
{
  using Vector = thrust::device_vector<int>;
  const Vector a{0, 2, 4}, b{0, 3, 3, 4};
  const Vector ref{0, 0, 2, 3, 3, 4, 4};

  Vector result(7);

  hipStream_t s;
  hipStreamCreate(&s);

  const auto end = thrust::merge(thrust::cuda::par.on(s), a.begin(), a.end(), b.begin(), b.end(), result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMergeCudaStreams);
