#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/uninitialized_copy.h>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__ void uninitialized_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::uninitialized_copy(exec, first, last, result);
}

template <typename ExecutionPolicy>
void TestUninitializedCopyDevice(ExecutionPolicy exec)
{
  using Vector = thrust::device_vector<int>;

  Vector v1{0, 1, 2, 3, 4};

  // copy to Vector
  Vector v2(5);
  uninitialized_copy_kernel<<<1, 1>>>(exec, v1.begin(), v1.end(), v2.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Vector ref{0, 1, 2, 3, 4};
  ASSERT_EQUAL(v2, ref);
}

void TestUninitializedCopyDeviceSeq()
{
  TestUninitializedCopyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedCopyDeviceSeq);

void TestUninitializedCopyDeviceDevice()
{
  TestUninitializedCopyDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedCopyDeviceDevice);
#endif

void TestUninitializedCopyCudaStreams()
{
  using Vector = thrust::device_vector<int>;

  Vector v1{0, 1, 2, 3, 4};

  // copy to Vector
  Vector v2(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::uninitialized_copy(thrust::cuda::par.on(s), v1.begin(), v1.end(), v2.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v2, v1);
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedCopyCudaStreams);

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Size, typename Iterator2>
__global__ void uninitialized_copy_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, Iterator2 result)
{
  thrust::uninitialized_copy_n(exec, first, n, result);
}

template <typename ExecutionPolicy>
void TestUninitializedCopyNDevice(ExecutionPolicy exec)
{
  using Vector = thrust::device_vector<int>;

  Vector v1{0, 1, 2, 3, 4};

  // copy to Vector
  Vector v2(5);
  uninitialized_copy_n_kernel<<<1, 1>>>(exec, v1.begin(), v1.size(), v2.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(v2, v1);
}

void TestUninitializedCopyNDeviceSeq()
{
  TestUninitializedCopyNDevice(thrust::seq);
}
DECLARE_UNITTEST(TestUninitializedCopyNDeviceSeq);

void TestUninitializedCopyNDeviceDevice()
{
  TestUninitializedCopyNDevice(thrust::device);
}
DECLARE_UNITTEST(TestUninitializedCopyNDeviceDevice);
#endif

void TestUninitializedCopyNCudaStreams()
{
  using Vector = thrust::device_vector<int>;

  Vector v1{0, 1, 2, 3, 4};

  // copy to Vector
  Vector v2(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::uninitialized_copy_n(thrust::cuda::par.on(s), v1.begin(), v1.size(), v2.begin());
  hipStreamSynchronize(s);
  ASSERT_EQUAL(v2, v1);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestUninitializedCopyNCudaStreams);
