#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/gather.h>

#include <algorithm>

#include <unittest/unittest.h>

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3>
__global__ void
gather_kernel(ExecutionPolicy exec, Iterator1 map_first, Iterator1 map_last, Iterator2 elements_first, Iterator3 result)
{
  thrust::gather(exec, map_first, map_last, elements_first, result);
}

template <typename T, typename ExecutionPolicy>
void TestGatherDevice(ExecutionPolicy exec, const size_t n)
{
  const size_t source_size = std::min((size_t) 10, 2 * n);

  // source vectors to gather from
  thrust::host_vector<T> h_source   = unittest::random_samples<T>(source_size);
  thrust::device_vector<T> d_source = h_source;

  // gather indices
  thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_map[i] = h_map[i] % source_size;
  }

  thrust::device_vector<unsigned int> d_map = h_map;

  // gather destination
  thrust::host_vector<T> h_output(n);
  thrust::device_vector<T> d_output(n);

  thrust::gather(h_map.begin(), h_map.end(), h_source.begin(), h_output.begin());

  gather_kernel<<<1, 1>>>(exec, d_map.begin(), d_map.end(), d_source.begin(), d_output.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_output, d_output);
}

template <typename T>
void TestGatherDeviceSeq(const size_t n)
{
  TestGatherDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherDeviceSeq);

template <typename T>
void TestGatherDeviceDevice(const size_t n)
{
  TestGatherDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherDeviceDevice);
#endif

void TestGatherCudaStreams()
{
  thrust::device_vector<int> map = {6, 2, 1, 7, 2}; // gather indices
  thrust::device_vector<int> src = {0, 1, 2, 3, 4, 5, 6, 7}; // source vector
  thrust::device_vector<int> dst = {0, 0, 0, 0, 0}; // destination vector

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::gather(thrust::cuda::par.on(s), map.begin(), map.end(), src.begin(), dst.begin());
  hipStreamSynchronize(s);

  thrust::device_vector<int> ref = {6, 2, 1, 7, 2}; // destination vector

  ASSERT_EQUAL(dst, ref);
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGatherCudaStreams);

#ifdef THRUST_TEST_DEVICE_SIDE
template <typename ExecutionPolicy,
          typename Iterator1,
          typename Iterator2,
          typename Iterator3,
          typename Iterator4,
          typename Predicate>
__global__ void gather_if_kernel(
  ExecutionPolicy exec,
  Iterator1 map_first,
  Iterator1 map_last,
  Iterator2 stencil_first,
  Iterator3 elements_first,
  Iterator4 result,
  Predicate pred)
{
  thrust::gather_if(exec, map_first, map_last, stencil_first, elements_first, result, pred);
}

template <typename T>
struct is_even_gather_if
{
  _CCCL_HOST_DEVICE bool operator()(const T i) const
  {
    return (i % 2) == 0;
  }
};

template <typename T, typename ExecutionPolicy>
void TestGatherIfDevice(ExecutionPolicy exec, const size_t n)
{
  const size_t source_size = std::min((size_t) 10, 2 * n);

  // source vectors to gather from
  thrust::host_vector<T> h_source   = unittest::random_samples<T>(source_size);
  thrust::device_vector<T> d_source = h_source;

  // gather indices
  thrust::host_vector<unsigned int> h_map = unittest::random_integers<unsigned int>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_map[i] = h_map[i] % source_size;
  }

  thrust::device_vector<unsigned int> d_map = h_map;

  // gather stencil
  thrust::host_vector<unsigned int> h_stencil = unittest::random_integers<unsigned int>(n);

  for (size_t i = 0; i < n; i++)
  {
    h_stencil[i] = h_stencil[i] % 2;
  }

  thrust::device_vector<unsigned int> d_stencil = h_stencil;

  // gather destination
  thrust::host_vector<T> h_output(n);
  thrust::device_vector<T> d_output(n);

  thrust::gather_if(
    h_map.begin(),
    h_map.end(),
    h_stencil.begin(),
    h_source.begin(),
    h_output.begin(),
    is_even_gather_if<unsigned int>());

  gather_if_kernel<<<1, 1>>>(
    exec,
    d_map.begin(),
    d_map.end(),
    d_stencil.begin(),
    d_source.begin(),
    d_output.begin(),
    is_even_gather_if<unsigned int>());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }

  ASSERT_EQUAL(h_output, d_output);
}

template <typename T>
void TestGatherIfDeviceSeq(const size_t n)
{
  TestGatherIfDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherIfDeviceSeq);

template <typename T>
void TestGatherIfDeviceDevice(const size_t n)
{
  TestGatherIfDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGatherIfDeviceDevice);
#endif

void TestGatherIfCudaStreams()
{
  thrust::device_vector<int> flg{0, 1, 0, 1, 0}; // predicate array
  thrust::device_vector<int> map{6, 2, 1, 7, 2}; // gather indices
  thrust::device_vector<int> src{0, 1, 2, 3, 4, 5, 6, 7}; // source vector
  thrust::device_vector<int> dst(5, 0); // destination vector

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::gather_if(thrust::cuda::par.on(s), map.begin(), map.end(), flg.begin(), src.begin(), dst.begin());
  hipStreamSynchronize(s);

  thrust::device_vector<int> ref{0, 2, 0, 7, 0}; // destination vector

  ASSERT_EQUAL(dst, ref);
  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGatherIfCudaStreams);
