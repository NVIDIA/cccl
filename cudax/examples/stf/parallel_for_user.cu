#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief An AXPY kernel implemented with a task of the CUDA stream backend
 * where the task accesses managed memory from the device. This tests
 * explicitly created managed memory, and passes it to a logical data.
 */

#include <cuda/experimental/stf.cuh>

#include <iostream>

using namespace cuda::experimental::stf;

double X0(size_t i)
{
  return sin((double) i);
}

double Y0(size_t i)
{
  return cos((double) i);
}

int main()
{
  context ctx = graph_ctx();
  const size_t N = 16;

  double *X, *Y, *Z;
  cuda_safe_call(hipMallocManaged(&X, N * sizeof(double)));
  cuda_safe_call(hipMallocManaged(&Y, N * sizeof(double)));
  cuda_safe_call(hipMallocManaged(&Z, N * sizeof(double)));

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
    Z[i] = Y0(i);
  }

  double alpha = 3.14;
  double beta = 1664.0;

  ctx.parallel_for(box(N))->*[alpha, X, Y]__device__(size_t i){
      Y[i] += alpha*X[i];
  };

  ctx.parallel_for(box(N))->*[beta, X, Z]__device__(size_t i){
      Z[i] += beta*X[i];
  };

  ctx.task_fence();

  ctx.parallel_for(box(N))->*[Y, Z]__device__(size_t i){
      Z[i] += Y[i];
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
//    assert(fabs(Y[i] - (Y0(i) + alpha * X0(i))) < 0.0001);
//    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
