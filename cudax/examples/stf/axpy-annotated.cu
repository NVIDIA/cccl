#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief This example illustrates how we can annotate tasks and logical data with debugging symbol
 *
 * CUDASTF_DOT_FILE=axpy.dot build/examples/axpy-annotated
 *
 * # Generate the visualization from this dot file in PDF or PNG format
 * dot -Tpdf axpy.dot -o axpy.pdf
 * dot -Tpng axpy.dot -o axpy.png
 *
 * # Generate visualization with events (for advanced users)
 * CUDASTF_DOT_IGNORE_PREREQS=0 CUDASTF_DOT_FILE=axpy-with-events.dot build/examples/axpy-annotated
 * dot -Tpng axpy-with-events.dot -o axpy-with-events.png
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

__global__ void axpy(double a, slice<const double> x, slice<double> y)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int i = tid; i < x.size(); i += nthreads)
  {
    y(i) += a * x(i);
  }
}

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

int main()
{
  context ctx;
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  double alpha = 3.14;

  auto lX = ctx.logical_data(X).set_symbol("X");
  auto lY = ctx.logical_data(Y).set_symbol("Y");

  /* Compute Y = Y + alpha X */
  ctx.task(lX.read(), lY.rw()).set_symbol("axpy")->*[&](hipStream_t s, auto dX, auto dY) {
    axpy<<<16, 128, 0, s>>>(alpha, dX, dY);
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    assert(fabs(Y[i] - (Y0(i) + alpha * X0(i))) < 0.0001);
    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
