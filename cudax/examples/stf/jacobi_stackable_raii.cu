#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Jacobi method with a while scope guard and explicit management of the conditional handle
 *
 */

#include <cuda/experimental/stf.cuh>

#include <iostream>

#include "cuda/experimental/__stf/utility/stackable_ctx.cuh"

using namespace cuda::experimental::stf;

int main([[maybe_unused]] int argc, [[maybe_unused]] char** argv)
{
#if _CCCL_CTK_BELOW(12, 4)
  fprintf(stderr, "Waiving test: conditional nodes are only available since CUDA 12.4.\n");
  return 0;
#else
  stackable_ctx ctx;

  size_t n   = 4096;
  size_t m   = 4096;
  double tol = 0.1;

  if (argc > 2)
  {
    n = atol(argv[1]);
    m = atol(argv[2]);
  }

  if (argc > 3)
  {
    tol = atof(argv[3]);
  }

  auto lA    = ctx.logical_data(shape_of<slice<double, 2>>(m, n));
  auto lAnew = ctx.logical_data(lA.shape());

  ctx.parallel_for(lA.shape(), lA.write(), lAnew.write()).set_symbol("init")->*
    [=] __device__(size_t i, size_t j, auto A, auto Anew) {
      A(i, j) = (i == j) ? 1.0 : -1.0;
    };

  hipEvent_t start, stop;

  cuda_safe_call(hipEventCreate(&start));
  cuda_safe_call(hipEventCreate(&stop));

  cuda_safe_call(hipEventRecord(start, ctx.fence()));

  size_t iter = 0;

  auto lresidual = ctx.logical_data(shape_of<scalar_view<double>>());

  {
    auto while_guard = ctx.while_graph_scope();

    ctx.parallel_for(inner<1>(lA.shape()), lA.read(), lAnew.write(), lresidual.reduce(reducer::maxval<double>{}))
        ->*[] __device__(size_t i, size_t j, auto A, auto Anew, auto& residual) {
              Anew(i, j)   = 0.25 * (A(i - 1, j) + A(i + 1, j) + A(i, j - 1) + A(i, j + 1));
              double error = fabs(A(i, j) - Anew(i, j));
              residual     = error;
            };

    ctx.parallel_for(inner<1>(lA.shape()), lA.rw(), lAnew.read())->*[] __device__(size_t i, size_t j, auto A, auto Anew) {
      A(i, j) = Anew(i, j);
    };

    auto handle = while_guard.cond_handle();
    ctx.parallel_for(box(1), lresidual.read())->*[handle, tol] __device__(size_t, auto residual) {
      bool converged = (*residual < tol);
      cudaGraphSetConditional(handle, !converged);
    };
  }

  // Store final residual for verification
  double final_residual = ctx.wait(lresidual);

  fprintf(stderr, "ITER %zu: converged residual %e\n", iter++, final_residual);

  cuda_safe_call(hipEventRecord(stop, ctx.fence()));

  ctx.finalize();

  EXPECT(final_residual <= tol); // Algorithm should have converged within tolerance

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Elapsed time: %f ms\n", elapsedTime);
#endif
}
