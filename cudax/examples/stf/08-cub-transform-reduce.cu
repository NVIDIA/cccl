#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Example of reduction implementing using CUB
 */

#include <hipcub/hipcub.hpp>

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

/**
 * This functor transforms a 1D index into the result of the transformation
 */
// Args... is for example slice<double>, slice<int>...
template <typename TransformOp, typename shape_t, typename... Args>
struct IndexToTransformedValue
{
  IndexToTransformedValue(TransformOp _op, shape_t s, ::std::tuple<Args...> _targs)
      : op(mv(_op))
      , shape(mv(s))
      , targs(mv(_targs))
  {}

  __host__ __device__ __forceinline__ auto operator()(const size_t& index) const
  {
    const auto explode_args = [&](auto&&... data) {
      CUDASTF_NO_DEVICE_STACK
      auto const explode_coords = [&](auto&&... coords) {
        return op(coords..., data...);
      };
      return ::std::apply(explode_coords, shape.index_to_coords(index));
    };
    return ::std::apply(explode_args, targs);
  }

  TransformOp op;
  const shape_t shape;
  ::std::tuple<Args...> targs;
};

/**
 * @brief Helper to transform a device lambda into a functor that we can use in CUB
 */
template <typename BinaryOp>
struct ReduceOpWrapper
{
  ReduceOpWrapper(BinaryOp _op)
      : op(mv(_op)) {};

  template <typename T>
  __device__ __forceinline__ T operator()(const T& a, const T& b) const
  {
    return op(a, b);
  }

  BinaryOp op;
};

// This should print the output of the transform op (which is an int)
template <typename It>
__global__ void TEST_KERNEL(It it)
{
  printf("it(%d) = %d\n", threadIdx.x, it[threadIdx.x]);
}

/**
 * @brief Remove the first entry of a std::tuple
 */
template <typename Tuple>
auto remove_first(const Tuple& t)
{
  return ::std::apply(
    [](auto&& head, auto&&... tail) {
      return ::std::make_tuple(::std::forward<decltype(tail)>(tail)...);
    },
    t);
}

template <typename Ctx, typename shape_t, typename TransformOp, typename BinaryOp, typename OutT, typename... Args>
auto stf_transform_reduce(
  Ctx& ctx, shape_t s, TransformOp&& transform_op, BinaryOp&& op, OutT init_val, logical_data<Args>... args)
{
  using ConvertionOp_t = IndexToTransformedValue<TransformOp, shape_t, Args...>;

  // The result of this operation is a logical data
  auto result = ctx.logical_data(shape_of<scalar<OutT>>());

  auto t = ctx.task(result.write(), args.read()...);
  t.start();
  hipStream_t stream = t.get_stream();

  auto deps = t.typed_deps();
  // We remove the first argument
  ConvertionOp_t conversion_op(transform_op, s, remove_first(deps));

  size_t num_elements = s.size();
  hipcub::CountingInputIterator<size_t> count_it(0);

  // Create an iterator wrapper
  hipcub::TransformInputIterator<OutT, ConvertionOp_t, decltype(count_it)> itr(count_it, conversion_op);

  // Ensure that the
  TEST_KERNEL<<<1, 8, 0, stream>>>(itr);

  // Determine temporary device storage requirements
  void* d_temp_storage      = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Reduce(
    d_temp_storage,
    temp_storage_bytes,
    itr, //*static_cast<decltype(itr) *>(nullptr), // TODO
    (OutT*) nullptr,
    num_elements,
    ReduceOpWrapper<BinaryOp>(op),
    init_val,
    0);

  cuda_safe_call(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));

  hipcub::DeviceReduce::Reduce(
    d_temp_storage,
    temp_storage_bytes,
    itr, // TODO
    (OutT*) ::std::get<0>(deps).addr,
    num_elements,
    ReduceOpWrapper<BinaryOp>(op),
    init_val,
    0);

  cuda_safe_call(hipFreeAsync(d_temp_storage, stream));

  t.end();

  return result;
}

template <typename Ctx>
void run()
{
  Ctx ctx;

  const size_t N = 1024 * 16;

  int ref_prod = 0;

  int* X = new int[N];
  int* Y = new int[N];

  for (int ind = 0; ind < N; ind++)
  {
    X[ind] = 2 + ind; // rand() % N;
    Y[ind] = 3 + ind; // rand() % N;
    ref_prod += X[ind] * Y[ind];
  }

  auto lX = ctx.logical_data(X, {N});
  auto lY = ctx.logical_data(Y, {N});

  auto lresult = stf_transform_reduce(
    ctx,
    lX.shape(),
    [] __device__(size_t i, auto x, auto y) {
      return x(i) * y(i);
    },
    [] __device__(const int& a, const int& b) {
      return a + b;
    },
    0,
    lX,
    lY);

  int result = ctx.wait(lresult);
  _CCCL_ASSERT(result == ref_prod, "Incorrect result");

  ctx.finalize();
}

int main()
{
  run<stream_ctx>();
  // run<graph_ctx>();
}
