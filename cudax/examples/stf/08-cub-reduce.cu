//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Example of reduction implementing using CUB
 */

#include <hipcub/hipcub.hpp>

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

template <typename BinaryOp>
struct OpWrapper
{
  OpWrapper(BinaryOp _op)
      : op(mv(_op)) {};

  template <typename T>
  __device__ __forceinline__ T operator()(const T& a, const T& b) const
  {
    return op(a, b);
  }

  BinaryOp op;
};

template <typename D, typename T, typename Ctx, typename BinaryOp>
auto reduce(Ctx& ctx, logical_data<D> data, BinaryOp&& op, T init_val)
{
  using out_t = typename shape_of<D>::element_type;
  auto result = ctx.logical_data(shape_of<scalar_view<out_t>>());

  if constexpr (reserved::view_of<D>::can_provide_raw_data)
  {
    // Determine temporary device storage requirements
    void* d_temp_storage      = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Reduce(
      d_temp_storage,
      temp_storage_bytes,
      (T*) nullptr,
      (T*) nullptr,
      data.shape().size(),
      OpWrapper<BinaryOp>(op),
      init_val,
      0);

    auto ltemp = ctx.logical_data(shape_of<slice<char>>(temp_storage_bytes));

    ctx.task(data.read(), result.write(), ltemp.write())
        ->*[&op, init_val, temp_storage_bytes](hipStream_t stream, auto d_data, auto d_result, auto d_temp) {
              size_t d_temp_size = shape(d_temp).size();

              hipcub::DeviceReduce::Reduce(
                (void*) d_temp.data_handle(),
                d_temp_size,
                reserved::view_of<D>::data(d_data),
                (T*) d_result.addr,
                reserved::view_of<D>::size(d_data),
                OpWrapper<BinaryOp>(op),
                init_val,
                stream);
            };
  }
  else
  {
    ctx.task(data.read(), result.write())->*[&op, init_val](hipStream_t stream, auto d_data, auto d_result) {
      // Determine temporary device storage requirements
      void* d_temp_storage      = nullptr;
      size_t temp_storage_bytes = 0;
      hipcub::DeviceReduce::Reduce(
        d_temp_storage,
        temp_storage_bytes,
        reserved::view_of<D>::begin(d_data),
        (T*) d_result.addr,
        reserved::view_of<D>::size(d_data),
        OpWrapper<BinaryOp>(op),
        init_val,
        0);

      cuda_safe_call(hipMallocAsync(&d_temp_storage, temp_storage_bytes, stream));

      hipcub::DeviceReduce::Reduce(
        d_temp_storage,
        temp_storage_bytes,
        reserved::view_of<D>::begin(d_data),
        (T*) d_result.addr,
        reserved::view_of<D>::size(d_data),
        OpWrapper<BinaryOp>(op),
        init_val,
        stream);

      cuda_safe_call(hipFreeAsync(d_temp_storage, stream));
    };
  }

  return result;
}

template <typename Ctx>
void run()
{
  Ctx ctx;

  const size_t N = 1024 * 16;

  int* X      = new int[N];
  int ref_tot = 0;

  for (size_t ind = 0; ind < N; ind++)
  {
    X[ind] = rand() % N;
    ref_tot += X[ind];
  }

  auto values = ctx.logical_data(X, {N});

  // int should be deduced from "values"...
  auto lresult = reduce(
    ctx,
    values,
    [] __device__(const int& a, const int& b) {
      return a + b;
    },
    0);

  int result = ctx.wait(lresult);
  _CCCL_ASSERT(result == ref_tot, "Incorrect result");

  ctx.finalize();
}

template <typename Ctx>
void run_2D()
{
  Ctx ctx;

  const size_t N  = 1024;
  const size_t N2 = N * N;

  int* X      = new int[N2];
  int ref_tot = 0;

  for (size_t ind = 0; ind < N2; ind++)
  {
    X[ind] = rand() % N2;
    ref_tot += X[ind];
  }

  auto values = ctx.logical_data(make_slice(X, std::tuple{N, N}, N));

  // int should be deduced from "values"...
  auto lresult = reduce(
    ctx,
    values,
    [] __device__(const int& a, const int& b) {
      return a + b;
    },
    0);

  int result = ctx.wait(lresult);
  _CCCL_ASSERT(result == ref_tot, "Incorrect result");

  ctx.finalize();
}

int main()
{
  run<stream_ctx>();
  run_2D<stream_ctx>();
  // run<graph_ctx>();
}
