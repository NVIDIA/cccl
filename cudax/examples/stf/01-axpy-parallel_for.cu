#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief An AXPY kernel implemented using the parallel_for construct
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

int main()
{
  context ctx;
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  double alpha = 3.14;

  auto lX = ctx.logical_data(X);
  auto lY = ctx.logical_data(Y);

  /* Compute Y = Y + alpha X */
  ctx.parallel_for(lY.shape(), lX.read(), lY.rw())->*[alpha] __device__(size_t i, auto dX, auto dY) {
    dY(i) += alpha * dX(i);
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    assert(fabs(Y[i] - (Y0(i) + alpha * X0(i))) < 0.0001);
    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
