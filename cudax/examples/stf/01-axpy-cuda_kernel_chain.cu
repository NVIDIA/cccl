#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Example of task implementing a chain of CUDA kernels
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

__global__ void axpy(double a, slice<const double> x, slice<double> y)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int i = tid; i < x.size(); i += nthreads)
  {
    y(i) += a * x(i);
  }
}

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

int main()
{
  context ctx    = graph_ctx();
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  double alpha = 3.14;
  double beta  = 4.5;
  double gamma = -4.1;

  auto lX = ctx.logical_data(X);
  auto lY = ctx.logical_data(Y);

  /* Compute Y = Y + alpha X, Y = Y + beta X and then  Y = Y + gamma X */
  ctx.cuda_kernel_chain(lX.read(), lY.rw())->*[&](auto dX, auto dY) {
    // clang-format off
        return std::vector<cuda_kernel_desc> {
            { axpy, 16, 128, 0, alpha, dX, dY },
            { axpy, 16, 128, 0, beta, dX, dY },
            { axpy, 16, 128, 0, gamma, dX, dY }
        };
    // clang-format on
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    assert(fabs(Y[i] - (Y0(i) + (alpha + beta + gamma) * X0(i))) < 0.0001);
    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
