//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Illustrate how to use the void data interface
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

int main()
{
  context ctx;

  auto ltask_res = ctx.logical_data(shape_of<void_interface>());
  ctx.task(ltask_res.write())->*[](hipStream_t, auto) {

  };

  void_interface sync;
  auto ltask2_res = ctx.logical_data(sync);
  ctx.task(ltask2_res.write(), ltask_res.read())->*[](hipStream_t, auto, auto) {

  };

  ctx.finalize();
}
