#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Jacobi method with parallel_for
 *
 */

#include <cuda/experimental/stf.cuh>

#include <iostream>

using namespace cuda::experimental::stf;

int main(int argc, char** argv)
{
  context ctx;

  size_t n        = 4096;
  size_t m        = 4096;
  size_t iter_max = 100;
  double tol      = 0.0000001;

  if (argc > 2)
  {
    n = atol(argv[1]);
    m = atol(argv[2]);
  }

  if (argc > 3)
  {
    iter_max = atoi(argv[3]);
  }

  if (argc > 4)
  {
    tol = atof(argv[4]);
  }

  auto lA    = ctx.logical_data(shape_of<slice<double, 2>>(m, n));
  auto lAnew = ctx.logical_data(lA.shape());

  ctx.parallel_for(lA.shape(), lA.write(), lAnew.write()).set_symbol("init")->*
    [=] __device__(size_t i, size_t j, auto A, auto Anew) {
      A(i, j) = (i == j) ? 10.0 : -1.0;
    };

  hipEvent_t start, stop;

  cuda_safe_call(hipEventCreate(&start));
  cuda_safe_call(hipEventCreate(&stop));

  cuda_safe_call(hipEventRecord(start, ctx.task_fence()));

  auto lresidual = ctx.logical_data(shape_of<scalar_view<double>>());

  size_t iter = 0;
  do
  {
    ctx.parallel_for(inner<1>(lA.shape()), lA.read(), lAnew.write(), lresidual.reduce(reducer::maxval<double>{}))
        ->*[] __device__(size_t i, size_t j, auto A, auto Anew, auto residual) {
              Anew(i, j) = 0.25 * (A(i - 1, j) + A(i + 1, j) + A(i, j - 1) + A(i, j + 1));
              residual   = ::std::max(residual, fabs(A(i, j) - Anew(i, j)));
            };

    ctx.parallel_for(inner<1>(lA.shape()), lA.rw(), lAnew.read())->*[] __device__(size_t i, size_t j, auto A, auto Anew) {
      A(i, j) = Anew(i, j);
    };

    iter++;

  } while (ctx.wait(lresidual) > tol && iter < iter_max);

  cuda_safe_call(hipEventRecord(stop, ctx.task_fence()));

  ctx.finalize();

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Elapsed time: %f ms\n", elapsedTime);
}
