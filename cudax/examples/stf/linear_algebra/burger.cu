#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Sparse conjugate gradient algorithm
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

#if !_CCCL_CTK_BELOW(12, 4)
using vector_t  = stackable_logical_data<slice<double>>;
using scalar_t  = stackable_logical_data<scalar_view<double>>;
using context_t = stackable_ctx;

struct csr_matrix
{
  csr_matrix(stackable_logical_data<slice<double>> _val_handle,
             stackable_logical_data<slice<size_t>> _row_handle,
             stackable_logical_data<slice<size_t>> _col_handle)
      : val_handle(mv(_val_handle))
      , row_handle(mv(_row_handle))
      , col_handle(mv(_col_handle))
  {}

  /* Description of the CSR */
  mutable stackable_logical_data<slice<double>> val_handle;
  mutable stackable_logical_data<slice<size_t>> row_handle;
  mutable stackable_logical_data<slice<size_t>> col_handle;
};

// Note that a and b might be the same logical data
void DOT(context_t& ctx, vector_t& a, vector_t& b, scalar_t& res)
{
  ctx.parallel_for(a.shape(), a.read(), b.read(), res.reduce(reducer::sum<double>{})).set_symbol("DOT")->*
    [] __device__(size_t i, auto da, auto db, double& dres) {
      dres += da(i) * db(i);
    };
};

void SPMV(context_t& ctx, csr_matrix& a, vector_t& x, vector_t& y)
{
  ctx.parallel_for(y.shape(), a.val_handle.read(), a.col_handle.read(), a.row_handle.read(), x.read(), y.write())
      .set_symbol("SPMV")
      ->*[] _CCCL_DEVICE(size_t row, auto da_val, auto da_col, auto da_row, auto dx, auto dy) {
            int row_start = da_row(row);
            int row_end   = da_row(row + 1);

            double sum = 0.0;
            for (int elt = row_start; elt < row_end; elt++)
            {
              sum += da_val(elt) * dx(da_col(elt));
            }

            dy(row) = sum;
          };
}

void build_tridiag_csr_structure(size_t* row_offsets, size_t* col_indices, size_t N)
{
  size_t n_unknowns = N - 2;
  size_t nnz        = 0;
  row_offsets[0]    = 0;

  for (size_t row = 0; row < n_unknowns; row++)
  {
    // For interior point i (global index = row + 1), the matrix equation involves:
    // - Left neighbor: matrix column = row-1 (if row > 0)
    // - Center: matrix column = row
    // - Right neighbor: matrix column = row+1 (if row < n_unknowns-1)

    if (row > 0)
    {
      col_indices[nnz++] = row - 1; // left neighbor
    }
    col_indices[nnz++] = row; // center
    if (row < n_unknowns - 1)
    {
      col_indices[nnz++] = row + 1; // right neighbor
    }
    row_offsets[row + 1] = nnz;
  }
}

void assemble_jacobian(context_t& ctx, vector_t U, vector_t values, size_t N, double h, double dt, double nu)
{
  size_t n_unknowns = N - 2;
  ctx.parallel_for(box(n_unknowns), U.read(), values.write())
      ->*[n_unknowns, h, dt, nu] __device__(size_t row, auto dU, auto dvalues) {
            size_t global = row + 1; // global grid index for this interior point
            double u_i    = dU[global];
            double u_ip1  = dU[global + 1];
            double u_im1  = dU[global - 1];

            double left   = -u_i / (2 * h) - nu / (h * h);
            double center = 1.0 / dt + (u_ip1 - u_im1) / (2 * h) + 2.0 * nu / (h * h);
            double right  = u_i / (2 * h) - nu / (h * h);

            // Calculate the starting index for this row's values in the CSR values array
            size_t val_idx = 0;
            if (row == 0)
            {
              // First row: starts at index 0, has center + right
              val_idx              = 0;
              dvalues[val_idx]     = center;
              dvalues[val_idx + 1] = right;
            }
            else if (row == n_unknowns - 1)
            {
              // Last row: starts at index 2 + 3*(n_unknowns-2), has left + center
              val_idx              = 2 + 3 * (n_unknowns - 2);
              dvalues[val_idx]     = left;
              dvalues[val_idx + 1] = center;
            }
            else
            {
              // Middle rows: start at index 2 + 3*(row-1), have left + center + right
              val_idx              = 2 + 3 * (row - 1);
              dvalues[val_idx]     = left;
              dvalues[val_idx + 1] = center;
              dvalues[val_idx + 2] = right;
            }
          };
}

// residual: length N-2
void compute_residual(
  context_t& ctx, vector_t U, vector_t U_prev, vector_t residual, size_t /* N */, double h, double dt, double nu)
{
  ctx.parallel_for(residual.shape(), residual.write(), U.read(), U_prev.read())
      ->*[h, dt, nu] __device__(size_t i, auto dresidual, auto dU, auto dU_prev) {
            size_t global = i + 1;
            double u_i    = dU[global];
            double u_ip1  = dU[global + 1];
            double u_im1  = dU[global - 1];

            double term_time = (u_i - dU_prev[global]) / dt;
            double term_conv = u_i * (u_ip1 - u_im1) / (2 * h);
            double term_diff = -nu * (u_im1 - 2 * u_i + u_ip1) / (h * h);

            dresidual(i) = term_time + term_conv + term_diff;
          };
}

void cg_solver(context_t& ctx, csr_matrix& A, vector_t& X, vector_t& B, double cg_tol = 1e-10)
{
  // Initial guess X = 0 (better for Newton corrections)
  ctx.parallel_for(X.shape(), X.write()).set_symbol("init_guess")->*[] _CCCL_DEVICE(size_t i, auto dX) {
    dX(i) = 0.0;
  };

  // Residual R initialized to B
  auto R = ctx.logical_data(B.shape());
  ctx.parallel_for(R.shape(), R.write(), B.read()).set_symbol("R=B")->*[] _CCCL_DEVICE(size_t i, auto dR, auto dB) {
    dR(i) = dB(i);
  };

  // R = R - A*X
  auto Ax = ctx.logical_data(X.shape()).set_symbol("Ax");
  SPMV(ctx, A, X, Ax);
  ctx.parallel_for(R.shape(), R.rw(), Ax.read()).set_symbol("R -= Ax")->*[] _CCCL_DEVICE(size_t i, auto dR, auto dAx) {
    dR(i) -= dAx(i);
  };

  // P = R;
  auto P = ctx.logical_data(R.shape()).set_symbol("P");
  ctx.parallel_for(P.shape(), P.write(), R.read()).set_symbol("P=R")->*[] _CCCL_DEVICE(size_t i, auto dP, auto dR) {
    dP(i) = dR(i);
  };

  // RSOLD = R'*R
  auto rsold = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("rsold");
  DOT(ctx, R, R, rsold);

  // CG iteration counter
  auto cg_iter = ctx.logical_data(shape_of<scalar_view<int>>()).set_symbol("cg_iter");
  ctx.parallel_for(box(1), cg_iter.write()).set_symbol("init_cg_iter")->*[] _CCCL_DEVICE(size_t i, auto diter) {
    *diter = 0;
  };

  {
    auto while_guard = ctx.while_graph_scope();

    // Ap = A*P
    auto Ap = ctx.logical_data(P.shape()).set_symbol("Ap");
    SPMV(ctx, A, P, Ap);

    // We don't compute alpha explicitly
    // alpha = rsold / (p' * Ap);
    auto pAp = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("pAp");
    DOT(ctx, P, Ap, pAp);

    // x = x + alpha * p;
    ctx.parallel_for(X.shape(), X.rw(), rsold.read(), pAp.read(), P.read()).set_symbol("X+=alpha*P")
        ->*[] _CCCL_DEVICE(size_t i, auto dX, auto drsold, auto dpAp, auto dP) {
              double alpha = (*drsold / *dpAp);
              dX(i) += alpha * dP(i);
            };

    // r = r - alpha * Ap;
    ctx.parallel_for(R.shape(), R.rw(), rsold.read(), pAp.read(), Ap.read()).set_symbol("R-=alpha*Ap")
        ->*[] _CCCL_DEVICE(size_t i, auto dR, auto drsold, auto dpAp, auto dAp) {
              double alpha = (*drsold / *dpAp);
              dR(i) -= alpha * dAp(i);
            };

    // rsnew = r' * r;
    auto rsnew = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("rsnew");
    DOT(ctx, R, R, rsnew);

    while_guard.update_cond(rsnew.read(), cg_iter.rw())->*[cg_tol] __device__(auto drsnew, auto diter) {
      (*diter)++; // increment iteration counter
      // printf("CG iter %d: RES %e (tol=%e)\n", *diter, *drsnew, *dtol);
      bool converged = (*drsnew < cg_tol * cg_tol);
      return !converged;
    };

    // p = r + (rsnew / rsold) * p;
    ctx.parallel_for(P.shape(), P.rw(), R.read(), rsnew.read(), rsold.read()).set_symbol("P=r+(rsnew/rsold)*P")
        ->*[] _CCCL_DEVICE(size_t i, auto dP, auto dR, auto drsnew, auto drsold) {
              dP(i) = dR(i) + (*drsnew / *drsold) * dP(i);
            };

    // update old residual
    ctx.parallel_for(box(1), rsold.write(), rsnew.read()).set_symbol("update_rsold")
        ->*[] _CCCL_DEVICE(size_t i, auto drsold, auto drsnew) {
              *drsold = *drsnew;
            };
  }

  fprintf(stderr,
          "CG solver converged after %d iterations (final residual %e, tolerance %e)\n",
          ctx.wait(cg_iter),
          std::sqrt(ctx.wait(rsold)),
          cg_tol);
}
#endif

int main([[maybe_unused]] int argc, [[maybe_unused]] char** argv)
{
#if _CCCL_CTK_BELOW(12, 4)
  fprintf(stderr, "Waiving test: conditional nodes are only available since CUDA 12.4.\n");
  return 0;
#else
  size_t N = 100000; // Large system - auto-scaled parameters maintain stability

  context_t ctx;

  if (argc > 1)
  {
    N = atoi(argv[1]);
    fprintf(stderr, "N = %zu\n", N);
  }

  double h          = 1.0 / (N - 1);
  size_t n_unknowns = N - 2;

  // Set reasonable parameters - implicit method allows larger time steps
  double nu           = 0.05;
  double dt_diffusion = 0.5 * h * h / nu; // Diffusion-limited time step
  double dt_fixed     = 0.001; // Fixed reasonable time step
  double dt           = std::max(dt_diffusion, dt_fixed); // Use larger of the two

  // For very fine grids, cap the time step to prevent tiny steps
  if (N > 10000)
  {
    dt = std::min(dt, 0.01); // Cap at 0.01 for large grids
  }

  size_t nsteps     = 10000;
  double total_time = nsteps * dt;

  fprintf(stderr, "=== Simulation Parameters ===\n");
  fprintf(stderr, "Grid: N=%zu, h=%e\n", N, h);
  fprintf(stderr, "Time: dt=%e, nsteps=%zu, total_time=%e\n", dt, nsteps, total_time);
  fprintf(stderr, "Physics: nu=%e\n", nu);
  fprintf(stderr, "Diffusion number: nu*dt/h^2 = %e\n", nu * dt / (h * h));
  fprintf(stderr, "System size: %zu unknowns, %zu non-zeros\n", n_unknowns, 3 * n_unknowns - 2);
  if (N > 1000000)
  {
    fprintf(stderr, "WARNING: Very large system! Consider using iterative preconditioners for N > 1M\n");
  }
  fprintf(stderr, "=============================\n");

  // First and last rows have 2 entries each, middle rows have 3 entries each
  // Total: 2 + 3*(n_unknowns-2) + 2 = 3*n_unknowns - 2
  size_t nz = 3 * n_unknowns - 2;

  size_t* row_offsets;
  size_t* col_indices;
  cuda_safe_call(hipHostAlloc(&row_offsets, (n_unknowns + 1) * sizeof(size_t), hipHostMallocMapped));
  cuda_safe_call(hipHostAlloc(&col_indices, nz * sizeof(size_t), hipHostMallocMapped));

  build_tridiag_csr_structure(row_offsets, col_indices, N);

  auto csr_row_offsets = ctx.logical_data(make_slice(row_offsets, n_unknowns + 1));
  auto csr_col_ind     = ctx.logical_data(make_slice(col_indices, nz));
  auto csr_values      = ctx.logical_data(shape_of<slice<double>>(nz));

  auto U      = ctx.logical_data(shape_of<slice<double>>(N));
  auto U_prev = ctx.logical_data(shape_of<slice<double>>(N));

  auto residual = ctx.logical_data(shape_of<slice<double>>(n_unknowns));
  auto rhs      = ctx.logical_data(shape_of<slice<double>>(n_unknowns));
  auto delta    = ctx.logical_data(shape_of<slice<double>>(n_unknowns));

  // Initial condition
  ctx.parallel_for(U_prev.shape(), U_prev.write())->*[h, N] __device__(size_t i, auto dU_prev) {
    double x = i * h;
    if (i == 0 || i == N - 1)
    {
      dU_prev(i) = 0.0; // Homogeneous Dirichlet boundary conditions
    }
    else
    {
      dU_prev(i) = sin(M_PI * x);
    }
  };

  // Parameters are now set above with auto-scaling
  for (size_t t = 0; t < nsteps; t++)
  {
    // initial guess: u = u_prev (with boundary conditions)
    ctx.parallel_for(U.shape(), U.write(), U_prev.read())->*[] __device__(size_t i, auto dU, auto dU_prev) {
      dU(i) = dU_prev(i);
    };

    size_t max_newton = 50;
    for (size_t newton = 0; newton < max_newton; newton++)
    {
      compute_residual(ctx, U, U_prev, residual, N, h, dt, nu);

      // Compute Newton residual norm for adaptive CG tolerance
      auto newton_norm2 = ctx.logical_data(shape_of<scalar_view<double>>());
      DOT(ctx, residual, residual, newton_norm2);
      double newton_residual = std::sqrt(ctx.wait(newton_norm2));

      // Adaptive CG tolerance: Eisenstat-Walker style
      double cg_tol = std::max(1e-12, std::min(0.1 * newton_residual, 1e-8));

      assemble_jacobian(ctx, U, csr_values, N, h, dt, nu);

      ctx.parallel_for(rhs.shape(), rhs.write(), residual.read())->*[] __device__(size_t i, auto drhs, auto dresidual) {
        drhs(i) = -dresidual(i);
      };

      csr_matrix A(csr_values, csr_row_offsets, csr_col_ind);

      // Solve A * delta = rhs with adaptive tolerance
      cg_solver(ctx, A, delta, rhs, cg_tol);

      // Update solution: interior unknowns get delta corrections, boundaries stay zero
      ctx.parallel_for(U.shape(), U.rw(), delta.read())->*[N] __device__(size_t i, auto dU, auto ddelta) {
        if (i == 0 || i == N - 1)
        {
          dU(i) = 0.0; // Enforce boundary conditions
        }
        else
        {
          dU(i) += ddelta(i - 1); // Interior: delta[i-1] corresponds to interior unknown at global index i
        }
      };

      // Convergence check (using already computed Newton residual)
      fprintf(stderr, "Newton iter %zu: residual norm = %e, CG tol = %e\n", newton, newton_residual, cg_tol);
      if (newton_residual < 1e-10)
      {
        break;
      }
    }

    // accept timestep
    ctx.parallel_for(U.shape(), U_prev.write(), U.read())->*[] __device__(size_t i, auto dU_prev, auto dU) {
      dU_prev(i) = dU(i);
    };

    // Dump solution for visualization (every 10 time steps to avoid too much output)
    if (t % 10 == 0 || t == nsteps - 1)
    {
      ctx.host_launch(U.read())->*[t, h, N, dt](auto hU) {
        char filename[256];
        snprintf(filename, sizeof(filename), "solution_t%04zu.dat", t);

        FILE* fp = fopen(filename, "w");
        if (fp)
        {
          fprintf(fp, "# Burger equation solution at step=%zu, physical_time=%.6e\n", t, t * dt);
          fprintf(fp, "# Format: x_coordinate  u(x,t)\n");
          fprintf(fp, "# Grid points: %zu\n", N);

          for (size_t i = 0; i < N; i++)
          {
            double x = i * h;
            fprintf(fp, "%.10e %.10e\n", x, hU(i));
          }

          fclose(fp);
          printf("Solution dumped to %s (t=%.4e)\n", filename, t * dt);
        }
        else
        {
          printf("Error: Could not open %s for writing\n", filename);
        }
      };
    }

    if (t % 1000 == 0 || t == nsteps - 1)
    {
      printf("Step %zu/%zu (t=%.4e) done\n", t, nsteps, t * dt);
    }
  }

  // Final solution dump
  printf("\n=== Simulation complete ===\n");
  printf("Solution files: solution_t*.dat (every 10th timestep)\n");
  printf("\nVisualization options:\n");
  printf("1. Python/matplotlib:\n");
  printf("  python3 -c \"\n");
  printf("import numpy as np; import matplotlib.pyplot as plt; import glob\n");
  printf("files = sorted(glob.glob('solution_t*.dat'))\n");
  printf("for f in files[::2]:  # plot every other file\n");
  printf("    data = np.loadtxt(f)\n");
  printf("    plt.plot(data[:,0], data[:,1], label=f.split('.')[0])\n");
  printf("plt.xlabel('x'); plt.ylabel('u(x,t)'); plt.legend(); plt.show()\n");
  printf("\"\n");
  printf("\n2. gnuplot:\n");
  printf("  gnuplot -e \"plot for [i=0:9] 'solution_t'.sprintf('%%04d',i*10).'.dat' u 1:2 w l title 'step '.i*10\"\n");
  printf("\n3. Simple inspection:\n");
  printf("  head -20 solution_t0000.dat  # initial condition\n");
  printf("  head -20 solution_t0099.dat  # final solution\n");

  ctx.finalize();
#endif
}
