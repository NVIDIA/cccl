//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Sparse conjugate gradient algorithm
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

using vector_t  = stackable_logical_data<slice<double>>;
using scalar_t  = stackable_logical_data<scalar_view<double>>;
using context_t = stackable_ctx;

struct csr_matrix
{
  csr_matrix(
    context_t& ctx, size_t num_rows, size_t num_nonzeros, double* values, size_t* row_offsets, size_t* column_indices)
  {
    val_handle = ctx.logical_data(make_slice(values, num_nonzeros));
    col_handle = ctx.logical_data(make_slice(column_indices, num_nonzeros));
    row_handle = ctx.logical_data(make_slice(row_offsets, num_rows + 1));

    val_handle.set_symbol("csr_val");
    col_handle.set_symbol("csr_col");
    row_handle.set_symbol("csr_row");
  }

  /* Description of the CSR */
  mutable stackable_logical_data<slice<double>> val_handle;
  mutable stackable_logical_data<slice<size_t>> row_handle;
  mutable stackable_logical_data<slice<size_t>> col_handle;
};

// Note that a and b might be the same logical data
void DOT(context_t& ctx, vector_t& a, vector_t& b, scalar_t& res)
{
  ctx.parallel_for(a.shape(), a.read(), b.read(), res.reduce(reducer::sum<double>{})).set_symbol("DOT")->*
    [] __device__(size_t i, auto da, auto db, double& dres) {
      dres += da(i) * db(i);
    };
};

void SPMV(context_t& ctx, csr_matrix& a, vector_t& x, vector_t& y)
{
  ctx.parallel_for(y.shape(), a.val_handle.read(), a.col_handle.read(), a.row_handle.read(), x.read(), y.write())
      .set_symbol("SPMV")
      ->*[] _CCCL_DEVICE(size_t row, auto da_val, auto da_col, auto da_row, auto dx, auto dy) {
            int row_start = da_row(row);
            int row_end   = da_row(row + 1);

            double sum = 0.0;
            for (int elt = row_start; elt < row_end; elt++)
            {
              sum += da_val(elt) * dx(da_col(elt));
            }

            dy(row) = sum;
          };
}

/* genTridiag: generate a random tridiagonal symmetric matrix
   from :
   https://github.com/NVIDIA/cuda-samples/blob/master/Samples/4_CUDA_Libraries/conjugateGradientCudaGraphs/conjugateGradientCudaGraphs.cu
 */
void genTridiag(size_t* I, size_t* J, double* val, size_t N, size_t nz)
{
  const double d = 2.0;

  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = drand48() + d;
  val[1] = drand48();
  int start;

  for (size_t i = 1; i < N; i++)
  {
    if (i > 1)
    {
      I[i] = I[i - 1] + 3;
    }
    else
    {
      I[1] = 2;
    }

    start        = (i - 1) * 3 + 2;
    J[start]     = i - 1;
    J[start + 1] = i;

    if (i < N - 1)
    {
      J[start + 2] = i + 1;
    }

    val[start]     = val[start - 1];
    val[start + 1] = drand48() + d;

    if (i < N - 1)
    {
      val[start + 2] = drand48();
    }
  }

  I[N] = nz;
}

void cg_solver(context_t& ctx, csr_matrix& A, vector_t& X, vector_t& B)
{
  // Initial guess X = 1
  ctx.parallel_for(X.shape(), X.write()).set_symbol("init_guess")->*[] _CCCL_DEVICE(size_t i, auto dX) {
    dX(i) = 1.0;
  };

  // Residual R initialized to B
  auto R = ctx.logical_data(B.shape());
  ctx.parallel_for(R.shape(), R.write(), B.read()).set_symbol("R=B")->*[] _CCCL_DEVICE(size_t i, auto dR, auto dB) {
    dR(i) = dB(i);
  };

  // R = R - A*X
  auto Ax = ctx.logical_data(X.shape()).set_symbol("Ax");
  SPMV(ctx, A, X, Ax);
  ctx.parallel_for(R.shape(), R.rw(), Ax.read()).set_symbol("R -= Ax")->*[] _CCCL_DEVICE(size_t i, auto dR, auto dAx) {
    dR(i) -= dAx(i);
  };

  // P = R;
  auto P = ctx.logical_data(R.shape()).set_symbol("P");
  ctx.parallel_for(P.shape(), P.write(), R.read()).set_symbol("P=R")->*[] _CCCL_DEVICE(size_t i, auto dP, auto dR) {
    dP(i) = dR(i);
  };

  // RSOLD = R'*R
  auto rsold = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("rsold");
  DOT(ctx, R, R, rsold);

  {
    auto while_guard = ctx.while_graph_scope();

    // Ap = A*P
    auto Ap = ctx.logical_data(P.shape()).set_symbol("Ap");
    SPMV(ctx, A, P, Ap);

    // We don't compute alpha explicitly
    // alpha = rsold / (p' * Ap);
    auto pAp = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("pAp");
    DOT(ctx, P, Ap, pAp);

    // x = x + alpha * p;
    ctx.parallel_for(X.shape(), X.rw(), rsold.read(), pAp.read(), P.read()).set_symbol("X+=alpha*P")
        ->*[] _CCCL_DEVICE(size_t i, auto dX, auto drsold, auto dpAp, auto dP) {
              double alpha = (*drsold / *dpAp);
              dX(i) += alpha * dP(i);
            };

    // r = r - alpha * Ap;
    ctx.parallel_for(R.shape(), R.rw(), rsold.read(), pAp.read(), Ap.read()).set_symbol("R-=alpha*Ap")
        ->*[] _CCCL_DEVICE(size_t i, auto dR, auto drsold, auto dpAp, auto dAp) {
              double alpha = (*drsold / *dpAp);
              dR(i) -= alpha * dAp(i);
            };

    // rsnew = r' * r;
    auto rsnew = ctx.logical_data(shape_of<scalar_view<double>>()).set_symbol("rsnew");
    DOT(ctx, R, R, rsnew);

    while_guard.update_cond(rsnew.read())->*[] __device__(auto drsnew) {
      printf("RES %e\n", *drsnew);
      bool converged = (*drsnew < 1e-13);
      return !converged;
    };

    // p = r + (rsnew / rsold) * p;
    ctx.parallel_for(P.shape(), P.rw(), R.read(), rsnew.read(), rsold.read()).set_symbol("P=r+(rsnew/rsold)*P")
        ->*[] _CCCL_DEVICE(size_t i, auto dP, auto dR, auto drsnew, auto drsold) {
              dP(i) = dR(i) + (*drsnew / *drsold) * dP(i);
            };

    // update old residual
    ctx.parallel_for(box(1), rsold.write(), rsnew.read()).set_symbol("update_rsold")
        ->*[] _CCCL_DEVICE(size_t i, auto drsold, auto drsnew) {
              *drsold = *drsnew;
            };
  }
}

int main(int argc, char** argv)
{
  size_t N = 10485760;

  context_t ctx;

  if (argc > 1)
  {
    N = atoi(argv[1]);
    fprintf(stderr, "N = %zu\n", N);
  }

  size_t nz = (N - 2) * 3 + 4;

  size_t* row_offsets;
  size_t* column_indices;
  double* values;
  cuda_safe_call(hipHostAlloc(&row_offsets, (N + 1) * sizeof(size_t), hipHostMallocMapped));
  cuda_safe_call(hipHostAlloc(&column_indices, nz * sizeof(size_t), hipHostMallocMapped));
  cuda_safe_call(hipHostAlloc(&values, nz * sizeof(double), hipHostMallocMapped));

  // Generate a random matrix that is supposed to be invertible
  genTridiag(row_offsets, column_indices, values, N, nz);

  csr_matrix A(ctx, N, nz, values, row_offsets, column_indices);

  auto X = ctx.logical_data(shape_of<slice<double>>(N)).set_symbol("X");
  auto B = ctx.logical_data(shape_of<slice<double>>(N)).set_symbol("B");

  // RHS
  ctx.parallel_for(B.shape(), B.write()).set_symbol("B assembly")->*[] __device__(size_t i, auto dB) {
    dB(i) = 1.0;
  };

  cg_solver(ctx, A, X, B);

  ctx.finalize();
}
