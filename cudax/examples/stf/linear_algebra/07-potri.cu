#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief This example implements the POTRI matrix inversion algorithm over multiple devices
 *
 *
 */
#include <cuda/experimental/stf.cuh>

#include <hipblas.h>
#include <hipsolver.h>
#include <nvtx3/nvToolsExt.h>

#define TILED

using namespace cuda::experimental::stf;

stream_ctx ctx;

static std::unordered_map<int, hipblasHandle_t> cublas_handles;
static std::unordered_map<int, hipsolverHandle_t> cusolver_handles;

/* Get a CUBLAS handle valid on the current device, or initialize it lazily */
hipblasHandle_t& get_cublas_handle()
{
  int dev = cuda_try<hipGetDevice>();

  auto& result = cublas_handles[dev];
  if (result == hipblasHandle_t())
  { // not found, default value inserted
    // Lazy initialization, and save the handle for future use
    cuda_try(hipblasCreate(&result));
  }
  return result;
}

/* Get a CUSOLVER handle valid on the current device, or initialize it lazily */
hipsolverHandle_t& get_cusolver_handle()
{
  int dev;
  cuda_try(hipGetDevice(&dev));

  auto& result = cusolver_handles[dev];
  if (result == hipsolverHandle_t())
  { // not found, default value inserted
    // Lazy initialization, and save the handle for future use
    cuda_try(hipsolverDnCreate(&result));
  }
  return result;
}

template <typename T>
class matrix
{
public:
  matrix(int NROWS, int NCOLS, int BLOCKSIZE_ROWS, int BLOCKSIZE_COLS, bool is_sym, const char* _symbol = "matrix")
  {
    symbol = _symbol;

    sym_matrix = is_sym;

    m  = NROWS;
    mb = BLOCKSIZE_ROWS;

    n  = NCOLS;
    nb = BLOCKSIZE_COLS;

    assert(m % mb == 0);
    assert(n % nb == 0);

    size_t s = m * n * sizeof(T);
    // cuda_try(hipHostMalloc(&h_array, m*n*sizeof(T)));
    // fprintf(stderr, "Allocating %ld x %ld x %ld = %ld bytes (%f GB) on host for %s\n", m, n, sizeof(T), s,
    //        s / (1024.0 * 1024.0 * 1024.0), _symbol);
    h_array = (T*) malloc(s);
    assert(h_array);
    cuda_try(hipHostRegister(h_array, s, hipHostRegisterPortable));
    // cuda_try(hipMalloc(&d_array, m*n*sizeof(T)));

    // Compute the number of blocks
    mt = m / mb;
    nt = n / nb;

    handles.resize(mt * nt);

    for (int colb = 0; colb < nt; colb++)
    {
      int low_rowb = sym_matrix ? colb : 0;
      for (int rowb = low_rowb; rowb < mt; rowb++)
      {
        T* addr_h = get_block_h(rowb, colb);
        auto& h   = get_handle(rowb, colb);

#ifdef TILED
        // tiles are stored contiguously
        size_t ld = mb;
#else
        size_t ld = m;
#endif
        std::ignore = ld; // work around compiler bug
        h           = ctx.logical_data(make_slice(addr_h, std::tuple{mb, nb}, ld));
        h.set_symbol(std::string(symbol) + "_" + std::to_string(rowb) + "_" + std::to_string(colb));
        h.set_write_back(false);
      }
    }

    cuda_try(hipGetDeviceCount(&ndevs));
    for (int a = 1; a * a <= ndevs; a++)
    {
      if (ndevs % a == 0)
      {
        grid_p = a;
        grid_q = ndevs / a;
      }
    }

    assert(grid_p * grid_q == ndevs);

    // std::cout << "FOUND " << ndevs << " DEVICES "
    //          << "p=" << grid_p << " q=" << grid_q << std::endl;
  }

  int get_preferred_devid(int row, int col)
  {
    return (row % grid_p) + (col % grid_q) * grid_p;
  }

  auto& get_handle(int row, int col)
  {
    return handles[row + col * mt];
  }

  size_t get_index(size_t row, size_t col)
  {
#ifdef TILED
    // Find which tile contains this element
    int tile_row = row / mb;
    int tile_col = col / nb;

    size_t tile_size = mb * nb;

    // Look for the index of the begining of the tile
    size_t tile_start = (tile_row + mt * tile_col) * tile_size;

    // Offset within the tile
    size_t offset = (row % mb) + (col % nb) * mb;

    return tile_start + offset;
#else
    return row + col * m;
#endif
  }

  T* get_block_h(int brow, int bcol)
  {
    size_t index = get_index(brow * mb, bcol * nb);
    return &h_array[index];
  }

  // Fill with func(Matrix*,row, col)
  template <typename Fun>
  void fill(Fun&& fun)
  {
    nvtxRangePushA("FILL");
    // Fill blocks by blocks
    for (int colb = 0; colb < nt; colb++)
    {
      int low_rowb = sym_matrix ? colb : 0;
      for (int rowb = low_rowb; rowb < mt; rowb++)
      {
        // Each task fills a block
        auto& h   = get_handle(rowb, colb);
        int devid = get_preferred_devid(rowb, colb);

        ctx.parallel_for(exec_place::device(devid), h.shape(), h.write()).set_symbol("INIT")->*
          [=] _CCCL_DEVICE(size_t lrow, size_t lcol, auto sA) {
            size_t row     = lrow + rowb * sA.extent(0);
            size_t col     = lcol + colb * sA.extent(1);
            sA(lrow, lcol) = fun(row, col);
          };
      }
    }
    nvtxRangePop();
  }

  // Print blocks
  void print()
  {
    // print blocks by blocks
    for (int colb = 0; colb < nt; colb++)
    {
      int low_rowb = sym_matrix ? colb : 0;
      for (int rowb = low_rowb; rowb < mt; rowb++)
      {
        // Each task fills a block
        ctx.host_launch(get_handle(rowb, colb).read())->*[=](auto sA) {
          for (int lcol = 0; lcol < sA.extent(1); lcol++)
          {
            size_t col = lcol + colb * sA.extent(1);
            for (int lrow = 0; lrow < sA.extent(0); lrow++)
            {
              size_t row = lrow + rowb * sA.extent(0);

              fprintf(stderr, "%d,%d : %le\n", row, col, sA(lrow, lcol));
            }
          }
        };
      }
    }
  }

  T* h_array;
  T* d_array;
  size_t m; // nrows
  size_t n; // ncols

  // Is this a sym matrix ? (lower assumed)
  bool sym_matrix;

  size_t mb; // block size (rows)
  size_t nb; // block size (cols)

  size_t mt; // number of column blocks
  size_t nt; // number of row blocks

  // abstract data handles
  std::vector<logical_data<slice<double, 2>>> handles;

  const char* symbol;

  // for the mapping
  int ndevs;
  int grid_p, grid_q;
};

void DPOTRF(hipblasFillMode_t uplo, matrix<double>& A, int A_row, int A_col)
{
  auto& Akk    = A.get_handle(A_row, A_col);
  size_t m_akk = Akk.shape().extent(0);
  // Note that the handle may be different from the actual handle...
  int Lwork_expected;
  cuda_safe_call(hipsolverDnDpotrf_bufferSize(get_cusolver_handle(), uplo, m_akk, nullptr, 0, &Lwork_expected));

  auto potrf_buffer = ctx.logical_data(shape_of<slice<double>>(Lwork_expected));
  potrf_buffer.set_allocator(ctx.get_default_allocator());

  auto devInfo = ctx.logical_data(shape_of<slice<int>>(1));

  auto t = ctx.task(Akk.rw(), potrf_buffer.write(), devInfo.write());
  t.set_symbol("DPOTRF");
  t->*[&](auto s, auto sAkk, auto buffer, auto info) {
    auto& h = get_cusolver_handle();
    cuda_try(hipsolverSetStream(h, s));

    cuda_try(hipsolverDnDpotrf(
      h,
      uplo,
      sAkk.extent(0),
      sAkk.data_handle(),
      sAkk.stride(1),
      buffer.data_handle(),
      buffer.extent(0),
      info.data_handle()));
  };
}

void DTRTRI(hipblasFillMode_t uplo, hipblasDiagType_t diag, matrix<double>& A, int A_row, int A_col)
{
  // Preallocate a buffer used by CUSOLVER
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  int64_t m_a00 = A.mb;
  assert(A.mb == A.nb);

  cuda_try(cusolverDnXtrtri_bufferSize(
    get_cusolver_handle(),
    uplo,
    diag,
    m_a00,
    HIP_R_64F /* DTRTRI */,
    nullptr,
    m_a00,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost));

  // We don't support allocating buffers of 0 bytes ... XXX
  if (workspaceInBytesOnHost == 0)
  {
    workspaceInBytesOnHost = 8;
  }

  auto d_buffer = ctx.logical_data(shape_of<slice<char>>(workspaceInBytesOnDevice));
  auto h_buffer = ctx.logical_data(shape_of<slice<char>>(workspaceInBytesOnHost));
  d_buffer.set_allocator(ctx.get_default_allocator());
  h_buffer.set_allocator(ctx.get_default_allocator());

  auto devInfo = ctx.logical_data(shape_of<slice<int>>(1));

  auto t =
    ctx.task(A.get_handle(A_row, A_col).rw(), d_buffer.write(), h_buffer.write(data_place::managed), devInfo.write());
  t.set_symbol("DTRTRI");
  t->*[&](auto s, auto sA, auto dbuffer, auto hbuffer, auto info) {
    auto& h = get_cusolver_handle();
    cuda_try(hipsolverSetStream(h, s));

    // DTRTRI(...)
    cuda_try(cusolverDnXtrtri(
      h,
      uplo,
      diag,
      sA.extent(0),
      HIP_R_64F /* DTRTRI */,
      sA.data_handle(),
      sA.stride(1),
      (double*) dbuffer.data_handle(),
      workspaceInBytesOnDevice,
      (double*) hbuffer.data_handle(),
      workspaceInBytesOnHost,
      info.data_handle()));
  };
}

/*
 * Note: this code was taken from CUSOLVER
 *
 * SLACPY copies all or part of a two-dimensional matrix A to another matrix B.
 *
 *  up     up_and_lo
 *  1          0          upper triangle, including diagonal
 *  0          0          lower triangle, including diagonal
 *  ?          1          whole matrix
 *
 * configuration:
 *   dim3 grids( m/VEC, m/BY )
 *   dim3 threads(VEC,BY)
 */
template <typename T_ELEM_SRC, typename T_ELEM_DST, int VEC_LOG, int BY_LOG>
__global__ void __launch_bounds__(1 << (VEC_LOG + BY_LOG))
  lacpy_kernel(int m, int n, const T_ELEM_SRC* A, size_t lda, T_ELEM_DST* B, size_t ldb, int up, int up_and_lo)
{
  const int VEC = (1 << VEC_LOG);
  const int BY  = (1 << BY_LOG);

  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  const int ibx = blockIdx.x * VEC;
  const int iby = blockIdx.y * BY;

  const int i = ibx + inx;
  const int j = iby + iny;

  if (ibx >= m)
  {
    return;
  }
  if (iby >= n)
  {
    return;
  }

  T_ELEM_SRC Areg = T_ELEM_SRC(0);

  if (up_and_lo)
  {
    /*
     * copy whole matrix
             DO 60 J = 1, N
                DO 50 I = 1, M
                   B( I, J ) = A( I, J )
       50       CONTINUE
       60    CONTINUE
    */
    if ((i < m) && (j < n))
    {
      Areg           = A[i + j * lda];
      B[i + j * ldb] = T_ELEM_DST(Areg);
    }
    return;
  }

  // only lower or upper triangle is copied.
  if (up)
  {
    /*
     * copy upper triangle, including diagonal
             DO 20 J = 1, N
                DO 10 I = 1, MIN( J, M )
                   B( I, J ) = A( I, J )
       10       CONTINUE
       20    CONTINUE
     */
    if ((i <= min(j, m - 1)) && (j < n))
    {
      Areg           = A[i + j * lda];
      B[i + j * ldb] = T_ELEM_DST(Areg);
    }
  }
  else
  {
    /*
     * copy lower triangle, including diagonal
             DO 40 J = 1, N
                DO 30 I = J, M
                   B( I, J ) = A( I, J )
       30       CONTINUE
       40    CONTINUE

     */
    if (((j <= i) && (i < m)) && (j < n))
    {
      Areg           = A[i + j * lda];
      B[i + j * ldb] = T_ELEM_DST(Areg);
    }
  }
}

/*
 * SLACPY copies all or part of a two-dimensional matrix A to another
 * matrix B.
 *
 * Input
 * -------
 *          UPLO is CHARACTER*1
 *          Specifies the part of the matrix A to be copied to B.
 *          = 'U':      Upper triangular part
 *          = 'L':      Lower triangular part
 *          Otherwise:  All of the matrix A
 *
 *          M is INTEGER
 *          The number of rows of the matrix A.
 *          M >= 0.
 *
 *          N is INTEGER
 *          The number of columns of the matrix A.
 *          N >= 0.
 *
 *          A is REAL array, dimension (LDA,N)
 *          The m by n matrix A.  If UPLO = 'U', only the upper triangle
 *          or trapezoid is accessed; if UPLO = 'L', only the lower
 *          triangle or trapezoid is accessed.
 *
 *          LDA is INTEGER
 *          The first dimension of the array A. LDA >= max(1,M).
 *
 *          B is REAL array, dimension (LDB,N)
 *          On exit, B = A in the locations specified by UPLO.
 *
 *          LDB is INTEGER
 *          The leading dimension of the array B.  LDB >= max(1,M).
 *
 */
template <typename T_ELEM_SRC, typename T_ELEM_DST>
hipsolverStatus_t cusolverDnXlacpy(
  hipblasFillMode_t uplo, // "UPPER", B = upper(A)
                         // "LOWER", B = lower(A)
                         // otherwise, B = A
  int m,
  int n,
  const T_ELEM_SRC* A,
  int lda,
  T_ELEM_DST* B,
  int ldb,
  hipStream_t stream)
{
  hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1   = hipSuccess;

  int up        = 0;
  int up_and_lo = 0;

  //  Quick return if possible
  if ((0 >= m) || (0 >= n))
  {
    return status;
  }

  /*
   *  up     up_and_lo
   *  1          0          upper triangle, including diagonal
   *  0          0          lower triangle, including diagonal
   *  ?          1          whole matrix
   */
  if (HIPBLAS_FILL_MODE_LOWER == uplo)
  {
    // Lower triangular part
    up = 0;
  }
  else if (HIPBLAS_FILL_MODE_UPPER == uplo)
  {
    // upper triangular part
    up = 1;
  }
  else
  {
    up_and_lo = 1; // Otherwise:  All of the matrix A
  }

  const int VEC_LOG = 5;
  const int BY_LOG  = 3;
  const int VEC     = (1 << VEC_LOG);
  const int BY      = (1 << BY_LOG);
  dim3 grids((m + VEC - 1) / VEC, (n + BY - 1) / BY);
  dim3 threads(VEC, BY);

  lacpy_kernel<T_ELEM_SRC, T_ELEM_DST, VEC_LOG, BY_LOG>
    <<<grids, threads, 0, stream>>>(m, n, A, (size_t) lda, B, (size_t) ldb, up, up_and_lo);

  cudaStat1 = hipGetLastError(); /* launch error */
  if (hipSuccess != cudaStat1)
  {
    fprintf(stderr, "Error (lacpy): %d\n", cudaStat1);
    status = HIPSOLVER_STATUS_EXECUTION_FAILED;
  }

  return status;
}

hipsolverStatus_t cusolverDnDlacpy(
  hipblasFillMode_t uplo, // "UPPER", B = upper(A)
                         // "LOWER", B = lower(A)
                         // otherwise, B = A
  int m,
  int n,
  const double* A,
  int lda,
  double* B,
  int ldb,
  hipStream_t stream)
{
  return cusolverDnXlacpy<double, double>(uplo, m, n, A, lda, B, ldb, stream);
}

// Pretend there is a CUBLAS interface for DLAAUM
void cublasDnDlaaum_bufferSize(hipblasHandle_t /*unused*/, int m, int n, size_t* Workspace_size)
{
  assert(Workspace_size);
  *Workspace_size = m * n * sizeof(double);
}

// Pretend there is a CUBLAS interface for DLAAUM
// A triangular
// Lower : A = A^T * A
// Upper : A = A A^T
void cublasDnDlaaum(
  hipblasHandle_t cublas_handle,
  hipblasFillMode_t uplo,
  int m,
  int n,
  double* A,
  int ldA,
  double* Workspace_d,
  size_t Workspace_size)
{
  hipStream_t stream;
  cuda_safe_call(hipblasGetStream(cublas_handle, &stream));

  // "Hand coded"
  // We use a full copy of A !
  // fprintf(stderr, "GOT Workspace_size %ld ... expected %d\n", Workspace_size, m * n * sizeof(double));
  std::ignore = Workspace_size;
  assert(Workspace_size >= m * n * sizeof(double));

  double* B = Workspace_d;
  int ldB   = m;

  // Blank the buffer
  cuda_safe_call(hipMemsetAsync(B, 0, m * n * sizeof(double), stream));

  // Copy A (with upper or lower 0 untouched)
  cusolverDnDlacpy(uplo, m, n, A, ldA, B, ldB, stream);

  hipblasDiagType_t diag = HIPBLAS_DIAG_NON_UNIT;
  const double one      = 1.0;

  auto side = (uplo == HIPBLAS_FILL_MODE_LOWER) ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

  // LOWER: TRMM(A,B) : B = op(A) * B = A^T * B with A triangular (B = C in CUBLAS), HIPBLAS_OP_T, HIPBLAS_SIDE_RIGHT
  // UPPER: TRMM(A,B) : B = B * op(A) = B A^T with A triangular (B = C in CUBLAS), HIPBLAS_OP_T, HIPBLAS_SIDE_RIGHT
  cuda_safe_call(hipblasDtrmm(cublas_handle, side, uplo, HIPBLAS_OP_T, diag, m, n, &one, A, ldA, B, ldB, B, ldB));

  // Copy B=AA^T back into A (with upper or lower 0 untouched)
  cusolverDnDlacpy(uplo, m, n, B, ldB, A, ldA, stream);
}

void DLAAUM(hipblasFillMode_t uplo, matrix<double>& A, int A_row, int A_col)
{
  int NB = A.mb;
  size_t Lwork;
  cublasDnDlaaum_bufferSize(get_cublas_handle(), NB, NB, &Lwork);

  auto d_buffer = ctx.logical_data(shape_of<slice<char>>(Lwork));

  auto t = ctx.task(A.get_handle(A_row, A_col).rw(), d_buffer.write());
  t.set_symbol("DLAAUM");
  t->*[&](auto s, auto sA, auto buffer) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    cublasDnDlaaum(
      h, uplo, sA.extent(0), sA.extent(1), sA.data_handle(), sA.stride(1), (double*) buffer.data_handle(), Lwork);
  };
}

void DGEMM(
  hipblasOperation_t transa,
  hipblasOperation_t transb,
  double alpha,
  matrix<double>& A,
  int A_row,
  int A_col,
  matrix<double>& B,
  int B_row,
  int B_col,
  double beta,
  matrix<double>& C,
  int C_row,
  int C_col)
{
  auto ignored = get_cublas_handle();
  auto t =
    ctx.task(A.get_handle(A_row, A_col).read(), B.get_handle(B_row, B_col).read(), C.get_handle(C_row, C_col).rw());
  t.set_symbol("DGEMM");
  t->*[&](auto s, auto sA, auto sB, auto sC) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    int k = (transa == HIPBLAS_OP_N) ? sA.extent(1) : sA.extent(0);
    cuda_try(hipblasDgemm(
      h,
      transa,
      transb,
      sC.extent(0),
      sC.extent(1),
      k,
      &alpha,
      sA.data_handle(),
      sA.stride(1),
      sB.data_handle(),
      sB.stride(1),
      &beta,
      sC.data_handle(),
      sC.stride(1)));
  };
}

void DSYMM(
  hipblasSideMode_t side,
  hipblasFillMode_t uplo,
  double alpha,
  matrix<double>& A,
  int A_row,
  int A_col,
  matrix<double>& B,
  int B_row,
  int B_col,
  double beta,
  matrix<double>& C,
  int C_row,
  int C_col)
{
  auto ignored = get_cublas_handle();
  auto t =
    ctx.task(A.get_handle(A_row, A_col).read(), B.get_handle(B_row, B_col).read(), C.get_handle(C_row, C_col).rw());
  t.set_symbol("DSYMM");
  t->*[&](auto s, auto sA, auto sB, auto sC) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    cuda_try(hipblasDsymm(
      h,
      side,
      uplo,
      sC.extent(0),
      sC.extent(1),
      &alpha,
      sA.data_handle(),
      sA.stride(1),
      sB.data_handle(),
      sB.stride(1),
      &beta,
      sC.data_handle(),
      sC.stride(1)));
  };
}

void DSYRK(
  hipblasFillMode_t uplo,
  hipblasOperation_t trans,
  double alpha,
  matrix<double>& A,
  int A_row,
  int A_col,
  double beta,
  matrix<double>& C,
  int C_row,
  int C_col)
{
  auto ignored = get_cublas_handle();
  auto t       = ctx.task(A.get_handle(A_row, A_col).read(), C.get_handle(C_row, C_col).rw());
  t.set_symbol("DSYRK");
  t->*[&](auto s, auto sA, auto sC) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    // number of rows of matrix op(A) and C
    int n = sC.extent(0);

    // number of columns of matrix op(A)
    int k = (trans == HIPBLAS_OP_N) ? sA.extent(1) : sA.extent(0);

    cuda_try(
      hipblasDsyrk(h, uplo, trans, n, k, &alpha, sA.data_handle(), sA.stride(1), &beta, sC.data_handle(), sC.stride(1)));
  };
}

void DTRSM(
  hipblasSideMode_t side,
  hipblasFillMode_t uplo,
  hipblasOperation_t transa,
  hipblasDiagType_t diag,
  double alpha,
  matrix<double>& A,
  int A_row,
  int A_col,
  matrix<double>& B,
  int B_row,
  int B_col)
{
  auto ignored = get_cublas_handle();
  auto t       = ctx.task(A.get_handle(A_row, A_col).read(), B.get_handle(B_row, B_col).rw());
  t.set_symbol("DTRSM");
  t->*[&](auto s, auto sA, auto sB) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    cuda_try(hipblasDtrsm(
      h,
      side,
      uplo,
      transa,
      diag,
      sB.extent(0),
      sB.extent(1),
      &alpha,
      sA.data_handle(),
      sA.stride(1),
      sB.data_handle(),
      sB.stride(1)));
  };
}

void DTRMM(
  hipblasSideMode_t side,
  hipblasFillMode_t uplo,
  hipblasOperation_t transa,
  hipblasDiagType_t diag,
  double alpha,
  matrix<double>& A,
  int A_row,
  int A_col,
  matrix<double>& B,
  int B_row,
  int B_col)
{
  auto ignored = get_cublas_handle();
  auto t       = ctx.task(A.get_handle(A_row, A_col).read(), B.get_handle(B_row, B_col).rw());
  t.set_symbol("DTRMM");
  t->*[&](auto s, auto sA, auto sB) {
    auto& h = get_cublas_handle();
    cuda_try(hipblasSetStream(h, s));

    // Note : CUBLAS DTRMM implementation is out of place but supports in place by using the same buffer B and C
    cuda_try(hipblasDtrmm(
      get_cublas_handle(),
      side,
      uplo,
      transa,
      diag,
      sB.extent(0),
      sB.extent(1),
      &alpha,
      sA.data_handle(),
      sA.stride(1),
      sB.data_handle(),
      sB.stride(1),
      sB.data_handle(),
      sB.stride(1) /* same as B*/));
  };
}

void PDNRM2_HOST(matrix<double>* A, double* result)
{
#ifdef HAVE_DOT
  ctx.get_dot()->set_current_color("red");
#endif

  for (int rowb = 0; rowb < A->mt; rowb++)
  {
    for (int colb = 0; colb < A->nt; colb++)
    {
      ctx.host_launch(A->get_handle(rowb, colb).read())->*[=](auto sA) {
        double res2 = 0.0;
        for (size_t col = 0; col < sA.extent(1); col++)
        {
          for (size_t row = 0; row < sA.extent(0); row++)
          {
            double v = sA(row, col);
            res2 += v * v;
          }
        }
        *result += res2;
      };
    }
  }
}

void PDPOTRF(matrix<double>& A)
{
#ifdef HAVE_DOT
  ctx.get_dot()->set_current_color("yellow");
#endif

  assert(A.m == A.n);
  assert(A.mt == A.nt);

  int NBLOCKS = A.mt;
  assert(A.mb == A.nb);

  nvtxRangePushA("SUBMIT_PDPOTRF");
  for (int K = 0; K < NBLOCKS; K++)
  {
    cuda_try(hipSetDevice(A.get_preferred_devid(K, K)));
    DPOTRF(HIPBLAS_FILL_MODE_LOWER, A, K, K);

    for (int row = K + 1; row < NBLOCKS; row++)
    {
      cuda_try(hipSetDevice(A.get_preferred_devid(row, K)));
      DTRSM(HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 1.0, A, K, K, A, row, K);

      for (int col = K + 1; col < row; col++)
      {
        cuda_try(hipSetDevice(A.get_preferred_devid(row, col)));
        DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_T, -1.0, A, row, K, A, col, K, 1.0, A, row, col);
      }

      cuda_try(hipSetDevice(A.get_preferred_devid(row, row)));
      DSYRK(HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, -1.0, A, row, K, 1.0, A, row, row);
    }
  }

  nvtxRangePop();
}

// Algorithm from PLASMA
void PDTRSM(hipblasSideMode_t side,
            hipblasFillMode_t uplo,
            hipblasOperation_t trans,
            hipblasDiagType_t diag,
            double alpha,
            matrix<double>& A,
            matrix<double>& B)
{
  //    std::cout << "[PDTRSM] START B MT " << B.mt << " NT " << B.nt << std::endl;

  nvtxRangePushA("SUBMIT_PDTRSM");

  if (side == HIPBLAS_SIDE_LEFT)
  {
    if (uplo == HIPBLAS_FILL_MODE_UPPER)
    {
      // TODO
      abort();
    }
    else
    {
      //===========================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_LOWER / HIPBLAS_OP_N
      //===========================================
      if (trans == HIPBLAS_OP_N)
      {
        for (int k = 0; k < B.mt; k++)
        {
          double lalpha = k == 0 ? alpha : 1.0;
          for (int n = 0; n < B.nt; n++)
          {
            cuda_try(hipSetDevice(A.get_preferred_devid(k, k)));
            DTRSM(side, uplo, trans, diag, lalpha, A, k, k, B, k, n);
          }
          for (int m = k + 1; m < B.mt; m++)
          {
            for (int n = 0; n < B.nt; n++)
            {
              cuda_try(hipSetDevice(A.get_preferred_devid(m, k)));
              DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, -1.0, A, m, k, B, k, n, lalpha, B, m, n);
            }
          }
        }
      }
      //================================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_LOWER / CUBLAS_OP_[C|T]
      //================================================
      else
      {
        for (int k = 0; k < B.mt; k++)
        {
          double lalpha = k == 0 ? alpha : 1.0;
          for (int n = 0; n < B.nt; n++)
          {
            cuda_try(hipSetDevice(A.get_preferred_devid(B.mt - k - 1, B.mt - k - 1)));
            DTRSM(side, uplo, trans, diag, lalpha, A, B.mt - k - 1, B.mt - k - 1, B, B.mt - k - 1, n);
          }
          for (int m = k + 1; m < B.mt; m++)
          {
            for (int n = 0; n < B.nt; n++)
            {
              cuda_try(hipSetDevice(A.get_preferred_devid(B.mt - k - 1, B.mt - 1 - m)));
              DGEMM(
                trans, HIPBLAS_OP_N, -1.0, A, B.mt - k - 1, B.mt - 1 - m, B, B.mt - k - 1, n, lalpha, B, B.mt - 1 - m, n);
            }
          }
        }
      }
    }
  }
  else
  {
    // TODO
    abort();
  }
  //    std::cout << "[PDTRSM] END" << std::endl;

  nvtxRangePop();
}

void PDPOTRS(matrix<double>& A, matrix<double>& B, hipblasFillMode_t uplo)
{
  nvtxRangePushA("SUBMIT_PDPOTRS");

#ifdef HAVE_DOT
  ctx.get_dot()->set_current_color("green");
#endif

  //    std::cout << "[PDPOTRS] START" << std::endl;
  // Call the parallel functions.
  PDTRSM(
    HIPBLAS_SIDE_LEFT, uplo, uplo == HIPBLAS_FILL_MODE_UPPER ? HIPBLAS_OP_T : HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 1.0, A, B);

#ifdef HAVE_DOT
  ctx.get_dot()->set_current_color("darkgreen");
#endif

  PDTRSM(
    HIPBLAS_SIDE_LEFT, uplo, uplo == HIPBLAS_FILL_MODE_UPPER ? HIPBLAS_OP_N : HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 1.0, A, B);
  //    std::cout << "[PDPOTRS] END" << std::endl;

  nvtxRangePop();
}

/***************************************************************************/ /**
                                                                               * Parallel tile matrix-matrix
                                                                               *multiplication.
                                                                               * @see plasma_omp_dgemm
                                                                               ******************************************************************************/
void PDGEMM(hipblasOperation_t transa,
            hipblasOperation_t transb,
            double alpha,
            matrix<double>& A,
            matrix<double>& B,
            double beta,
            matrix<double>& C)
{
#ifdef HAVE_DOT
  reserved::dot::set_current_color("blue");
#endif

  for (int m = 0; m < C.mt; m++)
  {
    for (int n = 0; n < C.nt; n++)
    {
      cuda_try(hipSetDevice(C.get_preferred_devid(m, n)));

      //=========================================
      // alpha*A*B does not contribute; scale C
      //=========================================
      int inner_k = transa == HIPBLAS_OP_N ? A.n : A.m;
      if (alpha == 0.0 || inner_k == 0)
      {
        DGEMM(transa, transb, alpha, A, 0, 0, B, 0, 0, beta, C, m, n);
      }
      else if (transa == HIPBLAS_OP_N)
      {
        //================================
        // HIPBLAS_OP_N / HIPBLAS_OP_N
        //================================
        if (transb == HIPBLAS_OP_N)
        {
          assert(A.nt == B.mt);
          for (int k = 0; k < A.nt; k++)
          {
            double zbeta = k == 0 ? beta : 1.0;
            DGEMM(transa, transb, alpha, A, m, k, B, k, n, zbeta, C, m, n);
          }
        }
        //=====================================
        // HIPBLAS_OP_N / HIPBLAS_OP_T
        //=====================================
        else
        {
          for (int k = 0; k < A.nt; k++)
          {
            double zbeta = k == 0 ? beta : 1.0;
            DGEMM(transa, transb, alpha, A, m, k, B, n, k, zbeta, C, m, n);
          }
        }
      }
      else
      {
        //=====================================
        // HIPBLAS_OP_T / HIPBLAS_OP_N
        //=====================================
        if (transb == HIPBLAS_OP_N)
        {
          for (int k = 0; k < A.mt; k++)
          {
            double zbeta = k == 0 ? beta : 1.0;
            DGEMM(transa, transb, alpha, A, k, m, B, k, n, zbeta, C, m, n);
          }
        }
        //==========================================
        // HIPBLAS_OP_T / HIPBLAS_OP_T
        //==========================================
        else
        {
          for (int k = 0; k < A.mt; k++)
          {
            double zbeta = k == 0 ? beta : 1.0;
            DGEMM(transa, transb, alpha, A, k, m, B, n, k, zbeta, C, m, n);
          }
        }
      }
    }
  }
}

/*
 * Algorithm taken from the PLASMA library
 */
// We assume a lower triangular matrix (uplo == HIPBLAS_FILL_MODE_LOWER)
void PDTRTRI(matrix<double>& A, hipblasFillMode_t uplo, hipblasDiagType_t diag)
{
  assert(uplo == HIPBLAS_FILL_MODE_LOWER);

  nvtxRangePushA("SUBMIT_PDTRTRI");

  for (int k = 0; k < A.nt; k++)
  {
    for (int m = k + 1; m < A.mt; m++)
    {
      cuda_try(hipSetDevice(A.get_preferred_devid(m, k)));
      DTRSM(HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, diag, -1.0, A, k, k, A, m, k);
    }
    for (int m = k + 1; m < A.mt; m++)
    {
      for (int n = 0; n < k; n++)
      {
        cuda_try(hipSetDevice(A.get_preferred_devid(m, n)));
        DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, 1.0, A, m, k, A, k, n, 1.0, A, m, n);
      }
    }
    for (int n = 0; n < k; n++)
    {
      cuda_try(hipSetDevice(A.get_preferred_devid(k, n)));
      DTRSM(HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, diag, 1.0, A, k, k, A, k, n);
    }

    // DTRTRI(...)
    cuda_try(hipSetDevice(A.get_preferred_devid(k, k)));
    DTRTRI(uplo, diag, A, k, k);
  }

  nvtxRangePop();
}

/*
 * Algorithm taken from the PLASMA library
 */
// We assume a lower triangular matrix (uplo == HIPBLAS_FILL_MODE_LOWER)
void PDLAUUM(matrix<double>& A, hipblasFillMode_t uplo)
{
  assert(uplo == HIPBLAS_FILL_MODE_LOWER);

  nvtxRangePushA("SUBMIT_PDLAUUM");

  for (int k = 0; k < A.mt; k++)
  {
    for (int n = 0; n < k; n++)
    {
      cuda_try(hipSetDevice(A.get_preferred_devid(n, n)));
      DSYRK(HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, 1.0, A, k, n, 1.0, A, n, n);

      for (int m = n + 1; m < k; m++)
      {
        cuda_try(hipSetDevice(A.get_preferred_devid(m, n)));
        DGEMM(HIPBLAS_OP_T, HIPBLAS_OP_N, 1.0, A, k, m, A, k, n, 1.0, A, m, n);
      }
    }
    for (int n = 0; n < k; n++)
    {
      cuda_try(hipSetDevice(A.get_preferred_devid(k, n)));
      DTRMM(HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, 1.0, A, k, k, A, k, n);
    }

    // LAAUM (Akk RW) (compute Akk^T * Akk)
    cuda_try(hipSetDevice(A.get_preferred_devid(k, k)));
    DLAAUM(uplo, A, k, k);
  }

  nvtxRangePop();
}

void PDSYMM(hipblasSideMode_t side,
            hipblasFillMode_t uplo,
            double alpha,
            matrix<double>& A,
            matrix<double>& B,
            double beta,
            matrix<double>& C)
{
  int k, m, n;
  double zbeta;
  double zone = (double) 1.0;

  for (m = 0; m < C.mt; m++)
  {
    for (n = 0; n < C.nt; n++)
    {
      cuda_try(hipSetDevice(C.get_preferred_devid(m, n)));
      /*
       *  HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_LOWER
       */
      if (side == HIPBLAS_SIDE_LEFT)
      {
        if (uplo == HIPBLAS_FILL_MODE_LOWER)
        {
          for (k = 0; k < C.mt; k++)
          {
            zbeta = k == 0 ? beta : zone;
            if (k < m)
            {
              DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, A, m, k, B, k, n, zbeta, C, m, n);
            }
            else
            {
              if (k == m)
              {
                DSYMM(side, uplo, alpha, A, k, k, B, k, n, zbeta, C, m, n);
              }
              else
              {
                DGEMM(HIPBLAS_OP_T, HIPBLAS_OP_N, alpha, A, k, m, B, k, n, zbeta, C, m, n);
              }
            }
          }
        }
        /*
         *  HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_UPPER
         */
        else
        {
          for (k = 0; k < C.mt; k++)
          {
            zbeta = k == 0 ? beta : zone;
            if (k < m)
            {
              DGEMM(HIPBLAS_OP_T, HIPBLAS_OP_N, alpha, A, k, m, B, k, n, zbeta, C, m, n);
            }
            else
            {
              if (k == m)
              {
                DSYMM(side, uplo, alpha, A, k, k, B, k, n, zbeta, C, m, n);
              }
              else
              {
                DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, A, m, k, B, k, n, zbeta, C, m, n);
              }
            }
          }
        }
      }
      /*
       *  HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_LOWER
       */
      else
      {
        if (uplo == HIPBLAS_FILL_MODE_LOWER)
        {
          for (k = 0; k < C.nt; k++)
          {
            zbeta = k == 0 ? beta : zone;
            if (k < n)
            {
              DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_T, alpha, B, m, k, A, n, k, zbeta, C, m, n);
            }
            else
            {
              if (k == n)
              {
                DSYMM(side, uplo, alpha, A, k, k, B, m, k, zbeta, C, m, n);
              }
              else
              {
                DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, B, m, k, A, k, n, zbeta, C, m, n);
              }
            }
          }
        }
        /*
         *  HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_UPPER
         */
        else
        {
          for (k = 0; k < C.nt; k++)
          {
            zbeta = k == 0 ? beta : zone;
            if (k < n)
            {
              DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, alpha, B, m, k, A, k, n, zbeta, C, m, n);
            }
            else
            {
              if (k == n)
              {
                DSYMM(side, uplo, alpha, A, k, k, B, m, k, zbeta, C, m, n);
              }
              else
              {
                DGEMM(HIPBLAS_OP_N, HIPBLAS_OP_T, alpha, B, m, k, A, n, k, zbeta, C, m, n);
              }
            }
          }
        }
      }
    }
  }
}

void PDTRMM(hipblasSideMode_t side,
            hipblasFillMode_t uplo,
            hipblasOperation_t trans,
            hipblasDiagType_t diag,
            double alpha,
            matrix<double>& A,
            matrix<double>& B)
{
  if (side == HIPBLAS_SIDE_LEFT)
  {
    if (uplo == HIPBLAS_FILL_MODE_UPPER)
    {
      //===========================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_UPPER / HIPBLAS_OP_N
      //===========================================
      if (trans == HIPBLAS_OP_N)
      {
        for (int m = 0; m < B.mt; m++)
        {
          for (int n = 0; n < B.nt; n++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, m, m, B, m, n);

            for (int k = m + 1; k < A.mt; k++)
            {
              DGEMM(trans, HIPBLAS_OP_N, alpha, A, m, k, B, k, n, 1.0, B, m, n);
            }
          }
        }
      }
      //================================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_UPPER / HIPBLAS_OP_T
      //================================================
      else
      {
        for (int m = B.mt - 1; m > -1; m--)
        {
          for (int n = 0; n < B.nt; n++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, m, m, B, m, n);

            for (int k = 0; k < m; k++)
            {
              DGEMM(trans, HIPBLAS_OP_N, alpha, A, k, m, B, k, n, 1.0, B, m, n);
            }
          }
        }
      }
    }
    else
    {
      //===========================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_LOWER / HIPBLAS_OP_N
      //===========================================
      if (trans == HIPBLAS_OP_N)
      {
        for (int m = B.mt - 1; m > -1; m--)
        {
          for (int n = 0; n < B.nt; n++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, m, m, B, m, n);

            for (int k = 0; k < m; k++)
            {
              DGEMM(trans, HIPBLAS_OP_N, alpha, A, m, k, B, k, n, 1.0, B, m, n);
            }
          }
        }
      }
      //================================================
      // HIPBLAS_SIDE_LEFT / HIPBLAS_FILL_MODE_LOWER / HIPBLAS_OP_T
      //================================================
      else
      {
        for (int m = 0; m < B.mt; m++)
        {
          for (int n = 0; n < B.nt; n++)
          {
            DTRMM(side, uplo, trans, diag, alpha, A, m, m, B, m, n);

            for (int k = m + 1; k < A.mt; k++)
            {
              DGEMM(trans, HIPBLAS_OP_N, alpha, A, k, m, B, k, n, 1.0, B, m, n);
            }
          }
        }
      }
    }
  }
  else
  {
    if (uplo == HIPBLAS_FILL_MODE_UPPER)
    {
      //============================================
      // HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_UPPER / HIPBLAS_OP_N
      //============================================
      if (trans == HIPBLAS_OP_N)
      {
        for (int n = B.nt - 1; n > -1; n--)
        {
          for (int m = 0; m < B.mt; m++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, n, n, B, m, n);

            for (int k = 0; k < n; k++)
            {
              DGEMM(HIPBLAS_OP_N, trans, alpha, B, m, k, A, k, n, 1.0, B, m, n);
            }
          }
        }
      }
      //=================================================
      // HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_UPPER / Plasma[_Conj]Trans
      //=================================================
      else
      {
        for (int n = 0; n < B.nt; n++)
        {
          for (int m = 0; m < B.mt; m++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, n, n, B, m, n);

            for (int k = n + 1; k < A.mt; k++)
            {
              DGEMM(HIPBLAS_OP_N, trans, alpha, B, m, k, A, n, k, 1.0, B, m, n);
            }
          }
        }
      }
    }
    else
    {
      //============================================
      // HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_LOWER / HIPBLAS_OP_N
      //============================================
      if (trans == HIPBLAS_OP_N)
      {
        for (int n = 0; n < B.nt; n++)
        {
          for (int m = 0; m < B.mt; m++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, n, n, B, m, n);

            for (int k = n + 1; k < A.mt; k++)
            {
              DGEMM(HIPBLAS_OP_N, trans, alpha, B, m, k, A, k, n, 1.0, B, m, n);
            }
          }
        }
      }
      //=================================================
      // HIPBLAS_SIDE_RIGHT / HIPBLAS_FILL_MODE_LOWER / Plasma[_Conj]Trans
      //=================================================
      else
      {
        for (int n = B.nt - 1; n > -1; n--)
        {
          for (int m = 0; m < B.mt; m++)
          {
            cuda_try(hipSetDevice(B.get_preferred_devid(m, n)));

            DTRMM(side, uplo, trans, diag, alpha, A, n, n, B, m, n);

            for (int k = 0; k < n; k++)
            {
              DGEMM(HIPBLAS_OP_N, trans, alpha, B, m, k, A, n, k, 1.0, B, m, n);
            }
          }
        }
      }
    }
  }
}

// Taken from Chameleon (INRIA)
// All the formula are reported in the LAPACK Lawn 41:
//     http://www.netlib.org/lapack/lawns/lawn41.ps
#define FMULS_POTRI(__n) ((double) (__n) * ((2. / 3.) + (double) (__n) * ((1. / 3.) * (double) (__n) + 1.)))
#define FADDS_POTRI(__n) ((double) (__n) * ((1. / 6.) + (double) (__n) * ((1. / 3.) * (double) (__n) - 0.5)))
double flops_dpotri(double __n)
{
  double flops = (FMULS_POTRI((__n)) + FADDS_POTRI((__n)));
  return flops;
}

void run(int N, int NB)
{
  // Use pools of preallocated blocks
  auto fixed_alloc = block_allocator<fixed_size_allocator>(ctx, NB * NB * sizeof(double));
  ctx.set_allocator(fixed_alloc);

  // Set up CUBLAS and CUSOLVER
  int ndevs;
  cuda_try(hipGetDeviceCount(&ndevs));

  for (size_t d = 0; d < ndevs; d++)
  {
    auto ldummy = ctx.logical_data(shape_of<slice<char>>(1));
    ctx.task(exec_place::device(d), ldummy.write())->*[](hipStream_t, auto) {
      get_cublas_handle();
      get_cusolver_handle();
    };

    ctx.task(exec_place::host, ldummy.write(data_place::managed))->*[](hipStream_t, auto) {};
  }

  cuda_try(hipSetDevice(0));

  hipStream_t timing_stream;
  cuda_try(hipStreamCreate(&timing_stream));

  matrix<double> A(N, N, NB, NB, true, "A");
  matrix<double> Aref(N, N, NB, NB, false, "Aref");

  // (Hilbert matrix + 2*N*Id) to have a diagonal dominant matrix
  auto hilbert = [=] _CCCL_HOST_DEVICE(size_t row, size_t col) {
    return 1.0 / (col + row + 1.0) + 2.0 * N * (col == row);
  };

  Aref.fill(hilbert);
  A.fill(hilbert);

  /* Right-hand side */
  matrix<double> B_potrs(N, 1, NB, 1, false, "B");
  matrix<double> Bref_potrs(N, 1, NB, 1, false, "Bref");

  auto rhs_vals = [] _CCCL_HOST_DEVICE(size_t row, size_t /*unused*/) {
    return 1.0 * (row + 1);
  };

  B_potrs.fill(rhs_vals);
  Bref_potrs.fill(rhs_vals);

  int check_result = 1;
  if (getenv("CHECK_RESULT"))
  {
    check_result = atoi(getenv("CHECK_RESULT"));
  }

  int check_result_potrs = check_result;
  if (getenv("CHECK_RESULT_POTRS"))
  {
    check_result_potrs = atoi(getenv("CHECK_RESULT_POTRS"));
  }

  //    // Compute ||Bref||
  double Bref_nrm2 = 0.0;
  double res_nrm2  = 0.0;

  if (check_result_potrs)
  {
    PDNRM2_HOST(&Bref_potrs, &Bref_nrm2);
  }

  hipEvent_t startEvent, stopEvent;

  cuda_safe_call(hipSetDevice(0));
  cuda_safe_call(hipStreamSynchronize(ctx.task_fence()));
  cuda_safe_call(hipEventCreate(&startEvent));
  cuda_safe_call(hipEventCreate(&stopEvent));
  cuda_safe_call(hipEventRecord(startEvent, ctx.task_fence()));

  ctx.get_dot()->set_current_color("green");
  PDPOTRF(A);
  ctx.get_dot()->set_current_color("white");

  /*
   *  POTRS
   */

  if (check_result_potrs)
  {
    // Solve AX = B and put the result in B
    PDPOTRS(A, B_potrs, HIPBLAS_FILL_MODE_LOWER);

    // Compute (AX - B)
    // Bref = (Aref*B - Bref)
    PDGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, 1.0, Aref, B_potrs, -1.0, Bref_potrs);

    // Compute ||AX - B|| = ||Bref||
    PDNRM2_HOST(&Bref_potrs, &res_nrm2);
  }

  /*
   *  POTRI
   */
  /* PDPOTRI = PDTRTRI + PDLAUUM */

  // PDTRTRI : La^-1 (invert A)
  //    fprintf(stderr, "A=La before POTRI\n");
  //    A.print();

  ctx.get_dot()->set_current_color("yellow");
  PDTRTRI(A, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_DIAG_NON_UNIT);
  ctx.get_dot()->set_current_color("white");

  //    fprintf(stderr, "A=La^-1 after POTRI\n");
  //    A.print();

  // Computes the lower part of A^tA (La^-t La^-1)
  ctx.get_dot()->set_current_color("blue");
  PDLAUUM(A, HIPBLAS_FILL_MODE_LOWER);
  ctx.get_dot()->set_current_color("white");

  double b_nrm2_potri   = 0.0;
  double res_nrm2_potri = 0.0;

  if (check_result)
  {
    /* Right-hand side */
    matrix<double> B_potri(N, 1, NB, 1, false, "B_potri");
    matrix<double> Bref_potri(N, 1, NB, 1, false, "Bref_potri");

    // auto rhs_vals = [](matrix<double>& mat, int row, int col) { return 1.0 * (row + 1); };
    B_potri.fill(rhs_vals);
    Bref_potri.fill(rhs_vals);

    // AX = B, X = A^-1 B
    // LLt X = B, X = (LLt)^-1 B = L^-t L^-1 B
    // Compute Bref_potri = (A^-1 B - B)
    PDNRM2_HOST(&Bref_potri, &b_nrm2_potri);

    // B = (A^-1)*B (A triangular lower, B_potri full)
    //    fprintf(stderr, "B_potri before PDTRMM\n");
    //    B_potri.print();
    //
    //    fprintf(stderr, "A before PDTRMM\n");
    //    A.print();

    // B_tmp = 0 (to avoid NaN*0.0)
    matrix<double> B_tmp(N, 1, NB, 1, false, "B_tmp");
    auto zero_vals = [] _CCCL_HOST_DEVICE(size_t /* unused */, size_t /*unused*/) {
      return 0.0;
    };
    B_tmp.fill(zero_vals);

    // B_tmp = A * B_potri + 0*B_tmp
    PDSYMM(HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, 1.0, A, B_potri, 0.0, B_tmp);

    //    fprintf(stderr, "B_potri after PDTRMM\n");
    //    B_potri.print();

    // res = A X - B
    PDGEMM(HIPBLAS_OP_N, HIPBLAS_OP_N, 1.0, Aref, B_tmp, -1.0, Bref_potri);

    //    fprintf(stderr, "Bref_potri after PDGEMM\n");
    //    Bref_potri.print();

    // Compute residual
    PDNRM2_HOST(&Bref_potri, &res_nrm2_potri);
  }

  cuda_safe_call(hipSetDevice(0));
  cuda_safe_call(hipEventRecord(stopEvent, ctx.task_fence()));

  ctx.finalize();

  if (check_result_potrs)
  {
    double residual = sqrt(res_nrm2) / sqrt(Bref_nrm2);
    // std::cout << "[POTRS] ||AX - B|| : " << sqrt(res_nrm2) << std::endl;
    // std::cout << "[POTRS] ||B|| : " << sqrt(Bref_nrm2) << std::endl;
    // std::cout << "[POTRS] RESIDUAL (||AX - B||/||B||) : " << residual << std::endl;
    EXPECT(residual < 0.01);
  }

  if (check_result)
  {
    double residual_potri = sqrt(res_nrm2_potri) / sqrt(b_nrm2_potri);
    // std::cout << "[POTRI] RESIDUAL ||A * ((A^-1)B) - B|| : " << sqrt(res_nrm2_potri) << std::endl;
    // std::cout << "[POTRI] RESIDUAL ||B|| : " << sqrt(b_nrm2_potri) << std::endl;
    // std::cout << "[POTRI] RESIDUAL (||A * ((A^-1)B) - B||/||B||) : " << residual_potri << std::endl;
    EXPECT(residual_potri < 0.0001);
  }

  //    // Compute Aref * A^-1 in Aref (A^-1 is lower triangular)
  //    PDTRMM(HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 1.0, A, Aref);

  //    // This should be almost identity
  //    Aref.print();

#if 0

    std::cout << "Print A^-1 after PDLAUUM : " << std::endl;
    A.print();

    std::cout << "RES after AX - B POTRI : " << std::endl;
    Bref_potri.print();

    // This should be almost identity
    Aref.print();
#endif

  float milliseconds;
  cuda_safe_call(hipEventElapsedTime(&milliseconds, startEvent, stopEvent));

  double gflops = flops_dpotri((double) N) / (1000000000.0);
  std::cout
    << "[PDPOTRI] ELAPSED: " << milliseconds << " ms, GFLOPS: " << gflops / (milliseconds / 1000.0) << std::endl;
}

int main(int argc, char** argv)
{
  int N  = 1024;
  int NB = 128;

  if (argc > 1)
  {
    N = atoi(argv[1]);
  }

  if (argc > 2)
  {
    NB = atoi(argv[2]);
  }

  assert(N % NB == 0);

  run(N, NB);
}
