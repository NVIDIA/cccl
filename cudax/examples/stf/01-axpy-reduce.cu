#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief An AXPY kernel implemented using the parallel_for construct
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

class sum {
public:
};

int main()
{
  context ctx;
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  double alpha = 3.14;

  auto lX = ctx.logical_data(X);
  auto lY = ctx.logical_data(Y);

  auto lsum = ctx.logical_data(shape_of<slice<double>>(1));
  auto lsum2 = ctx.logical_data(shape_of<slice<double>>(1));

  /* Compute Y = Y + alpha X */
  //ctx.parallel_for(lY.shape(), lX.read(), lY.rw(), lsum.write(), lsum2.template reduce<sum>())->*[alpha] __device__(size_t i, auto dX, auto dY, auto sum, auto sum2) {
  ctx.parallel_for(lY.shape(), lX.read(), lY.rw(), lsum.write(), lsum2.reduce(sum{}))->*[alpha] __device__(size_t i, auto dX, auto dY, auto sum, auto sum2) {
    dY(i) += alpha * dX(i);
    // sum(0) += dY(i);
    atomicAdd(sum.data_handle(), dY(i));
  };

  ctx.host_launch(lsum.read())->*[](auto sum) {
      fprintf(stderr, "SUM ... %lf\n", sum(0));
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    assert(fabs(Y[i] - (Y0(i) + alpha * X0(i))) < 0.0001);
    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
