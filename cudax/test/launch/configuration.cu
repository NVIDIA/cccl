#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

// Test translation of launch function arguments to hipLaunchConfig_t sent to cudaLaunchKernelEx internally
// We replace cudaLaunchKernelEx with a test function here through a macro to intercept the hipLaunchConfig_t
#define cudaLaunchKernelEx cudaLaunchKernelExTestReplacement
#include <cuda/experimental/launch.cuh>
#undef cudaLaunchKernelEx

#include <host_device.cuh>

static hipLaunchConfig_t expectedConfig;
static bool replacementCalled = false;

template <typename... ExpTypes, typename... ActTypes>
hipError_t
cudaLaunchKernelExTestReplacement(const hipLaunchConfig_t* config, void (*kernel)(ExpTypes...), ActTypes&&... args)
{
  replacementCalled = true;
  bool has_cluster  = false;

  CUDAX_CHECK(expectedConfig.numAttrs == config->numAttrs);
  CUDAX_CHECK(expectedConfig.blockDim == config->blockDim);
  CUDAX_CHECK(expectedConfig.gridDim == config->gridDim);
  CUDAX_CHECK(expectedConfig.stream == config->stream);
  CUDAX_CHECK(expectedConfig.dynamicSmemBytes == config->dynamicSmemBytes);

  for (unsigned int i = 0; i < expectedConfig.numAttrs; ++i)
  {
    auto& expectedAttr = expectedConfig.attrs[i];
    unsigned int j;
    for (j = 0; j < expectedConfig.numAttrs; ++j)
    {
      auto& actualAttr = config->attrs[j];
      if (expectedAttr.id == actualAttr.id)
      {
        switch (expectedAttr.id)
        {
          case cudaLaunchAttributeClusterDimension:
            CUDAX_CHECK(expectedAttr.val.clusterDim.x == actualAttr.val.clusterDim.x);
            CUDAX_CHECK(expectedAttr.val.clusterDim.y == actualAttr.val.clusterDim.y);
            CUDAX_CHECK(expectedAttr.val.clusterDim.z == actualAttr.val.clusterDim.z);
            has_cluster = true;
            break;
          case hipLaunchAttributeCooperative:
            CUDAX_CHECK(expectedAttr.val.cooperative == actualAttr.val.cooperative);
            break;
          case hipLaunchAttributePriority:
            CUDAX_CHECK(expectedAttr.val.priority == actualAttr.val.priority);
            break;
          default:
            CUDAX_CHECK(false);
            break;
        }
        break;
      }
    }
    INFO("Searched attribute is " << expectedAttr.id);
    CUDAX_CHECK(j != expectedConfig.numAttrs);
  }

  if (!has_cluster || !skip_device_exec(arch_filter<std::less<int>, 90>))
  {
    return cudaLaunchKernelEx(config, kernel, cuda::std::forward<ActTypes>(args)...);
  }
  else
  {
    return hipSuccess;
  }
}

__global__ void empty_kernel(int i) {}

template <bool HasCluster>
auto make_test_dims(const dim3& grid_dims, const dim3& block_dims, const dim3& cluster_dims = dim3())
{
  if constexpr (HasCluster)
  {
    return cudax::make_hierarchy(
      cudax::grid_dims(grid_dims), cudax::cluster_dims(cluster_dims), cudax::block_dims(block_dims));
  }
  else
  {
    return cudax::make_hierarchy(cudax::grid_dims(grid_dims), cudax::block_dims(block_dims));
  }
}

auto add_cluster(const dim3& cluster_dims, hipLaunchAttribute& attr)
{
  attr.id             = cudaLaunchAttributeClusterDimension;
  attr.val.clusterDim = {cluster_dims.x, cluster_dims.y, cluster_dims.z};
}

template <bool HasCluster, typename... Dims>
auto configuration_test(
  ::cuda::stream_ref stream, const dim3& grid_dims, const dim3& block_dims, const dim3& cluster_dims = dim3())
{
  auto dims             = make_test_dims<HasCluster>(grid_dims, block_dims, cluster_dims);
  expectedConfig        = {};
  expectedConfig.stream = stream.get();
  if constexpr (HasCluster)
  {
    expectedConfig.gridDim =
      dim3(grid_dims.x * cluster_dims.x, grid_dims.y * cluster_dims.y, grid_dims.z * cluster_dims.z);
  }
  else
  {
    expectedConfig.gridDim = grid_dims;
  }
  expectedConfig.blockDim = block_dims;

  SECTION("Simple cooperative launch")
  {
    hipLaunchAttribute attrs[2];
    auto config                             = cudax::make_config(dims, cudax::cooperative_launch());
    expectedConfig.numAttrs                 = 1 + HasCluster;
    expectedConfig.attrs                    = &attrs[0];
    expectedConfig.attrs[0].id              = hipLaunchAttributeCooperative;
    expectedConfig.attrs[0].val.cooperative = 1;
    if constexpr (HasCluster)
    {
      add_cluster(cluster_dims, expectedConfig.attrs[1]);
    }
    cudax::launch(stream, config, empty_kernel, 0);
  }

  SECTION("Priority and dynamic smem")
  {
    hipLaunchAttribute attrs[2];
    const int priority = 42;
    const int num_ints = 128;
    auto config =
      cudax::make_config(dims, cudax::launch_priority(priority), cudax::dynamic_shared_memory<int>(num_ints));
    expectedConfig.dynamicSmemBytes      = num_ints * sizeof(int);
    expectedConfig.numAttrs              = 1 + HasCluster;
    expectedConfig.attrs                 = &attrs[0];
    expectedConfig.attrs[0].id           = hipLaunchAttributePriority;
    expectedConfig.attrs[0].val.priority = priority;
    if constexpr (HasCluster)
    {
      add_cluster(cluster_dims, expectedConfig.attrs[1]);
    }
    cudax::launch(stream, config, empty_kernel, 0);
  }

  SECTION("Large dynamic smem")
  {
    // Exceed the default 48kB of shared to check if its properly handled
    // TODO move to launch option (available since CUDA 12.4)
    struct S
    {
      int arr[13 * 1024];
    };
    hipLaunchAttribute attrs[1];
    auto config                     = cudax::make_config(dims, cudax::dynamic_shared_memory<S, 1, true>());
    expectedConfig.dynamicSmemBytes = sizeof(S);
    expectedConfig.numAttrs         = HasCluster;
    expectedConfig.attrs            = &attrs[0];
    if constexpr (HasCluster)
    {
      add_cluster(cluster_dims, expectedConfig.attrs[0]);
    }
    cudax::launch(stream, config, empty_kernel, 0);
  }
  stream.wait();
}

TEST_CASE("Launch configuration", "[launch]")
{
  hipStream_t stream;
  CUDART(hipStreamCreate(&stream));
  SECTION("No cluster")
  {
    configuration_test<false>(stream, 8, 64);
  }
  SECTION("With cluster")
  {
    configuration_test<true>(stream, 8, 32, 2);
  }

  CUDART(hipStreamDestroy(stream));
  CUDAX_CHECK(replacementCalled);
}

TEST_CASE("Hierarchy construction in config", "[launch]")
{
  auto config = cudax::make_config(cudax::grid_dims<2>(), cudax::cooperative_launch());
  static_assert(config.dims.count(cudax::block) == 2);

  auto config_larger = cudax::make_config(cudax::grid_dims<2>(), cudax::block_dims(256), cudax::cooperative_launch());
  CUDAX_REQUIRE(config_larger.dims.count(cudax::thread) == 512);

  auto config_no_options = cudax::make_config(cudax::grid_dims(2), cudax::block_dims<128>());
  CUDAX_REQUIRE(config_no_options.dims.count(cudax::thread) == 256);

  [[maybe_unused]] auto config_no_dims = cudax::make_config(cudax::cooperative_launch());
  static_assert(cuda::std::is_same_v<decltype(config_no_dims.dims), cudax::uninit_t>);
}
