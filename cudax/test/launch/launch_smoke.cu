#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//
#include <cuda/atomic>

#include <cuda/experimental/launch.cuh>

#include <hip/hip_cooperative_groups.h>
#include <host_device.cuh>

__managed__ bool kernel_run_proof = false;

void check_kernel_run(hipStream_t stream)
{
  CUDART(hipStreamSynchronize(stream));
  CUDAX_CHECK(kernel_run_proof);
  kernel_run_proof = false;
}

struct functor_int_argument
{
  __device__ void operator()(int dummy)
  {
    kernel_run_proof = true;
  }
};

template <unsigned int BlockSize>
struct functor_taking_config
{
  template <typename Config>
  __device__ void operator()(Config conf, int grid_size)
  {
    static_assert(conf.dims.static_count(cudax::thread, cudax::block) == BlockSize);
    CUDAX_REQUIRE(conf.dims.count(cudax::block, cudax::grid) == grid_size);
    kernel_run_proof = true;
  }
};

template <unsigned int BlockSize>
struct functor_taking_dims
{
  template <typename Dimensions>
  __device__ void operator()(Dimensions dims, int grid_size)
  {
    static_assert(dims.static_count(cudax::thread, cudax::block) == BlockSize);
    CUDAX_REQUIRE(dims.count(cudax::block, cudax::grid) == grid_size);
    kernel_run_proof = true;
  }
};

__global__ void kernel_int_argument(int dummy)
{
  kernel_run_proof = true;
}

template <typename Config, unsigned int BlockSize>
__global__ void kernel_taking_config(Config conf, int grid_size)
{
  functor_taking_config<BlockSize>()(conf, grid_size);
}

template <typename Dims, unsigned int BlockSize>
__global__ void kernel_taking_dims(Dims dims, int grid_size)
{
  functor_taking_dims<BlockSize>()(dims, grid_size);
}

struct my_dynamic_smem_t
{
  int i;
};

template <typename SmemType>
struct dynamic_smem_single
{
  template <typename Config>
  __device__ void operator()(Config conf)
  {
    auto& dynamic_smem = cudax::dynamic_smem_ref(conf);
    static_assert(::cuda::std::is_same_v<SmemType&, decltype(dynamic_smem)>);
    CUDAX_REQUIRE(__isShared(&dynamic_smem));
    kernel_run_proof = true;
  }
};

template <typename SmemType, size_t Extent>
struct dynamic_smem_span
{
  template <typename Config>
  __device__ void operator()(Config conf, int size)
  {
    auto dynamic_smem = cudax::dynamic_smem_span(conf);
    static_assert(decltype(dynamic_smem)::extent == Extent);
    static_assert(::cuda::std::is_same_v<SmemType&, decltype(dynamic_smem[1])>);
    CUDAX_REQUIRE(dynamic_smem.size() == size);
    CUDAX_REQUIRE(__isShared(&dynamic_smem[1]));
    kernel_run_proof = true;
  }
};

struct launch_transform_to_int_convertible
{
  int value_;

  struct int_convertible
  {
    hipStream_t stream_;
    int value_;

    int_convertible(hipStream_t stream, int value) noexcept
        : stream_(stream)
        , value_(value)
    {
      // Check that the constructor runs before the kernel is launched
      CUDAX_CHECK_FALSE(kernel_run_proof);
    }

    // Immovable to ensure that __launch_transform doesn't copy the returned
    // object
    int_convertible(int_convertible&&) = delete;

    ~int_convertible() noexcept
    {
      // Check that the destructor runs after the kernel is launched
      CUDART(hipStreamSynchronize(stream_));
      CUDAX_CHECK(kernel_run_proof);
    }

    using __as_kernel_arg = int;

    // This is the value that will be passed to the kernel
    explicit operator int() const
    {
      return value_;
    }
  };

  _CCCL_NODISCARD_FRIEND int_convertible
  __cudax_launch_transform(::cuda::stream_ref stream, launch_transform_to_int_convertible self) noexcept
  {
    return int_convertible(stream.get(), self.value_);
  }
};

// Needs a separe function for Windows extended lambda
void launch_smoke_test()
{
  // Use raw stream to make sure it can be implicitly converted on call to launch
  hipStream_t stream;

  CUDART(hipStreamCreate(&stream));
  // Spell out all overloads to make sure they compile, include a check for implicit conversions
  SECTION("Launch overloads")
  {
    const int grid_size      = 4;
    constexpr int block_size = 256;
    auto dimensions          = cudax::make_hierarchy(cudax::grid_dims(grid_size), cudax::block_dims<256>());
    auto config              = cudax::make_config(dimensions);

    SECTION("Not taking dims")
    {
      auto lambda = [&](auto dims_or_conf) {
        const int dummy = 1;
        cudax::launch(stream, dims_or_conf, kernel_int_argument, dummy);
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, kernel_int_argument, 1);
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, kernel_int_argument, launch_transform_to_int_convertible{1});
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, functor_int_argument(), dummy);
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, functor_int_argument(), 1);
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, functor_int_argument(), launch_transform_to_int_convertible{1});
        check_kernel_run(stream);

        cudax::launch(stream, dims_or_conf, kernel_int_argument, 1U);
        check_kernel_run(stream);
        cudax::launch(stream, dims_or_conf, functor_int_argument(), 1U);
        check_kernel_run(stream);
      };
      lambda(config);
      lambda(dimensions);
    }

    SECTION("Config argument")
    {
      auto functor_instance = functor_taking_config<block_size>();
      auto kernel_instance  = kernel_taking_config<decltype(config), block_size>;

      cudax::launch(stream, config, functor_instance, grid_size);
      check_kernel_run(stream);
      cudax::launch(stream, config, functor_instance, ::cuda::std::move(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, config, functor_instance, launch_transform_to_int_convertible{grid_size});
      check_kernel_run(stream);

      cudax::launch(stream, config, kernel_instance, grid_size);
      check_kernel_run(stream);
      cudax::launch(stream, config, kernel_instance, ::cuda::std::move(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, config, kernel_instance, launch_transform_to_int_convertible{grid_size});
      check_kernel_run(stream);

      cudax::launch(stream, config, functor_instance, static_cast<unsigned int>(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, config, kernel_instance, static_cast<unsigned int>(grid_size));
      check_kernel_run(stream);
    }

    SECTION("Dimensions argument")
    {
      auto functor_instance = functor_taking_dims<block_size>();
      auto kernel_instance  = kernel_taking_dims<decltype(dimensions), block_size>;

      cudax::launch(stream, dimensions, functor_instance, grid_size);
      check_kernel_run(stream);
      cudax::launch(stream, dimensions, functor_instance, ::cuda::std::move(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, dimensions, functor_instance, launch_transform_to_int_convertible{grid_size});
      check_kernel_run(stream);

      cudax::launch(stream, dimensions, kernel_instance, grid_size);
      check_kernel_run(stream);
      cudax::launch(stream, dimensions, kernel_instance, ::cuda::std::move(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, dimensions, kernel_instance, launch_transform_to_int_convertible{grid_size});
      check_kernel_run(stream);

      cudax::launch(stream, dimensions, functor_instance, static_cast<unsigned int>(grid_size));
      check_kernel_run(stream);
      cudax::launch(stream, dimensions, kernel_instance, static_cast<unsigned int>(grid_size));
      check_kernel_run(stream);
    }
  }

  SECTION("Lambda")
  {
    cudax::launch(stream, cudax::block_dims<256>() & cudax::grid_dims(1), [] __device__(auto dims) {
      if (dims.rank(cudax::thread, cudax::block) == 0)
      {
        printf("Hello from the GPU\n");
        kernel_run_proof = true;
      }
    });
    check_kernel_run(stream);
  }

  SECTION("Dynamic shared memory option")
  {
    auto dims   = cudax::block_dims<32>() & cudax::grid_dims<1>();
    auto config = cudax::kernel_config(dims);

    auto test = [stream](const auto& input_config) {
      SECTION("Single element")
      {
        auto config = input_config.add(cudax::dynamic_shared_memory<my_dynamic_smem_t>());

        cudax::launch(stream, config, dynamic_smem_single<my_dynamic_smem_t>());
        check_kernel_run(stream);
      }

      SECTION("Dynamic span")
      {
        const int size = 2;
        auto config    = input_config.add(cudax::dynamic_shared_memory<my_dynamic_smem_t>(size));
        cudax::launch(stream, config, dynamic_smem_span<my_dynamic_smem_t, ::cuda::std::dynamic_extent>(), size);
        check_kernel_run(stream);
      }

      SECTION("Static span")
      {
        constexpr int size = 3;
        auto config        = input_config.add(cudax::dynamic_shared_memory<my_dynamic_smem_t, size>());
        cudax::launch(stream, config, dynamic_smem_span<my_dynamic_smem_t, size>(), size);
        check_kernel_run(stream);
      }
    };

    test(config);
    test(config.add(cudax::cooperative_launch(), cudax::launch_priority(0)));
  }

  CUDART(hipStreamSynchronize(stream));
  CUDART(hipStreamDestroy(stream));
}

TEST_CASE("Smoke", "[launch]")
{
  launch_smoke_test();
}

__global__ void check_expected_counts(unsigned int num_threads_in_block, unsigned int num_blocks_in_grid)
{
  CUDAX_REQUIRE(cudax::block.count(cudax::thread) == num_threads_in_block);
  CUDAX_REQUIRE(cudax::grid.count(cudax::block) == num_blocks_in_grid);
};

template <unsigned int Count = 0>
__global__ void shared_memory_expected_counts(unsigned int num_threads_in_block, unsigned int num_blocks_in_grid)
{
  CUDAX_REQUIRE(cudax::block.count(cudax::thread) == num_threads_in_block);
  CUDAX_REQUIRE(cudax::grid.count(cudax::block) == num_blocks_in_grid);

  if constexpr (Count != 0)
  {
    __shared__ int smem[Count];
    smem[threadIdx.x] = 1;
    CUDAX_REQUIRE(smem[threadIdx.x] == 1);
  }
}

__global__ void grid_sync_kernel(int i)
{
  auto grid = cooperative_groups::this_grid();
  grid.sync();
};

template <typename Dims>
inline void print_dims(const Dims& in)
{
  std::cout << in.count() << " block: " << in.count(cudax::thread, cudax::block) << " grid: " << in.count(cudax::block)
            << std::endl;
}

void meta_dims_test()
{
  hipStream_t stream;
  CUDART(hipStreamCreate(&stream));

  SECTION("Just at least")
  {
    constexpr unsigned int block_size = 256, grid_size = 4;
    auto dims = cudax::make_hierarchy(
      cudax::block_dims<block_size>(), cudax::grid_dims(cudax::at_least(block_size * grid_size, cudax::thread)));

    // Won't work until finalized
    // dims.count();

    // Does not touch a meta dims, so works
    static_assert(dims.count(cudax::thread, cudax::block) == block_size);

    auto dims_finalized = cudax::finalize(stream, dims, check_expected_counts);
    static_assert(::cuda::std::is_same_v<::cudax::finalized_t<decltype(dims)>, decltype(dims_finalized)>);

    CUDAX_REQUIRE(dims_finalized.count(cudax::block, cudax::grid) == grid_size);

    cudax::launch(stream, dims_finalized, check_expected_counts, block_size, grid_size);

    cudax::launch(stream, dims, check_expected_counts, block_size, grid_size);
  }

  SECTION("At least with adjacent level")
  {
    // Not the best usage, but should work too
    constexpr unsigned int block_size = 256, grid_size = 4;
    auto dims = cudax::make_hierarchy(
      cudax::block_dims<block_size>(), cudax::grid_dims(cudax::at_least(grid_size, cudax::block)));

    cudax::launch(stream, dims, check_expected_counts, block_size, grid_size);
  }

  SECTION("At least + best occupancy")
  {
    unsigned int target_count = 4420;
    auto dims                 = cudax::make_hierarchy(
      cudax::block_dims(cudax::max_occupancy()), cudax::grid_dims(cudax::at_least(target_count, cudax::thread)));

    auto dims_finalized = cudax::finalize(stream, dims, empty_kernel);
    static_assert(::cuda::std::is_same_v<::cudax::finalized_t<decltype(dims)>, decltype(dims_finalized)>);

    CUDAX_REQUIRE(dims_finalized.count(cudax::thread) >= target_count);

    cudax::launch(stream, dims_finalized, empty_kernel, 1);

    cudax::launch(stream, dims, empty_kernel, 1);
  }

  SECTION("max_coresident + best occupancy")
  {
    auto dims =
      cudax::make_hierarchy(cudax::block_dims(cudax::max_occupancy()), cudax::grid_dims(cudax::max_coresident()));

    auto config = cudax::make_config(dims, cudax::cooperative_launch());

    auto config_finalized = cudax::finalize(stream, config, grid_sync_kernel);
    static_assert(::cuda::std::is_same_v<::cudax::finalized_t<decltype(config)>, decltype(config_finalized)>);

    cudax::launch(stream, config_finalized, grid_sync_kernel, 1);

    auto config_finalized_with_arguments = cudax::finalize(stream, config, grid_sync_kernel, 1);
    static_assert(::cuda::std::is_same_v<decltype(config_finalized_with_arguments), decltype(config_finalized)>);

    cudax::launch(stream, config_finalized_with_arguments, grid_sync_kernel, 1);

    cudax::launch(stream, config, grid_sync_kernel, 1);

    auto lambda = [] __device__(auto conf, int dummy) {
      auto grid = cooperative_groups::this_grid();
      grid.sync();
      conf.dims.count();
    };

    auto finalized_for_lambda = cudax::finalize(stream, config, lambda, 1);
    static_assert(::cuda::std::is_same_v<::cudax::finalized_t<decltype(config)>, decltype(finalized_for_lambda)>);

    cudax::launch(stream, finalized_for_lambda, lambda, 1);

    cudax::launch(stream, config, lambda, 1);
  }

  SECTION("Dyn smem and max coresident")
  {
    constexpr unsigned int large_smem = 7 * 1024;
    auto dims = cudax::make_hierarchy(cudax::block_dims(128), cudax::grid_dims(cudax::max_coresident()));

    auto dims_finalized = cudax::finalize(stream, dims, shared_memory_expected_counts<>);

    auto dims_with_smem = cudax::finalize(stream, dims, shared_memory_expected_counts<large_smem>);

    auto config = cudax::make_config(dims, cudax::dynamic_shared_memory<int>(large_smem));

    auto config_finalized = cudax::finalize(stream, config, shared_memory_expected_counts<>);

    CUDAX_REQUIRE(
      dims_finalized.count(cudax::thread, cudax::block) == config_finalized.dims.count(cudax::thread, cudax::block));
    CUDAX_REQUIRE(
      dims_finalized.count(cudax::thread, cudax::block) == dims_with_smem.count(cudax::thread, cudax::block));

    // Confirm adding large dynamic shared reduced the number of blocks that can execute at the same time
    CUDAX_REQUIRE(dims_finalized.count(cudax::block) > config_finalized.dims.count(cudax::block));
    // But the number is the same no matter if smem is static or dynamic
    CUDAX_REQUIRE(dims_with_smem.count(cudax::block) == config_finalized.dims.count(cudax::block));

    cudax::launch(stream,
                  dims,
                  shared_memory_expected_counts<>,
                  dims_finalized.count(cudax::thread, cudax::block),
                  dims_finalized.count(cudax::block));

    cudax::launch(stream,
                  dims,
                  shared_memory_expected_counts<large_smem>,
                  dims_with_smem.count(cudax::thread, cudax::block),
                  dims_with_smem.count(cudax::block));

    cudax::launch(stream,
                  config,
                  shared_memory_expected_counts<>,
                  config_finalized.dims.count(cudax::thread, cudax::block),
                  config_finalized.dims.count(cudax::block));
  }
  CUDART(hipStreamSynchronize(stream));
  CUDART(hipStreamDestroy(stream));
}

TEST_CASE("Meta dimensions", "[launch]")
{
  meta_dims_test();
}
