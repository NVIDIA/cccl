//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/memory_resource>
#include <cuda/std/__algorithm_>
#include <cuda/std/array>
#include <cuda/std/cassert>
#include <cuda/std/initializer_list>
#include <cuda/std/tuple>
#include <cuda/std/type_traits>

#include <cuda/experimental/container.cuh>

#include <stdexcept>

#include "helper.h"
#include "types.h"

#if _CCCL_CUDACC_AT_LEAST(12, 6)
using test_types = c2h::type_list<cuda::std::tuple<cuda::mr::host_accessible>,
                                  cuda::std::tuple<cuda::mr::device_accessible>,
                                  cuda::std::tuple<cuda::mr::host_accessible, cuda::mr::device_accessible>>;
#else
using test_types = c2h::type_list<cuda::std::tuple<cuda::mr::device_accessible>>;
#endif

C2H_CCCLRT_TEST("cudax::async_buffer constructors", "[container][async_buffer]", test_types)
{
  using TestT    = c2h::get<0, TestType>;
  using Env      = typename extract_properties<TestT>::env;
  using Resource = typename extract_properties<TestT>::resource;
  using Buffer   = typename extract_properties<TestT>::async_buffer;
  using T        = typename Buffer::value_type;

  cudax::stream stream{cudax::device_ref{0}};
  Env env{Resource{}, stream};

  SECTION("Construction with explicit size")
  {
    { // from env, no allocation
      const Buffer buf{env};
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // from env and size, no allocation
      const Buffer buf{env, 0};
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, 0);
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // from env, size and value, no allocation
      const Buffer buf{env, 0, T{42}};
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer(stream, Resource{}, 0, T{42});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // from env and size
      const Buffer buf{env, 5};
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(equal_size_value(buf, 5, T(0)));
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, 5);
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(equal_size_value(buf, 5, T(0)));
    }

    { // from env, size and value
      const Buffer buf{env, 5, T{42}};
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(equal_size_value(buf, 5, T(42)));
    }
    {
      const auto buf = cudax::make_async_buffer(stream, Resource{}, 5, T{42});
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(equal_size_value(buf, 5, T(42)));
    }
  }

  { // from env and size with no_init, no allocation
    SECTION("from env and size with no_init, no allocation")
    {
      const Buffer buf{env, 0, cudax::no_init};
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, 0, cudax::no_init);
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // from env and size with no_init
      const Buffer buf{env, 5, cudax::no_init};
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(buf.data() != nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, 5, cudax::no_init);
      CUDAX_CHECK(buf.size() == 5);
      CUDAX_CHECK(buf.data() != nullptr);
    }
  }

  SECTION("Construction from iterators")
  {
    const cuda::std::array<T, 6> input{T(1), T(42), T(1337), T(0), T(12), T(-1)};
    { // can be constructed from two equal input iterators
      Buffer buf(env, input.begin(), input.begin());
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, input.begin(), input.begin());
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // can be constructed from two input iterators
      Buffer buf(env, input.begin(), input.end());
      CUDAX_CHECK(buf.size() == 6);
      CUDAX_CHECK(equal_range(buf));
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, input.begin(), input.end());
      CUDAX_CHECK(buf.size() == 6);
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("Construction from range")
  {
    { // can be constructed from an empty random access range
      Buffer buf(env, cuda::std::array<T, 0>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer<T>(stream, Resource{}, cuda::std::array<T, 0>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // can be constructed from a non-empty random access range
      Buffer buf(env, cuda::std::array<T, 6>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
    {
      const auto buf = cudax::make_async_buffer<T>(
        stream, Resource{}, cuda::std::array<T, 6>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("Construction from initializer_list")
  {
    { // can be constructed from an empty initializer_list
      const cuda::std::initializer_list<T> input{};
      Buffer buf(env, input);
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }
    {
      const auto buf = cudax::make_async_buffer(stream, Resource{}, cuda::std::initializer_list<T>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // can be constructed from a non-empty initializer_list
      const cuda::std::initializer_list<T> input{T(1), T(42), T(1337), T(0), T(12), T(-1)};
      Buffer buf(env, input);
      CUDAX_CHECK(buf.size() == 6);
      CUDAX_CHECK(equal_range(buf));
    }
    {
      const auto buf = cudax::make_async_buffer(
        stream, Resource{}, cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(buf.size() == 6);
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("copy construction")
  {
    static_assert(!cuda::std::is_nothrow_copy_constructible<Buffer>::value, "");
    { // can be copy constructed from empty input
      const Buffer input{env, 0};
      Buffer buf(input);
      CUDAX_CHECK(buf.empty());
    }

    { // can be copy constructed from non-empty input
      const Buffer input{env, {T(1), T(42), T(1337), T(0), T(12), T(-1)}};
      Buffer buf(input);
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("move construction")
  {
    static_assert(cuda::std::is_nothrow_move_constructible<Buffer>::value, "");

    { // can be move constructed with empty input
      Buffer input{env, 0};
      Buffer buf(cuda::std::move(input));
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(input.empty());
    }

    { // can be move constructed from non-empty input
      Buffer input{env, {T(1), T(42), T(1337), T(0), T(12), T(-1)}};

      // ensure that we steal the data
      const auto* allocation = input.data();
      Buffer buf(cuda::std::move(input));
      CUDAX_CHECK(buf.size() == 6);
      CUDAX_CHECK(buf.data() == allocation);
      CUDAX_CHECK(input.size() == 0);
      CUDAX_CHECK(input.data() == nullptr);
      CUDAX_CHECK(equal_range(buf));
    }
  }
  stream.sync();

#if 0 // Implement exception handling
#  if _CCCL_HAS_EXCEPTIONS()
  SECTION("Exception handling throwing bad_alloc")
  {
    using async_buffer = cudax::async_buffer<int>;

    try
    {
      async_buffer too_small(2 * capacity);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      async_buffer too_small(2 * capacity, 42);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      cuda::std::array<int, 2 * capacity> input{0, 1, 2, 3, 4, 5, 6, 7};
      async_buffer too_small(input.begin(), input.end());
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      cuda::std::initializer_list<int> input{0, 1, 2, 3, 4, 5, 6};
      async_buffer too_small(input);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      uncommon_range<int, 2 * capacity> input{{0, 1, 2, 3, 4, 5, 6, 7}};
      async_buffer too_small(input);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      sized_uncommon_range<int, 2 * capacity> input{{0, 1, 2, 3, 4, 5, 6, 7}};
      async_buffer too_small(input);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      cuda::std::array<int, 2 * capacity> input{0, 1, 2, 3, 4, 5, 6, 7};
      async_buffer too_small(input);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }
  }
#  endif // _CCCL_HAS_EXCEPTIONS()
#endif // 0
}
