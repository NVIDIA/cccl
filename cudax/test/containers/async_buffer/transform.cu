//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/device/device_transform.cuh>

#include <thrust/host_vector.h>
#include <thrust/sequence.h>

#include <cuda/memory_resource>
#include <cuda/std/tuple>
#include <cuda/std/type_traits>
#include <cuda/std/utility>

#include <cuda/experimental/container.cuh>

#include <algorithm>

#include "helper.h"
#include "types.h"

using hipcub::detail::transform::Algorithm;

template <Algorithm Alg>
struct policy_hub_for_alg
{
  struct max_policy : hipcub::ChainedPolicy<300, max_policy, max_policy>
  {
    static constexpr int min_bif         = 64 * 1024;
    static constexpr Algorithm algorithm = Alg;
    using algo_policy =
      ::cuda::std::_If<Alg == Algorithm::prefetch,
                       hipcub::detail::transform::prefetch_policy_t<256>,
                       hipcub::detail::transform::async_copy_policy_t<256>>;
  };
};

template <Algorithm Alg,
          typename Offset,
          typename... RandomAccessIteratorsIn,
          typename RandomAccessIteratorOut,
          typename TransformOp>
HIPCUB_RUNTIME_FUNCTION static hipError_t transform_many_with_alg(
  ::cuda::std::tuple<RandomAccessIteratorsIn...> inputs,
  RandomAccessIteratorOut output,
  Offset num_items,
  TransformOp transform_op,
  hipStream_t stream = nullptr)
{
  return hipcub::detail::transform::dispatch_t<hipcub::detail::transform::requires_stable_address::no,
                                            Offset,
                                            ::cuda::std::tuple<RandomAccessIteratorsIn...>,
                                            RandomAccessIteratorOut,
                                            TransformOp,
                                            policy_hub_for_alg<Alg>>{}
    .dispatch(inputs, output, num_items, transform_op, stream);
}

using algorithms =
  c2h::type_list<::cuda::std::integral_constant<Algorithm, Algorithm::prefetch>
#ifdef _CUB_HAS_TRANSFORM_UBLKCP
                 ,
                 ::cuda::std::integral_constant<Algorithm, Algorithm::ublkcp>
#endif // _CUB_HAS_TRANSFORM_UBLKCP
                 >;

#ifdef _CUB_HAS_TRANSFORM_UBLKCP
#  define FILTER_UBLKCP                                \
    if (alg == Algorithm::ublkcp && ptx_version < 900) \
    {                                                  \
      return;                                          \
    }
#else // _CUB_HAS_TRANSFORM_UBLKCP
#  define FILTER_UBLKCP
#endif // _CUB_HAS_TRANSFORM_UBLKCP

#define FILTER_UNSUPPORTED_ALGS                                           \
  int ptx_version = 0;                                                    \
  REQUIRE(hipcub::PtxVersion(ptx_version) == hipSuccess);                   \
  _CCCL_DIAG_PUSH                                                         \
  _CCCL_DIAG_SUPPRESS_MSVC(4127) /* conditional expression is constant */ \
  FILTER_UBLKCP                                                           \
  _CCCL_DIAG_POP

C2H_TEST("DeviceTransform::Transform cudax::async_device_buffer", "[device][device_transform]", algorithms)
{
  using type         = int;
  constexpr auto alg = c2h::get<0, TestType>::value;
  FILTER_UNSUPPORTED_ALGS
  const int num_items = 1 << 24;

  cudax::stream stream{};
  cudax::env_t<cuda::mr::device_accessible> env{cudax::device_memory_resource{}, stream};

  cudax::async_device_buffer<type> a{env, num_items, cudax::uninit};
  cudax::async_device_buffer<type> b{env, num_items, cudax::uninit};
  thrust::sequence(thrust::cuda::par_nosync.on(stream), a.begin(), a.end());
  thrust::sequence(thrust::cuda::par_nosync.on(stream), b.begin(), b.end());

  cudax::async_device_buffer<type> result{env, num_items, cudax::uninit};

  transform_many_with_alg<alg>(
    ::cuda::std::make_tuple(a.begin(), b.begin()), result.begin(), num_items, ::cuda::std::plus<type>{});

  // copy back to host
  thrust::host_vector<type> a_h(num_items);
  thrust::host_vector<type> b_h(num_items);
  thrust::host_vector<type> result_h(num_items);
  REQUIRE(hipMemcpyAsync(a_h.data(), a.data(), num_items * sizeof(type), hipMemcpyDeviceToHost, stream.get())
          == hipSuccess);
  REQUIRE(hipMemcpyAsync(b_h.data(), b.data(), num_items * sizeof(type), hipMemcpyDeviceToHost, stream.get())
          == hipSuccess);
  REQUIRE(
    hipMemcpyAsync(result_h.data(), result.data(), num_items * sizeof(type), hipMemcpyDeviceToHost, stream.get())
    == hipSuccess);
  stream.sync();

  // compute reference and verify
  thrust::host_vector<type> reference_h(num_items);
  std::transform(a_h.begin(), a_h.end(), b_h.begin(), reference_h.begin(), std::plus<type>{});
  REQUIRE(reference_h == result_h);
}
