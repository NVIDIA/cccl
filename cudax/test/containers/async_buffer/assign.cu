//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/memory_resource>
#include <cuda/std/__algorithm_>
#include <cuda/std/array>
#include <cuda/std/cassert>
#include <cuda/std/initializer_list>
#include <cuda/std/tuple>
#include <cuda/std/type_traits>

#include <cuda/experimental/container.cuh>

#include <stdexcept>

#include "helper.h"
#include "types.h"

#if _CCCL_CUDACC_AT_LEAST(12, 6)
using test_types = c2h::type_list<cuda::std::tuple<cuda::mr::host_accessible>,
                                  cuda::std::tuple<cuda::mr::device_accessible>,
                                  cuda::std::tuple<cuda::mr::host_accessible, cuda::mr::device_accessible>>;
#else
using test_types = c2h::type_list<cuda::std::tuple<cuda::mr::device_accessible>>;
#endif

C2H_TEST("cudax::async_buffer assign", "[container][async_buffer]", test_types)
{
  using TestT    = c2h::get<0, TestType>;
  using Env      = typename extract_properties<TestT>::env;
  using Resource = typename extract_properties<TestT>::resource;
  using Buffer   = typename extract_properties<TestT>::async_buffer;
  using T        = typename Buffer::value_type;

  cudax::stream stream{};
  Env env{Resource{}, stream};

  SECTION("cudax::async_buffer::assign_range random access range")
  {
    { // cudax::async_buffer::assign_range with an empty input
      Buffer buf{env};
      buf.assign_range(cuda::std::array<T, 0>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // cudax::async_buffer::assign_range with an empty input, shrinking
      Buffer buf{env, 10, T(-2)};
      buf.assign_range(cuda::std::array<T, 0>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() != nullptr);
    }

    { // cudax::async_buffer::assign_range with a non-empty input, shrinking
      Buffer buf{env, 10, T(-2)};
      buf.assign_range(cuda::std::array<T, 6>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }

    { // cudax::async_buffer::assign_range with a non-empty input, growing
      Buffer buf{env, 4, T(-2)};
      buf.assign_range(cuda::std::array<T, 6>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("cudax::async_buffer::assign(count, const T&)")
  {
    { // cudax::async_buffer::assign(count, const T&), zero count from empty
      Buffer buf{env};
      buf.assign(0, T(42));
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // cudax::async_buffer::assign(count, const T&), shrinking to empty
      Buffer buf{env, 10, T(-2)};
      buf.assign(0, T(42));
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() != nullptr);
    }

    { // cudax::async_buffer::assign(count, const T&), shrinking
      Buffer buf{env, 10, T(-2)};
      buf.assign(2, T(42));
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_size_value(buf, 2, T(42)));
    }

    { // cudax::async_buffer::assign(count, const T&), growing
      Buffer buf{env, 4, T(-2)};
      buf.assign(6, T(42));
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_size_value(buf, 6, T{42}));
    }
  }

  SECTION("cudax::async_buffer::assign(iter, iter) forward iterators")
  {
    const cuda::std::array<T, 6> input = {T(1), T(42), T(1337), T(0), T(12), T(-1)};
    { // cudax::async_buffer::assign(iter, iter), with forward iterators empty range
      Buffer buf{env};
      buf.assign(input.begin(), input.begin());
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // cudax::async_buffer::assign(iter, iter), with forward iterators shrinking to empty
      Buffer buf{env, 10, T(-2)};
      buf.assign(input.begin(), input.begin());
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() != nullptr);
    }

    { // cudax::async_buffer::assign(iter, iter), with forward iterators shrinking
      Buffer buf{env, 10, T(-2)};
      buf.assign(input.begin(), input.end());
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }

    { // cudax::async_buffer::assign(iter, iter), with forward iterators growing
      Buffer buf{env, 4, T(-2)};
      buf.assign(input.begin(), input.end());
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
  }

  SECTION("cudax::async_buffer::assign(initializer_list)")
  {
    { // cudax::async_buffer::assign(initializer_list), empty range
      Buffer buf{env};
      buf.assign(cuda::std::initializer_list<T>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() == nullptr);
    }

    { // cudax::async_buffer::assign(initializer_list), shrinking to empty
      Buffer buf{env, 10, T(-2)};
      buf.assign(cuda::std::initializer_list<T>{});
      CUDAX_CHECK(buf.empty());
      CUDAX_CHECK(buf.data() != nullptr);
    }

    { // cudax::async_buffer::assign(initializer_list), shrinking
      Buffer buf{env, 10, T(-2)};
      buf.assign(cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }

    { // cudax::async_buffer::assign(initializer_list), growing
      Buffer buf{env, 4, T(-2)};
      buf.assign(cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
      CUDAX_CHECK(!buf.empty());
      CUDAX_CHECK(equal_range(buf));
    }
  }
  stream.sync();

  SECTION("cudax::async_buffer::operator=(uninitialized_async_buffer&&)")
  {
    Buffer buf1{env};
    Buffer buf2{env};
    buf2.assign(cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
    CUDAX_CHECK(buf1.empty());
    CUDAX_CHECK(!buf2.empty());
    buf1 = std::move(buf2);
    CUDAX_CHECK(buf2.empty());
    CUDAX_CHECK(!buf1.empty());
    CUDAX_CHECK(equal_range(buf1));

    buf2 = Buffer{env, cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)}};
    CUDAX_CHECK(!buf2.empty());
    CUDAX_CHECK(equal_range(buf2));
  }

  SECTION("cudax::async_buffer::destroy")
  {
    Buffer buf{env};
    buf.assign(cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)});
    CUDAX_CHECK(!buf.empty());
    CUDAX_CHECK(equal_range(buf));
    buf.destroy();
    CUDAX_CHECK(buf.empty());
    CUDAX_CHECK(buf.data() == nullptr);

    buf = Buffer{env, cuda::std::initializer_list<T>{T(1), T(42), T(1337), T(0), T(12), T(-1)}};
    CUDAX_CHECK(!buf.empty());
    CUDAX_CHECK(equal_range(buf));
  }

#if 0 // Implement exceptions
#  if _CCCL_HAS_EXCEPTIONS()
  SECTION("cudax::async_buffer::assign exception handling")
  {
    try
    {
      too_small.assign(2 * capacity, 42);
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      too_small.assign(input.begin(), input.end());
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }

    try
    {
      too_small.assign(cuda::std::initializer_list<int>{0, 1, 2, 3, 4, 5, 6});
    }
    catch (const std::bad_alloc&)
    {}
    catch (...)
    {
      CUDAX_CHECK(false);
    }
  }
#  endif // _CCCL_HAS_EXCEPTIONS()
#endif // Implement exceptions
}
