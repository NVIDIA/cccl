//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>

#include <cuda/memory_resource>
#include <cuda/std/cstdint>
#include <cuda/std/span>
#include <cuda/std/type_traits>
#include <cuda/std/utility>

#include <cuda/experimental/buffer>

#include <catch2/catch.hpp>

struct do_not_construct
{
  do_not_construct()
  {
    CHECK(false);
  }
};

struct non_trivial
{
  int val_      = 0;
  non_trivial() = default;
  __host__ __device__ constexpr non_trivial(const int val) noexcept
      : val_(val)
  {}

  __host__ __device__ constexpr friend bool operator==(const non_trivial& lhs, const non_trivial& rhs)
  {
    return lhs.val_ == rhs.val_;
  }
};

struct my_property
{
  using value_type = int;
};
constexpr int get_property(const cuda::experimental::uninitialized_buffer<int, my_property>&, my_property)
{
  return 42;
}

TEMPLATE_TEST_CASE(
  "uninitialized_buffer", "[container]", char, short, int, long, long long, float, double, do_not_construct)
{
  using uninitialized_buffer = cuda::experimental::uninitialized_buffer<TestType, cuda::mr::device_accessible>;
  static_assert(!cuda::std::is_default_constructible<uninitialized_buffer>::value, "");
  static_assert(!cuda::std::is_copy_constructible<uninitialized_buffer>::value, "");
  static_assert(!cuda::std::is_copy_assignable<uninitialized_buffer>::value, "");

  cuda::mr::cuda_memory_resource resource{};

  SECTION("construction")
  {
    static_assert(!cuda::std::is_copy_constructible<uninitialized_buffer>::value, "");
    {
      uninitialized_buffer from_count{resource, 42};
      CHECK(from_count.data() != nullptr);
      CHECK(from_count.size() == 42);
    }
    {
      uninitialized_buffer input{resource, 42};
      const TestType* ptr = input.data();

      uninitialized_buffer from_rvalue{cuda::std::move(input)};
      CHECK(from_rvalue.data() == ptr);
      CHECK(from_rvalue.size() == 42);

      // Ensure that we properly reset the input buffer
      CHECK(input.data() == nullptr);
      CHECK(input.size() == 0);
    }
  }

  SECTION("assignment")
  {
    static_assert(!cuda::std::is_copy_assignable<uninitialized_buffer>::value, "");
    {
      cuda::mr::cuda_managed_memory_resource other_resource{};
      uninitialized_buffer input{other_resource, 42};
      uninitialized_buffer buf{resource, 1337};
      const auto* old_ptr       = buf.data();
      const auto* old_input_ptr = input.data();

      buf = cuda::std::move(input);
      CHECK(buf.data() != old_ptr);
      CHECK(buf.data() == old_input_ptr);
      CHECK(buf.size() == 42);
      CHECK(buf.resource() == other_resource);

      CHECK(input.data() == nullptr);
      CHECK(input.size() == 0);
    }

    { // Ensure self move assignment doesnt do anything
      uninitialized_buffer buf{resource, 1337};
      const auto* old_ptr = buf.data();

      buf = cuda::std::move(buf);
      CHECK(buf.data() == old_ptr);
      CHECK(buf.size() == 1337);
    }
  }

  SECTION("access")
  {
    uninitialized_buffer buf{resource, 42};
    CHECK(buf.data() != nullptr);
    CHECK(buf.size() == 42);
    CHECK(buf.begin() == buf.data());
    CHECK(buf.end() == buf.begin() + buf.size());
    CHECK(buf.resource() == resource);

    CHECK(cuda::std::as_const(buf).data() != nullptr);
    CHECK(cuda::std::as_const(buf).size() == 42);
    CHECK(cuda::std::as_const(buf).begin() == buf.data());
    CHECK(cuda::std::as_const(buf).end() == buf.begin() + buf.size());
    CHECK(cuda::std::as_const(buf).resource() == resource);
  }

  SECTION("properties")
  {
    static_assert(cuda::has_property<cuda::experimental::uninitialized_buffer<int, cuda::mr::device_accessible>,
                                     cuda::mr::device_accessible>,
                  "");
    static_assert(cuda::has_property<cuda::experimental::uninitialized_buffer<int, my_property>, my_property>, "");
  }

  SECTION("convertion to span")
  {
    uninitialized_buffer buf{resource, 42};
    const cuda::std::span<TestType> as_span{buf};
    CHECK(as_span.data() == buf.data());
    CHECK(as_span.size() == 42);
  }

  SECTION("Actually use memory")
  {
    if constexpr (!cuda::std::is_same_v<TestType, do_not_construct>)
    {
      uninitialized_buffer buf{resource, 42};
      thrust::fill(thrust::device, buf.begin(), buf.end(), TestType{2});
      const auto res = thrust::reduce(thrust::device, buf.begin(), buf.end(), TestType{0}, thrust::plus<int>());
      CHECK(res == TestType{84});
    }
  }
}
