#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>

#include <cuda/memory_resource>
#include <cuda/std/cstdint>
#include <cuda/std/span>
#include <cuda/std/type_traits>
#include <cuda/std/utility>

#include <cuda/experimental/buffer.cuh>
#include <cuda/experimental/launch.cuh>
#include <cuda/experimental/stream.cuh>

#include "testing.cuh"
#include <catch2/catch.hpp>

struct do_not_construct
{
  do_not_construct()
  {
    CUDAX_CHECK(false);
  }
};

struct non_trivial
{
  int val_      = 0;
  non_trivial() = default;
  __host__ __device__ constexpr non_trivial(const int val) noexcept
      : val_(val)
  {}

  __host__ __device__ constexpr friend bool operator==(const non_trivial& lhs, const non_trivial& rhs)
  {
    return lhs.val_ == rhs.val_;
  }
};

struct my_property
{
  using value_type = int;
};
constexpr int get_property(const cuda::experimental::uninitialized_buffer<int, my_property>&, my_property)
{
  return 42;
}

TEMPLATE_TEST_CASE(
  "uninitialized_buffer", "[container]", char, short, int, long, long long, float, double, do_not_construct)
{
  using uninitialized_buffer = cuda::experimental::uninitialized_buffer<TestType, cuda::mr::device_accessible>;
  static_assert(!cuda::std::is_default_constructible<uninitialized_buffer>::value, "");
  static_assert(!cuda::std::is_copy_constructible<uninitialized_buffer>::value, "");
  static_assert(!cuda::std::is_copy_assignable<uninitialized_buffer>::value, "");

  cuda::mr::device_memory_resource resource{};

  SECTION("construction")
  {
    static_assert(!cuda::std::is_copy_constructible<uninitialized_buffer>::value, "");
    {
      uninitialized_buffer from_count{resource, 42};
      CUDAX_CHECK(from_count.data() != nullptr);
      CUDAX_CHECK(from_count.size() == 42);
    }
    {
      uninitialized_buffer input{resource, 42};
      const TestType* ptr = input.data();

      uninitialized_buffer from_rvalue{cuda::std::move(input)};
      CUDAX_CHECK(from_rvalue.data() == ptr);
      CUDAX_CHECK(from_rvalue.size() == 42);

      // Ensure that we properly reset the input buffer
      CUDAX_CHECK(input.data() == nullptr);
      CUDAX_CHECK(input.size() == 0);
    }
  }

  SECTION("assignment")
  {
    static_assert(!cuda::std::is_copy_assignable<uninitialized_buffer>::value, "");
    {
      cuda::mr::managed_memory_resource other_resource{};
      uninitialized_buffer input{other_resource, 42};
      uninitialized_buffer buf{resource, 1337};
      const auto* old_ptr       = buf.data();
      const auto* old_input_ptr = input.data();

      buf = cuda::std::move(input);
      CUDAX_CHECK(buf.data() != old_ptr);
      CUDAX_CHECK(buf.data() == old_input_ptr);
      CUDAX_CHECK(buf.size() == 42);
      CUDAX_CHECK(buf.get_resource() == other_resource);

      CUDAX_CHECK(input.data() == nullptr);
      CUDAX_CHECK(input.size() == 0);
    }

    { // Ensure self move assignment doesnt do anything
      uninitialized_buffer buf{resource, 1337};
      const auto* old_ptr = buf.data();

      buf = cuda::std::move(buf);
      CUDAX_CHECK(buf.data() == old_ptr);
      CUDAX_CHECK(buf.size() == 1337);
    }
  }

  SECTION("access")
  {
    uninitialized_buffer buf{resource, 42};
    CUDAX_CHECK(buf.data() != nullptr);
    CUDAX_CHECK(buf.size() == 42);
    CUDAX_CHECK(buf.begin() == buf.data());
    CUDAX_CHECK(buf.end() == buf.begin() + buf.size());
    CUDAX_CHECK(buf.get_resource() == resource);

    CUDAX_CHECK(cuda::std::as_const(buf).data() != nullptr);
    CUDAX_CHECK(cuda::std::as_const(buf).size() == 42);
    CUDAX_CHECK(cuda::std::as_const(buf).begin() == buf.data());
    CUDAX_CHECK(cuda::std::as_const(buf).end() == buf.begin() + buf.size());
    CUDAX_CHECK(cuda::std::as_const(buf).get_resource() == resource);
  }

  SECTION("properties")
  {
    static_assert(cuda::has_property<cuda::experimental::uninitialized_buffer<int, cuda::mr::device_accessible>,
                                     cuda::mr::device_accessible>,
                  "");
    static_assert(cuda::has_property<cuda::experimental::uninitialized_buffer<int, my_property>, my_property>, "");
  }

  SECTION("convertion to span")
  {
    uninitialized_buffer buf{resource, 42};
    const cuda::std::span<TestType> as_span{buf};
    CUDAX_CHECK(as_span.data() == buf.data());
    CUDAX_CHECK(as_span.size() == 42);
  }

  SECTION("Actually use memory")
  {
    if constexpr (!cuda::std::is_same_v<TestType, do_not_construct>)
    {
      uninitialized_buffer buf{resource, 42};
      thrust::fill(thrust::device, buf.begin(), buf.end(), TestType{2});
      const auto res = thrust::reduce(thrust::device, buf.begin(), buf.end(), TestType{0}, thrust::plus<int>());
      CUDAX_CHECK(res == TestType{84});
    }
  }
}

__global__ void kernel(_CUDA_VSTD::span<int> data)
{
  // Touch the memory to be sure it's accessible
  CUDAX_CHECK(data.size() == 1024);
  data[0] = 42;
}

__global__ void const_kernel(_CUDA_VSTD::span<const int> data)
{
  // Touch the memory to be sure it's accessible
  CUDAX_CHECK(data.size() == 1024);
}

TEST_CASE("uninitialized_buffer is usable with cudax::launch", "[container]")
{
  SECTION("non-const")
  {
    const int grid_size = 4;
    cudax::uninitialized_buffer<int, ::cuda::mr::device_accessible> buffer{cuda::mr::device_memory_resource{}, 1024};
    auto dimensions = cudax::make_hierarchy(cudax::grid_dims(grid_size), cudax::block_dims<256>());

    cudax::stream stream;

    cudax::launch(stream, dimensions, kernel, buffer);
  }

  SECTION("const")
  {
    const int grid_size = 4;
    const cudax::uninitialized_buffer<int, ::cuda::mr::device_accessible> buffer{
      cuda::mr::device_memory_resource{}, 1024};
    auto dimensions = cudax::make_hierarchy(cudax::grid_dims(grid_size), cudax::block_dims<256>());

    cudax::stream stream;

    cudax::launch(stream, dimensions, const_kernel, buffer);
  }
}
