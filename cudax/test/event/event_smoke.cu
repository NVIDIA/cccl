//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/event.cuh>

#include "../hierarchy/testing_common.cuh"
#include <catch2/catch.hpp>

namespace
{
cudax::event_ref fn_takes_event_ref(cudax::event_ref ref)
{
  return ref;
}
} // namespace

TEST_CASE("can construct an event_ref from a hipEvent_t", "[event]")
{
  ::hipEvent_t event;
  CUDAX_REQUIRE(::hipEventCreate(&event) == ::hipSuccess);
  cudax::event_ref ref(event);
  CUDAX_REQUIRE(ref.get() == event);
  // test implicit converstion from hipEvent_t:
  cudax::event_ref ref2 = ::fn_takes_event_ref(event);
  CUDAX_REQUIRE(ref2.get() == event);
  CUDAX_REQUIRE(::hipEventDestroy(event) == ::hipSuccess);
}

TEST_CASE("can copy construct an event_ref and compare for equality", "[event]")
{
  ::hipEvent_t event;
  CUDAX_REQUIRE(::hipEventCreate(&event) == ::hipSuccess);
  const cudax::event_ref ref(event);
  const cudax::event_ref ref2 = ref;
  CUDAX_REQUIRE(ref2 == ref);
  CUDAX_REQUIRE(!(ref != ref2));
  CUDAX_REQUIRE((ref ? true : false)); // test contextual convertibility to bool
  CUDAX_REQUIRE(!!ref);
  CUDAX_REQUIRE(::hipEvent_t{} != ref);
  CUDAX_REQUIRE(::hipEventDestroy(event) == ::hipSuccess);
}
