//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/event.cuh>
#include <cuda/experimental/stream.cuh>

#include <catch2/catch.hpp>
#include <utility.cuh>

namespace
{
namespace test
{
cudax::event_ref fn_takes_event_ref(cudax::event_ref ref)
{
  return ref;
}
} // namespace test
} // namespace

static_assert(!_CUDA_VSTD::is_default_constructible_v<cudax::event_ref>);
static_assert(!_CUDA_VSTD::is_default_constructible_v<cudax::event>);
static_assert(!_CUDA_VSTD::is_default_constructible_v<cudax::timed_event>);

TEST_CASE("can construct an event_ref from a hipEvent_t", "[event]")
{
  ::hipEvent_t ev;
  CUDAX_REQUIRE(::hipEventCreate(&ev) == ::hipSuccess);
  cudax::event_ref ref(ev);
  CUDAX_REQUIRE(ref.get() == ev);
  CUDAX_REQUIRE(!!ref);
  // test implicit converstion from hipEvent_t:
  cudax::event_ref ref2 = ::test::fn_takes_event_ref(ev);
  CUDAX_REQUIRE(ref2.get() == ev);
  CUDAX_REQUIRE(::hipEventDestroy(ev) == ::hipSuccess);
  // test an empty event_ref:
  cudax::event_ref ref3(::hipEvent_t{});
  CUDAX_REQUIRE(ref3.get() == ::hipEvent_t{});
  CUDAX_REQUIRE(!ref3);
}

TEST_CASE("can copy construct an event_ref and compare for equality", "[event]")
{
  ::hipEvent_t ev;
  CUDAX_REQUIRE(::hipEventCreate(&ev) == ::hipSuccess);
  const cudax::event_ref ref(ev);
  const cudax::event_ref ref2 = ref;
  CUDAX_REQUIRE(ref2 == ref);
  CUDAX_REQUIRE(!(ref != ref2));
  CUDAX_REQUIRE((ref ? true : false)); // test contextual convertibility to bool
  CUDAX_REQUIRE(!!ref);
  CUDAX_REQUIRE(::hipEvent_t{} != ref);
  CUDAX_REQUIRE(::hipEventDestroy(ev) == ::hipSuccess);
  // copy from empty event_ref:
  const cudax::event_ref ref3(::hipEvent_t{});
  const cudax::event_ref ref4 = ref3;
  CUDAX_REQUIRE(ref4 == ref3);
  CUDAX_REQUIRE(!(ref3 != ref4));
  CUDAX_REQUIRE(!ref4);
}

TEST_CASE("can use event_ref to record and wait on an event", "[event]")
{
  ::hipEvent_t ev;
  CUDAX_REQUIRE(::hipEventCreate(&ev) == ::hipSuccess);
  const cudax::event_ref ref(ev);

  test::managed<int> i(0);
  cudax::stream stream;
  cudax::launch(stream, ::test::one_thread_dims, ::test::assign_42{}, i.get());
  ref.record(stream);
  ref.wait();
  CUDAX_REQUIRE(ref.is_done());
  CUDAX_REQUIRE(*i == 42);

  stream.wait();
  CUDAX_REQUIRE(::hipEventDestroy(ev) == ::hipSuccess);
}

TEST_CASE("can construct an event with a stream_ref", "[event]")
{
  cudax::stream stream;
  cudax::event ev(static_cast<cuda::stream_ref>(stream));
  CUDAX_REQUIRE(ev.get() != ::hipEvent_t{});
}

TEST_CASE("can wait on an event", "[event]")
{
  cudax::stream stream;
  ::test::managed<int> i(0);
  cudax::launch(stream, ::test::one_thread_dims, ::test::assign_42{}, i.get());
  cudax::event ev(stream);
  ev.wait();
  CUDAX_REQUIRE(ev.is_done());
  CUDAX_REQUIRE(*i == 42);
  stream.wait();
}

TEST_CASE("can take the difference of two timed_event objects", "[event]")
{
  cudax::stream stream;
  ::test::managed<int> i(0);
  cudax::timed_event start(stream);
  cudax::launch(stream, ::test::one_thread_dims, ::test::assign_42{}, i.get());
  cudax::timed_event end(stream);
  end.wait();
  CUDAX_REQUIRE(end.is_done());
  CUDAX_REQUIRE(*i == 42);
  auto elapsed = end - start;
  CUDAX_REQUIRE(elapsed.count() >= 0);
  STATIC_REQUIRE(_CUDA_VSTD::is_same_v<decltype(elapsed), _CUDA_VSTD::chrono::nanoseconds>);
  stream.wait();
}

TEST_CASE("can observe the event in not ready state", "[event]")
{
  ::test::managed<int> i(0);
  ::cuda::atomic_ref atomic_i(*i);

  cudax::stream stream;

  cudax::launch(stream, ::test::one_thread_dims, ::test::spin_until_80{}, i.get());
  cudax::event ev(stream);
  CUDAX_REQUIRE(!ev.is_done());
  atomic_i.store(80);
  ev.wait();
  CUDAX_REQUIRE(ev.is_done());
}
