//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/launch.cuh>
#include <cuda/experimental/stream.cuh>

#include <catch2/catch.hpp>
#include <utility.cuh>

TEST_CASE("Can create a stream and launch work into it", "[stream]")
{
  cudax::stream str;
  ::test::managed<int> i(0);
  cudax::launch(str, ::test::one_thread_dims, ::test::assign_42{}, i.get());
  str.wait();
  CUDAX_REQUIRE(*i == 42);
}

TEST_CASE("From native handle", "[stream]")
{
  hipStream_t handle;
  CUDART(hipStreamCreate(&handle));
  {
    auto stream = cudax::stream::from_native_handle(handle);

    ::test::managed<int> i(0);
    cudax::launch(stream, ::test::one_thread_dims, ::test::assign_42{}, i.get());
    stream.wait();
    CUDAX_REQUIRE(*i == 42);
    (void) stream.release();
  }
  CUDART(hipStreamDestroy(handle));
}

template <typename StreamType>
void add_dependency_test(const StreamType& waiter, const StreamType& waitee)
{
  CUDAX_REQUIRE(waiter != waitee);

  auto verify_dependency = [&](const auto& insert_dependency) {
    ::test::managed<int> i(0);
    ::cuda::atomic_ref atomic_i(*i);

    cudax::launch(waitee, ::test::one_thread_dims, ::test::spin_until_80{}, i.get());
    cudax::launch(waitee, ::test::one_thread_dims, ::test::assign_42{}, i.get());
    insert_dependency();
    cudax::launch(waiter, ::test::one_thread_dims, ::test::verify_42{}, i.get());
    CUDAX_REQUIRE(atomic_i.load() != 42);
    CUDAX_REQUIRE(!waiter.ready());
    atomic_i.store(80);
    waiter.wait();
    waitee.wait();
  };

  SECTION("Stream wait declared event")
  {
    verify_dependency([&]() {
      cudax::event ev(waitee);
      waiter.wait(ev);
    });
  }

  SECTION("Stream wait returned event")
  {
    verify_dependency([&]() {
      auto ev = waitee.record_event();
      waiter.wait(ev);
    });
  }

  SECTION("Stream wait returned timed event")
  {
    verify_dependency([&]() {
      auto ev = waitee.record_timed_event();
      waiter.wait(ev);
    });
  }

  SECTION("Stream wait stream")
  {
    verify_dependency([&]() {
      waiter.wait(waitee);
    });
  }
}

TEST_CASE("Can add dependency into a stream", "[stream]")
{
  cudax::stream waiter, waitee;

  add_dependency_test<cudax::stream>(waiter, waitee);
  add_dependency_test<cudax::stream_ref>(waiter, waitee);
}

TEST_CASE("Stream priority", "[stream]")
{
  cudax::stream stream_default_prio;
  CUDAX_REQUIRE(stream_default_prio.priority() == cudax::stream::default_priority);

  auto priority = cudax::stream::default_priority - 1;
  cudax::stream stream(0, priority);
  CUDAX_REQUIRE(stream.priority() == priority);
}

TEST_CASE("Stream get device", "[stream]")
{
  cudax::stream dev0_stream(cudax::device_ref{0});
  CUDAX_REQUIRE(dev0_stream.device() == 0);

  hipSetDevice(static_cast<int>(cudax::devices.size() - 1));
  hipStream_t stream_handle;
  CUDART(hipStreamCreate(&stream_handle));
  auto stream_cudart = cudax::stream::from_native_handle(stream_handle);
  CUDAX_REQUIRE(stream_cudart.device() == *std::prev(cudax::devices.end()));
  auto stream_ref_cudart = cudax::stream_ref(stream_handle);
  CUDAX_REQUIRE(stream_ref_cudart.device() == *std::prev(cudax::devices.end()));

  INFO("Can create a side stream using logical device")
  {
    if (test::cuda_driver_version() >= 12050)
    {
      auto ldev = dev0_stream.logical_device();
      CUDAX_REQUIRE(ldev.get_kind() == cudax::logical_device::kinds::device);
      cudax::stream side_stream(ldev);
      CUDAX_REQUIRE(side_stream.device() == dev0_stream.device());
    }
  }
}
