#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

//! \file
//!
//! \brief Freeze data and store it as a frozen_logical_data_untyped object

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

using namespace cuda::experimental::stf;

int X0(int i)
{
  return 17 * i + 45;
}

__global__ void mult(slice<int> s, int val)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int i = tid; i < s.size(); i += nthreads)
  {
    s(i) *= val;
  }
}

int main()
{
  stream_ctx ctx;

  hipStream_t stream = ctx.pick_stream();

  const int N = 16;
  int X[N];

  for (int i = 0; i < N; i++)
  {
    X[i] = X0(i);
  }

  auto lX = ctx.logical_data(X).set_symbol("X");
  auto lY = ctx.logical_data(lX.shape()).set_symbol("Y");

  for (int k = 0; k < 4; k++)
  {
    logical_data_untyped lX_untyped = lX;
    auto fx                         = ctx.freeze(lX_untyped, access_mode::rw, data_place::current_device());

    _CCCL_ASSERT(fx.get_access_mode() == access_mode::rw, "invalid access mode");

    auto dX = fx.template get<slice<int>>(data_place::current_device(), stream);
    mult<<<8, 4, 0, stream>>>(dX, 4);
    fx.unfreeze(stream);

    ctx.parallel_for(lX.shape(), lX.read(), lY.write()).set_symbol("Y=X")->*[] __device__(size_t i, auto x, auto y) {
      y(i) = x(i);
    };

    ctx.parallel_for(lX.shape(), lY.rw()).set_symbol("Y+=1")->*[] __device__(size_t i, auto y) {
      y(i) += 1;
    };
  }

  ctx.finalize();
}
