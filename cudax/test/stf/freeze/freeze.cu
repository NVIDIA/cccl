#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Freeze data in read-only fashion
 *
 */

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

using namespace cuda::experimental::stf;

int X0(int i)
{
  return 17 * i + 45;
}

__global__ void print(slice<int> s)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int i = tid; i < s.size(); i += nthreads)
  {
    printf("%d %d\n", i, s(i));
  }
}

int main()
{
  stream_ctx ctx;

  hipStream_t stream = ctx.pick_stream();

  const int N = 16;
  int X[N];

  for (int i = 0; i < N; i++)
  {
    X[i] = X0(i);
  }

  auto lX = ctx.logical_data(X).set_symbol("X");
  auto lY = ctx.logical_data(lX.shape()).set_symbol("Y");

  ctx.parallel_for(lX.shape(), lX.rw()).set_symbol("X=2X")->*[] __device__(size_t i, auto x) {
    x(i) *= 2;
  };

  auto fx = ctx.freeze(lX);

  auto dX = fx.get(data_place::current_device(), stream);

  print<<<8, 4, 0, stream>>>(dX);

  ctx.parallel_for(lX.shape(), lX.read(), lY.write()).set_symbol("Y=X")->*[] __device__(size_t i, auto x, auto y) {
    y(i) = x(i);
  };

  fx.unfreeze(stream);

  ctx.parallel_for(lX.shape(), lX.rw()).set_symbol("X+=1")->*[] __device__(size_t i, auto x) {
    x(i) += 1;
  };

  ctx.finalize();

  for (int i = 0; i < N; i++)
  {
    EXPECT(X[i] == 2 * X0(i) + 1);
  }
}
