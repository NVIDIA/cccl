#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

using namespace cuda::experimental::stf;

template <typename T>
__global__ void axpy(size_t n, T a, const T* x, T* y)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int ind = tid; ind < n; ind += nthreads)
  {
    y[ind] += a * x[ind];
  }
}

template <typename T>
__global__ void setup_vectors(size_t n, T* x, T* y)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (size_t ind = tid; ind < n; ind += nthreads)
  {
    x[ind] = 1.0 * ind;
    y[ind] = 2.0 * ind - 3.0;
  }
}

template <typename Ctx>
void run()
{
  Ctx ctx;
  const size_t n     = 12;
  const double alpha = 2.0;

  double *dX, *dY;
  cuda_safe_call(hipMalloc((void**) &dX, n * sizeof(double)));
  cuda_safe_call(hipMalloc((void**) &dY, n * sizeof(double)));

  // Use a kernel to setup values
  setup_vectors<<<16, 16>>>(n, dX, dY);
  cuda_safe_call(hipDeviceSynchronize());
  // We here provide device addresses and memory node 1 (which is assumed to
  // be device 0)
  auto handle_X = ctx.logical_data(make_slice(dX, n), data_place::device(0));
  auto handle_Y = ctx.logical_data(make_slice(dY, n), data_place::device(0));

  ctx.task(handle_X.read(), handle_Y.rw())->*[&](hipStream_t stream, auto X, auto Y) {
    axpy<<<16, 128, 0, stream>>>(n, alpha, X.data_handle(), Y.data_handle());
  };

  // Access Ask to use X, Y and Z on the host
  ctx.host_launch(handle_X.read(), handle_Y.read())->*[&](auto X, auto Y) {
    for (size_t ind = 0; ind < n; ind++)
    {
      // X unchanged
      EXPECT(fabs(X(ind) - 1.0 * ind) < 0.00001);
      // Y = Y + alpha X
      EXPECT(fabs(Y(ind) - (-3.0 + ind * (2.0 + alpha))) < 0.00001);
    }
  };

  ctx.finalize();
}

int main()
{
  run<stream_ctx>();
  run<graph_ctx>();
}
