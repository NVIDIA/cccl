#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Make sure we can compile with no arguments in CUDA kernel
 *
 */

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

__global__ void dummy() {}

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

int main()
{
  context ctx    = graph_ctx();
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  auto lX = ctx.logical_data(X);
  auto lY = ctx.logical_data(Y);

  // Ensure this works without arguments in the kernel
  ctx.cuda_kernel(lX.read(), lY.rw())->*[&](auto, auto) {
    return cuda_kernel_desc{dummy, 16, 128, 0};
  };

  ctx.finalize();
}
