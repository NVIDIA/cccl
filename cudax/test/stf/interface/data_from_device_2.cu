#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

using namespace cuda::experimental::stf;

template <typename T>
__global__ void axpy(int N, T a, const T* x, T* y)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int ind = tid; ind < N; ind += nthreads)
  {
    y[ind] += a * x[ind];
  }
}

template <typename T>
__global__ void setup_vectors(int N, T* x, T* y, T* z)
{
  int tid      = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;

  for (int ind = tid; ind < N; ind += nthreads)
  {
    x[ind] = 1.0 * ind;
    y[ind] = 2.0 * ind - 3.0;
    z[ind] = 7.0 * ind + 6.0;
  }
}

template <typename Ctx>
void run()
{
  Ctx ctx;

  const double alpha = 2.0;
  const int N        = 12;

  double *dX, *dY, *dZ;
  cuda_safe_call(hipMalloc((void**) &dX, N * sizeof(double)));
  cuda_safe_call(hipMalloc((void**) &dY, N * sizeof(double)));
  cuda_safe_call(hipMalloc((void**) &dZ, N * sizeof(double)));

  // Use a kernel to setup values
  setup_vectors<<<16, 16>>>(N, dX, dY, dZ);
  cuda_safe_call(hipDeviceSynchronize());

  // We here provide device addresses and memory node 1 (which is assumed to
  // be device 0)
  auto handle_X = ctx.logical_data(make_slice(dX, N), data_place::device(0));
  auto handle_Y = ctx.logical_data(make_slice(dY, N), data_place::device(0));
  auto handle_Z = ctx.logical_data(make_slice(dZ, N), data_place::device(0));

  ctx.task(handle_X.read(), handle_Y.rw())->*[&](hipStream_t stream, auto X, auto Y) {
    axpy<<<16, 128, 0, stream>>>(N, alpha, X.data_handle(), Y.data_handle());
  };

  ctx.task(handle_X.read(), handle_Z.rw())->*[&](hipStream_t stream, auto X, auto Z) {
    axpy<<<16, 128, 0, stream>>>(N, alpha, X.data_handle(), Z.data_handle());
  };

  ctx.task(handle_Y.read(), handle_Z.rw())->*[&](hipStream_t stream, auto Y, auto Z) {
    axpy<<<16, 128, 0, stream>>>(N, alpha, Y.data_handle(), Z.data_handle());
  };

  // Access Ask to use X, Y and Z on the host
  ctx.host_launch(handle_X.read(), handle_Y.read(), handle_Z.read())->*[&](auto X, auto Y, auto Z) {
    for (size_t ind = 0; ind < N; ind++)
    {
      // X unchanged
      EXPECT(fabs(X(ind) - 1.0 * ind) < 0.00001);
      // Y = Y + alpha X
      EXPECT(fabs(Y(ind) - (-3.0 + ind * (2.0 + alpha))) < 0.00001);
      // Z = Z + alpha (X + alpha Y)
      EXPECT(fabs(Z(ind) - ((6.0 - 3 * alpha) + ind * (7.0 + 3 * alpha + alpha * alpha))) < 0.00001);
    }
  };

  ctx.finalize();
}

int main()
{
  run<stream_ctx>();
  run<graph_ctx>();
}
