//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Ensure an error is raised if we try to ask for an unreasonnable
 *        amount of resources in a thread hierarchy spec
 */

#include <cuda/experimental/stf.cuh>

#include <csignal>

using namespace cuda::experimental::stf;

bool should_abort = false;

void cleanupRoutine(int /*unused*/)
{
  if (should_abort)
  {
    exit(EXIT_SUCCESS);
  }
  else
  {
    fprintf(stderr, "Unexpected SIGABRT !\n");
    exit(EXIT_FAILURE);
  }
}

int main()
{
  /* Setup an handler to catch the SIGABRT signal during the programming error */
#ifndef NDEBUG
#  if _CCCL_COMPILER(MSVC)
  signal(SIGABRT, &cleanupRoutine);
#  else // ^^^ _CCCL_COMPILER(MSVC) ^^^ / vvv !_CCCL_COMPILER(MSVC)
  struct sigaction sigabrt_action
  {};
  memset(&sigabrt_action, 0, sizeof(sigabrt_action));
  sigabrt_action.sa_handler = &cleanupRoutine;

  if (sigaction(SIGABRT, &sigabrt_action, nullptr) != 0)
  {
    perror("sigaction SIGABRT");
    exit(EXIT_FAILURE);
  }
#  endif // !_CCCL_COMPILER(MSVC)

  context ctx;

  int X[128];
  auto lX = ctx.logical_data(X);

  should_abort = true;

  // We are asking an unreasonnable amount of threads per block
  auto spec = con(con<128000>());
  ctx.launch(spec, lX.rw())->*[] __device__(auto th, auto X) {
    X[th.rank()] = th.rank();
  };

  assert(0 && "This should not be reached");
  return EXIT_FAILURE;
#endif
}
