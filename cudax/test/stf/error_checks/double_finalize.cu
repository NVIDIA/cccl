//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Ensure an error is detected if we can finalize more than once
 */

#include <cuda/experimental/stf.cuh>

#include <csignal>

using namespace cuda::experimental::stf;

bool should_abort = false;

void cleanupRoutine(int /*unused*/)
{
  if (should_abort)
  {
    exit(EXIT_SUCCESS);
  }
  else
  {
    fprintf(stderr, "Unexpected SIGABRT !\n");
    exit(EXIT_FAILURE);
  }
}

int main()
{
  // This test only works when assert() is enabled in
#ifndef NDEBUG
  /* Setup an handler to catch the SIGABRT signal during the programming error */
#  if _CCCL_COMPILER(MSVC)
  signal(SIGABRT, &cleanupRoutine);
#  else // ^^^ _CCCL_COMPILER(MSVC) ^^^ / vvv !_CCCL_COMPILER(MSVC)
  struct sigaction sigabrt_action
  {};
  memset(&sigabrt_action, 0, sizeof(sigabrt_action));
  sigabrt_action.sa_handler = &cleanupRoutine;

  if (sigaction(SIGABRT, &sigabrt_action, nullptr) != 0)
  {
    perror("sigaction SIGABRT");
    exit(EXIT_FAILURE);
  }
#  endif // !_CCCL_COMPILER(MSVC)

  context ctx;

  const int n = 12;
  double X[n];

  for (int ind = 0; ind < n; ind++)
  {
    X[ind] = 1.0 * ind;
  }

  // This creates a handle that is implicitly a vector of size n
  auto lX = ctx.logical_data(X);

  ctx.task(lX.rw())->*[](hipStream_t, auto) { /* no-op */ };

  ctx.finalize();

  should_abort = true;
  // We cannot call sync twice
  ctx.finalize();

  assert(0 && "This should not be reached");
  return EXIT_FAILURE;
#endif
}
