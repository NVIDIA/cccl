//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Ensure an error is detected when trying to declare a logical data
 *        with a managed memory data place while the data is not in managed
 *        memory
 */

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

#include <csignal>

using namespace cuda::experimental::stf;

bool should_abort = false;

void cleanupRoutine(int /*unused*/)
{
  if (should_abort)
  {
    exit(EXIT_SUCCESS);
  }
  else
  {
    fprintf(stderr, "Unexpected SIGABRT !\n");
    exit(EXIT_FAILURE);
  }
}

int main()
{
  /* Setup an handler to catch the SIGABRT signal during the programming error */
#ifndef NDEBUG
#  if _CCCL_COMPILER(MSVC)
  signal(SIGABRT, &cleanupRoutine);
#  else // ^^^ _CCCL_COMPILER(MSVC) ^^^ / vvv !_CCCL_COMPILER(MSVC)
  struct sigaction sigabrt_action
  {};
  memset(&sigabrt_action, 0, sizeof(sigabrt_action));
  sigabrt_action.sa_handler = &cleanupRoutine;

  if (sigaction(SIGABRT, &sigabrt_action, nullptr) != 0)
  {
    perror("sigaction SIGABRT");
    exit(EXIT_FAILURE);
  }
#  endif // !_CCCL_COMPILER(MSVC)

  stream_ctx ctx;

  logical_data<slice<int>> lX;

  should_abort = true;

  int X[128];
  lX = ctx.logical_data(X, data_place::managed);

  assert(0 && "This should not be reached");
  return EXIT_FAILURE;
#endif
}
