//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Ensure an error is detected when a task uses a logical data from a
 *        different context
 */

#include <cuda/experimental/stf.cuh>

#include <csignal>

using namespace cuda::experimental::stf;

bool should_abort = false;

void cleanupRoutine(int /*unused*/)
{
  if (should_abort)
  {
    exit(EXIT_SUCCESS);
  }
  else
  {
    fprintf(stderr, "Unexpected SIGABRT !\n");
    exit(EXIT_FAILURE);
  }
}

template <typename Ctx, size_t n>
void run(double (&X)[n])
{
  Ctx ctx1;
  auto lX = ctx1.logical_data(X);

  // We are now using lX in the wrong context
  should_abort = true;

  Ctx ctx2;
  ctx2.task(lX.rw())->*[&](hipStream_t /*unused*/, auto /*unused*/) {};

  assert(0 && "This should not be reached");
}

int main()
{
  /* Setup an handler to catch the SIGABRT signal during the programming error */
#if _CCCL_COMPILER(MSVC)
  signal(SIGABRT, &cleanupRoutine);
#else // ^^^ _CCCL_COMPILER(MSVC) ^^^ / vvv !_CCCL_COMPILER(MSVC)
  struct sigaction sigabrt_action
  {};
  memset(&sigabrt_action, 0, sizeof(sigabrt_action));
  sigabrt_action.sa_handler = &cleanupRoutine;

  if (sigaction(SIGABRT, &sigabrt_action, nullptr) != 0)
  {
    perror("sigaction SIGABRT");
    exit(EXIT_FAILURE);
  }
#endif // !_CCCL_COMPILER(MSVC)

  const int n = 12;
  double X[n];

  for (int ind = 0; ind < n; ind++)
  {
    X[ind] = 1.0 * ind;
  }

  // We can't run both stream and graph tests because either will abort the program. So choose one at random.
  srand(static_cast<unsigned>(time(nullptr)));
  if (rand() % 2 == 0)
  {
    run<stream_ctx>(X);
  }
  else
  {
    run<graph_ctx>(X);
  }

  assert(0 && "This should not be reached");
  return EXIT_FAILURE;
}
