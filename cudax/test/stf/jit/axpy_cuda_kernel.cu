#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief JIT version of the AXPY example
 *
 */

#include <cuda/experimental/stf.cuh>
#include <cuda/experimental/__stf/nvrtc/jit_utils.cuh>

using namespace cuda::experimental::stf;

double X0(int i)
{
  return sin((double) i);
}

double Y0(int i)
{
  return cos((double) i);
}

const char *header_template = R"(
)";

const char* axpy_kernel_template = R"(
#include <cuda/experimental/__stf/nvrtc/slice.cuh>

extern "C"
__global__ void %KERNEL_NAME%(%s dynX, %s dynY)
{
  %s X{dynX};
  %s Y{dynY};

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int dimx = blockDim.x * gridDim.x;

  for (size_t i = tidx; i < X.extent(0); i+= dimx)
  {
     Y(i) += %a * X(i);
  }
}

)";


int main()
{
  const size_t N = 16;
  double X[N], Y[N];

  for (size_t i = 0; i < N; i++)
  {
    X[i] = X0(i);
    Y[i] = Y0(i);
  }

  context ctx;
  double alpha = 3.14;

  auto lX = ctx.logical_data(X);
  auto lY = ctx.logical_data(Y);

  /* Compute Y = Y + alpha X */
  ctx.cuda_kernel(lX.read(), lY.rw())->*[alpha](auto dX, auto dY)
  {
    hipFunction_t axpy_kernel = lazy_jit(axpy_kernel_template, get_nvrtc_flags(), header_template, jit_reduced_type_name(dX), jit_reduced_type_name(dY), jit_typename(dX), jit_typename(dY), alpha);
    return cuda_kernel_desc{axpy_kernel, 1152, 160, 0, jit_reduce(dX), jit_reduce(dY)};
  };

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    assert(fabs(Y[i] - (Y0(i) + alpha * X0(i))) < 0.0001);
    assert(fabs(X[i] - X0(i)) < 0.0001);
  }
}
