#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief An example solving heat equation with finite differences using a
 * CUDA kernel
 */

#include <cuda/experimental/__stf/nvrtc/jit_utils.cuh>
#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

void dump_iter(slice<const double, 2> sUn, int iter)
{
  /* Create a binary file in the PPM format */
  char name[64];
  snprintf(name, 64, "heat_%06d.ppm", iter);
  FILE* f = fopen(name, "wb");
  fprintf(f, "P6\n%zu %zu\n255\n", sUn.extent(0), sUn.extent(1));
  for (size_t j = 0; j < sUn.extent(1); j++)
  {
    for (size_t i = 0; i < sUn.extent(0); i++)
    {
      int v = (int) (255.0 * sUn(i, j) / 100.0);
      // we assume values between 0.0 and 100.0 : max value is in red,
      // min is in blue
      unsigned char color[3];
      color[0] = static_cast<char>(v); /* red */
      color[1] = static_cast<char>(0); /* green */
      color[2] = static_cast<char>(255 - v); /* blue */
      fwrite(color, 1, 3, f);
    }
  }
  fclose(f);
}

const char* header_template = R"(
#include <cuda/experimental/__stf/nvrtc/slice.cuh>
#include <cuda/experimental/__stf/places/inner_shape.cuh>

using namespace cuda::experimental::stf;
)";

const char* heat_kernel_template = R"(
const double c = %a;
const double dx2 = %a;
const double dy2 = %a;

extern "C"
__global__ void %KERNEL_NAME%(slice<const double, 2> dyn_U, slice<double, 2> dyn_U1)
{
  %s U{dyn_U};
  %s U1{dyn_U1};

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;
  int dimx = blockDim.x * gridDim.x;
  int dimy = blockDim.y * gridDim.y;

  for (size_t i = tidx + 1; i < U.extent(0)-1; i+= dimx)
    for (size_t j = tidy + 1; j < U.extent(1)-1; j += dimy)
    {
      U1(i, j) = U(i, j) + c * ((U(i - 1, j) - 2 * U(i, j) + U(i + 1, j)) / dx2 + (U(i, j - 1) - 2 * U(i, j) + U(i, j + 1)) / dy2);
    }
}

)";

int main()
{
  context ctx;

  const size_t N = 800;

  auto lU  = ctx.logical_data(shape_of<slice<double, 2>>(N, N));
  auto lU1 = ctx.logical_data(lU.shape());

  // Initialize the Un field with boundary conditions, and a disk at a lower
  // temperature in the middle.
  parallel_for_scope_jit(ctx, exec_place::current_device(), lU.shape(), lU.write())->*[]() {
    const char* body =
      R"((size_t i, size_t j, auto U) {
            double rad = U.extent(0) / 8.0;
            double dx  = (double) i - U.extent(0) / 2;
            double dy  = (double) j - U.extent(1) / 2;

            U(i, j) = (dx * dx + dy * dy < rad * rad) ? 100.0 : 0.0;

            /* Set up boundary conditions */
            if (j == 0.0)
            {
              U(i, j) = 100.0;
            }
            if (j == U.extent(1) - 1)
            {
              U(i, j) = 0.0;
            }
            if (i == 0.0)
            {
              U(i, j) = 0.0;
            }
            if (i == U.extent(0) - 1)
            {
              U(i, j) = 0.0;
            }
       }
    )";
    return ::std::pair(::std::string(header_template), ::std::string(body));
  };

  // diffusion constant
  double a = 0.5;

  double dx  = 0.1;
  double dy  = 0.1;
  double dx2 = dx * dx;
  double dy2 = dy * dy;

  // time step
  double dt = dx2 * dy2 / (2.0 * a * (dx2 + dy2));

  double c = a * dt;

  int nsteps     = 1000;
  int image_freq = 100;

  for (int iter = 0; iter < nsteps; iter++)
  {
    if (image_freq > 0 && iter % image_freq == 0)
    {
      // Dump Un in a PPM file
      ctx.host_launch(lU.read())->*[=](auto U) {
        dump_iter(U, iter);
      };
    }

    // Update Un using Un1 value with a finite difference scheme
    parallel_for_scope_jit(ctx, exec_place::current_device(), inner<1>(lU.shape()), lU.read(), lU1.write())
        ->*[c, dx2, dy2]() {
              ::std::ostringstream body_stream;
              body_stream
                << R"(
      (size_t i, size_t j, auto U, auto U1) {
        const double c = )"
                << c << R"(;
        const double dx2 = )"
                << dx2 << R"(;
        const double dy2 = )"
                << dy2 << R"(;
        U1(i, j) = U(i, j) + c * ((U(i - 1, j) - 2 * U(i, j) + U(i + 1, j)) / dx2
                               + (U(i, j - 1) - 2 * U(i, j) + U(i, j + 1)) / dy2);
      })";
              return ::std::pair(::std::string(header_template), body_stream.str());
            };

    ::std::swap(lU, lU1);
  }

  ctx.finalize();
  return 0;
}
