//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

int main()
{
  context ctx;

  int nqpoints = 3;
  auto ltoken  = ctx.logical_token();

  ctx.parallel_for(exec_place::host, box(5), ltoken.read())->*[nqpoints] __host__ __device__(size_t, void_interface) {
    _CCCL_ASSERT(nqpoints == 3, "invalid value");
  };

  ctx.finalize();
}
