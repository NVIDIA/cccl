#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Experiment with local context nesting
 *
 */

#include <cuda/experimental/stf.cuh>

#include <thread>

#include "cuda/experimental/__stf/utility/stackable_ctx.cuh"

using namespace cuda::experimental::stf;

void worker(
  stackable_ctx sctx, int main_head, stackable_logical_data<slice<int>> lAi, stackable_logical_data<slice<int>> lB)
{
  sctx.set_head_offset(main_head);
  sctx.push();

  auto lC = sctx.logical_data_no_export(lB.shape());

  sctx.parallel_for(lC.shape(), lC.write(), lB.read())->*[] __device__(size_t k, auto c, auto b) {
    c(k) = b(k);
  };

  sctx.parallel_for(lAi.shape(), lAi.write())->*[] __device__(size_t k, auto ai) {
    ai(k) = k;
  };

  sctx.parallel_for(lAi.shape(), lAi.rw(), lC.read())->*[] __device__(size_t k, auto ai, auto c) {
    ai(k) += int(sin(cos(cos(10.0 * c(k)))));
  };

  sctx.pop();
}

int main()
{
  const size_t N = 1024000;
  stackable_ctx sctx;

  int array[N];
  for (size_t i = 0; i < N; i++)
  {
    array[i] = 1 + i * i;
  }

  auto lB = sctx.logical_data(array);

  lB.set_read_only();

  int main_head = sctx.get_head_offset();

  ::std::vector<stackable_logical_data<slice<int>>> lA;

  const int NTHREADS = 8;

  for (int i = 0; i < NTHREADS; ++i)
  {
    lA.push_back(sctx.logical_data(shape_of<slice<int>>(N)));
  }

  for (int k = 0; k < 30; k++)
  {
    ::std::vector<::std::thread> threads;

    for (int i = 0; i < NTHREADS; ++i)
    {
      threads.emplace_back(worker, sctx, main_head, lA[i], lB);
    }

    for (int i = 0; i < NTHREADS; ++i)
    {
      threads[i].join();
    }
  }

  sctx.finalize();
}
