#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>
#include <cuda/experimental/__stf/stream/stream_ctx.cuh>

using namespace cuda::experimental::stf;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

template <typename Ctx>
void run()
{
  Ctx ctx;
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  int numElements = 50000;
  size_t size     = numElements * sizeof(float);
  // printf("[Vector addition of %d elements]\n", numElements);

  // Allocate the host input vector A
  float* h_A = (float*) malloc(size);

  // Allocate the host input vector B
  float* h_B = (float*) malloc(size);

  // Allocate the host output vector C
  float* h_C = (float*) malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL)
  {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i)
  {
    h_A[i] = rand() / (float) RAND_MAX;
    h_B[i] = rand() / (float) RAND_MAX;
  }

  auto A_handle = ctx.logical_data(h_A, numElements);
  auto B_handle = ctx.logical_data(h_B, numElements);
  auto C_handle = ctx.logical_data(h_C, numElements);

  auto t = ctx.task(A_handle.read(), B_handle.read(), C_handle.rw());
  t->*[&](hipStream_t stream, auto d_A, auto d_B, auto d_C) {
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid   = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      d_A.data_handle(), d_B.data_handle(), d_C.data_handle(), numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  };

  auto t_host = ctx.host_launch(A_handle.read(), B_handle.read(), C_handle.read());
  t_host->*[&](auto, auto, auto) {
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
      if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
      {
        fprintf(stderr, "Result verification failed at element %d!\n", i);
        exit(EXIT_FAILURE);
      }
    }
  };

  ctx.finalize();

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
}

/**
 * Host main routine
 */
int main(void)
{
  run<stream_ctx>();
  run<graph_ctx>();
}
