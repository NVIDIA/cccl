#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 * @brief Test explicit uses of the API to change epoch and create a sequence
 *        of CUDA graphs
 */

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>

using namespace cuda::experimental::stf;

int main()
{
  graph_ctx ctx;

  const size_t N     = 8;
  const size_t NITER = 2;

  double A[N];
  for (size_t i = 0; i < N; i++)
  {
    A[i] = 1.0 * i;
  }

  auto lA = ctx.logical_data(A);

  for (size_t k = 0; k < NITER; k++)
  {
    ctx.parallel_for(blocked_partition(), exec_place::current_device(), lA.shape(), lA.rw())
        ->*[] __host__ __device__(size_t i, slice<double> A) { A(i) = cos(A(i)); };

    ctx.change_epoch();
  }

  ctx.finalize();

  for (size_t i = 0; i < N; i++)
  {
    double Ai_ref = 1.0 * i;
    for (size_t k = 0; k < NITER; k++)
    {
      Ai_ref = cos(Ai_ref);
    }

    EXPECT(fabs(A[i] - Ai_ref) < 0.01);
  }
}
