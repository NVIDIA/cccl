#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

/**
 * @file
 *
 * @brief Add tasks to a user-provided child graph from a while loop
 *
 * Adapted from https://developer.nvidia.com/blog/dynamic-control-flow-in-cuda-graphs-with-conditional-nodes/
 */

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>

using namespace cuda::experimental::stf;

#if _CCCL_CTK_AT_LEAST(12, 4)
__global__ void dummy() {}

__device__ int counter = 5;

__global__ void setHandle(cudaGraphConditionalHandle handle)
{
  unsigned int value = 0;
  // We could perform some work here and set value based on the result of that work.
  if (counter-- > 0)
  {
    // Set ‘value’ to non-zero if we want the conditional body to execute
    value = 1;
  }
  cudaGraphSetConditional(handle, value);
}
#endif // _CCCL_CTK_AT_LEAST(12, 4)

int main()
{
#if _CCCL_CTK_BELOW(12, 4)
  fprintf(stderr, "Waiving test: conditional nodes are only available since CUDA 12.4.\n");
#else
  hipStream_t stream;

  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  hipGraph_t graph;
  hipGraphNode_t kernelNode, conditionalNode;
  void* kernelArgs[1];

  hipGraphCreate(&graph, 0);

  cudaGraphConditionalHandle handle;
  cudaGraphConditionalHandleCreate(&handle, graph);

  // Use a kernel upstream of the conditional to set the handle value
  hipGraphNodeParams kParams = {};
  kParams.type                = hipGraphNodeTypeKernel;
  kParams.kernel.func         = (void*) setHandle;
  kParams.kernel.gridDim.x = kParams.kernel.gridDim.y = kParams.kernel.gridDim.z = 1;
  kParams.kernel.blockDim.x = kParams.kernel.blockDim.y = kParams.kernel.blockDim.z = 1;
  kParams.kernel.kernelParams                                                       = kernelArgs;
  kernelArgs[0]                                                                     = &handle;
  hipGraphAddNode(&kernelNode, graph, NULL, 0, &kParams);

  hipGraphNodeParams cParams = {};
  cParams.type                = hipGraphNodeTypeConditional;
  cParams.conditional.handle  = handle;
  cParams.conditional.type    = cudaGraphCondTypeSwitch;
  cParams.conditional.size    = 3;
  hipGraphAddNode(&conditionalNode, graph, &kernelNode, 1, &cParams);

  hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

  graph_ctx ctx(bodyGraph);

  auto lX = ctx.token();
  auto lY = ctx.token();
  auto lZ = ctx.token();

  ctx.cuda_kernel(lX.write())->*[]() {
    return cuda_kernel_desc{dummy, 1, 1, 0};
  };

  ctx.cuda_kernel(lX.read(), lY.write())->*[]() {
    return cuda_kernel_desc{dummy, 1, 1, 0};
  };

  ctx.cuda_kernel(lX.read(), lZ.write())->*[]() {
    return cuda_kernel_desc{dummy, 1, 1, 0};
  };

  ctx.cuda_kernel(lY.rw(), lZ.rw())->*[]() {
    return cuda_kernel_desc{dummy, 1, 1, 0};
  };

  ctx.finalize_as_graph();

  hipGraphExec_t graphExec = NULL;
  cuda_safe_call(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
  cuda_safe_call(hipGraphLaunch(graphExec, stream));
  cuda_safe_call(hipStreamSynchronize(stream));
  cuda_safe_call(hipGraphDebugDotPrint(graph, "test-while.dot", hipGraphDebugDotFlags(0)));
#endif // !_CCCL_CTK_BELOW(12, 4)
}
