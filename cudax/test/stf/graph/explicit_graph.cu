#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

//! @file
//! @brief Add tasks to a user-provided graph

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>

using namespace cuda::experimental::stf;

__global__ void dummy() {}

int main()
{
  hipGraph_t graph;
  hipGraphExec_t graphExec = NULL;
  hipStream_t stream;

  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  cuda_safe_call(hipGraphCreate(&graph, 0));

  graph_ctx ctx(graph);

  auto lX = ctx.token();
  auto lY = ctx.token();
  auto lZ = ctx.token();

  ctx.task(lX.write())->*[](hipStream_t s) {
    dummy<<<1, 1, 0, s>>>();
  };

  ctx.task(lX.read(), lY.write())->*[](hipStream_t s) {
    dummy<<<1, 1, 0, s>>>();
  };

  ctx.task(lX.read(), lZ.write())->*[](hipStream_t s) {
    dummy<<<1, 1, 0, s>>>();
  };

  ctx.task(lY.rw(), lZ.rw())->*[](hipStream_t s) {
    dummy<<<1, 1, 0, s>>>();
  };

  ctx.finalize_as_graph();

  cuda_safe_call(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
  cuda_safe_call(hipGraphLaunch(graphExec, stream));
  cuda_safe_call(hipStreamSynchronize(stream));
}
