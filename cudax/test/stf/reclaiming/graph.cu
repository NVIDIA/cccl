#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/__stf/graph/graph_ctx.cuh>

#if !_CCCL_COMPILER(MSVC)
using namespace cuda::experimental::stf;

__global__ void kernel()
{
  // No-op
}
#endif // !_CCCL_COMPILER(MSVC)

int main([[maybe_unused]] int argc, [[maybe_unused]] char** argv)
{
// TODO fix setenv
#if !_CCCL_COMPILER(MSVC)
  int nblocks       = 4;
  size_t block_size = 1024 * 1024;

  if (argc > 1)
  {
    nblocks = atoi(argv[1]);
  }

  if (argc > 2)
  {
    block_size = atoi(argv[2]);
  }

  // At most 1 buffer is allocated at the same time
  setenv("MAX_ALLOC_CNT", "1", 0);

  graph_ctx ctx;

  ::std::vector<logical_data<slice<char>>> handles(nblocks);

  char* h_buffer = new char[nblocks * block_size];

  for (int i = 0; i < nblocks; i++)
  {
    handles[i] = ctx.logical_data(make_slice(&h_buffer[i * block_size], block_size));
    handles[i].set_symbol("D_" + std::to_string(i));
  }

  // We only 2 buffers, we are forced to reuse the buffer from D0 for D2
  for (int i = 0; i < 3; i++)
  {
    ctx.task(handles[i % nblocks].rw())->*[&](hipStream_t s, auto /*unused*/) {
      kernel<<<1, 1, 0, s>>>();
    };
  }

  ctx.submit();

  if (argc > 3)
  {
    std::cout << "Generating DOT output in " << argv[3] << std::endl;
    ctx.print_to_dot(argv[1]);
  }

  ctx.finalize();
#endif // !_CCCL_COMPILER(MSVC)
}
