#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDASTF in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/stf.cuh>

using namespace cuda::experimental::stf;

static __global__ void cuda_sleep_kernel(long long int clock_cnt)
{
  long long int start_clock  = clock64();
  long long int clock_offset = 0;
  while (clock_offset < clock_cnt)
  {
    clock_offset = clock64() - start_clock;
  }
}

void cuda_sleep(double ms, hipStream_t stream)
{
  int device;
  hipGetDevice(&device);

  // hipDeviceAttributeClockRate: Peak clock frequency in kilohertz;
  int clock_rate;
  hipDeviceGetAttribute(&clock_rate, hipDeviceAttributeClockRate, device);

  long long int clock_cnt = (long long int) (ms * clock_rate);
  cuda_sleep_kernel<<<1, 1, 0, stream>>>(clock_cnt);
}

template <typename Ctx_t>
void run(int NTASKS, int ms)
{
  Ctx_t ctx;

  int dummy[1];
  auto handle = ctx.logical_data(dummy);

  hipEvent_t start, stop;
  cuda_safe_call(hipEventCreate(&start));
  cuda_safe_call(hipEventCreate(&stop));

  // warm-up
  ctx.task(handle.rw())->*[ms](hipStream_t stream, auto) {
    cuda_sleep(ms, stream);
  };

  cuda_safe_call(hipEventRecord(start, ctx.task_fence()));

  for (int iter = 0; iter < NTASKS; iter++)
  {
    ctx.task(handle.rw())->*[ms](hipStream_t stream, auto) {
      cuda_sleep(ms, stream);
    };
  }

  cuda_safe_call(hipEventRecord(stop, ctx.task_fence()));

  ctx.finalize();

  [[maybe_unused]] float elapsed;
  cuda_safe_call(hipEventElapsedTime(&elapsed, start, stop));

  [[maybe_unused]] float expected = 1.0f * NTASKS * ms;

  /* We cannot really expect this measurement to be accurate because the
   * thread(s) executing the code might be preempted on a system with a high load
   * (as during unit tests). So the best we can expect is that the elapsed time
   * is larger than the sleep time, but event the timer on the GPU is not
   * perfectly accurate so we do not make any strict assumptions about the
   * test, and just keep this test to demonstrate how to use the mechanisms,
   * and ensure they are functional . */
  // EXPECT(elapsed >= expected);
}

int main(int argc, char** argv)
{
  int NTASKS = 25;
  int ms     = 200;

  if (argc > 1)
  {
    NTASKS = atoi(argv[1]);
  }

  if (argc > 2)
  {
    ms = atoi(argv[2]);
  }

  run<context>(NTASKS, ms);
  run<stream_ctx>(NTASKS, ms);
  run<graph_ctx>(NTASKS, ms);
}
