#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//
#define LIBCUDACXX_ENABLE_EXCEPTIONS
#include <cuda/experimental/device.cuh>

#include "../hierarchy/testing_common.cuh"
#include "cuda/std/__type_traits/is_same.h"

void device_smoke_test()
{
  using cudax::device;

  SECTION("Device count")
  {
    int count = device::count();
    CUDAX_REQUIRE(count > 0);
  }

  SECTION("Attributes")
  {
    device dev0(0);

    SECTION("max_threads_per_block")
    {
      STATIC_REQUIRE(::hipDeviceAttributeMaxThreadsPerBlock == device::attrs::max_threads_per_block);
      STATIC_REQUIRE(::cuda::std::is_same_v<int, device::attrs::max_threads_per_block_t::type>);
      STATIC_REQUIRE(::cuda::std::is_same_v<int, device::attr_result_t<device::attrs::max_threads_per_block>>);

      auto max = dev0.attr(device::attrs::max_threads_per_block);
      STATIC_REQUIRE(::cuda::std::is_same_v<decltype(max), int>);
      CUDAX_REQUIRE(max > 0);
      CUDAX_REQUIRE(max == dev0.attr<hipDeviceAttributeMaxThreadsPerBlock>());
    }

    SECTION("compute_mode")
    {
      STATIC_REQUIRE(::hipDeviceAttributeComputeMode == device::attrs::compute_mode);
      STATIC_REQUIRE(::cuda::std::is_same_v<::hipComputeMode, device::attrs::compute_mode_t::type>);
      STATIC_REQUIRE(::cuda::std::is_same_v<::hipComputeMode, device::attr_result_t<device::attrs::compute_mode>>);
      STATIC_REQUIRE(::hipComputeModeDefault == device::attrs::compute_mode._default);
      STATIC_REQUIRE(::hipComputeModeProhibited == device::attrs::compute_mode.prohibited);
      STATIC_REQUIRE(::hipComputeModeExclusiveProcess == device::attrs::compute_mode.exclusive_process);

      auto mode = dev0.attr(device::attrs::compute_mode);
      STATIC_REQUIRE(::cuda::std::is_same_v<decltype(mode), ::hipComputeMode>);
      CUDAX_REQUIRE((mode == device::attrs::compute_mode._default || //
                     mode == device::attrs::compute_mode.prohibited || //
                     mode == device::attrs::compute_mode.exclusive_process));
      CUDAX_REQUIRE(mode == dev0.attr<::hipDeviceAttributeComputeMode>());
    }
  }
}

TEST_CASE("Smoke", "[device]")
{
  device_smoke_test();
}
