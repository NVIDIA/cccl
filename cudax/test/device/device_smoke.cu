//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/std/__type_traits/is_same.h>

#include <cuda/experimental/device.cuh>

#include <utility.cuh>

namespace
{
template <const auto& Attr, ::hipDeviceAttribute_t ExpectedAttr, class ExpectedResult>
[[maybe_unused]] auto test_device_attribute()
{
  cudax::device_ref dev0(0);
  STATIC_REQUIRE(Attr == ExpectedAttr);
  STATIC_REQUIRE(::cuda::std::is_same_v<cudax::device::attribute_result_t<Attr>, ExpectedResult>);

  auto result = dev0.attribute(Attr);
  STATIC_REQUIRE(::cuda::std::is_same_v<decltype(result), ExpectedResult>);
  CUDAX_REQUIRE(result == dev0.attribute<ExpectedAttr>());
  CUDAX_REQUIRE(result == Attr(dev0));
  return result;
}
} // namespace

C2H_CCCLRT_TEST("Smoke", "[device]")
{
  using cudax::device;
  using cudax::device_ref;

  SECTION("Compare")
  {
    CUDAX_REQUIRE(device_ref{0} == device_ref{0});
    CUDAX_REQUIRE(device_ref{0} == 0);
    CUDAX_REQUIRE(0 == device_ref{0});
    CUDAX_REQUIRE(device_ref{1} != device_ref{0});
    CUDAX_REQUIRE(device_ref{1} != 2);
    CUDAX_REQUIRE(1 != device_ref{2});
  }

  SECTION("Attributes")
  {
    ::test_device_attribute<device::attributes::max_threads_per_block, ::hipDeviceAttributeMaxThreadsPerBlock, int>();
    ::test_device_attribute<device::attributes::max_block_dim_x, ::hipDeviceAttributeMaxBlockDimX, int>();
    ::test_device_attribute<device::attributes::max_block_dim_y, ::hipDeviceAttributeMaxBlockDimY, int>();
    ::test_device_attribute<device::attributes::max_block_dim_z, ::hipDeviceAttributeMaxBlockDimZ, int>();
    ::test_device_attribute<device::attributes::max_grid_dim_x, ::hipDeviceAttributeMaxGridDimX, int>();
    ::test_device_attribute<device::attributes::max_grid_dim_y, ::hipDeviceAttributeMaxGridDimY, int>();
    ::test_device_attribute<device::attributes::max_grid_dim_z, ::hipDeviceAttributeMaxGridDimZ, int>();
    ::test_device_attribute<device::attributes::max_shared_memory_per_block, ::hipDeviceAttributeMaxSharedMemoryPerBlock, int>();
    ::test_device_attribute<device::attributes::total_constant_memory, ::hipDeviceAttributeTotalConstantMemory, int>();
    ::test_device_attribute<device::attributes::warp_size, ::hipDeviceAttributeWarpSize, int>();
    ::test_device_attribute<device::attributes::max_pitch, ::hipDeviceAttributeMaxPitch, int>();
    ::test_device_attribute<device::attributes::max_texture_1d_width, ::hipDeviceAttributeMaxTexture1DWidth, int>();
    ::test_device_attribute<device::attributes::max_texture_1d_linear_width, ::hipDeviceAttributeMaxTexture1DLinear, int>();
    ::test_device_attribute<device::attributes::max_texture_1d_mipmapped_width,
                            ::hipDeviceAttributeMaxTexture1DMipmap,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_width, ::hipDeviceAttributeMaxTexture2DWidth, int>();
    ::test_device_attribute<device::attributes::max_texture_2d_height, ::hipDeviceAttributeMaxTexture2DHeight, int>();
    ::test_device_attribute<device::attributes::max_texture_2d_linear_width, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<device::attributes::max_texture_2d_linear_height,
                            ::hipDeviceAttributeMaxTexture2DLinear,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_linear_pitch, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<device::attributes::max_texture_2d_mipmapped_width,
                            ::hipDeviceAttributeMaxTexture2DMipmap,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_mipmapped_height,
                            ::hipDeviceAttributeMaxTexture2DMipmap,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_3d_width, ::hipDeviceAttributeMaxTexture3DWidth, int>();
    ::test_device_attribute<device::attributes::max_texture_3d_height, ::hipDeviceAttributeMaxTexture3DHeight, int>();
    ::test_device_attribute<device::attributes::max_texture_3d_depth, ::hipDeviceAttributeMaxTexture3DDepth, int>();
    ::test_device_attribute<device::attributes::max_texture_3d_width_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attributes::max_texture_3d_height_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attributes::max_texture_3d_depth_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attributes::max_texture_cubemap_width, ::hipDeviceAttributeMaxTextureCubemap, int>();
    ::test_device_attribute<device::attributes::max_texture_1d_layered_width,
                            ::hipDeviceAttributeMaxTexture1DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_1d_layered_layers,
                            ::cudaDevAttrMaxTexture1DLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_layered_width,
                            ::hipDeviceAttributeMaxTexture2DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_layered_height,
                            ::hipDeviceAttributeMaxTexture2DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_2d_layered_layers,
                            ::cudaDevAttrMaxTexture2DLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_cubemap_layered_width,
                            ::hipDeviceAttributeMaxTextureCubemapLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_texture_cubemap_layered_layers,
                            ::cudaDevAttrMaxTextureCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_1d_width, ::hipDeviceAttributeMaxSurface1D, int>();
    ::test_device_attribute<device::attributes::max_surface_2d_width, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<device::attributes::max_surface_2d_height, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<device::attributes::max_surface_3d_width, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attributes::max_surface_3d_height, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attributes::max_surface_3d_depth, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attributes::max_surface_1d_layered_width,
                            ::hipDeviceAttributeMaxSurface1DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_1d_layered_layers,
                            ::cudaDevAttrMaxSurface1DLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_2d_layered_width,
                            ::hipDeviceAttributeMaxSurface2DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_2d_layered_height,
                            ::hipDeviceAttributeMaxSurface2DLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_2d_layered_layers,
                            ::cudaDevAttrMaxSurface2DLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_cubemap_width, ::hipDeviceAttributeMaxSurfaceCubemap, int>();
    ::test_device_attribute<device::attributes::max_surface_cubemap_layered_width,
                            ::hipDeviceAttributeMaxSurfaceCubemapLayered,
                            int>();
    ::test_device_attribute<device::attributes::max_surface_cubemap_layered_layers,
                            ::cudaDevAttrMaxSurfaceCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<device::attributes::max_registers_per_block, ::hipDeviceAttributeMaxRegistersPerBlock, int>();
    ::test_device_attribute<device::attributes::clock_rate, ::hipDeviceAttributeClockRate, int>();
    ::test_device_attribute<device::attributes::texture_alignment, ::hipDeviceAttributeTextureAlignment, int>();
    ::test_device_attribute<device::attributes::texture_pitch_alignment, ::hipDeviceAttributeTexturePitchAlignment, int>();
    ::test_device_attribute<device::attributes::gpu_overlap, ::hipDeviceAttributeAsyncEngineCount, bool>();
    ::test_device_attribute<device::attributes::multiprocessor_count, ::hipDeviceAttributeMultiprocessorCount, int>();
    ::test_device_attribute<device::attributes::kernel_exec_timeout, ::hipDeviceAttributeKernelExecTimeout, bool>();
    ::test_device_attribute<device::attributes::integrated, ::hipDeviceAttributeIntegrated, bool>();
    ::test_device_attribute<device::attributes::can_map_host_memory, ::hipDeviceAttributeCanMapHostMemory, bool>();
    ::test_device_attribute<device::attributes::compute_mode, ::hipDeviceAttributeComputeMode, ::hipComputeMode>();
    ::test_device_attribute<device::attributes::concurrent_kernels, ::hipDeviceAttributeConcurrentKernels, bool>();
    ::test_device_attribute<device::attributes::ecc_enabled, ::hipDeviceAttributeEccEnabled, bool>();
    ::test_device_attribute<device::attributes::pci_bus_id, ::hipDeviceAttributePciBusId, int>();
    ::test_device_attribute<device::attributes::pci_device_id, ::hipDeviceAttributePciDeviceId, int>();
    ::test_device_attribute<device::attributes::tcc_driver, ::hipDeviceAttributeTccDriver, bool>();
    ::test_device_attribute<device::attributes::memory_clock_rate, ::hipDeviceAttributeMemoryClockRate, int>();
    ::test_device_attribute<device::attributes::global_memory_bus_width, ::hipDeviceAttributeMemoryBusWidth, int>();
    ::test_device_attribute<device::attributes::l2_cache_size, ::hipDeviceAttributeL2CacheSize, int>();
    ::test_device_attribute<device::attributes::max_threads_per_multiprocessor,
                            ::hipDeviceAttributeMaxThreadsPerMultiProcessor,
                            int>();
    ::test_device_attribute<device::attributes::unified_addressing, ::hipDeviceAttributeUnifiedAddressing, bool>();
    ::test_device_attribute<device::attributes::compute_capability_major, ::hipDeviceAttributeComputeCapabilityMajor, int>();
    ::test_device_attribute<device::attributes::compute_capability_minor, ::hipDeviceAttributeComputeCapabilityMinor, int>();
    ::test_device_attribute<device::attributes::stream_priorities_supported,
                            ::hipDeviceAttributeStreamPrioritiesSupported,
                            bool>();
    ::test_device_attribute<device::attributes::global_l1_cache_supported, ::hipDeviceAttributeGlobalL1CacheSupported, bool>();
    ::test_device_attribute<device::attributes::local_l1_cache_supported, ::hipDeviceAttributeLocalL1CacheSupported, bool>();
    ::test_device_attribute<device::attributes::max_shared_memory_per_multiprocessor,
                            ::hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                            int>();
    ::test_device_attribute<device::attributes::max_registers_per_multiprocessor,
                            ::hipDeviceAttributeMaxRegistersPerMultiprocessor,
                            int>();
    ::test_device_attribute<device::attributes::managed_memory, ::hipDeviceAttributeManagedMemory, bool>();
    ::test_device_attribute<device::attributes::is_multi_gpu_board, ::hipDeviceAttributeIsMultiGpuBoard, bool>();
    ::test_device_attribute<device::attributes::multi_gpu_board_group_id, ::hipDeviceAttributeMultiGpuBoardGroupID, int>();
    ::test_device_attribute<device::attributes::host_native_atomic_supported,
                            ::hipDeviceAttributeHostNativeAtomicSupported,
                            bool>();
    ::test_device_attribute<device::attributes::single_to_double_precision_perf_ratio,
                            ::hipDeviceAttributeSingleToDoublePrecisionPerfRatio,
                            int>();
    ::test_device_attribute<device::attributes::pageable_memory_access, ::hipDeviceAttributePageableMemoryAccess, bool>();
    ::test_device_attribute<device::attributes::concurrent_managed_access, ::hipDeviceAttributeConcurrentManagedAccess, bool>();
    ::test_device_attribute<device::attributes::compute_preemption_supported,
                            ::hipDeviceAttributeComputePreemptionSupported,
                            bool>();
    ::test_device_attribute<device::attributes::can_use_host_pointer_for_registered_mem,
                            ::hipDeviceAttributeCanUseHostPointerForRegisteredMem,
                            bool>();
    ::test_device_attribute<device::attributes::cooperative_launch, ::hipDeviceAttributeCooperativeLaunch, bool>();
    ::test_device_attribute<device::attributes::can_flush_remote_writes, ::cudaDevAttrCanFlushRemoteWrites, bool>();
    ::test_device_attribute<device::attributes::host_register_supported, ::hipDeviceAttributeHostRegisterSupported, bool>();
    ::test_device_attribute<device::attributes::pageable_memory_access_uses_host_page_tables,
                            ::hipDeviceAttributePageableMemoryAccessUsesHostPageTables,
                            bool>();
    ::test_device_attribute<device::attributes::direct_managed_mem_access_from_host,
                            ::hipDeviceAttributeDirectManagedMemAccessFromHost,
                            bool>();
    ::test_device_attribute<device::attributes::max_shared_memory_per_block_optin,
                            ::hipDeviceAttributeSharedMemPerBlockOptin,
                            int>();
    ::test_device_attribute<device::attributes::max_blocks_per_multiprocessor,
                            ::hipDeviceAttributeMaxBlocksPerMultiprocessor,
                            int>();
    ::test_device_attribute<device::attributes::max_persisting_l2_cache_size,
                            ::cudaDevAttrMaxPersistingL2CacheSize,
                            int>();
    ::test_device_attribute<device::attributes::max_access_policy_window_size,
                            ::cudaDevAttrMaxAccessPolicyWindowSize,
                            int>();
    ::test_device_attribute<device::attributes::reserved_shared_memory_per_block,
                            ::cudaDevAttrReservedSharedMemoryPerBlock,
                            int>();
    ::test_device_attribute<device::attributes::sparse_cuda_array_supported,
                            ::cudaDevAttrSparseCudaArraySupported,
                            bool>();
    ::test_device_attribute<device::attributes::host_register_read_only_supported,
                            ::cudaDevAttrHostRegisterReadOnlySupported,
                            bool>();
    ::test_device_attribute<device::attributes::memory_pools_supported, ::hipDeviceAttributeMemoryPoolsSupported, bool>();
    ::test_device_attribute<device::attributes::gpu_direct_rdma_supported, ::cudaDevAttrGPUDirectRDMASupported, bool>();
    ::test_device_attribute<device::attributes::gpu_direct_rdma_flush_writes_options,
                            ::cudaDevAttrGPUDirectRDMAFlushWritesOptions,
                            ::hipFlushGPUDirectRDMAWritesOptions>();
    ::test_device_attribute<device::attributes::gpu_direct_rdma_writes_ordering,
                            ::cudaDevAttrGPUDirectRDMAWritesOrdering,
                            ::hipGPUDirectRDMAWritesOrdering>();
    ::test_device_attribute<device::attributes::memory_pool_supported_handle_types,
                            ::cudaDevAttrMemoryPoolSupportedHandleTypes,
                            ::hipMemAllocationHandleType>();
    ::test_device_attribute<device::attributes::deferred_mapping_cuda_array_supported,
                            ::cudaDevAttrDeferredMappingCudaArraySupported,
                            bool>();
    ::test_device_attribute<device::attributes::ipc_event_support, ::cudaDevAttrIpcEventSupport, bool>();

#if CUDART_VERSION >= 12020
    ::test_device_attribute<device::attributes::numa_config, ::cudaDevAttrNumaConfig, ::cudaDeviceNumaConfig>();
    ::test_device_attribute<device::attributes::numa_id, ::cudaDevAttrNumaId, int>();
#endif

    SECTION("compute_mode")
    {
      STATIC_REQUIRE(::hipComputeModeDefault == device::attributes::compute_mode.default_mode);
      STATIC_REQUIRE(::hipComputeModeProhibited == device::attributes::compute_mode.prohibited_mode);
      STATIC_REQUIRE(::hipComputeModeExclusiveProcess == device::attributes::compute_mode.exclusive_process_mode);

      auto mode = device_ref(0).attribute(device::attributes::compute_mode);
      CUDAX_REQUIRE((mode == device::attributes::compute_mode.default_mode || //
                     mode == device::attributes::compute_mode.prohibited_mode || //
                     mode == device::attributes::compute_mode.exclusive_process_mode));
    }

    SECTION("gpu_direct_rdma_flush_writes_options")
    {
      STATIC_REQUIRE(
        ::hipFlushGPUDirectRDMAWritesOptionHost == device::attributes::gpu_direct_rdma_flush_writes_options.host);
      STATIC_REQUIRE(
        ::hipFlushGPUDirectRDMAWritesOptionMemOps == device::attributes::gpu_direct_rdma_flush_writes_options.mem_ops);

      auto options = device_ref(0).attribute(device::attributes::gpu_direct_rdma_flush_writes_options);
      CUDAX_REQUIRE((options == device::attributes::gpu_direct_rdma_flush_writes_options.host || //
                     options == device::attributes::gpu_direct_rdma_flush_writes_options.mem_ops));
    }

    SECTION("gpu_direct_rdma_writes_ordering")
    {
      STATIC_REQUIRE(::hipGPUDirectRDMAWritesOrderingNone == device::attributes::gpu_direct_rdma_writes_ordering.none);
      STATIC_REQUIRE(
        ::hipGPUDirectRDMAWritesOrderingOwner == device::attributes::gpu_direct_rdma_writes_ordering.owner);
      STATIC_REQUIRE(
        ::hipGPUDirectRDMAWritesOrderingAllDevices == device::attributes::gpu_direct_rdma_writes_ordering.all_devices);

      auto ordering = device_ref(0).attribute(device::attributes::gpu_direct_rdma_writes_ordering);
      CUDAX_REQUIRE((ordering == device::attributes::gpu_direct_rdma_writes_ordering.none || //
                     ordering == device::attributes::gpu_direct_rdma_writes_ordering.owner || //
                     ordering == device::attributes::gpu_direct_rdma_writes_ordering.all_devices));
    }

    SECTION("memory_pool_supported_handle_types")
    {
      STATIC_REQUIRE(::hipMemHandleTypeNone == device::attributes::memory_pool_supported_handle_types.none);
      STATIC_REQUIRE(::hipMemHandleTypePosixFileDescriptor
                     == device::attributes::memory_pool_supported_handle_types.posix_file_descriptor);
      STATIC_REQUIRE(::hipMemHandleTypeWin32 == device::attributes::memory_pool_supported_handle_types.win32);
      STATIC_REQUIRE(::hipMemHandleTypeWin32Kmt == device::attributes::memory_pool_supported_handle_types.win32_kmt);
#if CUDART_VERSION >= 12040
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == 0x8);
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == device::attributes::memory_pool_supported_handle_types.fabric);
#else
      STATIC_REQUIRE(0x8 == device::attributes::memory_pool_supported_handle_types.fabric);
#endif

      constexpr int all_handle_types =
        device::attributes::memory_pool_supported_handle_types.none
        | device::attributes::memory_pool_supported_handle_types.posix_file_descriptor
        | device::attributes::memory_pool_supported_handle_types.win32
        | device::attributes::memory_pool_supported_handle_types.win32_kmt
        | device::attributes::memory_pool_supported_handle_types.fabric;
      auto handle_types = device_ref(0).attribute(device::attributes::memory_pool_supported_handle_types);
      CUDAX_REQUIRE(static_cast<int>(handle_types) <= static_cast<int>(all_handle_types));
    }

#if CUDART_VERSION >= 12020
    SECTION("numa_config")
    {
      STATIC_REQUIRE(::cudaDeviceNumaConfigNone == device::attributes::numa_config.none);
      STATIC_REQUIRE(::cudaDeviceNumaConfigNumaNode == device::attributes::numa_config.numa_node);

      auto config = device_ref(0).attribute(device::attributes::numa_config);
      CUDAX_REQUIRE((config == device::attributes::numa_config.none || //
                     config == device::attributes::numa_config.numa_node));
    }
#endif
    SECTION("Compute capability")
    {
      int compute_cap       = device_ref(0).attribute(device::attributes::compute_capability);
      int compute_cap_major = device_ref(0).attribute(device::attributes::compute_capability_major);
      int compute_cap_minor = device_ref(0).attribute(device::attributes::compute_capability_minor);
      CUDAX_REQUIRE(compute_cap == 100 * compute_cap_major + 10 * compute_cap_minor);
    }
  }
  SECTION("Name")
  {
    std::string name = device_ref(0).name();
    CUDAX_REQUIRE(name.length() != 0);
    CUDAX_REQUIRE(name[0] != 0);
  }
}

C2H_CCCLRT_TEST("global devices vector", "[device]")
{
  CUDAX_REQUIRE(cudax::devices.size() > 0);
  CUDAX_REQUIRE(cudax::devices.begin() != cudax::devices.end());
  CUDAX_REQUIRE(cudax::devices.begin() == cudax::devices.begin());
  CUDAX_REQUIRE(cudax::devices.end() == cudax::devices.end());
  CUDAX_REQUIRE(cudax::devices.size() == static_cast<size_t>(cudax::devices.end() - cudax::devices.begin()));

  CUDAX_REQUIRE(0 == cudax::devices[0].get());
  CUDAX_REQUIRE(cudax::device_ref{0} == cudax::devices[0]);

  CUDAX_REQUIRE(0 == (*cudax::devices.begin()).get());
  CUDAX_REQUIRE(cudax::device_ref{0} == *cudax::devices.begin());

  CUDAX_REQUIRE(0 == cudax::devices.begin()->get());
  CUDAX_REQUIRE(0 == cudax::devices.begin()[0].get());

  if (cudax::devices.size() > 1)
  {
    CUDAX_REQUIRE(1 == cudax::devices[1].get());
    CUDAX_REQUIRE(cudax::device_ref{0} != cudax::devices[1].get());

    CUDAX_REQUIRE(1 == (*std::next(cudax::devices.begin())).get());
    CUDAX_REQUIRE(1 == std::next(cudax::devices.begin())->get());
    CUDAX_REQUIRE(1 == cudax::devices.begin()[1].get());

    CUDAX_REQUIRE(cudax::devices.size() - 1 == static_cast<std::size_t>((*std::prev(cudax::devices.end())).get()));
    CUDAX_REQUIRE(cudax::devices.size() - 1 == static_cast<std::size_t>(std::prev(cudax::devices.end())->get()));
    CUDAX_REQUIRE(cudax::devices.size() - 1 == static_cast<std::size_t>(cudax::devices.end()[-1].get()));

    auto peers = cudax::devices[0].peer_devices();
    for (auto peer : peers)
    {
      CUDAX_REQUIRE(cudax::devices[0].has_peer_access_to(peer))
      CUDAX_REQUIRE(peer.has_peer_access_to(cudax::devices[0]));
    }
  }

#if _CCCL_HAS_EXCEPTIONS()
  try
  {
    [[maybe_unused]] const cudax::device& dev = cudax::devices[cudax::devices.size()];
    CUDAX_REQUIRE(false); // should not get here
  }
  catch (const std::out_of_range&)
  {
    CUDAX_REQUIRE(true); // expected
  }
#endif // _CCCL_HAS_EXCEPTIONS()
}
