//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/device.cuh>

#include "cuda/std/__type_traits/is_same.h"
#include <testing.cuh>

namespace
{
template <const auto& Attr, ::hipDeviceAttribute_t ExpectedAttr, class ExpectedResult>
[[maybe_unused]] auto test_device_attribute()
{
  cudax::device_ref dev0(0);
  STATIC_REQUIRE(Attr == ExpectedAttr);
  STATIC_REQUIRE(::cuda::std::is_same_v<cudax::device::attr_result_t<Attr>, ExpectedResult>);

  auto result = dev0.attr(Attr);
  STATIC_REQUIRE(::cuda::std::is_same_v<decltype(result), ExpectedResult>);
  CUDAX_REQUIRE(result == dev0.attr<ExpectedAttr>());
  CUDAX_REQUIRE(result == Attr(dev0));
  return result;
}
} // namespace

TEST_CASE("Smoke", "[device]")
{
  using cudax::device;
  using cudax::device_ref;

  SECTION("Compare")
  {
    CUDAX_REQUIRE(device_ref{0} == device_ref{0});
    CUDAX_REQUIRE(device_ref{0} == 0);
    CUDAX_REQUIRE(0 == device_ref{0});
    CUDAX_REQUIRE(device_ref{1} != device_ref{0});
    CUDAX_REQUIRE(device_ref{1} != 2);
    CUDAX_REQUIRE(1 != device_ref{2});
  }

  SECTION("Attributes")
  {
    ::test_device_attribute<device::attrs::max_threads_per_block, ::hipDeviceAttributeMaxThreadsPerBlock, int>();
    ::test_device_attribute<device::attrs::max_block_dim_x, ::hipDeviceAttributeMaxBlockDimX, int>();
    ::test_device_attribute<device::attrs::max_block_dim_y, ::hipDeviceAttributeMaxBlockDimY, int>();
    ::test_device_attribute<device::attrs::max_block_dim_z, ::hipDeviceAttributeMaxBlockDimZ, int>();
    ::test_device_attribute<device::attrs::max_grid_dim_x, ::hipDeviceAttributeMaxGridDimX, int>();
    ::test_device_attribute<device::attrs::max_grid_dim_y, ::hipDeviceAttributeMaxGridDimY, int>();
    ::test_device_attribute<device::attrs::max_grid_dim_z, ::hipDeviceAttributeMaxGridDimZ, int>();
    ::test_device_attribute<device::attrs::max_shared_memory_per_block, ::hipDeviceAttributeMaxSharedMemoryPerBlock, int>();
    ::test_device_attribute<device::attrs::total_constant_memory, ::hipDeviceAttributeTotalConstantMemory, int>();
    ::test_device_attribute<device::attrs::warp_size, ::hipDeviceAttributeWarpSize, int>();
    ::test_device_attribute<device::attrs::max_pitch, ::hipDeviceAttributeMaxPitch, int>();
    ::test_device_attribute<device::attrs::max_texture_1d_width, ::hipDeviceAttributeMaxTexture1DWidth, int>();
    ::test_device_attribute<device::attrs::max_texture_1d_linear_width, ::hipDeviceAttributeMaxTexture1DLinear, int>();
    ::test_device_attribute<device::attrs::max_texture_1d_mipmapped_width, ::hipDeviceAttributeMaxTexture1DMipmap, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_width, ::hipDeviceAttributeMaxTexture2DWidth, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_height, ::hipDeviceAttributeMaxTexture2DHeight, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_linear_width, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_linear_height, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_linear_pitch, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_mipmapped_width, ::hipDeviceAttributeMaxTexture2DMipmap, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_mipmapped_height,
                            ::hipDeviceAttributeMaxTexture2DMipmap,
                            int>();
    ::test_device_attribute<device::attrs::max_texture_3d_width, ::hipDeviceAttributeMaxTexture3DWidth, int>();
    ::test_device_attribute<device::attrs::max_texture_3d_height, ::hipDeviceAttributeMaxTexture3DHeight, int>();
    ::test_device_attribute<device::attrs::max_texture_3d_depth, ::hipDeviceAttributeMaxTexture3DDepth, int>();
    ::test_device_attribute<device::attrs::max_texture_3d_width_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attrs::max_texture_3d_height_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attrs::max_texture_3d_depth_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<device::attrs::max_texture_cubemap_width, ::hipDeviceAttributeMaxTextureCubemap, int>();
    ::test_device_attribute<device::attrs::max_texture_1d_layered_width, ::hipDeviceAttributeMaxTexture1DLayered, int>();
    ::test_device_attribute<device::attrs::max_texture_1d_layered_layers, ::cudaDevAttrMaxTexture1DLayeredLayers, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_layered_width, ::hipDeviceAttributeMaxTexture2DLayered, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_layered_height, ::hipDeviceAttributeMaxTexture2DLayered, int>();
    ::test_device_attribute<device::attrs::max_texture_2d_layered_layers, ::cudaDevAttrMaxTexture2DLayeredLayers, int>();
    ::test_device_attribute<device::attrs::max_texture_cubemap_layered_width,
                            ::hipDeviceAttributeMaxTextureCubemapLayered,
                            int>();
    ::test_device_attribute<device::attrs::max_texture_cubemap_layered_layers,
                            ::cudaDevAttrMaxTextureCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<device::attrs::max_surface_1d_width, ::hipDeviceAttributeMaxSurface1D, int>();
    ::test_device_attribute<device::attrs::max_surface_2d_width, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<device::attrs::max_surface_2d_height, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<device::attrs::max_surface_3d_width, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attrs::max_surface_3d_height, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attrs::max_surface_3d_depth, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<device::attrs::max_surface_1d_layered_width, ::hipDeviceAttributeMaxSurface1DLayered, int>();
    ::test_device_attribute<device::attrs::max_surface_1d_layered_layers, ::cudaDevAttrMaxSurface1DLayeredLayers, int>();
    ::test_device_attribute<device::attrs::max_surface_2d_layered_width, ::hipDeviceAttributeMaxSurface2DLayered, int>();
    ::test_device_attribute<device::attrs::max_surface_2d_layered_height, ::hipDeviceAttributeMaxSurface2DLayered, int>();
    ::test_device_attribute<device::attrs::max_surface_2d_layered_layers, ::cudaDevAttrMaxSurface2DLayeredLayers, int>();
    ::test_device_attribute<device::attrs::max_surface_cubemap_width, ::hipDeviceAttributeMaxSurfaceCubemap, int>();
    ::test_device_attribute<device::attrs::max_surface_cubemap_layered_width,
                            ::hipDeviceAttributeMaxSurfaceCubemapLayered,
                            int>();
    ::test_device_attribute<device::attrs::max_surface_cubemap_layered_layers,
                            ::cudaDevAttrMaxSurfaceCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<device::attrs::max_registers_per_block, ::hipDeviceAttributeMaxRegistersPerBlock, int>();
    ::test_device_attribute<device::attrs::clock_rate, ::hipDeviceAttributeClockRate, int>();
    ::test_device_attribute<device::attrs::texture_alignment, ::hipDeviceAttributeTextureAlignment, int>();
    ::test_device_attribute<device::attrs::texture_pitch_alignment, ::hipDeviceAttributeTexturePitchAlignment, int>();
    ::test_device_attribute<device::attrs::gpu_overlap, ::hipDeviceAttributeAsyncEngineCount, bool>();
    ::test_device_attribute<device::attrs::multiprocessor_count, ::hipDeviceAttributeMultiprocessorCount, int>();
    ::test_device_attribute<device::attrs::kernel_exec_timeout, ::hipDeviceAttributeKernelExecTimeout, bool>();
    ::test_device_attribute<device::attrs::integrated, ::hipDeviceAttributeIntegrated, bool>();
    ::test_device_attribute<device::attrs::can_map_host_memory, ::hipDeviceAttributeCanMapHostMemory, bool>();
    ::test_device_attribute<device::attrs::compute_mode, ::hipDeviceAttributeComputeMode, ::hipComputeMode>();
    ::test_device_attribute<device::attrs::concurrent_kernels, ::hipDeviceAttributeConcurrentKernels, bool>();
    ::test_device_attribute<device::attrs::ecc_enabled, ::hipDeviceAttributeEccEnabled, bool>();
    ::test_device_attribute<device::attrs::pci_bus_id, ::hipDeviceAttributePciBusId, int>();
    ::test_device_attribute<device::attrs::pci_device_id, ::hipDeviceAttributePciDeviceId, int>();
    ::test_device_attribute<device::attrs::tcc_driver, ::hipDeviceAttributeTccDriver, bool>();
    ::test_device_attribute<device::attrs::memory_clock_rate, ::hipDeviceAttributeMemoryClockRate, int>();
    ::test_device_attribute<device::attrs::global_memory_bus_width, ::hipDeviceAttributeMemoryBusWidth, int>();
    ::test_device_attribute<device::attrs::l2_cache_size, ::hipDeviceAttributeL2CacheSize, int>();
    ::test_device_attribute<device::attrs::max_threads_per_multiprocessor,
                            ::hipDeviceAttributeMaxThreadsPerMultiProcessor,
                            int>();
    ::test_device_attribute<device::attrs::unified_addressing, ::hipDeviceAttributeUnifiedAddressing, bool>();
    ::test_device_attribute<device::attrs::compute_capability_major, ::hipDeviceAttributeComputeCapabilityMajor, int>();
    ::test_device_attribute<device::attrs::compute_capability_minor, ::hipDeviceAttributeComputeCapabilityMinor, int>();
    ::test_device_attribute<device::attrs::stream_priorities_supported, ::hipDeviceAttributeStreamPrioritiesSupported, bool>();
    ::test_device_attribute<device::attrs::global_l1_cache_supported, ::hipDeviceAttributeGlobalL1CacheSupported, bool>();
    ::test_device_attribute<device::attrs::local_l1_cache_supported, ::hipDeviceAttributeLocalL1CacheSupported, bool>();
    ::test_device_attribute<device::attrs::max_shared_memory_per_multiprocessor,
                            ::hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                            int>();
    ::test_device_attribute<device::attrs::max_registers_per_multiprocessor,
                            ::hipDeviceAttributeMaxRegistersPerMultiprocessor,
                            int>();
    ::test_device_attribute<device::attrs::managed_memory, ::hipDeviceAttributeManagedMemory, bool>();
    ::test_device_attribute<device::attrs::is_multi_gpu_board, ::hipDeviceAttributeIsMultiGpuBoard, bool>();
    ::test_device_attribute<device::attrs::multi_gpu_board_group_id, ::hipDeviceAttributeMultiGpuBoardGroupID, int>();
    ::test_device_attribute<device::attrs::host_native_atomic_supported, ::hipDeviceAttributeHostNativeAtomicSupported, bool>();
    ::test_device_attribute<device::attrs::single_to_double_precision_perf_ratio,
                            ::hipDeviceAttributeSingleToDoublePrecisionPerfRatio,
                            int>();
    ::test_device_attribute<device::attrs::pageable_memory_access, ::hipDeviceAttributePageableMemoryAccess, bool>();
    ::test_device_attribute<device::attrs::concurrent_managed_access, ::hipDeviceAttributeConcurrentManagedAccess, bool>();
    ::test_device_attribute<device::attrs::compute_preemption_supported, ::hipDeviceAttributeComputePreemptionSupported, bool>();
    ::test_device_attribute<device::attrs::can_use_host_pointer_for_registered_mem,
                            ::hipDeviceAttributeCanUseHostPointerForRegisteredMem,
                            bool>();
    ::test_device_attribute<device::attrs::cooperative_launch, ::hipDeviceAttributeCooperativeLaunch, bool>();
    ::test_device_attribute<device::attrs::cooperative_multi_device_launch,
                            ::hipDeviceAttributeCooperativeMultiDeviceLaunch,
                            bool>();
    ::test_device_attribute<device::attrs::can_flush_remote_writes, ::cudaDevAttrCanFlushRemoteWrites, bool>();
    ::test_device_attribute<device::attrs::host_register_supported, ::hipDeviceAttributeHostRegisterSupported, bool>();
    ::test_device_attribute<device::attrs::pageable_memory_access_uses_host_page_tables,
                            ::hipDeviceAttributePageableMemoryAccessUsesHostPageTables,
                            bool>();
    ::test_device_attribute<device::attrs::direct_managed_mem_access_from_host,
                            ::hipDeviceAttributeDirectManagedMemAccessFromHost,
                            bool>();
    ::test_device_attribute<device::attrs::max_shared_memory_per_block_optin,
                            ::hipDeviceAttributeSharedMemPerBlockOptin,
                            int>();
    ::test_device_attribute<device::attrs::max_blocks_per_multiprocessor, ::hipDeviceAttributeMaxBlocksPerMultiprocessor, int>();
    ::test_device_attribute<device::attrs::max_persisting_l2_cache_size, ::cudaDevAttrMaxPersistingL2CacheSize, int>();
    ::test_device_attribute<device::attrs::max_access_policy_window_size, ::cudaDevAttrMaxAccessPolicyWindowSize, int>();
    ::test_device_attribute<device::attrs::reserved_shared_memory_per_block,
                            ::cudaDevAttrReservedSharedMemoryPerBlock,
                            int>();
    ::test_device_attribute<device::attrs::sparse_cuda_array_supported, ::cudaDevAttrSparseCudaArraySupported, bool>();
    ::test_device_attribute<device::attrs::host_register_read_only_supported,
                            ::cudaDevAttrHostRegisterReadOnlySupported,
                            bool>();
    ::test_device_attribute<device::attrs::memory_pools_supported, ::hipDeviceAttributeMemoryPoolsSupported, bool>();
    ::test_device_attribute<device::attrs::gpu_direct_rdma_supported, ::cudaDevAttrGPUDirectRDMASupported, bool>();
    ::test_device_attribute<device::attrs::gpu_direct_rdma_flush_writes_options,
                            ::cudaDevAttrGPUDirectRDMAFlushWritesOptions,
                            ::hipFlushGPUDirectRDMAWritesOptions>();
    ::test_device_attribute<device::attrs::gpu_direct_rdma_writes_ordering,
                            ::cudaDevAttrGPUDirectRDMAWritesOrdering,
                            ::hipGPUDirectRDMAWritesOrdering>();
    ::test_device_attribute<device::attrs::memory_pool_supported_handle_types,
                            ::cudaDevAttrMemoryPoolSupportedHandleTypes,
                            ::hipMemAllocationHandleType>();
    ::test_device_attribute<device::attrs::deferred_mapping_cuda_array_supported,
                            ::cudaDevAttrDeferredMappingCudaArraySupported,
                            bool>();
    ::test_device_attribute<device::attrs::ipc_event_support, ::cudaDevAttrIpcEventSupport, bool>();

#if CUDART_VERSION >= 12020
    ::test_device_attribute<device::attrs::numa_config, ::cudaDevAttrNumaConfig, ::cudaDeviceNumaConfig>();
    ::test_device_attribute<device::attrs::numa_id, ::cudaDevAttrNumaId, int>();
#endif

    SECTION("compute_mode")
    {
      STATIC_REQUIRE(::hipComputeModeDefault == device::attrs::compute_mode.default_mode);
      STATIC_REQUIRE(::hipComputeModeProhibited == device::attrs::compute_mode.prohibited_mode);
      STATIC_REQUIRE(::hipComputeModeExclusiveProcess == device::attrs::compute_mode.exclusive_process_mode);

      auto mode = device_ref(0).attr(device::attrs::compute_mode);
      CUDAX_REQUIRE((mode == device::attrs::compute_mode.default_mode || //
                     mode == device::attrs::compute_mode.prohibited_mode || //
                     mode == device::attrs::compute_mode.exclusive_process_mode));
    }

    SECTION("gpu_direct_rdma_flush_writes_options")
    {
      STATIC_REQUIRE(
        ::hipFlushGPUDirectRDMAWritesOptionHost == device::attrs::gpu_direct_rdma_flush_writes_options.host);
      STATIC_REQUIRE(
        ::hipFlushGPUDirectRDMAWritesOptionMemOps == device::attrs::gpu_direct_rdma_flush_writes_options.mem_ops);

      auto options = device_ref(0).attr(device::attrs::gpu_direct_rdma_flush_writes_options);
      CUDAX_REQUIRE((options == device::attrs::gpu_direct_rdma_flush_writes_options.host || //
                     options == device::attrs::gpu_direct_rdma_flush_writes_options.mem_ops));
    }

    SECTION("gpu_direct_rdma_writes_ordering")
    {
      STATIC_REQUIRE(::hipGPUDirectRDMAWritesOrderingNone == device::attrs::gpu_direct_rdma_writes_ordering.none);
      STATIC_REQUIRE(::hipGPUDirectRDMAWritesOrderingOwner == device::attrs::gpu_direct_rdma_writes_ordering.owner);
      STATIC_REQUIRE(
        ::hipGPUDirectRDMAWritesOrderingAllDevices == device::attrs::gpu_direct_rdma_writes_ordering.all_devices);

      auto ordering = device_ref(0).attr(device::attrs::gpu_direct_rdma_writes_ordering);
      CUDAX_REQUIRE((ordering == device::attrs::gpu_direct_rdma_writes_ordering.none || //
                     ordering == device::attrs::gpu_direct_rdma_writes_ordering.owner || //
                     ordering == device::attrs::gpu_direct_rdma_writes_ordering.all_devices));
    }

    SECTION("memory_pool_supported_handle_types")
    {
      STATIC_REQUIRE(::hipMemHandleTypeNone == device::attrs::memory_pool_supported_handle_types.none);
      STATIC_REQUIRE(::hipMemHandleTypePosixFileDescriptor
                     == device::attrs::memory_pool_supported_handle_types.posix_file_descriptor);
      STATIC_REQUIRE(::hipMemHandleTypeWin32 == device::attrs::memory_pool_supported_handle_types.win32);
      STATIC_REQUIRE(::hipMemHandleTypeWin32Kmt == device::attrs::memory_pool_supported_handle_types.win32_kmt);
#if CUDART_VERSION >= 12040
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == 0x8);
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == device::attrs::memory_pool_supported_handle_types.fabric);
#else
      STATIC_REQUIRE(0x8 == device::attrs::memory_pool_supported_handle_types.fabric);
#endif

      constexpr int all_handle_types =
        device::attrs::memory_pool_supported_handle_types.none
        | device::attrs::memory_pool_supported_handle_types.posix_file_descriptor
        | device::attrs::memory_pool_supported_handle_types.win32
        | device::attrs::memory_pool_supported_handle_types.win32_kmt
        | device::attrs::memory_pool_supported_handle_types.fabric;
      auto handle_types = device_ref(0).attr(device::attrs::memory_pool_supported_handle_types);
      CUDAX_REQUIRE(handle_types <= all_handle_types);
    }

#if CUDART_VERSION >= 12020
    SECTION("numa_config")
    {
      STATIC_REQUIRE(::cudaDeviceNumaConfigNone == device::attrs::numa_config.none);
      STATIC_REQUIRE(::cudaDeviceNumaConfigNumaNode == device::attrs::numa_config.numa_node);

      auto config = device_ref(0).attr(device::attrs::numa_config);
      CUDAX_REQUIRE((config == device::attrs::numa_config.none || //
                     config == device::attrs::numa_config.numa_node));
    }
#endif
    SECTION("Compute capability")
    {
      int compute_cap       = device_ref(0).attr(device::attrs::compute_capability);
      int compute_cap_major = device_ref(0).attr(device::attrs::compute_capability_major);
      int compute_cap_minor = device_ref(0).attr(device::attrs::compute_capability_minor);
      CUDAX_REQUIRE(compute_cap == 100 * compute_cap_major + 10 * compute_cap_minor);
    }
  }
  SECTION("Name")
  {
    std::string name = device_ref(0).get_name();
    CUDAX_REQUIRE(name.length() != 0);
    CUDAX_REQUIRE(name[0] != 0);
  }
}

TEST_CASE("global devices vector", "[device]")
{
  CUDAX_REQUIRE(cudax::devices.size() > 0);
  CUDAX_REQUIRE(cudax::devices.begin() != cudax::devices.end());
  CUDAX_REQUIRE(cudax::devices.begin() == cudax::devices.begin());
  CUDAX_REQUIRE(cudax::devices.end() == cudax::devices.end());
  CUDAX_REQUIRE(cudax::devices.size() == static_cast<size_t>(cudax::devices.end() - cudax::devices.begin()));

  CUDAX_REQUIRE(0 == cudax::devices[0].get());
  CUDAX_REQUIRE(cudax::device_ref{0} == cudax::devices[0]);

  CUDAX_REQUIRE(0 == (*cudax::devices.begin()).get());
  CUDAX_REQUIRE(cudax::device_ref{0} == *cudax::devices.begin());

  CUDAX_REQUIRE(0 == cudax::devices.begin()->get());
  CUDAX_REQUIRE(0 == cudax::devices.begin()[0].get());

  if (cudax::devices.size() > 1)
  {
    CUDAX_REQUIRE(1 == cudax::devices[1].get());
    CUDAX_REQUIRE(cudax::device_ref{0} != cudax::devices[1].get());

    CUDAX_REQUIRE(1 == (*std::next(cudax::devices.begin())).get());
    CUDAX_REQUIRE(1 == std::next(cudax::devices.begin())->get());
    CUDAX_REQUIRE(1 == cudax::devices.begin()[1].get());

    CUDAX_REQUIRE(cudax::devices.size() - 1 == (*std::prev(cudax::devices.end())).get());
    CUDAX_REQUIRE(cudax::devices.size() - 1 == std::prev(cudax::devices.end())->get());
    CUDAX_REQUIRE(cudax::devices.size() - 1 == cudax::devices.end()[-1].get());

    auto peers = cudax::devices[0].get_peers();
    for (auto peer : peers)
    {
      CUDAX_REQUIRE(cudax::devices[0].has_peer_access_to(peer))
      CUDAX_REQUIRE(peer.has_peer_access_to(cudax::devices[0]));
    }
  }

  try
  {
    [[maybe_unused]] const cudax::device& dev = cudax::devices.at(cudax::devices.size());
    CUDAX_REQUIRE(false); // should not get here
  }
  catch (const std::out_of_range&)
  {
    CUDAX_REQUIRE(true); // expected
  }
}
