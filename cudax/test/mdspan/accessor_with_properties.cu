#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cuda/std/mdspan>

#include <cuda/experimental/accessor.cuh>

#include <testing.cuh>

template <typename T, typename E, typename L, typename A>
__global__ void mdspan_accessor_kernel(cuda::std::mdspan<T, E, L, A> md)
{
  CUDAX_REQUIRE(md(threadIdx.x) == threadIdx.x);
}

using namespace cuda::experimental;

template <typename... Ts>
using type_list = ::cuda::std::__type_list<Ts...>;

template <typename... Ts>
using __type_cartesian_product = ::cuda::std::__type_cartesian_product<Ts...>;

using eviction_list =
  type_list<eviction_none_t, eviction_normal_t, eviction_first_t, eviction_last_t, eviction_last_use_t, eviction_no_alloc_t>;

using alignment_list = type_list<cuda::aligned_size_t<4>, cuda::aligned_size_t<8>>;

using prefetch_list = type_list<prefetch_default_t, prefetch_64B_t, prefetch_128B_t, prefetch_256B_t>;

using aliasing_list = type_list<ptr_may_alias_t, ptr_no_aliasing_t>;

using TypeLists = ::cuda::std::__type_cartesian_product<eviction_list, alignment_list, prefetch_list, aliasing_list>;

TEMPLATE_LIST_TEST_CASE("Accessor", "[device]", TypeLists)
{
  using EvictionPolicy = ::cuda::std::__type_at_c<0, TestType>;
  using Alignment      = ::cuda::std::__type_at_c<1, TestType>;
  using Prefetch       = ::cuda::std::__type_at_c<2, TestType>;
  using Restrict       = ::cuda::std::__type_at_c<3, TestType>;

  thrust::host_vector<int> h_vector(32);
  std::iota(h_vector.begin(), h_vector.end(), 0);
  thrust::device_vector<int> d_vector = h_vector;

  auto md            = cuda::std::mdspan(thrust::raw_pointer_cast(d_vector.data()), d_vector.size());
  auto md_with_props = cuda::experimental::add_properties(md, cuda::experimental::eviction_no_alloc);
  mdspan_accessor_kernel<<<1, 32>>>(md_with_props);
  hipDeviceSynchronize();
  CUDAX_REQUIRE(hipGetLastError() == hipSuccess);
}
