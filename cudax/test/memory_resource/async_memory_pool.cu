//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream_ref>

#include <cuda/experimental/memory_resource.cuh>

#include <catch2/catch.hpp>

namespace cudax = cuda::experimental;
using pool      = cudax::mr::async_memory_pool;
static_assert(!cuda::std::is_trivial<pool>::value, "");
static_assert(!cuda::std::is_trivially_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_constructible<pool>::value, "");
static_assert(!cuda::std::is_move_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_assignable<pool>::value, "");
static_assert(!cuda::std::is_move_assignable<pool>::value, "");
static_assert(!cuda::std::is_trivially_destructible<pool>::value, "");
static_assert(!cuda::std::is_empty<pool>::value, "");

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool, const int driver_version)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  constexpr int min_async_version = 11050;
  return driver_version < min_async_version ? disable_reuse == 0 : disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to querry hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

TEST_CASE("async_memory_pool construction", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = cudax::mr::async_memory_pool;
  SECTION("Construct from device id")
  {
    cudax::mr::async_memory_pool from_device{current_device};

    ::hipMemPool_t get = from_device.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with empty properties")
  {
    cudax::mr::async_memory_pool_properties props{};
    memory_pool from_defaulted_properties{current_device, props};

    ::hipMemPool_t get = from_defaulted_properties.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with initial pool size")
  {
    cudax::mr::async_memory_pool_properties props = {42, 20};
    memory_pool with_threshold{current_device, props};

    ::hipMemPool_t get = with_threshold.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  // Allocation handles are only supported after 11.2
#if !defined(_CCCL_CUDACC_BELOW_11_2)
  SECTION("Construct with allocation handle")
  {
    cudax::mr::async_memory_pool_properties props = {
      42, 20, cudax::mr::hipMemAllocationHandleType::hipMemHandleTypePosixFileDescriptor};
    memory_pool with_allocation_handle{current_device, props};

    ::hipMemPool_t get = with_allocation_handle.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
  }
#endif // !_CCCL_CUDACC_BELOW_11_2

  SECTION("Take ownership of native handle")
  {
    ::hipMemPoolProps pool_properties{};
    pool_properties.allocType     = ::hipMemAllocationTypePinned;
    pool_properties.handleTypes   = ::hipMemAllocationHandleType(hipMemAllocationHandleType::hipMemHandleTypeNone);
    pool_properties.location.type = ::hipMemLocationTypeDevice;
    pool_properties.location.id   = current_device;
    ::hipMemPool_t new_pool{};
    _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &new_pool, &pool_properties);

    cudax::mr::async_memory_pool from_handle = cudax::mr::async_memory_pool::from_native_handle(new_pool);
    CHECK(from_handle.get() == new_pool);
  }
}

TEST_CASE("async_memory_pool comparison", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to querry current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  cudax::mr::async_memory_pool first{current_device};
  { // comparison against a plain async_memory_pool
    cudax::mr::async_memory_pool second{current_device};
    CHECK(first == first);
    CHECK(first != second);
  }

  { // comparison against a hipMemPool_t
    CHECK(first == first.get());
    CHECK(first.get() == first);
    CHECK(first != current_default_pool);
    CHECK(current_default_pool != first);
  }
}
