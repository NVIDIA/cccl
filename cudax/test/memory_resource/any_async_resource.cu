//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/memory_resource.cuh>

#include <testing.cuh>

#include "test_resource.cuh"

#ifndef __CUDA_ARCH__

TEMPLATE_TEST_CASE_METHOD(test_fixture, "any_async_resource", "[container][resource]", big_resource, small_resource)
{
  using TestResource = TestType;
  static_assert(cuda::mr::synchronous_resource_with<TestResource, cudax::host_accessible>);
  constexpr bool is_big = sizeof(TestResource) > cuda::__default_small_object_size;

  SECTION("construct and destruct")
  {
    Counts expected{};
    CHECK(this->counts == expected);
    {
      cudax::any_async_resource<cudax::host_accessible> mr{TestResource{42, this}};
      expected.new_count += is_big;
      ++expected.object_count;
      ++expected.move_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += is_big;
    --expected.object_count;
    CHECK(this->counts == expected);
  }

  // Reset the counters:
  this->counts = Counts();

  SECTION("copy and move")
  {
    Counts expected{};
    CHECK(this->counts == expected);
    {
      cudax::any_async_resource<cudax::host_accessible> mr{TestResource{42, this}};
      expected.new_count += is_big;
      ++expected.object_count;
      ++expected.move_count;
      CHECK(this->counts == expected);

      auto mr2 = mr;
      expected.new_count += is_big;
      ++expected.copy_count;
      ++expected.object_count;
      CHECK(this->counts == expected);
      CHECK(mr == mr2);
      ++expected.equal_to_count;
      CHECK(this->counts == expected);

      auto mr3 = std::move(mr);
      expected.move_count += !is_big; // for big resources, move is a pointer swap
      CHECK(this->counts == expected);
      CHECK(mr2 == mr3);
      ++expected.equal_to_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += 2 * is_big;
    expected.object_count -= 2;
    CHECK(this->counts == expected);
  }

  // Reset the counters:
  this->counts = Counts();

  SECTION("allocate and deallocate_sync")
  {
    Counts expected{};
    CHECK(this->counts == expected);
    {
      cudax::any_async_resource<cudax::host_accessible> mr{TestResource{42, this}};
      expected.new_count += is_big;
      ++expected.object_count;
      ++expected.move_count;
      CHECK(this->counts == expected);

      void* ptr = mr.allocate_sync(bytes(50), align(8));
      CHECK(ptr == this);
      ++expected.allocate_count;
      CHECK(this->counts == expected);

      mr.deallocate_sync(ptr, bytes(50), align(8));
      ++expected.deallocate_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += is_big;
    --expected.object_count;
    CHECK(this->counts == expected);
  }

  // Reset the counters:
  this->counts = Counts();

  SECTION("allocate and deallocate")
  {
    Counts expected{};
    CHECK(this->counts == expected);
    {
      cudax::stream stream{cuda::device_ref{0}};
      cudax::any_async_resource<cudax::host_accessible> mr{TestResource{42, this}};
      expected.new_count += is_big;
      ++expected.object_count;
      ++expected.move_count;
      CHECK(this->counts == expected);

      void* ptr = mr.allocate(::cuda::stream_ref{stream}, bytes(50), align(8));
      CHECK(ptr == this);
      ++expected.allocate_async_count;
      CHECK(this->counts == expected);

      mr.deallocate(::cuda::stream_ref{stream}, ptr, bytes(50), align(8));
      ++expected.deallocate_async_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += is_big;
    --expected.object_count;
    CHECK(this->counts == expected);
  }

  // Reset the counters:
  this->counts = Counts();

  SECTION("conversion to resource_ref")
  {
    Counts expected{};
    {
      cudax::any_async_resource<cudax::host_accessible> mr{TestResource{42, this}};
      expected.new_count += is_big;
      ++expected.object_count;
      ++expected.move_count;
      CHECK(this->counts == expected);

      cudax::resource_ref<cudax::host_accessible> ref = mr;

      CHECK(this->counts == expected);
      auto* ptr = ref.allocate_sync(bytes(100), align(8));
      CHECK(ptr == this);
      ++expected.allocate_count;
      CHECK(this->counts == expected);
      ref.deallocate_sync(ptr, bytes(0), align(0));
      ++expected.deallocate_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += is_big;
    --expected.object_count;
    CHECK(this->counts == expected);
  }

  // Reset the counters:
  this->counts = Counts();

  SECTION("make_any_async_resource")
  {
    Counts expected{};
    CHECK(this->counts == expected);
    {
      cudax::any_async_resource<cudax::host_accessible> mr =
        cudax::make_any_async_resource<TestResource, cudax::host_accessible>(42, this);
      expected.new_count += is_big;
      ++expected.object_count;
      CHECK(this->counts == expected);
    }
    expected.delete_count += is_big;
    --expected.object_count;
    CHECK(this->counts == expected);
  }
  // Reset the counters:
  this->counts = Counts();
}

#endif // __CUDA_ARCH__
