#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream_ref>

#include <cuda/experimental/launch.cuh>
#include <cuda/experimental/memory_resource.cuh>
#include <cuda/experimental/stream.cuh>

#include <stdexcept>

#include <testing.cuh>

#if _CCCL_CUDACC_AT_LEAST(13, 0)
#  define TEST_TYPES cudax::managed_memory_pool, cudax::device_memory_pool, cudax::pinned_memory_pool
#elif _CCCL_CUDACC_AT_LEAST(12, 6)
#  define TEST_TYPES cudax::device_memory_pool, cudax::pinned_memory_pool
#else
#  define TEST_TYPES cudax::device_memory_pool
#endif

namespace cudax = cuda::experimental;

template <typename PoolType>
void pool_static_asserts()
{
  static_assert(!cuda::std::is_trivial<PoolType>::value, "");
  static_assert(!cuda::std::is_trivially_default_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_copy_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_move_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_copy_assignable<PoolType>::value, "");
  static_assert(!cuda::std::is_move_assignable<PoolType>::value, "");
  static_assert(!cuda::std::is_trivially_destructible<PoolType>::value, "");
  static_assert(!cuda::std::is_empty<PoolType>::value, "");
}

#if _CCCL_CUDACC_AT_LEAST(13, 0)
template void pool_static_asserts<cudax::managed_memory_pool>();
#endif
#if _CCCL_CUDACC_AT_LEAST(12, 6)
template void pool_static_asserts<cudax::pinned_memory_pool>();
#endif
template void pool_static_asserts<cudax::device_memory_pool>();

#if _CCCL_CUDACC_AT_LEAST(13, 0)
static_assert(cuda::std::is_default_constructible<cudax::managed_memory_pool>::value, "");
static_assert(cuda::std::is_default_constructible<cudax::pinned_memory_pool>::value, "");
#endif
static_assert(!cuda::std::is_default_constructible<cudax::device_memory_pool>::value, "");

template <typename PoolType>
PoolType construct_pool([[maybe_unused]] int device_id, cudax::memory_pool_properties props = {})
{
  if constexpr (cuda::std::is_same_v<PoolType, cudax::device_memory_pool>)
  {
    return cudax::device_memory_pool(device_id, props);
  }
  else
  {
#if _CCCL_CTK_AT_LEAST(12, 6)
    if constexpr (cuda::std::is_same_v<PoolType, cudax::pinned_memory_pool>)
    {
      return cudax::pinned_memory_pool(0, props);
    }
    else
    {
#  if _CCCL_CTK_AT_LEAST(13, 0)
      return cudax::managed_memory_pool(props);
#  endif // _CCCL_CTK_AT_LEAST(13, 0)
    }
#endif // _CCCL_CTK_AT_LEAST(12, 6)
  }
}

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  return disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to query hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

C2H_CCCLRT_TEST_LIST("device_memory_pool construction", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = TestType;
  SECTION("Construct from device id")
  {
    memory_pool from_device = construct_pool<memory_pool>(current_device);

    ::hipMemPool_t get = from_device.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, cuda::std::numeric_limits<size_t>::max()));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with empty properties")
  {
    cudax::memory_pool_properties props{};
    memory_pool from_defaulted_properties = construct_pool<memory_pool>(current_device, props);

    ::hipMemPool_t get = from_defaulted_properties.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, cuda::std::numeric_limits<size_t>::max()));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with initial pool size")
  {
    cudax::memory_pool_properties props = {20, 42};
    memory_pool with_threshold          = construct_pool<memory_pool>(current_device, props);

    ::hipMemPool_t get = with_threshold.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse
    CHECK(ensure_disable_reuse(get));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  if (cuda::std::is_same_v<memory_pool, cudax::device_memory_pool>)
  {
  }

  SECTION("Take ownership of native handle")
  {
    ::hipMemPoolProps pool_properties{};
    pool_properties.handleTypes = ::hipMemAllocationHandleType(hipMemAllocationHandleType::hipMemHandleTypeNone);
    if (cuda::std::is_same_v<memory_pool, cudax::device_memory_pool>)
    {
      pool_properties.allocType     = ::hipMemAllocationTypePinned;
      pool_properties.location.type = ::hipMemLocationTypeDevice;
      pool_properties.location.id   = current_device;
    }
#if _CCCL_CUDACC_AT_LEAST(12, 6)
    else if (cuda::std::is_same_v<memory_pool, cudax::pinned_memory_pool>)
    {
      pool_properties.allocType     = ::hipMemAllocationTypePinned;
      pool_properties.location.type = hipMemLocationTypeHostNuma;
      pool_properties.location.id   = 0;
    }
#  if _CCCL_CUDACC_AT_LEAST(13, 0)
    else if (cuda::std::is_same_v<memory_pool, cudax::managed_memory_pool>)
    {
      pool_properties.allocType     = ::cudaMemAllocationTypeManaged;
      pool_properties.location.type = hipMemLocationTypeNone;
      pool_properties.location.id   = 0;
    }
#  endif
#endif
    else
    {
      REQUIRE(false);
    }
    ::hipMemPool_t new_pool{};
    _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &new_pool, &pool_properties);

    memory_pool from_handle = memory_pool::from_native_handle(new_pool);
    CHECK(from_handle.get() == new_pool);
  }
}

C2H_CCCLRT_TEST_LIST("device_memory_pool comparison", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = TestType;
  memory_pool first = construct_pool<memory_pool>(current_device);
  { // comparison against a plain device_memory_pool
    memory_pool second = construct_pool<memory_pool>(current_device);
    CHECK(first == first);
    CHECK(first != second);
  }
}

C2H_CCCLRT_TEST_LIST("device_memory_pool accessors", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  using memory_pool     = TestType;
  using memory_resource = typename memory_pool::reference_type;
  SECTION("device_memory_pool::set_attribute")
  {
    memory_pool pool = construct_pool<memory_pool>(current_device);

    { // hipMemPoolReuseFollowEventDependencies
      // Get the attribute value
      bool attr = pool.attribute(cudax::memory_pool_attributes::reuse_follow_event_dependencies);

      // Set it to the opposite
      pool.set_attribute(cudax::memory_pool_attributes::reuse_follow_event_dependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(cudax::memory_pool_attributes::reuse_follow_event_dependencies);
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(cudax::memory_pool_attributes::reuse_follow_event_dependencies, attr);
    }

    { // hipMemPoolReuseAllowOpportunistic
      // Get the attribute value
      bool attr = pool.attribute(cudax::memory_pool_attributes::reuse_allow_opportunistic);

      // Set it to the opposite
      pool.set_attribute(cudax::memory_pool_attributes::reuse_allow_opportunistic, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(cudax::memory_pool_attributes::reuse_allow_opportunistic);
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(cudax::memory_pool_attributes::reuse_allow_opportunistic, attr);
    }

    { // hipMemPoolReuseAllowInternalDependencies
      // Get the attribute value
      bool attr = pool.attribute(cudax::memory_pool_attributes::reuse_allow_internal_dependencies);

      // Set it to the opposite
      pool.set_attribute(cudax::memory_pool_attributes::reuse_allow_internal_dependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(cudax::memory_pool_attributes::reuse_allow_internal_dependencies);
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(cudax::memory_pool_attributes::reuse_allow_internal_dependencies, attr);
    }

    { // hipMemPoolAttrReleaseThreshold
      // Get the attribute value
      size_t attr = pool.attribute(cudax::memory_pool_attributes::release_threshold);

      // Set it to something else
      pool.set_attribute(cudax::memory_pool_attributes::release_threshold, 2 * attr);

      // Retrieve again and verify it was changed
      size_t new_attr = pool.attribute(cudax::memory_pool_attributes::release_threshold);
      CHECK(new_attr == 2 * attr);

      // Set it back
      pool.set_attribute(cudax::memory_pool_attributes::release_threshold, attr);
    }

    // prime the pool to a given size
    memory_resource resource{pool};
    cudax::stream stream{cuda::device_ref{0}};

    // Allocate a buffer to prime
    auto* ptr = resource.allocate(stream, 256 * sizeof(int));
    stream.sync();

    { // hipMemPoolAttrReservedMemHigh
      // Get the attribute value
      size_t attr = pool.attribute(cudax::memory_pool_attributes::reserved_mem_high);

      // Set it to zero as everything else is illegal
      pool.set_attribute(cudax::memory_pool_attributes::reserved_mem_high, 0);

      // Retrieve again and verify it was changed, which it wasn't...
      size_t new_attr = pool.attribute(cudax::memory_pool_attributes::reserved_mem_high);
      CHECK(new_attr == attr);

#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(cudax::memory_pool_attributes::reserved_mem_high, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "This attribute can't be set to a non-zero value.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    { // hipMemPoolAttrUsedMemHigh
      // Get the attribute value
      size_t attr = pool.attribute(cudax::memory_pool_attributes::used_mem_high);

      // Set it to zero as everything else is illegal
      pool.set_attribute(cudax::memory_pool_attributes::used_mem_high, 0);

      // Retrieve again and verify it was changed, which it wasn't...
      size_t new_attr = pool.attribute(cudax::memory_pool_attributes::used_mem_high);
      CHECK(new_attr == attr);

#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(cudax::memory_pool_attributes::used_mem_high, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "This attribute can't be set to a non-zero value.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    // Reallocate as the checks above have screwed with the allocation count
    resource.deallocate(stream, ptr, 256 * sizeof(int));
    ptr = resource.allocate(stream, 2048 * sizeof(int));
    stream.sync();

    { // hipMemPoolAttrReservedMemCurrent
      // Get the attribute value
      size_t attr = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
      CHECK(attr >= 2048 * sizeof(int));
      // hipMemPoolAttrReservedMemCurrent cannot be set
    }

    { // hipMemPoolAttrUsedMemCurrent
      // Get the attribute value
      size_t attr = pool.attribute(cudax::memory_pool_attributes::used_mem_current);
      CHECK(attr == 2048 * sizeof(int));
      // hipMemPoolAttrUsedMemCurrent cannot be set
    }

    // Free the last allocation
    resource.deallocate(stream, ptr, 2048 * sizeof(int));
    stream.sync();
  }

  SECTION("device_memory_pool::trim_to")
  {
    memory_pool pool = construct_pool<memory_pool>(current_device);

    // prime the pool to a given size
    memory_resource resource{pool};
    cudax::stream stream{cuda::device_ref{0}};

    // Allocate 2 buffers
    auto* ptr1 = resource.allocate(stream, 2048 * sizeof(int));
    auto* ptr2 = resource.allocate(stream, 2048 * sizeof(int));
    resource.deallocate(stream, ptr1, 2048 * sizeof(int));
    stream.sync();

    // Ensure that we still hold some memory, otherwise everything is freed
    auto backing_size = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(backing_size >= 4096 * sizeof(int));

    // Trim the pool to something smaller than currently held
    pool.trim_to(1024);

    // Should be a noop
    auto noop_backing_size = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(backing_size == noop_backing_size);

    // Trim to larger than ever allocated
    pool.trim_to(backing_size * 24);

    // Should be a noop
    auto another_noop_backing_size = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(backing_size == another_noop_backing_size);

    // Trim to smaller than current backing but larger than current allocated
    pool.trim_to(2560 * sizeof(int));

    // Check the backing size again
    auto new_backing_size = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(new_backing_size <= backing_size);
    CHECK(new_backing_size >= 4096 * sizeof(int));

    // Free the last allocation
    resource.deallocate(stream, ptr2, 2048 * sizeof(int));
    stream.sync();

    // By default the pool should not release anything without a trim call
    auto no_backing = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(no_backing == new_backing_size);

    // We can still trim the pool without effect
    pool.trim_to(2560 * sizeof(int));

    auto still_no_backing = pool.attribute(cudax::memory_pool_attributes::reserved_mem_current);
    CHECK(still_no_backing == new_backing_size);
  }
}

C2H_CCCLRT_TEST("device_memory_pool::enable_access", "[memory_resource]")
{
  if (cuda::devices.size() > 1)
  {
    auto peers = cuda::devices[0].peers();
    if (peers.size() > 0)
    {
      cudax::device_memory_pool pool{cuda::devices[0]};
      CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));

      pool.enable_access_from(peers);
      CUDAX_CHECK(pool.is_accessible_from(peers.front()));

      pool.disable_access_from(peers.front());
      CUDAX_CHECK(!pool.is_accessible_from(peers.front()));

      if (peers.size() > 1)
      {
        CUDAX_CHECK(pool.is_accessible_from(peers[1]));
      }
    }
  }
}

#if _CCCL_CUDACC_AT_LEAST(12, 6)
C2H_CCCLRT_TEST("pinned_memory_pool::enable_access", "[memory_resource]")
{
  cudax::pinned_memory_pool pool{0};
  CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));

  // Currently bugged, need to wait for driver fix
  // pool.disable_access_from(cuda::devices[0]);
  // CUDAX_CHECK(!pool.is_accessible_from(cuda::devices[0]));

  // pool.enable_access_from(cuda::devices[0]);
  // CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));
}
#endif

C2H_CCCLRT_TEST("device_memory_pool with allocation handle", "[memory_resource]")
{
  cudax::memory_pool_properties props              = {20, 42, ::hipMemHandleTypePosixFileDescriptor};
  cudax::device_memory_pool with_allocation_handle = cudax::device_memory_pool(cuda::device_ref{0}, props);

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(
      ::hipDeviceGetDefaultMemPool, "Failed to call hipDeviceGetDefaultMemPool", &current_default_pool, 0);
  }

  ::hipMemPool_t get = with_allocation_handle.get();
  CHECK(get != current_default_pool);

  // Ensure we use the right release threshold
  CHECK(ensure_release_threshold(get, props.release_threshold));

  // Ensure that we disable reuse
  CHECK(ensure_disable_reuse(get));

  // Ensure that we disable export
  CHECK(ensure_export_handle(get, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
}

#if _CCCL_CUDACC_AT_LEAST(12, 6)
C2H_CCCLRT_TEST("pinned_memory_pool with allocation handle", "[memory_resource]")
{
  cudax::memory_pool_properties props              = {20, 42, ::hipMemHandleTypePosixFileDescriptor};
  cudax::pinned_memory_pool with_allocation_handle = cudax::pinned_memory_pool(0, props);

  ::hipMemPool_t get = with_allocation_handle.get();
  CHECK(get != cudax::pinned_memory_resource{}.get());

  // Ensure we use the right release threshold
  CHECK(ensure_release_threshold(get, props.release_threshold));

  // Ensure that we disable reuse
  CHECK(ensure_disable_reuse(get));

  // Ensure that we disable export
  CHECK(ensure_export_handle(get, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
}
#endif // _CCCL_CUDACC_AT_LEAST(12, 6)

// managed memory pool does not support allocation handles yet.
