//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream_ref>

#include <cuda/experimental/launch.cuh>
#include <cuda/experimental/memory_resource.cuh>
#include <cuda/experimental/stream.cuh>

#include <stdexcept>

#include <testing.cuh>

#if _CCCL_CUDACC_AT_LEAST(12, 6)
#  define TEST_TYPES cudax::device_memory_pool, cudax::pinned_memory_pool
#else
#  define TEST_TYPES cudax::device_memory_pool
#endif

namespace cudax = cuda::experimental;

template <typename PoolType>
void pool_static_asserts()
{
  static_assert(!cuda::std::is_trivial<PoolType>::value, "");
  static_assert(!cuda::std::is_trivially_default_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_copy_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_move_constructible<PoolType>::value, "");
  static_assert(!cuda::std::is_copy_assignable<PoolType>::value, "");
  static_assert(!cuda::std::is_move_assignable<PoolType>::value, "");
  static_assert(!cuda::std::is_trivially_destructible<PoolType>::value, "");
  static_assert(!cuda::std::is_empty<PoolType>::value, "");
}

template void pool_static_asserts<cudax::device_memory_pool>();
#if _CCCL_CUDACC_AT_LEAST(12, 6)
template void pool_static_asserts<cudax::pinned_memory_pool>();
#endif

static_assert(!cuda::std::is_default_constructible<cudax::device_memory_pool>::value, "");
#if _CCCL_CUDACC_AT_LEAST(12, 6)
static_assert(cuda::std::is_default_constructible<cudax::pinned_memory_pool>::value, "");
#endif

// TODO should this be part of the public API?
template <typename PoolType>
using memory_resource_for_pool =
  cuda::std::conditional_t<cuda::std::is_same_v<PoolType, cudax::device_memory_pool>,
                           cudax::device_memory_resource,
#if _CCCL_CUDACC_AT_LEAST(12, 6)
                           cudax::pinned_memory_resource
#else
                           void
#endif
                           >;

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool, const int driver_version)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  constexpr int min_async_version = 11050;
  return driver_version < min_async_version ? disable_reuse == 0 : disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to query hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

C2H_TEST_LIST("device_memory_pool construction", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = TestType;
  SECTION("Construct from device id")
  {
    memory_pool from_device{current_device};

    ::hipMemPool_t get = from_device.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with empty properties")
  {
    cudax::memory_pool_properties props{};
    memory_pool from_defaulted_properties{current_device, props};

    ::hipMemPool_t get = from_defaulted_properties.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with initial pool size")
  {
    cudax::memory_pool_properties props = {42, 20};
    memory_pool with_threshold{current_device, props};

    ::hipMemPool_t get = with_threshold.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  // Allocation handles are only supported after 11.2
  SECTION("Construct with allocation handle")
  {
    cudax::memory_pool_properties props = {
      42, 20, cudax::hipMemAllocationHandleType::hipMemHandleTypePosixFileDescriptor};
    memory_pool with_allocation_handle{current_device, props};

    ::hipMemPool_t get = with_allocation_handle.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
  }

  SECTION("Take ownership of native handle")
  {
    ::hipMemPoolProps pool_properties{};
    pool_properties.allocType   = ::hipMemAllocationTypePinned;
    pool_properties.handleTypes = ::hipMemAllocationHandleType(hipMemAllocationHandleType::hipMemHandleTypeNone);
    if (cuda::std::is_same_v<memory_pool, cudax::device_memory_pool>)
    {
      pool_properties.location.type = ::hipMemLocationTypeDevice;
      pool_properties.location.id   = current_device;
    }
    else
    {
#if _CCCL_CUDACC_AT_LEAST(12, 6)
      pool_properties.location.type = hipMemLocationTypeHostNuma;
      pool_properties.location.id   = 0;
#else
      REQUIRE(false);
#endif
    }
    ::hipMemPool_t new_pool{};
    _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &new_pool, &pool_properties);

    memory_pool from_handle = memory_pool::from_native_handle(new_pool);
    CHECK(from_handle.get() == new_pool);
  }
}

C2H_TEST_LIST("device_memory_pool comparison", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = TestType;
  memory_pool first{current_device};
  { // comparison against a plain device_memory_pool
    memory_pool second{current_device};
    CHECK(first == first);
    CHECK(first != second);
  }

  { // comparison against a hipMemPool_t
    CHECK(first == first.get());
    CHECK(first.get() == first);
    CHECK(first != current_default_pool);
    CHECK(current_default_pool != first);
  }
}

C2H_TEST_LIST("device_memory_pool accessors", "[memory_resource]", TEST_TYPES)
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = TestType;
  SECTION("device_memory_pool::set_attribute")
  {
    memory_pool pool{current_device};

    { // hipMemPoolReuseFollowEventDependencies
      // Get the attribute value
      bool attr = pool.attribute(::hipMemPoolReuseFollowEventDependencies) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseFollowEventDependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(::hipMemPoolReuseFollowEventDependencies) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseFollowEventDependencies, attr);
    }

    { // hipMemPoolReuseAllowOpportunistic
      // Get the attribute value
      bool attr = pool.attribute(::hipMemPoolReuseAllowOpportunistic) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseAllowOpportunistic, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(::hipMemPoolReuseAllowOpportunistic) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseAllowOpportunistic, attr);
    }

    { // hipMemPoolReuseAllowInternalDependencies
      // Get the attribute value
      bool attr = pool.attribute(::hipMemPoolReuseAllowInternalDependencies) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseAllowInternalDependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.attribute(::hipMemPoolReuseAllowInternalDependencies) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseAllowInternalDependencies, attr);
    }

    { // hipMemPoolAttrReleaseThreshold
      // Get the attribute value
      size_t attr = pool.attribute(::hipMemPoolAttrReleaseThreshold);

      // Set it to something else
      pool.set_attribute(::hipMemPoolAttrReleaseThreshold, 2 * attr);

      // Retrieve again and verify it was changed
      size_t new_attr = pool.attribute(::hipMemPoolAttrReleaseThreshold);
      CHECK(new_attr == 2 * attr);

      // Set it back
      pool.set_attribute(::hipMemPoolAttrReleaseThreshold, attr);
    }

    // prime the pool to a given size
    memory_resource_for_pool<memory_pool> resource{pool};
    cudax::stream stream{cuda::device_ref{0}};

    // Allocate a buffer to prime
    auto* ptr = resource.allocate(stream, 256 * sizeof(int));
    stream.sync();

    { // hipMemPoolAttrReservedMemHigh
      // Get the attribute value
      size_t attr = pool.attribute(::hipMemPoolAttrReservedMemHigh);

      // Set it to zero as everything else is illegal
      pool.set_attribute(::hipMemPoolAttrReservedMemHigh, 0);

      // Retrieve again and verify it was changed, which it wasn't...
      size_t new_attr = pool.attribute(::hipMemPoolAttrReservedMemHigh);
      CHECK(new_attr == attr);

#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(::hipMemPoolAttrReservedMemHigh, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "set_attribute: It is illegal to set this attribute to a non-zero value.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    { // hipMemPoolAttrUsedMemHigh
      // Get the attribute value
      size_t attr = pool.attribute(::hipMemPoolAttrUsedMemHigh);

      // Set it to zero as everything else is illegal
      pool.set_attribute(::hipMemPoolAttrUsedMemHigh, 0);

      // Retrieve again and verify it was changed, which it wasn't...
      size_t new_attr = pool.attribute(::hipMemPoolAttrUsedMemHigh);
      CHECK(new_attr == attr);

#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(::hipMemPoolAttrUsedMemHigh, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "set_attribute: It is illegal to set this attribute to a non-zero value.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    // Reallocate as the checks above have screwed with the allocation count
    resource.deallocate(stream, ptr, 256 * sizeof(int));
    ptr = resource.allocate(stream, 2048 * sizeof(int));
    stream.sync();

    { // hipMemPoolAttrReservedMemCurrent
      // Get the attribute value
      size_t attr = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
      CHECK(attr >= 2048 * sizeof(int));
      // hipMemPoolAttrReservedMemCurrent cannot be set
#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        pool.set_attribute(::hipMemPoolAttrReservedMemCurrent, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "Invalid attribute passed to set_attribute.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    { // hipMemPoolAttrUsedMemCurrent
      // Get the attribute value
      size_t attr = pool.attribute(::hipMemPoolAttrUsedMemCurrent);
      CHECK(attr == 2048 * sizeof(int));
      // hipMemPoolAttrUsedMemCurrent cannot be set
#if _CCCL_HAS_EXCEPTIONS()
      try
      {
        pool.set_attribute(::hipMemPoolAttrUsedMemCurrent, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "Invalid attribute passed to set_attribute.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
#endif // _CCCL_HAS_EXCEPTIONS()
    }

    // Free the last allocation
    resource.deallocate(stream, ptr, 2048 * sizeof(int));
    stream.sync();
  }

  SECTION("device_memory_pool::trim_to")
  {
    memory_pool pool{current_device};

    // prime the pool to a given size
    memory_resource_for_pool<memory_pool> resource{pool};
    cudax::stream stream{cuda::device_ref{0}};

    // Allocate 2 buffers
    auto* ptr1 = resource.allocate(stream, 2048 * sizeof(int));
    auto* ptr2 = resource.allocate(stream, 2048 * sizeof(int));
    resource.deallocate(stream, ptr1, 2048 * sizeof(int));
    stream.sync();

    // Ensure that we still hold some memory, otherwise everything is freed
    auto backing_size = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size >= 4096 * sizeof(int));

    // Trim the pool to something smaller than currently held
    pool.trim_to(1024);

    // Should be a noop
    auto noop_backing_size = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size == noop_backing_size);

    // Trim to larger than ever allocated
    pool.trim_to(backing_size * 24);

    // Should be a noop
    auto another_noop_backing_size = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size == another_noop_backing_size);

    // Trim to smaller than current backing but larger than current allocated
    pool.trim_to(2560 * sizeof(int));

    // Check the backing size again
    auto new_backing_size = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(new_backing_size <= backing_size);
    CHECK(new_backing_size >= 4096 * sizeof(int));

    // Free the last allocation
    resource.deallocate(stream, ptr2, 2048 * sizeof(int));
    stream.sync();

    // There is nothing allocated anymore, so all memory is released
    auto no_backing = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(no_backing == 0);

    // We can still trim the pool without effect
    pool.trim_to(2560 * sizeof(int));

    auto still_no_backing = pool.attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(still_no_backing == 0);
  }
}

C2H_TEST("device_memory_pool::enable_access", "[memory_resource]")
{
  if (cuda::devices.size() > 1)
  {
    auto peers = cuda::devices[0].peer_devices();
    if (peers.size() > 0)
    {
      cudax::device_memory_pool pool{cuda::devices[0]};
      CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));

      pool.enable_access_from(peers);
      CUDAX_CHECK(pool.is_accessible_from(peers.front()));

      pool.disable_access_from(peers.front());
      CUDAX_CHECK(!pool.is_accessible_from(peers.front()));

      if (peers.size() > 1)
      {
        CUDAX_CHECK(pool.is_accessible_from(peers[1]));
      }
    }
  }
}

#if _CCCL_CUDACC_AT_LEAST(12, 6)
C2H_TEST("pinned_memory_pool::enable_access", "[memory_resource]")
{
  cudax::pinned_memory_pool pool{};
  CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));

  // Currently bugged, need to wait for driver fix
  // pool.disable_access_from(cuda::devices[0]);
  // CUDAX_CHECK(!pool.is_accessible_from(cuda::devices[0]));

  // pool.enable_access_from(cuda::devices[0]);
  // CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));
}
#endif
