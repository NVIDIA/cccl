//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream_ref>

#include <cuda/experimental/launch.cuh>
#include <cuda/experimental/memory_resource.cuh>
#include <cuda/experimental/stream.cuh>

#include <stdexcept>

#include <catch2/catch.hpp>
#include <testing.cuh>

namespace cudax = cuda::experimental;
using pool      = cudax::device_memory_pool;
static_assert(!cuda::std::is_trivial<pool>::value, "");
static_assert(!cuda::std::is_trivially_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_constructible<pool>::value, "");
static_assert(!cuda::std::is_move_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_assignable<pool>::value, "");
static_assert(!cuda::std::is_move_assignable<pool>::value, "");
static_assert(!cuda::std::is_trivially_destructible<pool>::value, "");
static_assert(!cuda::std::is_empty<pool>::value, "");

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool, const int driver_version)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  constexpr int min_async_version = 11050;
  return driver_version < min_async_version ? disable_reuse == 0 : disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to querry hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

TEST_CASE("device_memory_pool construction", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to query current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = cudax::device_memory_pool;
  SECTION("Construct from device id")
  {
    cudax::device_memory_pool from_device{current_device};

    ::hipMemPool_t get = from_device.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with empty properties")
  {
    cudax::memory_pool_properties props{};
    memory_pool from_defaulted_properties{current_device, props};

    ::hipMemPool_t get = from_defaulted_properties.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with initial pool size")
  {
    cudax::memory_pool_properties props = {42, 20};
    memory_pool with_threshold{current_device, props};

    ::hipMemPool_t get = with_threshold.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  // Allocation handles are only supported after 11.2
#if _CCCL_CUDACC_AT_LEAST(11, 2)
  SECTION("Construct with allocation handle")
  {
    cudax::memory_pool_properties props = {
      42, 20, cudax::hipMemAllocationHandleType::hipMemHandleTypePosixFileDescriptor};
    memory_pool with_allocation_handle{current_device, props};

    ::hipMemPool_t get = with_allocation_handle.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
  }
#endif // _CCCL_CUDACC_AT_LEAST(11, 2)

  SECTION("Take ownership of native handle")
  {
    ::hipMemPoolProps pool_properties{};
    pool_properties.allocType     = ::hipMemAllocationTypePinned;
    pool_properties.handleTypes   = ::hipMemAllocationHandleType(hipMemAllocationHandleType::hipMemHandleTypeNone);
    pool_properties.location.type = ::hipMemLocationTypeDevice;
    pool_properties.location.id   = current_device;
    ::hipMemPool_t new_pool{};
    _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &new_pool, &pool_properties);

    cudax::device_memory_pool from_handle = cudax::device_memory_pool::from_native_handle(new_pool);
    CHECK(from_handle.get() == new_pool);
  }
}

TEST_CASE("device_memory_pool comparison", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to querry current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  cudax::device_memory_pool first{current_device};
  { // comparison against a plain device_memory_pool
    cudax::device_memory_pool second{current_device};
    CHECK(first == first);
    CHECK(first != second);
  }

  { // comparison against a hipMemPool_t
    CHECK(first == first.get());
    CHECK(first.get() == first);
    CHECK(first != current_default_pool);
    CHECK(current_default_pool != first);
  }
}

TEST_CASE("device_memory_pool accessors", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to querry current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  SECTION("device_memory_pool::set_attribute")
  {
    cudax::device_memory_pool pool{current_device};

    { // hipMemPoolReuseFollowEventDependencies
      // Get the attribute value
      bool attr = pool.get_attribute(::hipMemPoolReuseFollowEventDependencies) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseFollowEventDependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.get_attribute(::hipMemPoolReuseFollowEventDependencies) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseFollowEventDependencies, attr);
    }

    { // hipMemPoolReuseAllowOpportunistic
      // Get the attribute value
      bool attr = pool.get_attribute(::hipMemPoolReuseAllowOpportunistic) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseAllowOpportunistic, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.get_attribute(::hipMemPoolReuseAllowOpportunistic) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseAllowOpportunistic, attr);
    }

    { // hipMemPoolReuseAllowInternalDependencies
      // Get the attribute value
      bool attr = pool.get_attribute(::hipMemPoolReuseAllowInternalDependencies) != 0;

      // Set it to the opposite
      pool.set_attribute(::hipMemPoolReuseAllowInternalDependencies, !attr);

      // Retrieve again and verify it was changed
      bool new_attr = pool.get_attribute(::hipMemPoolReuseAllowInternalDependencies) != 0;
      CHECK(attr == !new_attr);

      // Set it back
      pool.set_attribute(::hipMemPoolReuseAllowInternalDependencies, attr);
    }

    { // hipMemPoolAttrReleaseThreshold
      // Get the attribute value
      size_t attr = pool.get_attribute(::hipMemPoolAttrReleaseThreshold);

      // Set it to something else
      pool.set_attribute(::hipMemPoolAttrReleaseThreshold, 2 * attr);

      // Retrieve again and verify it was changed
      size_t new_attr = pool.get_attribute(::hipMemPoolAttrReleaseThreshold);
      CHECK(new_attr == 2 * attr);

      // Set it back
      pool.set_attribute(::hipMemPoolAttrReleaseThreshold, attr);
    }

    // prime the pool to a given size
    cudax::device_memory_resource resource{pool};
    cudax::stream stream{};

    // Allocate a buffer to prime
    auto* ptr = resource.allocate_async(256 * sizeof(int), stream);
    stream.wait();

    { // hipMemPoolAttrReservedMemHigh
      // Get the attribute value
      size_t attr = pool.get_attribute(::hipMemPoolAttrReservedMemHigh);

      // Set it to zero as everything else is illegal
      pool.set_attribute(::hipMemPoolAttrReservedMemHigh, 0);

      // Retrieve again and verify it was changed, which it wasnt...
      size_t new_attr = pool.get_attribute(::hipMemPoolAttrReservedMemHigh);
      CHECK(new_attr == attr);

      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(::hipMemPoolAttrReservedMemHigh, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(),
                     "device_memory_pool::set_attribute: It is illegal to set this attribute to a non-zero value.")
              == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
    }

    { // hipMemPoolAttrUsedMemHigh
      // Get the attribute value
      size_t attr = pool.get_attribute(::hipMemPoolAttrUsedMemHigh);

      // Set it to zero as everything else is illegal
      pool.set_attribute(::hipMemPoolAttrUsedMemHigh, 0);

      // Retrieve again and verify it was changed, which it wasnt...
      size_t new_attr = pool.get_attribute(::hipMemPoolAttrUsedMemHigh);
      CHECK(new_attr == attr);

      try
      {
        // Ensure we catch the contract violation
        pool.set_attribute(::hipMemPoolAttrUsedMemHigh, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(),
                     "device_memory_pool::set_attribute: It is illegal to set this attribute to a non-zero value.")
              == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
    }

    // Reallocate as the checks above have screwed with the allocation count
    resource.deallocate_async(ptr, 256 * sizeof(int), stream);
    ptr = resource.allocate_async(2048 * sizeof(int), stream);
    stream.wait();

    { // hipMemPoolAttrReservedMemCurrent
      // Get the attribute value
      size_t attr = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
      CHECK(attr >= 2048 * sizeof(int));
      // hipMemPoolAttrReservedMemCurrent cannot be set
      try
      {
        pool.set_attribute(::hipMemPoolAttrReservedMemCurrent, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "Invalid attribute passed to device_memory_pool::set_attribute.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
    }

    { // hipMemPoolAttrUsedMemCurrent
      // Get the attribute value
      size_t attr = pool.get_attribute(::hipMemPoolAttrUsedMemCurrent);
      CHECK(attr == 2048 * sizeof(int));
      // hipMemPoolAttrUsedMemCurrent cannot be set
      try
      {
        pool.set_attribute(::hipMemPoolAttrUsedMemCurrent, attr);
        CHECK(false);
      }
      catch (::std::invalid_argument& err)
      {
        CHECK(strcmp(err.what(), "Invalid attribute passed to device_memory_pool::set_attribute.") == 0);
      }
      catch (...)
      {
        CHECK(false);
      }
    }

    // Free the last allocation
    resource.deallocate_async(ptr, 2048 * sizeof(int), stream);
    stream.wait();
  }

  SECTION("device_memory_pool::trim_to")
  {
    cudax::device_memory_pool pool{current_device};
    // prime the pool to a given size
    cudax::device_memory_resource resource{pool};
    cudax::stream stream{};

    // Allocate 2 buffers
    auto* ptr1 = resource.allocate_async(2048 * sizeof(int), stream);
    auto* ptr2 = resource.allocate_async(2048 * sizeof(int), stream);
    resource.deallocate_async(ptr1, 2048 * sizeof(int), stream);
    stream.wait();

    // Ensure that we still hold some memory, otherwise everything is freed
    auto backing_size = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size >= 4096 * sizeof(int));

    // Trim the pool to something smaller than currently held
    pool.trim_to(1024);

    // Should be a noop
    auto noop_backing_size = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size == noop_backing_size);

    // Trim to larger than ever allocated
    pool.trim_to(backing_size * 24);

    // Should be a noop
    auto another_noop_backing_size = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(backing_size == another_noop_backing_size);

    // Trim to smaller than current backing but larger than current allocated
    pool.trim_to(2560 * sizeof(int));

    // Check the backing size again
    auto new_backing_size = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(new_backing_size <= backing_size);
    CHECK(new_backing_size >= 4096 * sizeof(int));

    // Free the last allocation
    resource.deallocate_async(ptr2, 2048 * sizeof(int), stream);
    stream.wait();

    // There is nothing allocated anymore, so all memory is released
    auto no_backing = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(no_backing == 0);

    // We can still trim the pool without effect
    pool.trim_to(2560 * sizeof(int));

    auto still_no_backing = pool.get_attribute(::hipMemPoolAttrReservedMemCurrent);
    CHECK(still_no_backing == 0);
  }

  SECTION("device_memory_pool::enable_peer_access")
  {
    if (cudax::devices.size() > 1)
    {
      auto peers = cudax::devices[0].get_peers();
      if (peers.size() > 0)
      {
        cudax::device_memory_pool pool{cudax::devices[0]};
        CUDAX_CHECK(pool.is_accessible_from(cudax::devices[0]));

        pool.enable_peer_access_from(peers);
        CUDAX_CHECK(pool.is_accessible_from(peers.front()));

        pool.disable_peer_access_from(peers.front());
        CUDAX_CHECK(!pool.is_accessible_from(peers.front()));

        if (peers.size() > 1)
        {
          CUDAX_CHECK(pool.is_accessible_from(peers[1]));
        }
      }
    }
  }
}
