//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include "common.cuh"

TEST_CASE("Fill", "[data_manipulation]")
{
  cudax::stream _stream;
  SECTION("Host resource")
  {
    cudax::pinned_memory_resource host_resource;
    cudax::uninitialized_buffer<int, cuda::mr::device_accessible> buffer(host_resource, buffer_size);

    cudax::fill_bytes(_stream, buffer, fill_byte);

    check_result_and_erase(_stream, cuda::std::span(buffer));
  }

  SECTION("Device resource")
  {
    cudax::device_memory_resource device_resource;
    cudax::uninitialized_buffer<int, cuda::mr::device_accessible> buffer(device_resource, buffer_size);
    cudax::fill_bytes(_stream, buffer, fill_byte);

    std::vector<int> host_vector(42);
    CUDART(hipMemcpyAsync(
      host_vector.data(), buffer.data(), buffer.size() * sizeof(int), hipMemcpyDefault, _stream.get()));

    check_result_and_erase(_stream, host_vector);
  }
  SECTION("Launch transform")
  {
    cudax::pinned_memory_resource host_resource;
    cudax::weird_buffer buffer(host_resource, buffer_size);

    cudax::fill_bytes(_stream, buffer, fill_byte);
    check_result_and_erase(_stream, cuda::std::span(buffer.data, buffer.size));
  }
}

TEST_CASE("Mdspan Fill", "[data_manipulation]")
{
  cudax::stream stream;
  {
    cuda::std::dextents<size_t, 3> dynamic_extents{1, 2, 3};
    auto buffer = make_buffer_for_mdspan(dynamic_extents, 0);
    cuda::std::mdspan<int, decltype(dynamic_extents)> dynamic_mdspan(buffer.data(), dynamic_extents);

    cudax::fill_bytes(stream, dynamic_mdspan, fill_byte);
    check_result_and_erase(stream, cuda::std::span(buffer.data(), buffer.size()));
  }
  {
    cuda::std::extents<size_t, 2, cuda::std::dynamic_extent, 4> mixed_extents{1};
    auto buffer = make_buffer_for_mdspan(mixed_extents, 0);
    cuda::std::mdspan<int, decltype(mixed_extents)> mixed_mdspan(buffer.data(), mixed_extents);

    cudax::fill_bytes(stream, cuda::std::move(mixed_mdspan), fill_byte);
    check_result_and_erase(stream, cuda::std::span(buffer.data(), buffer.size()));
  }
  {
    using static_extents = cuda::std::extents<size_t, 2, 3, 4>;
    auto size            = cuda::std::layout_left::mapping<static_extents>().required_span_size();
    cudax::weird_buffer<cuda::std::mdspan<int, static_extents>> buffer(cudax::pinned_memory_resource{}, size);

    cudax::fill_bytes(stream, buffer, fill_byte);
    check_result_and_erase(stream, cuda::std::span(buffer.data, buffer.size));
  }
}
