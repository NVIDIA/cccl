//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include "common.cuh"

TEST_CASE("1d Copy", "[data_manipulation]")
{
  cudax::stream _stream;

  SECTION("Device resource")
  {
    cudax::device_memory_resource device_resource;
    std::vector<int> host_vector(buffer_size);

    {
      cudax::uninitialized_async_buffer<int, cuda::mr::device_accessible> buffer(device_resource, _stream, buffer_size);
      cudax::fill_bytes(_stream, buffer, fill_byte);

      cudax::copy_bytes(_stream, buffer, host_vector);
      check_result_and_erase(_stream, host_vector);

      cudax::copy_bytes(_stream, std::move(buffer), host_vector);
      check_result_and_erase(_stream, host_vector);
    }
    {
      cudax::uninitialized_async_buffer<int, cuda::mr::device_accessible> not_yet_const_buffer(
        device_resource, _stream, buffer_size);
      cudax::fill_bytes(_stream, not_yet_const_buffer, fill_byte);

      const auto& const_buffer = not_yet_const_buffer;

      cudax::copy_bytes(_stream, const_buffer, host_vector);
      check_result_and_erase(_stream, host_vector);

      cudax::copy_bytes(_stream, const_buffer, cuda::std::span(host_vector));
      check_result_and_erase(_stream, host_vector);
    }
  }

  SECTION("Host and managed resource")
  {
    cudax::managed_memory_resource managed_resource;
    cudax::pinned_memory_resource host_resource;

    {
      cudax::uninitialized_buffer<int, cuda::mr::host_accessible> host_buffer(host_resource, buffer_size);
      cudax::uninitialized_buffer<int, cuda::mr::device_accessible> device_buffer(managed_resource, buffer_size);

      cudax::fill_bytes(_stream, host_buffer, fill_byte);

      cudax::copy_bytes(_stream, host_buffer, device_buffer);
      check_result_and_erase(_stream, device_buffer);

      cudax::copy_bytes(_stream, cuda::std::span(host_buffer), device_buffer);
      check_result_and_erase(_stream, device_buffer);
    }

    {
      cudax::uninitialized_buffer<int, cuda::mr::host_accessible> not_yet_const_host_buffer(host_resource, buffer_size);
      cudax::uninitialized_buffer<int, cuda::mr::device_accessible> device_buffer(managed_resource, buffer_size);
      cudax::fill_bytes(_stream, not_yet_const_host_buffer, fill_byte);

      const auto& const_host_buffer = not_yet_const_host_buffer;

      cudax::copy_bytes(_stream, const_host_buffer, device_buffer);
      check_result_and_erase(_stream, device_buffer);

      cudax::copy_bytes(_stream, cuda::std::span(const_host_buffer), device_buffer);
      check_result_and_erase(_stream, device_buffer);
    }
  }
  SECTION("Launch transform")
  {
    cudax::pinned_memory_resource host_resource;
    cudax::weird_buffer input(host_resource, buffer_size);
    cudax::weird_buffer output(host_resource, buffer_size);

    memset(input.data, fill_byte, input.size * sizeof(int));

    cudax::copy_bytes(_stream, input, output);
    check_result_and_erase(_stream, cuda::std::span(output.data, output.size));
  }

  SECTION("Asymetric size")
  {
    cudax::pinned_memory_resource host_resource;
    cudax::uninitialized_buffer<int, cuda::mr::host_accessible> host_buffer(host_resource, 1);
    cudax::fill_bytes(_stream, host_buffer, fill_byte);

    ::std::vector<int> vec(buffer_size, 0xbeef);

    cudax::copy_bytes(_stream, host_buffer, vec);
    _stream.wait();

    CUDAX_REQUIRE(vec[0] == get_expected_value(fill_byte));
    CUDAX_REQUIRE(vec[1] == 0xbeef);
  }
}

template <typename SrcLayout = cuda::std::layout_right,
          typename DstLayout = SrcLayout,
          typename SrcExtents,
          typename DstExtents>
void test_mdspan_copy_bytes(
  cudax::stream_ref stream, SrcExtents src_extents = SrcExtents(), DstExtents dst_extents = DstExtents())
{
  auto src_buffer = make_buffer_for_mdspan<SrcLayout>(src_extents, 1);
  auto dst_buffer = make_buffer_for_mdspan<DstLayout>(dst_extents, 0);

  cuda::std::mdspan<int, SrcExtents, SrcLayout> src(src_buffer.data(), src_extents);
  cuda::std::mdspan<int, DstExtents, DstLayout> dst(dst_buffer.data(), dst_extents);

  for (int i = 0; i < static_cast<int>(src.extent(1)); i++)
  {
    src(0, i) = i;
  }

  cudax::copy_bytes(stream, std::move(src), dst);
  stream.wait();

  for (int i = 0; i < static_cast<int>(dst.extent(1)); i++)
  {
    CUDAX_CHECK(dst(0, i) == i);
  }
}

TEST_CASE("Mdspan copy", "[data_manipulation]")
{
  cudax::stream stream;

  SECTION("Different extents")
  {
    auto static_extents = cuda::std::extents<size_t, 3, 4>();
    test_mdspan_copy_bytes(stream, static_extents, static_extents);
    test_mdspan_copy_bytes<cuda::std::layout_left>(stream, static_extents, static_extents);

    auto dynamic_extents = cuda::std::dextents<size_t, 2>(3, 4);
    test_mdspan_copy_bytes(stream, dynamic_extents, dynamic_extents);
    test_mdspan_copy_bytes(stream, static_extents, dynamic_extents);
    test_mdspan_copy_bytes<cuda::std::layout_left>(stream, static_extents, dynamic_extents);

    auto mixed_extents = cuda::std::extents<int, cuda::std::dynamic_extent, 4>(3);
    test_mdspan_copy_bytes(stream, dynamic_extents, mixed_extents);
    test_mdspan_copy_bytes(stream, mixed_extents, static_extents);
    test_mdspan_copy_bytes<cuda::std::layout_left>(stream, mixed_extents, static_extents);
  }

  SECTION("Launch transform")
  {
    auto mixed_extents =
      cuda::std::extents<size_t, 1024, cuda::std::dynamic_extent, 2, cuda::std::dynamic_extent>(1024, 2);
    [[maybe_unused]] auto static_extents = cuda::std::extents<size_t, 1024, 1024, 2, 2>();
    auto mdspan_buffer                   = make_buffer_for_mdspan(mixed_extents, 1);
    cuda::std::mdspan<int, decltype(mixed_extents)> mdspan(mdspan_buffer.data(), mixed_extents);
    cudax::weird_buffer<cuda::std::mdspan<int, decltype(static_extents)>> buffer{
      cudax::pinned_memory_resource{}, mdspan.mapping().required_span_size()};

    cudax::copy_bytes(stream, mdspan, buffer);
    stream.wait();
    CUDAX_REQUIRE(!memcmp(mdspan_buffer.data(), buffer.data, mdspan_buffer.size()));
  }
}
