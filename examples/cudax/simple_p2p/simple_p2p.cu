#include "hip/hip_runtime.h"
/*
 * This sample demonstrates a combination of Peer-to-Peer (P2P) and
 * Unified Virtual Address Space (UVA) features new to SDK 4.0
 */

// includes, system
#include <cuda/experimental/device.cuh>

#include <stdio.h>
#include <stdlib.h>

__global__ void SimpleKernel(float* src, float* dst)
{
  // Just a dummy kernel, doing enough for us to verify that everything
  // worked
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx]      = src[idx] * 2.0f;
}

int main(int argc, char** argv)
{
  printf("[%s] - Starting...\n", argv[0]);

  // Number of GPUs
  printf("Checking for multiple GPUs...\n");
  printf("CUDA-capable device count: %llu\n", cudax::devices.size());

  if (cudax::devices.size())
  {
    printf("Two or more GPUs with Peer-to-Peer access capability are required for "
           "%s.\n",
           argv[0]);
    printf("Waiving test.\n");
    exit(2);
  }

  // Query device properties
  hipDeviceProp_t prop[64];
  int gpuid[2]; // we want to find the first two GPU's that can support P2P

  for (int i = 0; i < gpu_n; i++)
  {
    checkCudaErrors(hipGetDeviceProperties(&prop[i], i));
  }
  // Check possibility for peer access
  printf("\nChecking GPU(s) for support of peer to peer memory access...\n");

  int can_access_peer;
  int p2pCapableGPUs[2]; // We take only 1 pair of P2P capable GPUs
  p2pCapableGPUs[0] = p2pCapableGPUs[1] = -1;

  // Show all the combinations of supported P2P GPUs
  for (int i = 0; i < gpu_n; i++)
  {
    for (int j = 0; j < gpu_n; j++)
    {
      if (i == j)
      {
        continue;
      }
      checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, i, j));
      printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n",
             prop[i].name,
             i,
             prop[j].name,
             j,
             can_access_peer ? "Yes" : "No");
      if (can_access_peer && p2pCapableGPUs[0] == -1)
      {
        p2pCapableGPUs[0] = i;
        p2pCapableGPUs[1] = j;
      }
    }
  }

  if (p2pCapableGPUs[0] == -1 || p2pCapableGPUs[1] == -1)
  {
    printf("Two or more GPUs with Peer-to-Peer access capability are required for "
           "%s.\n",
           argv[0]);
    printf("Peer to Peer access is not available amongst GPUs in the system, "
           "waiving test.\n");

    exit(EXIT_WAIVED);
  }

  // Use first pair of p2p capable GPUs detected.
  gpuid[0] = p2pCapableGPUs[0];
  gpuid[1] = p2pCapableGPUs[1];

  // Enable peer access
  printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0], gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[1], 0));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[0], 0));

  // Allocate buffers
  const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
  printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n", int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  float* g0;
  checkCudaErrors(hipMalloc(&g0, buf_size));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  float* g1;
  checkCudaErrors(hipMalloc(&g1, buf_size));
  float* h0;
  checkCudaErrors(hipHostMalloc(&h0, buf_size)); // Automatically portable with UVA

  // Create CUDA event handles
  printf("Creating event handles...\n");
  hipEvent_t start_event, stop_event;
  float time_memcpy;
  int eventflags = hipEventBlockingSync;
  checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
  checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

  // P2P memcopy() benchmark
  checkCudaErrors(hipEventRecord(start_event, 0));

  for (int i = 0; i < 100; i++)
  {
    // With UVA we don't need to specify source and target devices, the
    // runtime figures this out by itself from the pointers
    // Ping-pong copy between GPUs
    if (i % 2 == 0)
    {
      checkCudaErrors(hipMemcpy(g1, g0, buf_size, hipMemcpyDefault));
    }
    else
    {
      checkCudaErrors(hipMemcpy(g0, g1, buf_size, hipMemcpyDefault));
    }
  }

  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
  printf("hipMemcpyPeer / hipMemcpy between GPU%d and GPU%d: %.2fGB/s\n",
         gpuid[0],
         gpuid[1],
         (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f / 1024.0f / 1024.0f);

  // Prepare host buffer and copy to GPU 0
  printf("Preparing host buffer and memcpy to GPU%d...\n", gpuid[0]);

  for (int i = 0; i < buf_size / sizeof(float); i++)
  {
    h0[i] = float(i % 4096);
  }

  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipMemcpy(g0, h0, buf_size, hipMemcpyDefault));

  // Kernel launch configuration
  const dim3 threads(512, 1);
  const dim3 blocks((buf_size / sizeof(float)) / threads.x, 1);

  // Run kernel on GPU 1, reading input from the GPU 0 buffer, writing
  // output to the GPU 1 buffer
  printf("Run kernel on GPU%d, taking source data from GPU%d and writing to "
         "GPU%d...\n",
         gpuid[1],
         gpuid[0],
         gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[1]));
  SimpleKernel<<<blocks, threads>>>(g0, g1);

  checkCudaErrors(hipDeviceSynchronize());

  // Run kernel on GPU 0, reading input from the GPU 1 buffer, writing
  // output to the GPU 0 buffer
  printf("Run kernel on GPU%d, taking source data from GPU%d and writing to "
         "GPU%d...\n",
         gpuid[0],
         gpuid[1],
         gpuid[0]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  SimpleKernel<<<blocks, threads>>>(g1, g0);

  checkCudaErrors(hipDeviceSynchronize());

  // Copy data back to host and verify
  printf("Copy data back to host from GPU%d and verify results...\n", gpuid[0]);
  checkCudaErrors(hipMemcpy(h0, g0, buf_size, hipMemcpyDefault));

  int error_count = 0;

  for (int i = 0; i < buf_size / sizeof(float); i++)
  {
    // Re-generate input data and apply 2x '* 2.0f' computation of both
    // kernel runs
    if (h0[i] != float(i % 4096) * 2.0f * 2.0f)
    {
      printf("Verification error @ element %i: val = %f, ref = %f\n", i, h0[i], (float(i % 4096) * 2.0f * 2.0f));

      if (error_count++ > 10)
      {
        break;
      }
    }
  }

  // Disable peer access (also unregisters memory for non-UVA cases)
  printf("Disabling peer access...\n");
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[1]));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[0]));

  // Cleanup and shutdown
  printf("Shutting down...\n");
  checkCudaErrors(hipEventDestroy(start_event));
  checkCudaErrors(hipEventDestroy(stop_event));
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipFree(g0));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipFree(g1));
  checkCudaErrors(hipHostFree(h0));

  for (int i = 0; i < gpu_n; i++)
  {
    checkCudaErrors(hipSetDevice(i));
  }

  if (error_count != 0)
  {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }
  else
  {
    printf("Test passed\n");
    exit(EXIT_SUCCESS);
  }
}
