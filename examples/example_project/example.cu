#include "hip/hip_runtime.h"

#include <cstdio>
#include <thrust/device_vector.h>
#include <cub/block/block_reduce.cuh>
#include <cuda/atomic>

__global__ void sumKernel(int const* data, int* result, std::size_t N)
{
    using BlockReduce = hipcub::BlockReduce<int, 256> ;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int sum = 0;
    if (index < N) {
        sum += data[index];
    }

    sum = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0){
        cuda::atomic_ref<int, cuda::thread_scope_device> atomic_result(*result);
        atomic_result.fetch_add(sum, cuda::memory_order_relaxed);
    }
}

int main()
{
    std::size_t N = 1000;
    thrust::device_vector<int> data(N, 1);
    thrust::device_vector<int> result(1);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    sumKernel<<<numBlocks, blockSize>>>(thrust::raw_pointer_cast(data.data()),
                                        thrust::raw_pointer_cast(result.data()), N);

    auto err = hipDeviceSynchronize();
    if(err != hipSuccess){
        std::cout << "Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Sum: " << result[0] << std::endl;

    assert(result[0] == N);

    return 0;
}