#include "hip/hip_runtime.h"


/*
This is a simple example demonstrating the use of CCCL functionality from Thrust, CUB, and libcu++.

The example computes the sum of an array of integers using a simple parallel reduction. Each thread block
computes the sum of a subset of the array using cuB::BlockRecuce. The sum of each block is then reduced 
to a single value using an atomic add via cuda::atomic_ref from libcu++. The result is stored in a device_vector
from Thrust. The sum is then printed to the console.
*/

#include <cstdio>
#include <thrust/device_vector.h>
#include <cub/block/block_reduce.cuh>
#include <cuda/atomic>

constexpr int block_size = 256;

__global__ void sumKernel(int const* data, int* result, std::size_t N)
{
    using BlockReduce = hipcub::BlockReduce<int, block_size> ;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int sum = 0;
    if (index < N) {
        sum += data[index];
    }

    sum = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0){
        cuda::atomic_ref<int, cuda::thread_scope_device> atomic_result(*result);
        atomic_result.fetch_add(sum, cuda::memory_order_relaxed);
    }
}

int main()
{
    std::size_t N = 1000;
    thrust::device_vector<int> data(N, 1);
    thrust::device_vector<int> result(1);

    int numBlocks = (N + block_size - 1) / block_size;

    sumKernel<<<numBlocks, block_size>>>(thrust::raw_pointer_cast(data.data()),
                                         thrust::raw_pointer_cast(result.data()), N);

    auto err = hipDeviceSynchronize();
    if(err != hipSuccess){
        std::cout << "Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Sum: " << result[0] << std::endl;

    assert(result[0] == N);

    return 0;
}
