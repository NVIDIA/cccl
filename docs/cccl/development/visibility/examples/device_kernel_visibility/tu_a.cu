#include "kernel.cuh"

void b_launch(void (*k)(char, size_t*), char c, size_t* d_out);

void a_launch(void (*k)(char, int*), char c, int* d_out)
{
  void* ptr = reinterpret_cast<void*>(k);

  printf("a: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("a: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("a: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  hipMemset(d_out, 0, sizeof(int));
  k<<<1, 1>>>(c, d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("a: FAILED to launch kernel\n");
  }
  else
  {
    printf("a: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("a: FAILED to synchronize stream\n");
  }
  else
  {
    printf("a: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("a: FAILED to copy from device to host\n");
  }
  else
  {
    printf("a: copied from device to host\n");
  }

  printf("a: out: %d\n", h_out);
  if (h_out != 42)
  {
    printf("a: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("a: kernel was launched: out == 42\n");
  }

  printf("\n");
}

void a()
{
  hipGetLastError();

  size_t* d_out{};
  hipMalloc(&d_out, sizeof(size_t));
  hipMemset(d_out, 0, sizeof(size_t));

  void* ptr = reinterpret_cast<void*>(kernel<size_t>);

  printf("a: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("a: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("a: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('a', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("b: FAILED to launch kernel\n");
  }
  else
  {
    printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("a: FAILED to synchronize stream\n");
  }
  else
  {
    printf("a: synchronized stream\n");
  }

  size_t h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(size_t), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("a: FAILED to copy from device to host\n");
  }
  else
  {
    printf("a: copied from device to host\n");
  }

  printf("a: out: %d\n", static_cast<int>(h_out));
  if (h_out != 42)
  {
    printf("a: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("a: kernel was launched: out == 42\n");
  }

  hipMemset(d_out, 0, sizeof(size_t));
  printf("\n");

  printf("a: defers launch to b\n");
  b_launch(kernel<size_t>, 'b', d_out);
}
