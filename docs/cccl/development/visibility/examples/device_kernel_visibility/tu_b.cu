#include "kernel.cuh"

void a_launch(void (*k)(char, int*), char c, int* d_out);

void b_launch(void (*k)(char, size_t*), char c, size_t* d_out)
{
  void* ptr = reinterpret_cast<void*>(k);

  printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  k<<<1, 1>>>(c, d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("b: FAILED to launch kernel\n");
  }
  else
  {
    printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    printf("b: synchronized stream\n");
  }

  size_t h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(size_t), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    printf("b: copied from device to host\n");
  }

  printf("b: out: %d\n", static_cast<int>(h_out));
  if (h_out != 42)
  {
    printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("b: kernel was launched: out == 42\n");
  }

  printf("\n");
}

void b()
{
  hipGetLastError();

  int* d_out{};
  hipMalloc(&d_out, sizeof(int));
  hipMemset(d_out, 0, sizeof(int));

  void* ptr = reinterpret_cast<void*>(kernel<int>);

  printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('b', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("b: FAILED to launch kernel\n");
  }
  else
  {
    printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    printf("b: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    printf("b: copied from device to host\n");
  }

  printf("b: out: %d\n", h_out);

  if (h_out != 42)
  {
    printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("b: kernel was launched: out == 42\n");
  }

  hipMemset(d_out, 0, sizeof(int));
  printf("\n");

  printf("b: defers launch to a\n");
  a_launch(kernel<int>, 'b', d_out);
}
