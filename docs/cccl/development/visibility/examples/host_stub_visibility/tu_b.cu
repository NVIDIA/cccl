#include "kernel.cuh"

void b()
{
  int* d_out{};
  hipMalloc(&d_out, sizeof(int));
  hipMemset(d_out, 0, sizeof(int));

  void* ptr = reinterpret_cast<void*>(kernel<int>);

  printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('b', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("b: FAILED to launch kernel\n");
  }
  else
  {
    printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    printf("b: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    printf("b: copied from device to host\n");
  }

  printf("b: out: %d\n", h_out);

  if (h_out != 42)
  {
    printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("b: kernel was launched: out == 42\n");
  }

  printf("\n");
}
