#include "kernel.cuh"

void a()
{
  int* d_out{};
  hipMalloc(&d_out, sizeof(int));
  hipMemset(d_out, 0, sizeof(int));

  void* ptr = reinterpret_cast<void*>(kernel<int>);

  printf("a: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    printf("a: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    printf("a: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('a', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    printf("b: FAILED to launch kernel\n");
  }
  else
  {
    printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    printf("a: FAILED to synchronize stream\n");
  }
  else
  {
    printf("a: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    printf("a: FAILED to copy from device to host\n");
  }
  else
  {
    printf("a: copied from device to host\n");
  }

  printf("a: out: %d\n", h_out);
  if (h_out != 42)
  {
    printf("a: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    printf("a: kernel was launched: out == 42\n");
  }

  printf("\n");
}
