#include "kernel.cuh"

void a_launch(void (*k)(char, int*), char c, int* d_out);

void b_launch(void (*k)(char, size_t*), char c, size_t* d_out)
{
  void* ptr = reinterpret_cast<void*>(k);

  std::printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    std::printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    std::printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  k<<<1, 1>>>(c, d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    std::printf("b: FAILED to launch kernel\n");
  }
  else
  {
    std::printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    std::printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    std::printf("b: synchronized stream\n");
  }

  size_t h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(size_t), hipMemcpyDeviceToHost) != hipSuccess)
  {
    std::printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    std::printf("b: copied from device to host\n");
  }

  std::printf("b: out: %d\n", static_cast<int>(h_out));
  if (h_out != 42)
  {
    std::printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    std::printf("b: kernel was launched: out == 42\n");
  }

  std::printf("\n");
}

void b()
{
  hipGetLastError();

  int* d_out{};
  hipMalloc(&d_out, sizeof(int));
  hipMemset(d_out, 0, sizeof(int));

  void* ptr = reinterpret_cast<void*>(kernel<int>);

  std::printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    std::printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    std::printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('b', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    std::printf("b: FAILED to launch kernel\n");
  }
  else
  {
    std::printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    std::printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    std::printf("b: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    std::printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    std::printf("b: copied from device to host\n");
  }

  std::printf("b: out: %d\n", h_out);

  if (h_out != 42)
  {
    std::printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    std::printf("b: kernel was launched: out == 42\n");
  }

  hipMemset(d_out, 0, sizeof(int));
  std::printf("\n");

  std::printf("b: defers launch to a\n");
  a_launch(kernel<int>, 'b', d_out);
}
