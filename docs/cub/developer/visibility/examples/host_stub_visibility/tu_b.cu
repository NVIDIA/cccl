#include "kernel.cuh"

void b()
{
  int* d_out{};
  hipMalloc(&d_out, sizeof(int));
  hipMemset(d_out, 0, sizeof(int));

  void* ptr = reinterpret_cast<void*>(kernel<int>);

  std::printf("b: kernel stub address: %p\n", ptr);

  hipFunction_t func{};
  if (hipError_t error = hipGetFuncBySymbol(&func, ptr))
  {
    std::printf("b: kernel NOT found in mapping: %s\n", hipGetErrorString(error));
  }
  else
  {
    std::printf("b: kernel is in mapping: %s\n", hipGetErrorString(error));
  }

  kernel<<<1, 1>>>('b', d_out);

  if (hipPeekAtLastError() != hipSuccess)
  {
    std::printf("b: FAILED to launch kernel\n");
  }
  else
  {
    std::printf("b: launched kernel\n");
  }

  if (hipStreamSynchronize(0) != hipSuccess)
  {
    std::printf("b: FAILED to synchronize stream\n");
  }
  else
  {
    std::printf("b: synchronized stream\n");
  }

  int h_out{};
  if (hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
  {
    std::printf("b: FAILED to copy from device to host\n");
  }
  else
  {
    std::printf("b: copied from device to host\n");
  }

  std::printf("b: out: %d\n", h_out);

  if (h_out != 42)
  {
    std::printf("b: kernel was NOT actually launched: out != 42\n");
  }
  else
  {
    std::printf("b: kernel was launched: out == 42\n");
  }

  std::printf("\n");
}
