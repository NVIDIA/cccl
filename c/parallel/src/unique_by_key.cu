//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/block/block_scan.cuh>
#include <cub/detail/choose_offset.cuh>
#include <cub/detail/launcher/cuda_driver.cuh>
#include <cub/detail/ptx-json-parser.h>
#include <cub/device/device_select.cuh>

#include <format>
#include <vector>

#include <cccl/c/unique_by_key.h>
#include <kernels/iterators.h>
#include <kernels/operators.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>
#include <util/build_utils.h>
#include <util/context.h>
#include <util/indirect_arg.h>
#include <util/scan_tile_state.h>
#include <util/tuning.h>
#include <util/types.h>

struct op_wrapper;
struct device_unique_by_key_policy;
using OffsetT = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be unsigned long long");

struct num_selected_storage_t;

namespace unique_by_key
{
struct unique_by_key_runtime_tuning_policy
{
  hipcub::detail::RuntimeUniqueByKeyAgentPolicy unique_by_key;

  auto UniqueByKey() const
  {
    return unique_by_key;
  }

  using UniqueByKeyPolicyT = hipcub::detail::RuntimeUniqueByKeyAgentPolicy;
  using MaxPolicy          = unique_by_key_runtime_tuning_policy;

  template <typename F>
  hipError_t Invoke(int, F& op)
  {
    return op.template Invoke<unique_by_key_runtime_tuning_policy>(*this);
  }
};

enum class unique_by_key_iterator_t
{
  input_keys    = 0,
  input_values  = 1,
  output_keys   = 2,
  output_values = 3,
  num_selected  = 4
};

template <typename StorageT = storage_t>
std::string get_iterator_name(cccl_iterator_t iterator, unique_by_key_iterator_t which_iterator)
{
  if (iterator.type == cccl_iterator_kind_t::CCCL_POINTER)
  {
    return cccl_type_enum_to_name<StorageT>(iterator.value_type.type, true);
  }
  else
  {
    std::string iterator_t;
    switch (which_iterator)
    {
      case unique_by_key_iterator_t::input_keys:
        return "input_keys_iterator_state_t";
        break;
      case unique_by_key_iterator_t::input_values:
        return "input_values_iterator_state_t";
        break;
      case unique_by_key_iterator_t::output_keys:
        return "output_keys_iterator_t";
        break;
      case unique_by_key_iterator_t::output_values:
        return "output_values_iterator_t";
        break;
      case unique_by_key_iterator_t::num_selected:
        return "output_num_selected_iterator_t";
        break;
    }

    return iterator_t;
  }
}

std::string get_compact_init_kernel_name(cccl_iterator_t output_num_selected_it)
{
  std::string offset_t;
  check(cccl_type_name_from_nvrtc<OffsetT>(&offset_t));

  const std::string num_selected_iterator_t =
    get_iterator_name(output_num_selected_it, unique_by_key_iterator_t::num_selected);

  return std::format(
    "hipcub::detail::scan::DeviceCompactInitKernel<hipcub::ScanTileState<{0}>, {1}>", offset_t, num_selected_iterator_t);
}

std::string get_sweep_kernel_name(
  cccl_iterator_t input_keys_it,
  cccl_iterator_t input_values_it,
  cccl_iterator_t output_keys_it,
  cccl_iterator_t output_values_it,
  cccl_iterator_t output_num_selected_it)
{
  std::string chained_policy_t;
  check(cccl_type_name_from_nvrtc<device_unique_by_key_policy>(&chained_policy_t));

  const std::string input_keys_iterator_t = get_iterator_name(input_keys_it, unique_by_key_iterator_t::input_keys);
  const std::string input_values_iterator_t =
    get_iterator_name<items_storage_t>(input_values_it, unique_by_key_iterator_t::input_values);
  const std::string output_keys_iterator_t = get_iterator_name(output_keys_it, unique_by_key_iterator_t::output_keys);
  const std::string output_values_iterator_t =
    get_iterator_name<items_storage_t>(output_values_it, unique_by_key_iterator_t::output_values);
  const std::string output_num_selected_iterator_t =
    get_iterator_name<num_selected_storage_t>(output_num_selected_it, unique_by_key_iterator_t::num_selected);

  std::string offset_t;
  check(cccl_type_name_from_nvrtc<OffsetT>(&offset_t));

  auto tile_state_t = std::format("hipcub::ScanTileState<{0}>", offset_t);

  std::string equality_op_t;
  check(cccl_type_name_from_nvrtc<op_wrapper>(&equality_op_t));

  return std::format(
    "hipcub::detail::unique_by_key::DeviceUniqueByKeySweepKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}, {7}, {8}, "
    "device_unique_by_key_vsmem_helper>",
    chained_policy_t,
    input_keys_iterator_t,
    input_values_iterator_t,
    output_keys_iterator_t,
    output_values_iterator_t,
    output_num_selected_iterator_t,
    tile_state_t,
    equality_op_t,
    offset_t);
}

struct unique_by_key_kernel_source
{
  cccl_device_unique_by_key_build_result_t& build;

  hipKernel_t UniqueByKeySweepKernel() const
  {
    return build.sweep_kernel;
  }

  hipKernel_t CompactInitKernel() const
  {
    return build.compact_init_kernel;
  }

  scan_tile_state TileState()
  {
    return {build.description_bytes_per_tile, build.payload_bytes_per_tile};
  }
};

struct dynamic_vsmem_helper_t
{
  template <typename PolicyT, typename... Ts>
  static int BlockThreads(PolicyT policy)
  {
    return policy.BlockThreads();
  }

  template <typename PolicyT, typename... Ts>
  static int ItemsPerThread(PolicyT policy)
  {
    return policy.ItemsPerThread();
  }

  template <typename PolicyT, typename... Ts>
  static ::cuda::std::size_t VSMemPerBlock(PolicyT /*policy*/)
  {
    return 0;
  }
};

} // namespace unique_by_key

hipError_t cccl_device_unique_by_key_build_ex(
  cccl_device_unique_by_key_build_result_t* build_ptr,
  cccl_iterator_t input_keys_it,
  cccl_iterator_t input_values_it,
  cccl_iterator_t output_keys_it,
  cccl_iterator_t output_values_it,
  cccl_iterator_t output_num_selected_it,
  cccl_op_t op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path,
  cccl_build_config* config)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "test";

    const int cc                              = cc_major * 10 + cc_minor;
    const auto input_keys_it_value_t          = cccl_type_enum_to_name(input_keys_it.value_type.type);
    const auto input_values_it_value_t        = cccl_type_enum_to_name(input_values_it.value_type.type);
    const auto output_keys_it_value_t         = cccl_type_enum_to_name(output_keys_it.value_type.type);
    const auto output_values_it_value_t       = cccl_type_enum_to_name(output_values_it.value_type.type);
    const auto output_num_selected_it_value_t = cccl_type_enum_to_name(output_num_selected_it.value_type.type);
    const auto offset_cpp                     = cccl_type_enum_to_name(cccl_type_enum::CCCL_UINT64);
    const cccl_type_info offset_t{sizeof(OffsetT), alignof(OffsetT), cccl_type_enum::CCCL_UINT64};

    const std::string input_keys_iterator_src = make_kernel_input_iterator(
      offset_cpp,
      get_iterator_name(input_keys_it, unique_by_key::unique_by_key_iterator_t::input_keys),
      input_keys_it_value_t,
      input_keys_it);
    const std::string input_values_iterator_src = make_kernel_input_iterator(
      offset_cpp,
      get_iterator_name(input_values_it, unique_by_key::unique_by_key_iterator_t::input_values),
      input_values_it_value_t,
      input_values_it);
    const std::string output_keys_iterator_src = make_kernel_output_iterator(
      offset_cpp,
      get_iterator_name(output_keys_it, unique_by_key::unique_by_key_iterator_t::output_keys),
      output_keys_it_value_t,
      output_keys_it);
    const std::string output_values_iterator_src = make_kernel_output_iterator(
      offset_cpp,
      get_iterator_name(output_values_it, unique_by_key::unique_by_key_iterator_t::output_values),
      output_values_it_value_t,
      output_values_it);
    const std::string output_num_selected_iterator_src = make_kernel_output_iterator(
      offset_cpp,
      get_iterator_name(output_num_selected_it, unique_by_key::unique_by_key_iterator_t::num_selected),
      output_num_selected_it_value_t,
      output_num_selected_it);

    const std::string op_src = make_kernel_user_comparison_operator(input_keys_it_value_t, op);

    std::string policy_hub_expr =
      std::format("hipcub::detail::unique_by_key::policy_hub<{}, {}>", input_keys_it_value_t, input_values_it_value_t);

    std::string final_src = std::format(
      R"XXX(
#include <cub/device/dispatch/tuning/tuning_unique_by_key.cuh>
#include <cub/device/dispatch/kernels/kernel_scan.cuh>
#include <cub/device/dispatch/kernels/kernel_unique_by_key.cuh>
#include <cub/agent/single_pass_scan_operators.cuh>
struct __align__({1}) storage_t {{
  char data[{0}];
}};
struct __align__({3}) items_storage_t {{
  char data[{2}];
}};
struct __align__({5}) num_out_storage_t {{
  char data[{4}];
}};
{6}
{7}
{8}
{9}
{10}
{11}
using device_unique_by_key_policy = {12}::MaxPolicy;

struct device_unique_by_key_vsmem_helper {{
  template<typename ActivePolicyT, typename... Ts>
  struct VSMemHelperDefaultFallbackPolicyT {{
    using agent_policy_t = device_unique_by_key_policy::ActivePolicy::UniqueByKeyPolicyT;
    using agent_t = hipcub::detail::unique_by_key::AgentUniqueByKey<agent_policy_t, Ts...>;
    using static_temp_storage_t = typename hipcub::detail::unique_by_key::AgentUniqueByKey<agent_policy_t, Ts...>::TempStorage;
    static _CCCL_DEVICE _CCCL_FORCEINLINE static_temp_storage_t& get_temp_storage(
      static_temp_storage_t& static_temp_storage, hipcub::detail::vsmem_t& vsmem, ::cuda::std::size_t linear_block_id)
    {{
        return static_temp_storage;
    }}
    template <bool needs_vsmem_ = false, ::cuda::std::enable_if_t<!needs_vsmem_, int> = 0>
    static _CCCL_DEVICE _CCCL_FORCEINLINE bool discard_temp_storage(static_temp_storage_t& temp_storage)
    {{
      return false;
    }}
  }};
}};

#include <cub/detail/ptx-json/json.h>
__device__ consteval auto& policy_generator() {{
  return ptx_json::id<ptx_json::string("device_unique_by_key_policy")>()
    = hipcub::detail::unique_by_key::UniqueByKeyPolicyWrapper<device_unique_by_key_policy::ActivePolicy>::EncodedPolicy();
}}
)XXX",
      input_keys_it.value_type.size, // 0
      input_keys_it.value_type.alignment, // 1
      input_values_it.value_type.size, // 2
      input_values_it.value_type.alignment, // 3
      output_values_it.value_type.size, // 4
      output_values_it.value_type.alignment, // 5
      input_keys_iterator_src, // 6
      input_values_iterator_src, // 7
      output_keys_iterator_src, // 8
      output_values_iterator_src, // 9
      output_num_selected_iterator_src, // 10
      op_src, // 11
      policy_hub_expr); // 12

#if false // CCCL_DEBUGGING_SWITCH
      fflush(stderr);
      printf("\nCODE4NVRTC BEGIN\n%sCODE4NVRTC END\n", final_src.c_str());
      fflush(stdout);
#endif

    std::string compact_init_kernel_name = unique_by_key::get_compact_init_kernel_name(output_num_selected_it);
    std::string sweep_kernel_name        = unique_by_key::get_sweep_kernel_name(
      input_keys_it, input_values_it, output_keys_it, output_values_it, output_num_selected_it);
    std::string compact_init_kernel_lowered_name;
    std::string sweep_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    std::vector<const char*> args = {
      arch.c_str(),
      cub_path,
      thrust_path,
      libcudacxx_path,
      ctk_path,
      "-rdc=true",
      "-dlto",
      "-DCUB_DISABLE_CDP",
      "-DCUB_ENABLE_POLICY_PTX_JSON",
      "-std=c++20"};

    cccl::detail::extend_args_with_build_config(args, config);

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_linkable_list linkable_list;
    nvrtc_linkable_list_appender appender{linkable_list};

    appender.append_operation(op);
    appender.add_iterator_definition(input_keys_it);
    appender.add_iterator_definition(input_values_it);
    appender.add_iterator_definition(output_keys_it);
    appender.add_iterator_definition(output_values_it);
    appender.add_iterator_definition(output_num_selected_it);

    nvrtc_link_result result =
      begin_linking_nvrtc_program(num_lto_args, lopts)
        ->add_program(nvrtc_translation_unit{final_src.c_str(), name})
        ->add_expression({compact_init_kernel_name})
        ->add_expression({sweep_kernel_name})
        ->compile_program({args.data(), args.size()})
        ->get_name({compact_init_kernel_name, compact_init_kernel_lowered_name})
        ->get_name({sweep_kernel_name, sweep_kernel_lowered_name})
        ->link_program()
        ->add_link_list(linkable_list)
        ->finalize_program();

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(
      &build_ptr->compact_init_kernel, build_ptr->library, compact_init_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(&build_ptr->sweep_kernel, build_ptr->library, sweep_kernel_lowered_name.c_str()));

    auto [description_bytes_per_tile,
          payload_bytes_per_tile] = get_tile_state_bytes_per_tile(offset_t, offset_cpp, args.data(), args.size(), arch);

    nlohmann::json runtime_policy =
      hipcub::detail::ptx_json::parse("device_unique_by_key_policy", {result.data.get(), result.size});

    using hipcub::detail::RuntimeUniqueByKeyAgentPolicy;
    auto ubk_policy = RuntimeUniqueByKeyAgentPolicy::from_json(runtime_policy, "UniqueByKeyPolicyT");

    build_ptr->cc                         = cc;
    build_ptr->cubin                      = (void*) result.data.release();
    build_ptr->cubin_size                 = result.size;
    build_ptr->description_bytes_per_tile = description_bytes_per_tile;
    build_ptr->payload_bytes_per_tile     = payload_bytes_per_tile;
    build_ptr->runtime_policy             = new unique_by_key::unique_by_key_runtime_tuning_policy{ubk_policy};
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_unique_by_key_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

hipError_t cccl_device_unique_by_key(
  cccl_device_unique_by_key_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_keys_in,
  cccl_iterator_t d_values_in,
  cccl_iterator_t d_keys_out,
  cccl_iterator_t d_values_out,
  cccl_iterator_t d_num_selected_out,
  cccl_op_t op,
  uint64_t num_items,
  hipStream_t stream)
{
  hipError_t error = hipSuccess;
  bool pushed    = false;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    auto exec_status = hipcub::DispatchUniqueByKey<
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      OffsetT,
      unique_by_key::unique_by_key_runtime_tuning_policy,
      unique_by_key::unique_by_key_kernel_source,
      hipcub::detail::CudaDriverLauncherFactory,
      unique_by_key::dynamic_vsmem_helper_t,
      indirect_arg_t,
      indirect_arg_t>::
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_keys_in,
        d_values_in,
        d_keys_out,
        d_values_out,
        d_num_selected_out,
        op,
        num_items,
        stream,
        {build},
        hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        *reinterpret_cast<unique_by_key::unique_by_key_runtime_tuning_policy*>(build.runtime_policy));

    error = static_cast<hipError_t>(exec_status);
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_unique_by_key(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_unique_by_key_build(
  cccl_device_unique_by_key_build_result_t* build,
  cccl_iterator_t d_keys_in,
  cccl_iterator_t d_values_in,
  cccl_iterator_t d_keys_out,
  cccl_iterator_t d_values_out,
  cccl_iterator_t d_num_selected_out,
  cccl_op_t op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  return cccl_device_unique_by_key_build_ex(
    build,
    d_keys_in,
    d_values_in,
    d_keys_out,
    d_values_out,
    d_num_selected_out,
    op,
    cc_major,
    cc_minor,
    cub_path,
    thrust_path,
    libcudacxx_path,
    ctk_path,
    nullptr);
}

hipError_t cccl_device_unique_by_key_cleanup(cccl_device_unique_by_key_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    std::unique_ptr<char[]> policy(reinterpret_cast<char*>(build_ptr->runtime_policy));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_unique_by_key_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}
