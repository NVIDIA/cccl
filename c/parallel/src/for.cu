//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/grid/grid_even_share.cuh>
#include <cub/util_device.cuh>

#include <format>
#include <type_traits>

#include <cccl/c/for.h>
#include <cccl/c/types.h>
#include <for/for_op_helper.h>
#include <nvrtc/command_list.h>
#include <util/context.h>
#include <util/errors.h>
#include <util/types.h>

struct op_wrapper;
struct device_reduce_policy;

using OffsetT = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

static hipError_t
Invoke(cccl_iterator_t d_in, size_t num_items, cccl_op_t op, int /*cc*/, hipFunction_t static_kernel, hipStream_t stream)
{
  hipError_t error = hipSuccess;

  if (num_items == 0)
  {
    return error;
  }

  auto for_kernel_state = make_for_kernel_state(op, d_in);

  void* args[] = {&num_items, for_kernel_state.get()};

  int thread_count = 256;
  int block_count  = (num_items + 511) / 512;
  check(hipModuleLaunchKernel(static_kernel, block_count, 1, 1, thread_count, 1, 1, 0, stream, args, 0));

  // Check for failure to launch
  error = HipcubDebug(hipPeekAtLastError());

  return error;
}

struct for_each_wrapper;

static std::string get_device_for_kernel_name()
{
  std::string offset_t;
  std::string function_op_t;
  check(nvrtcGetTypeName<for_each_wrapper>(&function_op_t));
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  return std::format("hipcub::detail::for_each::static_kernel<device_for_policy, {0}, {1}>", offset_t, function_op_t);
}

hipError_t cccl_device_for_build(
  cccl_device_for_build_result_t* build_ptr,
  cccl_iterator_t d_data,
  cccl_op_t op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  hipError_t error = hipSuccess;

  try
  {
    if (d_data.type == cccl_iterator_kind_t::CCCL_ITERATOR)
    {
      throw std::runtime_error(std::string("Iterators are unsupported in for_each currently"));
    }

    const char* name = "test";

    const int cc = cc_major * 10 + cc_minor;

    const std::string for_kernel_name   = get_device_for_kernel_name();
    const std::string device_for_kernel = get_for_kernel(op, d_data);

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    constexpr size_t num_args  = 8;
    const char* args[num_args] = {
      arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto", "-DCUB_DISABLE_CDP"};

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    std::string lowered_name;

    auto cl =
      begin_linking_nvrtc_program(num_lto_args, lopts)
        ->add_program(nvrtc_translation_unit{device_for_kernel, name})
        ->add_expression({for_kernel_name})
        ->compile_program({args, num_args})
        ->get_name({for_kernel_name, lowered_name})
        ->link_program()
        ->add_link({op.ltoir, op.ltoir_size});

    nvrtc_link_result result{};

    if (cccl_iterator_kind_t::CCCL_ITERATOR == d_data.type)
    {
      result = cl->add_link({d_data.advance.ltoir, d_data.advance.ltoir_size})
                 ->add_link({d_data.dereference.ltoir, d_data.dereference.ltoir_size})
                 ->finalize_program();
    }
    else
    {
      result = cl->finalize_program();
    }

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build_ptr->static_kernel, build_ptr->library, lowered_name.c_str()));

    build_ptr->cc         = cc;
    build_ptr->cubin      = (void*) result.data.release();
    build_ptr->cubin_size = result.size;
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }
  return error;
}

hipError_t cccl_device_for(
  cccl_device_for_build_result_t build, cccl_iterator_t d_data, uint64_t num_items, cccl_op_t op, hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;

  try
  {
    pushed           = try_push_context();
    auto exec_status = Invoke(d_data, num_items, op, build.cc, (hipFunction_t) build.static_kernel, stream);
    error            = static_cast<hipError_t>(exec_status);
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_for_cleanup(cccl_device_for_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (...)
  {
    return hipErrorUnknown;
  }

  return hipSuccess;
}
