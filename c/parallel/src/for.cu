#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/grid/grid_even_share.cuh>
#include <cub/util_device.cuh>

#include <format>
#include <type_traits>
#include <vector>

#include <cccl/c/for.h>
#include <cccl/c/types.h>
#include <for/for_op_helper.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>
#include <util/build_utils.h>
#include <util/context.h>
#include <util/errors.h>
#include <util/types.h>

struct op_wrapper;
struct device_reduce_policy;

using OffsetT = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

static hipError_t
Invoke(cccl_iterator_t d_in, size_t num_items, cccl_op_t op, int /*cc*/, hipFunction_t static_kernel, hipStream_t stream)
{
  hipError_t error = hipSuccess;

  if (num_items == 0)
  {
    return error;
  }

  auto for_kernel_state = make_for_kernel_state(op, d_in);

  void* args[] = {&num_items, for_kernel_state.get()};

  const unsigned int thread_count = 256;
  const size_t items_per_block    = 512;
  const size_t block_sz           = cuda::ceil_div(num_items, items_per_block);

  if (block_sz > std::numeric_limits<unsigned int>::max())
  {
    return hipErrorInvalidValue;
  }
  const unsigned int block_count = static_cast<unsigned int>(block_sz);

  check(hipModuleLaunchKernel(static_kernel, block_count, 1, 1, thread_count, 1, 1, 0, stream, args, 0));

  // Check for failure to launch
  error = HipcubDebug(hipPeekAtLastError());

  return error;
}

struct for_each_wrapper;

static std::string get_device_for_kernel_name()
{
  std::string offset_t;
  std::string function_op_t;
  check(cccl_type_name_from_nvrtc<for_each_wrapper>(&function_op_t));
  check(cccl_type_name_from_nvrtc<OffsetT>(&offset_t));

  return std::format("hipcub::detail::for_each::static_kernel<device_for_policy, {0}, {1}>", offset_t, function_op_t);
}

hipError_t cccl_device_for_build_ex(
  cccl_device_for_build_result_t* build_ptr,
  cccl_iterator_t d_data,
  cccl_op_t op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path,
  cccl_build_config* config)
{
  hipError_t error = hipSuccess;

  try
  {
    if (d_data.type == cccl_iterator_kind_t::CCCL_ITERATOR)
    {
      throw std::runtime_error(std::string("Iterators are unsupported in for_each currently"));
    }

    const char* name = "test";

    const int cc = cc_major * 10 + cc_minor;

    const std::string for_kernel_name   = get_device_for_kernel_name();
    const std::string device_for_kernel = get_for_kernel(op, d_data);

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    std::vector<const char*> args = {
      arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto", "-DCUB_DISABLE_CDP"};

    cccl::detail::extend_args_with_build_config(args, config);

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    std::string lowered_name;

    // Collect all LTO-IRs to be linked
    nvrtc_linkable_list linkable_list;
    nvrtc_linkable_list_appender appender{linkable_list};

    // Add operation if it's LTO-IR (C++ source not yet supported in for)
    appender.append_operation(op);

    // Add iterator definitions if present
    if (cccl_iterator_kind_t::CCCL_ITERATOR == d_data.type)
    {
      appender.append_operation(d_data.advance);
      appender.append_operation(d_data.dereference);
    }

    nvrtc_link_result result =
      begin_linking_nvrtc_program(num_lto_args, lopts)
        ->add_program(nvrtc_translation_unit{device_for_kernel, name})
        ->add_expression({for_kernel_name})
        ->compile_program({args.data(), args.size()})
        ->get_name({for_kernel_name, lowered_name})
        ->link_program()
        ->add_link_list(linkable_list)
        ->finalize_program();

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build_ptr->static_kernel, build_ptr->library, lowered_name.c_str()));

    build_ptr->cc         = cc;
    build_ptr->cubin      = (void*) result.data.release();
    build_ptr->cubin_size = result.size;
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }
  return error;
}

hipError_t cccl_device_for(
  cccl_device_for_build_result_t build, cccl_iterator_t d_data, uint64_t num_items, cccl_op_t op, hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;

  try
  {
    pushed           = try_push_context();
    auto exec_status = Invoke(d_data, num_items, op, build.cc, (hipFunction_t) build.static_kernel, stream);
    error            = static_cast<hipError_t>(exec_status);
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_for_build(
  cccl_device_for_build_result_t* build,
  cccl_iterator_t d_data,
  cccl_op_t op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  return cccl_device_for_build_ex(
    build, d_data, op, cc_major, cc_minor, cub_path, thrust_path, libcudacxx_path, ctk_path, nullptr);
}

hipError_t cccl_device_for_cleanup(cccl_device_for_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (...)
  {
    return hipErrorUnknown;
  }

  return hipSuccess;
}
