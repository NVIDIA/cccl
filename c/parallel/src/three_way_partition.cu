//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh> // hipcub::detail::choose_offset_t
#include <cub/detail/launcher/cuda_driver.cuh> // hipcub::detail::CudaDriverLauncherFactory
#include <cub/device/dispatch/dispatch_three_way_partition.cuh> // hipcub::DispatchThreeWayPartitionIf
#include <cub/device/dispatch/kernels/three_way_partition.cuh> // DeviceThreeWayPartition kernels
#include <cub/device/dispatch/tuning/tuning_three_way_partition.cuh> // policy_hub

#include <exception>
#include <format>
#include <string>
#include <string_view>
#include <type_traits> // std::is_same_v
#include <vector>

#include "jit_templates/templates/input_iterator.h"
#include "jit_templates/templates/operation.h"
#include "jit_templates/templates/output_iterator.h"
#include "jit_templates/traits.h"
#include "util/context.h"
#include "util/errors.h"
#include "util/indirect_arg.h"
#include "util/runtime_policy.h"
#include "util/types.h"
#include <cccl/c/three_way_partition.h>
#include <cccl/c/types.h>
#include <nlohmann/json.hpp>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>
#include <util/build_utils.h>

struct device_three_way_partition_policy;
using OffsetT = ptrdiff_t;
static_assert(std::is_same_v<hipcub::detail::choose_signed_offset<OffsetT>::type, OffsetT>, "OffsetT must be long");

// check we can map OffsetT to cuda::std::int64_t
static_assert(std::is_signed_v<OffsetT>);
static_assert(sizeof(OffsetT) == sizeof(cuda::std::int64_t));

namespace three_way_partition
{

struct three_way_partition_runtime_tuning_policy
{
  hipcub::detail::RuntimeThreeWayPartitionAgentPolicy three_way_partition;

  auto ThreeWayPartition() const
  {
    return three_way_partition;
  }

  using MaxPolicy = three_way_partition_runtime_tuning_policy;

  template <typename F>
  hipError_t Invoke(int, F& op)
  {
    return op.template Invoke<three_way_partition_runtime_tuning_policy>(*this);
  }
};

struct three_way_partition_kernel_source
{
  cccl_device_three_way_partition_build_result_t& build;

  hipKernel_t ThreeWayPartitionInitKernel() const
  {
    return build.three_way_partition_init_kernel;
  }

  hipKernel_t ThreeWayPartitionKernel() const
  {
    return build.three_way_partition_kernel;
  }
};

std::string get_three_way_partition_init_kernel_name(std::string_view num_selected_out_iterator_name)
{
  constexpr std::string_view scan_tile_state_t = "hipcub::detail::three_way_partition::ScanTileStateT";
  return std::format("hipcub::detail::three_way_partition::DeviceThreeWayPartitionInitKernel<{0}, {1}>",
                     scan_tile_state_t, // 0
                     num_selected_out_iterator_name); // 1
}

std::string get_three_way_partition_kernel_name(
  std::string_view d_in_iterator_name,
  std::string_view d_first_part_out_iterator_name,
  std::string_view d_second_part_out_iterator_name,
  std::string_view d_unselected_out_iterator_name,
  std::string_view d_num_selected_out_iterator_name,
  std::string_view select_first_part_op_name,
  std::string_view select_second_part_op_name)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_three_way_partition_policy>(&chained_policy_t));

  constexpr std::string_view scan_tile_state_t = "hipcub::detail::three_way_partition::ScanTileStateT";

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  const std::string streaming_context_t =
    std::format("hipcub::detail::three_way_partition::streaming_context_t<{0}>", offset_t);

  return std::format(
    "hipcub::detail::three_way_partition::DeviceThreeWayPartitionKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}, {7}, {8}, {9}, "
    "{10}>",
    chained_policy_t, // 0
    d_in_iterator_name, // 1
    d_first_part_out_iterator_name, // 2
    d_second_part_out_iterator_name, // 3
    d_unselected_out_iterator_name, // 4
    d_num_selected_out_iterator_name, // 5
    scan_tile_state_t, // 6
    select_first_part_op_name, // 7
    select_second_part_op_name, // 8
    "hipcub::detail::three_way_partition::per_partition_offset_t", // 9
    streaming_context_t // 10
  );
}

std::string get_three_way_partition_policy_delay_constructor(const nlohmann::json& partition_policy)
{
  auto delay_ctor_info = partition_policy["DelayConstructor"];

  std::string delay_ctor_params;
  for (auto&& param : delay_ctor_info["params"])
  {
    delay_ctor_params.append(to_string(param) + ", ");
  }
  delay_ctor_params.erase(delay_ctor_params.size() - 2); // remove last ", "

  return std::format("hipcub::detail::{}<{}>", delay_ctor_info["name"].get<std::string>(), delay_ctor_params);
}

std::string inject_delay_constructor_into_three_way_policy(
  const std::string& three_way_partition_policy_str, const std::string& delay_constructor_type)
{
  // Insert before the final closing of the struct (right before the sequence "};")
  const std::string needle = "};";
  const auto pos           = three_way_partition_policy_str.rfind(needle);
  if (pos == std::string::npos)
  {
    return three_way_partition_policy_str; // unexpected; return as-is
  }
  const std::string insertion =
    std::format("\n  struct detail {{ using delay_constructor_t = {}; }}; \n", delay_constructor_type);
  std::string out = three_way_partition_policy_str;
  out.insert(pos, insertion);
  return out;
}
} // namespace three_way_partition

struct three_way_partition_input_iterator_tag;
struct three_way_partition_first_part_output_iterator_tag;
struct three_way_partition_second_part_output_iterator_tag;
struct three_way_partition_unselected_output_iterator_tag;
struct three_way_partition_num_selected_output_iterator_tag;
struct three_way_partition_select_first_part_operation_tag;
struct three_way_partition_select_second_part_operation_tag;

hipError_t cccl_device_three_way_partition_build_ex(
  cccl_device_three_way_partition_build_result_t* build_ptr,
  cccl_iterator_t d_in,
  cccl_iterator_t d_first_part_out,
  cccl_iterator_t d_second_part_out,
  cccl_iterator_t d_unselected_out,
  cccl_iterator_t d_num_selected_out,
  cccl_op_t select_first_part_op,
  cccl_op_t select_second_part_op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path,
  cccl_build_config* config)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "device_three_way_partition";

    const int cc = cc_major * 10 + cc_minor;

    const auto [d_in_iterator_name, d_in_iterator_src] =
      get_specialization<three_way_partition_input_iterator_tag>(template_id<input_iterator_traits>(), d_in);
    const auto [d_first_part_out_iterator_name, d_first_part_out_iterator_src] =
      get_specialization<three_way_partition_first_part_output_iterator_tag>(
        template_id<output_iterator_traits>(), d_first_part_out, d_first_part_out.value_type);
    const auto [d_second_part_out_iterator_name, d_second_part_out_iterator_src] =
      get_specialization<three_way_partition_second_part_output_iterator_tag>(
        template_id<output_iterator_traits>(), d_second_part_out, d_second_part_out.value_type);
    const auto [d_unselected_out_iterator_name, d_unselected_out_iterator_src] =
      get_specialization<three_way_partition_unselected_output_iterator_tag>(
        template_id<output_iterator_traits>(), d_unselected_out, d_unselected_out.value_type);
    const auto [d_num_selected_out_iterator_name, d_num_selected_out_iterator_src] =
      get_specialization<three_way_partition_num_selected_output_iterator_tag>(
        template_id<output_iterator_traits>(), d_num_selected_out, d_num_selected_out.value_type);

    cccl_type_info selector_result_t{sizeof(bool), alignof(bool), cccl_type_enum::CCCL_BOOLEAN};

    const auto [select_first_part_op_name, select_first_part_op_src] =
      get_specialization<three_way_partition_select_first_part_operation_tag>(
        template_id<user_operation_traits>(), select_first_part_op, selector_result_t, d_in.value_type);
    const auto [select_second_part_op_name, select_second_part_op_src] =
      get_specialization<three_way_partition_select_second_part_operation_tag>(
        template_id<user_operation_traits>(), select_second_part_op, selector_result_t, d_in.value_type);

    const auto offset_t = cccl_type_enum_to_name(cccl_type_enum::CCCL_INT64);

    const std::string dependent_definitions_src = std::format(
      R"XXX(
struct __align__({1}) storage_t {{
  char data[{0}];
}};
{2}
{3}
{4}
{5}
{6}
{7}
{8}
)XXX",
      d_in.value_type.size, // 0
      d_in.value_type.alignment, // 1
      d_in_iterator_src, // 2
      d_first_part_out_iterator_src, // 3
      d_second_part_out_iterator_src, // 4
      d_unselected_out_iterator_src, // 5
      d_num_selected_out_iterator_src, // 6
      select_first_part_op_src, // 7
      select_second_part_op_src); // 8

    const std::string ptx_arch = std::format("-arch=compute_{}{}", cc_major, cc_minor);

    constexpr size_t ptx_num_args      = 6;
    const char* ptx_args[ptx_num_args] = {
      ptx_arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true"};

    static constexpr std::string_view policy_wrapper_expr_tmpl =
      R"XXXX(hipcub::detail::three_way_partition::MakeThreeWayPartitionPolicyWrapper(hipcub::detail::three_way_partition::policy_hub<{0}, {1}>::MaxPolicy::ActivePolicy{{}}))XXXX";

    const std::string key_t = cccl_type_enum_to_name(d_in.value_type.type);

    const auto policy_wrapper_expr = std::format(
      policy_wrapper_expr_tmpl,
      key_t, // 0
      offset_t); // 1

    static constexpr std::string_view ptx_query_tu_src_tmpl = R"XXXX(
#include <cub/device/dispatch/kernels/three_way_partition.cuh>
#include <cub/device/dispatch/tuning/tuning_three_way_partition.cuh>
{0}
{1}
)XXXX";

    const auto ptx_query_tu_src =
      std::format(ptx_query_tu_src_tmpl, jit_template_header_contents, dependent_definitions_src);

    nlohmann::json runtime_policy = get_policy(policy_wrapper_expr, ptx_query_tu_src, ptx_args);

    using hipcub::detail::RuntimeThreeWayPartitionAgentPolicy;
    auto [three_way_partition_policy, three_way_partition_policy_str] =
      RuntimeThreeWayPartitionAgentPolicy::from_json(runtime_policy, "ThreeWayPartitionPolicy");

    const std::string three_way_partition_policy_delay_constructor =
      three_way_partition::get_three_way_partition_policy_delay_constructor(runtime_policy);

    const std::string injected_three_way_partition_policy_str =
      three_way_partition::inject_delay_constructor_into_three_way_policy(
        three_way_partition_policy_str, three_way_partition_policy_delay_constructor);
    constexpr std::string_view program_preamble_template = R"XXX(
#include <cub/device/dispatch/kernels/three_way_partition.cuh>
{0}
{1}
struct device_three_way_partition_policy {{
  struct ActivePolicy {{
    {2}
  }};
}};
)XXX";

    std::string final_src = std::format(
      program_preamble_template,
      jit_template_header_contents, // 0
      dependent_definitions_src, // 1
      injected_three_way_partition_policy_str); // 2

    std::string three_way_partition_init_kernel_name =
      three_way_partition::get_three_way_partition_init_kernel_name(d_num_selected_out_iterator_name);
    std::string three_way_partition_kernel_name = three_way_partition::get_three_way_partition_kernel_name(
      d_in_iterator_name,
      d_first_part_out_iterator_name,
      d_second_part_out_iterator_name,
      d_unselected_out_iterator_name,
      d_num_selected_out_iterator_name,
      select_first_part_op_name,
      select_second_part_op_name);
    std::string three_way_partition_init_kernel_lowered_name;
    std::string three_way_partition_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    std::vector<const char*> args = {
      arch.c_str(),
      cub_path,
      thrust_path,
      libcudacxx_path,
      ctk_path,
      "-rdc=true",
      "-dlto",
      "-DCUB_DISABLE_CDP",
      "-std=c++20"};

    cccl::detail::extend_args_with_build_config(args, config);

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_linkable_list linkable_list;
    nvrtc_linkable_list_appender appender{linkable_list};

    appender.append_operation(select_first_part_op);
    appender.append_operation(select_second_part_op);
    appender.add_iterator_definition(d_in);
    appender.add_iterator_definition(d_first_part_out);
    appender.add_iterator_definition(d_second_part_out);
    appender.add_iterator_definition(d_unselected_out);
    appender.add_iterator_definition(d_num_selected_out);

    nvrtc_link_result result =
      begin_linking_nvrtc_program(num_lto_args, lopts)
        ->add_program(nvrtc_translation_unit{final_src.c_str(), name})
        ->add_expression({three_way_partition_init_kernel_name})
        ->add_expression({three_way_partition_kernel_name})
        ->compile_program({args.data(), args.size()})
        ->get_name({three_way_partition_init_kernel_name, three_way_partition_init_kernel_lowered_name})
        ->get_name({three_way_partition_kernel_name, three_way_partition_kernel_lowered_name})
        ->link_program()
        ->add_link_list(linkable_list)
        ->finalize_program();

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build_ptr->three_way_partition_init_kernel,
                             build_ptr->library,
                             three_way_partition_init_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(
      &build_ptr->three_way_partition_kernel, build_ptr->library, three_way_partition_kernel_lowered_name.c_str()));

    build_ptr->cc         = cc;
    build_ptr->cubin      = (void*) result.data.release();
    build_ptr->cubin_size = result.size;
    build_ptr->runtime_policy =
      new three_way_partition::three_way_partition_runtime_tuning_policy{three_way_partition_policy};
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_three_way_partition_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

hipError_t cccl_device_three_way_partition(
  cccl_device_three_way_partition_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_first_part_out,
  cccl_iterator_t d_second_part_out,
  cccl_iterator_t d_unselected_out,
  cccl_iterator_t d_num_selected_out,
  cccl_op_t select_first_part_op,
  cccl_op_t select_second_part_op,
  uint64_t num_items,
  hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    auto exec_status = hipcub::DispatchThreeWayPartitionIf<
      indirect_arg_t, // InputIteratorT
      indirect_arg_t, // FirstOutputIteratorT
      indirect_arg_t, // SecondOutputIteratorT
      indirect_arg_t, // UnselectedOutputIteratorT
      indirect_arg_t, // NumSelectedIteratorT
      indirect_arg_t, // SelectFirstPartOp
      indirect_arg_t, // SelectSecondPartOp
      OffsetT, // OffsetT
      three_way_partition::three_way_partition_runtime_tuning_policy, // PolicyHub
      three_way_partition::three_way_partition_kernel_source, // KernelSource
      hipcub::detail::CudaDriverLauncherFactory>::
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_in,
        d_first_part_out,
        d_second_part_out,
        d_unselected_out,
        d_num_selected_out,
        select_first_part_op,
        select_second_part_op,
        num_items,
        stream,
        /* kernel_source */ {build},
        /* launcher_factory */ hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        /* policy */
        *reinterpret_cast<three_way_partition::three_way_partition_runtime_tuning_policy*>(build.runtime_policy));

    error = static_cast<hipError_t>(exec_status);
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_three_way_partition(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_three_way_partition_cleanup(cccl_device_three_way_partition_build_result_t* bld_ptr)
{
  try
  {
    if (bld_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }
    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(bld_ptr->cubin));
    std::unique_ptr<char[]> policy(reinterpret_cast<char*>(bld_ptr->runtime_policy));
    check(hipLibraryUnload(bld_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_three_way_partition_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}

hipError_t cccl_device_three_way_partition_build(
  cccl_device_three_way_partition_build_result_t* build_ptr,
  cccl_iterator_t d_in,
  cccl_iterator_t d_first_part_out,
  cccl_iterator_t d_second_part_out,
  cccl_iterator_t d_unselected_out,
  cccl_iterator_t d_num_selected_out,
  cccl_op_t select_first_part_op,
  cccl_op_t select_second_part_op,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  return cccl_device_three_way_partition_build_ex(
    build_ptr,
    d_in,
    d_first_part_out,
    d_second_part_out,
    d_unselected_out,
    d_num_selected_out,
    select_first_part_op,
    select_second_part_op,
    cc_major,
    cc_minor,
    cub_path,
    thrust_path,
    libcudacxx_path,
    ctk_path,
    nullptr);
}
