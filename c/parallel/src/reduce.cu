//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/detail/launcher/cuda_driver.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/grid/grid_even_share.cuh>
#include <cub/util_device.cuh>

#include <cuda/std/__algorithm_>
#include <cuda/std/cstdint>
#include <cuda/std/functional> // ::cuda::std::identity
#include <cuda/std/variant>

#include <format>
#include <memory>

#include "jit_templates/templates/input_iterator.h"
#include "jit_templates/templates/operation.h"
#include "jit_templates/templates/output_iterator.h"
#include "jit_templates/traits.h"
#include "util/context.h"
#include "util/errors.h"
#include "util/indirect_arg.h"
#include "util/runtime_policy.h"
#include "util/types.h"
#include <cccl/c/reduce.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>

struct device_reduce_policy;
using TransformOpT = ::cuda::std::identity;
using OffsetT      = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

namespace reduce
{

struct reduce_runtime_tuning_policy
{
  hipcub::detail::RuntimeReduceAgentPolicy single_tile;
  hipcub::detail::RuntimeReduceAgentPolicy reduce;

  auto SingleTile() const
  {
    return single_tile;
  }
  auto Reduce() const
  {
    return reduce;
  }

  using MaxPolicy = reduce_runtime_tuning_policy;

  template <typename F>
  hipError_t Invoke(int, F& op)
  {
    return op.template Invoke<reduce_runtime_tuning_policy>(*this);
  }
};

static cccl_type_info get_accumulator_type(cccl_op_t /*op*/, cccl_iterator_t /*input_it*/, cccl_value_t init)
{
  // TODO Should be decltype(op(init, *input_it)) but haven't implemented type arithmetic yet
  //      so switching back to the old accumulator type logic for now
  return init.type;
}

std::string get_single_tile_kernel_name(
  std::string_view input_iterator_t,
  std::string_view output_iterator_t,
  std::string_view reduction_op_t,
  cccl_value_t init,
  std::string_view accum_cpp_t,
  bool is_second_kernel)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  const std::string init_t = cccl_type_enum_to_name(init.type.type);

  std::string offset_t;
  if (is_second_kernel)
  {
    // Second kernel is always invoked with an int offset.
    // See the definition of the local variable `reduce_grid_size`
    // in DispatchReduce::InvokePasses.
    check(nvrtcGetTypeName<int>(&offset_t));
  }
  else
  {
    check(nvrtcGetTypeName<OffsetT>(&offset_t));
  }

  return std::format(
    "hipcub::detail::reduce::DeviceReduceSingleTileKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}>",
    chained_policy_t,
    input_iterator_t,
    output_iterator_t,
    offset_t,
    reduction_op_t,
    init_t,
    accum_cpp_t);
}

std::string get_device_reduce_kernel_name(
  std::string_view reduction_op_t, std::string_view input_iterator_t, std::string_view accum_t)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  std::string transform_op_t;
  check(nvrtcGetTypeName<cuda::std::__identity>(&transform_op_t));

  return std::format(
    "hipcub::detail::reduce::DeviceReduceKernel<{0}, {1}, {2}, {3}, {4}, {5}>",
    chained_policy_t,
    input_iterator_t,
    offset_t,
    reduction_op_t,
    accum_t,
    transform_op_t);
}

struct reduce_kernel_source
{
  cccl_device_reduce_build_result_t& build;

  std::size_t AccumSize() const
  {
    return build.accumulator_size;
  }
  hipKernel_t SingleTileKernel() const
  {
    return build.single_tile_kernel;
  }
  hipKernel_t SingleTileSecondKernel() const
  {
    return build.single_tile_second_kernel;
  }
  hipKernel_t ReductionKernel() const
  {
    return build.reduction_kernel;
  }
};
} // namespace reduce

struct reduce_iterator_tag;
struct reduction_operation_tag;

hipError_t cccl_device_reduce_build(
  cccl_device_reduce_build_result_t* build,
  cccl_iterator_t input_it,
  cccl_iterator_t output_it,
  cccl_op_t op,
  cccl_value_t init,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "device_reduce";

    const int cc                 = cc_major * 10 + cc_minor;
    const cccl_type_info accum_t = reduce::get_accumulator_type(op, input_it, init);
    const auto accum_cpp         = cccl_type_enum_to_name(accum_t.type);

    const auto [input_iterator_name, input_iterator_src] =
      get_specialization<reduce_iterator_tag>(template_id<input_iterator_traits>(), input_it);
    const auto [output_iterator_name, output_iterator_src] =
      get_specialization<reduce_iterator_tag>(template_id<output_iterator_traits>(), output_it, accum_t);

    const auto [op_name, op_src] =
      get_specialization<reduction_operation_tag>(template_id<binary_user_operation_traits>(), op, accum_t);

    const std::string ptx_arch = std::format("-arch=compute_{}{}", cc_major, cc_minor);

    constexpr size_t ptx_num_args      = 5;
    const char* ptx_args[ptx_num_args] = {ptx_arch.c_str(), cub_path, thrust_path, libcudacxx_path, "-rdc=true"};

    const std::string src = std::format(
      R"XXX(
#include <cub/block/block_reduce.cuh>
{5}
struct __align__({1}) storage_t {{
  char data[{0}];
}};
{2}
{3}
{4}
)XXX",
      input_it.value_type.size, // 0
      input_it.value_type.alignment, // 1
      input_iterator_src, // 2
      output_iterator_src, // 3
      op_src, // 4
      jit_template_header_contents); // 5

    const auto offset_t           = cccl_type_enum_to_name(cccl_type_enum::CCCL_UINT64);
    nlohmann::json runtime_policy = get_policy(
      std::format("hipcub::detail::reduce::MakeReducePolicyWrapper(hipcub::detail::reduce::policy_hub<{}, {}, "
                  "{}>::MaxPolicy::ActivePolicy{{}})",
                  accum_cpp,
                  offset_t,
                  op_name),
      "#include <cub/device/dispatch/tuning/tuning_reduce.cuh>\n" + src,
      ptx_args);

    using hipcub::detail::RuntimeReduceAgentPolicy;
    auto [reduce_policy, reduce_policy_str] = RuntimeReduceAgentPolicy::from_json(runtime_policy, "ReducePolicy");
    auto [st_policy, st_policy_str]         = RuntimeReduceAgentPolicy::from_json(runtime_policy, "SingleTilePolicy");

    std::string final_src = std::format(
      "#include <cub/device/dispatch/kernels/reduce.cuh>\n"
      "{0}\n"
      "struct device_reduce_policy {{\n"
      "  struct ActivePolicy {{\n"
      "    {1}\n"
      "    {2}\n"
      "  }};\n"
      "}};",
      src,
      reduce_policy_str,
      st_policy_str);

#if false // CCCL_DEBUGGING_SWITCH
    fflush(stderr);
    printf("\nCODE4NVRTC BEGIN\n%sCODE4NVRTC END\n", final_src.c_str());
    fflush(stdout);
#endif

    std::string single_tile_kernel_name =
      reduce::get_single_tile_kernel_name(input_iterator_name, output_iterator_name, op_name, init, accum_cpp, false);
    std::string single_tile_second_kernel_name = reduce::get_single_tile_kernel_name(
      cccl_type_enum_to_name(accum_t.type, true), output_iterator_name, op_name, init, accum_cpp, true);
    std::string reduction_kernel_name = reduce::get_device_reduce_kernel_name(op_name, input_iterator_name, accum_cpp);
    std::string single_tile_kernel_lowered_name;
    std::string single_tile_second_kernel_lowered_name;
    std::string reduction_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    constexpr size_t num_args  = 9;
    const char* args[num_args] = {
      arch.c_str(),
      cub_path,
      thrust_path,
      libcudacxx_path,
      ctk_path,
      "-rdc=true",
      "-dlto",
      "-DCUB_DISABLE_CDP",
      "-std=c++20"};

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_ltoir_list ltoir_list;
    nvrtc_ltoir_list_appender appender{ltoir_list};

    appender.append({op.ltoir, op.ltoir_size});
    appender.add_iterator_definition(input_it);
    appender.add_iterator_definition(output_it);

    nvrtc_link_result result =
      make_nvrtc_command_list()
        .add_program(nvrtc_translation_unit{final_src.c_str(), name})
        .add_expression({single_tile_kernel_name})
        .add_expression({single_tile_second_kernel_name})
        .add_expression({reduction_kernel_name})
        .compile_program({args, num_args})
        .get_name({single_tile_kernel_name, single_tile_kernel_lowered_name})
        .get_name({single_tile_second_kernel_name, single_tile_second_kernel_lowered_name})
        .get_name({reduction_kernel_name, reduction_kernel_lowered_name})
        .cleanup_program()
        .add_link_list(ltoir_list)
        .finalize_program(num_lto_args, lopts);

    hipLibraryLoadData(&build->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build->single_tile_kernel, build->library, single_tile_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(
      &build->single_tile_second_kernel, build->library, single_tile_second_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(&build->reduction_kernel, build->library, reduction_kernel_lowered_name.c_str()));

    build->cc               = cc;
    build->cubin            = (void*) result.data.release();
    build->cubin_size       = result.size;
    build->accumulator_size = accum_t.size;
    build->runtime_policy   = new reduce::reduce_runtime_tuning_policy{st_policy, reduce_policy};
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

hipError_t cccl_device_reduce(
  cccl_device_reduce_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    auto exec_status = hipcub::DispatchReduce<
      indirect_arg_t, // InputIteratorT
      indirect_arg_t, // OutputIteratorT
      ::cuda::std::size_t, // OffsetT
      indirect_arg_t, // ReductionOpT
      indirect_arg_t, // InitT
      void, // AccumT
      ::cuda::std::__identity, // TransformOpT
      reduce::reduce_runtime_tuning_policy, // PolicyHub
      reduce::reduce_kernel_source, // KernelSource
      hipcub::detail::CudaDriverLauncherFactory>:: // KernelLauncherFactory
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_in,
        d_out,
        num_items,
        op,
        init,
        stream,
        {},
        {build},
        hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        *reinterpret_cast<reduce::reduce_runtime_tuning_policy*>(build.runtime_policy));

    error = static_cast<hipError_t>(exec_status);
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_reduce_cleanup(cccl_device_reduce_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    std::unique_ptr<char[]> policy(reinterpret_cast<char*>(build_ptr->runtime_policy));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}
