#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/detail/launcher/cuda_driver.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/grid/grid_even_share.cuh>
#include <cub/util_device.cuh>

#include <cuda/std/__algorithm_>
#include <cuda/std/cstdint>
#include <cuda/std/functional> // ::cuda::std::identity
#include <cuda/std/variant>

#include <format>
#include <memory>

#include "kernels/iterators.h"
#include "kernels/operators.h"
#include "util/context.h"
#include "util/errors.h"
#include "util/indirect_arg.h"
#include "util/types.h"
#include <cccl/c/reduce.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>

struct op_wrapper;
struct device_reduce_policy;
using TransformOpT = ::cuda::std::identity;
using OffsetT      = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

struct input_iterator_t;
struct output_iterator_t;

namespace reduce
{

struct reduce_runtime_tuning_policy
{
  int block_size;
  int items_per_thread;
  int vector_load_length;

  reduce_runtime_tuning_policy SingleTile() const
  {
    return *this;
  }
  reduce_runtime_tuning_policy Reduce() const
  {
    return *this;
  }

  int ItemsPerThread() const
  {
    return items_per_thread;
  }
  int BlockThreads() const
  {
    return block_size;
  }
};

struct reduce_tuning_t
{
  int cc;
  int block_size;
  int items_per_thread;
  int vector_load_length;
};

template <typename Tuning, int N>
Tuning find_tuning(int cc, const Tuning (&tunings)[N])
{
  for (const Tuning& tuning : tunings)
  {
    if (cc >= tuning.cc)
    {
      return tuning;
    }
  }

  return tunings[N - 1];
}

reduce_runtime_tuning_policy get_policy(int cc, cccl_type_info accumulator_type)
{
  constexpr reduce_tuning_t chain[] = {{60, 256, 16, 4}, {35, 256, 20, 4}};

  auto [_, block_size, items_per_thread, vector_load_length] = find_tuning(cc, chain);

  // Implement part of MemBoundScaling
  int candidate_ipt  = static_cast<int>(items_per_thread * 4 / accumulator_type.size);
  items_per_thread   = cuda::std::clamp(candidate_ipt, 1, items_per_thread * 2);
  int max_block_size = static_cast<int>((((1024 * 48) / (accumulator_type.size * items_per_thread)) + 31) / 32 * 32);
  block_size         = _CUDA_VSTD::min(block_size, max_block_size);

  return {block_size, items_per_thread, vector_load_length};
}

static cccl_type_info get_accumulator_type(cccl_op_t /*op*/, cccl_iterator_t /*input_it*/, cccl_value_t init)
{
  // TODO Should be decltype(op(init, *input_it)) but haven't implemented type arithmetic yet
  //      so switching back to the old accumulator type logic for now
  return init.type;
}

template <typename Type>
std::string get_iterator_name()
{
  std::string iterator_t{};
  check(nvrtcGetTypeName<Type>(&iterator_t));
  return iterator_t;
}

std::string get_input_iterator_name()
{
  return get_iterator_name<input_iterator_t>();
}

std::string get_output_iterator_name()
{
  return get_iterator_name<output_iterator_t>();
}

std::string get_single_tile_kernel_name(
  cccl_iterator_t input_it, cccl_iterator_t output_it, cccl_op_t op, cccl_value_t init, bool is_second_kernel)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  const cccl_type_info accum_t  = get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  const std::string input_iterator_t =
    is_second_kernel ? cccl_type_enum_to_name(accum_t.type, true)
    : input_it.type == cccl_iterator_kind_t::CCCL_POINTER //
      ? cccl_type_enum_to_name(input_it.value_type.type, true) //
      : get_input_iterator_name();
  const std::string output_iterator_t =
    output_it.type == cccl_iterator_kind_t::CCCL_POINTER //
      ? cccl_type_enum_to_name(output_it.value_type.type, true) //
      : get_output_iterator_name();
  const std::string init_t = cccl_type_enum_to_name(init.type.type);

  std::string offset_t;
  if (is_second_kernel)
  {
    // Second kernel is always invoked with an int offset.
    // See the definition of the local variable `reduce_grid_size`
    // in DispatchReduce::InvokePasses.
    check(nvrtcGetTypeName<int>(&offset_t));
  }
  else
  {
    check(nvrtcGetTypeName<OffsetT>(&offset_t));
  }

  std::string reduction_op_t;
  check(nvrtcGetTypeName<op_wrapper>(&reduction_op_t));

  return std::format(
    "hipcub::detail::reduce::DeviceReduceSingleTileKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}>",
    chained_policy_t,
    input_iterator_t,
    output_iterator_t,
    offset_t,
    reduction_op_t,
    init_t,
    accum_cpp_t);
}

std::string get_device_reduce_kernel_name(cccl_op_t op, cccl_iterator_t input_it, cccl_value_t init)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  const std::string input_iterator_t =
    input_it.type == cccl_iterator_kind_t::CCCL_POINTER //
      ? cccl_type_enum_to_name(input_it.value_type.type, true) //
      : get_input_iterator_name();

  const std::string accum_t = cccl_type_enum_to_name(get_accumulator_type(op, input_it, init).type);

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  std::string reduction_op_t;
  check(nvrtcGetTypeName<op_wrapper>(&reduction_op_t));

  std::string transform_op_t;
  check(nvrtcGetTypeName<cuda::std::__identity>(&transform_op_t));

  return std::format(
    "hipcub::detail::reduce::DeviceReduceKernel<{0}, {1}, {2}, {3}, {4}, {5}>",
    chained_policy_t,
    input_iterator_t,
    offset_t,
    reduction_op_t,
    accum_t,
    transform_op_t);
}

template <auto* GetPolicy>
struct dynamic_reduce_policy_t
{
  using MaxPolicy = dynamic_reduce_policy_t;

  template <typename F>
  hipError_t Invoke(int device_ptx_version, F& op)
  {
    return op.template Invoke<reduce_runtime_tuning_policy>(GetPolicy(device_ptx_version, accumulator_type));
  }

  cccl_type_info accumulator_type;
};

struct reduce_kernel_source
{
  cccl_device_reduce_build_result_t& build;

  std::size_t AccumSize() const
  {
    return build.accumulator_size;
  }
  hipKernel_t SingleTileKernel() const
  {
    return build.single_tile_kernel;
  }
  hipKernel_t SingleTileSecondKernel() const
  {
    return build.single_tile_second_kernel;
  }
  hipKernel_t ReductionKernel() const
  {
    return build.reduction_kernel;
  }
};
} // namespace reduce

hipError_t cccl_device_reduce_build(
  cccl_device_reduce_build_result_t* build_ptr,
  cccl_iterator_t input_it,
  cccl_iterator_t output_it,
  cccl_op_t op,
  cccl_value_t init,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "test";

    const int cc                 = cc_major * 10 + cc_minor;
    const cccl_type_info accum_t = reduce::get_accumulator_type(op, input_it, init);
    const auto policy            = reduce::get_policy(cc, accum_t);
    const auto accum_cpp         = cccl_type_enum_to_name(accum_t.type);
    const auto input_it_value_t  = cccl_type_enum_to_name(input_it.value_type.type);
    const auto offset_t          = cccl_type_enum_to_name(cccl_type_enum::CCCL_UINT64);

    const auto input_iterator_typename  = reduce::get_input_iterator_name();
    const auto output_iterator_typename = reduce::get_output_iterator_name();

    const std::string input_iterator_src =
      make_kernel_input_iterator(offset_t, input_iterator_typename, input_it_value_t, input_it);
    const std::string output_iterator_src =
      make_kernel_output_iterator(offset_t, output_iterator_typename, accum_cpp, output_it);

    const std::string op_src = make_kernel_user_binary_operator(accum_cpp, op);

    const std::string src = std::format(
      R"XXX(
#include <cub/block/block_reduce.cuh>
#include <cub/device/dispatch/kernels/reduce.cuh>
struct __align__({1}) storage_t {{
  char data[{0}];
}};
{4}
{5}
struct agent_policy_t {{
  static constexpr int ITEMS_PER_THREAD = {2};
  static constexpr int BLOCK_THREADS = {3};
  static constexpr int VECTOR_LOAD_LENGTH = {7};
  static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BLOCK_REDUCE_WARP_REDUCTIONS;
  static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::LOAD_LDG;
}};
struct device_reduce_policy {{
  struct ActivePolicy {{
    using ReducePolicy = agent_policy_t;
    using SingleTilePolicy = agent_policy_t;
  }};
}};
{6}
)XXX",
      input_it.value_type.size, // 0
      input_it.value_type.alignment, // 1
      policy.items_per_thread, // 2
      policy.block_size, // 3
      input_iterator_src, // 4
      output_iterator_src, // 5
      op_src, // 6
      policy.vector_load_length); // 7

#if false // CCCL_DEBUGGING_SWITCH
    fflush(stderr);
    printf("\nCODE4NVRTC BEGIN\n%sCODE4NVRTC END\n", src.c_str());
    fflush(stdout);
#endif

    std::string single_tile_kernel_name = reduce::get_single_tile_kernel_name(input_it, output_it, op, init, false);
    std::string single_tile_second_kernel_name =
      reduce::get_single_tile_kernel_name(input_it, output_it, op, init, true);
    std::string reduction_kernel_name = reduce::get_device_reduce_kernel_name(op, input_it, init);
    std::string single_tile_kernel_lowered_name;
    std::string single_tile_second_kernel_lowered_name;
    std::string reduction_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    constexpr size_t num_args  = 7;
    const char* args[num_args] = {arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto"};

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_ltoir_list ltoir_list;
    nvrtc_ltoir_list_appender appender{ltoir_list};

    appender.append({op.ltoir, op.ltoir_size});
    appender.add_iterator_definition(input_it);
    appender.add_iterator_definition(output_it);

    nvrtc_link_result result =
      make_nvrtc_command_list()
        .add_program(nvrtc_translation_unit{src.c_str(), name})
        .add_expression({single_tile_kernel_name})
        .add_expression({single_tile_second_kernel_name})
        .add_expression({reduction_kernel_name})
        .compile_program({args, num_args})
        .get_name({single_tile_kernel_name, single_tile_kernel_lowered_name})
        .get_name({single_tile_second_kernel_name, single_tile_second_kernel_lowered_name})
        .get_name({reduction_kernel_name, reduction_kernel_lowered_name})
        .cleanup_program()
        .add_link_list(ltoir_list)
        .finalize_program(num_lto_args, lopts);

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(
      hipLibraryGetKernel(&build_ptr->single_tile_kernel, build_ptr->library, single_tile_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(
      &build_ptr->single_tile_second_kernel, build_ptr->library, single_tile_second_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(&build_ptr->reduction_kernel, build_ptr->library, reduction_kernel_lowered_name.c_str()));

    build_ptr->cc               = cc;
    build_ptr->cubin            = (void*) result.data.release();
    build_ptr->cubin_size       = result.size;
    build_ptr->accumulator_size = accum_t.size;
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

hipError_t cccl_device_reduce(
  cccl_device_reduce_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  unsigned long long num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    hipcub::DispatchReduce<indirect_arg_t, // InputIteratorT
                        indirect_arg_t, // OutputIteratorT
                        ::cuda::std::size_t, // OffsetT
                        indirect_arg_t, // ReductionOpT
                        indirect_arg_t, // InitT
                        void, // AccumT
                        ::cuda::std::__identity, // TransformOpT
                        reduce::dynamic_reduce_policy_t<&reduce::get_policy>, // PolicyHub
                        reduce::reduce_kernel_source, // KernelSource
                        hipcub::detail::CudaDriverLauncherFactory>:: // KernelLauncherFactory
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_in,
        d_out,
        num_items,
        op,
        init,
        stream,
        {},
        {build},
        hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        {reduce::get_accumulator_type(op, d_in, init)});
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

hipError_t cccl_device_reduce_cleanup(cccl_device_reduce_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_reduce_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}
