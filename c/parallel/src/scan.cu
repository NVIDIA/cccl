//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/detail/launcher/cuda_driver.cuh>
#include <cub/device/dispatch/dispatch_scan.cuh>
#include <cub/thread/thread_load.cuh>
#include <cub/util_arch.cuh>
#include <cub/util_temporary_storage.cuh>
#include <cub/util_type.cuh>

#include <format>
#include <iostream>
#include <optional>
#include <string>
#include <type_traits>

#include <hip/hiprtc.h>

#include "cub/util_device.cuh"
#include "kernels/iterators.h"
#include "kernels/operators.h"
#include "util/context.h"
#include "util/errors.h"
#include "util/indirect_arg.h"
#include "util/scan_tile_state.h"
#include "util/types.h"
#include <cccl/c/scan.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>

struct op_wrapper;
struct device_scan_policy;
using OffsetT = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

struct input_iterator_state_t;
struct output_iterator_t;

namespace scan
{

struct scan_runtime_tuning_policy
{
  int block_size;
  int items_per_thread;
  hipcub::CacheLoadModifier load_modifier;

  scan_runtime_tuning_policy Scan() const
  {
    return *this;
  }

  int ItemsPerThread() const
  {
    return items_per_thread;
  }

  int BlockThreads() const
  {
    return block_size;
  }

  hipcub::CacheLoadModifier LoadModifier() const
  {
    return load_modifier;
  }

  void CheckLoadModifier() const
  {
    if (LoadModifier() == hipcub::CacheLoadModifier::LOAD_LDG)
    {
      throw std::runtime_error("The memory consistency model does not apply to texture "
                               "accesses");
    }
  }
};

template <typename Tuning, int N>
Tuning find_tuning(int cc, const Tuning (&tunings)[N])
{
  for (const Tuning& tuning : tunings)
  {
    if (cc >= tuning.cc)
    {
      return tuning;
    }
  }

  return tunings[N - 1];
}

scan_runtime_tuning_policy get_policy(int /*cc*/, cccl_type_info /*accumulator_type*/)
{
  // TODO: we should update this once we figure out a way to reuse
  // tuning logic from C++. Alternately, we should implement
  // something better than a hardcoded default:
  return {128, 4, hipcub::LOAD_DEFAULT};
}

static cccl_type_info get_accumulator_type(cccl_op_t /*op*/, cccl_iterator_t /*input_it*/, cccl_value_t init)
{
  // TODO Should be decltype(op(init, *input_it)) but haven't implemented type arithmetic yet
  //      so switching back to the old accumulator type logic for now
  return init.type;
}

std::string get_input_iterator_name()
{
  std::string iterator_t;
  check(nvrtcGetTypeName<input_iterator_state_t>(&iterator_t));
  return iterator_t;
}

std::string get_output_iterator_name()
{
  std::string iterator_t;
  check(nvrtcGetTypeName<output_iterator_t>(&iterator_t));
  return iterator_t;
}

std::string
get_init_kernel_name(cccl_iterator_t input_it, cccl_iterator_t /*output_it*/, cccl_op_t op, cccl_value_t init)
{
  const cccl_type_info accum_t  = scan::get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  return std::format("hipcub::detail::scan::DeviceScanInitKernel<hipcub::ScanTileState<{0}>>", accum_cpp_t);
}

std::string get_scan_kernel_name(
  cccl_iterator_t input_it, cccl_iterator_t output_it, cccl_op_t op, cccl_value_t init, bool force_inclusive)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_scan_policy>(&chained_policy_t));

  const cccl_type_info accum_t  = scan::get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  const std::string input_iterator_t =
    (input_it.type == cccl_iterator_kind_t::CCCL_POINTER //
       ? cccl_type_enum_to_name(input_it.value_type.type, true) //
       : scan::get_input_iterator_name());
  const std::string output_iterator_t =
    output_it.type == cccl_iterator_kind_t::CCCL_POINTER //
      ? cccl_type_enum_to_name(output_it.value_type.type, true) //
      : scan::get_output_iterator_name();
  const std::string init_t = cccl_type_enum_to_name(init.type.type);

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  std::string scan_op_t;
  check(nvrtcGetTypeName<op_wrapper>(&scan_op_t));

  auto tile_state_t = std::format("hipcub::ScanTileState<{0}>", accum_cpp_t);
  return std::format(
    "hipcub::detail::scan::DeviceScanKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}, {7}, {8}, {9}>",
    chained_policy_t, // 0
    input_iterator_t, // 1
    output_iterator_t, // 2
    tile_state_t, // 3
    scan_op_t, // 4
    init_t, // 5
    offset_t, // 6
    accum_cpp_t, // 7
    force_inclusive ? "true" : "false", // 8
    init_t); // 9
}

template <auto* GetPolicy>
struct dynamic_scan_policy_t
{
  using MaxPolicy = dynamic_scan_policy_t;

  template <typename F>
  hipError_t Invoke(int device_ptx_version, F& op)
  {
    return op.template Invoke<scan_runtime_tuning_policy>(GetPolicy(device_ptx_version, accumulator_type));
  }

  cccl_type_info accumulator_type;
};

struct scan_kernel_source
{
  cccl_device_scan_build_result_t& build;

  std::size_t AccumSize() const
  {
    return build.accumulator_type.size;
  }
  hipKernel_t InitKernel() const
  {
    return build.init_kernel;
  }
  hipKernel_t ScanKernel() const
  {
    return build.scan_kernel;
  }
  scan_tile_state TileState()
  {
    return {build.description_bytes_per_tile, build.payload_bytes_per_tile};
  }
};
} // namespace scan

hipError_t cccl_device_scan_build(
  cccl_device_scan_build_result_t* build_ptr,
  cccl_iterator_t input_it,
  cccl_iterator_t output_it,
  cccl_op_t op,
  cccl_value_t init,
  bool force_inclusive,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "test";

    const int cc                 = cc_major * 10 + cc_minor;
    const cccl_type_info accum_t = scan::get_accumulator_type(op, input_it, init);
    const auto policy            = scan::get_policy(cc, accum_t);
    const auto accum_cpp         = cccl_type_enum_to_name(accum_t.type);
    const auto input_it_value_t  = cccl_type_enum_to_name(input_it.value_type.type);
    const auto offset_t          = cccl_type_enum_to_name(cccl_type_enum::CCCL_UINT64);

    const std::string input_iterator_src =
      make_kernel_input_iterator(offset_t, "input_iterator_state_t", input_it_value_t, input_it);
    const std::string output_iterator_src =
      make_kernel_output_iterator(offset_t, "output_iterator_t", accum_cpp, output_it);

    const std::string op_src = make_kernel_user_binary_operator(accum_cpp, accum_cpp, accum_cpp, op);

    constexpr std::string_view src_template = R"XXX(
#include <cub/block/block_scan.cuh>
#include <cub/device/dispatch/kernels/scan.cuh>
#include <cub/agent/single_pass_scan_operators.cuh>
struct __align__({1}) storage_t {{
  char data[{0}];
}};
{4}
{5}
struct agent_policy_t {{
  static constexpr int ITEMS_PER_THREAD = {2};
  static constexpr int BLOCK_THREADS = {3};
  static constexpr hipcub::BlockLoadAlgorithm LOAD_ALGORITHM = hipcub::BLOCK_LOAD_WARP_TRANSPOSE;
  static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::LOAD_DEFAULT;
  static constexpr hipcub::BlockStoreAlgorithm STORE_ALGORITHM = hipcub::BLOCK_STORE_WARP_TRANSPOSE;
  static constexpr hipcub::BlockScanAlgorithm SCAN_ALGORITHM = hipcub::BLOCK_SCAN_WARP_SCANS;
  struct detail {{
    using delay_constructor_t = hipcub::detail::default_delay_constructor_t<{7}>;
  }};
}};
struct device_scan_policy {{
  struct ActivePolicy {{
    using ScanPolicyT = agent_policy_t;
  }};
}};
{6}
)XXX";

    const std::string& src = std::format(
      src_template,
      input_it.value_type.size, // 0
      input_it.value_type.alignment, // 1
      policy.items_per_thread, // 2
      policy.block_size, // 3
      input_iterator_src, // 4
      output_iterator_src, // 5
      op_src, // 6
      accum_cpp); // 7

#if false // CCCL_DEBUGGING_SWITCH
    fflush(stderr);
    printf("\nCODE4NVRTC BEGIN\n%sCODE4NVRTC END\n", src.c_str());
    fflush(stdout);
#endif

    std::string init_kernel_name = scan::get_init_kernel_name(input_it, output_it, op, init);
    std::string scan_kernel_name = scan::get_scan_kernel_name(input_it, output_it, op, init, force_inclusive);
    std::string init_kernel_lowered_name;
    std::string scan_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    constexpr size_t num_args  = 8;
    const char* args[num_args] = {
      arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto", "-DCUB_DISABLE_CDP"};

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_ltoir_list ltoir_list;
    nvrtc_ltoir_list_appender appender{ltoir_list};

    appender.append({op.ltoir, op.ltoir_size});
    appender.add_iterator_definition(input_it);
    appender.add_iterator_definition(output_it);

    nvrtc_link_result result =
      make_nvrtc_command_list()
        .add_program(nvrtc_translation_unit{src.c_str(), name})
        .add_expression({init_kernel_name})
        .add_expression({scan_kernel_name})
        .compile_program({args, num_args})
        .get_name({init_kernel_name, init_kernel_lowered_name})
        .get_name({scan_kernel_name, scan_kernel_lowered_name})
        .cleanup_program()
        .add_link_list(ltoir_list)
        .finalize_program(num_lto_args, lopts);

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build_ptr->init_kernel, build_ptr->library, init_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(&build_ptr->scan_kernel, build_ptr->library, scan_kernel_lowered_name.c_str()));

    auto [description_bytes_per_tile,
          payload_bytes_per_tile] = get_tile_state_bytes_per_tile(accum_t, accum_cpp, args, num_args, arch);

    build_ptr->cc                         = cc;
    build_ptr->cubin                      = (void*) result.data.release();
    build_ptr->cubin_size                 = result.size;
    build_ptr->accumulator_type           = accum_t;
    build_ptr->force_inclusive            = force_inclusive;
    build_ptr->description_bytes_per_tile = description_bytes_per_tile;
    build_ptr->payload_bytes_per_tile     = payload_bytes_per_tile;
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

template <hipcub::ForceInclusive EnforceInclusive>
hipError_t cccl_device_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    auto exec_status = hipcub::DispatchScan<
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      ::cuda::std::size_t,
      void,
      EnforceInclusive,
      scan::dynamic_scan_policy_t<&scan::get_policy>,
      scan::scan_kernel_source,
      hipcub::detail::CudaDriverLauncherFactory>::
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_in,
        d_out,
        op,
        init,
        num_items,
        stream,
        {build},
        hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        {scan::get_accumulator_type(op, d_in, init)});

    error = static_cast<hipError_t>(exec_status);
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }
  if (pushed)
  {
    hipCtx_t cu_context;
    hipCtxPopCurrent(&cu_context);
  }
  return error;
}

hipError_t cccl_device_exclusive_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  assert(!build.force_inclusive);
  return cccl_device_scan<hipcub::ForceInclusive::No>(
    build, d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, op, init, stream);
}

hipError_t cccl_device_inclusive_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  assert(build.force_inclusive);
  return cccl_device_scan<hipcub::ForceInclusive::Yes>(
    build, d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, op, init, stream);
}

hipError_t cccl_device_scan_cleanup(cccl_device_scan_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }
    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}
