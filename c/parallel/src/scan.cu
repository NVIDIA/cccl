//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/detail/launcher/cuda_driver.cuh>
#include <cub/device/dispatch/dispatch_scan.cuh>
#include <cub/thread/thread_load.cuh>
#include <cub/util_arch.cuh>
#include <cub/util_device.cuh>
#include <cub/util_temporary_storage.cuh>
#include <cub/util_type.cuh>

#include <format>
#include <iostream>
#include <optional>
#include <string>
#include <type_traits>
#include <vector>

#include <hip/hiprtc.h>

#include <cccl/c/scan.h>
#include <kernels/iterators.h>
#include <kernels/operators.h>
#include <nvrtc/command_list.h>
#include <nvrtc/ltoir_list_appender.h>
#include <util/build_utils.h>
#include <util/context.h>
#include <util/errors.h>
#include <util/indirect_arg.h>
#include <util/runtime_policy.h>
#include <util/scan_tile_state.h>
#include <util/types.h>

struct op_wrapper;
struct device_scan_policy;
using OffsetT = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

struct input_iterator_state_t;
struct output_iterator_t;

namespace scan
{

struct scan_runtime_tuning_policy
{
  hipcub::detail::RuntimeScanAgentPolicy scan;

  auto Scan() const
  {
    return scan;
  }

  void CheckLoadModifier() const
  {
    if (scan.LoadModifier() == hipcub::CacheLoadModifier::LOAD_LDG)
    {
      throw std::runtime_error("The memory consistency model does not apply to texture "
                               "accesses");
    }
  }

  using MaxPolicy = scan_runtime_tuning_policy;

  template <typename F>
  hipError_t Invoke(int, F& op)
  {
    return op.template Invoke<scan_runtime_tuning_policy>(*this);
  }
};

static cccl_type_info get_accumulator_type(cccl_op_t /*op*/, cccl_iterator_t /*input_it*/, cccl_value_t init)
{
  // TODO Should be decltype(op(init, *input_it)) but haven't implemented type arithmetic yet
  //      so switching back to the old accumulator type logic for now
  return init.type;
}

std::string get_input_iterator_name()
{
  std::string iterator_t;
  check(cccl_type_name_from_nvrtc<input_iterator_state_t>(&iterator_t));
  return iterator_t;
}

std::string get_output_iterator_name()
{
  std::string iterator_t;
  check(cccl_type_name_from_nvrtc<output_iterator_t>(&iterator_t));
  return iterator_t;
}

std::string
get_init_kernel_name(cccl_iterator_t input_it, cccl_iterator_t /*output_it*/, cccl_op_t op, cccl_value_t init)
{
  const cccl_type_info accum_t  = scan::get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  return std::format("hipcub::detail::scan::DeviceScanInitKernel<hipcub::ScanTileState<{0}>>", accum_cpp_t);
}

std::string get_scan_kernel_name(
  cccl_iterator_t input_it, cccl_iterator_t output_it, cccl_op_t op, cccl_value_t init, bool force_inclusive)
{
  std::string chained_policy_t;
  check(cccl_type_name_from_nvrtc<device_scan_policy>(&chained_policy_t));

  const cccl_type_info accum_t  = scan::get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  const std::string input_iterator_t =
    (input_it.type == cccl_iterator_kind_t::CCCL_POINTER //
       ? cccl_type_enum_to_name(input_it.value_type.type, true) //
       : scan::get_input_iterator_name());
  const std::string output_iterator_t =
    output_it.type == cccl_iterator_kind_t::CCCL_POINTER //
      ? cccl_type_enum_to_name(output_it.value_type.type, true) //
      : scan::get_output_iterator_name();
  const std::string init_t = cccl_type_enum_to_name(init.type.type);

  std::string offset_t;
  check(cccl_type_name_from_nvrtc<OffsetT>(&offset_t));

  std::string scan_op_t;
  check(cccl_type_name_from_nvrtc<op_wrapper>(&scan_op_t));

  auto tile_state_t = std::format("hipcub::ScanTileState<{0}>", accum_cpp_t);
  return std::format(
    "hipcub::detail::scan::DeviceScanKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}, {7}, {8}, {9}>",
    chained_policy_t, // 0
    input_iterator_t, // 1
    output_iterator_t, // 2
    tile_state_t, // 3
    scan_op_t, // 4
    init_t, // 5
    offset_t, // 6
    accum_cpp_t, // 7
    force_inclusive ? "true" : "false", // 8
    init_t); // 9
}

template <auto* GetPolicy>
struct dynamic_scan_policy_t
{
  using MaxPolicy = dynamic_scan_policy_t;

  template <typename F>
  hipError_t Invoke(int device_ptx_version, F& op)
  {
    return op.template Invoke<scan_runtime_tuning_policy>(GetPolicy(device_ptx_version, accumulator_type));
  }

  cccl_type_info accumulator_type;
};

struct scan_kernel_source
{
  cccl_device_scan_build_result_t& build;

  std::size_t AccumSize() const
  {
    return build.accumulator_type.size;
  }
  hipKernel_t InitKernel() const
  {
    return build.init_kernel;
  }
  hipKernel_t ScanKernel() const
  {
    return build.scan_kernel;
  }
  scan_tile_state TileState()
  {
    return {build.description_bytes_per_tile, build.payload_bytes_per_tile};
  }
};
} // namespace scan

hipError_t cccl_device_scan_build_ex(
  cccl_device_scan_build_result_t* build_ptr,
  cccl_iterator_t input_it,
  cccl_iterator_t output_it,
  cccl_op_t op,
  cccl_value_t init,
  bool force_inclusive,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path,
  cccl_build_config* config)
{
  hipError_t error = hipSuccess;

  try
  {
    const char* name = "test";

    const int cc                 = cc_major * 10 + cc_minor;
    const cccl_type_info accum_t = scan::get_accumulator_type(op, input_it, init);
    const auto accum_cpp         = cccl_type_enum_to_name(accum_t.type);
    const auto input_it_value_t  = cccl_type_enum_to_name(input_it.value_type.type);
    const auto offset_t          = cccl_type_enum_to_name(cccl_type_enum::CCCL_UINT64);

    const std::string input_iterator_src =
      make_kernel_input_iterator(offset_t, "input_iterator_state_t", input_it_value_t, input_it);
    const std::string output_iterator_src =
      make_kernel_output_iterator(offset_t, "output_iterator_t", accum_cpp, output_it);

    const std::string op_src = make_kernel_user_binary_operator(accum_cpp, accum_cpp, accum_cpp, op);

    constexpr std::string_view src_template = R"XXX(
#include <cub/block/block_scan.cuh>
#include <cub/device/dispatch/kernels/scan.cuh>
#include <cub/agent/single_pass_scan_operators.cuh>
struct __align__({1}) storage_t {{
  char data[{0}];
}};
{2}
{3}
{4}
)XXX";

    const std::string& src = std::format(
      src_template,
      input_it.value_type.size, // 0
      input_it.value_type.alignment, // 1
      input_iterator_src, // 2
      output_iterator_src, // 3
      op_src); // 4

    const auto output_it_value_t = cccl_type_enum_to_name(output_it.value_type.type);

    const std::string ptx_arch = std::format("-arch=compute_{}{}", cc_major, cc_minor);

    std::vector<const char*> ptx_args = {
      ptx_arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true"};

    cccl::detail::extend_args_with_build_config(ptx_args, config);

    std::string policy_hub_expr = std::format(
      "hipcub::detail::scan::policy_hub<{}, {}, {}, {}, {}>",
      input_it_value_t,
      output_it_value_t,
      accum_cpp,
      offset_t,
      "op_wrapper");

    nlohmann::json runtime_policy = get_policy(
      std::format("hipcub::detail::scan::MakeScanPolicyWrapper({}::MaxPolicy::ActivePolicy{{}})", policy_hub_expr),
      "#include <cub/device/dispatch/tuning/tuning_scan.cuh>\n" + src,
      ptx_args);

    auto delay_ctor_info = runtime_policy["DelayConstructor"];
    std::string delay_ctor_params;
    for (auto&& param : delay_ctor_info["params"])
    {
      delay_ctor_params.append(to_string(param) + ", ");
    }
    delay_ctor_params.erase(delay_ctor_params.size() - 2); // remove last ", "
    auto delay_ctor_t =
      std::format("hipcub::detail::{}<{}>", delay_ctor_info["name"].get<std::string>(), delay_ctor_params);

    using hipcub::detail::RuntimeScanAgentPolicy;
    auto [scan_policy,
          scan_policy_str] = RuntimeScanAgentPolicy::from_json(runtime_policy, "ScanPolicyT", delay_ctor_t);

    std::string final_src = std::format(
      R"XXX(
{0}
struct device_scan_policy {{
  struct ActivePolicy {{
    {1}
  }};
}};
)XXX",
      src,
      scan_policy_str);

#if false // CCCL_DEBUGGING_SWITCH
    fflush(stderr);
    printf("\nCODE4NVRTC BEGIN\n%sCODE4NVRTC END\n", final_src.c_str());
    fflush(stdout);
#endif

    std::string init_kernel_name = scan::get_init_kernel_name(input_it, output_it, op, init);
    std::string scan_kernel_name = scan::get_scan_kernel_name(input_it, output_it, op, init, force_inclusive);
    std::string init_kernel_lowered_name;
    std::string scan_kernel_lowered_name;

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    std::vector<const char*> args = {
      arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto", "-DCUB_DISABLE_CDP"};

    cccl::detail::extend_args_with_build_config(args, config);

    constexpr size_t num_lto_args   = 2;
    const char* lopts[num_lto_args] = {"-lto", arch.c_str()};

    // Collect all LTO-IRs to be linked.
    nvrtc_linkable_list linkable_list;
    nvrtc_linkable_list_appender appender{linkable_list};

    appender.append_operation(op);
    appender.add_iterator_definition(input_it);
    appender.add_iterator_definition(output_it);

    nvrtc_link_result result =
      begin_linking_nvrtc_program(num_lto_args, lopts)
        ->add_program(nvrtc_translation_unit{final_src.c_str(), name})
        ->add_expression({init_kernel_name})
        ->add_expression({scan_kernel_name})
        ->compile_program({args.data(), args.size()})
        ->get_name({init_kernel_name, init_kernel_lowered_name})
        ->get_name({scan_kernel_name, scan_kernel_lowered_name})
        ->link_program()
        ->add_link_list(linkable_list)
        ->finalize_program();

    hipLibraryLoadData(&build_ptr->library, result.data.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build_ptr->init_kernel, build_ptr->library, init_kernel_lowered_name.c_str()));
    check(hipLibraryGetKernel(&build_ptr->scan_kernel, build_ptr->library, scan_kernel_lowered_name.c_str()));

    auto [description_bytes_per_tile,
          payload_bytes_per_tile] = get_tile_state_bytes_per_tile(accum_t, accum_cpp, args.data(), args.size(), arch);

    build_ptr->cc                         = cc;
    build_ptr->cubin                      = (void*) result.data.release();
    build_ptr->cubin_size                 = result.size;
    build_ptr->accumulator_type           = accum_t;
    build_ptr->force_inclusive            = force_inclusive;
    build_ptr->description_bytes_per_tile = description_bytes_per_tile;
    build_ptr->payload_bytes_per_tile     = payload_bytes_per_tile;
    build_ptr->runtime_policy             = new scan::scan_runtime_tuning_policy{scan_policy};
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan_build(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }

  return error;
}

template <hipcub::ForceInclusive EnforceInclusive>
hipError_t cccl_device_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    auto exec_status = hipcub::DispatchScan<
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      indirect_arg_t,
      ::cuda::std::size_t,
      void,
      EnforceInclusive,
      scan::scan_runtime_tuning_policy,
      scan::scan_kernel_source,
      hipcub::detail::CudaDriverLauncherFactory>::
      Dispatch(
        d_temp_storage,
        *temp_storage_bytes,
        d_in,
        d_out,
        op,
        init,
        num_items,
        stream,
        {build},
        hipcub::detail::CudaDriverLauncherFactory{cu_device, build.cc},
        *reinterpret_cast<scan::scan_runtime_tuning_policy*>(build.runtime_policy));

    error = static_cast<hipError_t>(exec_status);
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan(): %s\n", exc.what());
    fflush(stdout);
    error = hipErrorUnknown;
  }
  if (pushed)
  {
    hipCtx_t cu_context;
    hipCtxPopCurrent(&cu_context);
  }
  return error;
}

hipError_t cccl_device_exclusive_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  assert(!build.force_inclusive);
  return cccl_device_scan<hipcub::ForceInclusive::No>(
    build, d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, op, init, stream);
}

hipError_t cccl_device_inclusive_scan(
  cccl_device_scan_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  uint64_t num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream)
{
  assert(build.force_inclusive);
  return cccl_device_scan<hipcub::ForceInclusive::Yes>(
    build, d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, op, init, stream);
}

hipError_t cccl_device_scan_build(
  cccl_device_scan_build_result_t* build_ptr,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  cccl_op_t op,
  cccl_value_t init,
  bool force_inclusive,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path)
{
  return cccl_device_scan_build_ex(
    build_ptr,
    d_in,
    d_out,
    op,
    init,
    force_inclusive,
    cc_major,
    cc_minor,
    cub_path,
    thrust_path,
    libcudacxx_path,
    ctk_path,
    nullptr);
}

hipError_t cccl_device_scan_cleanup(cccl_device_scan_build_result_t* build_ptr)
{
  try
  {
    if (build_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }
    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(build_ptr->cubin));
    check(hipLibraryUnload(build_ptr->library));
  }
  catch (const std::exception& exc)
  {
    fflush(stderr);
    printf("\nEXCEPTION in cccl_device_scan_cleanup(): %s\n", exc.what());
    fflush(stdout);
    return hipErrorUnknown;
  }

  return hipSuccess;
}
