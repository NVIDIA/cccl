#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cub/detail/choose_offset.cuh>
#include <cub/grid/grid_even_share.cuh>
#include <cub/util_device.cuh>

#include <cuda/std/cstdint>
#include <cuda/std/functional>

#include <format>
#include <iostream>
#include <memory>

#include <cccl/c/reduce.h>
#include <nvJitLink.h>
#include <hip/hiprtc.h>

void check(hiprtcResult result)
{
  if (result != HIPRTC_SUCCESS)
  {
    throw std::runtime_error(std::string("NVRTC error: ") + hiprtcGetErrorString(result));
  }
}

void check(hipError_t result)
{
  if (result != hipSuccess)
  {
    const char* str = nullptr;
    hipDrvGetErrorString(result, &str);
    throw std::runtime_error(std::string("CUDA error: ") + str);
  }
}

void check(nvJitLinkResult result)
{
  if (result != NVJITLINK_SUCCESS)
  {
    throw std::runtime_error(std::string("nvJitLink error: ") + std::to_string(result));
  }
}

struct op_wrapper;
struct device_reduce_policy;
using TransformOpT = ::cuda::std::__identity;
using OffsetT      = unsigned long long;
static_assert(std::is_same_v<hipcub::detail::choose_offset_t<OffsetT>, OffsetT>, "OffsetT must be size_t");

struct nothing_t
{};

struct runtime_tuning_policy
{
  int block_size;
  int items_per_thread;
  int vector_load_length;
};

struct storage_t;
struct input_iterator_state_t;
struct output_iterator_t;

char const* cccl_type_enum_to_string(cccl_type_enum type)
{
  switch (type)
  {
    case cccl_type_enum::INT8:
      return "::cuda::std::int8_t";
    case cccl_type_enum::INT16:
      return "::cuda::std::int16_t";
    case cccl_type_enum::INT32:
      return "::cuda::std::int32_t";
    case cccl_type_enum::INT64:
      return "::cuda::std::int64_t";
    case cccl_type_enum::UINT8:
      return "::cuda::std::uint8_t";
    case cccl_type_enum::UINT16:
      return "::cuda::std::uint16_t";
    case cccl_type_enum::UINT32:
      return "::cuda::std::uint32_t";
    case cccl_type_enum::UINT64:
      return "::cuda::std::uint64_t";
    case cccl_type_enum::FLOAT32:
      return "float";
    case cccl_type_enum::FLOAT64:
      return "double";
    case cccl_type_enum::STORAGE:
      return "storage_t";
  }
  return "unknown";
}

std::string cccl_type_enum_to_name(cccl_type_enum type, bool is_pointer = false)
{
  std::string result;

  if (is_pointer)
  {
    switch (type)
    {
      case cccl_type_enum::INT8:

        check(nvrtcGetTypeName<::cuda::std::int8_t*>(&result));
        break;
      case cccl_type_enum::INT16:
        check(nvrtcGetTypeName<::cuda::std::int16_t*>(&result));
        break;
      case cccl_type_enum::INT32:
        check(nvrtcGetTypeName<::cuda::std::int32_t*>(&result));
        break;
      case cccl_type_enum::INT64:
        check(nvrtcGetTypeName<::cuda::std::int64_t*>(&result));
        break;
      case cccl_type_enum::UINT8:
        check(nvrtcGetTypeName<::cuda::std::uint8_t*>(&result));
        break;
      case cccl_type_enum::UINT16:
        check(nvrtcGetTypeName<::cuda::std::uint16_t*>(&result));
        break;
      case cccl_type_enum::UINT32:
        check(nvrtcGetTypeName<::cuda::std::uint32_t*>(&result));
        break;
      case cccl_type_enum::UINT64:
        check(nvrtcGetTypeName<::cuda::std::uint64_t*>(&result));
        break;
      case cccl_type_enum::FLOAT32:
        check(nvrtcGetTypeName<float*>(&result));
        break;
      case cccl_type_enum::FLOAT64:
        check(nvrtcGetTypeName<double*>(&result));
        break;
      case cccl_type_enum::STORAGE:
        check(nvrtcGetTypeName<storage_t*>(&result));
        break;
    }
  }
  else
  {
    switch (type)
    {
      case cccl_type_enum::INT8:
        check(nvrtcGetTypeName<::cuda::std::int8_t>(&result));
        break;
      case cccl_type_enum::INT16:
        check(nvrtcGetTypeName<::cuda::std::int16_t>(&result));
        break;
      case cccl_type_enum::INT32:
        check(nvrtcGetTypeName<::cuda::std::int32_t>(&result));
        break;
      case cccl_type_enum::INT64:
        check(nvrtcGetTypeName<::cuda::std::int64_t>(&result));
        break;
      case cccl_type_enum::UINT8:
        check(nvrtcGetTypeName<::cuda::std::uint8_t>(&result));
        break;
      case cccl_type_enum::UINT16:
        check(nvrtcGetTypeName<::cuda::std::uint16_t>(&result));
        break;
      case cccl_type_enum::UINT32:
        check(nvrtcGetTypeName<::cuda::std::uint32_t>(&result));
        break;
      case cccl_type_enum::UINT64:
        check(nvrtcGetTypeName<::cuda::std::uint64_t>(&result));
        break;
      case cccl_type_enum::FLOAT32:
        check(nvrtcGetTypeName<float>(&result));
        break;
      case cccl_type_enum::FLOAT64:
        check(nvrtcGetTypeName<double>(&result));
        break;
      case cccl_type_enum::STORAGE:
        check(nvrtcGetTypeName<storage_t>(&result));
        break;
    }
  }

  return result;
}

struct reduce_tuning_t
{
  int cc;
  int block_size;
  int items_per_thread;
  int vector_load_length;
};

template <int N>
reduce_tuning_t find_tuning(int cc, const reduce_tuning_t (&tunings)[N])
{
  for (const reduce_tuning_t& tuning : tunings)
  {
    if (cc >= tuning.cc)
    {
      return tuning;
    }
  }

  return tunings[N - 1];
}

runtime_tuning_policy get_policy(int cc, cccl_type_info accumulator_type, cccl_type_info input_type)
{
  reduce_tuning_t chain[] = {{60, 256, 16, 4}, {35, 256, 20, 4}};

  auto [_, block_size, items_per_thread, vector_load_length] = find_tuning(cc, chain);

  // Implement part of MemBoundScaling
  items_per_thread = CUB_MAX(1, CUB_MIN(items_per_thread * 4 / accumulator_type.size, items_per_thread * 2));
  block_size       = CUB_MIN(block_size, (((1024 * 48) / (accumulator_type.size * items_per_thread)) + 31) / 32 * 32);

  return {block_size, items_per_thread, vector_load_length};
}

cccl_type_info get_accumulator_type(cccl_op_t op, cccl_iterator_t input_it, cccl_value_t init)
{
  // TODO Should be decltype(op(init, *input_it)) but haven't implemented type arithmetic yet
  //      so switching back to the old accumulator type logic for now
  return init.type;
}

hipError_t InvokeSingleTile(
  void* d_temp_storage,
  std::size_t& temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  unsigned long long num_items,
  cccl_op_t op,
  cccl_value_t init,
  int cc,
  hipKernel_t single_tile_kernel,
  hipStream_t stream)
{
  const runtime_tuning_policy policy = get_policy(cc, d_in.value_type, d_in.value_type);

  hipError_t error = hipSuccess;
  do
  {
    if (d_temp_storage == nullptr)
    {
      temp_storage_bytes = 1;
      break;
    }

    nothing_t nothing{};
    TransformOpT transform_op{};
    void* op_state = op.type == cccl_op_kind_t::stateless ? &nothing : op.state;
    void* in_ptr   = d_in.type == cccl_iterator_kind_t::pointer ? &d_in.state : d_in.state;
    void* out_ptr  = d_out.type == cccl_iterator_kind_t::pointer ? &d_out.state : d_out.state;
    void* args[]   = {in_ptr, out_ptr, &num_items, op_state, init.state, &transform_op};

    check(hipModuleLaunchKernel((hipFunction_t) single_tile_kernel, 1, 1, 1, policy.block_size, 1, 1, 0, stream, args, 0));

    // Check for failure to launch
    error = HipcubDebug(hipPeekAtLastError());
    if (hipSuccess != error)
    {
      break;
    }
  } while (0);

  return error;
}

hipError_t InvokePasses(
  void* d_temp_storage,
  std::size_t& temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  unsigned long long num_items,
  cccl_op_t op,
  cccl_value_t init,
  int cc,
  hipKernel_t reduce_kernel,
  hipKernel_t single_tile_kernel,
  hipDevice_t device,
  hipStream_t stream)
{
  const cccl_type_info accum_t       = get_accumulator_type(op, d_in, init);
  const runtime_tuning_policy policy = get_policy(cc, accum_t, d_in.value_type);

  hipError_t error = hipSuccess;
  do
  {
    void* in_ptr  = d_in.type == cccl_iterator_kind_t::pointer ? &d_in.state : d_in.state;
    void* out_ptr = d_out.type == cccl_iterator_kind_t::pointer ? &d_out.state : d_out.state;

    // Get SM count
    int sm_count;
    check(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device));

    // Init regular kernel configuration
    const auto tile_size = policy.block_size * policy.items_per_thread;

    // Older drivers have issues handling hipKernel_t in the occupancy queries, get the hipFunction_t instead.
    // Assumes that the current device is properly set, it needs to be set for the occupancy queries anyway
    hipFunction_t reduce_kernel_fn;
    check(cuKernelGetFunction(&reduce_kernel_fn, reduce_kernel));

    int sm_occupancy = 1;
    check(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&sm_occupancy, reduce_kernel_fn, policy.block_size, 0));

    int reduce_device_occupancy = sm_occupancy * sm_count;

    // Even-share work distribution
    int max_blocks = reduce_device_occupancy * CUB_SUBSCRIPTION_FACTOR(0);
    hipcub::GridEvenShare<OffsetT> even_share;
    even_share.DispatchInit(num_items, max_blocks, tile_size);

    // Temporary storage allocation requirements
    void* allocations[1]       = {};
    size_t allocation_sizes[1] = {
      max_blocks * static_cast<std::size_t>(d_in.value_type.size) // bytes needed for privatized block reductions
    };

    // Alias the temporary allocations from the single storage blob (or
    // compute the necessary size of the blob)
    error = HipcubDebug(hipcub::AliasTemporaries(d_temp_storage, temp_storage_bytes, allocations, allocation_sizes));
    if (hipSuccess != error)
    {
      break;
    }

    if (d_temp_storage == nullptr)
    {
      // Return if the caller is simply requesting the size of the storage
      // allocation
      return hipSuccess;
    }

    // Get grid size for device_reduce_sweep_kernel
    OffsetT reduce_grid_size = even_share.grid_size;

    // Invoke DeviceReduceKernel
    // reduce_kernel<<<reduce_grid_size, ActivePolicyT::ReducePolicy::BLOCK_THREADS>>>(
    //    d_in, d_block_reductions, num_items, even_share, ReductionOpT{}, TransformOpT{});

    nothing_t nothing{};
    void* op_state = op.type == cccl_op_kind_t::stateless ? &nothing : op.state;

    TransformOpT transform_op{};
    void* reduce_args[] = {in_ptr, &allocations[0], &num_items, &even_share, op_state, &transform_op};

    check(hipModuleLaunchKernel(
      (hipFunction_t) reduce_kernel, reduce_grid_size, 1, 1, policy.block_size, 1, 1, 0, stream, reduce_args, 0));

    // Check for failure to launch
    error = HipcubDebug(hipPeekAtLastError());
    if (hipSuccess != error)
    {
      break;
    }

    // single_tile_kernel<<<1, ActivePolicyT::SingleTilePolicy::BLOCK_THREADS>>>(
    //     d_block_reductions, d_out, reduce_grid_size, ReductionOpT{}, 0, TransformOpT{});

    void* single_tile_kernel_args[] = {&allocations[0], out_ptr, &reduce_grid_size, op_state, init.state, &transform_op};

    check(hipModuleLaunchKernel(
      (hipFunction_t) single_tile_kernel, 1, 1, 1, policy.block_size, 1, 1, 0, stream, single_tile_kernel_args, 0));

    // Check for failure to launch
    error = HipcubDebug(hipPeekAtLastError());
    if (hipSuccess != error)
    {
      break;
    }
  } while (0);

  return error;
}

hipError_t Invoke(
  void* d_temp_storage,
  std::size_t& temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  OffsetT num_items,
  cccl_op_t op,
  cccl_value_t init,
  int cc,
  hipKernel_t single_tile_kernel,
  hipKernel_t single_tile_second_kernel,
  hipKernel_t reduce_kernel,
  hipDevice_t device,
  hipStream_t stream)
{
  const cccl_type_info accum_t = get_accumulator_type(op, d_in, init);
  runtime_tuning_policy policy = get_policy(cc, accum_t, d_in.value_type);

  // Force kernel code-generation in all compiler passes
  if (num_items <= (policy.block_size * policy.items_per_thread))
  {
    // Small, single tile size
    return InvokeSingleTile(
      d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, op, init, cc, single_tile_kernel, stream);
  }
  else
  {
    // Multi-tile pass
    return InvokePasses(
      d_temp_storage,
      temp_storage_bytes,
      d_in,
      d_out,
      num_items,
      op,
      init,
      cc,
      reduce_kernel,
      single_tile_second_kernel,
      device,
      stream);
  }
}

std::string get_input_iterator_name()
{
  std::string iterator_t;
  check(nvrtcGetTypeName<input_iterator_state_t>(&iterator_t));
  return iterator_t;
}

std::string get_output_iterator_name()
{
  std::string iterator_t;
  check(nvrtcGetTypeName<output_iterator_t>(&iterator_t));
  return iterator_t;
}

std::string get_single_tile_kernel_name(
  cccl_iterator_t input_it, cccl_iterator_t output_it, cccl_op_t op, cccl_value_t init, bool is_second_kernel)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  const cccl_type_info accum_t  = get_accumulator_type(op, input_it, init);
  const std::string accum_cpp_t = cccl_type_enum_to_name(accum_t.type);
  const std::string input_iterator_t =
    is_second_kernel ? cccl_type_enum_to_name(accum_t.type, true)
    : input_it.type == cccl_iterator_kind_t::pointer //
      ? cccl_type_enum_to_name(input_it.value_type.type, true) //
      : get_input_iterator_name();
  const std::string output_iterator_t =
    output_it.type == cccl_iterator_kind_t::pointer //
      ? cccl_type_enum_to_name(output_it.value_type.type, true) //
      : get_output_iterator_name();
  const std::string init_t = cccl_type_enum_to_name(init.type.type);

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  std::string reduction_op_t;
  check(nvrtcGetTypeName<op_wrapper>(&reduction_op_t));

  return std::format(
    "hipcub::DeviceReduceSingleTileKernel<{0}, {1}, {2}, {3}, {4}, {5}, {6}>",
    chained_policy_t,
    input_iterator_t,
    output_iterator_t,
    offset_t,
    reduction_op_t,
    init_t,
    accum_cpp_t);
}

std::string get_device_reduce_kernel_name(cccl_op_t op, cccl_iterator_t input_it, cccl_value_t init)
{
  std::string chained_policy_t;
  check(nvrtcGetTypeName<device_reduce_policy>(&chained_policy_t));

  const std::string input_iterator_t =
    input_it.type == cccl_iterator_kind_t::pointer //
      ? cccl_type_enum_to_name(input_it.value_type.type, true) //
      : get_input_iterator_name();

  const std::string accum_t = cccl_type_enum_to_name(get_accumulator_type(op, input_it, init).type);

  std::string offset_t;
  check(nvrtcGetTypeName<OffsetT>(&offset_t));

  std::string reduction_op_t;
  check(nvrtcGetTypeName<op_wrapper>(&reduction_op_t));

  std::string transform_op_t;
  check(nvrtcGetTypeName<cuda::std::__identity>(&transform_op_t));

  return std::format(
    "hipcub::DeviceReduceKernel<{0}, {1}, {2}, {3}, {4}, {5}>",
    chained_policy_t,
    input_iterator_t,
    offset_t,
    reduction_op_t,
    accum_t,
    transform_op_t);
}

bool try_push_context()
{
  hipCtx_t context = nullptr;

  check(hipCtxGetCurrent(&context));

  if (context == nullptr)
  {
    const int default_device = 0;
    check(hipDevicePrimaryCtxRetain(&context, default_device));
    check(hipCtxPushCurrent(context));

    return true;
  }

  return false;
}

extern "C" CCCL_C_API hipError_t cccl_device_reduce_build(
  cccl_device_reduce_build_result_t* build,
  cccl_iterator_t input_it,
  cccl_iterator_t output_it,
  cccl_op_t op,
  cccl_value_t init,
  int cc_major,
  int cc_minor,
  const char* cub_path,
  const char* thrust_path,
  const char* libcudacxx_path,
  const char* ctk_path) noexcept
{
  hipError_t error = hipSuccess;

  try
  {
    hiprtcProgram prog{};
    const char* name = "test";

    const int cc                       = cc_major * 10 + cc_minor;
    const cccl_type_info accum_t       = get_accumulator_type(op, input_it, init);
    const std::string accum_cpp        = cccl_type_enum_to_string(accum_t.type);
    const runtime_tuning_policy policy = get_policy(cc, accum_t, input_it.value_type);
    const std::string input_it_value_t = cccl_type_enum_to_string(input_it.value_type.type);
    const std::string offset_t         = cccl_type_enum_to_string(cccl_type_enum::UINT64);

    const std::string input_iterator_src =
      input_it.type == cccl_iterator_kind_t::pointer
        ? std::string{}
        : std::format(
            "extern \"C\" __device__ {3} {4}(const void *self_ptr);\n"
            "extern \"C\" __device__ void {5}(void *self_ptr, {0} offset);\n"
            "struct __align__({2}) input_iterator_state_t {{\n;"
            "  using iterator_category = cuda::std::random_access_iterator_tag;\n"
            "  using value_type = {3};\n"
            "  using difference_type = {0};\n"
            "  using pointer = {3}*;\n"
            "  using reference = {3}&;\n"
            "  __device__ value_type operator*() const {{ return {4}(this); }}\n"
            "  __device__ input_iterator_state_t& operator+=(difference_type diff) {{\n"
            "      {5}(this, diff);\n"
            "      return *this;\n"
            "  }}\n"
            "  __device__ value_type operator[](difference_type diff) const {{\n"
            "      return *(*this + diff);\n"
            "  }}\n"
            "  __device__ input_iterator_state_t operator+(difference_type diff) const {{\n"
            "      input_iterator_state_t result = *this;\n"
            "      result += diff;\n"
            "      return result;\n"
            "  }}\n"
            "  char data[{1}];\n"
            "}};\n",
            offset_t, // 0
            input_it.size, // 1
            input_it.alignment, // 2
            input_it_value_t, // 3
            input_it.dereference.name, // 4
            input_it.advance.name); // 5

    const std::string output_iterator_src =
      output_it.type == cccl_iterator_kind_t::pointer
        ? std::string{}
        : std::format(
            "extern \"C\" __device__ void {2}(const void *self_ptr, {1} x);\n"
            "extern \"C\" __device__ void {3}(void *self_ptr, {0} offset);\n"
            "struct __align__({5}) output_iterator_state_t{{\n"
            "  char data[{4}];\n"
            "}};\n"
            "struct output_iterator_proxy_t {{\n"
            "  __device__ output_iterator_proxy_t operator=({1} x) {{\n"
            "    {2}(&state, x);\n"
            "    return *this;\n"
            "  }}\n"
            "  output_iterator_state_t state;\n"
            "}};\n"
            "struct output_iterator_t {{\n"
            "  using iterator_category = cuda::std::random_access_iterator_tag;\n"
            "  using difference_type   = {0};\n"
            "  using value_type        = void;\n"
            "  using pointer           = output_iterator_proxy_t*;\n"
            "  using reference         = output_iterator_proxy_t;\n"
            "  __device__ output_iterator_proxy_t operator*() const {{ return {{state}}; }}\n"
            "  __device__ output_iterator_t& operator+=(difference_type diff) {{\n"
            "      {3}(&state, diff);\n"
            "      return *this;\n"
            "  }}\n"
            "  __device__ output_iterator_proxy_t operator[](difference_type diff) const {{\n"
            "    output_iterator_t result = *this;\n"
            "    result += diff;\n"
            "    return {{ result.state }};\n"
            "  }}\n"
            "  __device__ output_iterator_t operator+(difference_type diff) const {{\n"
            "    output_iterator_t result = *this;\n"
            "    result += diff;\n"
            "    return result;\n"
            "  }}\n"
            "  output_iterator_state_t state;\n"
            "}};",
            offset_t, // 0
            accum_cpp, // 1
            output_it.dereference.name, // 2
            output_it.advance.name, // 3
            output_it.size, // 4
            output_it.alignment); // 5

    const std::string op_src =
      op.type == cccl_op_kind_t::stateless
        ? std::format(
            "extern \"C\" __device__ {0} {1}({0} lhs, {0} rhs);\n"
            "struct op_wrapper {{\n"
            "  __device__ {0} operator()({0} lhs, {0} rhs) const {{\n"
            "    return {1}(lhs, rhs);\n"
            "  }}\n"
            "}};\n",
            accum_cpp,
            op.name)
        : std::format(
            "struct __align__({2}) op_state {{\n"
            "  char data[{3}];\n"
            "}};"
            "extern \"C\" __device__ {0} {1}(op_state *state, {0} lhs, {0} rhs);\n"
            "struct op_wrapper {{\n"
            "  op_state state;\n"
            "  __device__ {0} operator()({0} lhs, {0} rhs) {{\n"
            "    return {1}(&state, lhs, rhs);\n"
            "  }}\n"
            "}};\n",
            accum_cpp,
            op.name,
            op.alignment,
            op.size);

    const std::string src = std::format(
      "#include <cub/block/block_reduce.cuh>\n"
      "#include <cub/device/dispatch/kernels/reduce.cuh>\n"
      "struct __align__({1}) storage_t {{\n"
      "  char data[{0}];\n"
      "}};\n"
      "{4}\n"
      "{5}\n"
      "struct agent_policy_t {{\n"
      "  static constexpr int ITEMS_PER_THREAD = {2};\n"
      "  static constexpr int BLOCK_THREADS = {3};\n"
      "  static constexpr int VECTOR_LOAD_LENGTH = {7};\n"
      "  static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BLOCK_REDUCE_WARP_REDUCTIONS;\n"
      "  static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::LOAD_LDG;\n"
      "}};\n"
      "struct device_reduce_policy {{\n"
      "  struct ActivePolicy {{\n"
      "    using ReducePolicy = agent_policy_t;\n"
      "    using SingleTilePolicy = agent_policy_t;\n"
      "  }};\n"
      "}};\n"
      "{6};\n",
      input_it.value_type.size, // 0
      input_it.value_type.alignment, // 1
      policy.items_per_thread, // 2
      policy.block_size, // 3
      input_iterator_src, // 4
      output_iterator_src, // 5
      op_src, // 6
      policy.vector_load_length); // 7

    check(hiprtcCreateProgram(&prog, src.c_str(), name, 0, nullptr, nullptr));

    std::string single_tile_kernel_name = get_single_tile_kernel_name(input_it, output_it, op, init, false);
    check(hiprtcAddNameExpression(prog, single_tile_kernel_name.c_str()));

    std::string single_tile_second_kernel_name = get_single_tile_kernel_name(input_it, output_it, op, init, true);
    check(hiprtcAddNameExpression(prog, single_tile_second_kernel_name.c_str()));

    std::string reduction_kernel_name = get_device_reduce_kernel_name(op, input_it, init);
    check(hiprtcAddNameExpression(prog, reduction_kernel_name.c_str()));

    const std::string arch = std::format("-arch=sm_{0}{1}", cc_major, cc_minor);

    constexpr int num_args     = 7;
    const char* args[num_args] = {arch.c_str(), cub_path, thrust_path, libcudacxx_path, ctk_path, "-rdc=true", "-dlto"};

    std::size_t log_size{};
    hiprtcResult compile_result = hiprtcCompileProgram(prog, num_args, args);

    check(hiprtcGetProgramLogSize(prog, &log_size));

    std::unique_ptr<char[]> log{new char[log_size]};
    check(hiprtcGetProgramLog(prog, log.get()));

    if (log_size > 1)
    {
      std::cerr << log.get() << std::endl;
    }

    const char* single_tile_kernel_lowered_name;
    check(hiprtcGetLoweredName(prog, single_tile_kernel_name.c_str(), &single_tile_kernel_lowered_name));

    const char* single_tile_second_kernel_lowered_name;
    check(hiprtcGetLoweredName(prog, single_tile_second_kernel_name.c_str(), &single_tile_second_kernel_lowered_name));

    const char* reduction_kernel_lowered_name;
    check(hiprtcGetLoweredName(prog, reduction_kernel_name.c_str(), &reduction_kernel_lowered_name));

    // Copy lowered names to a std::unique_ptr to ensure they can be used after
    // the program is destroyed

    std::unique_ptr<char[]> single_tile_kernel_lowered_name_ptr{new char[strlen(single_tile_kernel_lowered_name) + 1]};
    strcpy(single_tile_kernel_lowered_name_ptr.get(), single_tile_kernel_lowered_name);

    std::unique_ptr<char[]> single_tile_second_kernel_lowered_name_ptr{
      new char[strlen(single_tile_second_kernel_lowered_name) + 1]};
    strcpy(single_tile_second_kernel_lowered_name_ptr.get(), single_tile_second_kernel_lowered_name);

    std::unique_ptr<char[]> reduction_kernel_lowered_name_ptr{new char[strlen(reduction_kernel_lowered_name) + 1]};
    strcpy(reduction_kernel_lowered_name_ptr.get(), reduction_kernel_lowered_name);

    check(compile_result);

    std::size_t ltoir_size{};
    check(nvrtcGetLTOIRSize(prog, &ltoir_size));
    std::unique_ptr<char[]> ltoir{new char[ltoir_size]};
    check(nvrtcGetLTOIR(prog, ltoir.get()));
    check(hiprtcDestroyProgram(&prog));

    nvJitLinkHandle handle;
    const char* lopts[] = {"-lto", arch.c_str()};
    check(nvJitLinkCreate(&handle, 2, lopts));

    check(nvJitLinkAddData(handle, NVJITLINK_INPUT_LTOIR, ltoir.get(), ltoir_size, name));
    check(nvJitLinkAddData(handle, NVJITLINK_INPUT_LTOIR, op.ltoir, op.ltoir_size, name));

    if (input_it.type == cccl_iterator_kind_t::iterator)
    {
      check(nvJitLinkAddData(handle, NVJITLINK_INPUT_LTOIR, input_it.advance.ltoir, input_it.advance.ltoir_size, name));
      check(nvJitLinkAddData(
        handle, NVJITLINK_INPUT_LTOIR, input_it.dereference.ltoir, input_it.dereference.ltoir_size, name));
    }

    if (output_it.type == cccl_iterator_kind_t::iterator)
    {
      check(
        nvJitLinkAddData(handle, NVJITLINK_INPUT_LTOIR, output_it.advance.ltoir, output_it.advance.ltoir_size, name));
      check(nvJitLinkAddData(
        handle, NVJITLINK_INPUT_LTOIR, output_it.dereference.ltoir, output_it.dereference.ltoir_size, name));
    }

    check(nvJitLinkComplete(handle));

    std::size_t cubin_size{};
    check(nvJitLinkGetLinkedCubinSize(handle, &cubin_size));
    std::unique_ptr<char[]> cubin{new char[cubin_size]};
    check(nvJitLinkGetLinkedCubin(handle, cubin.get()));
    check(nvJitLinkDestroy(&handle));

    hipLibraryLoadData(&build->library, cubin.get(), nullptr, nullptr, 0, nullptr, nullptr, 0);
    check(hipLibraryGetKernel(&build->single_tile_kernel, build->library, single_tile_kernel_lowered_name_ptr.get()));
    check(hipLibraryGetKernel(
      &build->single_tile_second_kernel, build->library, single_tile_second_kernel_lowered_name_ptr.get()));
    check(hipLibraryGetKernel(&build->reduction_kernel, build->library, reduction_kernel_lowered_name_ptr.get()));

    build->cc         = cc;
    build->cubin      = cubin.release();
    build->cubin_size = cubin_size;
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }

  return error;
}

extern "C" CCCL_C_API hipError_t cccl_device_reduce(
  cccl_device_reduce_build_result_t build,
  void* d_temp_storage,
  size_t* temp_storage_bytes,
  cccl_iterator_t d_in,
  cccl_iterator_t d_out,
  unsigned long long num_items,
  cccl_op_t op,
  cccl_value_t init,
  hipStream_t stream) noexcept
{
  bool pushed    = false;
  hipError_t error = hipSuccess;
  try
  {
    pushed = try_push_context();

    hipDevice_t cu_device;
    check(hipCtxGetDevice(&cu_device));

    Invoke(
      d_temp_storage,
      *temp_storage_bytes,
      d_in,
      d_out,
      num_items,
      op,
      init,
      build.cc,
      build.single_tile_kernel,
      build.single_tile_second_kernel,
      build.reduction_kernel,
      cu_device,
      stream);
  }
  catch (...)
  {
    error = hipErrorUnknown;
  }

  if (pushed)
  {
    hipCtx_t dummy;
    hipCtxPopCurrent(&dummy);
  }

  return error;
}

extern "C" CCCL_C_API hipError_t cccl_device_reduce_cleanup(cccl_device_reduce_build_result_t* bld_ptr)
{
  try
  {
    if (bld_ptr == nullptr)
    {
      return hipErrorInvalidValue;
    }

    std::unique_ptr<char[]> cubin(reinterpret_cast<char*>(bld_ptr->cubin));
    check(hipLibraryUnload(bld_ptr->library));
  }
  catch (...)
  {
    return hipErrorUnknown;
  }

  return hipSuccess;
}
