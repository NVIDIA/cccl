// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

//! Simple example of hipcub::DeviceTopK::MinPairs().
//! Find the top-k smallest float keys paired with a corresponding array of int values.
//! To compile using the command line:
//!   nvcc -arch=sm_XX example_device_topk_pairs.cu -I../.. -lcudart -O3

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/device/device_topk.cuh>
#include <cub/util_allocator.cuh>

#include <thrust/host_vector.h>

#include <algorithm>
#include <cstdio>

#include "../../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and aliases
//---------------------------------------------------------------------

bool g_verbose = false; // Whether to display input/output to console
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

// Simple key-value pairing for floating point types.
// Treats positive and negative zero as equivalent.
struct Pair
{
  float key;
  int value;

  bool operator<(const Pair& b) const
  {
    bool res = key < b.key;
    if (key == b.key)
    {
      res = value < b.value;
    }
    return res;
  }
};

// Initialize key-value sorting problem.
void Initialize(float* h_keys, int* h_values, float* h_reference_keys, int* h_reference_values, int num_items, int k)
{
  Pair* h_pairs           = new Pair[num_items];
  Pair* h_reference_pairs = new Pair[k];

  for (int i = 0; i < num_items; ++i)
  {
    RandomBits(h_keys[i]);
    RandomBits(h_values[i]);
    h_pairs[i].key   = h_keys[i];
    h_pairs[i].value = h_values[i];
  }

  if (g_verbose)
  {
    printf("Input keys:\n");
    DisplayResults(h_keys, num_items);
    printf("\n\n");

    printf("Input values:\n");
    DisplayResults(h_values, num_items);
    printf("\n\n");
  }

  std::partial_sort_copy(h_pairs, h_pairs + num_items, h_reference_pairs, h_reference_pairs + k);

  for (int i = 0; i < k; ++i)
  {
    h_reference_keys[i]   = h_reference_pairs[i].key;
    h_reference_values[i] = h_reference_pairs[i].value;
  }

  delete[] h_pairs;
  delete[] h_reference_pairs;
}

//  In some case the results of topK is unordered. Sort the results to compare with ground truth.
void SortUnorderedRes(float* h_res_keys, float* d_keys_out, int* h_res_values, int* d_values_out, int k)
{
  CubDebugExit(hipMemcpy(h_res_keys, d_keys_out, sizeof(float) * k, hipMemcpyDeviceToHost));
  CubDebugExit(hipMemcpy(h_res_values, d_values_out, sizeof(int) * k, hipMemcpyDeviceToHost));
  Pair* h_res_pairs = new Pair[k];
  for (int i = 0; i < k; ++i)
  {
    h_res_pairs[i].key   = h_res_keys[i];
    h_res_pairs[i].value = h_res_values[i];
  }
  std::stable_sort(h_res_pairs, h_res_pairs + k);
  for (int i = 0; i < k; ++i)
  {
    h_res_keys[i]   = h_res_pairs[i].key;
    h_res_values[i] = h_res_pairs[i].value;
  }
}

//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
int main(int argc, char** argv)
{
  int num_items = 10240;
  int k         = 10;
  // Initialize command line
  CommandLineArgs args(argc, argv);
  g_verbose = args.CheckCmdLineFlag("v");
  args.GetCmdLineArgument("n", num_items);
  args.GetCmdLineArgument("k", k);
  // Print usage
  if (args.CheckCmdLineFlag("help"))
  {
    printf("%s "
           "[--n=<input items> "
           "[--k=<output items> "
           "[--device=<device-id>] "
           "[--v] "
           "\n",
           argv[0]);
    exit(0);
  }

  // Initialize device
  CubDebugExit(args.DeviceInit());

  printf("hipcub::DeviceTopK::MinPairs() find %d smallest items from %d items (%d-byte keys %d-byte values)\n",
         k,
         num_items,
         int(sizeof(float)),
         int(sizeof(int)));
  fflush(stdout);

  // Allocate host arrays
  thrust::host_vector<float> h_keys_vector(num_items);
  thrust::host_vector<float> h_reference_keys_vector(k);
  thrust::host_vector<float> h_res_keys_vector(k);
  thrust::host_vector<int> h_values_vector(num_items);
  thrust::host_vector<int> h_reference_values_vector(k);
  thrust::host_vector<int> h_res_values_vector(k);

  float* h_keys           = thrust::raw_pointer_cast(h_keys_vector.data());
  float* h_reference_keys = thrust::raw_pointer_cast(h_reference_keys_vector.data());
  float* h_res_keys       = thrust::raw_pointer_cast(h_res_keys_vector.data());
  int* h_values           = thrust::raw_pointer_cast(h_values_vector.data());
  int* h_reference_values = thrust::raw_pointer_cast(h_reference_values_vector.data());
  int* h_res_values       = thrust::raw_pointer_cast(h_res_values_vector.data());

  // Initialize problem and solution on host
  Initialize(h_keys, h_values, h_reference_keys, h_reference_values, num_items, k);

  // Allocate device arrays
  float* d_keys_in = nullptr;
  CubDebugExit(g_allocator.DeviceAllocate((void**) &d_keys_in, sizeof(float) * num_items));
  int* d_values_in = nullptr;
  CubDebugExit(g_allocator.DeviceAllocate((void**) &d_values_in, sizeof(int) * num_items));

  // Initialize device input
  CubDebugExit(hipMemcpy(d_keys_in, h_keys, sizeof(float) * num_items, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_values_in, h_values, sizeof(int) * num_items, hipMemcpyHostToDevice));

  // Allocate device output array and num selected
  float* d_keys_out = nullptr;
  int* d_values_out = nullptr;
  CubDebugExit(g_allocator.DeviceAllocate((void**) &d_keys_out, sizeof(float) * k));
  CubDebugExit(g_allocator.DeviceAllocate((void**) &d_values_out, sizeof(int) * k));

  // Allocate temporary storage
  size_t temp_storage_bytes = 0;
  void* d_temp_storage      = nullptr;

  // Specify that we do not require a specific output order and do not require deterministic results
  auto requirements =
    cuda::execution::require(cuda::execution::determinism::not_guaranteed, cuda::execution::output_ordering::unsorted);

  CubDebugExit(DeviceTopK::MinPairs(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, num_items, k, requirements));
  CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  // Initialize device arrays
  CubDebugExit(hipMemcpy(d_keys_in, h_keys, sizeof(float) * num_items, hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy(d_values_in, h_values, sizeof(int) * num_items, hipMemcpyHostToDevice));

  // Run
  CubDebugExit(DeviceTopK::MinPairs(
    d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, num_items, k, requirements));

  // Check for correctness (and display results, if specified)
  SortUnorderedRes(h_res_keys, d_keys_out, h_res_values, d_values_out, k);
  if (g_verbose)
  {
    printf("Output keys:\n");
    DisplayResults(h_res_keys, k);
    printf("\n\n");

    printf("Output values:\n");
    DisplayResults(h_res_values, k);
    printf("\n\n");
  }
  int compare = CompareResults(h_reference_keys, h_res_keys, k, g_verbose);
  AssertEquals(0, compare);
  compare = CompareResults(h_reference_values, h_res_values, k, g_verbose);
  AssertEquals(0, compare);

  // Cleanup
  if (d_keys_in)
  {
    CubDebugExit(g_allocator.DeviceFree(d_keys_in));
    d_keys_in = nullptr;
  }
  if (d_values_in)
  {
    CubDebugExit(g_allocator.DeviceFree(d_values_in));
    d_values_in = nullptr;
  }
  if (d_keys_out)
  {
    CubDebugExit(g_allocator.DeviceFree(d_keys_out));
    d_keys_out = nullptr;
  }
  if (d_values_out)
  {
    CubDebugExit(g_allocator.DeviceFree(d_values_out));
    d_values_out = nullptr;
  }
  if (d_temp_storage)
  {
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    d_temp_storage = nullptr;
  }

  printf("\n\n");

  return 0;
}
