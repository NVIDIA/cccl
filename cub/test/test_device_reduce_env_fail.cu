// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_reduce.cuh>

#include <cuda/__execution/determinism.h>
#include <cuda/__execution/require.h>

#include <iostream>

int main()
{
  namespace stdexec = cuda::std::execution;

  int* ptr{};
  auto env = stdexec::env{cuda::execution::determinism::run_to_run};

  // expected-error {{"Determinism should be used inside requires to have an effect."}}
  auto error = hipcub::DeviceReduce::Reduce(ptr, ptr, 0, cuda::std::plus<>{}, 0, env);
  if (error != hipSuccess)
  {
    std::cerr << "hipcub::DeviceReduce::Reduce failed with status: " << error << std::endl;
  }
}
