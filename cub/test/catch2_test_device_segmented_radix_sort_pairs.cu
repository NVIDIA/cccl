/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_segmented_radix_sort.cuh>
#include <cub/util_type.cuh>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/memory.h>
#include <thrust/scatter.h>

#include <algorithm>
#include <limits>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"
#include "thrust/detail/raw_pointer_cast.h"

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedRadixSort::SortPairs, sort_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedRadixSort::SortPairsDescending, sort_pairs_descending);

using custom_value_t = c2h::custom_type_t<c2h::equal_comparable_t>;
using value_types    = c2h::type_list<cuda::std::uint8_t, cuda::std::uint64_t, custom_value_t>;

// Index types used for OffsetsT testing
using offset_types = c2h::type_list<cuda::std::int32_t, cuda::std::uint64_t>;

CUB_TEST("DeviceSegmentedRadixSort::SortPairs: Basic testing",
         "[pairs][segmented][radix][sort][device]",
         value_types,
         offset_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = c2h::get<1, TestType>;

  constexpr std::size_t min_num_items = 1 << 5;
  constexpr std::size_t max_num_items = 1 << 20;
  const std::size_t num_items         = GENERATE_COPY(take(3, random(min_num_items, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(2, random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  c2h::device_vector<offset_t> offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(CUB_SEED(num_segment_seeds), offsets, static_cast<offset_t>(num_items));

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  REQUIRE(ref_keys == out_keys);
  REQUIRE(ref_values == out_values);
}

CUB_TEST("DeviceSegmentedRadixSort::SortPairs: DoubleBuffer API", "[pairs][segmented][radix][sort][device]", value_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = cuda::std::int32_t;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(1, random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  c2h::device_vector<offset_t> offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(CUB_SEED(num_segment_seeds), offsets, static_cast<offset_t>(num_items));

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  hipcub::DoubleBuffer<key_t> key_buffer(
    thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()));
  hipcub::DoubleBuffer<value_t> value_buffer(
    thrust::raw_pointer_cast(in_values.data()), thrust::raw_pointer_cast(out_values.data()));

  double_buffer_segmented_sort_t action(is_descending);
  action.initialize();
  launch(action,
         key_buffer,
         value_buffer,
         static_cast<int>(num_items),
         static_cast<int>(num_segments),
         // Mix pointers/iterators for segment info to test using different iterable types:
         thrust::raw_pointer_cast(offsets.data()),
         offsets.cbegin() + 1,
         begin_bit<key_t>(),
         end_bit<key_t>());

  key_buffer.selector   = action.selector();
  value_buffer.selector = action.selector();
  action.finalize();

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  auto& keys   = key_buffer.selector == 0 ? in_keys : out_keys;
  auto& values = value_buffer.selector == 0 ? in_values : out_values;

  REQUIRE(ref_keys == keys);
  REQUIRE(ref_values == values);
}

CUB_TEST("DeviceSegmentedRadixSort::SortPairs: unspecified ranges",
         "[pairs][segmented][radix][sort][device]",
         value_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = cuda::std::int32_t;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(1, random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  c2h::device_vector<offset_t> begin_offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(CUB_SEED(num_segment_seeds), begin_offsets, static_cast<offset_t>(num_items));

  // Create separate begin/end offsets arrays and remove some of the segments by
  // setting both offsets to 0.
  c2h::device_vector<offset_t> end_offsets(begin_offsets.cbegin() + 1, begin_offsets.cend());
  begin_offsets.pop_back();

  {
    std::size_t num_empty_segments = num_segments / 16;
    c2h::device_vector<std::size_t> indices(num_empty_segments);
    c2h::gen(CUB_SEED(1), indices, std::size_t{0}, num_segments - 1);
    auto begin = thrust::make_constant_iterator(key_t{0});
    auto end   = begin + num_empty_segments;
    thrust::scatter(c2h::device_policy, begin, end, indices.cbegin(), begin_offsets.begin());
    thrust::scatter(c2h::device_policy, begin, end, indices.cbegin(), end_offsets.begin());
  }

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(begin_offsets.data()),
      end_offsets.cbegin(),
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(begin_offsets.data()),
      end_offsets.cbegin(),
      begin_bit<key_t>(),
      end_bit<key_t>());
  }

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, begin_offsets, end_offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  REQUIRE((ref_keys == out_keys) == true);
  REQUIRE((ref_values == out_values) == true);
}
