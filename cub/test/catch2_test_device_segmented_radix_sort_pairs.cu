#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include "insert_nested_NVTX_range_guard.h"

#include <cub/device/device_segmented_radix_sort.cuh>
#include <cub/util_type.cuh>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/memory.h>
#include <thrust/scatter.h>

#include <algorithm>
#include <limits>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_segmented_sort_helper.cuh"
#include "catch2_test_launch_helper.h"
#include "thrust/detail/raw_pointer_cast.h"
#include <c2h/catch2_test_helper.h>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedRadixSort::SortPairs, sort_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedRadixSort::SortPairsDescending, sort_pairs_descending);

using custom_value_t = c2h::custom_type_t<c2h::equal_comparable_t>;
using value_types    = c2h::type_list<cuda::std::uint8_t, cuda::std::uint64_t, custom_value_t>;

// Index types used for OffsetsT testing
C2H_TEST("DeviceSegmentedRadixSort::SortPairs: Basic testing",
         "[pairs][segmented][radix][sort][device]",
         value_types,
         offset_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = c2h::get<1, TestType>;

  constexpr std::size_t min_num_items = 1 << 5;
  constexpr std::size_t max_num_items = 1 << 20;

  // Use c2h::adjust_seed_count to reduce runtime with sanitizers:
  const std::size_t num_items = GENERATE_COPY(take(c2h::adjust_seed_count(3), random(min_num_items, max_num_items)));
  const std::size_t num_segments =
    GENERATE_COPY(take(c2h::adjust_seed_count(2), random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(C2H_SEED(num_value_seeds), in_values);

  c2h::device_vector<offset_t> offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(C2H_SEED(num_segment_seeds), offsets, static_cast<offset_t>(num_items));

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  REQUIRE(ref_keys == out_keys);
  REQUIRE(ref_values == out_values);
}

C2H_TEST("DeviceSegmentedRadixSort::SortPairs: DoubleBuffer API", "[pairs][segmented][radix][sort][device]", value_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = cuda::std::int32_t;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(1, random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(C2H_SEED(num_value_seeds), in_values);

  c2h::device_vector<offset_t> offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(C2H_SEED(num_segment_seeds), offsets, static_cast<offset_t>(num_items));

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  hipcub::DoubleBuffer<key_t> key_buffer(
    thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()));
  hipcub::DoubleBuffer<value_t> value_buffer(
    thrust::raw_pointer_cast(in_values.data()), thrust::raw_pointer_cast(out_values.data()));

  double_buffer_segmented_sort_t action(is_descending);
  action.initialize();
  launch(action,
         key_buffer,
         value_buffer,
         static_cast<int>(num_items),
         static_cast<int>(num_segments),
         // Mix pointers/iterators for segment info to test using different iterable types:
         thrust::raw_pointer_cast(offsets.data()),
         offsets.cbegin() + 1,
         begin_bit<key_t>(),
         end_bit<key_t>());

  key_buffer.selector   = action.selector();
  value_buffer.selector = action.selector();
  action.finalize();

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  auto& keys   = key_buffer.selector == 0 ? in_keys : out_keys;
  auto& values = value_buffer.selector == 0 ? in_values : out_values;

  REQUIRE(ref_keys == keys);
  REQUIRE(ref_values == values);
}

C2H_TEST("DeviceSegmentedRadixSort::SortPairs: unspecified ranges",
         "[pairs][segmented][radix][sort][device]",
         value_types)
{
  using key_t    = cuda::std::uint32_t;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = cuda::std::int32_t;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(1, random(std::size_t{2}, num_items / 2)));

  c2h::device_vector<key_t> in_keys(num_items);
  const int num_key_seeds = 1;
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);

  c2h::device_vector<value_t> in_values(num_items);
  const int num_value_seeds = 1;
  c2h::gen(C2H_SEED(num_value_seeds), in_values);

  // Initialize the output vectors by copying the inputs since not all items
  // may belong to a segment.
  c2h::device_vector<key_t> out_keys(in_keys);
  c2h::device_vector<value_t> out_values(in_values);

  c2h::device_vector<offset_t> begin_offsets(num_segments + 1);
  const int num_segment_seeds = 1;
  generate_segment_offsets(C2H_SEED(num_segment_seeds), begin_offsets, static_cast<offset_t>(num_items));

  // Create separate begin/end offsets arrays and remove some of the segments by
  // setting both offsets to 0.
  c2h::device_vector<offset_t> end_offsets(begin_offsets.cbegin() + 1, begin_offsets.cend());
  begin_offsets.pop_back();

  {
    std::size_t num_empty_segments = num_segments / 16;
    c2h::device_vector<std::size_t> indices(num_empty_segments);
    c2h::gen(C2H_SEED(1), indices, std::size_t{0}, num_segments - 1);
    auto begin = thrust::make_constant_iterator(key_t{0});
    auto end   = begin + num_empty_segments;
    thrust::scatter(c2h::device_policy, begin, end, indices.cbegin(), begin_offsets.begin());
    thrust::scatter(c2h::device_policy, begin, end, indices.cbegin(), end_offsets.begin());
  }

  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, num_segments, is_descending);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(begin_offsets.data()),
      end_offsets.cbegin(),
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      static_cast<int>(num_items),
      static_cast<int>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(begin_offsets.data()),
      end_offsets.cbegin(),
      begin_bit<key_t>(),
      end_bit<key_t>());
  }

  auto refs        = segmented_radix_sort_reference(in_keys, in_values, is_descending, begin_offsets, end_offsets);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  REQUIRE((ref_keys == out_keys) == true);
  REQUIRE((ref_values == out_values) == true);
}

C2H_TEST("DeviceSegmentedSortPairs: very large num. items and num. segments",
         "[pairs][segmented][sort][device][skip-cs-initcheck][skip-cs-racecheck][skip-cs-synccheck]",
         all_offset_types)
try
{
  using key_t                        = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using value_t                      = cuda::std::uint8_t;
  using segment_offset_t             = std::int64_t;
  using offset_t                     = c2h::get<0, TestType>;
  using segment_iterator_t           = segment_index_to_offset_op<offset_t, segment_offset_t>;
  constexpr std::size_t segment_size = 1000000;
  constexpr std::size_t uint32_max   = cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : cuda::std::numeric_limits<offset_t>::max();
  constexpr segment_offset_t num_empty_segments = uint32_max;
  const segment_offset_t num_segments           = num_empty_segments + cuda::ceil_div(num_items, segment_size);
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  // Generate input
  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<value_t> in_values(num_items);
  constexpr auto max_histo_size = 250;
  segmented_verification_helper<key_t> verification_helper{max_histo_size};
  verification_helper.prepare_input_data(in_keys);
  thrust::copy(in_keys.cbegin(), in_keys.cend(), in_values.begin());

  // Initialize the output vectors by copying the inputs since not all items may belong to a segment.
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::device_vector<value_t> out_values(num_items);

  auto offsets = thrust::make_transform_iterator(
    thrust::make_counting_iterator(std::size_t{0}),
    segment_iterator_t{num_empty_segments, num_segments, segment_size, num_items});
  auto offsets_plus_1 = offsets + 1;

  sort_pairs(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    thrust::raw_pointer_cast(in_values.data()),
    thrust::raw_pointer_cast(out_values.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    offsets,
    offsets_plus_1,
    begin_bit<key_t>(),
    end_bit<key_t>());

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys, offsets + num_empty_segments, num_segments - num_empty_segments);

  // Verify values were sorted along with the keys
  REQUIRE(thrust::equal(out_keys.cbegin(), out_keys.cend(), out_values.cbegin()));
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}

// Currently, size of a single segment in DeviceRadixSort is limited to INT_MAX
#if defined(CCCL_TEST_ENABLE_LARGE_SEGMENTED_SORT)
C2H_TEST("DeviceSegmentedSort::SortPairs: very large segments",
         "[pairs][segmented][sort][device][skip-cs-initcheck][skip-cs-racecheck][skip-cs-synccheck]",
         all_offset_types)
try
{
  using key_t                      = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using value_t                    = cuda::std::uint8_t;
  using segment_offset_t           = std::int32_t;
  using offset_t                   = c2h::get<0, TestType>;
  constexpr std::size_t uint32_max = cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr int num_key_seeds      = 1;
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : cuda::std::numeric_limits<offset_t>::max();
  constexpr segment_offset_t num_segments = 2;
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<value_t> in_values(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  thrust::copy(in_keys.cbegin(), in_keys.cend(), in_values.begin());
  c2h::device_vector<value_t> out_values(num_items);
  c2h::device_vector<offset_t> offsets(num_segments + 1);
  offsets[0] = 0;
  offsets[1] = static_cast<offset_t>(num_items);
  offsets[2] = static_cast<offset_t>(num_items);

  // Prepare information for later verification
  short_key_verification_helper<key_t> verification_helper{};
  verification_helper.prepare_verification_data(in_keys);

  sort_pairs(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    thrust::raw_pointer_cast(in_values.data()),
    thrust::raw_pointer_cast(out_values.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    thrust::raw_pointer_cast(offsets.data()),
    offsets.cbegin() + 1,
    begin_bit<key_t>(),
    end_bit<key_t>());

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys);

  // Verify values were sorted along with the keys
  REQUIRE(thrust::equal(out_keys.cbegin(), out_keys.cend(), out_values.cbegin()));
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}
#endif // defined(CCCL_TEST_ENABLE_LARGE_SEGMENTED_SORT)
