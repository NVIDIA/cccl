#define NVTX3_CPP_REQUIRE_EXPLICIT_VERSION
#include <cub/device/device_for.cuh> // internal include of NVTX

#include <cuda/iterator>
#include <cuda/std/functional>

#include <nvtx3/nvtx3.hpp> // user-side include of NVTX, retrieved elsewhere

int main()
{
  nvtx3::v1::scoped_range range("user-range"); // user-side use of explicit NVTX API

  cuda::counting_iterator<int> it{0};
  hipcub::DeviceFor::ForEach(it, it + 16, ::cuda::std::negate<int>{}); // internal use of NVTX
  hipDeviceSynchronize();
}
