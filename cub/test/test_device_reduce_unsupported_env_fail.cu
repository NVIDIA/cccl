// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_reduce.cuh>

#include <cuda/__execution/determinism.h>
#include <cuda/__execution/require.h>
#include <cuda/std/complex>

int main()
{
  namespace stdexec = cuda::std::execution;

  cuda::std::complex<float>* ptr{};
  auto env = cuda::execution::require(cuda::execution::determinism::gpu_to_gpu);

  // expected-error {{"gpu_to_gpu determinism is only supported for integral types, or float and double types with
  // ::cuda::std::plus operator, or any floating point types with ::cuda::minimum<> or ::cuda::maximum<> operators."}}
  hipcub::DeviceReduce::Reduce(ptr, ptr, 0, cuda::std::plus<float>{}, cuda::std::complex<float>{}, env);
}
