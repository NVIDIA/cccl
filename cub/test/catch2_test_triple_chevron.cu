#include "hip/hip_runtime.h"
#include <thrust/count.h>
#include <cub/detail/triple_chevron_launch.cuh>
#include <cuda/std/tuple>
#include <cstdio>  // For printf

// Has to go after all cub headers. Otherwise, this test won't catch unused
// variables in cub kernels.
#include "catch2_test_cdp_helper.h"
#include "catch2_test_helper.h"

// %PARAM% TEST_CDP cdp 0:1

template <class T>
__global__ void mult_two_kernel(const T *d_in, T *d_out, int num_items)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(blockIdx.x == 0 && threadIdx.x == 0) {
        printf("Inside kernel: d_in[%d] = %d, d_out[%d] = %d\n", i, d_in[i], i, d_out[i]);
    }

    if (i < num_items)
    {
        d_out[i] = d_in[i] * T{2};
        if(blockIdx.x == 0 && threadIdx.x == 0) {
            printf("After computation: d_out[%d] = %d\n", i, d_out[i]);
        }
    }
}


struct cdp_chevron_invoker { 
  static constexpr int threads_in_block = 256;

  template <class T, class KernelT>
  HIPCUB_RUNTIME_FUNCTION static hipError_t invoke(std::uint8_t *d_temp_storage,
                                                 std::size_t &temp_storage_bytes,
                                                 KernelT kernel,
                                                 const T *d_in,
                                                 T *d_out,
                                                 int num_items,
                                                 bool on_device)
  {
    NV_IF_TARGET(NV_IS_HOST,
                 (if (on_device) { return hipErrorLaunchFailure; }),
                 (if (!on_device) { return hipErrorLaunchFailure; }));

    if (d_temp_storage == nullptr)
    {
      temp_storage_bytes = static_cast<std::size_t>(num_items);
      return hipSuccess;
    }
  
    if (temp_storage_bytes != static_cast<std::size_t>(num_items))
    {
      return hipErrorInvalidValue;
    }

    const int blocks_in_grid = (num_items + threads_in_block - 1) / threads_in_block;

    return hipcub::detail::triple_chevron(blocks_in_grid, threads_in_block, 0, 0)
      .doit(kernel, d_in, d_out, num_items);
  }

  template <class T>
  HIPCUB_RUNTIME_FUNCTION static hipError_t create(std::uint8_t *d_temp_storage,
                                               std::size_t &temp_storage_bytes,
                                               const T *d_in,
                                               T *d_out,
                                               int num_items,
                                               bool device_invoke)
  {
    return invoke(d_temp_storage,
                  temp_storage_bytes,
                  mult_two_kernel<T>,
                  d_in,
                  d_out,
                  num_items,
                  device_invoke);
  }
};

struct cdp_invocable { 

  template <class T>
  HIPCUB_RUNTIME_FUNCTION hipError_t operator()(uint8_t* temp, size_t bytes, T d_in, T d_out, int n, bool on_device) const {
       return cdp_chevron_invoker::create(
          temp,
          bytes, 
          d_in, 
          d_out,
          n, 
          on_device
       );
  }
};


__global__ void add_kernel(int a, float b, double* out) {
    *out = a + b;
}

CUB_TEST("CDP wrapper works with custom invocables and cdp_launch, on both host and device", "[test][utils]")
{
  int n = 42;
  thrust::device_vector<int> in(n, 21);
  thrust::device_vector<int> out(n);

  int *d_in  = thrust::raw_pointer_cast(in.data());
  int *d_out = thrust::raw_pointer_cast(out.data());

  constexpr bool on_device = TEST_CDP;

  {
    cdp_launch(cdp_invocable{}, d_in, d_out, n, on_device);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    std::vector<int> h_out(n);
    thrust::copy(out.begin(), out.end(), h_out.begin());
    for(int i = 0; i < std::min(5, n); ++i) {
        printf("h_out[%d] = %d\n", i, h_out[i]);
    }
    const auto actual   = static_cast<std::size_t>(thrust::count(out.begin(), out.end(), 42));
    printf("Thrust count result: %zu\n", actual);
    const auto expected = static_cast<std::size_t>(n);

    REQUIRE(actual == expected);
  }


}


CUB_TEST("Rough draft of testing Chevron launches successfully ", "[test][utils]") {
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  
  hipEvent_t event;
  hipEventCreate(&event);

  int n = 1000;
  thrust::device_vector<int> in(n, 5);
  thrust::device_vector<int> out(n);

  int *d_in  = thrust::raw_pointer_cast(in.data());
  int *d_out = thrust::raw_pointer_cast(out.data());

  hipMemcpyAsync(d_out, d_in, n * sizeof(float), hipMemcpyHostToDevice, stream1); 
  hipEventRecord(event, stream1); 

  const int block_size = 256;
  const int grid_size = (n * block_size - 1) / block_size;
  auto chev = hipcub::detail::triple_chevron(grid_size, block_size, 0, stream2);

  hipStreamWaitEvent(stream2, event, 0); // sync streams
  chev.doit(mult_two_kernel<int>, d_in, d_out, n);

  {
    const auto actual   = static_cast<std::size_t>(thrust::count(out.begin(), out.end(), 10));
    const auto expected   = static_cast<std::size_t>(n);
    REQUIRE(actual==expected);
  }
  hipEventDestroy(event);

}

CUB_TEST("Triple Chevron with missing configuration returns hipErrorMissingConfiguration", "[test][utils]") {
  int n = 42;
  thrust::device_vector<int> in(n, 21);
  thrust::device_vector<int> out(n);
  int *d_in  = thrust::raw_pointer_cast(in.data());
  int *d_out = thrust::raw_pointer_cast(out.data());

  auto chevron = hipcub::detail::triple_chevron(0, 0);
  auto err = chevron.doit(mult_two_kernel<int>, d_in, d_out, n);
  REQUIRE( HipcubDebug(hipErrorMissingConfiguration) == hipErrorMissingConfiguration );
}

CUB_TEST("Triple Chevron respects required dynamic shared memory allocation", "[test][utils]") {
  int n = 42;
  thrust::device_vector<int> in(n, 21);
  thrust::device_vector<int> out(n);
  int *d_in  = thrust::raw_pointer_cast(in.data());
  int *d_out = thrust::raw_pointer_cast(out.data());

  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
  auto cap = deviceProperties.sharedMemPerBlock;

  const int block_size = 2566666;
  const int grid_size = (n * block_size - 1) / block_size;
  auto chevron = hipcub::detail::triple_chevron(grid_size, block_size, cap); 
  auto err = chevron.doit(mult_two_kernel<int>, d_in, d_out, n);
  REQUIRE( err == hipSuccess );
}

CUB_TEST("Triple Chevron properly forwards parameters", "[test][utils]") {
  double result;
  double *d_result;

  hipMalloc(&d_result, sizeof(double));
  auto chev = hipcub::detail::triple_chevron(1, 1); 
  chev.doit(add_kernel, 5, 3.5f, d_result);
  hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);

  REQUIRE(result == 8.5f);
}