#include <cub/util_debug.cuh>
#include <cub/util_device.cuh>

#include "catch2_test_helper.h"

TEST_CASE("HipcubDebug returns input error", "[debug][utils]")
{
  REQUIRE(HipcubDebug(hipSuccess) == hipSuccess);
  REQUIRE(HipcubDebug(hipErrorInvalidConfiguration) == hipErrorInvalidConfiguration);
}

TEST_CASE("HipcubDebug returns new errors", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) != hipSuccess);
}

TEST_CASE("HipcubDebug prefers input errors", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipErrorOutOfMemory) != hipSuccess);
}

TEST_CASE("HipcubDebug resets last error", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) == hipSuccess);
}
