#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_find_if.cuh>
#include <cub/thread/thread_operators.cuh>
#include <cub/util_namespace.cuh>
#include <cub/util_type.cuh>

#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include <algorithm>
#include <cstdint>
#include <iostream>

#include "c2h/custom_type.cuh"
#include "catch2_test_device_reduce.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"
#include <nv/target>

// %PARAM% TEST_LAUNCH lid 0:1

// DECLARE_LAUNCH_WRAPPER(hipcub::DeviceFind::FindIf, device_findif);

// List of types to test
using custom_t =
  c2h::custom_type_t<c2h::accumulateable_t,
                     c2h::equal_comparable_t,
                     c2h::lexicographical_less_comparable_t,
                     c2h::lexicographical_greater_comparable_t>;

using full_type_list = c2h::type_list<type_pair<std::uint8_t, std::int32_t>, type_pair<std::int8_t>>;
// clang-format on

enum class gen_data_t : int
{
  /// Uniform random data generation
  GEN_TYPE_RANDOM,
  /// Constant value as input data
  GEN_TYPE_CONST
};

template <typename InputIt, typename OutputIt, typename BinaryOp>
void compute_find_if_reference(InputIt first, InputIt last, OutputIt& result, BinaryOp op)
{
  auto pos = thrust::find_if(first, last, op);
  result   = pos - first;
}

template <typename T>
struct equals_2
{
  __device__ __host__ bool operator()(T i)
  {
    return i == 2;
  }
};

CUB_TEST("Device find if works", "[device]", full_type_list)
{
  using params   = params_t<TestType>;
  using input_t  = typename params::item_t;
  using output_t = typename params::output_t;
  using offset_t = int32_t;

  constexpr offset_t min_items = 1;
  constexpr offset_t max_items = 1000000;

  // Generate the input sizes to test for
  const offset_t num_items = GENERATE_COPY(
    take(3, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));

  // Input data generation to test
  const gen_data_t data_gen_mode = GENERATE_COPY(gen_data_t::GEN_TYPE_RANDOM, gen_data_t::GEN_TYPE_CONST);

  // Generate input data
  c2h::device_vector<input_t> in_items(num_items);
  if (data_gen_mode == gen_data_t::GEN_TYPE_RANDOM)
  {
    c2h::gen(CUB_SEED(2), in_items);
  }
  else
  {
    input_t default_constant{};
    init_default_constant(default_constant);
    thrust::fill(c2h::device_policy, in_items.begin(), in_items.end(), default_constant);
  }
  auto d_in_it = thrust::raw_pointer_cast(in_items.data());

  SECTION("find if")
  {
    using op_t = equals_2<std::int32_t>;

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(1);
    compute_find_if_reference(host_items.begin(), host_items.end(), expected_result[0], op_t{});

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes{};

    // Run test
    c2h::device_vector<output_t> out_result(1);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());

    hipcub::DeviceFind::FindIf(
      d_temp_storage, temp_storage_bytes, unwrap_it(d_in_it), unwrap_it(d_out_it), op_t{}, num_items);

    thrust::device_vector<uint8_t> temp_storage(temp_storage_bytes);
    d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

    hipcub::DeviceFind::FindIf(
      d_temp_storage, temp_storage_bytes, unwrap_it(d_in_it), unwrap_it(d_out_it), op_t{}, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);
  }
}
