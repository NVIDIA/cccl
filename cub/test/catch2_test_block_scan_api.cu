#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/block/block_scan.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cuda/std/numeric>

#include <c2h/catch2_test_helper.h>

constexpr int num_items_per_thread = 2;
constexpr int block_num_threads    = 64;

// example-begin inclusive-scan-array-init-value
__global__ void InclusiveBlockScanKernel(int* output)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * 2), // item 0
    -1 * ((int) threadIdx.x * 2 + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}

  // Collectively compute the block-wide inclusive scan max
  block_scan_t(temp_storage).InclusiveScan(thread_data, thread_data, initial_value, cuda::maximum<>{});

  // output: {[1, 1], [2, 2], [4, 4], ... [126, 126]}
  // ...
  // example-end inclusive-scan-array-init-value
  output[threadIdx.x * 2]     = thread_data[0];
  output[threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based inclusive scan works with initial value", "[scan][block]")
{
  c2h::device_vector<int> d_out(block_num_threads * num_items_per_thread);

  InclusiveBlockScanKernel<<<1, block_num_threads>>>(thrust::raw_pointer_cast(d_out.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  for (size_t i = 0; i < expected.size() - 1; i += 2)
  {
    expected[i]     = static_cast<int>(i);
    expected[i + 1] = static_cast<int>(i);
  }

  // When initial value = 1 for the given input the first two
  // elements of the result are equal to 1.
  expected[0] = 1;
  expected[1] = 1;

  REQUIRE(expected == d_out);
}

// example-begin inclusive-scan-array-aggregate-init-value
__global__ void InclusiveBlockScanKernelAggregate(int* output, int* d_block_aggregate)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * 2), // item 0
    -1 * ((int) threadIdx.x * 2 + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}

  // Collectively compute the block-wide inclusive scan max
  int block_aggregate;
  block_scan_t(temp_storage).InclusiveScan(thread_data, thread_data, initial_value, cuda::maximum<>{}, block_aggregate);

  // output: {[1, 1], [2, 2], [4, 4], ... [126, 126]}
  // block_aggregate = 126;
  // ...
  // example-end inclusive-scan-array-aggregate-init-value

  *d_block_aggregate          = block_aggregate;
  output[threadIdx.x * 2]     = thread_data[0];
  output[threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based inclusive scan with block aggregate works with initial value", "[scan][block]")
{
  c2h::device_vector<int> d_out(block_num_threads * num_items_per_thread);

  c2h::device_vector<int> d_block_aggregate(1);
  InclusiveBlockScanKernelAggregate<<<1, block_num_threads>>>(
    thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_block_aggregate.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  for (size_t i = 0; i < expected.size() - 1; i += 2)
  {
    expected[i]     = static_cast<int>(i);
    expected[i + 1] = static_cast<int>(i);
  }

  // When initial value = 1 for the given input the first two
  // elements of the result are equal to 1.
  expected[0] = 1;
  expected[1] = 1;

  REQUIRE(d_out == expected);
  REQUIRE(d_block_aggregate[0] == 126);
}

constexpr int num_blocks = 3;

// example-begin inclusive-scan-partial-tile-array-init-value
__global__ void InclusiveBlockScanPartialTileKernel(int* output)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * 2), // item 0
    -1 * ((int) threadIdx.x * 2 + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}
  int valid_items = -7 + (int) blockIdx.x * 69;
  // 1st block: -7 (effectively 0); 2nd block: 62; 3rd block: 131 (effectively 128)

  // Collectively compute the block-wide inclusive scan max
  block_scan_t(temp_storage)
    .InclusiveScanPartialTile(thread_data, thread_data, initial_value, cuda::maximum<>{}, valid_items);

  // 1st block output: {[0, -1], [2, -3], [4, -5], ...                           [126, -127]}
  // 2nd block output: {[1,  1], [2,  2], [4,  4], ... [60, 60], [62, -63], ..., [126, -127]}
  // 3rd block output: {[1,  1], [2,  2], [4,  4], ...                           [126,  126]}
  // ...
  // example-end inclusive-scan-partial-tile-array-init-value
  const int block_offset                     = block_num_threads * num_items_per_thread * blockIdx.x;
  output[block_offset + threadIdx.x * 2]     = thread_data[0];
  output[block_offset + threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based partial inclusive scan works with initial value", "[scan][block]")
{
  c2h::device_vector<int> d_out(num_blocks * block_num_threads * num_items_per_thread);

  InclusiveBlockScanPartialTileKernel<<<num_blocks, block_num_threads>>>(thrust::raw_pointer_cast(d_out.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  for (int block = 0; block < num_blocks; ++block)
  {
    constexpr int num_items_per_block = block_num_threads * num_items_per_thread;
    const int block_offset            = num_items_per_block * block;
    const int valid_items             = -7 + block * 69;
    const int bounded_valid_items     = cuda::std::clamp(valid_items, 0, block_num_threads * num_items_per_thread);

    for (int i = 0; i < bounded_valid_items; ++i)
    {
      expected[block_offset + i] = cuda::std::max(cuda::round_down(i, 2), 1);
    }
    for (int i = bounded_valid_items; i < num_items_per_block; ++i)
    {
      expected[block_offset + i] = (i % 2 == 0) ? i : -i;
    }
  }

  REQUIRE(expected == d_out);
}

// example-begin inclusive-scan-partial-tile-array-aggregate-init-value
__global__ void InclusiveBlockScanPartialTileKernelAggregate(int* output, int* d_block_aggregate)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * 2), // item 0
    -1 * ((int) threadIdx.x * 2 + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}
  int valid_items = -7 + (int) blockIdx.x * 69;
  // 1st block: -7 (effectively 0); 2nd block: 62; 3rd block: 131 (effectively 128)

  // Collectively compute the block-wide inclusive scan max
  int block_aggregate;
  block_scan_t(temp_storage)
    .InclusiveScanPartialTile(thread_data, thread_data, initial_value, cuda::maximum<>{}, valid_items, block_aggregate);

  // 1st block output: {[0, -1], [2, -3], [4, -5], ...                          [126, -127]}; block_aggregate:   ?
  // 2nd block output: {[1,  1], [2,  2], [4,  4], ... [60, 60], [62, -63], ... [126, -127]}; block_aggergate:  60
  // 3rd block output: {[1,  1], [2,  2], [4,  4], ...                          [126,  126]}; block_aggregate: 126
  // ...
  // example-end inclusive-scan-partial-tile-array-aggregate-init-value

  d_block_aggregate[blockIdx.x]              = block_aggregate;
  const int block_offset                     = block_num_threads * num_items_per_thread * blockIdx.x;
  output[block_offset + threadIdx.x * 2]     = thread_data[0];
  output[block_offset + threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based partial inclusive scan with block aggregate works with initial value", "[scan][block]")
{
  c2h::device_vector<int> d_out(num_blocks * block_num_threads * num_items_per_thread);

  c2h::device_vector<int> d_block_aggregate(num_blocks);
  InclusiveBlockScanPartialTileKernelAggregate<<<num_blocks, block_num_threads>>>(
    thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_block_aggregate.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  c2h::host_vector<int> expected_agg(d_block_aggregate.size());
  for (int block = 0; block < num_blocks; ++block)
  {
    constexpr int num_items_per_block = block_num_threads * num_items_per_thread;
    const int block_offset            = num_items_per_block * block;
    const int valid_items             = -7 + block * 69;
    const int bounded_valid_items     = cuda::std::clamp(valid_items, 0, block_num_threads * num_items_per_thread);

    for (int i = 0; i < bounded_valid_items; ++i)
    {
      expected[block_offset + i] = cuda::std::max(cuda::round_down(i, 2), 1);
    }
    for (int i = bounded_valid_items; i < num_items_per_block; ++i)
    {
      expected[block_offset + i] = (i % 2 == 0) ? i : -i;
    }

    if (valid_items > 0)
    {
      expected_agg[block] = expected[block_offset + bounded_valid_items - 1];
    }
    else
    {
      // Undefined
      expected_agg[block] = d_block_aggregate[block];
    }
  }

  REQUIRE(d_out == expected);
  REQUIRE(d_block_aggregate == expected_agg);
}
