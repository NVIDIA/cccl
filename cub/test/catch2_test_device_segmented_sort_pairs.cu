/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <catch2_segmented_sort_helper.cuh>
#include <catch2_test_helper.h>

// FIXME: Graph launch disabled, algorithm syncs internally. WAR exists for device-launch, figure out how to enable for
// graph launch.

// %PARAM% TEST_LAUNCH lid 0:1

using pair_types =
  c2h::type_list<c2h::type_list<bool, std::uint8_t>,
                 c2h::type_list<std::int8_t, std::uint64_t>,
                 c2h::type_list<double, float>
#if TEST_HALF_T
                 ,
                 c2h::type_list<half_t, std::int8_t>
#endif
#if TEST_BF_T
                 ,
                 c2h::type_list<bfloat16_t, float>
#endif
                 >;

CUB_TEST("DeviceSegmentedSortPairs: No segments", "[pairs][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT   = std::uint8_t;
  using ValueT = std::uint8_t;

  const bool stable_sort     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    stable_sort,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    int{},
    int{},
    nullptr,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

CUB_TEST("DeviceSegmentedSortPairs: Empty segments", "[pairs][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT   = std::uint8_t;
  using ValueT = std::uint8_t;

  const int num_segments     = GENERATE(take(2, random(1 << 2, 1 << 22)));
  const bool sort_stable     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  c2h::device_vector<int> offsets(num_segments + 1, int{});
  const int* d_offsets = thrust::raw_pointer_cast(offsets.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    sort_stable,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    int{},
    num_segments,
    d_offsets,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

CUB_TEST("DeviceSegmentedSortPairs: Same size segments, derived keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int segment_size = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)));

  const int segments = GENERATE_COPY(take(2, random(1 << 0, 1 << 5)), //
                                     take(2, random(1 << 5, 1 << 10)));

  test_same_size_segments_derived<KeyT, ValueT>(segment_size, segments);
}

CUB_TEST("DeviceSegmentedSortPairs: Randomly sized segments, derived keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)),
    take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_derived<KeyT, ValueT>(CUB_SEED(1), max_items, max_segment, segments);
}

CUB_TEST("DeviceSegmentedSortPairs: Randomly sized segments, random keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_random<KeyT, ValueT>(CUB_SEED(1), max_items, max_segment, segments);
}

CUB_TEST("DeviceSegmentedSortPairs: Edge case segments, random keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  test_edge_case_segments_random<KeyT, ValueT>(CUB_SEED(4));
}

CUB_TEST("DeviceSegmentedSortPairs: Unspecified segments, random key/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  test_unspecified_segments_random<KeyT, ValueT>(CUB_SEED(4));
}
