#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include "catch2_radix_sort_helper.cuh"
// above header needs to be included first

#include "catch2_segmented_sort_helper.cuh"
#include <c2h/catch2_test_helper.h>

// FIXME: Graph launch disabled, algorithm syncs internally. WAR exists for device-launch, figure out how to enable for
// graph launch.
// %PARAM% TEST_LAUNCH lid 0:1

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedSort::StableSortPairs, stable_sort_pairs);

using pair_types =
  c2h::type_list<c2h::type_list<bool, std::uint8_t>,
                 c2h::type_list<std::int8_t, std::uint64_t>,
                 c2h::type_list<double, float>
#if TEST_HALF_T
                 ,
                 c2h::type_list<half_t, std::int8_t>
#endif
#if TEST_BF_T
                 ,
                 c2h::type_list<bfloat16_t, float>
#endif
                 >;

C2H_TEST("DeviceSegmentedSortPairs: No segments", "[pairs][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT   = std::uint8_t;
  using ValueT = std::uint8_t;

  const bool stable_sort     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    stable_sort,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    int{},
    int{},
    nullptr,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortPairs: Empty segments", "[pairs][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT   = std::uint8_t;
  using ValueT = std::uint8_t;

  const int num_segments     = GENERATE(take(2, random(1 << 2, 1 << 22)));
  const bool sort_stable     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  c2h::device_vector<int> offsets(num_segments + 1, int{});
  const int* d_offsets = thrust::raw_pointer_cast(offsets.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<ValueT> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    sort_stable,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    static_cast<ValueT*>(nullptr),
    int{},
    num_segments,
    d_offsets,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortPairs: Same size segments, derived keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int segment_size = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)));

  const int segments = GENERATE_COPY(take(2, random(1 << 0, 1 << 5)), //
                                     take(2, random(1 << 5, 1 << 10)));

  test_same_size_segments_derived<KeyT, ValueT>(segment_size, segments);
}

C2H_TEST("DeviceSegmentedSortPairs: Randomly sized segments, derived keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)),
    take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_derived<KeyT, ValueT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortPairs: Randomly sized segments, random keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_random<KeyT, ValueT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortPairs: Edge case segments, random keys/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  test_edge_case_segments_random<KeyT, ValueT>(C2H_SEED(4));
}

C2H_TEST("DeviceSegmentedSortPairs: Unspecified segments, random key/values",
         "[pairs][segmented][sort][device]",
         pair_types)
{
  using PairT  = c2h::get<0, TestType>;
  using KeyT   = c2h::get<0, PairT>;
  using ValueT = c2h::get<1, PairT>;

  test_unspecified_segments_random<KeyT, ValueT>(C2H_SEED(4));
}

C2H_TEST("DeviceSegmentedSortPairs: very large num. items and num. segments",
         "[pairs][segmented][sort][device]",
         all_offset_types)
try
{
  using key_t                        = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using value_t                      = cuda::std::uint8_t;
  using segment_offset_t             = std::int64_t;
  using offset_t                     = c2h::get<0, TestType>;
  using segment_iterator_t           = segment_index_to_offset_op<offset_t, segment_offset_t>;
  constexpr std::size_t segment_size = 1000000;
  constexpr std::size_t uint32_max   = ::cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : ::cuda::std::numeric_limits<offset_t>::max();
  constexpr segment_offset_t num_empty_segments = uint32_max;
  const segment_offset_t num_segments           = num_empty_segments + ::cuda::ceil_div(num_items, segment_size);
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  // Generate input
  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<value_t> in_values(num_items);
  constexpr auto max_histo_size = 250;
  segmented_verification_helper<key_t> verification_helper{max_histo_size};
  verification_helper.prepare_input_data(in_keys);
  thrust::copy(in_keys.cbegin(), in_keys.cend(), in_values.begin());

  // Initialize the output vectors by copying the inputs since not all items may belong to a segment.
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::device_vector<value_t> out_values(num_items);

  auto offsets = thrust::make_transform_iterator(
    thrust::make_counting_iterator(std::size_t{0}),
    segment_iterator_t{num_empty_segments, num_segments, segment_size, num_items});
  auto offsets_plus_1 = offsets + 1;

  stable_sort_pairs(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    thrust::raw_pointer_cast(in_values.data()),
    thrust::raw_pointer_cast(out_values.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    offsets,
    offsets_plus_1);

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys, offsets + num_empty_segments, num_segments - num_empty_segments);

  // Verify values were sorted along with the keys
  REQUIRE(thrust::equal(out_keys.cbegin(), out_keys.cend(), out_values.cbegin()));
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}

C2H_TEST("DeviceSegmentedSort::SortPairs: very large segments", "[pairs][segmented][sort][device]", all_offset_types)
try
{
  using key_t                      = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using value_t                    = cuda::std::uint8_t;
  using segment_offset_t           = std::int32_t;
  using offset_t                   = c2h::get<0, TestType>;
  constexpr std::size_t uint32_max = ::cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr int num_key_seeds      = 1;
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : ::cuda::std::numeric_limits<offset_t>::max();
  constexpr segment_offset_t num_segments = 2;
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<value_t> in_values(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  thrust::copy(in_keys.cbegin(), in_keys.cend(), in_values.begin());
  c2h::device_vector<value_t> out_values(num_items);
  c2h::device_vector<offset_t> offsets(num_segments + 1);
  offsets[0] = 0;
  offsets[1] = static_cast<offset_t>(num_items);
  offsets[2] = static_cast<offset_t>(num_items);

  // Prepare information for later verification
  short_key_verification_helper<key_t> verification_helper{};
  verification_helper.prepare_verification_data(in_keys);

  stable_sort_pairs(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    thrust::raw_pointer_cast(in_values.data()),
    thrust::raw_pointer_cast(out_values.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    thrust::raw_pointer_cast(offsets.data()),
    offsets.cbegin() + 1);

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys);

  // Verify values were sorted along with the keys
  REQUIRE(thrust::equal(out_keys.cbegin(), out_keys.cend(), out_values.cbegin()));
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}
