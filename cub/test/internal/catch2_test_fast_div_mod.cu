#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include <cub/config.cuh>

#if __cccl_lib_mdspan

#  include "c2h/catch2_test_helper.h"
#  include "c2h/utility.h"
#  include "catch2_test_launch_helper.h"

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceFor::ForEachInExtents, device_for_each_in_extents);

/***********************************************************************************************************************
 * TEST CASES
 **********************************************************************************************************************/

using index_types =
  c2h::type_list<int8_t,
                 uint8_t,
                 int16_t,
                 uint16_t,
                 int32_t,
                 uint32_t
#  if CUB_IS_INT128_ENABLED
                 ,
                 int64_t,
                 uint64_t
#  endif
                 >;

C2H_TEST("FastDivMod", "[FastDivMod][Random]", index_types)
{
  using hipcub::detail::fast_div_mod;
  using index_type         = c2h::get<0, TestType>;
  constexpr auto max_value = cuda::std::numeric_limits<index_type>::max();
  auto divisor             = GENERATE(take(100, random(index_type{1}, max_value)));
  auto dividend            = GENERATE(take(100, random(index_type{1}, max_value)));
  fast_div_mod<index_type> div_mod(divisor);
  static_cast<void>(div_mod(dividend));
}

C2H_TEST("FastDivMod", "[FastDivMod][EdgeCases]", index_types)
{
  using hipcub::detail::fast_div_mod;
  using index_type         = c2h::get<0, TestType>;
  constexpr auto max_value = cuda::std::numeric_limits<index_type>::max();
  // divisor/dividend == max
  fast_div_mod<index_type> div_mod_max(max_value);
  static_cast<void>(div_mod_max(max_value));
  // divisor == 0, /dividend == max
  fast_div_mod<index_type> div_mod_min(max_value);
  static_cast<void>(div_mod_min(10));
}

#endif // __cccl_lib_mdspan
