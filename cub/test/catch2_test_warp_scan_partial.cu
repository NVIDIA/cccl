#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

#include <cub/util_arch.cuh>
#include <cub/util_macro.cuh>
#include <cub/warp/warp_scan.cuh>

#include <cuda/cmath>

#include "thread_reduce/catch2_test_thread_reduce_helper.cuh"
#include <c2h/catch2_test_helper.h>

constexpr int num_seeds = 3;

template <int LOGICAL_WARP_THREADS, int TOTAL_WARPS, class T, class ActionT>
__global__ void
warp_combine_scan_kernel(T* in, T* inclusive_out, T* exclusive_out, ActionT action, int valid_items, T filler)
{
  using warp_scan_t = hipcub::WarpScan<T, LOGICAL_WARP_THREADS>;
  using storage_t   = typename warp_scan_t::TempStorage;

  __shared__ storage_t storage[TOTAL_WARPS];

  const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

  // Get warp index
  int warp_id = tid / LOGICAL_WARP_THREADS;

  T inc_out     = filler;
  T exc_out     = filler;
  T thread_data = in[tid];

  warp_scan_t scan(storage[warp_id]);

  action(scan, thread_data, inc_out, exc_out, valid_items);

  inclusive_out[tid] = inc_out;
  exclusive_out[tid] = exc_out;
}

template <int LOGICAL_WARP_THREADS, int TOTAL_WARPS, class T, class ActionT>
void warp_combine_scan(
  c2h::device_vector<T>& in,
  c2h::device_vector<T>& inclusive_out,
  c2h::device_vector<T>& exclusive_out,
  ActionT action,
  int valid_items,
  T filler)
{
  warp_combine_scan_kernel<LOGICAL_WARP_THREADS, TOTAL_WARPS, T, ActionT><<<1, LOGICAL_WARP_THREADS * TOTAL_WARPS>>>(
    thrust::raw_pointer_cast(in.data()),
    thrust::raw_pointer_cast(inclusive_out.data()),
    thrust::raw_pointer_cast(exclusive_out.data()),
    action,
    valid_items,
    filler);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

template <int LOGICAL_WARP_THREADS, int TOTAL_WARPS, class T, class ActionT>
__global__ void warp_scan_kernel(T* in, T* out, ActionT action, int valid_items)
{
  using warp_scan_t = hipcub::WarpScan<T, LOGICAL_WARP_THREADS>;
  using storage_t   = typename warp_scan_t::TempStorage;

  __shared__ storage_t storage[TOTAL_WARPS];

  const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

  // Get warp index
  int warp_id = tid / LOGICAL_WARP_THREADS;

  T thread_data = in[tid];

  warp_scan_t scan(storage[warp_id]);

  action(scan, thread_data, valid_items);

  out[tid] = thread_data;
}

template <int LOGICAL_WARP_THREADS, int TOTAL_WARPS, class T, class ActionT>
void warp_scan(c2h::device_vector<T>& in, c2h::device_vector<T>& out, ActionT action, int valid_items)
{
  warp_scan_kernel<LOGICAL_WARP_THREADS, TOTAL_WARPS, T, ActionT><<<1, LOGICAL_WARP_THREADS * TOTAL_WARPS>>>(
    thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()), action, valid_items);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

enum class scan_mode
{
  exclusive,
  inclusive
};

template <scan_mode Mode>
struct sum_op_t
{
  template <class WarpScanT, class T>
  __device__ void operator()(WarpScanT& scan, T& thread_data, int valid_items) const
  {
    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, cuda::std::plus<>{}, valid_items);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, cuda::std::plus<>{}, valid_items);
    }
  }
};

template <class T, scan_mode Mode>
struct sum_aggregate_op_t
{
  int m_target_thread_id;
  T* m_d_warp_aggregate;

  template <int LOGICAL_WARP_THREADS>
  __device__ void operator()(hipcub::WarpScan<T, LOGICAL_WARP_THREADS>& scan, T& thread_data, int valid_items) const
  {
    T warp_aggregate{};

    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, cuda::std::plus<>{}, valid_items, warp_aggregate);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, cuda::std::plus<>{}, valid_items, warp_aggregate);
    }

    const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

    if (tid % LOGICAL_WARP_THREADS == m_target_thread_id)
    {
      m_d_warp_aggregate[tid / LOGICAL_WARP_THREADS] = warp_aggregate;
    }
  }
};

template <scan_mode Mode>
struct min_op_t
{
  template <class T, class WarpScanT>
  __device__ void operator()(WarpScanT& scan, T& thread_data, int valid_items) const
  {
    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, cuda::minimum<>{}, valid_items);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, cuda::minimum<>{}, valid_items);
    }
  }
};

template <class T, scan_mode Mode>
struct min_aggregate_op_t
{
  int m_target_thread_id;
  T* m_d_warp_aggregate;

  template <int LOGICAL_WARP_THREADS>
  __device__ void operator()(hipcub::WarpScan<T, LOGICAL_WARP_THREADS>& scan, T& thread_data, int valid_items) const
  {
    T warp_aggregate{};

    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, cuda::minimum<>{}, valid_items, warp_aggregate);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, cuda::minimum<>{}, valid_items, warp_aggregate);
    }

    const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

    if (tid % LOGICAL_WARP_THREADS == m_target_thread_id)
    {
      m_d_warp_aggregate[tid / LOGICAL_WARP_THREADS] = warp_aggregate;
    }
  }
};

template <class T, scan_mode Mode>
struct min_init_value_op_t
{
  T initial_value;
  template <class WarpScanT>
  __device__ void operator()(WarpScanT& scan, T& thread_data, int valid_items) const
  {
    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, initial_value, cuda::minimum<>{}, valid_items);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, initial_value, cuda::minimum<>{}, valid_items);
    }
  }
};

template <class T, scan_mode Mode>
struct min_init_value_aggregate_op_t
{
  int m_target_thread_id;
  T initial_value;
  T* m_d_warp_aggregate;

  template <int LOGICAL_WARP_THREADS>
  __device__ void operator()(hipcub::WarpScan<T, LOGICAL_WARP_THREADS>& scan, T& thread_data, int valid_items) const
  {
    T warp_aggregate{};

    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, initial_value, cuda::minimum<>{}, valid_items, warp_aggregate);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, initial_value, cuda::minimum<>{}, valid_items, warp_aggregate);
    }

    const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

    if (tid % LOGICAL_WARP_THREADS == m_target_thread_id)
    {
      m_d_warp_aggregate[tid / LOGICAL_WARP_THREADS] = warp_aggregate;
    }
  }
};

struct min_scan_op_t
{
  template <class T, class WarpScanT>
  __device__ void
  operator()(WarpScanT& scan, T& thread_data, T& inclusive_output, T& exclusive_output, int valid_items) const
  {
    scan.ScanPartial(thread_data, inclusive_output, exclusive_output, cuda::minimum<>{}, valid_items);
  }
};

template <class T>
struct min_init_value_scan_op_t
{
  T initial_value;
  template <class WarpScanT>
  __device__ void
  operator()(WarpScanT& scan, T& thread_data, T& inclusive_output, T& exclusive_output, int valid_items) const
  {
    scan.ScanPartial(thread_data, inclusive_output, exclusive_output, initial_value, cuda::minimum<>{}, valid_items);
  }
};

template <scan_mode Mode>
struct merge_op_t
{
  bool* error_flag_ptr;
  template <class WarpScanT>
  __device__ void operator()(WarpScanT& scan, segment& thread_data, int valid_items) const
  {
    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, merge_segments_op{error_flag_ptr}, valid_items);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, merge_segments_op{error_flag_ptr}, valid_items);
    }
  }
};

template <scan_mode Mode>
struct merge_aggregate_op_t
{
  int m_target_thread_id;
  segment* m_d_warp_aggregate;
  bool* error_flag_ptr;

  template <int LOGICAL_WARP_THREADS>
  __device__ void
  operator()(hipcub::WarpScan<segment, LOGICAL_WARP_THREADS>& scan, segment& thread_data, int valid_items) const
  {
    segment warp_aggregate{};

    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(
        thread_data, thread_data, merge_segments_op{error_flag_ptr}, valid_items, warp_aggregate);
    }
    else
    {
      scan.InclusiveScanPartial(
        thread_data, thread_data, merge_segments_op{error_flag_ptr}, valid_items, warp_aggregate);
    }

    const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

    if (tid % LOGICAL_WARP_THREADS == m_target_thread_id)
    {
      m_d_warp_aggregate[tid / LOGICAL_WARP_THREADS] = warp_aggregate;
    }
  }
};

template <scan_mode Mode>
struct merge_init_value_op_t
{
  segment initial_value;
  bool* error_flag_ptr;

  template <class WarpScanT>
  __device__ void operator()(WarpScanT& scan, segment& thread_data, int valid_items) const
  {
    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(thread_data, thread_data, initial_value, merge_segments_op{error_flag_ptr}, valid_items);
    }
    else
    {
      scan.InclusiveScanPartial(thread_data, thread_data, initial_value, merge_segments_op{error_flag_ptr}, valid_items);
    }
  }
};

template <scan_mode Mode>
struct merge_init_value_aggregate_op_t
{
  int m_target_thread_id;
  segment initial_value;
  segment* m_d_warp_aggregate;
  bool* error_flag_ptr;

  template <int LOGICAL_WARP_THREADS>
  __device__ void
  operator()(hipcub::WarpScan<segment, LOGICAL_WARP_THREADS>& scan, segment& thread_data, int valid_items) const
  {
    segment warp_aggregate{};

    if constexpr (Mode == scan_mode::exclusive)
    {
      scan.ExclusiveScanPartial(
        thread_data, thread_data, initial_value, merge_segments_op{error_flag_ptr}, valid_items, warp_aggregate);
    }
    else
    {
      scan.InclusiveScanPartial(
        thread_data, thread_data, initial_value, merge_segments_op{error_flag_ptr}, valid_items, warp_aggregate);
    }

    const int tid = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);

    if (tid % LOGICAL_WARP_THREADS == m_target_thread_id)
    {
      m_d_warp_aggregate[tid / LOGICAL_WARP_THREADS] = warp_aggregate;
    }
  }
};

struct merge_scan_op_t
{
  bool* error_flag_ptr;

  template <class WarpScanT>
  __device__ void operator()(
    WarpScanT& scan, segment& thread_data, segment& inclusive_output, segment& exclusive_output, int valid_items) const
  {
    scan.ScanPartial(thread_data, inclusive_output, exclusive_output, merge_segments_op{error_flag_ptr}, valid_items);
  }
};

struct merge_init_value_scan_op_t
{
  segment initial_value;
  bool* error_flag_ptr;

  template <class WarpScanT>
  __device__ void operator()(
    WarpScanT& scan, segment& thread_data, segment& inclusive_output, segment& exclusive_output, int valid_items) const
  {
    scan.ScanPartial(
      thread_data, inclusive_output, exclusive_output, initial_value, merge_segments_op{error_flag_ptr}, valid_items);
  }
};

template <class T, class ScanOpT>
c2h::host_vector<T> compute_host_reference(
  scan_mode mode,
  c2h::host_vector<T>& result,
  int logical_warp_threads,
  ScanOpT scan_op,
  int valid_items,
  T initial_value = T{})
{
  if (result.empty())
  {
    return c2h::host_vector<T>{};
  }
  // TODO : assert result.size() % logical_warp_threads == 0

  // The accumulator variable is used to calculate warp_aggregate without
  // taking initial_value into consideration in both exclusive and inclusive scan.
  int num_warps = cuda::ceil_div(static_cast<int>(result.size()), logical_warp_threads);
  c2h::host_vector<T> warp_accumulator(num_warps);
  if (mode == scan_mode::exclusive)
  {
    for (int w = 0; w < num_warps; ++w)
    {
      T* output     = result.data() + w * logical_warp_threads;
      T accumulator = output[0];
      T current     = static_cast<T>(scan_op(initial_value, output[0]));
      output[0]     = initial_value;
      for (int i = 1; i < cuda::std::clamp(valid_items, 0, logical_warp_threads); i++)
      {
        accumulator = static_cast<T>(scan_op(accumulator, output[i]));
        T tmp       = output[i];
        output[i]   = current;
        current     = static_cast<T>(scan_op(current, tmp));
      }
      warp_accumulator[w] = accumulator;
    }
  }
  else
  {
    for (int w = 0; w < num_warps; ++w)
    {
      T* output     = result.data() + w * logical_warp_threads;
      T accumulator = output[0];
      T current     = static_cast<T>(scan_op(initial_value, output[0]));
      output[0]     = current;
      for (int i = 1; i < cuda::std::clamp(valid_items, 0, logical_warp_threads); i++)
      {
        T tmp       = output[i];
        current     = static_cast<T>(scan_op(current, tmp));
        accumulator = static_cast<T>(scan_op(accumulator, tmp));
        output[i]   = current;
      }
      warp_accumulator[w] = accumulator;
    }
  }

  return warp_accumulator;
}

using types                = c2h::type_list<std::uint8_t, std::uint16_t, std::int32_t, std::int64_t>;
using logical_warp_threads = c2h::enum_type_list<int, 32, 16, 9, 2>;
using modes                = c2h::enum_type_list<scan_mode, scan_mode::exclusive, scan_mode::inclusive>;

using vec_types = c2h::type_list<
#if _CCCL_CTK_AT_LEAST(13, 0)
  ulonglong4_16a,
#else // _CCCL_CTK_AT_LEAST(13, 0)
  ulonglong4,
#endif // _CCCL_CTK_AT_LEAST(13, 0)
  uchar3,
  short2>;

using invalid_types = c2h::type_list<segment>;

template <int logical_warp_threads>
struct total_warps_t
{
private:
  static constexpr int max_warps      = 2;
  static constexpr bool is_arch_warp  = (logical_warp_threads == hipcub::detail::warp_threads);
  static constexpr bool is_pow_of_two = ((logical_warp_threads & (logical_warp_threads - 1)) == 0);
  static constexpr int total_warps    = (is_arch_warp || is_pow_of_two) ? max_warps : 1;

public:
  static constexpr int value()
  {
    return total_warps;
  }
};

template <class TestType>
struct params_t
{
  using type = typename c2h::get<0, TestType>;

  static constexpr int logical_warp_threads = c2h::get<1, TestType>::value;
  static constexpr scan_mode mode           = c2h::get<2, TestType>::value;
  static constexpr int total_warps          = total_warps_t<logical_warp_threads>::value();
  static constexpr int tile_size            = total_warps * logical_warp_threads;
};

C2H_TEST("Partial warp scan works with sum", "[scan][warp]", types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  warp_scan<params::logical_warp_threads, params::total_warps>(d_in, d_out, sum_op_t<params::mode>{}, valid_items);

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(params::mode, h_out, params::logical_warp_threads, std::plus<type>{}, valid_items);
  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE_APPROX_EQ(h_out, d_out);
}

C2H_TEST("Partial warp scan works with vec_types", "[scan][warp]", vec_types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  warp_scan<params::logical_warp_threads, params::total_warps>(d_in, d_out, sum_op_t<params::mode>{}, valid_items);

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(params::mode, h_out, params::logical_warp_threads, std::plus<type>{}, valid_items);
  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
}

C2H_TEST("Partial warp scan works with custom types",
         "[scan][warp]",
         c2h::type_list<c2h::custom_type_t<c2h::accumulateable_t, c2h::equal_comparable_t>>,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  warp_scan<params::logical_warp_threads, params::total_warps>(d_in, d_out, sum_op_t<params::mode>{}, valid_items);

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(params::mode, h_out, params::logical_warp_threads, std::plus<type>{}, valid_items);
  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
}

C2H_TEST("Partial warp scan returns valid warp aggregate",
         "[scan][warp]",
         c2h::type_list<c2h::custom_type_t<c2h::accumulateable_t, c2h::equal_comparable_t>>,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_warp_aggregates(params::total_warps);
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  c2h::gen(C2H_SEED(num_seeds), d_in);
  CAPTURE(d_in);

  const int target_thread_id = GENERATE_COPY(take(2, random(0, params::logical_warp_threads - 1)));

  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    sum_aggregate_op_t<type, params::mode>{target_thread_id, thrust::raw_pointer_cast(d_warp_aggregates.data())},
    valid_items);

  c2h::host_vector<type> h_out = d_in;

  auto h_warp_aggregates =
    compute_host_reference(params::mode, h_out, params::logical_warp_threads, std::plus<type>{}, valid_items);
  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
  REQUIRE(h_warp_aggregates == d_warp_aggregates);
}

// TODO : Do we need all the types?
C2H_TEST("Partial warp scan works with custom scan op", "[scan][warp]", types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  warp_scan<params::logical_warp_threads, params::total_warps>(d_in, d_out, min_op_t<params::mode>{}, valid_items);

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(
    params::mode,
    h_out,
    params::logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    cuda::std::numeric_limits<type>::max());

  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE_APPROX_EQ(h_out, d_out);
}

C2H_TEST("Partial warp custom op scan returns valid warp aggregate", "[scan][warp]", types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_warp_aggregates(params::total_warps);
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  const int target_thread_id = GENERATE_COPY(take(2, random(0, params::logical_warp_threads - 1)));

  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    min_aggregate_op_t<type, params::mode>{target_thread_id, thrust::raw_pointer_cast(d_warp_aggregates.data())},
    valid_items);

  c2h::host_vector<type> h_out = d_in;

  auto h_warp_aggregates = compute_host_reference(
    params::mode,
    h_out,
    params::logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    cuda::std::numeric_limits<type>::max());

  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
  REQUIRE(h_warp_aggregates == d_warp_aggregates);
}

C2H_TEST("Partial warp custom op scan works with initial value", "[scan][warp]", types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  const type initial_value = static_cast<type>(GENERATE_COPY(take(2, random(0, params::tile_size))));
  CAPTURE(d_in, initial_value);

  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in, d_out, min_init_value_op_t<type, params::mode>{initial_value}, valid_items);

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(
    params::mode,
    h_out,
    params::logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    initial_value);

  REQUIRE_APPROX_EQ(h_out, d_out);
}

C2H_TEST("Partial warp custom op scan with initial value returns valid warp aggregate",
         "[scan][warp]",
         types,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_warp_aggregates(params::total_warps);
  c2h::device_vector<type> d_out(params::tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(params::tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  const int target_thread_id = GENERATE_COPY(take(2, random(0, params::logical_warp_threads - 1)));
  const type initial_value   = static_cast<type>(GENERATE_COPY(take(2, random(0, params::tile_size))));

  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    min_init_value_aggregate_op_t<type, params::mode>{
      target_thread_id, initial_value, thrust::raw_pointer_cast(d_warp_aggregates.data())},
    valid_items);

  c2h::host_vector<type> h_out = d_in;

  auto h_warp_aggregates = compute_host_reference(
    params::mode,
    h_out,
    params::logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    initial_value);

  REQUIRE(h_out == d_out);
  REQUIRE(h_warp_aggregates == d_warp_aggregates);
}

C2H_TEST("Partial warp combination scan works with custom scan op", "[scan][warp]", logical_warp_threads)
{
  constexpr int logical_warp_threads = c2h::get<0, TestType>();
  constexpr int total_warps          = total_warps_t<logical_warp_threads>::value();
  constexpr int tile_size            = logical_warp_threads * total_warps;
  using type                         = int;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, logical_warp_threads))),
    take(1, random(logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, logical_warp_threads, logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::min(valid_items, logical_warp_threads);
  const type filler =
    GENERATE_COPY(take(1, random(cuda::std::numeric_limits<type>::lowest(), cuda::std::numeric_limits<type>::max())));
  CAPTURE(valid_items, logical_warp_threads, filler, c2h::type_name<type>());
  c2h::device_vector<type> d_inclusive_out(tile_size, thrust::no_init);
  c2h::device_vector<type> d_exclusive_out(tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  warp_combine_scan<logical_warp_threads, total_warps>(
    d_in, d_inclusive_out, d_exclusive_out, min_scan_op_t{}, valid_items, filler);

  c2h::host_vector<type> h_exclusive_out = d_in;
  c2h::host_vector<type> h_inclusive_out = d_in;
  for (int i = 0; i < tile_size; i += logical_warp_threads)
  {
    thrust::fill(
      h_exclusive_out.begin() + i + bounded_valid_items, h_exclusive_out.begin() + i + logical_warp_threads, filler);
    thrust::fill(
      h_inclusive_out.begin() + i + bounded_valid_items, h_inclusive_out.begin() + i + logical_warp_threads, filler);
  }

  compute_host_reference(
    scan_mode::exclusive,
    h_exclusive_out,
    logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    cuda::std::numeric_limits<type>::max());

  compute_host_reference(
    scan_mode::inclusive,
    h_inclusive_out,
    logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    cuda::std::numeric_limits<type>::max());

  // According to WarpScan::Scan documentation -
  // Because no initial value is supplied, the exclusive_output computed for warp-lane0 is
  // undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  for (size_t i = 0; i < h_exclusive_out.size(); i += logical_warp_threads)
  {
    d_exclusive_out[i] = h_exclusive_out[i];
  }

  REQUIRE(h_inclusive_out == d_inclusive_out);
  REQUIRE(h_exclusive_out == d_exclusive_out);
}

C2H_TEST("Partial warp combination custom scan works with initial value", "[scan][warp]", logical_warp_threads)
{
  constexpr int logical_warp_threads = c2h::get<0, TestType>();
  constexpr int total_warps          = total_warps_t<logical_warp_threads>::value();
  constexpr int tile_size            = logical_warp_threads * total_warps;
  using type                         = int;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, logical_warp_threads))),
    take(1, random(logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, logical_warp_threads, logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::min(valid_items, logical_warp_threads);
  const type filler =
    GENERATE_COPY(take(1, random(cuda::std::numeric_limits<type>::lowest(), cuda::std::numeric_limits<type>::max())));
  CAPTURE(valid_items, logical_warp_threads, filler, c2h::type_name<type>());
  c2h::device_vector<type> d_inclusive_out(tile_size, thrust::no_init);
  c2h::device_vector<type> d_exclusive_out(tile_size, thrust::no_init);
  c2h::device_vector<type> d_in(tile_size, thrust::no_init);
  c2h::gen(C2H_SEED(num_seeds), d_in);

  const type initial_value = GENERATE_COPY(take(2, random(0, total_warps * logical_warp_threads)));
  CAPTURE(d_in, initial_value);

  warp_combine_scan<logical_warp_threads, total_warps>(
    d_in, d_inclusive_out, d_exclusive_out, min_init_value_scan_op_t<type>{initial_value}, valid_items, filler);

  c2h::host_vector<type> h_exclusive_out = d_in;
  c2h::host_vector<type> h_inclusive_out = d_in;
  for (size_t i = 0; i < tile_size; i += logical_warp_threads)
  {
    thrust::fill(
      h_exclusive_out.begin() + i + bounded_valid_items, h_exclusive_out.begin() + i + logical_warp_threads, filler);
    thrust::fill(
      h_inclusive_out.begin() + i + bounded_valid_items, h_inclusive_out.begin() + i + logical_warp_threads, filler);
  }

  compute_host_reference(
    scan_mode::exclusive,
    h_exclusive_out,
    logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    initial_value);

  compute_host_reference(
    scan_mode::inclusive,
    h_inclusive_out,
    logical_warp_threads,
    [](type l, type r) {
      return std::min(l, r);
    },
    valid_items,
    initial_value);

  REQUIRE(h_inclusive_out == d_inclusive_out);
  REQUIRE(h_exclusive_out == d_exclusive_out);
}

C2H_TEST(
  "Partial warp scan does not apply op to invalid elements", "[scan][warp]", invalid_types, logical_warp_threads, modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::clamp(valid_items, 0, params::logical_warp_threads);
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < params::tile_size; i += params::logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  c2h::device_vector<bool> error_flag(1);
  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in, d_out, merge_op_t<params::mode>{thrust::raw_pointer_cast(error_flag.data())}, valid_items);
  REQUIRE(false == error_flag.front());
  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(
    params::mode, h_out, params::logical_warp_threads, merge_segments_op{}, valid_items, segment{1, 1});

  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
}

C2H_TEST("Partial warp scan does not apply op to invalid elements and returns valid warp aggregate",
         "[scan][warp]",
         invalid_types,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::clamp(valid_items, 0, params::logical_warp_threads);
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_warp_aggregates(params::total_warps);
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < params::tile_size; i += params::logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  const int target_thread_id = GENERATE_COPY(take(2, random(0, params::logical_warp_threads - 1)));

  c2h::device_vector<bool> error_flag(1);
  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    merge_aggregate_op_t<params::mode>{
      target_thread_id, thrust::raw_pointer_cast(d_warp_aggregates.data()), thrust::raw_pointer_cast(error_flag.data())},
    valid_items);
  REQUIRE(false == error_flag.front());

  c2h::host_vector<type> h_out = d_in;

  auto h_warp_aggregates = compute_host_reference(
    params::mode, h_out, params::logical_warp_threads, merge_segments_op{}, valid_items, segment{1, 1});

  // From the documentation -
  // Computes an exclusive prefix scan using the specified binary scan functor
  // across the calling warp. Because no initial value is supplied, the output
  // computed for warp-lane0 is undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  if constexpr (params::mode == scan_mode::exclusive)
  {
    for (size_t i = 0; i < h_out.size(); i += params::logical_warp_threads)
    {
      d_out[i] = h_out[i];
    }
  }
  REQUIRE(h_out == d_out);
  REQUIRE(h_warp_aggregates == d_warp_aggregates);
}

C2H_TEST("Partial warp scan does not apply op to invalid elements and works with initial value",
         "[scan][warp]",
         invalid_types,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::clamp(valid_items, 0, params::logical_warp_threads);
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < params::tile_size; i += params::logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  const type initial_value = segment{0, 1};

  c2h::device_vector<bool> error_flag(1);
  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    merge_init_value_op_t<params::mode>{initial_value, thrust::raw_pointer_cast(error_flag.data())},
    valid_items);
  REQUIRE(false == error_flag.front());

  c2h::host_vector<type> h_out = d_in;

  compute_host_reference(
    params::mode, h_out, params::logical_warp_threads, merge_segments_op{}, valid_items, initial_value);

  REQUIRE(h_out == d_out);
}

C2H_TEST("Partial warp scan with initial value does not apply op to invalid elements and returns valid warp aggregate",
         "[scan][warp]",
         invalid_types,
         logical_warp_threads,
         modes)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, params::logical_warp_threads))),
    take(1, random(params::logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, params::logical_warp_threads, params::logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::clamp(valid_items, 0, params::logical_warp_threads);
  CAPTURE(valid_items, params::mode, params::logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_warp_aggregates(params::total_warps);
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<type> d_in(params::tile_size);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < params::tile_size; i += params::logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  const int target_thread_id = GENERATE_COPY(take(2, random(0, params::logical_warp_threads - 1)));
  const type initial_value   = segment{0, 1};

  c2h::device_vector<bool> error_flag(1);
  warp_scan<params::logical_warp_threads, params::total_warps>(
    d_in,
    d_out,
    merge_init_value_aggregate_op_t<params::mode>{
      target_thread_id,
      initial_value,
      thrust::raw_pointer_cast(d_warp_aggregates.data()),
      thrust::raw_pointer_cast(error_flag.data())},
    valid_items);
  REQUIRE(false == error_flag.front());

  c2h::host_vector<type> h_out = d_in;

  auto h_warp_aggregates = compute_host_reference(
    params::mode, h_out, params::logical_warp_threads, merge_segments_op{}, valid_items, initial_value);

  REQUIRE(h_out == d_out);
  REQUIRE(h_warp_aggregates == d_warp_aggregates);
}

C2H_TEST("Partial warp combination scan does not apply op to invalid elements", "[scan][warp]", logical_warp_threads)
{
  constexpr int logical_warp_threads = c2h::get<0, TestType>();
  constexpr int total_warps          = total_warps_t<logical_warp_threads>::value();
  constexpr int tile_size            = logical_warp_threads * total_warps;
  using type                         = segment;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, logical_warp_threads))),
    take(1, random(logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, logical_warp_threads, logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::min(valid_items, logical_warp_threads);
  CAPTURE(valid_items, logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_inclusive_out(tile_size);
  c2h::device_vector<type> d_exclusive_out(tile_size);
  c2h::device_vector<type> d_in(tile_size);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < tile_size; i += logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  c2h::device_vector<bool> error_flag(1);
  warp_combine_scan<logical_warp_threads, total_warps>(
    d_in,
    d_inclusive_out,
    d_exclusive_out,
    merge_scan_op_t{thrust::raw_pointer_cast(error_flag.data())},
    valid_items,
    segment{});
  REQUIRE(false == error_flag.front());

  c2h::host_vector<type> h_exclusive_out = d_in;
  c2h::host_vector<type> h_inclusive_out = d_in;

  compute_host_reference(
    scan_mode::exclusive, h_exclusive_out, logical_warp_threads, merge_segments_op{}, valid_items, segment{1, 1});

  compute_host_reference(
    scan_mode::inclusive, h_inclusive_out, logical_warp_threads, merge_segments_op{}, valid_items, segment{1, 1});

  // According to WarpScan::Scan documentation -
  // Because no initial value is supplied, the exclusive_output computed for warp-lane0 is
  // undefined.

  // When comparing device output, the corresponding undefined data points need
  // to be fixed

  for (size_t i = 0; i < h_exclusive_out.size(); i += logical_warp_threads)
  {
    d_exclusive_out[i] = h_exclusive_out[i];
  }

  REQUIRE(h_inclusive_out == d_inclusive_out);
  REQUIRE(h_exclusive_out == d_exclusive_out);
}

C2H_TEST("Partial warp combination custom scan does not apply op to invalid elements and works with initial value",
         "[scan][warp]",
         logical_warp_threads)
{
  constexpr int logical_warp_threads = c2h::get<0, TestType>();
  constexpr int total_warps          = total_warps_t<logical_warp_threads>::value();
  constexpr int tile_size            = logical_warp_threads * total_warps;
  using type                         = segment;

  const int valid_items = GENERATE_COPY(
    take(1, random(2, cuda::std::max(2, logical_warp_threads))),
    take(1, random(logical_warp_threads + 2, cuda::std::numeric_limits<int>::max())),
    values({1, logical_warp_threads, logical_warp_threads + 1}));
  const int bounded_valid_items = cuda::std::min(valid_items, logical_warp_threads);
  CAPTURE(valid_items, logical_warp_threads, c2h::type_name<type>());
  c2h::device_vector<type> d_inclusive_out(total_warps * logical_warp_threads);
  c2h::device_vector<type> d_exclusive_out(total_warps * logical_warp_threads);
  c2h::device_vector<type> d_in(total_warps * logical_warp_threads);
  const auto in_it = cuda::make_transform_iterator(
    thrust::make_zip_iterator(cuda::counting_iterator<segment::offset_t>{1},
                              cuda::counting_iterator<segment::offset_t>{2}),
    tuple_to_segment_op{});
  for (size_t i = 0; i < tile_size; i += logical_warp_threads)
  {
    thrust::copy(in_it, in_it + bounded_valid_items, d_in.begin() + i);
  }

  const type initial_value = segment{0, 1};

  c2h::device_vector<bool> error_flag(1);
  warp_combine_scan<logical_warp_threads, total_warps>(
    d_in,
    d_inclusive_out,
    d_exclusive_out,
    merge_init_value_scan_op_t{initial_value, thrust::raw_pointer_cast(error_flag.data())},
    valid_items,
    segment{});
  REQUIRE(false == error_flag.front());

  c2h::host_vector<type> h_exclusive_out = d_in;
  c2h::host_vector<type> h_inclusive_out = d_in;

  compute_host_reference(
    scan_mode::exclusive, h_exclusive_out, logical_warp_threads, merge_segments_op{}, valid_items, initial_value);

  compute_host_reference(
    scan_mode::inclusive, h_inclusive_out, logical_warp_threads, merge_segments_op{}, valid_items, initial_value);

  REQUIRE(h_inclusive_out == d_inclusive_out);
  REQUIRE(h_exclusive_out == d_exclusive_out);
}
