#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/iterator/arg_index_input_iterator.cuh>
#include <cub/iterator/cache_modified_input_iterator.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/iterator/tex_obj_input_iterator.cuh>
#include <cub/iterator/transform_input_iterator.cuh>
#include <cub/util_allocator.cuh>
#include <cub/util_type.cuh>

#include <cuda/std/__cccl/dialect.h>

#include "catch2_test_helper.h"
#include "test_util.h"

using integral_types = c2h::type_list<signed char, short, int, long, long long>;

using types = metal::append<
  integral_types,
  float,
  double,
  char2,
  short2,
  int2,
  long2,
  longlong2,
  float2,
  double2,
  char3,
  short3,
  int3,
  long3,
  longlong3,
  float3,
  double3,
  char4,
  short4,
  int4,
  long4,
  longlong4,
  float4,
  double4,
  TestFoo,
  TestBar>;

template <typename InputIteratorT, typename T>
__global__ void test_iterator_kernel(InputIteratorT d_in, T* d_out, InputIteratorT* d_itrs)
{
  d_out[0] = *d_in; // Value at offset 0
  d_out[1] = d_in[100]; // Value at offset 100
  d_out[2] = *(d_in + 1000); // Value at offset 1000
  d_out[3] = *(d_in + 10000); // Value at offset 10000

  d_in++;
  d_out[4] = d_in[0]; // Value at offset 1

  d_in += 20;
  d_out[5]  = d_in[0]; // Value at offset 21
  d_itrs[0] = d_in; // Iterator at offset 21

  d_in -= 10;
  d_out[6] = d_in[0]; // Value at offset 11;

  d_in -= 11;
  d_out[7]  = d_in[0]; // Value at offset 0
  d_itrs[1] = d_in; // Iterator at offset 0
}

template <typename InputIteratorT, typename T>
void test_iterator(InputIteratorT d_in, const c2h::host_vector<T>& h_reference)
{
  // c2h::device_vector<InputIteratorT> and operator== below are so expensive to instantiate, they add **minutes** of
  // compile-time (measured with clang++-18 in CUDA mode).
#if 0
  c2h::device_vector<T> d_out(h_reference.size());
  c2h::device_vector<InputIteratorT> d_itrs(2, d_in); // need any iterator to copy, because InputIteratorT may not be
                                                      // default-constructible

  test_iterator_kernel<<<1, 1>>>(d_in, thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_itrs.data()));
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());

  c2h::host_vector<InputIteratorT> h_itrs = d_itrs;
  CHECK(h_reference == c2h::host_vector<T>(d_out)); // comparing host_vectors compiles a lot faster than mixed vectors
  CHECK(d_in + 21 == h_itrs[0]);
  CHECK(d_in == h_itrs[1]);
#else
  c2h::device_vector<T> d_out(h_reference.size());
  InputIteratorT* d_itrs;
  CHECK(hipSuccess == hipMalloc(&d_itrs, sizeof(InputIteratorT) * 2));

  test_iterator_kernel<<<1, 1>>>(d_in, thrust::raw_pointer_cast(d_out.data()), d_itrs);
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());

  InputIteratorT h_itrs[2]{d_in, d_in};
  CHECK(hipSuccess == hipMemcpy(h_itrs, d_itrs, sizeof(InputIteratorT) * 2, hipMemcpyDeviceToHost));

  CHECK(0 == CompareDeviceResults(h_reference.data(), thrust::raw_pointer_cast(d_out.data()), h_reference.size()));
  CHECK(d_in + 21 == h_itrs[0]);
  CHECK(d_in == h_itrs[1]);

  CHECK(hipSuccess == hipFree(d_itrs));
#endif
}

CUB_TEST("Test constant iterator", "[iterator]", integral_types)
{
  using T                = c2h::get<0, TestType>;
  const T base           = static_cast<T>(GENERATE(0, 99));
  const auto h_reference = c2h::host_vector<T>{base, base, base, base, base, base, base, base};
  test_iterator(hipcub::ConstantInputIterator<T>(base), h_reference);
}

CUB_TEST("Test counting iterator", "[iterator]", integral_types)
{
  using T                = c2h::get<0, TestType>;
  const T base           = static_cast<T>(GENERATE(0, 99));
  const auto h_reference = c2h::host_vector<T>{
    static_cast<T>(base + 0),
    static_cast<T>(base + 100),
    static_cast<T>(base + 1000),
    static_cast<T>(base + 10000),
    static_cast<T>(base + 1),
    static_cast<T>(base + 21),
    static_cast<T>(base + 11),
    static_cast<T>(base + 0)};
  test_iterator(hipcub::CountingInputIterator<T>(base), h_reference);
}

using cache_modifiers =
  c2h::enum_type_list<hipcub::CacheLoadModifier,
                      hipcub::LOAD_DEFAULT,
                      hipcub::LOAD_CA,
                      hipcub::LOAD_CG,
                      hipcub::LOAD_CS,
                      hipcub::LOAD_CV,
                      hipcub::LOAD_LDG,
                      hipcub::LOAD_VOLATILE>;

CUB_TEST("Test cache modified iterator", "[iterator]", types, cache_modifiers)
{
  using T                       = c2h::get<0, TestType>;
  constexpr auto cache_modifier = c2h::get<1, TestType>::value;
  constexpr int TEST_VALUES     = 11000;

  c2h::host_vector<T> h_data(TEST_VALUES);
  for (int i = 0; i < TEST_VALUES; ++i)
  {
    RandomBits(h_data[i]);
  }
  c2h::device_vector<T> d_data(h_data);

  const auto h_reference = c2h::host_vector<T>{
    h_data[0], h_data[100], h_data[1000], h_data[10000], h_data[1], h_data[21], h_data[11], h_data[0]};
  test_iterator(
    hipcub::CacheModifiedInputIterator<cache_modifier, T>(const_cast<const T*>(thrust::raw_pointer_cast(d_data.data()))),
    h_reference);
}

template <typename T>
struct TransformOp
{
  _CCCL_HOST_DEVICE T operator()(T input) const
  {
    T addend;
    InitValue(INTEGER_SEED, addend, 1);
    return input + addend;
  }
};

CUB_TEST("Test transform iterator", "[iterator]", types)
{
  using T                   = c2h::get<0, TestType>;
  constexpr int TEST_VALUES = 11000;

  c2h::host_vector<T> h_data(TEST_VALUES);
  for (int i = 0; i < TEST_VALUES; ++i)
  {
    InitValue(INTEGER_SEED, h_data[i], i);
  }
  c2h::device_vector<T> d_data(h_data.begin(), h_data.end());

  TransformOp<T> op;
  const auto h_reference = c2h::host_vector<T>{
    op(h_data[0]),
    op(h_data[100]),
    op(h_data[1000]),
    op(h_data[10000]),
    op(h_data[1]),
    op(h_data[21]),
    op(h_data[11]),
    op(h_data[0])};
  test_iterator(hipcub::TransformInputIterator<T, TransformOp<T>, const T*>(
                  const_cast<const T*>(const_cast<const T*>(thrust::raw_pointer_cast(d_data.data()))), op),
                h_reference);
}

CUB_TEST("Test tex-obj texture iterator", "[iterator]", types)
{
  using T                            = c2h::get<0, TestType>;
  constexpr unsigned int TEST_VALUES = 11000;

  c2h::host_vector<T> h_data(TEST_VALUES);
  for (unsigned int i = 0; i < TEST_VALUES; ++i)
  {
    RandomBits(h_data[i]);
  }
  c2h::device_vector<T> d_data(h_data.begin(), h_data.end());

  const auto h_reference = c2h::host_vector<T>{
    h_data[0], h_data[100], h_data[1000], h_data[10000], h_data[1], h_data[21], h_data[11], h_data[0]};
  hipcub::TexObjInputIterator<T> d_obj_itr;
  CubDebugExit(
    d_obj_itr.BindTexture(const_cast<const T*>(thrust::raw_pointer_cast(d_data.data())), sizeof(T) * TEST_VALUES));
  test_iterator(d_obj_itr, h_reference);
}

CUB_TEST("Test texture transform iterator", "[iterator]", types)
{
  using T                   = c2h::get<0, TestType>;
  constexpr int TEST_VALUES = 11000;

  c2h::host_vector<T> h_data(TEST_VALUES);
  for (int i = 0; i < TEST_VALUES; ++i)
  {
    InitValue(INTEGER_SEED, h_data[i], i);
  }
  c2h::device_vector<T> d_data(h_data.begin(), h_data.end());

  TransformOp<T> op;
  const auto h_reference = c2h::host_vector<T>{
    op(h_data[0]),
    op(h_data[100]),
    op(h_data[1000]),
    op(h_data[10000]),
    op(h_data[1]),
    op(h_data[21]),
    op(h_data[11]),
    op(h_data[0])};

  using TextureIterator = hipcub::TexObjInputIterator<T>;
  TextureIterator d_tex_itr;
  CubDebugExit(
    d_tex_itr.BindTexture(const_cast<const T*>(thrust::raw_pointer_cast(d_data.data())), sizeof(T) * TEST_VALUES));
  hipcub::TransformInputIterator<T, TransformOp<T>, TextureIterator> xform_itr(d_tex_itr, op);
  test_iterator(xform_itr, h_reference);
  CubDebugExit(d_tex_itr.UnbindTexture());
}
