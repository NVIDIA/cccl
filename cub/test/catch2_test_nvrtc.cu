#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <hip/hip_runtime.h>

#include <string>

#include <c2h/catch2_test_helper.h>
#include <hip/hiprtc.h>
#include <nvrtc_args.h>

TEST_CASE("Test nvrtc", "[test][nvrtc]")
{
  hiprtcProgram prog{};

  const char* src  = R"asdf(
    #include <cub/warp/warp_reduce.cuh>
    #include <cub/warp/warp_scan.cuh>
    #include <cub/warp/warp_exchange.cuh>
    #include <cub/warp/warp_load.cuh>
    #include <cub/warp/warp_store.cuh>
    #include <cub/warp/warp_merge_sort.cuh>
    #include <cub/block/block_adjacent_difference.cuh>
    #include <cub/block/block_discontinuity.cuh>
    #include <cub/block/block_exchange.cuh>
    #include <cub/block/block_histogram.cuh>
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
    #include <cub/block/block_merge_sort.cuh>
    #include <cub/block/block_radix_rank.cuh>
    #include <cub/block/block_radix_sort.cuh>
    #include <cub/block/block_reduce.cuh>
    #include <cub/block/block_scan.cuh>
    #include <cub/device/dispatch/kernels/reduce.cuh>
    #include <cub/device/dispatch/kernels/for_each.cuh>
    #include <cub/device/dispatch/kernels/scan.cuh>
    #include <cub/device/dispatch/kernels/merge_sort.cuh>

    extern "C" __global__ void kernel(int *ptr, int *errors)
    {
      constexpr int items_per_thread = 4;
      constexpr int threads_per_block = 128;
      using warp_load_t = hipcub::WarpLoad<int, items_per_thread>;
      using warp_load_storage_t = warp_load_t::TempStorage;

      using warp_exchange_t = hipcub::WarpExchange<int, items_per_thread>;
      using warp_exchange_storage_t = warp_exchange_t::TempStorage;

      using warp_reduce_t = hipcub::WarpReduce<int>;
      using warp_reduce_storage_t = warp_reduce_t::TempStorage;

      using warp_merge_sort_t = hipcub::WarpMergeSort<int, items_per_thread>;
      using warp_merge_sort_storage_t = warp_merge_sort_t::TempStorage;

      using warp_scan_t = hipcub::WarpScan<int>;
      using warp_scan_storage_t = warp_scan_t::TempStorage;

      using warp_store_t = hipcub::WarpStore<int, items_per_thread>;
      using warp_store_storage_t = warp_store_t::TempStorage;

      __shared__ warp_load_storage_t warp_load_storage;
      __shared__ warp_exchange_storage_t warp_exchange_storage;
      __shared__ warp_reduce_storage_t warp_reduce_storage;
      __shared__ warp_merge_sort_storage_t warp_merge_sort_storage;
      __shared__ warp_scan_storage_t warp_scan_storage;
      __shared__ warp_store_storage_t warp_store_storage;

      int items[items_per_thread];
      if (threadIdx.x < 32)
      {
        // Test warp load
        warp_load_t(warp_load_storage).Load(ptr, items);

        for (int i = 0; i < items_per_thread; i++)
        {
          if (items[i] != (i + threadIdx.x * items_per_thread))
          {
            atomicAdd(errors, 1);
          }
        }

        // Test warp exchange
        warp_exchange_t(warp_exchange_storage).BlockedToStriped(items, items);

        for (int i = 0; i < items_per_thread; i++)
        {
          if (items[i] != (i * 32 + threadIdx.x))
          {
            atomicAdd(errors, 1);
          }
        }

        // Test warp reduce
        const int sum = warp_reduce_t(warp_reduce_storage).Sum(items[0]);
        if (threadIdx.x == 0)
        {
          if (sum != (32 * (32 - 1) / 2))
          {
            atomicAdd(errors, 1);
          }
        }

        // Test warp scan
        int prefix_sum{};
        warp_scan_t(warp_scan_storage).InclusiveSum(items[0], prefix_sum);
        if (prefix_sum != (threadIdx.x * (threadIdx.x + 1) / 2))
        {
          atomicAdd(errors, 1);
        }

        // Test warp merge sort
        warp_merge_sort_t(warp_merge_sort_storage).Sort(
          items,
          [](int a, int b) { return a < b; });

        for (int i = 0; i < items_per_thread; i++)
        {
          if (items[i] != (i + threadIdx.x * items_per_thread))
          {
            atomicAdd(errors, 1);
          }
        }

        // Test warp store
        warp_store_t(warp_store_storage).Store(ptr, items);
      }
      __syncthreads();

      using block_load_t = hipcub::BlockLoad<int, threads_per_block, items_per_thread>;
      using block_load_storage_t = block_load_t::TempStorage;

      using block_exchange_t = hipcub::BlockExchange<int, threads_per_block, items_per_thread>;
      using block_exchange_storage_t = block_exchange_t::TempStorage;

      using block_reduce_t = hipcub::BlockReduce<int, threads_per_block>;
      using block_reduce_storage_t = block_reduce_t::TempStorage;

      using block_scan_t = hipcub::BlockScan<int, threads_per_block>;
      using block_scan_storage_t = block_scan_t::TempStorage;

      using block_radix_sort_t = hipcub::BlockRadixSort<int, threads_per_block, items_per_thread>;
      using block_radix_sort_storage_t = block_radix_sort_t::TempStorage;

      using block_store_t = hipcub::BlockStore<int, threads_per_block, items_per_thread>;
      using block_store_storage_t = block_store_t::TempStorage;

      __shared__ block_load_storage_t block_load_storage;
      __shared__ block_exchange_storage_t block_exchange_storage;
      __shared__ block_reduce_storage_t block_reduce_storage;
      __shared__ block_scan_storage_t block_scan_storage;
      __shared__ block_radix_sort_storage_t block_radix_sort_storage;
      __shared__ block_store_storage_t block_store_storage;

      // Test block load
      block_load_t(block_load_storage).Load(ptr, items);

      for (int i = 0; i < items_per_thread; i++)
      {
        if (items[i] != (i + threadIdx.x * items_per_thread))
        {
          atomicAdd(errors, 1);
        }
      }

      // Test block exchange
      block_exchange_t(block_exchange_storage).BlockedToStriped(items, items);

      for (int i = 0; i < items_per_thread; i++)
      {
        if (items[i] != (i * threads_per_block + threadIdx.x))
        {
          atomicAdd(errors, 1);
        }
      }

      // Test block reduce
      const int sum = block_reduce_t(block_reduce_storage).Sum(items[0]);
      if (threadIdx.x == 0)
      {
        if (sum != (threads_per_block * (threads_per_block - 1) / 2))
        {
          atomicAdd(errors, 1);
        }
      }

      // Test block scan
      int prefix_sum{};
      block_scan_t(block_scan_storage).InclusiveSum(items[0], prefix_sum);
      if (prefix_sum != (threadIdx.x * (threadIdx.x + 1) / 2))
      {
        atomicAdd(errors, 1);
      }

      // Test block radix sort
      block_radix_sort_t(block_radix_sort_storage).SortDescending(items);

      // Test block store
      block_store_t(block_store_storage).Store(ptr, items);
    }
)asdf";
  const char* name = "test";

  REQUIRE(HIPRTC_SUCCESS == hiprtcCreateProgram(&prog, src, name, 0, nullptr, nullptr));

  int ptx_version{};
  hipcub::PtxVersion(ptx_version);
  const std::string arch = std::string("-arch=sm_") + std::to_string(ptx_version / 10);
  const std::string std  = std::string("-std=c++") + std::to_string(_CCCL_STD_VER - 2000);

  constexpr int num_includes         = 6;
  const char* includes[num_includes] = {
    NVRTC_CUB_PATH, NVRTC_THRUST_PATH, NVRTC_LIBCUDACXX_PATH, NVRTC_CTK_PATH, arch.c_str(), std.c_str()};

  std::size_t log_size{};
  hiprtcResult compile_result = hiprtcCompileProgram(prog, num_includes, includes);

  REQUIRE(HIPRTC_SUCCESS == hiprtcGetProgramLogSize(prog, &log_size));

  std::unique_ptr<char[]> log{new char[log_size]};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetProgramLog(prog, log.get()));
  INFO("nvrtc log = " << log.get());
  REQUIRE(HIPRTC_SUCCESS == compile_result);

  std::size_t code_size{};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetBitcodeSize(prog, &code_size));

  std::unique_ptr<char[]> code{new char[code_size]};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetBitcode(prog, code.get()));
  REQUIRE(HIPRTC_SUCCESS == hiprtcDestroyProgram(&prog));

  hipCtx_t context{};
  hipDevice_t device{};
  hipModule_t module{};
  hipFunction_t kernel{};

  REQUIRE(hipSuccess == hipInit(0));
  REQUIRE(hipSuccess == hipDeviceGet(&device, 0));
  REQUIRE(hipSuccess == hipCtxCreate(&context, 0, device));
  REQUIRE(hipSuccess == hipModuleLoadDataEx(&module, code.get(), 0, 0, 0));
  REQUIRE(hipSuccess == hipModuleGetFunction(&kernel, module, "kernel"));

  // Generate input for execution, and create output buffers.
  constexpr int threads_in_block = 128;
  constexpr int items_per_thread = 4;
  constexpr int tile_size        = threads_in_block * items_per_thread;

  hipDeviceptr_t d_ptr{};
  REQUIRE(hipSuccess == hipMalloc(&d_ptr, tile_size * sizeof(int)));

  hipDeviceptr_t d_err{};
  REQUIRE(hipSuccess == hipMalloc(&d_err, sizeof(int)));

  int h_ptr[tile_size];
  for (int i = 0; i < tile_size; i++)
  {
    h_ptr[i] = i;
  }
  REQUIRE(hipSuccess == hipMemcpyHtoD(d_ptr, h_ptr, tile_size * sizeof(int)));

  int h_err{0};
  REQUIRE(hipSuccess == hipMemcpyHtoD(d_err, &h_err, sizeof(int)));

  void* args[] = {&d_ptr, &d_err};

  REQUIRE(hipSuccess == hipModuleLaunchKernel(kernel, 1, 1, 1, threads_in_block, 1, 1, 0, nullptr, args, 0));
  REQUIRE(hipSuccess == hipCtxSynchronize());
  REQUIRE(hipSuccess == hipMemcpyDtoH(h_ptr, d_ptr, tile_size * sizeof(int)));
  REQUIRE(hipSuccess == hipMemcpyDtoH(&h_err, d_err, sizeof(int)));

  REQUIRE(h_err == 0);
  for (int i = 0; i < tile_size; i++)
  {
    const int actual   = h_ptr[i];
    const int expected = tile_size - i - 1;
    REQUIRE(actual == expected);
  }

  REQUIRE(hipSuccess == hipFree(d_ptr));
  REQUIRE(hipSuccess == hipFree(d_err));
  REQUIRE(hipSuccess == hipModuleUnload(module));
  REQUIRE(hipSuccess == hipCtxDestroy(context));
}
