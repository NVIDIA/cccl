#include <cub/device/device_for.cuh> // internal include of NVTX

#include <cuda/iterator>
#include <cuda/std/functional>

#include <nvtx3/nvtx3.hpp> // user-side include of NVTX, retrieved elsewhere

int main()
{
  nvtx3::scoped_range range("user-range"); // user-side use of unversioned NVTX API

  cuda::counting_iterator<int> it{0};
  hipcub::DeviceFor::ForEach(it, it + 16, ::cuda::std::negate<int>{}); // internal use of NVTX
  hipDeviceSynchronize();
}
