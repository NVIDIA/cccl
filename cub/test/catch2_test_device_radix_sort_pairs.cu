/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/memory.h>

#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>

#include <cuda/std/type_traits>

#include <algorithm>
#include <limits>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceRadixSort::SortPairs, sort_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceRadixSort::SortPairsDescending, sort_pairs_descending);

using custom_value_t = c2h::custom_type_t<c2h::equal_comparable_t>;
using value_types = c2h::type_list<cuda::std::uint8_t, cuda::std::uint64_t, custom_value_t>;

using num_items_types =
  c2h::type_list<cuda::std::uint32_t, cuda::std::int32_t, cuda::std::uint64_t, cuda::std::int64_t>;

CUB_TEST("DeviceRadixSort::SortPairs: Basic testing", "[pairs][radix][sort][device]", value_types, num_items_types)
{
  using key_t = cuda::std::uint32_t;
  using value_t = c2h::get<0, TestType>;
  using num_items_t = c2h::get<1, TestType>;

  constexpr num_items_t min_num_items = 1 << 5;
  constexpr num_items_t max_num_items = 1 << 20;
  const num_items_t num_items =
    GENERATE_COPY(num_items_t{0}, num_items_t{1}, take(5, random(min_num_items, max_num_items)));

  thrust::device_vector<key_t> in_keys(num_items);
  thrust::device_vector<key_t> out_keys(num_items);

  thrust::device_vector<value_t> in_values(num_items);
  thrust::device_vector<value_t> out_values(num_items);

  const int num_key_seeds = 1;
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  const bool is_descending = GENERATE(false, true);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      num_items,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(thrust::raw_pointer_cast(in_keys.data()),
               thrust::raw_pointer_cast(out_keys.data()),
               thrust::raw_pointer_cast(in_values.data()),
               thrust::raw_pointer_cast(out_values.data()),
               num_items,
               begin_bit<key_t>(),
               end_bit<key_t>());
  }

  auto refs = radix_sort_reference(in_keys, in_values, is_descending);
  auto &ref_keys = refs.first;
  auto &ref_values = refs.second;

  REQUIRE(ref_keys == out_keys);
  REQUIRE(ref_values == out_values);
}

CUB_TEST("DeviceRadixSort::SortPairs: DoubleBuffer API", "[pairs][radix][sort][device]", value_types)
{
  using key_t = cuda::std::uint32_t;
  using value_t = c2h::get<0, TestType>;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));

  thrust::device_vector<key_t> in_keys(num_items);
  thrust::device_vector<key_t> out_keys(num_items);

  thrust::device_vector<value_t> in_values(num_items);
  thrust::device_vector<value_t> out_values(num_items);

  const int num_key_seeds = 1;
  const int num_value_seeds = 1;
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);
  c2h::gen(CUB_SEED(num_value_seeds), in_values);

  const bool is_descending = GENERATE(false, true);

  hipcub::DoubleBuffer<key_t> key_buffer(
    thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()));
  hipcub::DoubleBuffer<value_t> value_buffer(
    thrust::raw_pointer_cast(in_values.data()), thrust::raw_pointer_cast(out_values.data()));

  double_buffer_sort_t action(is_descending);
  action.initialize();
  launch(action,
         key_buffer,
         value_buffer,
         num_items,
         begin_bit<key_t>(),
         end_bit<key_t>());

  key_buffer.selector = action.selector();
  value_buffer.selector = action.selector();
  action.finalize();

  auto refs = radix_sort_reference(in_keys, in_values, is_descending);
  auto &ref_keys = refs.first;
  auto &ref_values = refs.second;

  auto& keys = key_buffer.selector == 0 ? in_keys : out_keys;
  auto& values = value_buffer.selector == 0 ? in_values : out_values;

  REQUIRE(ref_keys == keys);
  REQUIRE(ref_values == values);
}
