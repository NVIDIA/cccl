/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first
#include "catch2_radix_sort_helper.cuh"
#include <catch2_segmented_sort_helper.cuh>
#include <catch2_test_helper.h>

// FIXME: Graph launch disabled, algorithm syncs internally. WAR exists for device-launch, figure out how to enable for
// graph launch.

// TODO replace with DeviceSegmentedRadixSort::If interface once https://github.com/NVIDIA/cccl/issues/50 is addressed
// Temporary wrapper that allows specializing the DeviceSegmentedRadixSort algorithm for different offset types
template <bool IS_DESCENDING, typename KeyT, typename BeginOffsetIteratorT, typename EndOffsetIteratorT, typename NumItemsT>
HIPCUB_RUNTIME_FUNCTION _CCCL_FORCEINLINE static hipError_t dispatch_segmented_sort_wrapper(
  void* d_temp_storage,
  size_t& temp_storage_bytes,
  const KeyT* d_keys_in,
  KeyT* d_keys_out,
  NumItemsT num_items,
  NumItemsT num_segments,
  BeginOffsetIteratorT d_begin_offsets,
  EndOffsetIteratorT d_end_offsets,
  hipStream_t stream = 0)
{
  hipcub::DoubleBuffer<KeyT> d_keys(const_cast<KeyT*>(d_keys_in), d_keys_out);
  hipcub::DoubleBuffer<hipcub::NullType> d_values;
  return cub::
    DispatchSegmentedSort<IS_DESCENDING, KeyT, hipcub::NullType, NumItemsT, BeginOffsetIteratorT, EndOffsetIteratorT>::
      Dispatch(
        d_temp_storage,
        temp_storage_bytes,
        d_keys,
        d_values,
        num_items,
        num_segments,
        d_begin_offsets,
        d_end_offsets,
        false,
        stream);
}

// %PARAM% TEST_LAUNCH lid 0:1

DECLARE_LAUNCH_WRAPPER(dispatch_segmented_sort_wrapper<true>, dispatch_segmented_sort_descending);
DECLARE_LAUNCH_WRAPPER(dispatch_segmented_sort_wrapper<false>, dispatch_segmented_sort);

using key_types =
  c2h::type_list<bool,
                 std::uint8_t,
                 std::uint64_t
#if TEST_HALF_T
                 ,
                 half_t
#endif
#if TEST_BF_T
                 ,
                 bfloat16_t
#endif
                 >;

CUB_TEST("DeviceSegmentedSortKeys: No segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const bool stable_sort     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    stable_sort,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    int{},
    nullptr,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

CUB_TEST("DeviceSegmentedSortKeys: Empty segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const int num_segments     = GENERATE(take(2, random(1 << 2, 1 << 22)));
  const bool sort_stable     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  c2h::device_vector<int> offsets(num_segments + 1, int{});
  const int* d_offsets = thrust::raw_pointer_cast(offsets.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    sort_stable,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    num_segments,
    d_offsets,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

CUB_TEST("DeviceSegmentedSortKeys: Same size segments, derived keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int segment_size = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)));

  const int segments = GENERATE_COPY(take(2, random(1 << 0, 1 << 5)), //
                                     take(2, random(1 << 5, 1 << 10)));

  test_same_size_segments_derived<KeyT>(segment_size, segments);
}

CUB_TEST("DeviceSegmentedSortKeys: Randomly sized segments, derived keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)),
    take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_derived<KeyT>(CUB_SEED(1), max_items, max_segment, segments);
}

CUB_TEST("DeviceSegmentedSortKeys: Randomly sized segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_random<KeyT>(CUB_SEED(1), max_items, max_segment, segments);
}

CUB_TEST("DeviceSegmentedSortKeys: Edge case segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_edge_case_segments_random<KeyT>(CUB_SEED(4));
}

CUB_TEST("DeviceSegmentedSortKeys: Unspecified segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_unspecified_segments_random<KeyT>(CUB_SEED(4));
}

// we can reuse the same structure of DeviceSegmentedRadixSortKeys for simplicity
CUB_TEST("DeviceSegmentedSortKeys: 64-bit num. items and num. segments", "[keys][segmented][sort][device]")
{
  using key_t    = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using offset_t = cuda::std::int64_t; // the test requires ~30 GB GPU memory including temporary buffer size

  constexpr std::size_t min_num_items = std::size_t{1} << 31;
  constexpr std::size_t max_num_items = min_num_items + (std::size_t{1} << 20);
  constexpr int num_key_seeds         = 1;
  constexpr int num_segment_seeds     = 1;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(min_num_items, max_num_items)));
  const std::size_t num_segments      = GENERATE_COPY(take(1, random(min_num_items, max_num_items)));
  const bool is_descending            = GENERATE(false, true);
  CAPTURE(num_items, num_segments, is_descending);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::device_vector<offset_t> offsets(num_segments + 1);
  c2h::gen(CUB_SEED(num_key_seeds), in_keys);
  generate_segment_offsets(CUB_SEED(num_segment_seeds), offsets, static_cast<offset_t>(num_items));

  if (is_descending)
  {
    dispatch_segmented_sort(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1);
  }
  else
  {
    dispatch_segmented_sort(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      // Mix pointers/iterators for segment info to test using different iterable types:
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1);
  }
  // compoute the reference only if the routine is able to terminate correctly
  auto ref_keys = segmented_radix_sort_reference(in_keys, is_descending, offsets);
  REQUIRE((ref_keys == out_keys) == true);
}
