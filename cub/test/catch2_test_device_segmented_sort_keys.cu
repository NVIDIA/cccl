#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"

#include <cub/device/device_segmented_sort.cuh>
#include <cub/util_type.cuh>

#include "catch2_radix_sort_helper.cuh"
#include "catch2_segmented_sort_helper.cuh"
#include <c2h/bfloat16.cuh>
#include <c2h/catch2_test_helper.h>
#include <c2h/half.cuh>

// FIXME: Graph launch disabled, algorithm syncs internally. WAR exists for device-launch, figure out how to enable for
// graph launch.
// %PARAM% TEST_LAUNCH lid 0:1

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedSort::StableSortKeys, stable_sort_keys);

using key_types =
  c2h::type_list<bool,
                 std::uint8_t,
                 std::uint64_t
#if TEST_HALF_T()
                 ,
                 half_t
#endif // TEST_HALF_T()
#if TEST_BF_T()
                 ,
                 bfloat16_t
#endif // TEST_BF_T()
                 >;

C2H_TEST("DeviceSegmentedSortKeys: No segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const bool stable_sort     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    stable_sort,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    int{},
    nullptr,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortKeys: Empty segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const int num_segments     = GENERATE(take(2, random(1 << 2, 1 << 22)));
  const bool sort_stable     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  c2h::device_vector<int> offsets(num_segments + 1, int{});
  const int* d_offsets = thrust::raw_pointer_cast(offsets.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    sort_stable,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    num_segments,
    d_offsets,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortKeys: Same size segments, derived keys",
         "[keys][segmented][sort][device][skip-cs-racecheck]",
         key_types)
{
  using KeyT = c2h::get<0, TestType>;

  // Use adjust_seed_count to limit the number of passes run during sanitizer tests.
  const int segment_size = GENERATE_COPY(
    take(c2h::adjust_seed_count(2), random(1 << 0, 1 << 5)), //
    take(c2h::adjust_seed_count(2), random(1 << 5, 1 << 10)),
    take(c2h::adjust_seed_count(2), random(1 << 10, 1 << 15)));

  const int segments = GENERATE_COPY( //
    take(c2h::adjust_seed_count(2), random(1 << 0, 1 << 5)), //
    take(c2h::adjust_seed_count(2), random(1 << 5, 1 << 10)));

  test_same_size_segments_derived<KeyT>(segment_size, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Randomly sized segments, derived keys",
         "[keys][segmented][sort][device][skip-cs-racecheck]",
         key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  // Use adjust_seed_count to limit the number of passes run during sanitizer tests.
  const int segments = GENERATE_COPY(
    take(c2h::adjust_seed_count(2), random(1 << 0, 1 << 5)), //
    take(c2h::adjust_seed_count(2), random(1 << 5, 1 << 10)),
    take(c2h::adjust_seed_count(2), random(1 << 10, 1 << 15)),
    take(c2h::adjust_seed_count(2), random(1 << 15, 1 << 20)));

  test_random_size_segments_derived<KeyT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Randomly sized segments, random keys",
         "[keys][segmented][sort][device][skip-cs-initcheck][skip-cs-racecheck]",
         key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  // Use adjust_seed_count to limit the number of passes run during sanitizer tests.
  const int segments = GENERATE_COPY(take(c2h::adjust_seed_count(2), random(1 << 15, 1 << 20)));

  test_random_size_segments_random<KeyT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Edge case segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_edge_case_segments_random<KeyT>(C2H_SEED(4));
}

C2H_TEST("DeviceSegmentedSortKeys: Unspecified segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_unspecified_segments_random<KeyT>(C2H_SEED(4));
}

C2H_TEST("DeviceSegmentedSortKeys: very large number of segments",
         "[keys][segmented][sort][device][skip-cs-memcheck][skip-cs-racecheck][skip-cs-initcheck]",
         all_offset_types)
try
{
  using key_t                        = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using segment_offset_t             = std::int64_t;
  using offset_t                     = c2h::get<0, TestType>;
  using segment_iterator_t           = segment_index_to_offset_op<offset_t, segment_offset_t>;
  constexpr std::size_t segment_size = 1000000;
  constexpr std::size_t uint32_max   = cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : cuda::std::numeric_limits<offset_t>::max();
  constexpr segment_offset_t num_empty_segments = uint32_max;
  const segment_offset_t num_segments           = num_empty_segments + cuda::ceil_div(num_items, segment_size);
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);

  // Generate input keys
  constexpr auto max_histo_size = 250;
  segmented_verification_helper<key_t> verification_helper{max_histo_size};
  verification_helper.prepare_input_data(in_keys);

  auto offsets = thrust::make_transform_iterator(
    thrust::make_counting_iterator(std::size_t{0}),
    segment_iterator_t{num_empty_segments, num_segments, segment_size, num_items});

  stable_sort_keys(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    offsets,
    offsets + 1);

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys, offsets + num_empty_segments, num_segments - num_empty_segments);
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}

C2H_TEST("DeviceSegmentedSort::SortKeys: very large segments",
         "[keys][segmented][sort][device][skip-cs-memcheck][skip-cs-racecheck][skip-cs-initcheck]",
         all_offset_types)
try
{
  using key_t                      = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using segment_offset_t           = std::int32_t;
  using offset_t                   = c2h::get<0, TestType>;
  constexpr std::size_t uint32_max = cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr int num_key_seeds      = 1;
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : cuda::std::numeric_limits<offset_t>::max();
  const segment_offset_t num_segments = 2;
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  c2h::device_vector<offset_t> offsets(num_segments + 1);
  offsets[0] = 0;
  offsets[1] = static_cast<offset_t>(num_items);
  offsets[2] = static_cast<offset_t>(num_items);

  // Prepare information for later verification
  short_key_verification_helper<key_t> verification_helper{};
  verification_helper.prepare_verification_data(in_keys);

  stable_sort_keys(
    thrust::raw_pointer_cast(in_keys.data()),
    thrust::raw_pointer_cast(out_keys.data()),
    static_cast<offset_t>(num_items),
    static_cast<segment_offset_t>(num_segments),
    thrust::raw_pointer_cast(offsets.data()),
    offsets.cbegin() + 1);

  // Verify the keys are sorted correctly
  verification_helper.verify_sorted(out_keys);
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, insufficient GPU memory. " << e.what() << "\n";
}
