#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_select.cuh>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <algorithm>

#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

template <class T>
inline T to_bound(const unsigned long long bound)
{
  return static_cast<T>(bound);
}

template <>
inline ulonglong2 to_bound(const unsigned long long bound)
{
  return {bound, bound};
}

template <>
inline ulonglong4 to_bound(const unsigned long long bound)
{
  return {bound, bound, bound, bound};
}

template <>
inline long2 to_bound(const unsigned long long bound)
{
  return {static_cast<long>(bound), static_cast<long>(bound)};
}

template <>
inline c2h::custom_type_t<c2h::equal_comparable_t> to_bound(const unsigned long long bound)
{
  c2h::custom_type_t<c2h::equal_comparable_t> val;
  val.key = bound;
  val.val = bound;
  return val;
}

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSelect::Unique, select_unique);

// %PARAM% TEST_LAUNCH lid 0:1:2

struct equal_to_default_t
{
  template <typename T>
  __host__ __device__ bool operator()(const T& a) const
  {
    return a == T{};
  }
};

using all_types =
  c2h::type_list<std::uint8_t,
                 std::uint16_t,
                 std::uint32_t,
                 std::uint64_t,
                 ulonglong2,
                 ulonglong4,
                 int,
                 long2,
                 c2h::custom_type_t<c2h::equal_comparable_t>>;

using types = c2h::type_list<std::uint8_t, std::uint32_t>;

CUB_TEST("DeviceSelect::Unique can run with empty input", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(in.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == 0);
}

CUB_TEST("DeviceSelect::Unique handles none equal", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(thrust::counting_iterator<type>(0), thrust::discard_iterator<>(), d_first_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == num_items);
}

CUB_TEST("DeviceSelect::Unique handles all equal", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items, static_cast<type>(1));
  c2h::device_vector<type> out(1);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(in.begin(), out.begin(), d_first_num_selected_out, num_items);

  // At least one item is selected
  REQUIRE(num_selected_out[0] == 1);
  REQUIRE(out[0] == in[0]);
}

CUB_TEST("DeviceSelect::Unique does not change input", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(CUB_SEED(2), in, to_bound<type>(0), to_bound<type>(42));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // copy input first
  c2h::device_vector<type> reference = in;

  select_unique(in.begin(), out.begin(), d_first_num_selected_out, num_items);

  REQUIRE(reference == in);
}

CUB_TEST("DeviceSelect::Unique works with iterators", "[device][select_unique]", all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(CUB_SEED(2), in, to_bound<type>(0), to_bound<type>(42));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(in.begin(), out.begin(), d_first_num_selected_out, num_items);

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  const auto boundary              = std::unique(reference.begin(), reference.end());
  REQUIRE((boundary - reference.begin()) == num_selected_out[0]);

  out.resize(num_selected_out[0]);
  reference.resize(num_selected_out[0]);
  REQUIRE(reference == out);
}

CUB_TEST("DeviceSelect::Unique works with pointers", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(CUB_SEED(2), in, to_bound<type>(0), to_bound<type>(42));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(
    thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()), d_first_num_selected_out, num_items);

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  const auto boundary              = std::unique(reference.begin(), reference.end());
  REQUIRE((boundary - reference.begin()) == num_selected_out[0]);

  out.resize(num_selected_out[0]);
  reference.resize(num_selected_out[0]);
  REQUIRE(reference == out);
}

template <class T>
struct convertible_from_T
{
  T val_;

  convertible_from_T() = default;
  __host__ __device__ convertible_from_T(const T& val) noexcept
      : val_(val)
  {}
  __host__ __device__ convertible_from_T& operator=(const T& val) noexcept
  {
    val_ = val;
  }
  // Converting back to T helps satisfy all the machinery that T supports
  __host__ __device__ operator T() const noexcept
  {
    return val_;
  }
};

CUB_TEST("DeviceSelect::Unique works with a different output type", "[device][select_unique]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<convertible_from_T<type>> out(num_items);
  c2h::gen(CUB_SEED(2), in, to_bound<type>(0), to_bound<type>(42));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_unique(in.begin(), out.begin(), d_first_num_selected_out, num_items);

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  const auto boundary              = std::unique(reference.begin(), reference.end());
  REQUIRE((boundary - reference.begin()) == num_selected_out[0]);

  out.resize(num_selected_out[0]);
  reference.resize(num_selected_out[0]);
  REQUIRE(reference == out);
}
