/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/util_arch.cuh>
#include <cub/util_device.cuh>
#include <cub/util_macro.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>

#include "catch2_test_helper.h"
#include "catch2_test_launch_helper.h"

BEGIN_HIPCUB_NAMESPACE

CUB_DETAIL_KERNEL_ATTRIBUTES void write_ptx_version_kernel(int* d_kernel_cuda_arch)
{
  *d_kernel_cuda_arch = HIPCUB_ARCH;
}

HIPCUB_RUNTIME_FUNCTION static hipError_t get_cuda_arch_from_kernel(
  void* d_temp_storage, size_t& temp_storage_bytes, int* d_kernel_cuda_arch, int* ptx_version, hipStream_t stream = 0)
{
  if (d_temp_storage == nullptr)
  {
    temp_storage_bytes = 1;
    return hipSuccess;
  }
  write_ptx_version_kernel<<<1, 1, 0, stream>>>(d_kernel_cuda_arch);
  return hipcub::PtxVersion(*ptx_version);
}

END_HIPCUB_NAMESPACE

// %PARAM% TEST_LAUNCH lid 0:1:2
DECLARE_LAUNCH_WRAPPER(hipcub::get_cuda_arch_from_kernel, get_cuda_arch_from_kernel);

CUB_TEST("CUB correctly identifies the ptx version the kernel was compiled for", "[util][dispatch]")
{
  constexpr std::size_t single_item = 1;
  c2h::device_vector<int> cuda_arch(single_item);

  int* ptx_version{};
  hipHostMalloc(&ptx_version, sizeof(*ptx_version));

  // Query the arch the kernel was actually compiled for
  get_cuda_arch_from_kernel(thrust::raw_pointer_cast(cuda_arch.data()), ptx_version);
  int kernel_cuda_arch = cuda_arch[0];

  // Host hipcub::PtxVersion
  int host_ptx_version{};
  hipcub::PtxVersion(host_ptx_version);

  // Ensure variable was properly populated
  REQUIRE(0 != kernel_cuda_arch);

  // Ensure that the ptx version corresponds to the arch the kernel was compiled for
  REQUIRE(*ptx_version == kernel_cuda_arch);
  REQUIRE(host_ptx_version == kernel_cuda_arch);
}
