#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"

#include <cub/detail/rfa.cuh>
#include <cub/device/dispatch/dispatch_reduce_deterministic.cuh>
#include <cub/util_type.cuh>

#include <thrust/device_vector.h>

#include <numeric>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.h>
#include <c2h/generators.h>

// %PARAM% TEST_LAUNCH lid 0:1:2

using float_type_list = c2h::type_list<float, double>;

template <typename InputIteratorT, typename OutputIteratorT, typename NumItemsT>
HIPCUB_RUNTIME_FUNCTION static hipError_t DeterministicSum(
  void* d_temp_storage,
  size_t& temp_storage_bytes,
  InputIteratorT d_in,
  OutputIteratorT d_out,
  NumItemsT num_items,
  hipStream_t stream = 0)
{
  _CCCL_NVTX_RANGE_SCOPE_IF(d_temp_storage, "hipcub::DeviceReduce::DeterministicSum");

  // Signed integer type for global offsets
  using OffsetT = hipcub::detail::choose_offset_t<NumItemsT>;

  // The output value type
  using OutputT = hipcub::detail::non_void_value_t<OutputIteratorT, hipcub::detail::it_value_t<InputIteratorT>>;

  using InitT = OutputT;

  return hipcub::detail::DispatchReduceDeterministic<InputIteratorT, OutputIteratorT, OffsetT>::Dispatch(
    d_temp_storage,
    temp_storage_bytes,
    d_in,
    d_out,
    static_cast<OffsetT>(num_items),
    InitT{}, // zero-initialize
    stream);
}

template <int NOMINAL_BLOCK_THREADS_4B, int NOMINAL_ITEMS_PER_THREAD_4B>
struct AgentReducePolicy
{
  /// Number of items per vectorized load
  static constexpr int VECTOR_LOAD_LENGTH = 4;

  /// Cooperative block-wide reduction algorithm to use
  static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BlockReduceAlgorithm::BLOCK_REDUCE_RAKING;

  /// Cache load modifier for reading input elements
  static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::CacheLoadModifier::LOAD_DEFAULT;
  constexpr static int ITEMS_PER_THREAD                 = NOMINAL_ITEMS_PER_THREAD_4B;
  constexpr static int BLOCK_THREADS                    = NOMINAL_BLOCK_THREADS_4B;
};

template <int ItemsPerThread, int BlockSize>
struct hub_t
{
  struct Policy : hipcub::ChainedPolicy<300, Policy, Policy>
  {
    constexpr static int ITEMS_PER_THREAD = ItemsPerThread;

    using ReducePolicy = AgentReducePolicy<BlockSize, ItemsPerThread>;

    // SingleTilePolicy
    using SingleTilePolicy = ReducePolicy;

    // SegmentedReducePolicy
    using SegmentedReducePolicy = ReducePolicy;
  };

  using MaxPolicy = Policy;
};

DECLARE_LAUNCH_WRAPPER(DeterministicSum, deterministic_sum);

// TODO (srinivasyadav18): Replace with macro `REQUIRE_APPROX_EQ_EPSILON` once the PR
// https://github.com/NVIDIA/cccl/pull/4842 is merged
template <typename T>
bool approx_eq(const T& expected, const T& actual, const double tolerance = 0.01)
{
  double diff     = std::abs(static_cast<double>(expected) - static_cast<double>(actual));
  double rel_diff = diff / std::abs(static_cast<double>(expected));
  return rel_diff < tolerance;
}

C2H_TEST("Deterministic Device reduce works with float and double on gpu", "[reduce][deterministic]", float_type_list)
{
  using type          = typename c2h::get<0, TestType>;
  const int num_items = 1 << 20;
  c2h::device_vector<type> d_input(num_items);
  c2h::gen(C2H_SEED(2), d_input, static_cast<type>(-1000.0), static_cast<type>(1000.0));

  c2h::device_vector<type> d_output(1);

  const type* d_input_ptr = thrust::raw_pointer_cast(d_input.data());

  deterministic_sum(d_input_ptr, d_output.begin(), num_items);

  c2h::host_vector<type> h_input = d_input;

  // Requires `std::accumulate` to produce deterministic result which is required for comparison
  // with the device RFA result.
  // NOTE: `std::reduce` is not equivalent
  const type h_expected           = std::accumulate(h_input.begin(), h_input.end(), type{}, ::cuda::std::plus<type>());
  c2h::host_vector<type> h_output = d_output;

  REQUIRE(approx_eq(h_expected, h_output[0]));
}

C2H_TEST("Deterministic Device reduce works with float and double and is deterministic on gpu with different policies ",
         "[reduce][deterministic]",
         float_type_list)
{
  using type              = typename c2h::get<0, TestType>;
  constexpr int min_items = 1;
  constexpr int max_items = 50000;

  const int num_items = GENERATE_COPY(
    take(3, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));

  CAPTURE(num_items);

  c2h::device_vector<type> input(num_items);

  const type min_val = static_cast<type>(-1000.0f);
  const type max_val = static_cast<type>(1000.0f);

  c2h::gen(C2H_SEED(2), input, min_val, max_val);
  c2h::device_vector<type> output_p1(1);
  c2h::device_vector<type> output_p2(1);

  using input_it_t   = const type*;
  input_it_t d_input = thrust::raw_pointer_cast(input.data());

  using output_it_t = decltype(output_p1.begin());
  using init_t      = hipcub::detail::rfa::InitT<input_it_t, output_it_t>;
  using accum_t     = hipcub::detail::rfa::AccumT<::cuda::std::plus<>, init_t, input_it_t>;
  using transform_t = ::cuda::std::__identity;

  using deterministic_dispatch_t_p1 =
    hipcub::detail::DispatchReduceDeterministic<input_it_t, output_it_t, int, init_t, accum_t, transform_t, hub_t<1, 128>>;

  using deterministic_dispatch_t_p2 =
    hipcub::detail::DispatchReduceDeterministic<input_it_t, output_it_t, int, init_t, accum_t, transform_t, hub_t<2, 256>>;

  std::size_t temp_storage_bytes{};

  auto error =
    deterministic_dispatch_t_p1::Dispatch(nullptr, temp_storage_bytes, d_input, output_p1.begin(), num_items);
  REQUIRE(error == hipSuccess);

  c2h::device_vector<std::uint8_t> temp_storage_p1(temp_storage_bytes);

  error = deterministic_dispatch_t_p1::Dispatch(
    thrust::raw_pointer_cast(temp_storage_p1.data()), temp_storage_bytes, d_input, output_p1.begin(), num_items);
  REQUIRE(error == hipSuccess);

  type const res_p1 = output_p1[0];

  error = deterministic_dispatch_t_p2::Dispatch(nullptr, temp_storage_bytes, d_input, output_p2.begin(), num_items);
  REQUIRE(error == hipSuccess);

  c2h::device_vector<std::uint8_t> temp_storage_p2(temp_storage_bytes);

  error = deterministic_dispatch_t_p2::Dispatch(
    thrust::raw_pointer_cast(temp_storage_p2.data()), temp_storage_bytes, d_input, output_p2.begin(), num_items);
  REQUIRE(error == hipSuccess);

  type const res_p2 = output_p2[0];

  c2h::host_vector<type> h_input = input;
  const type h_expected          = std::accumulate(h_input.begin(), h_input.end(), type{}, ::cuda::std::plus<type>());

  // device RFA result should be approximately equal to host result
  REQUIRE(approx_eq(h_expected, res_p1));

  // Both device RFA results should be strictly equal, as RFA is deterministic
  REQUIRE(res_p1 == res_p2);
}

C2H_TEST("Deterministic Device reduce works with float and double on gpu using device_vector iterators",
         "[reduce][deterministic]",
         float_type_list)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = 1 << 10;
  c2h::device_vector<type> d_input(num_items);
  c2h::gen(C2H_SEED(2), d_input, static_cast<type>(-1000.0), static_cast<type>(1000.0));

  c2h::device_vector<type> d_output(1);

  deterministic_sum(d_input.begin(), d_output.begin(), num_items);

  c2h::host_vector<type> h_input = d_input;

  // Requires `std::accumulate` to produce deterministic result which is required for comparison
  // with the device RFA result.
  // NOTE: `std::reduce` is not equivalent
  const type h_expected           = std::accumulate(h_input.begin(), h_input.end(), type{}, ::cuda::std::plus<type>());
  c2h::host_vector<type> h_output = d_output;

  REQUIRE(approx_eq(h_expected, h_output[0]));
}
