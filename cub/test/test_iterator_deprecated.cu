#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of iterator utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

// This file tests deprecated CUB APIs. Silence deprecation warnings:
#define CUB_IGNORE_DEPRECATED_API

#include <cub/iterator/tex_ref_input_iterator.cuh>
#include <cub/util_type.cuh>
#include <cub/util_allocator.cuh>

#include <iterator>
#include <cstdio>
#include <typeinfo>

#include "test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;
CachingDeviceAllocator  g_allocator(true);

//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Test random access input iterator
 */
template <
    typename InputIteratorT,
    typename T>
__global__ void Kernel(
    InputIteratorT    d_in,
    T                 *d_out,
    InputIteratorT    *d_itrs)
{
    d_out[0] = *d_in;               // Value at offset 0
    d_out[1] = d_in[100];           // Value at offset 100
    d_out[2] = *(d_in + 1000);      // Value at offset 1000
    d_out[3] = *(d_in + 10000);     // Value at offset 10000

    d_in++;
    d_out[4] = d_in[0];             // Value at offset 1

    d_in += 20;
    d_out[5] = d_in[0];             // Value at offset 21
    d_itrs[0] = d_in;               // Iterator at offset 21

    d_in -= 10;
    d_out[6] = d_in[0];             // Value at offset 11;

    d_in -= 11;
    d_out[7] = d_in[0];             // Value at offset 0
    d_itrs[1] = d_in;               // Iterator at offset 0
}



//---------------------------------------------------------------------
// Host testing subroutines
//---------------------------------------------------------------------


/**
 * Run iterator test on device
 */
template <
    typename        InputIteratorT,
    typename        T,
    int             TEST_VALUES>
void Test(
    InputIteratorT  d_in,
    T               (&h_reference)[TEST_VALUES])
{
    // Allocate device arrays
    T                 *d_out    = NULL;
    InputIteratorT    *d_itrs   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out,     sizeof(T) * TEST_VALUES));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_itrs,    sizeof(InputIteratorT) * 2));

    int compare;

    // Run unguarded kernel
    Kernel<<<1, 1>>>(d_in, d_out, d_itrs);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Check results
    compare = CompareDeviceResults(h_reference, d_out, TEST_VALUES, g_verbose, g_verbose);
    printf("\tValues: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 21
    InputIteratorT h_itr = d_in + 21;
    compare = CompareDeviceResults(&h_itr, d_itrs, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 0
    compare = CompareDeviceResults(&d_in, d_itrs + 1, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (d_out)
    {
        CubDebugExit(g_allocator.DeviceFree(d_out));
    }
    if (d_itrs)
    {
        CubDebugExit(g_allocator.DeviceFree(d_itrs));
    }
}

/**
 * Test tex-ref texture iterator
 */
template <typename T, typename CastT>
void TestTexRef()
{
    printf("\nTesting tex-ref iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES                   = 11000;
    constexpr unsigned int DUMMY_OFFSET         = 500;
    constexpr unsigned int DUMMY_TEST_VALUES    = TEST_VALUES - DUMMY_OFFSET;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data   = NULL;
    T *d_dummy  = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dummy, sizeof(T) * DUMMY_TEST_VALUES));
    CubDebugExit(hipMemcpy(d_dummy, h_data + DUMMY_OFFSET, sizeof(T) * DUMMY_TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    // Create and bind ref-based test iterator
    TexRefInputIterator<T, __LINE__> d_ref_itr;
    CubDebugExit(d_ref_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    // Create and bind dummy iterator of same type to check with interferance
    TexRefInputIterator<T, __LINE__> d_ref_itr2;
    CubDebugExit(d_ref_itr2.BindTexture((CastT*) d_dummy, sizeof(T) * DUMMY_TEST_VALUES));

    Test(d_ref_itr, h_reference);

    CubDebugExit(d_ref_itr.UnbindTexture());
    CubDebugExit(d_ref_itr2.UnbindTexture());

    if (h_data)
    {
        delete[] h_data;
    }
    if (d_data)
    {
        CubDebugExit(g_allocator.DeviceFree(d_data));
    }
    if (d_dummy)
    {
        CubDebugExit(g_allocator.DeviceFree(d_dummy));
    }
}

/**
 * Run non-integer tests
 */
template <typename T, typename CastT>
void Test()
{
    TestTexRef<T, CastT>();
}

/**
 * Run tests
 */
template <typename T>
void Test()
{
    // Test non-const type
    Test<T, T>();

    // Test non-const type
    Test<T, const T>();
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Evaluate different data types
    Test<signed char>();
    Test<short>();
    Test<int>();
    Test<long>();
    Test<long long>();
    Test<float>();
    Test<double>();

    Test<char2>();
    Test<short2>();
    Test<int2>();
    Test<long2>();
    Test<longlong2>();
    Test<float2>();
    Test<double2>();

    Test<char3>();
    Test<short3>();
    Test<int3>();
    Test<long3>();
    Test<longlong3>();
    Test<float3>();
    Test<double3>();

    Test<char4>();
    Test<short4>();
    Test<int4>();
    Test<long4>();
    Test<longlong4>();
    Test<float4>();
    Test<double4>();

    Test<TestFoo>();
    Test<TestBar>();

    printf("\nTest complete\n");
    fflush(stdout);

    return 0;
}
