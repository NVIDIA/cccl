/*******************************************************************************
 * Copyright (c) 2011-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "test_util.h"

#include <cub/thread/thread_operators.cuh>

template <class T>
T Make(int val)
{
  return T{val};
}

template <bool>
class BaseT
{
protected:
  int m_val{};

public:
  BaseT(int val)
      : m_val{val}
  {}
};

template <>
class BaseT<true>
{
protected:
  int m_val{};

public:
  BaseT(int val)
      : m_val{val}
  {}

  __host__ __device__ operator int() const { return m_val; }
};

#define CUSTOM_TYPE_FACTORY(NAME, RT, OP, CONVERTABLE)                         \
  class Custom##NAME##T : public BaseT<CONVERTABLE>                            \
  {                                                                            \
    explicit Custom##NAME##T(int val)                                          \
        : BaseT<CONVERTABLE>(val)                                              \
    {}                                                                         \
                                                                               \
    friend Custom##NAME##T Make<Custom##NAME##T>(int);                         \
                                                                               \
  public:                                                                      \
    __host__ __device__ RT operator OP(int val) const                          \
    {                                                                          \
      return m_val OP val;                                                     \
    }                                                                          \
  }

//                  NAME  RT    OP  CONVERTABLE
CUSTOM_TYPE_FACTORY(Eq,   bool, ==, false);
CUSTOM_TYPE_FACTORY(Ineq, bool, !=, false);
CUSTOM_TYPE_FACTORY(Sum,  int,  +,  false);
CUSTOM_TYPE_FACTORY(Diff, int,  -,  false);
CUSTOM_TYPE_FACTORY(Div,  int,  /,  false);
CUSTOM_TYPE_FACTORY(Gt,   bool, >,  true);
CUSTOM_TYPE_FACTORY(Lt,   bool, <,  true);

void TestEquality()
{
  hipcub::Equality op{}; 

  const int const_magic_val = 42;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, const_magic_val), true);
  AssertEquals(op(const_magic_val, magic_val), true);
  AssertEquals(op(const_magic_val, magic_val + 1), false);

  AssertEquals(op(Make<CustomEqT>(magic_val), magic_val), true);
  AssertEquals(op(Make<CustomEqT>(magic_val), magic_val + 1), false);
}

void TestInequality()
{
  hipcub::Inequality op{}; 

  const int const_magic_val = 42;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, const_magic_val), false);
  AssertEquals(op(const_magic_val, magic_val), false);
  AssertEquals(op(const_magic_val, magic_val + 1), true);

  AssertEquals(op(Make<CustomIneqT>(magic_val), magic_val), false);
  AssertEquals(op(Make<CustomIneqT>(magic_val), magic_val + 1), true);
}

void TestInequalityWrapper()
{
  hipcub::Equality wrapped_op{}; 
  hipcub::InequalityWrapper<hipcub::Equality> op{wrapped_op};

  const int const_magic_val = 42;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, const_magic_val), false);
  AssertEquals(op(const_magic_val, magic_val), false);
  AssertEquals(op(const_magic_val, magic_val + 1), true);

  AssertEquals(op(Make<CustomEqT>(magic_val), magic_val), false);
  AssertEquals(op(Make<CustomEqT>(magic_val), magic_val + 1), true);
}

#define CUSTOM_SYNC_T(NAME, RT, OP)                                            \
  struct Custom ## NAME ## Sink                                                \
  {                                                                            \
    template <class T>                                                         \
    __host__ __device__ RT operator OP (T &&) const                            \
    {                                                                          \
      return RT{};                                                             \
    }                                                                          \
  }

CUSTOM_SYNC_T(SumInt, int, +);
CUSTOM_SYNC_T(SumCustomInt, CustomSumIntSink, +);

CUSTOM_SYNC_T(DiffInt, int, -);
CUSTOM_SYNC_T(DiffCustomInt, CustomDiffIntSink, -);

CUSTOM_SYNC_T(DivInt, int, /);
CUSTOM_SYNC_T(DivCustomInt, CustomDivIntSink, /);

template <class ExpectedT, class ActualT>
void StaticSame()
{
  static_assert(std::is_same<ExpectedT, ActualT>::value, "shall match");
}

void TestSum()
{
  hipcub::Sum op{};

  const int const_magic_val = 40;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, 2), 42);
  AssertEquals(op(magic_val, 2), 42);
  AssertEquals(op(Make<CustomSumT>(magic_val), 2), 42);

  StaticSame<decltype(op(42, 42)), int>();
  StaticSame<decltype(op(1, 1.0)), double>();
  StaticSame<decltype(op(CustomSumIntSink{}, 1.0)), int>();
  StaticSame<decltype(op(CustomSumCustomIntSink{}, 1.0)), CustomSumIntSink>();
}

void TestDifference()
{
  hipcub::Difference op{};

  const int const_magic_val = 44;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, 2), 42);
  AssertEquals(op(magic_val, 2), 42);

  AssertEquals(op(Make<CustomDiffT>(magic_val), 2), 42);

  StaticSame<decltype(op(42, 42)), int>();
  StaticSame<decltype(op(1, 1.0)), double>();
  StaticSame<decltype(op(CustomDiffIntSink{}, 1.0)), int>();
  StaticSame<decltype(op(CustomDiffCustomIntSink{}, 1.0)), CustomDiffIntSink>();
}

void TestDivision()
{
  hipcub::Division op{};

  const int const_magic_val = 44;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, 2), 22);
  AssertEquals(op(magic_val, 2), 22);

  AssertEquals(op(Make<CustomDivT>(magic_val), 2), 22);

  StaticSame<decltype(op(42, 42)), int>();
  StaticSame<decltype(op(1, 1.0)), double>();
  StaticSame<decltype(op(CustomDivIntSink{}, 1.0)), int>();
  StaticSame<decltype(op(CustomDivCustomIntSink{}, 1.0)), CustomDivIntSink>();
}

void TestMax()
{
  hipcub::Max op{};

  const int const_magic_val = 42;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, 2), 42);
  AssertEquals(op(magic_val, 2), 42);

  AssertEquals(op(2, Make<CustomGtT>(magic_val)), 42);

  StaticSame<decltype(op(42, 42)), int>();
  StaticSame<decltype(op(1, 1.0)), double>();
  StaticSame<decltype(op(1, Make<CustomGtT>(magic_val))), int>();
}

void TestMin()
{
  hipcub::Min op{};

  const int const_magic_val = 42;
  int magic_val = const_magic_val;

  AssertEquals(op(const_magic_val, 2), 2);
  AssertEquals(op(magic_val, 2), 2);

  AssertEquals(op(2, Make<CustomLtT>(magic_val)), 2);

  StaticSame<decltype(op(42, 42)), int>();
  StaticSame<decltype(op(1, 1.0)), double>();
  StaticSame<decltype(op(1, Make<CustomLtT>(magic_val))), int>();
}

int main()
{
  TestEquality();
  TestInequality();
  TestInequalityWrapper();
  TestSum();
  TestDifference();
  TestDivision();
  TestMax();
  TestMin();

  return 0;
}
