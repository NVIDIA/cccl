#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

#include "insert_nested_NVTX_range_guard.h"

#include <cub/device/device_merge.cuh>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/sort.h>

#include <algorithm>

#include <test_util.h>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.h>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceMerge::MergePairs, merge_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceMerge::MergeKeys, merge_keys);

using types = c2h::type_list<std::uint8_t, std::int16_t, std::uint32_t, double>;

template <typename Key,
          typename Offset,
          typename CompareOp = cuda::std::less<Key>,
          typename MergeKeys = decltype(::merge_keys)>
void test_keys(Offset size1 = 3623, Offset size2 = 6346, CompareOp compare_op = {}, MergeKeys merge_keys = ::merge_keys)
{
  CAPTURE(c2h::type_name<Key>(), c2h::type_name<Offset>(), size1, size2);

  c2h::device_vector<Key> keys1_d(size1);
  c2h::device_vector<Key> keys2_d(size2);

  c2h::gen(C2H_SEED(1), keys1_d);
  c2h::gen(C2H_SEED(1), keys2_d);

  thrust::sort(c2h::device_policy, keys1_d.begin(), keys1_d.end(), compare_op);
  thrust::sort(c2h::device_policy, keys2_d.begin(), keys2_d.end(), compare_op);
  // CAPTURE(keys1_d, keys2_d);

  c2h::device_vector<Key> result_d(size1 + size2);
  merge_keys(thrust::raw_pointer_cast(keys1_d.data()),
             static_cast<Offset>(keys1_d.size()),
             thrust::raw_pointer_cast(keys2_d.data()),
             static_cast<Offset>(keys2_d.size()),
             thrust::raw_pointer_cast(result_d.data()),
             compare_op);

  c2h::host_vector<Key> keys1_h = keys1_d;
  c2h::host_vector<Key> keys2_h = keys2_d;
  c2h::host_vector<Key> reference_h(size1 + size2);
  std::merge(keys1_h.begin(), keys1_h.end(), keys2_h.begin(), keys2_h.end(), reference_h.begin(), compare_op);

  // FIXME(bgruber): comparing std::vectors (slower than thrust vectors) but compiles a lot faster
  CHECK((detail::to_vec(reference_h) == detail::to_vec(c2h::host_vector<Key>(result_d))));
}

C2H_TEST("DeviceMerge::MergeKeys key types", "[merge][device]", types)
{
  using key_t    = c2h::get<0, TestType>;
  using offset_t = int;
  test_keys<key_t, offset_t>();
}

C2H_TEST("DeviceMerge::MergeKeys works for large number of items",
         "[merge][device][skip-cs-racecheck][skip-cs-initcheck][skip-cs-synccheck]")
try
{
  using key_t    = char;
  using offset_t = int64_t;

  // Clamp 64-bit offset type problem sizes to just slightly larger than 2^32 items
  const auto num_items_int_max = static_cast<offset_t>(cuda::std::numeric_limits<std::int32_t>::max());

  // Generate the input sizes to test for
  const offset_t num_items_lhs =
    GENERATE_COPY(values({num_items_int_max + offset_t{1000000}, num_items_int_max - 1, offset_t{3}}));
  const offset_t num_items_rhs =
    GENERATE_COPY(values({num_items_int_max + offset_t{1000000}, num_items_int_max, offset_t{3}}));

  test_keys<key_t, offset_t>(num_items_lhs, num_items_rhs, cuda::std::less<>{});
}
catch (const std::bad_alloc&)
{
  // allocation failure is not a test failure, so we can run tests on smaller GPUs
}

C2H_TEST("DeviceMerge::MergeKeys input sizes", "[merge][device]")
{
  using key_t    = int;
  using offset_t = int;
  // TODO(bgruber): maybe less combinations
  const auto size1 = offset_t{GENERATE(0, 1, 23, 123, 3234)};
  const auto size2 = offset_t{GENERATE(0, 1, 52, 556, 56767)};
  test_keys<key_t>(size1, size2);
}

// cannot put those in an anon namespace, or nvcc complains that the kernels have internal linkage
using unordered_t = c2h::custom_type_t<c2h::equal_comparable_t>;
struct order
{
  __host__ __device__ auto operator()(const unordered_t& a, const unordered_t& b) const -> bool
  {
    return a.key < b.key;
  }
};

C2H_TEST("DeviceMerge::MergeKeys no operator<", "[merge][device]")
{
  using key_t    = unordered_t;
  using offset_t = int;
  test_keys<key_t, offset_t, order>();
}

namespace
{
template <typename... Its>
auto zip(Its... its) -> decltype(thrust::make_zip_iterator(its...))
{
  return thrust::make_zip_iterator(its...);
}

template <typename Value>
struct key_to_value
{
  template <typename Key>
  __host__ __device__ auto operator()(const Key& k) const -> Value
  {
    Value v{};
    convert(k, v, 0);
    return v;
  }

  template <typename Key>
  __host__ __device__ static void convert(const Key& k, Value& v, ...)
  {
    v = static_cast<Value>(k);
  }

  template <template <typename> class... Policies>
  __host__ __device__ static void convert(const c2h::custom_type_t<Policies...>& k, Value& v, int)
  {
    v = static_cast<Value>(k.val);
  }

  template <typename Key, template <typename> class... Policies>
  __host__ __device__ static void convert(const Key& k, c2h::custom_type_t<Policies...>& v, int)
  {
    v     = {};
    v.val = static_cast<decltype(v.val)>(k);
  }
};
} // namespace

template <typename Key,
          typename Value,
          typename Offset,
          typename CompareOp  = cuda::std::less<Key>,
          typename MergePairs = decltype(::merge_pairs)>
void test_pairs(
  Offset size1 = 200, Offset size2 = 625, CompareOp compare_op = {}, MergePairs merge_pairs = ::merge_pairs)
{
  CAPTURE(c2h::type_name<Key>(), c2h::type_name<Value>(), c2h::type_name<Offset>(), size1, size2);

  // we start with random but sorted keys
  c2h::device_vector<Key> keys1_d(size1);
  c2h::device_vector<Key> keys2_d(size2);
  c2h::gen(C2H_SEED(1), keys1_d);
  c2h::gen(C2H_SEED(1), keys2_d);
  thrust::sort(c2h::device_policy, keys1_d.begin(), keys1_d.end(), compare_op);
  thrust::sort(c2h::device_policy, keys2_d.begin(), keys2_d.end(), compare_op);

  // the values must be functionally dependent on the keys (equal key => equal value), since merge is unstable
  c2h::device_vector<Value> values1_d(size1);
  c2h::device_vector<Value> values2_d(size2);
  thrust::transform(c2h::device_policy, keys1_d.begin(), keys1_d.end(), values1_d.begin(), key_to_value<Value>{});
  thrust::transform(c2h::device_policy, keys2_d.begin(), keys2_d.end(), values2_d.begin(), key_to_value<Value>{});
  //  CAPTURE(keys1_d, keys2_d, values1_d, values2_d);

  // compute CUB result
  c2h::device_vector<Key> result_keys_d(size1 + size2);
  c2h::device_vector<Value> result_values_d(size1 + size2);
  merge_pairs(
    thrust::raw_pointer_cast(keys1_d.data()),
    thrust::raw_pointer_cast(values1_d.data()),
    static_cast<Offset>(keys1_d.size()),
    thrust::raw_pointer_cast(keys2_d.data()),
    thrust::raw_pointer_cast(values2_d.data()),
    static_cast<Offset>(keys2_d.size()),
    thrust::raw_pointer_cast(result_keys_d.data()),
    thrust::raw_pointer_cast(result_values_d.data()),
    compare_op);

  // compute reference result
  c2h::host_vector<Key> reference_keys_h(size1 + size2);
  c2h::host_vector<Value> reference_values_h(size1 + size2);
  {
    c2h::host_vector<Key> keys1_h     = keys1_d;
    c2h::host_vector<Value> values1_h = values1_d;
    c2h::host_vector<Key> keys2_h     = keys2_d;
    c2h::host_vector<Value> values2_h = values2_d;
    using value_t                     = typename decltype(zip(keys1_h.begin(), values1_h.begin()))::value_type;
    std::merge(zip(keys1_h.begin(), values1_h.begin()),
               zip(keys1_h.end(), values1_h.end()),
               zip(keys2_h.begin(), values2_h.begin()),
               zip(keys2_h.end(), values2_h.end()),
               zip(reference_keys_h.begin(), reference_values_h.begin()),
               [&](const value_t& a, const value_t& b) {
                 return compare_op(thrust::get<0>(a), thrust::get<0>(b));
               });
  }

  // FIXME(bgruber): comparing std::vectors (slower than thrust vectors) but compiles a lot faster
  CHECK((detail::to_vec(reference_keys_h) == detail::to_vec(c2h::host_vector<Key>(result_keys_d))));
  CHECK((detail::to_vec(reference_values_h) == detail::to_vec(c2h::host_vector<Value>(result_values_d))));
}

C2H_TEST("DeviceMerge::MergePairs key types", "[merge][device]", types)
{
  using key_t    = c2h::get<0, TestType>;
  using value_t  = int;
  using offset_t = int;
  test_pairs<key_t, value_t, offset_t>();
}

// TODO(bgruber): fine tune the type sizes again to hit the fallback and the vsmem policies
// C2H_TEST("DeviceMerge::MergePairs large key types", "[merge][device]", large_types)
// {
//   using key_t    = c2h::get<0, TestType>;
//   using value_t  = int;
//   using offset_t = int;
//   test_pairs<key_t, value_t, offset_t>();
// }

C2H_TEST("DeviceMerge::MergePairs value types", "[merge][device]", types)
{
  using key_t    = int;
  using value_t  = c2h::get<0, TestType>;
  using offset_t = int;
  test_pairs<key_t, value_t, offset_t>();
}

C2H_TEST("DeviceMerge::MergePairs input sizes", "[merge][device]")
{
  using key_t      = int;
  using value_t    = int;
  using offset_t   = int;
  const auto size1 = offset_t{GENERATE(0, 1, 23, 123, 3234234)};
  const auto size2 = offset_t{GENERATE(0, 1, 52, 556, 56767)};
  test_pairs<key_t, value_t>(size1, size2);
}

// this test exceeds 4GiB of memory and the range of 32-bit integers
C2H_TEST("DeviceMerge::MergePairs really large input",
         "[merge][device][skip-cs-racecheck][skip-cs-initcheck][skip-cs-synccheck]")
try
{
  using key_t     = char;
  using value_t   = char;
  const auto size = std::int64_t{1} << GENERATE(30, 31, 32, 33);
  test_pairs<key_t, value_t>(size, size, cuda::std::less<>{});
}
catch (const std::bad_alloc&)
{
  // allocation failure is not a test failure, so we can run tests on smaller GPUs
}

C2H_TEST("DeviceMerge::MergePairs iterators", "[merge][device]")
{
  using key_t             = int;
  using value_t           = int;
  using offset_t          = int;
  const offset_t size1    = 363;
  const offset_t size2    = 634;
  const auto values_start = 123456789;

  auto key_it   = thrust::counting_iterator<key_t>{};
  auto value_it = thrust::counting_iterator<key_t>{values_start};

  // compute CUB result
  c2h::device_vector<key_t> result_keys_d(size1 + size2);
  c2h::device_vector<value_t> result_values_d(size1 + size2);
  merge_pairs(
    key_it,
    value_it,
    size1,
    key_it,
    value_it,
    size2,
    result_keys_d.begin(),
    result_values_d.begin(),
    cuda::std::less<key_t>{});

  // check result
  c2h::host_vector<key_t> result_keys_h     = result_keys_d;
  c2h::host_vector<value_t> result_values_h = result_values_d;
  const auto smaller_size                   = std::min(size1, size2);
  for (offset_t i = 0; i < static_cast<offset_t>(result_keys_h.size()); i++)
  {
    if (i < 2 * smaller_size)
    {
      CHECK(result_keys_h[i + 0] == i / 2);
      CHECK(result_values_h[i + 0] == values_start + i / 2);
    }
    else
    {
      CHECK(result_keys_h[i] == i - smaller_size);
      CHECK(result_values_h[i] == values_start + i - smaller_size);
    }
  }
}
