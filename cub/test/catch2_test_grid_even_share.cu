#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/grid/grid_even_share.cuh>
#include <cub/grid/grid_mapping.cuh>

#include <cuda/std/__algorithm/min.h>
#include <cuda/std/type_traits>

#include <c2h/catch2_test_helper.h>
#include <c2h/generators.h>

// %PARAM% TEST_LAUNCH lid 0:1:2

using offset_types = c2h::type_list<int32_t, int64_t, uint32_t, uint64_t>;

C2H_TEST("GridEvenShare handles edge cases (zero/negative items)", "[grid][even_share][edge_cases]", offset_types)
{
  using offset_t = typename c2h::get<0, TestType>;

  hipcub::GridEvenShare<offset_t> grid_share;

  const offset_t num_items = []() {
    if constexpr (cuda::std::is_signed_v<offset_t>)
    {
      return GENERATE_COPY(values({-1, 0, 1}));
    }
    else
    {
      return GENERATE_COPY(values({0, 1}));
    }
  }();

  const int max_grid_size = GENERATE_COPY(values({-1, 0, 1}));
  const int tile_items    = GENERATE_COPY(values({-1, 0, 1}));

  // Skip if all parameters are positive (covered by the normal operation test)
  if (num_items > 0 && max_grid_size > 0 && tile_items > 0)
  {
    return;
  }

  grid_share.DispatchInit(num_items, max_grid_size, tile_items);

  REQUIRE(grid_share.num_items == 0);
  REQUIRE(grid_share.grid_size == 0);
  REQUIRE(grid_share.block_offset == 0);
  REQUIRE(grid_share.block_end == 0);
}

C2H_TEST("GridEvenShare works with num_items > 0", "[grid][even_share]", offset_types)
{
  using offset_t = typename c2h::get<0, TestType>;

  hipcub::GridEvenShare<offset_t> grid_share;

  const offset_t num_items = GENERATE_COPY(values({1, 20, 37, 100, 2000, 1 << 20}));
  const int max_grid_size  = GENERATE_COPY(values({1, 20, 37, 100, 2000, 1 << 20}));
  const int tile_items     = GENERATE_COPY(values({1, 20, 37, 100, 2000, 1 << 20}));

  grid_share.DispatchInit(num_items, max_grid_size, tile_items);

  REQUIRE(grid_share.num_items == num_items);
  REQUIRE(
    grid_share.grid_size == cuda::std::min(max_grid_size, static_cast<int>(cuda::ceil_div(num_items, tile_items))));
  REQUIRE(grid_share.block_offset == num_items);
  REQUIRE(grid_share.block_end == num_items);
}
