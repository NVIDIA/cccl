#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

#include <cub/device/device_copy.cuh>
#include <cub/util_macro.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

#include <cstdint>

#include "catch2_large_problem_helper.cuh"
#include "catch2_test_device_memcpy_batched_common.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.h>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceCopy::Batched, copy_batched);

/**
 * @brief Function object class template that takes an offset and returns an iterator at the given
 * offset relative to a fixed base iterator.
 *
 * @tparam IteratorT The random-access iterator type to be returned
 */
template <typename IteratorT>
struct offset_to_transform_it
{
  template <typename OffsetT>
  __host__ __device__ __forceinline__ auto operator()(OffsetT offset) const
  {
    return thrust::make_transform_output_iterator(base_it + offset, ::cuda::std::identity{});
  }
  IteratorT base_it;
};

template <typename T>
struct offset_to_constant_it
{
  template <typename OffsetT>
  __host__ __device__ __forceinline__ auto operator()(OffsetT offset) const
  {
    return thrust::make_constant_iterator(static_cast<T>(offset));
  }
};

struct object_with_non_trivial_ctor
{
  static constexpr std::int32_t magic_constant = 923390;

  std::int32_t field;
  std::int32_t magic;

  __host__ __device__ object_with_non_trivial_ctor()
  {
    magic = magic_constant;
    field = 0;
  }
  __host__ __device__ object_with_non_trivial_ctor(std::int32_t f)
  {
    magic = magic_constant;
    field = f;
  }

  object_with_non_trivial_ctor(const object_with_non_trivial_ctor& x) = default;

  __host__ __device__ object_with_non_trivial_ctor& operator=(const object_with_non_trivial_ctor& x)
  {
    if (magic == magic_constant)
    {
      field = x.field;
    }
    return *this;
  }
};

static std::ostream& operator<<(std::ostream& os, const object_with_non_trivial_ctor& val)
{
  os << '(' << val.field << ',' << val.magic << ')';
  return os;
}

__host__ __device__ __forceinline__ bool
operator==(const object_with_non_trivial_ctor& lhs, const object_with_non_trivial_ctor& rhs)
{
  return lhs.field == rhs.field && lhs.magic == rhs.magic;
}

C2H_TEST("DeviceCopy::Batched works", "[copy]")
try
{
  // Type used for indexing into the array of ranges
  using range_offset_t = uint32_t;

  // Type used for indexing into individual items of a range (large enough to cover the max range's size)
  using range_size_t = uint32_t;

  // Type used for indexing into items over *all* the ranges' sizes
  using item_offset_t = uint32_t;

  // Total number of items that are targeted to be copied on each run
  constexpr range_offset_t target_copy_size = 32U << 20;

  // Pairs of [min, max] range sizes
  auto range_size_range = GENERATE_COPY(
    table<std::int32_t, std::int32_t>(
      {{0, 1},
       {1, 2},
       {0, 32},
       {1, 1024},
       {1, 32 * 1024},
       {128 * 1024, 256 * 1024},
       {target_copy_size, target_copy_size}}),
    take(4,
         map(
           [](const std::vector<std::int32_t>& chunk) {
             std::int32_t lhs = chunk[0];
             std::int32_t rhs = chunk[1];
             // Optionally ensure lhs < rhs, for example:
             return (lhs < rhs) ? std::make_tuple(lhs, rhs) : std::make_tuple(rhs, lhs);
           },
           chunk(2, random(1, 1000000)))));

  const auto min_range_size       = static_cast<range_size_t>(std::get<0>(range_size_range));
  const auto max_range_size       = static_cast<range_size_t>(std::get<1>(range_size_range));
  const double average_range_size = (min_range_size + max_range_size) / 2.0;
  const auto num_ranges           = static_cast<range_offset_t>(target_copy_size / average_range_size);

  c2h::device_vector<range_size_t> d_range_sizes(num_ranges);

  // Generate the range sizes: Make sure range sizes are a multiple of the most granular unit (one AtomicT) being
  // copied (round down)
  c2h::gen(C2H_SEED(2), d_range_sizes, min_range_size, max_range_size);
  item_offset_t num_total_items = thrust::reduce(d_range_sizes.cbegin(), d_range_sizes.cend());

  // Shuffle output range destination-offsets
  auto d_range_dst_offsets = get_shuffled_buffer_offsets<range_offset_t, item_offset_t>(d_range_sizes, C2H_SEED(1));

  // Generate random input data and initialize output data
  c2h::device_vector<std::uint8_t> d_in(num_total_items);
  c2h::device_vector<std::uint8_t> d_out(num_total_items, 42);
  c2h::gen(C2H_SEED(1), d_in);

  // Prepare host-side input data for verification
  c2h::host_vector<std::uint8_t> h_in(d_in);
  c2h::host_vector<std::uint8_t> h_out(num_total_items);
  c2h::host_vector<range_size_t> h_range_sizes(d_range_sizes);
  c2h::host_vector<item_offset_t> h_dst_offsets(d_range_dst_offsets);

  // Prepare d_range_srcs
  offset_to_constant_it<std::uint8_t> offset_to_index_op{};
  auto d_range_srcs =
    thrust::make_transform_iterator(thrust::make_counting_iterator(range_offset_t{0}), offset_to_index_op);

  // Prepare d_range_dsts
  offset_to_transform_it<std::uint8_t*> dst_transform_op{
    static_cast<std::uint8_t*>(thrust::raw_pointer_cast(d_out.data()))};
  auto d_range_dsts = thrust::make_transform_iterator(d_range_dst_offsets.begin(), dst_transform_op);

  // Invoke device-side algorithm
  copy_batched(d_range_srcs, d_range_dsts, d_range_sizes.begin(), num_ranges);

  // Prepare CPU-side result for verification
  for (range_offset_t i = 0; i < num_ranges; i++)
  {
    auto out_begin = h_out.begin() + h_dst_offsets[i];
    auto out_end   = out_begin + h_range_sizes[i];
    std::fill(out_begin, out_end, static_cast<std::uint8_t>(i));
  }

  REQUIRE(d_out == h_out);
}
catch (std::bad_alloc& e)
{
  std::cerr << "Caught bad_alloc: " << e.what() << std::endl;
}

C2H_TEST("DeviceCopy::Batched works for a very large range", "[copy]")
try
{
  using data_t        = uint64_t;
  using byte_offset_t = uint64_t;
  using buffer_size_t = uint64_t;

  byte_offset_t large_target_copy_size = static_cast<byte_offset_t>(std::numeric_limits<uint32_t>::max()) + (32 << 20);
  byte_offset_t num_items              = large_target_copy_size;

  // Input iterator for the items of a single range
  auto input_data_it = thrust::make_counting_iterator(data_t{42});

  // Prepare helper to check results
  auto check_result_helper = detail::large_problem_test_helper(num_items);
  auto check_result_it     = check_result_helper.get_flagging_output_iterator(input_data_it);

  // Run test
  const auto num_buffers = 1;
  auto d_buffer_srcs     = thrust::make_constant_iterator(input_data_it);
  auto d_buffer_dsts     = thrust::make_constant_iterator(check_result_it);
  auto d_buffer_sizes    = thrust::make_constant_iterator(num_items);
  copy_batched(d_buffer_srcs, d_buffer_dsts, d_buffer_sizes, num_buffers);

  // Verify result
  check_result_helper.check_all_results_correct();
}
catch (std::bad_alloc& e)
{
  std::cerr << "Caught bad_alloc: " << e.what() << std::endl;
}

C2H_TEST("DeviceCopy::Batched works for non-trivial ctors", "[copy]")
{
  using iterator = c2h::device_vector<object_with_non_trivial_ctor>::iterator;

  constexpr std::int32_t num_buffers = 3;
  c2h::device_vector<object_with_non_trivial_ctor> in(num_buffers, object_with_non_trivial_ctor(99));
  c2h::device_vector<object_with_non_trivial_ctor> out(num_buffers);

  c2h::device_vector<iterator> in_iter{in.begin(), in.begin() + 1, in.begin() + 2};
  c2h::device_vector<iterator> out_iter{out.begin(), out.begin() + 1, out.begin() + 2};

  auto sizes = thrust::make_constant_iterator(1);

  copy_batched(in_iter.begin(), out_iter.begin(), sizes, num_buffers);

  REQUIRE(in == out);
}
