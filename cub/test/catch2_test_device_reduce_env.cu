// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

// Should precede any includes
struct stream_registry_factory_t;
#define CUB_DETAIL_DEFAULT_KERNEL_LAUNCHER_FACTORY stream_registry_factory_t

#include "insert_nested_NVTX_range_guard.h"

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include "catch2_test_env_launch_helper.h"

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Reduce, device_reduce);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Sum, device_reduce_sum);

// %PARAM% TEST_LAUNCH lid 0:1:2

#include <cuda/__execution/determinism.h>
#include <cuda/__execution/require.h>

#include <c2h/catch2_test_helper.h>

namespace stdexec = cuda::std::execution;

// Launcher helper always passes an environment.
// We need a test of simple use to check if default environment works.
// ifdef it out not to spend time compiling and running it twice.
#if TEST_LAUNCH == 0
struct block_size_check_t
{
  int* ptr;

  __device__ int operator()(int a, int b)
  {
    *ptr = blockDim.x;
    return a + b;
  }
};

struct block_size_retreiver_t
{
  int* ptr;

  template <class ActivePolicyT>
  hipError_t Invoke()
  {
    *ptr = ActivePolicyT::SingleTilePolicy::BLOCK_THREADS;
    return hipSuccess;
  }
};

TEST_CASE("Device reduce works with default environment", "[reduce][device]")
{
  using num_items_t = int;
  using value_t     = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;
  using policy_t    = hipcub::detail::reduce::default_tuning::fn<value_t, offset_t, block_size_check_t>::MaxPolicy;

  int current_device{};
  REQUIRE(hipSuccess == hipGetDevice(&current_device));

  int ptx_version{};
  REQUIRE(hipSuccess == hipcub::PtxVersion(ptx_version, current_device));

  int target_block_size{};
  block_size_retreiver_t block_size_retreiver{&target_block_size};
  REQUIRE(hipSuccess == policy_t::Invoke(ptx_version, block_size_retreiver));

  num_items_t num_items = 1;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};
  auto d_in  = thrust::make_constant_iterator(value_t{1});
  auto d_out = thrust::device_vector<value_t>(1);

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Reduce(d_in, d_out.begin(), num_items, block_size_check, value_t{0}));
  REQUIRE(d_out[0] == num_items);

  // Make sure we use default tuning
  REQUIRE(d_block_size[0] == target_block_size);
}

TEST_CASE("Device sum works with default environment", "[reduce][device]")
{
  using num_items_t = int;
  using value_t     = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;

  int current_device{};
  REQUIRE(hipSuccess == hipGetDevice(&current_device));

  int ptx_version{};
  REQUIRE(hipSuccess == hipcub::PtxVersion(ptx_version, current_device));

  num_items_t num_items = 1;

  auto d_in  = thrust::make_constant_iterator(value_t{1});
  auto d_out = thrust::device_vector<value_t>(1);

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Sum(d_in, d_out.begin(), num_items));
  REQUIRE(d_out[0] == num_items);
}

template <int BlockThreads>
struct reduce_tuning : hipcub::detail::reduce::tuning<reduce_tuning<BlockThreads>>
{
  template <class /* AccumT */, class /* Offset */, class /* OpT */>
  struct fn
  {
    struct Policy500 : hipcub::ChainedPolicy<500, Policy500, Policy500>
    {
      struct ReducePolicy
      {
        static constexpr int VECTOR_LOAD_LENGTH = 1;

        static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BLOCK_REDUCE_WARP_REDUCTIONS;

        static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::LOAD_DEFAULT;

        static constexpr int ITEMS_PER_THREAD = 1;
        static constexpr int BLOCK_THREADS    = BlockThreads;
      };

      using SingleTilePolicy      = ReducePolicy;
      using SegmentedReducePolicy = ReducePolicy;
    };

    using MaxPolicy = Policy500;
  };
};

struct get_scan_tuning_query_t
{};

struct scan_tuning
{
  [[nodiscard]] _CCCL_TRIVIAL_API constexpr auto query(const get_scan_tuning_query_t&) const noexcept
  {
    return *this;
  }

  // Make sure this is not used
  template <class /* AccumT */, class /* Offset */, class /* OpT */>
  struct fn
  {};
};

using block_sizes = c2h::type_list<cuda::std::integral_constant<int, 32>, cuda::std::integral_constant<int, 64>>;

C2H_TEST("Device reduce can be tuned", "[reduce][device]", block_sizes)
{
  constexpr int target_block_size = c2h::get<0, TestType>::value;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};

  auto num_items = 1;
  auto d_in      = thrust::make_constant_iterator(1);
  auto d_out     = thrust::device_vector<int>(1);

  // We are expecting that `scan_tuning` is ignored
  auto env = cuda::execution::__tune(reduce_tuning<target_block_size>{}, scan_tuning{});

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Reduce(d_in, d_out.begin(), num_items, block_size_check, 0, env));
  REQUIRE(d_out[0] == num_items);
  REQUIRE(d_block_size[0] == target_block_size);
}

C2H_TEST("Device sum can be tuned", "[reduce][device]", block_sizes)
{
  constexpr int target_block_size = c2h::get<0, TestType>::value;

  auto num_items = 1;
  auto d_in      = thrust::make_constant_iterator(1);
  auto d_out     = thrust::device_vector<int>(1);

  // We are expecting that `scan_tuning` is ignored
  auto env = cuda::execution::__tune(reduce_tuning<target_block_size>{}, scan_tuning{});

  REQUIRE(hipSuccess == hipcub::DeviceReduce::Sum(d_in, d_out.begin(), num_items, env));
  REQUIRE(d_out[0] == num_items);
}
#endif

using requirements =
  c2h::type_list<cuda::execution::determinism::gpu_to_gpu_t,
                 cuda::execution::determinism::run_to_run_t,
                 cuda::execution::determinism::not_guaranteed_t>;

C2H_TEST("Device reduce uses environment", "[reduce][device]", requirements)
{
  using determinism_t = c2h::get<0, TestType>;
  using accumulator_t = float;
  using op_t          = cuda::std::plus<>;
  using num_items_t   = int;
  using offset_t      = hipcub::detail::choose_offset_t<num_items_t>;
  using transform_t   = ::cuda::std::identity;
  using init_t        = accumulator_t;

  num_items_t num_items = GENERATE(1 << 4, 1 << 24);
  auto d_in             = thrust::make_constant_iterator(1.0f);
  auto d_out            = thrust::device_vector<accumulator_t>(1);

  init_t init = 0;
  size_t expected_bytes_allocated{};

  // To check if a given algorithm implementation is used, we check if associated kernels are invoked.
  auto kernels = [&]() {
    // TODO(gevtushenko): split `not_guaranteed` kernels once atomic reduce is merged
    if constexpr (std::is_same_v<determinism_t, cuda::execution::determinism::run_to_run_t>
                  || std::is_same_v<determinism_t, cuda::execution::determinism::not_guaranteed_t>)
    {
      REQUIRE(
        hipSuccess
        == hipcub::DeviceReduce::Reduce(nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items, op_t{}, init));

      using policy_t = hipcub::detail::reduce::policy_hub<accumulator_t, offset_t, op_t>::MaxPolicy;
      return cuda::std::array<void*, 3>{
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            decltype(d_in),
            decltype(d_out.begin()),
            offset_t,
            op_t,
            init_t,
            accumulator_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceKernel<policy_t, decltype(d_in), offset_t, op_t, accumulator_t, transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            accumulator_t*,
            decltype(d_out.begin()),
            int, // always used with int offset
            op_t,
            init_t,
            accumulator_t>)};
    }
    else
    {
      using policy_t              = hipcub::detail::rfa::policy_hub<accumulator_t, offset_t, op_t>::MaxPolicy;
      using deterministic_add_t   = hipcub::detail::rfa::deterministic_sum_t<accumulator_t>;
      using reduction_op_t        = deterministic_add_t;
      using deterministic_accum_t = deterministic_add_t::DeterministicAcc;
      using output_it_t = thrust::transform_output_iterator<hipcub::detail::rfa::rfa_float_transform_t<accumulator_t>,
                                                            decltype(d_out.begin())>;

      using dispatch_t = hipcub::detail::
        DispatchReduceDeterministic<decltype(d_in), decltype(d_out.begin()), offset_t, init_t, transform_t, accumulator_t>;

      REQUIRE(
        hipSuccess == dispatch_t::Dispatch(nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items, init));

      return cuda::std::array<void*, 3>{
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceSingleTileKernel<
            policy_t,
            decltype(d_in),
            output_it_t,
            offset_t,
            reduction_op_t,
            init_t,
            deterministic_accum_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceKernel<
            policy_t,
            decltype(d_in),
            offset_t,
            reduction_op_t,
            deterministic_accum_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceSingleTileKernel<
            policy_t,
            accumulator_t*,
            output_it_t,
            int, // always used with int offset
            reduction_op_t,
            init_t,
            deterministic_accum_t,
            transform_t>)};
    }
  }();

  // Equivalent to `cuexec::require(cuexec::determinism::run_to_run)` and
  //               `cuexec::require(cuexec::determinism::not_guaranteed)`
  auto env = stdexec::env{cuda::execution::require(determinism_t{}), // determinism
                          allowed_kernels(kernels), // allowed kernels for the given determinism
                          expected_allocation_size(expected_bytes_allocated)}; // temp storage size

  device_reduce(d_in, d_out.begin(), num_items, op_t{}, init, env);

  REQUIRE(d_out[0] == num_items);
}

C2H_TEST("Device sum uses environment", "[reduce][device]", requirements)
{
  using determinism_t = c2h::get<0, TestType>;
  using accumulator_t = float;
  using op_t          = cuda::std::plus<>;
  using num_items_t   = int;
  using offset_t      = hipcub::detail::choose_offset_t<num_items_t>;
  using transform_t   = ::cuda::std::identity;
  using init_t        = accumulator_t;

  num_items_t num_items = GENERATE(1 << 4, 1 << 24);
  auto d_in             = thrust::make_constant_iterator(1.0f);
  auto d_out            = thrust::device_vector<accumulator_t>(1);

  size_t expected_bytes_allocated{};

  // To check if a given algorithm implementation is used, we check if associated kernels are invoked.
  auto kernels = [&]() {
    // TODO(gevtushenko): split `not_guaranteed` kernels once atomic reduce is merged
    if constexpr (std::is_same_v<determinism_t, cuda::execution::determinism::run_to_run_t>
                  || std::is_same_v<determinism_t, cuda::execution::determinism::not_guaranteed_t>)
    {
      REQUIRE(hipSuccess == hipcub::DeviceReduce::Sum(nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items));

      using policy_t = hipcub::detail::reduce::policy_hub<accumulator_t, offset_t, op_t>::MaxPolicy;
      return cuda::std::array<void*, 3>{
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            decltype(d_in),
            decltype(d_out.begin()),
            offset_t,
            op_t,
            init_t,
            accumulator_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceKernel<policy_t, decltype(d_in), offset_t, op_t, accumulator_t, transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeviceReduceSingleTileKernel<
            policy_t,
            accumulator_t*,
            decltype(d_out.begin()),
            int, // always used with int offset
            op_t,
            init_t,
            accumulator_t>)};
    }
    else
    {
      using policy_t              = hipcub::detail::rfa::policy_hub<accumulator_t, offset_t, op_t>::MaxPolicy;
      using deterministic_add_t   = hipcub::detail::rfa::deterministic_sum_t<accumulator_t>;
      using reduction_op_t        = deterministic_add_t;
      using deterministic_accum_t = deterministic_add_t::DeterministicAcc;
      using output_it_t = thrust::transform_output_iterator<hipcub::detail::rfa::rfa_float_transform_t<accumulator_t>,
                                                            decltype(d_out.begin())>;

      using dispatch_t = hipcub::detail::
        DispatchReduceDeterministic<decltype(d_in), decltype(d_out.begin()), offset_t, init_t, transform_t, accumulator_t>;

      REQUIRE(hipSuccess
              == dispatch_t::Dispatch(nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items, init_t{}));

      return cuda::std::array<void*, 3>{
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceSingleTileKernel<
            policy_t,
            decltype(d_in),
            output_it_t,
            offset_t,
            reduction_op_t,
            init_t,
            deterministic_accum_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceKernel<
            policy_t,
            decltype(d_in),
            offset_t,
            reduction_op_t,
            deterministic_accum_t,
            transform_t>),
        reinterpret_cast<void*>(
          hipcub::detail::reduce::DeterministicDeviceReduceSingleTileKernel<
            policy_t,
            accumulator_t*,
            output_it_t,
            int, // always used with int offset
            reduction_op_t,
            init_t,
            deterministic_accum_t,
            transform_t>)};
    }
  }();

  // Equivalent to `cuexec::require(cuexec::determinism::run_to_run)` and
  //               `cuexec::require(cuexec::determinism::not_guaranteed)`
  auto env = stdexec::env{cuda::execution::require(determinism_t{}), // determinism
                          allowed_kernels(kernels), // allowed kernels for the given determinism
                          expected_allocation_size(expected_bytes_allocated)}; // temp storage size

  device_reduce_sum(d_in, d_out.begin(), num_items, env);

  REQUIRE(d_out[0] == num_items);
}
