// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

// Should precede any includes
struct stream_registry_factory_t;
#define CUB_DETAIL_DEFAULT_KERNEL_LAUNCHER_FACTORY stream_registry_factory_t

#include "insert_nested_NVTX_range_guard.h"

#include <cub/block/block_load.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/block/block_store.cuh>
#include <cub/device/device_scan.cuh>

#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include "catch2_test_env_launch_helper.h"

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveScan, device_scan_exclusive);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveSum, device_scan_exclusive_sum);

// %PARAM% TEST_LAUNCH lid 0:1:2

#include <cuda/__execution/determinism.h>
#include <cuda/__execution/require.h>

#include <c2h/catch2_test_helper.h>

namespace stdexec = cuda::std::execution;

// Launcher helper always passes an environment.
// We need a test of simple use to check if default environment works.
// ifdef it out not to spend time compiling and running it twice.
// #if TEST_LAUNCH == 0
#if 0
struct block_size_check_t
{
  int* ptr;

  __device__ int operator()(int a, int b)
  {
    *ptr = blockDim.x;
    return a + b;
  }
};

struct block_size_retreiver_t
{
  int* ptr;

  template <class ActivePolicyT>
  hipError_t Invoke()
  {
    *ptr = ActivePolicyT::ScanPolicyT::BLOCK_THREADS;
    return hipSuccess;
  }
};

TEST_CASE("Device scan exclusive scan works with default environment", "[scan][device]")
{
  using num_items_t = int;
  using value_t     = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;

  using policy_t =
    hipcub::detail::scan::default_tuning::fn<value_t, value_t, value_t, offset_t, block_size_check_t>::MaxPolicy;

  int current_device{};
  REQUIRE(hipSuccess == hipGetDevice(&current_device));

  int ptx_version{};
  REQUIRE(hipSuccess == hipcub::PtxVersion(ptx_version, current_device));

  int target_block_size{};
  block_size_retreiver_t block_size_retreiver{&target_block_size};
  REQUIRE(hipSuccess == policy_t::Invoke(ptx_version, block_size_retreiver));

  num_items_t num_items = 1;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};
  auto d_in  = thrust::make_constant_iterator(value_t{1});
  auto d_out = thrust::device_vector<value_t>(1);

  auto init = value_t{0};
  REQUIRE(hipSuccess == hipcub::DeviceScan::ExclusiveScan(d_in, d_out.begin(), block_size_check, init, num_items));
  REQUIRE(d_out[0] == init);

  // Make sure we use default tuning
  REQUIRE(d_block_size[0] == target_block_size);
}

TEST_CASE("Device scan exclusive sum works with default environment", "[sum][device]")
{
  using num_items_t = int;
  using value_t     = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;

  int current_device{};
  REQUIRE(hipSuccess == hipGetDevice(&current_device));

  int ptx_version{};
  REQUIRE(hipSuccess == hipcub::PtxVersion(ptx_version, current_device));

  num_items_t num_items = 1;

  auto d_in  = thrust::make_constant_iterator(value_t{1});
  auto d_out = thrust::device_vector<value_t>(1);

  REQUIRE(hipSuccess == hipcub::DeviceScan::ExclusiveSum(d_in, d_out.begin(), num_items));
  REQUIRE(d_out[0] == value_t{0});
}

template <int BlockThreads>
struct scan_tuning : hipcub::detail::scan::tuning<scan_tuning<BlockThreads>>
{
  template <class /* InputValueT */, class /* OutputValueT */, class AccumT, class /* Offset */, class /* ScanOpT */>
  struct fn
  {
    struct Policy500 : hipcub::ChainedPolicy<500, Policy500, Policy500>
    {
      struct ScanPolicyT
      {
        static constexpr int BLOCK_THREADS                      = BlockThreads;
        static constexpr int ITEMS_PER_THREAD                   = 1;
        static constexpr hipcub::BlockLoadAlgorithm LOAD_ALGORITHM = hipcub::BlockLoadAlgorithm::BLOCK_LOAD_WARP_TRANSPOSE;

        static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::CacheLoadModifier::LOAD_DEFAULT;
        static constexpr hipcub::BlockStoreAlgorithm STORE_ALGORITHM =
          hipcub::BlockStoreAlgorithm::BLOCK_STORE_WARP_TRANSPOSE;
        static constexpr hipcub::BlockScanAlgorithm SCAN_ALGORITHM = hipcub::BlockScanAlgorithm::BLOCK_SCAN_RAKING;

        struct detail
        {
          using delay_constructor_t = hipcub::detail::default_delay_constructor_t<AccumT>;
        };
      };
    };

    using MaxPolicy = Policy500;
  };
};

struct get_reduce_tuning_query_t
{};

struct reduce_tuning
{
  [[nodiscard]] _CCCL_NODEBUG_API constexpr auto query(const get_reduce_tuning_query_t&) const noexcept
  {
    return *this;
  }

  // Make sure this is not used
  template <class /* InputValueT */,
            class /* OutputValueT */,
            class /* AccumT */,
            class /* Offset */,
            class /* ScanOpT */>
  struct fn
  {};
};

using block_sizes = c2h::type_list<cuda::std::integral_constant<int, 32>, cuda::std::integral_constant<int, 64>>;

C2H_TEST("Device scan exclusive-scan can be tuned", "[scan][device]", block_sizes)
{
  constexpr int target_block_size = c2h::get<0, TestType>::value;
  c2h::device_vector<int> d_block_size(1);
  block_size_check_t block_size_check{thrust::raw_pointer_cast(d_block_size.data())};

  auto num_items = 3;
  auto d_in      = thrust::make_constant_iterator(1);
  auto d_out     = thrust::device_vector<int>(num_items);

  // We are expecting that `reduce_tuning` is ignored
  auto env = cuda::execution::__tune(scan_tuning<target_block_size>{}, reduce_tuning{});

  REQUIRE(hipSuccess == hipcub::DeviceScan::ExclusiveScan(d_in, d_out.begin(), block_size_check, 0, num_items, env));

  for (int i = 0; i < num_items; i++)
  {
    REQUIRE(d_out[i] == i);
  }
  REQUIRE(d_block_size[0] == target_block_size);
}

C2H_TEST("Device scan exclusive-sum can be tuned", "[scan][device]", block_sizes)
{
  constexpr int target_block_size = c2h::get<0, TestType>::value;

  auto num_items = target_block_size;
  auto d_in      = thrust::make_constant_iterator(1);
  auto d_out     = thrust::device_vector<int>(num_items);

  // We are expecting that `reduce_tuning` is ignored
  auto env = cuda::execution::__tune(scan_tuning<target_block_size>{}, reduce_tuning{});

  REQUIRE(hipSuccess == hipcub::DeviceScan::ExclusiveSum(d_in, d_out.begin(), num_items, env));

  for (int i = 0; i < num_items; i++)
  {
    REQUIRE(d_out[i] == i);
  }
}

#endif

using requirements = c2h::type_list<cuda::execution::determinism::run_to_run_t>;

C2H_TEST("Device scan exclusive-scan uses environment", "[scan][device]", requirements)
{
  using determinism_t = c2h::get<0, TestType>;

  using scan_op_t   = cuda::std::plus<>;
  using num_items_t = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;

  num_items_t num_items = 10;
  auto d_in             = thrust::make_constant_iterator(1.0f);
  auto d_out            = thrust::device_vector<float>(num_items);

  using input_it_t  = decltype(d_in);
  using output_it_t = decltype(d_out.begin());

  using init_t        = hipcub::detail::it_value_t<input_it_t>;
  using input_value_t = hipcub::detail::InputValue<init_t>;

  using accum_t =
    ::cuda::std::__accumulator_t<scan_op_t,
                                 hipcub::detail::it_value_t<input_it_t>,
                                 ::cuda::std::_If<::cuda::std::is_same_v<input_value_t, hipcub::NullType>,
                                                  hipcub::detail::it_value_t<input_it_t>,
                                                  typename input_value_t::value_type>>;

  init_t init{};
  size_t expected_bytes_allocated{};

  // To check if a given algorithm implementation is used, we check if associated kernels are invoked.
  auto kernels = [&]() {
    if constexpr (cuda::std::is_same_v<determinism_t, cuda::execution::determinism::run_to_run_t>)
    {
      REQUIRE(hipSuccess
              == hipcub::DeviceScan::ExclusiveScan(
                nullptr, expected_bytes_allocated, d_in, d_out.begin(), scan_op_t{}, init, num_items));

      using policy_t =
        hipcub::detail::scan::policy_hub<hipcub::detail::it_value_t<input_it_t>,
                                      hipcub::detail::it_value_t<output_it_t>,
                                      accum_t,
                                      offset_t,
                                      scan_op_t>::MaxPolicy;

      using scan_tile_state_t = typename hipcub::ScanTileState<accum_t>;

      auto kernel1 = reinterpret_cast<void*>(
        hipcub::detail::scan::DeviceScanKernel<
          policy_t,
          input_it_t,
          output_it_t,
          scan_tile_state_t,
          scan_op_t,
          hipcub::detail::InputValue<init_t>,
          offset_t,
          accum_t,
          false,
          input_value_t::value_type>);

      auto kernel2 = reinterpret_cast<void*>(hipcub::detail::scan::DeviceScanInitKernel<scan_tile_state_t>);

      return cuda::std::array<void*, 2>{kernel1, kernel2};
    }
  }();

  // Equivalent to `cuexec::require(cuexec::determinism::run_to_run)` and
  //               `cuexec::require(cuexec::determinism::not_guaranteed)`
  auto env = stdexec::env{cuda::execution::require(determinism_t{}), // determinism
                          allowed_kernels(kernels), // allowed kernels for the given determinism
                          expected_allocation_size(expected_bytes_allocated)}; // temp storage size

  device_scan_exclusive(d_in, d_out.begin(), scan_op_t{}, init, num_items, env);

  for (int i = 0; i < num_items; i++)
  {
    REQUIRE(d_out[i] == i);
  }
}

C2H_TEST("Device scan exclusive-sum uses environment", "[scan][device]", requirements)
{
  using determinism_t = c2h::get<0, TestType>;

  using scan_op_t   = cuda::std::plus<>;
  using num_items_t = int;
  using offset_t    = hipcub::detail::choose_offset_t<num_items_t>;

  num_items_t num_items = 10;
  auto d_in             = thrust::make_constant_iterator(1.0f);
  auto d_out            = thrust::device_vector<float>(num_items);

  using input_it_t  = decltype(d_in);
  using output_it_t = decltype(d_out.begin());

  using init_t        = hipcub::detail::it_value_t<input_it_t>;
  using input_value_t = hipcub::detail::InputValue<init_t>;

  using accum_t =
    ::cuda::std::__accumulator_t<scan_op_t,
                                 hipcub::detail::it_value_t<input_it_t>,
                                 ::cuda::std::_If<::cuda::std::is_same_v<input_value_t, hipcub::NullType>,
                                                  hipcub::detail::it_value_t<input_it_t>,
                                                  typename input_value_t::value_type>>;

  size_t expected_bytes_allocated{};

  // To check if a given algorithm implementation is used, we check if associated kernels are invoked.
  auto kernels = [&]() {
    if constexpr (cuda::std::is_same_v<determinism_t, cuda::execution::determinism::run_to_run_t>)
    {
      REQUIRE(hipSuccess
              == hipcub::DeviceScan::ExclusiveSum(nullptr, expected_bytes_allocated, d_in, d_out.begin(), num_items));

      using policy_t =
        hipcub::detail::scan::policy_hub<hipcub::detail::it_value_t<input_it_t>,
                                      hipcub::detail::it_value_t<output_it_t>,
                                      accum_t,
                                      offset_t,
                                      scan_op_t>::MaxPolicy;

      using scan_tile_state_t = typename hipcub::ScanTileState<accum_t>;

      auto kernel1 = reinterpret_cast<void*>(
        hipcub::detail::scan::DeviceScanKernel<
          policy_t,
          input_it_t,
          output_it_t,
          scan_tile_state_t,
          scan_op_t,
          hipcub::detail::InputValue<init_t>,
          offset_t,
          accum_t,
          false,
          input_value_t::value_type>);

      auto kernel2 = reinterpret_cast<void*>(hipcub::detail::scan::DeviceScanInitKernel<scan_tile_state_t>);

      return cuda::std::array<void*, 2>{kernel1, kernel2};
    }
  }();

  // Equivalent to `cuexec::require(cuexec::determinism::run_to_run)` and
  //               `cuexec::require(cuexec::determinism::not_guaranteed)`
  auto env = stdexec::env{cuda::execution::require(determinism_t{}), // determinism
                          allowed_kernels(kernels), // allowed kernels for the given determinism
                          expected_allocation_size(expected_bytes_allocated)}; // temp storage size

  device_scan_exclusive_sum(d_in, d_out.begin(), num_items, env);

  for (int i = 0; i < num_items; i++)
  {
    REQUIRE(d_out[i] == i);
  }
}
