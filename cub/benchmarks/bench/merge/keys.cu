// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_merge.cuh>

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/iterator/tabulate_output_iterator.h>
#include <thrust/sort.h>

#include <cuda/std/utility>

#include <cstdint>

#include "merge_common.cuh"
#include <nvbench_helper.cuh>

// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:2:1
// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK_POW2 tpb 6:10:1

template <typename KeyT, typename OffsetT>
void keys(nvbench::state& state, nvbench::type_list<KeyT, OffsetT>)
{
  using key_t            = KeyT;
  using value_t          = hipcub::NullType;
  using key_input_it_t   = key_t*;
  using value_input_it_t = value_t*;
  using key_it_t         = key_t*;
  using value_it_t       = value_t*;
  using offset_t         = OffsetT;
  using compare_op_t     = less_t;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t>;
  using dispatch_t = hipcub::hipcub::detail::merge::
    dispatch_t<key_it_t, value_it_t, key_it_t, value_it_t, key_it_t, value_it_t, offset_t, compare_op_t, policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::detail::merge::
    dispatch_t<key_it_t, value_it_t, key_it_t, value_it_t, key_it_t, value_it_t, offset_t, compare_op_t>;
#endif // TUNE_BASE

  // Retrieve axis parameters
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  // We generate data distributions in the range [0, 255], which, with lower entropy, get skewed towards 0.
  // We use this to generate increasingly large *consecutive* segments of data that are getting selected from the lhs
  thrust::device_vector<uint8_t> rnd_selector_val = generate(elements, entropy);
  uint8_t threshold                               = 128;
  select_if_less_than_t select_lhs_op{false, threshold};
  select_if_less_than_t select_rhs_op{true, threshold};

  // The following algorithm only works under the precondition that there's at least 50% of the data in the lhs
  // If that's not the case, we simply swap the logic for selecting into lhs and rhs
  const auto num_items_selected_into_lhs =
    static_cast<offset_t>(thrust::count_if(rnd_selector_val.begin(), rnd_selector_val.end(), select_lhs_op));
  if (num_items_selected_into_lhs < elements / 2)
  {
    using ::cuda::std::swap;
    swap(select_lhs_op, select_rhs_op);
  }

  // We want lhs and rhs to be of equal size. We also want to have skewed distributions, such that we put different
  // workloads on the binary search part. For this reason, we identify the index from the input, referred to as pivot
  // point, after which the lhs is "full". We compose the rhs by selecting all items up to the pivot point that were not
  // selected for lhs and *all* items after the pivot point.
  constexpr std::size_t num_pivot_points = 1;
  thrust::device_vector<offset_t> pivot_point(num_pivot_points);
  const auto num_items_lhs = elements / 2;
  const auto num_items_rhs = elements - num_items_lhs;
  auto counting_it         = thrust::make_counting_iterator(offset_t{0});
  thrust::copy_if(
    counting_it,
    counting_it + elements,
    rnd_selector_val.begin(),
    thrust::make_tabulate_output_iterator(write_pivot_point_t<offset_t>{
      static_cast<offset_t>(num_items_lhs), thrust::raw_pointer_cast(pivot_point.data())}),
    select_lhs_op);

  thrust::device_vector<key_t> keys_lhs(num_items_lhs);
  thrust::device_vector<key_t> keys_rhs(num_items_rhs);
  thrust::device_vector<key_t> keys_out(elements);

  // Generate increasing input range to sample from
  thrust::device_vector<key_t> increasing_input = generate(elements);
  thrust::sort(increasing_input.begin(), increasing_input.end());

  // Select lhs from input up to pivot point
  offset_t pivot_point_val = pivot_point[0];
  auto const end_lhs       = thrust::copy_if(
    increasing_input.cbegin(),
    increasing_input.cbegin() + pivot_point_val,
    rnd_selector_val.cbegin(),
    keys_lhs.begin(),
    select_lhs_op);

  // Select rhs items from input up to pivot point
  auto const end_rhs = thrust::copy_if(
    increasing_input.cbegin(),
    increasing_input.cbegin() + pivot_point_val,
    rnd_selector_val.cbegin(),
    keys_rhs.begin(),
    select_rhs_op);
  // From pivot point copy all remaining items to rhs
  thrust::copy(increasing_input.cbegin() + pivot_point_val, increasing_input.cbegin() + elements, end_rhs);

  key_t* d_keys_lhs = thrust::raw_pointer_cast(keys_lhs.data());
  key_t* d_keys_rhs = thrust::raw_pointer_cast(keys_rhs.data());
  key_t* d_keys_out = thrust::raw_pointer_cast(keys_out.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<KeyT>(elements);
  state.add_global_memory_writes<KeyT>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::dispatch(
    nullptr,
    temp_size,
    d_keys_lhs,
    nullptr,
    num_items_lhs,
    d_keys_rhs,
    nullptr,
    num_items_rhs,
    d_keys_out,
    nullptr,
    compare_op_t{},
    hipStream_t{});

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::dispatch(
      temp_storage,
      temp_size,
      d_keys_lhs,
      nullptr,
      num_items_lhs,
      d_keys_rhs,
      nullptr,
      num_items_rhs,
      d_keys_out,
      nullptr,
      compare_op_t{},
      launch.get_stream());
  });
}

#ifdef TUNE_KeyT
using key_types = nvbench::type_list<TUNE_KeyT>;
#else // !defined(TUNE_KeyT)
using key_types = fundamental_types;
#endif // TUNE_KeyT

NVBENCH_BENCH_TYPES(keys, NVBENCH_TYPE_AXES(key_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"});
