// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_merge_sort.cuh>

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/iterator/tabulate_output_iterator.h>

#include <cuda/std/utility>

#include <cstdint>

#include "thrust/detail/raw_pointer_cast.h"
#include <nvbench_helper.cuh>

template <typename T>
struct to_key_t
{
  template <typename IndexT>
  __host__ __device__ __forceinline__ T operator()(IndexT index) const
  {
    return static_cast<T>(index);
  }
};

struct select_if_less_than_t
{
  bool negate;
  uint8_t threshold;

  __host__ __device__ __forceinline__ bool operator()(uint8_t val) const
  {
    return negate ? !(val < threshold) : val < threshold;
  }
};

template <typename OffsetT>
struct write_pivot_point_t
{
  OffsetT threshold;
  OffsetT* pivot_point;

  __host__ __device__ __forceinline__ void operator()(OffsetT output_index, OffsetT input_index) const
  {
    if (output_index == threshold)
    {
      *pivot_point = input_index;
    }
  }
};

// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:2:1
// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK_POW2 tpb 6:10:1

#ifndef TUNE_BASE
#  define TUNE_THREADS_PER_BLOCK (1 << TUNE_THREADS_PER_BLOCK_POW2)
#endif

#if !TUNE_BASE
#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_DIRECT
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

#  if TUNE_LOAD == 0
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_DEFAULT
#  elif TUNE_LOAD == 1
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_LDG
#  else // TUNE_LOAD == 2
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_CA
#  endif // TUNE_LOAD

template <typename KeyT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using merge_policy =
      hipcub::agent_policy_t<TUNE_THREADS_PER_BLOCK,
                          hipcub::Nominal4BItemsToItems<KeyT>(TUNE_ITEMS_PER_THREAD),
                          TUNE_LOAD_ALGORITHM,
                          TUNE_LOAD_MODIFIER,
                          TUNE_STORE_ALGORITHM>;
  };

  using MaxPolicy = policy_t;
};
#endif // TUNE_BASE

template <typename KeyT, typename ValueT, typename OffsetT>
void pairs(nvbench::state& state, nvbench::type_list<KeyT, ValueT, OffsetT>)
{
  using key_t            = KeyT;
  using value_t          = ValueT;
  using key_input_it_t   = key_t*;
  using value_input_it_t = value_t*;
  using key_it_t         = key_t*;
  using value_it_t       = value_t*;
  using offset_t         = OffsetT;
  using compare_op_t     = less_t;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t>;
  using dispatch_t = cub::
    DispatchMergeSort<key_it_t, value_it_t, key_it_t, value_it_t, key_it_t, value_it_t, offset_t, compare_op_t, policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::detail::merge::
    dispatch_t<key_it_t, value_it_t, key_it_t, value_it_t, key_it_t, value_it_t, offset_t, compare_op_t>;
#endif // TUNE_BASE

  // Retrieve axis parameters
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  // We generate data distributions in the range [0, 255] that, with lower entropy, get skewed towards 0
  // We use this to generate increasingly large *consecutive* segments of data that are getting selected from the lhs
  thrust::device_vector<uint8_t> rnd_selector_val = generate(elements, entropy);
  uint8_t threshold                               = 128;
  select_if_less_than_t select_lhs_op{false, threshold};
  select_if_less_than_t select_rhs_op{true, threshold};

  // The following algorithm only works under the precondition that there's at least 50% of the data in the lhs
  // If that's not the case, we simply swap the logic for selecting into lhs and rhs
  const auto num_items_selected_into_lhs =
    static_cast<offset_t>(thrust::count_if(rnd_selector_val.begin(), rnd_selector_val.end(), select_lhs_op));
  if (num_items_selected_into_lhs < elements / 2)
  {
    using ::cuda::std::swap;
    swap(select_lhs_op, select_rhs_op);
  }

  // We want lhs and rhs to be of equal size. We also want to have skewed distributions, such that we put different
  // workloads on the binary search part. For this reason, we identify the index from the input, referred to as pivot
  // point, after which the lhs is "full". We compose the rhs by selecting all unselected items up to the pivot point
  // and *all* items after the pivot point.
  constexpr std::size_t num_pivot_points = 1;
  thrust::device_vector<offset_t> pivot_point(num_pivot_points);
  const auto num_items_lhs = elements / 2;
  const auto num_items_rhs = elements - num_items_lhs;
  auto counting_it         = thrust::make_counting_iterator(offset_t{0});
  thrust::copy_if(
    counting_it,
    counting_it + elements,
    rnd_selector_val.begin(),
    thrust::make_tabulate_output_iterator(write_pivot_point_t<offset_t>{
      static_cast<offset_t>(num_items_lhs), thrust::raw_pointer_cast(pivot_point.data())}),
    select_lhs_op);

  thrust::device_vector<key_t> keys_lhs(num_items_lhs);
  thrust::device_vector<key_t> keys_rhs(num_items_rhs);
  thrust::device_vector<key_t> keys_out(elements);
  thrust::device_vector<value_t> values_lhs(num_items_lhs);
  thrust::device_vector<value_t> values_rhs(num_items_rhs);
  thrust::device_vector<value_t> values_out(elements);

  // Fancy iterator to generate key_t in strictly increasing order
  auto data_gen_it = thrust::make_transform_iterator(counting_it, to_key_t<key_t>{});

  // Select lhs from input up to pivot point
  offset_t pivot_point_val = pivot_point[0];
  auto const end_lhs       = thrust::copy_if(
    data_gen_it, data_gen_it + pivot_point_val, rnd_selector_val.cbegin(), keys_lhs.begin(), select_lhs_op);
  // Select rhs items from input up to pivot point
  auto const end_rhs = thrust::copy_if(
    data_gen_it, data_gen_it + pivot_point_val, rnd_selector_val.cbegin(), keys_rhs.begin(), select_rhs_op);
  // From pivot point copy all remaining items to rhs
  thrust::copy(data_gen_it + pivot_point_val, data_gen_it + elements, end_rhs);

  key_t* d_keys_lhs     = thrust::raw_pointer_cast(keys_lhs.data());
  key_t* d_keys_rhs     = thrust::raw_pointer_cast(keys_rhs.data());
  key_t* d_keys_out     = thrust::raw_pointer_cast(keys_out.data());
  value_t* d_values_lhs = thrust::raw_pointer_cast(values_lhs.data());
  value_t* d_values_rhs = thrust::raw_pointer_cast(values_rhs.data());
  value_t* d_values_out = thrust::raw_pointer_cast(values_out.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<KeyT>(elements);
  state.add_global_memory_reads<ValueT>(elements);
  state.add_global_memory_writes<KeyT>(elements);
  state.add_global_memory_writes<ValueT>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::dispatch(
    nullptr,
    temp_size,
    d_keys_lhs,
    d_values_lhs,
    num_items_lhs,
    d_keys_rhs,
    d_values_rhs,
    num_items_rhs,
    d_keys_out,
    d_values_out,
    compare_op_t{},
    hipStream_t{});

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::dispatch(
      temp_storage,
      temp_size,
      d_keys_lhs,
      d_values_lhs,
      num_items_lhs,
      d_keys_rhs,
      d_values_rhs,
      num_items_rhs,
      d_keys_out,
      d_values_out,
      compare_op_t{},
      launch.get_stream());
  });
}

#ifdef TUNE_KeyT
using key_types = nvbench::type_list<TUNE_KeyT>;
#else // !defined(TUNE_KeyT)
using key_types = fundamental_types;
#endif // TUNE_KeyT

#ifdef TUNE_ValueT
using value_types = nvbench::type_list<TUNE_ValueT>;
#else // !defined(TUNE_ValueT)
using value_types = nvbench::type_list<int8_t, int16_t, int32_t, int64_t
#  if NVBENCH_HELPER_HAS_I128
// nvcc currently hangs for __int128 value type with the fallback policy of {CTA: 64, IPT: 1}. NVBug 4384075
//  ,
//  int128_t
#  endif
                                       >;
#endif // TUNE_ValueT

// using key_types = nvbench::type_list<int32_t>;
// using value_types = nvbench::type_list<int32_t>;

NVBENCH_BENCH_TYPES(pairs, NVBENCH_TYPE_AXES(key_types, value_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "ValueT{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"});
