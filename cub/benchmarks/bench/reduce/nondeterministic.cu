// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/dispatch/dispatch_reduce_nondeterministic.cuh>

#include <nvbench_helper.cuh>

#include <nvbench/range.cuh>
#include <nvbench/types.cuh>

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 3:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32
// %RANGE% TUNE_ITEMS_PER_VEC_LOAD_POW2 ipv 1:2:1

#ifndef TUNE_BASE
#  define TUNE_ITEMS_PER_VEC_LOAD (1 << TUNE_ITEMS_PER_VEC_LOAD_POW2)
#endif

#if !TUNE_BASE
template <typename AccumT, typename OffsetT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int threads_per_block  = TUNE_THREADS_PER_BLOCK;
    static constexpr int items_per_thread   = TUNE_ITEMS_PER_THREAD;
    static constexpr int items_per_vec_load = TUNE_ITEMS_PER_VEC_LOAD;

    using ReducePolicy =
      hipcub::AgentReducePolicy<threads_per_block,
                             items_per_thread,
                             AccumT,
                             items_per_vec_load,
                             hipcub::BLOCK_REDUCE_WARP_REDUCTIONS,
                             hipcub::LOAD_DEFAULT>;

    // SingleTilePolicy
    using SingleTilePolicy = ReducePolicy;

    // SegmentedReducePolicy
    using SegmentedReducePolicy = ReducePolicy;

    // ReduceNondeterministicPolicy
    using ReduceNondeterministicPolicy = ReducePolicy;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

template <typename T, typename OffsetT>
void nondeterministic_sum(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using accum_t     = T;
  using input_it_t  = const T*;
  using output_it_t = T*;
  using offset_t    = hipcub::detail::choose_offset_t<OffsetT>;
  using output_t    = T;
  using op_t        = cuda::std::plus<>;
  using init_t      = T;
  using dispatch_t  = hipcub::detail::DispatchReduceNondeterministic<
     input_it_t,
     output_it_t,
     offset_t,
     op_t,
     init_t,
     accum_t
#if !TUNE_BASE
    ,
    policy_hub_t<accum_t, offset_t>
#endif // TUNE_BASE
    >;

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements{io}"));

  thrust::device_vector<T> in = generate(elements);
  thrust::device_vector<T> out(1);

  input_it_t d_in   = thrust::raw_pointer_cast(in.data());
  output_it_t d_out = thrust::raw_pointer_cast(out.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(1);

  // Allocate temporary storage:
  std::size_t temp_size;
  dispatch_t::Dispatch(
    nullptr, temp_size, d_in, d_out, static_cast<offset_t>(elements), op_t{}, init_t{}, 0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size, thrust::no_init);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::gpu | nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      temp_storage, temp_size, d_in, d_out, static_cast<offset_t>(elements), op_t{}, init_t{}, launch.get_stream());
  });
}

#ifdef TUNE_T
using value_types = nvbench::type_list<TUNE_T>;
#else
using value_types = nvbench::type_list<int32_t, int64_t, float, double>;
#endif

NVBENCH_BENCH_TYPES(nondeterministic_sum, NVBENCH_TYPE_AXES(value_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4));
