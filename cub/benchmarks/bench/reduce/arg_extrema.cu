// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_reduce.cuh>
#include <cub/device/dispatch/dispatch_streaming_reduce.cuh>

#include <cuda/std/type_traits>

#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32
// %RANGE% TUNE_ITEMS_PER_VEC_LOAD_POW2 ipv 1:2:1

#if !TUNE_BASE
#  define TUNE_ITEMS_PER_VEC_LOAD (1 << TUNE_ITEMS_PER_VEC_LOAD_POW2)
template <typename AccumT, typename OffsetT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int threads_per_block  = TUNE_THREADS_PER_BLOCK;
    static constexpr int items_per_thread   = TUNE_ITEMS_PER_THREAD;
    static constexpr int items_per_vec_load = TUNE_ITEMS_PER_VEC_LOAD;

    using ReducePolicy =
      hipcub::AgentReducePolicy<threads_per_block,
                             items_per_thread,
                             AccumT,
                             items_per_vec_load,
                             hipcub::BLOCK_REDUCE_WARP_REDUCTIONS,
                             hipcub::LOAD_DEFAULT>;

    using SingleTilePolicy      = ReducePolicy;
    using SegmentedReducePolicy = ReducePolicy;
  };
#endif // !TUNE_BASE

  template <typename T, typename OpT>
  void arg_reduce(nvbench::state& state, nvbench::type_list<T, OpT>)
  {
    // Offset type used within the kernel and to index within one partition
    using per_partition_offset_t = int;

    // Offset type used to index within the total input in the range [d_in, d_in + num_items)
    using global_offset_t = ::cuda::std::int64_t;

    // The value type of the KeyValuePair<global_offset_t, output_value_t> returned by the ArgIndexInputIterator
    using output_value_t = T;

    // Iterator providing the values being reduced
    using values_it_t = T*;

    // Iterator providing the input items for the reduction
    using input_it_t = values_it_t;

    // Type used for the final result
    using output_tuple_t = hipcub::KeyValuePair<global_offset_t, T>;

    auto const init = ::cuda::std::is_same<OpT, hipcub::ArgMin>::value ? hipcub::Traits<T>::Max() : hipcub::Traits<T>::Lowest();

#if !TUNE_BASE
    using policy_t   = policy_hub_t<output_tuple_t, per_partition_offset_t>;
    using dispatch_t = hipcub::detail::reduce::dispatch_streaming_arg_reduce_t<
      input_it_t,
      output_tuple_t*,
      per_partition_offset_t,
      global_offset_t,
      OpT,
      T,
      policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::detail::reduce::
    dispatch_streaming_arg_reduce_t<input_it_t, output_tuple_t*, per_partition_offset_t, global_offset_t, OpT, T>;
#endif // TUNE_BASE

    // Retrieve axis parameters
    const auto elements         = static_cast<std::size_t>(state.get_int64("Elements{io}"));
    thrust::device_vector<T> in = generate(elements);
    thrust::device_vector<output_tuple_t> out(1);

    values_it_t d_in      = thrust::raw_pointer_cast(in.data());
    output_tuple_t* d_out = thrust::raw_pointer_cast(out.data());

    // Enable throughput calculations and add "Size" column to results.
    state.add_element_count(elements);
    state.add_global_memory_reads<T>(elements, "Size");
    state.add_global_memory_writes<output_tuple_t>(1);

    // Allocate temporary storage:
    std::size_t temp_size;
    dispatch_t::Dispatch(
      nullptr, temp_size, d_in, d_out, static_cast<global_offset_t>(elements), OpT{}, init, 0 /* stream */);

    thrust::device_vector<nvbench::uint8_t> temp(temp_size);
    auto* temp_storage = thrust::raw_pointer_cast(temp.data());

    state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
      dispatch_t::Dispatch(
        temp_storage, temp_size, d_in, d_out, static_cast<global_offset_t>(elements), OpT{}, init, launch.get_stream());
    });
  }

  using op_types = nvbench::type_list<hipcub::ArgMin, hipcub::ArgMax>;

  NVBENCH_BENCH_TYPES(arg_reduce, NVBENCH_TYPE_AXES(fundamental_types, op_types))
    .set_name("base")
    .set_type_axes_names({"T{ct}", "Operation{ct}"})
    .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4));
