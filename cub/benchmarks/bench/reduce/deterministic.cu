/******************************************************************************
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/dispatch/dispatch_reduce_deterministic.cuh>

#include <nvbench_helper.cuh>

#include <nvbench/range.cuh>
#include <nvbench/types.cuh>

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 3:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32

#if !TUNE_BASE

struct AgentReducePolicy
{
  /// Number of items per vectorized load
  static constexpr int VECTOR_LOAD_LENGTH = 4;

  /// Cooperative block-wide reduction algorithm to use
  static constexpr hipcub::BlockReduceAlgorithm BLOCK_ALGORITHM = hipcub::BlockReduceAlgorithm::BLOCK_REDUCE_RAKING;

  /// Cache load modifier for reading input elements
  static constexpr hipcub::CacheLoadModifier LOAD_MODIFIER = hipcub::CacheLoadModifier::LOAD_DEFAULT;
  constexpr static int ITEMS_PER_THREAD                 = TUNE_ITEMS_PER_THREAD;
  constexpr static int BLOCK_THREADS                    = TUNE_THREADS_PER_BLOCK;
};

struct policy_hub_t
{
  struct Policy350 : hipcub::ChainedPolicy<350, Policy350, Policy350>
  {
    constexpr static int ITEMS_PER_THREAD = TUNE_ITEMS_PER_THREAD;

    using DeterministicReducePolicy = AgentReducePolicy;

    // SingleTilePolicy
    using SingleTilePolicy = DeterministicReducePolicy;
  };

  using MaxPolicy = Policy350;
};
#endif // !TUNE_BASE

template <class T, typename OffsetT>
void deterministic_sum(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using input_it_t  = const T*;
  using output_it_t = T*;
  using offset_t    = hipcub::detail::choose_offset_t<OffsetT>;

  using init_t      = hipcub::detail::rfa::InitT<input_it_t, output_it_t>;
  using accum_t     = hipcub::detail::rfa::AccumT<::cuda::std::plus<>, init_t, input_it_t>;
  using transform_t = ::cuda::std::__identity;

  using dispatch_t = hipcub::detail::DispatchReduceDeterministic<
    input_it_t,
    output_it_t,
    offset_t,
    init_t,
    accum_t,
    transform_t
#if !TUNE_BASE
    ,
    policy_hub_t
#endif
    >;

  const auto elements       = static_cast<T>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  thrust::device_vector<T> in = generate(elements, entropy);
  thrust::device_vector<T> out(1);

  input_it_t d_in   = thrust::raw_pointer_cast(in.data());
  output_it_t d_out = thrust::raw_pointer_cast(out.data());
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(out.size());

  std::size_t temp_storage_bytes{};
  dispatch_t::Dispatch(nullptr, temp_storage_bytes, d_in, d_out, static_cast<offset_t>(elements), {}, 0);

  thrust::device_vector<nvbench::uint8_t> temp_storage(temp_storage_bytes);
  auto* d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  state.exec(nvbench::exec_tag::no_batch | nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      d_temp_storage, temp_storage_bytes, d_in, d_out, static_cast<offset_t>(elements), {}, launch.get_stream());
  });
}

using types = nvbench::type_list<float, double>;

NVBENCH_BENCH_TYPES(deterministic_sum, NVBENCH_TYPE_AXES(types, offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.544", "0.201"});
