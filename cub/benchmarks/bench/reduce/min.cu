#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// This benchmark is intended to cover DPX instructions on Hopper+ architectures. It specifically uses cuda::minimum<>
// instead of a user-defined operator, which CUB recognizes to select an optimized code path.

// Tuning parameters found for ::cuda::minimum<> apply equally for ::cuda::maximum<>
// Tuning parameters found for signed integer types apply equally for unsigned integer types
// TODO(bgruber): do tuning parameters found for int16_t apply equally for __half or __hip_bfloat16 on SM90+?

#include <cuda/functional>

#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32
// %RANGE% TUNE_ITEMS_PER_VEC_LOAD_POW2 ipv 1:2:1

// TODO(bgruber): let's add __half and __hip_bfloat16 eventually when they compile, since we have fast paths for them.
using value_types = fundamental_types;
using op_t        = ::cuda::minimum<>;
#include "base.cuh"
