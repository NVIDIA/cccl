#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/detail/choose_offset.cuh>
#include <cub/device/device_scan.cuh>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cuda/iterator>
#include <cuda/std/cmath>
#include <cuda/std/limits>
#include <cuda/std/utility>

#include <look_back_helper.cuh>
#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS ipt 7:24:1
// %RANGE% TUNE_THREADS tpb 128:1024:32
// %RANGE% TUNE_MAGIC_NS ns 0:2048:4
// %RANGE% TUNE_DELAY_CONSTRUCTOR_ID dcid 0:7:1
// %RANGE% TUNE_L2_WRITE_LATENCY_NS l2w 0:1200:5
// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:1:1

#if !TUNE_BASE
#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_DIRECT
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

#  if TUNE_LOAD == 0
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_DEFAULT
#  elif TUNE_LOAD == 1
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_CA
#  endif // TUNE_LOAD

struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using ScanByKeyPolicyT = hipcub::AgentScanByKeyPolicy<
      TUNE_THREADS,
      TUNE_ITEMS,
      // TODO Tune
      TUNE_LOAD_ALGORITHM,
      TUNE_LOAD_MODIFIER,
      hipcub::BLOCK_SCAN_WARP_SCANS,
      TUNE_STORE_ALGORITHM,
      delay_constructor_t>;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

/* Given input sequence of values, compute sequence of
 * pairs corresponding to running minimum and running maximum values.
 */

/*! @brief Structure to hold minimum and maximum */
template <typename T>
struct MinMax
{
private:
  T m_min{cuda::std::numeric_limits<T>::max()};
  T m_max{cuda::std::numeric_limits<T>::min()};

public:
  __host__ __device__ MinMax() = default;
  __host__ __device__ MinMax(T minimum, T maximum)
      : m_min(minimum)
      , m_max(maximum)
  {}

  T __host__ __device__ minimum() const
  {
    return m_min;
  }
  T __host__ __device__ maximum() const
  {
    return m_max;
  }
};

/*! @brief Scan operator combining min-max pairs */
struct ScanOp
{
  template <typename T>
  MinMax<T> __host__ __device__ operator()(MinMax<T> v1, MinMax<T> v2) const
  {
    auto min_r = cuda::minimum{}(v1.minimum(), v2.minimum());
    auto max_r = cuda::maximum{}(v1.maximum(), v2.maximum());
    return {min_r, max_r};
  }
};

template <typename T>
struct EmbedOp
{
  MinMax<T> __host__ __device__ operator()(T v) const
  {
    return {v, v};
  }
};

template <typename T>
struct ExtractMin
{
  T __host__ __device__ operator()(MinMax<T> pair) const
  {
    return pair.minimum();
  }
};

template <typename T>
struct ExtractMax
{
  T __host__ __device__ operator()(MinMax<T> pair) const
  {
    return pair.maximum();
  }
};

template <typename T, typename OffsetT>
void benchmark_impl(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using wrapped_init_t = hipcub::NullType;
  using value_t        = T;
  using pair_t         = MinMax<value_t>;
  using op_t           = ScanOp;
  using accum_t        = pair_t;
  using input_raw_t    = const value_t*;
  using input_it_t     = cuda::transform_iterator<EmbedOp<value_t>, input_raw_t>;
  using output_it_t    = pair_t*;
  using offset_t       = hipcub::detail::choose_offset_t<OffsetT>;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<accum_t>;
  using dispatch_t = cub::
    DispatchScan<input_it_t, output_it_t, op_t, wrapped_init_t, offset_t, accum_t, hipcub::ForceInclusive::No, policy_t>;
#else
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, wrapped_init_t, offset_t, accum_t, hipcub::ForceInclusive::No>;
#endif

  const auto elements = static_cast<std::size_t>(state.get_int64("Elements{io}"));

  thrust::device_vector<pair_t> output(elements);
  thrust::device_vector<value_t> input = generate(elements);

  input_raw_t d_input  = thrust::raw_pointer_cast(input.data());
  output_it_t d_output = thrust::raw_pointer_cast(output.data());

  input_it_t inp_it(d_input, EmbedOp<value_t>{});

  state.add_element_count(elements);
  state.add_global_memory_reads<value_t>(elements, "Size");
  state.add_global_memory_writes<pair_t>(elements);

  size_t tmp_size;
  dispatch_t::Dispatch(
    nullptr, tmp_size, inp_it, d_output, op_t{}, wrapped_init_t{}, input.size(), state.get_cuda_stream().get_stream());

  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);
  nvbench::uint8_t* d_tmp = thrust::raw_pointer_cast(tmp.data());

  state.exec(nvbench::exec_tag::gpu | nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(d_tmp, tmp_size, inp_it, d_output, op_t{}, wrapped_init_t{}, input.size(), launch.get_stream());
  });

  // verification

  thrust::device_vector<value_t> ref_mins(elements);
  thrust::device_vector<value_t> ref_maxs(elements);

  hipcub::DeviceScan::InclusiveScanInit(
    nullptr,
    tmp_size,
    d_input,
    ref_mins.begin(),
    cuda::minimum<>{},
    cuda::std::numeric_limits<value_t>::max(),
    input.size(),
    state.get_cuda_stream().get_stream());

  thrust::device_vector<nvbench::uint8_t> tmp1(tmp_size);
  nvbench::uint8_t* d_tmp1 = thrust::raw_pointer_cast(tmp1.data());

  hipcub::DeviceScan::InclusiveScanInit(
    d_tmp1,
    tmp_size,
    d_input,
    ref_mins.begin(),
    cuda::minimum<>{},
    cuda::std::numeric_limits<value_t>::max(),
    input.size(),
    state.get_cuda_stream().get_stream());

  hipcub::DeviceScan::InclusiveScanInit(
    nullptr,
    tmp_size,
    d_input,
    ref_maxs.begin(),
    cuda::minimum<>{},
    cuda::std::numeric_limits<value_t>::max(),
    input.size(),
    state.get_cuda_stream().get_stream());

  thrust::device_vector<nvbench::uint8_t> tmp2(tmp_size);
  nvbench::uint8_t* d_tmp2 = thrust::raw_pointer_cast(tmp2.data());

  hipcub::DeviceScan::InclusiveScanInit(
    d_tmp2,
    tmp_size,
    d_input,
    ref_maxs.begin(),
    cuda::maximum<>{},
    cuda::std::numeric_limits<value_t>::min(),
    input.size(),
    state.get_cuda_stream().get_stream());

  thrust::device_vector<value_t> computed_mins(elements);
  thrust::device_vector<value_t> computed_maxs(elements);

  hipcub::DeviceTransform::Transform(
    d_output, computed_mins.begin(), input.size(), ExtractMin<T>{}, state.get_cuda_stream().get_stream());
  hipcub::DeviceTransform::Transform(
    d_output, computed_maxs.begin(), input.size(), ExtractMax<T>{}, state.get_cuda_stream().get_stream());

  assert(computed_mins == ref_mins);
  assert(computed_maxs == ref_maxs);
}

using bench_types =
  nvbench::type_list<cuda::std::uint32_t,
                     nvbench::int32_t,
                     cuda::std::uint64_t,
                     nvbench::int64_t,
                     nvbench::float32_t,
                     nvbench::float64_t>;

NVBENCH_BENCH_TYPES(benchmark_impl, NVBENCH_TYPE_AXES(bench_types, offset_types))
  .set_name("running-min-max")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4));
