#include "hip/hip_runtime.h"
// This file demonstrates how to use <nv/target> and how to avoid common
// pitfalls regarding compiler/dialect compatibility.

//=======================================================================================================================
#include <nv/target>

// The below are part of libcu++ and are exposed for users that would like a simpler method of targeting host/device
// code on NVCC, NVC++ or GCC/Clang/MSVC even when the NVCC compiler isn't present.

// These macros are to be used in lieu of common #if defined(__CUDA_ARCH__) statements and
// are only to be used inside of function scopes

/* Macros defined when including <nv/target> or virtually any libcu++ header
  NV_IF_TARGET(query, true, ...)        | Queries compilation mode and emits code if true                |
  NV_IF_ELSE_TARGET(query, true, false) | As above, but can also emit different code when false          |
  NV_DISPATCH_TARGET(...)               | Similar to a switch statement emitting code for multiple modes |
*/

/* Queryable properties defined by <nv/target>
Can be imagined as __CUDA_ARCH__ >= SM_XX
  NV_PROVIDES_SM_35
  NV_PROVIDES_SM_37
  NV_PROVIDES_SM_50
  NV_PROVIDES_SM_52
  NV_PROVIDES_SM_53
  NV_PROVIDES_SM_60
  NV_PROVIDES_SM_61
  NV_PROVIDES_SM_62
  NV_PROVIDES_SM_70
  NV_PROVIDES_SM_72
  NV_PROVIDES_SM_75
  NV_PROVIDES_SM_80
  NV_PROVIDES_SM_86
  NV_PROVIDES_SM_87

Similar to above, but instead __CUDA_ARCH__ == SM_XX
  NV_IS_EXACTLY_SM_35
  NV_IS_EXACTLY_SM_37
  NV_IS_EXACTLY_SM_50
  NV_IS_EXACTLY_SM_52
  NV_IS_EXACTLY_SM_53
  NV_IS_EXACTLY_SM_60
  NV_IS_EXACTLY_SM_61
  NV_IS_EXACTLY_SM_62
  NV_IS_EXACTLY_SM_70
  NV_IS_EXACTLY_SM_72
  NV_IS_EXACTLY_SM_75
  NV_IS_EXACTLY_SM_80
  NV_IS_EXACTLY_SM_86
  NV_IS_EXACTLY_SM_87

Queries whether if host or device code is being compiled
  NV_IS_HOST
  NV_IS_DEVICE

Static true/false values for fallbacks or user manipulation
  NV_ANY_TARGET
  NV_NO_TARGET
*/

//=======================================================================================================================
// NV_IF_ELSE_TARGET(query, true statement, false statement)
__host__ __device__ int my_popc(unsigned int v)
{
  // NV_IF_ELSE_TARGET accepts three arguments, a query and two statement.
  // Here we check if we're compiling for device code. This function acts as a backend for both CUDA and host CPU popc.
  NV_IF_ELSE_TARGET(
    NV_IS_DEVICE,
    return __popc(v); // Is false, use CUDA intrinsic
    , // Notice comma signifying end of block
    return __builtin_popc(v); // Is host, use GCC builtin
  )
}
// Note the commas seperating statements, if preprocessed code is written out the macro will be preprocessed into the
// below:
/*
  my_popc(unsigned int v) {
    {return __popc(v);}
  }
*/

//=======================================================================================================================
// NV_IF_TARGET(query, true statement) OR NV_IF_TARGET(q, t, ...)
__host__ __device__ void some_algorithm()
{
  // NV_IF_TARGET accepts two arguments, a query and a statement. (and an optional false statement in >=C++11)
  NV_IF_TARGET(NV_IS_DEVICE,
               do_device_specific_work(); // Code only emitted if compiling for device
  )
}

//=======================================================================================================================
// NV_DISPATCH_TARGET(...) - Available only in C++11 and up due to variadic macros
__host__ __device__ void my_memset(void* p, uint8_t v, uint64_t c)
{
  // Target dispatch accepts pairs of queries and statements.
  // The first postive query encountered will be emitted while others are ignored.
  NV_DISPATCH_TARGET(
    NV_PROVIDES_SM_80,
    // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#data-movement-and-conversion-instructions-cp-async
    if (v == 0) zero_fill(p, 0, c); // zero fill using cp.async available on SM_80
    else memset(p, v, c);
    , // Notice comma signifying end of block
    NV_ANY_TARGET, // Uncoditionally use memset in other cases
    memset(p, v, c);)
}

//***********************************************************************************************************************

// # Common pitfalls:

//=======================================================================================================================
// Embedding preprocessor statements as an argument. Perform textual manipulation outside of the macro.
/*
NV_IF_TARGET(
  NV_IS_DEVICE,
// This will break immediately on most compilers
#  if defined(ENABLE_SM_80_FEATURE)
    sm80_function();
#  else
    device_function();
#  endif
)

// Instead one could write the above as:
#if defined(ENABLE_SM_80_FEATURE)
# define OPTIMAL_DEVICE_FUNCTION() sm80_function()
#else
# define OPTIMAL_DEVICE_FUNCTION() device_function()
#endif

NV_IF_TARGET(
  NV_IS_DEVICE,
  OPTIMAL_DEVICE_FUNCTION();
)
*/

//=======================================================================================================================
// Some statements may have unguarded commas, e.g. lambdas or aggregate assignment
// Supported with C++11 and up ONLY, as it requires variadic macro processing
/*
NV_IF_TARGET(
  NV_IS_DEVICE,
    ( // You may wrap a statement or series of statements with a parenthesis to guard commas from any macro machinery
      int input[] = {x, y, z...};
      my_algorithm(input);
    )
)
*/
