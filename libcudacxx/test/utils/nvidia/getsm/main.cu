
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CALL(...) \
    do { \
        hipError_t err = __VA_ARGS__; \
        if (err != hipSuccess) \
        { \
            printf("CUDA ERROR: %s: %s\n", \
                   hipGetErrorName(err), hipGetErrorString(err)); \
            return err; \
        } \
    } while (false)

int main()
{
    int selected_device;
    CUDA_CALL(hipGetDevice(&selected_device));

    hipDeviceProp_t device_prop;
    CUDA_CALL(hipGetDeviceProperties(&device_prop, selected_device));

    FILE * output = fopen("sm", "w");
    fprintf(output, "%d%d\n", device_prop.major, device_prop.minor);
    fclose(output);
}
