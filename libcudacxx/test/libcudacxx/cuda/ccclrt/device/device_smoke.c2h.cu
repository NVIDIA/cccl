//===----------------------------------------------------------------------===//
//
// Part of libcu++, the C++ Standard Library for your entire system,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/devices>
#include <cuda/std/__type_traits/is_same.h>

#include <testing.cuh>

namespace
{
template <const auto& Attr, ::hipDeviceAttribute_t ExpectedAttr, class ExpectedResult>
[[maybe_unused]] auto test_device_attribute()
{
  cuda::device_ref dev0(0);
  STATIC_REQUIRE(Attr == ExpectedAttr);
  STATIC_REQUIRE(::cuda::std::is_same_v<cuda::device_attribute_result_t<Attr>, ExpectedResult>);

  auto result = dev0.attribute(Attr);
  STATIC_REQUIRE(::cuda::std::is_same_v<decltype(result), ExpectedResult>);
  CCCLRT_REQUIRE(result == dev0.attribute<ExpectedAttr>());
  CCCLRT_REQUIRE(result == Attr(dev0));
  return result;
}
} // namespace

C2H_CCCLRT_TEST("Smoke", "[device]")
{
  namespace attributes = cuda::device_attributes;
  using cuda::device_ref;

  SECTION("Compare")
  {
    CCCLRT_REQUIRE(device_ref{0} == device_ref{0});
    CCCLRT_REQUIRE(device_ref{0} == 0);
    CCCLRT_REQUIRE(0 == device_ref{0});
    CCCLRT_REQUIRE(device_ref{1} != device_ref{0});
    CCCLRT_REQUIRE(device_ref{1} != 2);
    CCCLRT_REQUIRE(1 != device_ref{2});
  }

  SECTION("Attributes")
  {
    ::test_device_attribute<attributes::max_threads_per_block, ::hipDeviceAttributeMaxThreadsPerBlock, int>();
    ::test_device_attribute<attributes::max_block_dim_x, ::hipDeviceAttributeMaxBlockDimX, int>();
    ::test_device_attribute<attributes::max_block_dim_y, ::hipDeviceAttributeMaxBlockDimY, int>();
    ::test_device_attribute<attributes::max_block_dim_z, ::hipDeviceAttributeMaxBlockDimZ, int>();
    ::test_device_attribute<attributes::max_grid_dim_x, ::hipDeviceAttributeMaxGridDimX, int>();
    ::test_device_attribute<attributes::max_grid_dim_y, ::hipDeviceAttributeMaxGridDimY, int>();
    ::test_device_attribute<attributes::max_grid_dim_z, ::hipDeviceAttributeMaxGridDimZ, int>();
    ::test_device_attribute<attributes::max_shared_memory_per_block, ::hipDeviceAttributeMaxSharedMemoryPerBlock, int>();
    ::test_device_attribute<attributes::total_constant_memory, ::hipDeviceAttributeTotalConstantMemory, int>();
    ::test_device_attribute<attributes::warp_size, ::hipDeviceAttributeWarpSize, int>();
    ::test_device_attribute<attributes::max_pitch, ::hipDeviceAttributeMaxPitch, int>();
    ::test_device_attribute<attributes::max_texture_1d_width, ::hipDeviceAttributeMaxTexture1DWidth, int>();
    ::test_device_attribute<attributes::max_texture_1d_linear_width, ::hipDeviceAttributeMaxTexture1DLinear, int>();
    ::test_device_attribute<attributes::max_texture_1d_mipmapped_width, ::hipDeviceAttributeMaxTexture1DMipmap, int>();
    ::test_device_attribute<attributes::max_texture_2d_width, ::hipDeviceAttributeMaxTexture2DWidth, int>();
    ::test_device_attribute<attributes::max_texture_2d_height, ::hipDeviceAttributeMaxTexture2DHeight, int>();
    ::test_device_attribute<attributes::max_texture_2d_linear_width, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<attributes::max_texture_2d_linear_height, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<attributes::max_texture_2d_linear_pitch, ::hipDeviceAttributeMaxTexture2DLinear, int>();
    ::test_device_attribute<attributes::max_texture_2d_mipmapped_width, ::hipDeviceAttributeMaxTexture2DMipmap, int>();
    ::test_device_attribute<attributes::max_texture_2d_mipmapped_height, ::hipDeviceAttributeMaxTexture2DMipmap, int>();
    ::test_device_attribute<attributes::max_texture_3d_width, ::hipDeviceAttributeMaxTexture3DWidth, int>();
    ::test_device_attribute<attributes::max_texture_3d_height, ::hipDeviceAttributeMaxTexture3DHeight, int>();
    ::test_device_attribute<attributes::max_texture_3d_depth, ::hipDeviceAttributeMaxTexture3DDepth, int>();
    ::test_device_attribute<attributes::max_texture_3d_width_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<attributes::max_texture_3d_height_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<attributes::max_texture_3d_depth_alt, ::hipDeviceAttributeMaxTexture3DAlt, int>();
    ::test_device_attribute<attributes::max_texture_cubemap_width, ::hipDeviceAttributeMaxTextureCubemap, int>();
    ::test_device_attribute<attributes::max_texture_1d_layered_width, ::hipDeviceAttributeMaxTexture1DLayered, int>();
    ::test_device_attribute<attributes::max_texture_1d_layered_layers, ::cudaDevAttrMaxTexture1DLayeredLayers, int>();
    ::test_device_attribute<attributes::max_texture_2d_layered_width, ::hipDeviceAttributeMaxTexture2DLayered, int>();
    ::test_device_attribute<attributes::max_texture_2d_layered_height, ::hipDeviceAttributeMaxTexture2DLayered, int>();
    ::test_device_attribute<attributes::max_texture_2d_layered_layers, ::cudaDevAttrMaxTexture2DLayeredLayers, int>();
    ::test_device_attribute<attributes::max_texture_cubemap_layered_width,
                            ::hipDeviceAttributeMaxTextureCubemapLayered,
                            int>();
    ::test_device_attribute<attributes::max_texture_cubemap_layered_layers,
                            ::cudaDevAttrMaxTextureCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<attributes::max_surface_1d_width, ::hipDeviceAttributeMaxSurface1D, int>();
    ::test_device_attribute<attributes::max_surface_2d_width, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<attributes::max_surface_2d_height, ::hipDeviceAttributeMaxSurface2D, int>();
    ::test_device_attribute<attributes::max_surface_3d_width, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<attributes::max_surface_3d_height, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<attributes::max_surface_3d_depth, ::hipDeviceAttributeMaxSurface3D, int>();
    ::test_device_attribute<attributes::max_surface_1d_layered_width, ::hipDeviceAttributeMaxSurface1DLayered, int>();
    ::test_device_attribute<attributes::max_surface_1d_layered_layers, ::cudaDevAttrMaxSurface1DLayeredLayers, int>();
    ::test_device_attribute<attributes::max_surface_2d_layered_width, ::hipDeviceAttributeMaxSurface2DLayered, int>();
    ::test_device_attribute<attributes::max_surface_2d_layered_height, ::hipDeviceAttributeMaxSurface2DLayered, int>();
    ::test_device_attribute<attributes::max_surface_2d_layered_layers, ::cudaDevAttrMaxSurface2DLayeredLayers, int>();
    ::test_device_attribute<attributes::max_surface_cubemap_width, ::hipDeviceAttributeMaxSurfaceCubemap, int>();
    ::test_device_attribute<attributes::max_surface_cubemap_layered_width,
                            ::hipDeviceAttributeMaxSurfaceCubemapLayered,
                            int>();
    ::test_device_attribute<attributes::max_surface_cubemap_layered_layers,
                            ::cudaDevAttrMaxSurfaceCubemapLayeredLayers,
                            int>();
    ::test_device_attribute<attributes::max_registers_per_block, ::hipDeviceAttributeMaxRegistersPerBlock, int>();
    ::test_device_attribute<attributes::clock_rate, ::hipDeviceAttributeClockRate, int>();
    ::test_device_attribute<attributes::texture_alignment, ::hipDeviceAttributeTextureAlignment, int>();
    ::test_device_attribute<attributes::texture_pitch_alignment, ::hipDeviceAttributeTexturePitchAlignment, int>();
    ::test_device_attribute<attributes::gpu_overlap, ::hipDeviceAttributeAsyncEngineCount, bool>();
    ::test_device_attribute<attributes::multiprocessor_count, ::hipDeviceAttributeMultiprocessorCount, int>();
    ::test_device_attribute<attributes::kernel_exec_timeout, ::hipDeviceAttributeKernelExecTimeout, bool>();
    ::test_device_attribute<attributes::integrated, ::hipDeviceAttributeIntegrated, bool>();
    ::test_device_attribute<attributes::can_map_host_memory, ::hipDeviceAttributeCanMapHostMemory, bool>();
    ::test_device_attribute<attributes::compute_mode, ::hipDeviceAttributeComputeMode, ::hipComputeMode>();
    ::test_device_attribute<attributes::concurrent_kernels, ::hipDeviceAttributeConcurrentKernels, bool>();
    ::test_device_attribute<attributes::ecc_enabled, ::hipDeviceAttributeEccEnabled, bool>();
    ::test_device_attribute<attributes::pci_bus_id, ::hipDeviceAttributePciBusId, int>();
    ::test_device_attribute<attributes::pci_device_id, ::hipDeviceAttributePciDeviceId, int>();
    ::test_device_attribute<attributes::tcc_driver, ::hipDeviceAttributeTccDriver, bool>();
    ::test_device_attribute<attributes::l2_cache_size, ::hipDeviceAttributeL2CacheSize, int>();
    ::test_device_attribute<attributes::max_threads_per_multiprocessor, ::hipDeviceAttributeMaxThreadsPerMultiProcessor, int>();
    ::test_device_attribute<attributes::unified_addressing, ::hipDeviceAttributeUnifiedAddressing, bool>();
    ::test_device_attribute<attributes::compute_capability_major, ::hipDeviceAttributeComputeCapabilityMajor, int>();
    ::test_device_attribute<attributes::compute_capability_minor, ::hipDeviceAttributeComputeCapabilityMinor, int>();
    ::test_device_attribute<attributes::stream_priorities_supported, ::hipDeviceAttributeStreamPrioritiesSupported, bool>();
    ::test_device_attribute<attributes::global_l1_cache_supported, ::hipDeviceAttributeGlobalL1CacheSupported, bool>();
    ::test_device_attribute<attributes::local_l1_cache_supported, ::hipDeviceAttributeLocalL1CacheSupported, bool>();
    ::test_device_attribute<attributes::max_shared_memory_per_multiprocessor,
                            ::hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
                            int>();
    ::test_device_attribute<attributes::max_registers_per_multiprocessor,
                            ::hipDeviceAttributeMaxRegistersPerMultiprocessor,
                            int>();
    ::test_device_attribute<attributes::is_multi_gpu_board, ::hipDeviceAttributeIsMultiGpuBoard, bool>();
    ::test_device_attribute<attributes::multi_gpu_board_group_id, ::hipDeviceAttributeMultiGpuBoardGroupID, int>();
    ::test_device_attribute<attributes::host_native_atomic_supported, ::hipDeviceAttributeHostNativeAtomicSupported, bool>();
    ::test_device_attribute<attributes::single_to_double_precision_perf_ratio,
                            ::hipDeviceAttributeSingleToDoublePrecisionPerfRatio,
                            int>();
    ::test_device_attribute<attributes::pageable_memory_access, ::hipDeviceAttributePageableMemoryAccess, bool>();
    ::test_device_attribute<attributes::concurrent_managed_access, ::hipDeviceAttributeConcurrentManagedAccess, bool>();
    ::test_device_attribute<attributes::compute_preemption_supported, ::hipDeviceAttributeComputePreemptionSupported, bool>();
    ::test_device_attribute<attributes::can_use_host_pointer_for_registered_mem,
                            ::hipDeviceAttributeCanUseHostPointerForRegisteredMem,
                            bool>();
    ::test_device_attribute<attributes::cooperative_launch, ::hipDeviceAttributeCooperativeLaunch, bool>();
    ::test_device_attribute<attributes::can_flush_remote_writes, ::cudaDevAttrCanFlushRemoteWrites, bool>();
    ::test_device_attribute<attributes::host_register_supported, ::hipDeviceAttributeHostRegisterSupported, bool>();
    ::test_device_attribute<attributes::pageable_memory_access_uses_host_page_tables,
                            ::hipDeviceAttributePageableMemoryAccessUsesHostPageTables,
                            bool>();
    ::test_device_attribute<attributes::direct_managed_mem_access_from_host,
                            ::hipDeviceAttributeDirectManagedMemAccessFromHost,
                            bool>();
    ::test_device_attribute<attributes::max_shared_memory_per_block_optin,
                            ::hipDeviceAttributeSharedMemPerBlockOptin,
                            int>();
    ::test_device_attribute<attributes::max_blocks_per_multiprocessor, ::hipDeviceAttributeMaxBlocksPerMultiprocessor, int>();
    ::test_device_attribute<attributes::max_persisting_l2_cache_size, ::cudaDevAttrMaxPersistingL2CacheSize, int>();
    ::test_device_attribute<attributes::max_access_policy_window_size, ::cudaDevAttrMaxAccessPolicyWindowSize, int>();
    ::test_device_attribute<attributes::reserved_shared_memory_per_block,
                            ::cudaDevAttrReservedSharedMemoryPerBlock,
                            int>();
    ::test_device_attribute<attributes::sparse_cuda_array_supported, ::cudaDevAttrSparseCudaArraySupported, bool>();
    ::test_device_attribute<attributes::host_register_read_only_supported,
                            ::cudaDevAttrHostRegisterReadOnlySupported,
                            bool>();
    ::test_device_attribute<attributes::memory_pools_supported, ::hipDeviceAttributeMemoryPoolsSupported, bool>();
    ::test_device_attribute<attributes::gpu_direct_rdma_supported, ::cudaDevAttrGPUDirectRDMASupported, bool>();
    ::test_device_attribute<attributes::gpu_direct_rdma_flush_writes_options,
                            ::cudaDevAttrGPUDirectRDMAFlushWritesOptions,
                            ::hipFlushGPUDirectRDMAWritesOptions>();
    ::test_device_attribute<attributes::gpu_direct_rdma_writes_ordering,
                            ::cudaDevAttrGPUDirectRDMAWritesOrdering,
                            ::hipGPUDirectRDMAWritesOrdering>();
    ::test_device_attribute<attributes::memory_pool_supported_handle_types,
                            ::cudaDevAttrMemoryPoolSupportedHandleTypes,
                            ::hipMemAllocationHandleType>();
    ::test_device_attribute<attributes::deferred_mapping_cuda_array_supported,
                            ::cudaDevAttrDeferredMappingCudaArraySupported,
                            bool>();
    ::test_device_attribute<attributes::ipc_event_support, ::cudaDevAttrIpcEventSupport, bool>();

#if _CCCL_CTK_AT_LEAST(12, 2)
    ::test_device_attribute<attributes::numa_config, ::cudaDevAttrNumaConfig, ::cudaDeviceNumaConfig>();
    ::test_device_attribute<attributes::numa_id, ::cudaDevAttrNumaId, int>();
#endif // _CCCL_CTK_AT_LEAST(12, 2)

    SECTION("compute_mode")
    {
      STATIC_REQUIRE(::hipComputeModeDefault == attributes::compute_mode.default_mode);
      STATIC_REQUIRE(::hipComputeModeProhibited == attributes::compute_mode.prohibited_mode);
      STATIC_REQUIRE(::hipComputeModeExclusiveProcess == attributes::compute_mode.exclusive_process_mode);

      auto mode = device_ref(0).attribute(attributes::compute_mode);
      CCCLRT_REQUIRE((mode == attributes::compute_mode.default_mode || //
                      mode == attributes::compute_mode.prohibited_mode || //
                      mode == attributes::compute_mode.exclusive_process_mode));
    }

    SECTION("gpu_direct_rdma_flush_writes_options")
    {
      STATIC_REQUIRE(::hipFlushGPUDirectRDMAWritesOptionHost == attributes::gpu_direct_rdma_flush_writes_options.host);
      STATIC_REQUIRE(
        ::hipFlushGPUDirectRDMAWritesOptionMemOps == attributes::gpu_direct_rdma_flush_writes_options.mem_ops);

      [[maybe_unused]] auto options = device_ref(0).attribute(attributes::gpu_direct_rdma_flush_writes_options);
#if !_CCCL_COMPILER(MSVC)
      CCCLRT_REQUIRE((options == attributes::gpu_direct_rdma_flush_writes_options.host || //
                      options == attributes::gpu_direct_rdma_flush_writes_options.mem_ops));
#endif
    }

    SECTION("gpu_direct_rdma_writes_ordering")
    {
      STATIC_REQUIRE(::hipGPUDirectRDMAWritesOrderingNone == attributes::gpu_direct_rdma_writes_ordering.none);
      STATIC_REQUIRE(::hipGPUDirectRDMAWritesOrderingOwner == attributes::gpu_direct_rdma_writes_ordering.owner);
      STATIC_REQUIRE(
        ::hipGPUDirectRDMAWritesOrderingAllDevices == attributes::gpu_direct_rdma_writes_ordering.all_devices);

      auto ordering = device_ref(0).attribute(attributes::gpu_direct_rdma_writes_ordering);
      CCCLRT_REQUIRE((ordering == attributes::gpu_direct_rdma_writes_ordering.none || //
                      ordering == attributes::gpu_direct_rdma_writes_ordering.owner || //
                      ordering == attributes::gpu_direct_rdma_writes_ordering.all_devices));
    }

    SECTION("memory_pool_supported_handle_types")
    {
      STATIC_REQUIRE(::hipMemHandleTypeNone == attributes::memory_pool_supported_handle_types.none);
      STATIC_REQUIRE(
        ::hipMemHandleTypePosixFileDescriptor == attributes::memory_pool_supported_handle_types.posix_file_descriptor);
      STATIC_REQUIRE(::hipMemHandleTypeWin32 == attributes::memory_pool_supported_handle_types.win32);
      STATIC_REQUIRE(::hipMemHandleTypeWin32Kmt == attributes::memory_pool_supported_handle_types.win32_kmt);
#if _CCCL_CTK_AT_LEAST(12, 4)
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == 0x8);
      STATIC_REQUIRE(::cudaMemHandleTypeFabric == attributes::memory_pool_supported_handle_types.fabric);
#else // ^^^ _CCCL_CTK_AT_LEAST(12, 4) ^^^ / vvv _CCCL_CTK_BELOW(12, 4) vvv
      STATIC_REQUIRE(0x8 == attributes::memory_pool_supported_handle_types.fabric);
#endif // ^^^ _CCCL_CTK_BELOW(12, 4) ^^^

      constexpr int all_handle_types =
        attributes::memory_pool_supported_handle_types.none
        | attributes::memory_pool_supported_handle_types.posix_file_descriptor
        | attributes::memory_pool_supported_handle_types.win32
        | attributes::memory_pool_supported_handle_types.win32_kmt
        | attributes::memory_pool_supported_handle_types.fabric;
      auto handle_types = device_ref(0).attribute(attributes::memory_pool_supported_handle_types);
      CCCLRT_REQUIRE(static_cast<int>(handle_types) <= static_cast<int>(all_handle_types));
    }

#if _CCCL_CTK_AT_LEAST(12, 2)
    SECTION("numa_config")
    {
      STATIC_REQUIRE(::cudaDeviceNumaConfigNone == attributes::numa_config.none);
      STATIC_REQUIRE(::cudaDeviceNumaConfigNumaNode == attributes::numa_config.numa_node);

      auto config = device_ref(0).attribute(attributes::numa_config);
      CCCLRT_REQUIRE((config == attributes::numa_config.none || //
                      config == attributes::numa_config.numa_node));
    }
#endif // _CCCL_CTK_AT_LEAST(12, 2)

    SECTION("Compute capability")
    {
      int compute_cap       = device_ref(0).attribute(attributes::compute_capability);
      int compute_cap_major = device_ref(0).attribute(attributes::compute_capability_major);
      int compute_cap_minor = device_ref(0).attribute(attributes::compute_capability_minor);
      CCCLRT_REQUIRE(compute_cap == 10 * compute_cap_major + compute_cap_minor);
    }
  }
  SECTION("Name")
  {
    std::string name = device_ref(0).name();
    CCCLRT_REQUIRE(name.length() != 0);
    CCCLRT_REQUIRE(name[0] != 0);
  }
}

C2H_CCCLRT_TEST("global devices vector", "[device]")
{
  CCCLRT_REQUIRE(cuda::devices.size() > 0);
  CCCLRT_REQUIRE(cuda::devices.begin() != cuda::devices.end());
  CCCLRT_REQUIRE(cuda::devices.begin() == cuda::devices.begin());
  CCCLRT_REQUIRE(cuda::devices.end() == cuda::devices.end());
  CCCLRT_REQUIRE(cuda::devices.size() == static_cast<size_t>(cuda::devices.end() - cuda::devices.begin()));

  CCCLRT_REQUIRE(0 == cuda::devices[0].get());
  CCCLRT_REQUIRE(cuda::device_ref{0} == cuda::devices[0]);

  CCCLRT_REQUIRE(0 == (*cuda::devices.begin()).get());
  CCCLRT_REQUIRE(cuda::device_ref{0} == *cuda::devices.begin());

  CCCLRT_REQUIRE(0 == cuda::devices.begin()->get());
  CCCLRT_REQUIRE(0 == cuda::devices.begin()[0].get());

  if (cuda::devices.size() > 1)
  {
    CCCLRT_REQUIRE(1 == cuda::devices[1].get());
    CCCLRT_REQUIRE(cuda::device_ref{0} != cuda::devices[1].get());

    CCCLRT_REQUIRE(1 == (*std::next(cuda::devices.begin())).get());
    CCCLRT_REQUIRE(1 == std::next(cuda::devices.begin())->get());
    CCCLRT_REQUIRE(1 == cuda::devices.begin()[1].get());

    CCCLRT_REQUIRE(cuda::devices.size() - 1 == static_cast<std::size_t>((*std::prev(cuda::devices.end())).get()));
    CCCLRT_REQUIRE(cuda::devices.size() - 1 == static_cast<std::size_t>(std::prev(cuda::devices.end())->get()));
    CCCLRT_REQUIRE(cuda::devices.size() - 1 == static_cast<std::size_t>(cuda::devices.end()[-1].get()));

    auto peers = cuda::devices[0].peer_devices();
    for (auto peer : peers)
    {
      CCCLRT_REQUIRE(cuda::devices[0].has_peer_access_to(peer));
      CCCLRT_REQUIRE(peer.has_peer_access_to(cuda::devices[0]));
    }
  }

#if _CCCL_HAS_EXCEPTIONS()
  try
  {
    [[maybe_unused]] const cuda::physical_device& dev = cuda::devices[cuda::devices.size()];
    CCCLRT_REQUIRE(false); // should not get here
  }
  catch (const std::out_of_range&)
  {
    CCCLRT_REQUIRE(true); // expected
  }
#endif // _CCCL_HAS_EXCEPTIONS()
}
