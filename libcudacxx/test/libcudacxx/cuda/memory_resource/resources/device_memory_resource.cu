#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/memory_resource>
#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream>

#include <stdexcept>

#include <testing.cuh>
#include <utility.cuh>

static_assert(!cuda::std::is_trivial<cuda::device_memory_pool_ref>::value, "");
static_assert(!cuda::std::is_trivially_default_constructible<cuda::device_memory_pool_ref>::value, "");
static_assert(cuda::std::is_copy_constructible<cuda::device_memory_pool_ref>::value, "");
static_assert(cuda::std::is_move_constructible<cuda::device_memory_pool_ref>::value, "");
static_assert(cuda::std::is_copy_assignable<cuda::device_memory_pool_ref>::value, "");
static_assert(cuda::std::is_move_assignable<cuda::device_memory_pool_ref>::value, "");
static_assert(cuda::std::is_trivially_destructible<cuda::device_memory_pool_ref>::value, "");
static_assert(!cuda::std::is_empty<cuda::device_memory_pool_ref>::value, "");

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool, const int driver_version)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  constexpr int min_async_version = 11050;
  return driver_version < min_async_version ? disable_reuse == 0 : disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to query hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

C2H_CCCLRT_TEST("device_memory_resource construction", "[memory_resource]")
{
  int current_device = 0;
  cuda::__ensure_current_context guard{cuda::device_ref{current_device}};

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using test_resource = cuda::device_memory_pool_ref;
  SECTION("Default construction")
  {
    {
      test_resource default_constructed = cuda::device_default_memory_pool(cuda::device_ref{0});
      CHECK(default_constructed.get() == current_default_pool);
    }

    // Ensure that the pool was not destroyed by allocating something
    void* ptr{nullptr};
    _CCCL_TRY_CUDA_API(
      ::hipMallocAsync,
      "Failed to allocate with pool passed to cuda::device_memory_pool_ref",
      &ptr,
      42,
      current_default_pool,
      ::hipStream_t{0});
    CHECK(ptr != nullptr);

    _CCCL_ASSERT_CUDA_API(
      ::hipFreeAsync, "Failed to deallocate with pool passed to cuda::device_memory_pool_ref", ptr, ::hipStream_t{0});
  }

  SECTION("Construct from mempool handle")
  {
    ::hipMemPoolProps pool_properties{};
    pool_properties.allocType     = ::hipMemAllocationTypePinned;
    pool_properties.handleTypes   = ::hipMemAllocationHandleType(0);
    pool_properties.location.type = ::hipMemLocationTypeDevice;
    pool_properties.location.id   = current_device;
    hipMemPool_t cuda_pool_handle{};
    _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &cuda_pool_handle, &pool_properties);

    {
      test_resource from_cudaMemPool{cuda_pool_handle};
      CHECK(from_cudaMemPool.get() == cuda_pool_handle);
      CHECK(from_cudaMemPool.get() != current_default_pool);
    }

    // Ensure that the pool was not destroyed by allocating something
    void* ptr{nullptr};
    _CCCL_TRY_CUDA_API(
      ::hipMallocAsync,
      "Failed to allocate with pool passed to cuda::device_memory_pool_ref",
      &ptr,
      42,
      current_default_pool,
      ::hipStream_t{0});
    CHECK(ptr != nullptr);

    _CCCL_ASSERT_CUDA_API(
      ::hipFreeAsync, "Failed to deallocate with pool passed to cuda::device_memory_pool_ref", ptr, ::hipStream_t{0});
  }

  SECTION("Construct with initial pool size")
  {
    cuda::memory_pool_properties props = {
      42,
    };
    cuda::device_memory_pool from_initial_pool_size{current_device, props};

    ::hipMemPool_t get = from_initial_pool_size.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, cuda::std::numeric_limits<size_t>::max()));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with release threshold")
  {
    cuda::memory_pool_properties props = {
      20,
      42,
    };
    cuda::device_memory_pool with_threshold{current_device, props};

    ::hipMemPool_t get = with_threshold.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, ::hipMemHandleTypeNone));
  }

  // Allocation handles are only supported after 11.2
  SECTION("Construct with allocation handle")
  {
    cuda::memory_pool_properties props = {
      20,
      42,
      ::hipMemHandleTypePosixFileDescriptor,
    };
    cuda::device_memory_pool with_allocation_handle{current_device, props};

    ::hipMemPool_t get = with_allocation_handle.get();
    CHECK(get != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(get, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(get, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(get, props.allocation_handle_type));
  }
}

static void ensure_device_ptr(void* ptr)
{
  CHECK(ptr != nullptr);
  hipPointerAttribute_t attributes;
  hipError_t status = hipPointerGetAttributes(&attributes, ptr);
  CHECK(status == hipSuccess);
  CHECK(attributes.type == hipMemoryTypeDevice);
}

C2H_CCCLRT_TEST("device_memory_resource allocation", "[memory_resource]")
{
  hipStream_t raw_stream;
  {
    cuda::__ensure_current_context guard{cuda::device_ref{0}};
    hipStreamCreate(&raw_stream);
  }
  cuda::device_memory_pool_ref res = cuda::device_default_memory_pool(cuda::device_ref{0});

  { // allocate_sync / deallocate_sync
    auto* ptr = res.allocate_sync(42);
    static_assert(cuda::std::is_same<decltype(ptr), void*>::value, "");
    ensure_device_ptr(ptr);

    res.deallocate_sync(ptr, 42);
  }

  { // allocate_sync / deallocate_sync with alignment
    auto* ptr = res.allocate_sync(42, 4);
    static_assert(cuda::std::is_same<decltype(ptr), void*>::value, "");
    ensure_device_ptr(ptr);

    res.deallocate_sync(ptr, 42, 4);
  }

  { // allocate / deallocate
    cuda::stream_ref stream{raw_stream};

    auto* ptr = res.allocate(stream, 42);
    static_assert(cuda::std::is_same<decltype(ptr), void*>::value, "");

    stream.sync();
    ensure_device_ptr(ptr);

    res.deallocate(stream, ptr, 42);
  }

  { // allocate / deallocate with alignment
    cuda::stream_ref stream{raw_stream};

    auto* ptr = res.allocate(stream, 42, 4);
    static_assert(cuda::std::is_same<decltype(ptr), void*>::value, "");

    stream.sync();
    ensure_device_ptr(ptr);

    res.deallocate(stream, ptr, 42, 4);
  }

#if _CCCL_HAS_EXCEPTIONS()
  { // allocate with too small alignment
    while (true)
    {
      try
      {
        [[maybe_unused]] auto* ptr = res.allocate_sync(5, 42);
      }
      catch (std::invalid_argument&)
      {
        break;
      }
      CHECK(false);
    }
  }

  { // allocate with non matching alignment
    while (true)
    {
      try
      {
        [[maybe_unused]] auto* ptr = res.allocate_sync(5, 1337);
      }
      catch (std::invalid_argument&)
      {
        break;
      }
      CHECK(false);
    }
  }
  { // allocate with too small alignment
    while (true)
    {
      try
      {
        [[maybe_unused]] auto* ptr = res.allocate(raw_stream, 5, 42);
      }
      catch (std::invalid_argument&)
      {
        break;
      }
      CHECK(false);
    }
  }

  { // allocate with non matching alignment
    while (true)
    {
      try
      {
        [[maybe_unused]] auto* ptr = res.allocate(raw_stream, 5, 1337);
      }
      catch (std::invalid_argument&)
      {
        break;
      }
      CHECK(false);
    }
  }
#endif // _CCCL_HAS_EXCEPTIONS()
  {
    cuda::__ensure_current_context guard{cuda::device_ref{0}};
    hipStreamDestroy(raw_stream);
  }
}

enum class AccessibilityType
{
  Device,
  Host,
};

template <AccessibilityType Accessibility>
struct resource
{
  void* allocate_sync(size_t, size_t)
  {
    return nullptr;
  }
  void deallocate_sync(void*, size_t, size_t) {}

  bool operator==(const resource&) const
  {
    return true;
  }
  bool operator!=(const resource& other) const
  {
    return false;
  }

  template <AccessibilityType Accessibilty2                                         = Accessibility,
            cuda::std::enable_if_t<Accessibilty2 == AccessibilityType::Device, int> = 0>
  friend void get_property(const resource&, ::cuda::mr::device_accessible) noexcept
  {}
};
static_assert(cuda::mr::synchronous_resource<resource<AccessibilityType::Host>>, "");
static_assert(!cuda::mr::synchronous_resource_with<resource<AccessibilityType::Host>, ::cuda::mr::device_accessible>,
              "");
static_assert(cuda::mr::synchronous_resource<resource<AccessibilityType::Device>>, "");
static_assert(cuda::mr::synchronous_resource_with<resource<AccessibilityType::Device>, ::cuda::mr::device_accessible>,
              "");

template <AccessibilityType Accessibility>
struct test_resource : public resource<Accessibility>
{
  void* allocate(cuda::stream_ref, size_t, size_t)
  {
    return nullptr;
  }
  void deallocate(cuda::stream_ref, void*, size_t, size_t) {}
};
static_assert(cuda::mr::resource<test_resource<AccessibilityType::Host>>, "");
static_assert(!cuda::mr::resource_with<test_resource<AccessibilityType::Host>, ::cuda::mr::device_accessible>, "");
static_assert(cuda::mr::resource<test_resource<AccessibilityType::Device>>, "");
static_assert(cuda::mr::resource_with<test_resource<AccessibilityType::Device>, ::cuda::mr::device_accessible>, "");

C2H_CCCLRT_TEST("device_memory_resource comparison", "[memory_resource]")
{
  int current_device = 0;
  cuda::__ensure_current_context guard{cuda::device_ref{current_device}};

  cuda::device_memory_pool_ref first = cuda::device_default_memory_pool(cuda::device_ref{0});
  { // comparison against a plain device_memory_pool_ref
    cuda::device_memory_pool_ref second = cuda::device_default_memory_pool(cuda::device_ref{0});
    CHECK((first == second));
    CHECK(!(first != second));
  }

  { // comparison against a plain device_memory_pool_ref with a different pool
    hipMemPool_t cuda_pool_handle{};
    {
      ::hipMemPoolProps pool_properties{};
      pool_properties.allocType     = ::hipMemAllocationTypePinned;
      pool_properties.handleTypes   = ::hipMemAllocationHandleType(0);
      pool_properties.location.type = ::hipMemLocationTypeDevice;
      pool_properties.location.id   = current_device;
      _CCCL_TRY_CUDA_API(::hipMemPoolCreate, "Failed to call hipMemPoolCreate", &cuda_pool_handle, &pool_properties);
    }
    cuda::device_memory_pool_ref second{cuda_pool_handle};
    CHECK((first != second));
    CHECK(!(first == second));
  }

  { // comparison against a device_memory_pool_ref wrapped inside a synchronous_resource_ref<device_accessible>
    cuda::device_memory_pool_ref second = cuda::device_default_memory_pool(cuda::device_ref{0});
    cuda::mr::synchronous_resource_ref<::cuda::mr::device_accessible> second_ref{second};
    CHECK((first == second_ref));
    CHECK(!(first != second_ref));
    CHECK((second_ref == first));
    CHECK(!(second_ref != first));
  }

  { // comparison against a device_memory_pool_ref wrapped inside a resource_ref
    cuda::device_memory_pool_ref second = cuda::device_default_memory_pool(cuda::device_ref{0});
    cuda::mr::resource_ref<::cuda::mr::device_accessible> second_ref{second};

    CHECK((first == second_ref));
    CHECK(!(first != second_ref));
    CHECK((second_ref == first));
    CHECK(!(second_ref != first));
  }

  { // comparison against a different resource through synchronous_resource_ref
    resource<AccessibilityType::Host> host_resource{};
    resource<AccessibilityType::Device> device_resource{};
    CHECK(!(first == host_resource));
    CHECK((first != host_resource));
    CHECK(!(first == device_resource));
    CHECK((first != device_resource));

    CHECK(!(host_resource == first));
    CHECK((host_resource != first));
    CHECK(!(device_resource == first));
    CHECK((device_resource != first));
  }

  { // comparison against a different resource through synchronous_resource_ref
    test_resource<AccessibilityType::Host> host_async_resource{};
    test_resource<AccessibilityType::Device> device_async_resource{};
    CHECK(!(first == host_async_resource));
    CHECK((first != host_async_resource));
    CHECK(!(first == device_async_resource));
    CHECK((first != device_async_resource));

    CHECK(!(host_async_resource == first));
    CHECK((host_async_resource != first));
    CHECK(!(device_async_resource == first));
    CHECK((device_async_resource != first));
  }
}

C2H_CCCLRT_TEST("Async memory resource access", "")
{
  /* diable until we move the launch API to libcudacxx
  if (cuda::devices.size() > 1)
  {
    auto peers = cuda::devices[0].peers();
    if (peers.size() > 0)
    {
      cuda::device_memory_pool pool{cuda::devices[0]};
      cuda::stream_ref stream{peers.front()};
      CUDAX_CHECK(pool.is_accessible_from(cuda::devices[0]));

      auto allocate_and_check_access = [&](auto& resource) {
        auto* ptr1  = resource.allocate(stream, sizeof(int));
        auto* ptr2  = resource.allocate_sync(sizeof(int));
        auto config = cudax::distribute<1>(1);
        cudax::launch(stream, config, test::assign_42{}, (int*) ptr1);
        cudax::launch(stream, config, test::assign_42{}, (int*) ptr2);
        stream.sync();
        resource.deallocate(stream, ptr1, sizeof(int));
        resource.deallocate_sync(ptr2, sizeof(int));
      };

      pool.enable_access_from(peers);

      CUDAX_CHECK(pool.is_accessible_from(peers.front()));
      allocate_and_check_access(pool);

      cudax::device_memory_pool_ref resource{pool};
      CUDAX_CHECK(resource.is_accessible_from(peers.front()));
      allocate_and_check_access(resource);

      pool.disable_access_from(peers.front());
      CUDAX_CHECK(!pool.is_accessible_from(peers.front()));
      CUDAX_CHECK(!resource.is_accessible_from(peers.front()));

      if (peers.size() > 1)
      {
        CUDAX_CHECK(pool.is_accessible_from(peers[1]));
      }

      pool.disable_access_from(peers);

      pool.enable_access_from(peers.front());
      CUDAX_CHECK(pool.is_accessible_from(peers.front()));
      CUDAX_CHECK(resource.is_accessible_from(peers.front()));

      // Check if enable can include the device on which the pool resides
      {
        std::vector peers_ext(peers.begin(), peers.end());
        peers_ext.push_back(cuda::devices[0]);
        pool.enable_access_from(peers_ext);

        // Check the resource using the default pool
        cuda::device_memory_pool_ref default_pool_resource = cuda::device_default_memory_pool(cuda::device_ref{0});
        cuda::device_memory_pool_ref another_default_pool_resource =
          cuda::device_default_memory_pool(cuda::device_ref{0});

        pool.enable_access_from(peers_ext.front());

        CUDAX_CHECK(default_pool_resource.is_accessible_from(peers_ext.front()));
        allocate_and_check_access(default_pool_resource);
        CUDAX_CHECK(another_default_pool_resource.is_accessible_from(peers_ext.front()));
        allocate_and_check_access(another_default_pool_resource);
      }
    }
  }
  */
}
