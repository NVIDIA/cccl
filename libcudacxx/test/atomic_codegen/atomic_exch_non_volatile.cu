#include "hip/hip_runtime.h"
#include <cuda/atomic>

__global__ void exch_device_relaxed_non_volatile(int* data, int* out, int n)
{
  auto ref = cuda::atomic_ref<int, cuda::thread_scope_device>{*(data)};
  *out     = ref.exchange(n, cuda::std::memory_order_relaxed);
}

/*

; SM8X-LABEL: .target sm_80
; SM8X:      .visible .entry [[FUNCTION:_.*exch_device_relaxed_non_volatile.*]](
; SM8X-DAG:  ld.param.u64 %rd[[#ATOM:]], [[[FUNCTION]]_param_0];
; SM8X-DAG:  ld.param.u64 %rd[[#EXPECTED:]], [[[FUNCTION]]_param_1];
; SM8X-DAG:  ld.param.u32 %r[[#INPUT:]], [[[FUNCTION]]_param_2];
; SM8X-NEXT: cvta.to.global.u64 %rd[[#GOUT:]], %rd[[#EXPECTED]];
; SM8X-NEXT: //
; SM8X-NEXT: atom.exch.relaxed.gpu.b32 %r[[#DEST:]],[%rd[[#ATOM]]],%r[[#INPUT]];
; SM8X-NEXT: //
; SM8X-NEXT: st.global.u32 [%rd[[#GOUT]]], %r[[#DEST]];
; SM8X-NEXT: ret;

*/
