#include "hip/hip_runtime.h"
#include <cuda/atomic>

__global__ void load_relaxed_device_non_volatile(int* data, int* out)
{
  auto ref = cuda::atomic_ref<int, cuda::thread_scope_device>{*(data)};
  *out     = ref.load(cuda::std::memory_order_relaxed);
}

/*

; SM8X-LABEL: .target sm_80
; SM8X:      .visible .entry [[FUNCTION:_.*load_relaxed_device_non_volatile.*]](
; SM8X-DAG:  ld.param.u64 %rd[[#ATOM:]], [[[FUNCTION]]_param_0];
; SM8X-DAG:  ld.param.u64 %rd[[#EXPECTED:]], [[[FUNCTION]]_param_1];
; SM8X-NEXT: cvta.to.global.u64 %rd[[#GOUT:]], %rd[[#EXPECTED]];
; SM8X-NEXT: //
; SM8X-NEXT: ld.relaxed.gpu.b32 %r[[#DEST:]],[%rd[[#ATOM]]];
; SM8X-NEXT: //
; SM8X-NEXT: st.global.u32 [%rd[[#GOUT]]], %r[[#DEST]];
; SM8X-NEXT: ret;

*/
