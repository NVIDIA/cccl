#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2011-2025, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_copy.cuh>
#include <cub/util_type.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scan.h>
#include <thrust/tabulate.h>

#include <cuda/std/complex>
#include <cuda/std/cstdint>
#include <cuda/type_traits>

#include <c2h/bfloat16.cuh>
#include <c2h/custom_type.h>
#include <c2h/detail/generators.cuh>
#include <c2h/device_policy.h>
#include <c2h/extended_types.h>
#include <c2h/fill_striped.h>
#include <c2h/generators.h>
#include <c2h/half.cuh>
#include <c2h/vector.h>

#if C2H_HAS_CURAND
#  include <hiprand.h>
#else
#  include <thrust/random.h>
#endif

namespace c2h::detail
{

#if !C2H_HAS_CURAND
struct i_to_rnd_t
{
  __host__ __device__ i_to_rnd_t(thrust::default_random_engine engine)
      : m_engine(engine)
  {}

  thrust::default_random_engine m_engine{};

  template <typename IndexType>
  __host__ __device__ float operator()(IndexType n)
  {
    m_engine.discard(n);
    return thrust::uniform_real_distribution<float>{0.0f, 1.0f}(m_engine);
  }
};
#endif // !C2H_HAS_CURAND

template <typename T>
struct random_to_item_t<cuda::std::complex<T>, false>
{
  cuda::std::complex<T> m_min;
  cuda::std::complex<T> m_max;

  __host__ __device__ random_to_item_t(cuda::std::complex<T> min, cuda::std::complex<T> max)
      : m_min(min)
      , m_max(max)
  {}

  __device__ cuda::std::complex<T> operator()(float random_value) const
  {
    return (m_max - m_min) * cuda::std::complex<T>(random_value) + m_min;
  }
};

void generator_t::generate()
{
#if C2H_HAS_CURAND
  hiprandGenerateUniform(m_gen, thrust::raw_pointer_cast(m_distribution.data()), m_distribution.size());
#else
  thrust::tabulate(device_policy, m_distribution.begin(), m_distribution.end(), i_to_rnd_t{m_re});
  m_re.discard(m_distribution.size());
#endif
}

float* generator_t::prepare_random_generator(seed_t seed, std::size_t num_items)
{
  m_distribution.resize(num_items);

#if C2H_HAS_CURAND
  hiprandSetPseudoRandomGeneratorSeed(m_gen, seed.get());
#else
  m_re.seed(seed.get());
#endif

  generate();

  return thrust::raw_pointer_cast(m_distribution.data());
}

struct random_to_custom_t
{
  static constexpr std::size_t m_max_key = std::numeric_limits<std::size_t>::max();

  __device__ void operator()(std::size_t idx) const
  {
    auto out = reinterpret_cast<custom_type_state_t*>(m_out + idx * m_element_size);
    out->key = static_cast<std::size_t>(static_cast<float>(m_max_key) * m_in[idx * 2 + 0]);
    out->val = static_cast<std::size_t>(static_cast<float>(m_max_key) * m_in[idx * 2 + 1]);
  }

  float* m_in{};
  char* m_out{};
  std::size_t m_element_size{};
};

void gen_custom_type_state(
  seed_t seed,
  char* d_out,
  custom_type_state_t /* min */,
  custom_type_state_t /* max */,
  std::size_t elements,
  std::size_t element_size)
{
  // FIXME(bgruber): implement min/max handling for custom_type_state_t
  float* d_in = generator.prepare_random_generator(seed, elements * 2);
  thrust::for_each(device_policy,
                   thrust::counting_iterator<std::size_t>{0},
                   thrust::counting_iterator<std::size_t>{elements},
                   random_to_custom_t{d_in, d_out, element_size});
}

template <class T>
struct greater_equal_op
{
  T val;

  __device__ bool operator()(T x)
  {
    return x >= val;
  }
};

template <typename T>
struct spaced_out_it_op
{
  char* base_it;
  std::size_t element_size;

  __host__ __device__ __forceinline__ T& operator()(std::size_t offset) const
  {
    return *reinterpret_cast<T*>(base_it + (element_size * offset));
  }
};

template <typename T>
struct offset_to_iterator_t
{
  char* base_it;
  std::size_t element_size;

  __host__
    __device__ __forceinline__ thrust::transform_iterator<spaced_out_it_op<T>, thrust::counting_iterator<std::size_t>>
    operator()(std::size_t offset) const
  {
    // The pointer to the beginning of this "buffer" (aka a series of same "keys")
    auto base_ptr = base_it + (element_size * offset);

    // We need to make sure that the i-th element within this "buffer" is spaced out by
    // `element_size`
    auto counting_it = thrust::make_counting_iterator(std::size_t{0});
    spaced_out_it_op<T> space_out_op{base_ptr, element_size};
    return thrust::make_transform_iterator(counting_it, space_out_op);
  }
};

template <class T>
struct repeat_index_t
{
  __host__ __device__ __forceinline__ thrust::constant_iterator<T> operator()(std::size_t i)
  {
    return thrust::constant_iterator<T>(static_cast<T>(i));
  }
};

template <>
struct repeat_index_t<custom_type_state_t>
{
  __host__ __device__ __forceinline__ thrust::constant_iterator<custom_type_state_t> operator()(std::size_t i)
  {
    custom_type_state_t item{};
    item.key = i;
    item.val = i;
    return thrust::constant_iterator<custom_type_state_t>(item);
  }
};

template <typename OffsetT>
struct offset_to_size_t
{
  const OffsetT* offsets;

  __host__ __device__ __forceinline__ std::size_t operator()(std::size_t i)
  {
    return offsets[i + 1] - offsets[i];
  }
};

/**
 * @brief Initializes key-segment ranges from an offsets-array like the one given by
 * `gen_uniform_offset`.
 */
template <typename OffsetT, typename KeyT>
void init_key_segments(::cuda::std::span<const OffsetT> segment_offsets, KeyT* d_out, std::size_t element_size)
{
  OffsetT total_segments   = static_cast<OffsetT>(segment_offsets.size() - 1);
  const OffsetT* d_offsets = segment_offsets.data();

  thrust::counting_iterator<int> iota(0);
  offset_to_iterator_t<KeyT> dst_transform_op{reinterpret_cast<char*>(d_out), element_size};

  auto d_range_srcs  = thrust::make_transform_iterator(iota, repeat_index_t<KeyT>{});
  auto d_range_dsts  = thrust::make_transform_iterator(d_offsets, dst_transform_op);
  auto d_range_sizes = thrust::make_transform_iterator(iota, offset_to_size_t<OffsetT>{d_offsets});

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
  std::uint8_t* d_temp_storage   = nullptr;
  std::size_t temp_storage_bytes = 0;
  // TODO(bgruber): replace by a non-CUB implementation
  hipcub::DeviceCopy::Batched(
    d_temp_storage, temp_storage_bytes, d_range_srcs, d_range_dsts, d_range_sizes, total_segments);

  device_vector<std::uint8_t> temp_storage(temp_storage_bytes, thrust::no_init);
  d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  // TODO(bgruber): replace by a non-CUB implementation
  hipcub::DeviceCopy::Batched(
    d_temp_storage, temp_storage_bytes, d_range_srcs, d_range_dsts, d_range_sizes, total_segments);
  hipDeviceSynchronize();
#else // THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
  static_assert(sizeof(OffsetT) == 0, "Need to implement a non-CUB version of hipcub::DeviceCopy::Batched");
  // TODO(bgruber): implement and *test* a non-CUB version, here is a sketch:
  // thrust::for_each(
  //   thrust::device,
  //   thrust::counting_iterator<OffsetT>{0},
  //   thrust::counting_iterator<OffsetT>{total_segments},
  //   [&](OffsetT i) {
  //     const auto value = d_range_srcs[i];
  //     const auto start = d_range_sizes[i];
  //     const auto end   = d_range_sizes[i + 1];
  //     for (auto j = start; j < end; ++j)
  //     {
  //       d_range_dsts[j] = value;
  //     }
  //   });
#endif // THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
}

template void
init_key_segments(::cuda::std::span<const std::uint32_t> segment_offsets, std::int32_t* out, std::size_t element_size);
template void
init_key_segments(::cuda::std::span<const std::uint32_t> segment_offsets, std::uint8_t* out, std::size_t element_size);
template void
init_key_segments(::cuda::std::span<const std::uint32_t> segment_offsets, float* out, std::size_t element_size);
template void init_key_segments(
  ::cuda::std::span<const std::uint32_t> segment_offsets, custom_type_state_t* out, std::size_t element_size);
#if TEST_HALF_T()
template void
init_key_segments(::cuda::std::span<const std::uint32_t> segment_offsets, half_t* out, std::size_t element_size);
#endif // TEST_HALF_T()

#if TEST_BF_T()
template void
init_key_segments(::cuda::std::span<const std::uint32_t> segment_offsets, bfloat16_t* out, std::size_t element_size);
#endif // TEST_BF_T()

template <typename T>
std::size_t gen_uniform_offsets(
  seed_t seed, cuda::std::span<T> segment_offsets, T total_elements, T min_segment_size, T max_segment_size)
{
  gen_values_between(seed, segment_offsets, min_segment_size, max_segment_size);
  *thrust::device_ptr<T>(&segment_offsets[total_elements]) = total_elements + 1;
  thrust::exclusive_scan(device_policy, segment_offsets.begin(), segment_offsets.end(), segment_offsets.begin());
  const auto iter =
    thrust::find_if(device_policy, segment_offsets.begin(), segment_offsets.end(), greater_equal_op<T>{total_elements});
  *thrust::device_ptr<T>(&*iter) = total_elements;
  return iter - segment_offsets.begin() + 1;
}

template std::size_t gen_uniform_offsets(
  seed_t seed,
  cuda::std::span<int32_t> segment_offsets,
  int32_t total_elements,
  int32_t min_segment_size,
  int32_t max_segment_size);
template std::size_t gen_uniform_offsets(
  seed_t seed,
  cuda::std::span<uint32_t> segment_offsets,
  uint32_t total_elements,
  uint32_t min_segment_size,
  uint32_t max_segment_size);
template std::size_t gen_uniform_offsets(
  seed_t seed,
  cuda::std::span<int64_t> segment_offsets,
  int64_t total_elements,
  int64_t min_segment_size,
  int64_t max_segment_size);
template std::size_t gen_uniform_offsets(
  seed_t seed,
  cuda::std::span<uint64_t> segment_offsets,
  uint64_t total_elements,
  uint64_t min_segment_size,
  uint64_t max_segment_size);

template <typename T>
void gen_values_between(seed_t seed, ::cuda::std::span<T> data, T min, T max)
{
  const auto* dist = generator.prepare_random_generator(seed, data.size());
  thrust::transform(device_policy, dist, dist + data.size(), data.begin(), random_to_item_t<T>(min, max));
}

template <typename T>
struct counter_to_cyclic_item_t
{
  std::size_t n;

  template <typename CounterT>
  __device__ T operator()(CounterT id)
  {
    // This has to be a type for which extended floating point types like __hip_fp8_e5m2_fnuz provide an overload
    return static_cast<T>(static_cast<float>(static_cast<uint64_t>(id) % n));
  }
};

template <typename T>
void gen_values_cyclic(modulo_t mod, ::cuda::std::span<T> data)
{
  thrust::tabulate(device_policy, data.begin(), data.end(), counter_to_cyclic_item_t<T>{mod.get()});
}

#define INSTANTIATE_RND(TYPE) \
  template void gen_values_between<TYPE>(seed_t, ::cuda::std::span<TYPE> data, TYPE min, TYPE max)
#define INSTANTIATE_MOD(TYPE) template void gen_values_cyclic<TYPE>(modulo_t, ::cuda::std::span<TYPE> data)

#define INSTANTIATE(TYPE) \
  INSTANTIATE_RND(TYPE);  \
  INSTANTIATE_MOD(TYPE)

INSTANTIATE(std::uint8_t);
INSTANTIATE(std::uint16_t);
INSTANTIATE(std::uint32_t);
INSTANTIATE(std::uint64_t);

INSTANTIATE(std::int8_t);
INSTANTIATE(std::int16_t);
INSTANTIATE(std::int32_t);
INSTANTIATE(std::int64_t);

#if _CCCL_HAS_NVFP8()
INSTANTIATE(__hip_fp8_e5m2_fnuz);
INSTANTIATE(__hip_fp8_e4m3_fnuz);
#endif // _CCCL_HAS_NVFP8()
INSTANTIATE(float);
INSTANTIATE(double);
INSTANTIATE(cuda::std::complex<float>);
INSTANTIATE(cuda::std::complex<double>);

INSTANTIATE(bool);
INSTANTIATE(char);

#if TEST_HALF_T()
INSTANTIATE(half_t);
INSTANTIATE(__half);
#  if _CCCL_CUDACC_AT_LEAST(12, 2)
INSTANTIATE(cuda::std::complex<__half>);
#  endif
#endif // TEST_HALF_T()

#if TEST_BF_T()
INSTANTIATE(bfloat16_t);
INSTANTIATE(__hip_bfloat16);
#  if _CCCL_CUDACC_AT_LEAST(12, 2)
INSTANTIATE(cuda::std::complex<__hip_bfloat16>);
#  endif
#endif // TEST_BF_T()

#if TEST_INT128()
INSTANTIATE(__int128_t);
INSTANTIATE(__uint128_t);
#endif // TEST_INT128()

#undef INSTANTIATE_RND
#undef INSTANTIATE_MOD
#undef INSTANTIATE
} // namespace c2h::detail
